// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --default-preprocessor --roc %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK-NOT: #include <hip/hip_runtime.h>
#include <memory>


// CHECK-NOT: #include <hip/hip_runtime.h>

#if CUDA_VERSION >= 7050
// CHECK: #include "hip/hip_fp16.h"
#include "hip/hip_fp16.h"
#endif
