// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// CHECK: #include <hip/hip_runtime.h>
// CHECK: #include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"
// CHECK: #include <stdio.h>
#include "hipblas.h"
#include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"
#include <stdio.h>
