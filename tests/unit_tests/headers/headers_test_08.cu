// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// CHECK: #include <hip/hip_runtime.h>
// CHECK: #include <iostream>
// CHECK: #include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"
// CHECK: #include <stdio.h>
// CHECK-NOT: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK-NOT: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK-NOT: #include <hip/hip_runtime.h>
#include <iostream>
#include "hipblas.h"
#include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"
#include <stdio.h>
