// RUN: %run_test hipify "%s" "%t" %hipify_args 1 --roc %clang_args

// CHECK: #include <hip/hip_runtime.h>
// CHECK: #include <memory>

// CHECK-NOT: #include <hip/hip_runtime.h>
// CHECK-NOT: #include <hip/hip_runtime.h>

// CHECK: #include "hip/hip_runtime_api.h"
// CHECK: #include "hip/hip/channel_descriptor.h"
// CHECK: #include "hip/hip/device_functions.h"
// CHECK: #include "hip/hip/driver_types.h"
// CHECK: #include "hip/hip_complex.h"
// CHECK: #include "hip/hip_texture_types.h"
// CHECK: #include "hip/hip_vector_types.h"

// CHECK: #include <iostream>

// CHECK: #include <stdio.h>

// CHECK: #include "rocrand/rocrand.h"
// CHECK: #include "rocrand/rocrand_kernel.h"

// CHECK: #include <algorithm>

// CHECK: #include "rocrand/rocrand_discrete.h"
// CHECK: #include "rocrand/rocrand_common.h"
// CHECK: #include "rocrand/rocrand_log_normal.h"
// CHECK: #include "rocrand/rocrand_mrg32k3a.h"
// CHECK: #include "rocrand/rocrand_mtgp32.h"
// CHECK: #include "rocrand/rocrand_mtgp32_11213.h"
// CHECK: #include "rocrand/rocrand_normal.h"
// CHECK: #include "rocrand/rocrand_philox4x32_10.h"
// CHECK: #include "rocrand/rocrand_poisson.h"
// CHECK: #include "rocrand/rocrand_xorwow_precomputed.h"
// CHECK: #include "rocrand/rocrand_uniform.h"

// CHECK-NOT: #include "rocrand/rocrand.h"
// CHECK-NOT: #include "rocrand/rocrand_kernel.h"
// CHECK-NOT: #include "rocrand/rocrand_discrete.h"
// CHECK-NOT: #include "rocrand/rocrand_mtgp32.h"
// CHECK-NOT: #include "rocrand/rocrand_normal.h"

// CHECK-NOT: #include "hiprand_discrete.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand/hiprand_mtgp32_host.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "rocrand/rocrand_mtgp32_11213.h"
// CHECK-NOT: #include "hiprand_normal.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand_poisson.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand_uniform.h"

// CHECK: #include <string>

// CHECK: #include "hipfft/hipfft.h"
// CHECK: #include "rocsparse.h"

#include <hip/hip_runtime.h>
// CHECK-NOT: #include <hip/hip_runtime.h>

#include <memory>

#include <hip/hip_runtime.h>
// CHECK-NOT: #include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"
#include "hip/channel_descriptor.h"
#include "hip/device_functions.h"
#include "hip/driver_types.h"
#include "hip/hip_complex.h"
#include "hip/hip_texture_types.h"
#include "hip/hip_vector_types.h"

#include <iostream>

#include <stdio.h>

#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <algorithm>

#include "hiprand_discrete.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand/hiprand_mtgp32_host.h"
#include "hiprand/hiprand_kernel.h"
#include "rocrand/rocrand_mtgp32_11213.h"
#include "hiprand_normal.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand_poisson.h"
#include "hiprand/hiprand_kernel.h"
#include "hiprand_uniform.h"

#include <string>

#include "hipfft/hipfft.h"

#include "hipsparse.h"
