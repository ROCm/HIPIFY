// RUN: %run_test hipify "%s" "%t" %hipify_args 1 --skip-excluded-preprocessor-conditional-blocks %clang_args

// CHECK: #include <hip/hip_runtime.h>
// CHECK: #include "hipblas.h"
// CHECK-NOT: #include "cublas_v2.h"
// CHECK: #include <stdio.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"
#include <stdio.h>
