// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// CHECK: #include <hip/hip_math_constants.h>
// CHECK-NOT: #include <cuda_runtime.h>
// CHECK: #include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <stdio.h>
