// RUN: %run_test hipify "%s" "%t" %hipify_args 1 -roc %clang_args

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "rocblas.h"
#include "hipblas.h"
#define M 6
#define N 5
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))
// CHECK: static __inline__ void modify(rocblas_handle handle, float *m, int ldm, int
static __inline__ void modify(hipblasHandle_t handle, float *m, int ldm, int
  n, int p, int q, float alpha, float beta) {
  // CHECK: rocblas_sscal(handle, n - p + 1, &alpha, &m[IDX2F(p, q, ldm)], ldm);
  // CHECK: rocblas_sscal(handle, ldm - p + 1, &beta, &m[IDX2F(p, q, ldm)], 1);
  hipblasSscal(handle, n - p + 1, &alpha, &m[IDX2F(p, q, ldm)], ldm);
  hipblasSscal(handle, ldm - p + 1, &beta, &m[IDX2F(p, q, ldm)], 1);
}
int main(void) {
  // CHECK: hipError_t cudaStat;
  // CHECK: rocblas_status stat;
  // CHECK: rocblas_handle handle;
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  int i, j;
  float* devPtrA;
  float* a = 0;
  a = (float *)malloc(M * N * sizeof(*a));
  if (!a) {
    printf("host memory allocation failed");
    return EXIT_FAILURE;
  }
  for (j = 1; j <= N; j++) {
    for (i = 1; i <= M; i++) {
      a[IDX2F(i, j, M)] = (float)((i - 1) * M + j);
    }
  }
  // CHECK: cudaStat = hipMalloc((void**)&devPtrA, M*N * sizeof(*a));
  cudaStat = hipMalloc((void**)&devPtrA, M*N * sizeof(*a));
  // CHECK: if (cudaStat != hipSuccess) {
  if (cudaStat != hipSuccess) {
    printf("device memory allocation failed");
    return EXIT_FAILURE;
  }
  // CHECK: stat = rocblas_create_handle(&handle);
  stat = hipblasCreate(&handle);
  // CHECK: if (stat != rocblas_status_success) {
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS initialization failed\n");
    return EXIT_FAILURE;
  }
  // CHECK: stat = rocblas_set_matrix(M, N, sizeof(*a), a, M, devPtrA, M);
  stat = hipblasSetMatrix(M, N, sizeof(*a), a, M, devPtrA, M);
  // CHECK: if (stat != rocblas_status_success) {
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("data download failed");
    // CHECK: hipFree(devPtrA);
    // CHECK: rocblas_destroy_handle(handle);
    hipFree(devPtrA);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
  modify(handle, devPtrA, M, N, 2, 3, 16.0f, 12.0f);
  // CHECK: stat = rocblas_get_matrix(M, N, sizeof(*a), devPtrA, M, a, M);
  stat = hipblasGetMatrix(M, N, sizeof(*a), devPtrA, M, a, M);
  // CHECK: if (stat != rocblas_status_success) {
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("data upload failed");
    // CHECK: hipFree(devPtrA);
    // CHECK: rocblas_destroy_handle(handle);
    hipFree(devPtrA);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
  // CHECK: hipFree(devPtrA);
  // CHECK: rocblas_destroy_handle(handle);
  hipFree(devPtrA);
  hipblasDestroy(handle);
  for (j = 1; j <= N; j++) {
    for (i = 1; i <= M; i++) {
      printf("%7.0f", a[IDX2F(i, j, M)]);
    }
    printf("\n");
  }
  free(a);
  return EXIT_SUCCESS;
}
