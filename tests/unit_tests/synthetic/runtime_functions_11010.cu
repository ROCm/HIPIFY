// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>

int main() {
  printf("12.11010. CUDA Runtime API Functions synthetic test for CUDA >= 11010\n");

  size_t bytes = 0;
  size_t width = 0;
  size_t wOffset = 0;
  void* image = nullptr;
  void* src = nullptr;
  void* dst = nullptr;

  // CHECK: hipError_t result = hipSuccess;
  hipError_t result = hipSuccess;

  // CHECK: hipMemcpyKind MemcpyKind;
  hipMemcpyKind MemcpyKind;

#if CUDA_VERSION >= 10000
  // CHECK: hipGraphNode_t graphNode, graphNode_2;
  hipGraphNode_t graphNode, graphNode_2;

  // CHECK: hipGraph_t Graph_t, Graph_t_2;
  hipGraph_t Graph_t, Graph_t_2;

  // CHECK: hipGraphExec_t GraphExec_t;
  hipGraphExec_t GraphExec_t;
#endif

#if CUDA_VERSION >= 11010
  // CUDA: extern __host__ cudaError_t CUDARTAPI cudaGraphAddMemcpyNodeToSymbol(cudaGraphNode_t* pGraphNode, cudaGraph_t graph, const cudaGraphNode_t* pDependencies, size_t numDependencies, const void* symbol, const void* src, size_t count, size_t offset, enum cudaMemcpyKind kind);
  // HIP: hipError_t hipGraphAddMemcpyNodeToSymbol(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, const void* symbol, const void* src, size_t count, size_t offset, hipMemcpyKind kind);
  // CHECK: result = hipGraphAddMemcpyNodeToSymbol(&graphNode, Graph_t, &graphNode_2, width, HIP_SYMBOL(image), src, bytes, wOffset, MemcpyKind);
  result = hipGraphAddMemcpyNodeToSymbol(&graphNode, Graph_t, &graphNode_2, width, HIP_SYMBOL(image), src, bytes, wOffset, MemcpyKind);

  // CUDA: extern __host__ cudaError_t CUDARTAPI cudaGraphAddMemcpyNodeFromSymbol(cudaGraphNode_t* pGraphNode, cudaGraph_t graph, const cudaGraphNode_t* pDependencies, size_t numDependencies, void* dst, const void* symbol, size_t count, size_t offset, enum cudaMemcpyKind kind);
  // HIP: hipError_t hipGraphAddMemcpyNodeFromSymbol(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, void* dst, const void* symbol, size_t count, size_t offset, hipMemcpyKind kind);
  // CHECK: result = hipGraphAddMemcpyNodeFromSymbol(&graphNode, Graph_t, &graphNode_2, width, dst, HIP_SYMBOL(image), bytes, wOffset, MemcpyKind);
  result = hipGraphAddMemcpyNodeFromSymbol(&graphNode, Graph_t, &graphNode_2, width, dst, HIP_SYMBOL(image), bytes, wOffset, MemcpyKind);

  // CUDA: extern __host__ cudaError_t CUDARTAPI cudaGraphMemcpyNodeSetParamsToSymbol(cudaGraphNode_t node, const void* symbol, const void* src, size_t count, size_t offset, enum cudaMemcpyKind kind);
  // HIP: hipError_t hipGraphMemcpyNodeSetParamsToSymbol(hipGraphNode_t node, const void* symbol, const void* src, size_t count, size_t offset, hipMemcpyKind kind);
  // CHECK: result = hipGraphMemcpyNodeSetParamsToSymbol(graphNode, HIP_SYMBOL(image), src, bytes, wOffset, MemcpyKind);
  result = hipGraphMemcpyNodeSetParamsToSymbol(graphNode, HIP_SYMBOL(image), src, bytes, wOffset, MemcpyKind);

  // CUDA: extern __host__ cudaError_t CUDARTAPI cudaGraphMemcpyNodeSetParamsFromSymbol(cudaGraphNode_t node, void* dst, const void* symbol, size_t count, size_t offset, enum cudaMemcpyKind kind);
  // HIP: hipError_t hipGraphMemcpyNodeSetParamsFromSymbol(hipGraphNode_t node, void* dst, const void* symbol, size_t count, size_t offset, hipMemcpyKind kind);
  // CHECK: result = hipGraphMemcpyNodeSetParamsFromSymbol(graphNode, dst, HIP_SYMBOL(image), bytes, wOffset, MemcpyKind);
  result = hipGraphMemcpyNodeSetParamsFromSymbol(graphNode, dst, HIP_SYMBOL(image), bytes, wOffset, MemcpyKind);

  // CUDA: extern __host__ cudaError_t CUDARTAPI cudaGraphExecMemcpyNodeSetParamsToSymbol(cudaGraphExec_t hGraphExec, cudaGraphNode_t node, const void* symbol, const void* src, size_t count, size_t offset, enum cudaMemcpyKind kind);
  // HIP: hipError_t hipGraphExecMemcpyNodeSetParamsToSymbol(hipGraphExec_t hGraphExec, hipGraphNode_t node, const void* symbol, const void* src, size_t count, size_t offset, hipMemcpyKind kind);
  // CHECK: result = hipGraphExecMemcpyNodeSetParamsToSymbol(GraphExec_t, graphNode, HIP_SYMBOL(image), src, bytes, wOffset, MemcpyKind);
  result = hipGraphExecMemcpyNodeSetParamsToSymbol(GraphExec_t, graphNode, HIP_SYMBOL(image), src, bytes, wOffset, MemcpyKind);

  // CUDA: extern __host__ cudaError_t CUDARTAPI cudaGraphExecMemcpyNodeSetParamsFromSymbol(cudaGraphExec_t hGraphExec, cudaGraphNode_t node, void* dst, const void* symbol, size_t count, size_t offset, enum cudaMemcpyKind kind);
  // HIP: hipError_t hipGraphExecMemcpyNodeSetParamsFromSymbol(hipGraphExec_t hGraphExec, hipGraphNode_t node, void* dst, const void* symbol, size_t count, size_t offset, hipMemcpyKind kind);
  // CHECK: result = hipGraphExecMemcpyNodeSetParamsFromSymbol(GraphExec_t, graphNode, dst, HIP_SYMBOL(image), bytes, wOffset, MemcpyKind);
  result = hipGraphExecMemcpyNodeSetParamsFromSymbol(GraphExec_t, graphNode, dst, HIP_SYMBOL(image), bytes, wOffset, MemcpyKind);
#endif

  return 0;
}
