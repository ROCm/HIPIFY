// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>
#if defined(_WIN32)
  #include "windows.h"
  #include <GL/glew.h>
#elif CUDA_VERSION <= 10000
  #include <GL/glew.h>
#endif
#include "cudaGL.h"
#include "cudaProfiler.h"

int main() {
  printf("09. CUDA Driver API Functions synthetic test\n");

  unsigned int flags = 0;
  unsigned int flags_2 = 0;
  int dim = 0;
  int count = 0;
  int iBlockSize = 0;
  int iBlockSize_2 = 0;
  size_t bytes = 0;
  size_t bytes_2 = 0;
  void* image = nullptr;
  std::string name = "str";
  uint32_t u_value = 0;
  float ms = 0;
  float ms_2 = 0;
  int* value = 0;
  int* value_2 = 0;
  GLuint gl_uint = 0;
  GLenum gl_enum = 0;
#if defined(_WIN32)
  unsigned long long ull = 0;
#else
  unsigned long ull = 0;
#endif
  unsigned long long ull_2 = 0;
  // CHECK: hipDevice_t device;
  // CHECK-NEXT: hipCtx_t context;
  // CHECK-NEXT: hipFuncCache_t func_cache;
  // CHECK-NEXT: hipLimit_t limit;
  // CHECK-NEXT: hipSharedMemConfig pconfig;
  // CHECK-NEXT: hipFunction_t function;
  // CHECK-NEXT: hipFunction_attribute function_attribute;
  // CHECK-NEXT: hipModule_t module_;
  // CHECK-NEXT: hipDeviceptr_t deviceptr;
  // CHECK-NEXT: hipDeviceptr_t deviceptr_2;
  // CHECK-NEXT: hipTexRef texref;
  // CHECK-NEXT: hipJitOption jit_option;
  // CHECK-NEXT: hipArray_t array_;
  // CHECK-NEXT: HIP_ARRAY3D_DESCRIPTOR ARRAY3D_DESCRIPTOR;
  // CHECK-NEXT: HIP_ARRAY_DESCRIPTOR ARRAY_DESCRIPTOR;
  // CHECK-NEXT: hipIpcEventHandle_t ipcEventHandle;
  // CHECK-NEXT: hipEvent_t event_;
  // CHECK-NEXT: hipEvent_t event_start;
  // CHECK-NEXT: hipEvent_t event_end;
  // CHECK-NEXT: hipIpcMemHandle_t ipcMemHandle;
  // CHECK-NEXT: hip_Memcpy2D MEMCPY2D;
  // CHECK-NEXT: HIP_MEMCPY3D MEMCPY3D;
  // CHECK-NEXT: hipStream_t stream;
  // CHECK-NEXT: hipMipmappedArray_t mipmappedArray;
  // CHECK-NEXT: hipStreamCallback_t streamCallback;
  // CHECK-NEXT: hipPointer_attribute pointer_attribute;
  // CHECK-NEXT: void* occupancyB2DSize;
  // CHECK-NEXT: hipGraphicsResource_t graphicsResource;
  // CHECK-NEXT: hipUUID uuid;
  hipDevice_t device;
  hipCtx_t context;
  hipFuncCache_t func_cache;
  hipLimit_t limit;
  hipSharedMemConfig pconfig;
  hipFunction_t function;
  hipFunction_attribute function_attribute;
  hipModule_t module_;
  hipDeviceptr_t deviceptr;
  hipDeviceptr_t deviceptr_2;
  hipTexRef texref;
  hipJitOption jit_option;
  hipArray_t array_;
  HIP_ARRAY3D_DESCRIPTOR ARRAY3D_DESCRIPTOR;
  HIP_ARRAY_DESCRIPTOR ARRAY_DESCRIPTOR;
  hipIpcEventHandle_t ipcEventHandle;
  hipEvent_t event_;
  hipEvent_t event_start;
  hipEvent_t event_end;
  hipIpcMemHandle_t ipcMemHandle;
  hip_Memcpy2D MEMCPY2D;
  HIP_MEMCPY3D MEMCPY3D;
  hipStream_t stream;
  hipMipmappedArray_t mipmappedArray;
  hipStreamCallback_t streamCallback;
  hipPointer_attribute pointer_attribute;
  void* occupancyB2DSize;
  hipGraphicsResource_t graphicsResource;
  hipUUID uuid;

  // CHECK: hipError_t result;
  hipError_t result;

  unsigned int gridDimX = 0, gridDimY = 0, gridDimZ = 0, blockDimX = 0, blockDimY = 0, blockDimZ = 0, sharedMemBytes = 0;
  void* kernelParams = nullptr, * extra = nullptr;

  // CUDA: hipError_t CUDAAPI hipInit(unsigned int Flags);
  // HIP: hipError_t hipInit(unsigned int flags);
  // CHECK: result = hipInit(flags);
  result = hipInit(flags);

  int driverVersion = 0;
  // CUDA: hipError_t CUDAAPI hipDriverGetVersion(int *driverVersion);
  // HIP: hipError_t hipDriverGetVersion(int* driverVersion);
  // CHECK: result = hipDriverGetVersion(&driverVersion);
  result = hipDriverGetVersion(&driverVersion);

  int ordinal = 0;
  // CUDA: hipError_t CUDAAPI hipDeviceGet(hipDevice_t *device, int ordinal);
  // HIP: hipError_t hipDeviceGet(hipDevice_t* device, int ordinal);
  // CHECK: result = hipDeviceGet(&device, ordinal);
  result = hipDeviceGet(&device, ordinal);

  int pi = 0;
  // CHECK: hipDeviceAttribute_t device_attribute = hipDeviceAttributePciBusId;
  hipDeviceAttribute_t device_attribute = hipDeviceAttributePciBusId;
  // CUDA: hipError_t CUDAAPI hipDeviceGetAttribute(int *pi, hipDeviceAttribute_t attrib, hipDevice_t dev);
  // HIP: hipError_t hipDeviceGetAttribute(int* pi, hipDeviceAttribute_t attr, int deviceId);
  // CHECK: result = hipDeviceGetAttribute(&pi, device_attribute, device);
  result = hipDeviceGetAttribute(&pi, device_attribute, device);

  // CUDA: hipError_t CUDAAPI hipGetDeviceCount(int *count);
  // HIP: hipError_t hipGetDeviceCount(int* count);
  // CHECK: result = hipGetDeviceCount(&count);
  result = hipGetDeviceCount(&count);

  // CUDA: hipError_t CUDAAPI hipDeviceTotalMem(size_t *bytes, hipDevice_t dev);
  // HIP: hipError_t hipDeviceTotalMem(size_t* bytes, hipDevice_t device);
  // CHECK: result = hipDeviceTotalMem(&bytes, device);
  // CHECK-NEXT: result = hipDeviceTotalMem(&bytes, device);
  result = hipDeviceTotalMem(&bytes, device);
  result = hipDeviceTotalMem(&bytes, device);

  int major = 0, minor = 0;
  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipDeviceComputeCapability(int *major, int *minor, hipDevice_t dev);
  // HIP: hipError_t hipDeviceComputeCapability(int* major, int* minor, hipDevice_t device);
  // CHECK: result = hipDeviceComputeCapability(&major, &minor, device);
  result = hipDeviceComputeCapability(&major, &minor, device);

  int active = 0;
  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxGetState(hipDevice_t dev, unsigned int *flags, int *active);
  // HIP: hipError_t hipDevicePrimaryCtxGetState(hipDevice_t dev, unsigned int* flags, int* active);
  // CHECK: result = hipDevicePrimaryCtxGetState(device, &flags, &active);
  result = hipDevicePrimaryCtxGetState(device, &flags, &active);

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxRelease(hipDevice_t dev);
  // HIP: hipError_t hipDevicePrimaryCtxRelease(hipDevice_t dev);
  // CHECK: result = hipDevicePrimaryCtxRelease(device);
  result = hipDevicePrimaryCtxRelease(device);

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxReset(hipDevice_t dev);
  // HIP: hipError_t hipDevicePrimaryCtxReset(hipDevice_t dev);
  // CHECK: result = hipDevicePrimaryCtxReset(device);
  result = hipDevicePrimaryCtxReset(device);

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxRetain(hipCtx_t *pctx, hipDevice_t dev);
  // HIP: hipError_t hipDevicePrimaryCtxRetain(hipCtx_t* pctx, hipDevice_t dev);
  // CHECK: result = hipDevicePrimaryCtxRetain(&context, device);
  result = hipDevicePrimaryCtxRetain(&context, device);

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxSetFlags(hipDevice_t dev, unsigned int flags);
  // HIP: hipError_t hipDevicePrimaryCtxSetFlags(hipDevice_t dev, unsigned int flags);
  // CHECK: result = hipDevicePrimaryCtxSetFlags(device, flags);
  result = hipDevicePrimaryCtxSetFlags(device, flags);

  // CUDA: hipError_t CUDAAPI hipCtxCreate(hipCtx_t *pctx, unsigned int flags, hipDevice_t dev);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxCreate(hipCtx_t *ctx, unsigned int flags, hipDevice_t device);
  // CHECK: result = hipCtxCreate(&context, flags, device);
  // CHECK-NEXT: result = hipCtxCreate(&context, flags, device);
  result = hipCtxCreate(&context, flags, device);
  result = hipCtxCreate(&context, flags, device);

  // CUDA: hipError_t CUDAAPI hipCtxDestroy(hipCtx_t ctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxDestroy(hipCtx_t ctx);
  // CHECK: result = hipCtxDestroy(context);
  // CHECK-NEXT: result = hipCtxDestroy(context);
  result = hipCtxDestroy(context);
  result = hipCtxDestroy(context);

  unsigned int version = 0;
  // CUDA: hipError_t CUDAAPI hipCtxGetApiVersion(hipCtx_t ctx, unsigned int *version);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetApiVersion(hipCtx_t ctx, int* apiVersion);
  // CHECK: result = hipCtxGetApiVersion(context, &version);
  result = hipCtxGetApiVersion(context, &version);

  // CUDA: hipError_t CUDAAPI hipCtxGetCacheConfig(hipFuncCache_t *pconfig);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetCacheConfig(hipFuncCache_t* cacheConfig);
  // CHECK: result = hipCtxGetCacheConfig(&func_cache);
  result = hipCtxGetCacheConfig(&func_cache);

  // CUDA: hipError_t CUDAAPI hipCtxGetCurrent(hipCtx_t *pctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetCurrent(hipCtx_t* ctx);
  // CHECK: result = hipCtxGetCurrent(&context);
  result = hipCtxGetCurrent(&context);

  // CUDA: hipError_t CUDAAPI hipCtxGetDevice(hipDevice_t *device);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetDevice(hipDevice_t* device);
  // CHECK: result = hipCtxGetDevice(&device);
  result = hipCtxGetDevice(&device);

  // CUDA: hipError_t CUDAAPI hipCtxGetFlags(unsigned int *flags);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetFlags(unsigned int* flags);
  // CHECK: result = hipCtxGetFlags(&flags);
  result = hipCtxGetFlags(&flags);

  size_t pvalue = 0;
  // CUDA: hipError_t CUDAAPI hipDeviceGetLimit(size_t *pvalue, hipLimit_t limit);
  // HIP: hipError_t hipDeviceGetLimit(size_t* pValue, enum hipLimit_t limit);
  // CHECK: result = hipDeviceGetLimit(&pvalue, limit);
  result = hipDeviceGetLimit(&pvalue, limit);

  // CUDA: hipError_t CUDAAPI hipCtxGetSharedMemConfig(hipSharedMemConfig *pConfig);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetSharedMemConfig(hipSharedMemConfig* pConfig);
  // CHECK: result = hipCtxGetSharedMemConfig(&pconfig);
  result = hipCtxGetSharedMemConfig(&pconfig);

  int leastPriority = 0, greatestPriority = 0;
  // CUDA: hipError_t CUDAAPI hipDeviceGetStreamPriorityRange(int *leastPriority, int *greatestPriority);
  // HIP: hipError_t hipDeviceGetStreamPriorityRange(int* leastPriority, int* greatestPriority);
  // CHECK: result = hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);
  result = hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);

  // CUDA: hipError_t CUDAAPI hipCtxPopCurrent(hipCtx_t *pctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxPopCurrent(hipCtx_t* ctx);
  // CHECK: result = hipCtxPopCurrent(&context);
  // CHECK-NEXT: result = hipCtxPopCurrent(&context);
  result = hipCtxPopCurrent(&context);
  result = hipCtxPopCurrent(&context);

  // CUDA: hipError_t CUDAAPI hipCtxPushCurrent(hipCtx_t ctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxPushCurrent(hipCtx_t ctx);
  // CHECK: result = hipCtxPushCurrent(context);
  // CHECK-NEXT: result = hipCtxPushCurrent(context);
  result = hipCtxPushCurrent(context);
  result = hipCtxPushCurrent(context);

  // CUDA: hipError_t CUDAAPI hipCtxSetCacheConfig(hipFuncCache_t config);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxSetCacheConfig(hipFuncCache_t cacheConfig);
  // CHECK: result = hipCtxSetCacheConfig(func_cache);
  result = hipCtxSetCacheConfig(func_cache);

  // CUDA: hipError_t CUDAAPI hipCtxSetCurrent(hipCtx_t ctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxSetCurrent(hipCtx_t ctx);
  // CHECK: result = hipCtxSetCurrent(context);
  result = hipCtxSetCurrent(context);

  // CUDA: hipError_t CUDAAPI hipCtxSetSharedMemConfig(hipSharedMemConfig config);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxSetSharedMemConfig(hipSharedMemConfig config);
  // CHECK: result = hipCtxSetSharedMemConfig(pconfig);
  result = hipCtxSetSharedMemConfig(pconfig);

  // CUDA: hipError_t CUDAAPI hipCtxSynchronize(void);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxSynchronize(void);
  // CHECK: result = hipCtxSynchronize();
  result = hipCtxSynchronize();

  // CUDA: hipError_t CUDAAPI hipModuleGetFunction(hipFunction_t *hfunc, hipModule_t hmod, const char *name);
  // HIP: hipError_t hipModuleGetFunction(hipFunction_t* function, hipModule_t module, const char* kname);
  // CHECK: result = hipModuleGetFunction(&function, module_, name.c_str());
  result = hipModuleGetFunction(&function, module_, name.c_str());

  // CUDA: hipError_t CUDAAPI hipModuleGetGlobal(hipDeviceptr_t *dptr, size_t *bytes, hipModule_t hmod, const char *name);
  // HIP: hipError_t hipModuleGetGlobal(hipDeviceptr_t* dptr, size_t* bytes, hipModule_t hmod, const char* name);
  // CHECK: result = hipModuleGetGlobal(&deviceptr, &bytes, module_, name.c_str());
  // CHECK-NEXT: result = hipModuleGetGlobal(&deviceptr, &bytes, module_, name.c_str());
  result = hipModuleGetGlobal(&deviceptr, &bytes, module_, name.c_str());
  result = hipModuleGetGlobal(&deviceptr, &bytes, module_, name.c_str());

  // CUDA: hipError_t CUDAAPI hipModuleGetTexRef(hipTexRef *pTexRef, hipModule_t hmod, const char *name);
  // HIP: hipError_t hipModuleGetTexRef(textureReference** texRef, hipModule_t hmod, const char* name);
  // CHECK: result = hipModuleGetTexRef(&texref, module_, name.c_str());
  result = hipModuleGetTexRef(&texref, module_, name.c_str());

  // CUDA: hipError_t CUDAAPI hipModuleLoad(hipModule_t *module, const char *fname);
  // HIP: hipError_t hipModuleLoad(hipModule_t* module, const char* fname);
  // CHECK: result = hipModuleLoad(&module_, name.c_str());
  result = hipModuleLoad(&module_, name.c_str());

  // CUDA: hipError_t CUDAAPI hipModuleLoadData(hipModule_t *module, const void *image);
  // HIP: hipError_t hipModuleLoadData(hipModule_t* module, const void* image);
  // CHECK: result = hipModuleLoadData(&module_, image);
  result = hipModuleLoadData(&module_, image);

  unsigned int numOptions = 0;
  void* optionValues = nullptr;
  // CUDA: hipError_t CUDAAPI hipModuleLoadDataEx(hipModule_t *module, const void *image, unsigned int numOptions, hipJitOption *options, void **optionValues);
  // HIP: hipError_t hipModuleLoadDataEx(hipModule_t* module, const void* image, unsigned int numOptions, hipJitOption* options, void** optionValues);
  // CHECK: result = hipModuleLoadDataEx(&module_, image, numOptions, &jit_option, &optionValues);
  result = hipModuleLoadDataEx(&module_, image, numOptions, &jit_option, &optionValues);

  // CUDA: hipError_t CUDAAPI hipModuleUnload(hipModule_t hmod);
  // HIP: hipError_t hipModuleUnload(hipModule_t module);
  // CHECK: result = hipModuleUnload(module_);
  result = hipModuleUnload(module_);

  // CUDA: hipError_t CUDAAPI hipArray3DCreate(hipArray_t *pHandle, const HIP_ARRAY3D_DESCRIPTOR *pAllocateArray);
  // HIP: hipError_t hipArray3DCreate(hipArray** array, const HIP_ARRAY3D_DESCRIPTOR* pAllocateArray);
  // CHECK: result = hipArray3DCreate(&array_, &ARRAY3D_DESCRIPTOR);
  // CHECK-NEXT: result = hipArray3DCreate(&array_, &ARRAY3D_DESCRIPTOR);
  result = hipArray3DCreate(&array_, &ARRAY3D_DESCRIPTOR);
  result = hipArray3DCreate(&array_, &ARRAY3D_DESCRIPTOR);

  // CUDA: hipError_t CUDAAPI hipArrayCreate(hipArray_t *pHandle, const HIP_ARRAY_DESCRIPTOR *pAllocateArray);
  // HIP: hipError_t hipArrayCreate(hipArray** pHandle, const HIP_ARRAY_DESCRIPTOR* pAllocateArray);
  // CHECK: result = hipArrayCreate(&array_, &ARRAY_DESCRIPTOR);
  // CHECK: result = hipArrayCreate(&array_, &ARRAY_DESCRIPTOR);
  result = hipArrayCreate(&array_, &ARRAY_DESCRIPTOR);
  result = hipArrayCreate(&array_, &ARRAY_DESCRIPTOR);

  // CUDA: hipError_t CUDAAPI hipArrayDestroy(hipArray_t hArray);
  // HIP: hipError_t hipArrayDestroy(hipArray* array);
  // CHECK: result = hipArrayDestroy(array_);
  result = hipArrayDestroy(array_);

  std::string pciBusId;
  // CUDA: hipError_t CUDAAPI hipDeviceGetByPCIBusId(hipDevice_t *dev, const char *pciBusId);
  // HIP: hipError_t hipDeviceGetByPCIBusId(int* device, const char* pciBusId);
  // CHECK: result = hipDeviceGetByPCIBusId(&device, pciBusId.c_str());
  result = hipDeviceGetByPCIBusId(&device, pciBusId.c_str());

  int len = 0;
  char* pciBusId_ = const_cast<char*>(pciBusId.c_str());
  // CUDA: hipError_t CUDAAPI hipDeviceGetPCIBusId(char *pciBusId, int len, hipDevice_t dev);
  // HIP: hipError_t hipDeviceGetPCIBusId(char* pciBusId, int len, int device);
  // CHECK: result = hipDeviceGetPCIBusId(pciBusId_, len, device);
  result = hipDeviceGetPCIBusId(pciBusId_, len, device);

  // CUDA: hipError_t CUDAAPI hipIpcCloseMemHandle(hipDeviceptr_t dptr);
  // HIP: hipError_t hipIpcCloseMemHandle(void* devPtr);
  // CHECK: result = hipIpcCloseMemHandle(deviceptr);
  result = hipIpcCloseMemHandle(deviceptr);

  // CUDA: hipError_t CUDAAPI hipIpcGetEventHandle(hipIpcEventHandle_t *pHandle, hipEvent_t event);
  // HIP: hipError_t hipIpcGetEventHandle(hipIpcEventHandle_t* handle, hipEvent_t event);
  // CHECK: result = hipIpcGetEventHandle(&ipcEventHandle, event_);
  result = hipIpcGetEventHandle(&ipcEventHandle, event_);

  // CUDA: hipError_t CUDAAPI hipIpcGetMemHandle(hipIpcMemHandle_t *pHandle, hipDeviceptr_t dptr);
  // HIP: hipError_t hipIpcGetMemHandle(hipIpcMemHandle_t* handle, void* devPtr);
  // CHECK: result = hipIpcGetMemHandle(&ipcMemHandle, deviceptr);
  result = hipIpcGetMemHandle(&ipcMemHandle, deviceptr);

  // CUDA: hipError_t CUDAAPI hipIpcOpenEventHandle(hipEvent_t *phEvent, hipIpcEventHandle_t handle);
  // HIP: hipError_t hipIpcOpenEventHandle(hipEvent_t* event, hipIpcEventHandle_t handle);
  // CHECK: result = hipIpcOpenEventHandle(&event_, ipcEventHandle);
  result = hipIpcOpenEventHandle(&event_, ipcEventHandle);

  // CUDA: hipError_t CUDAAPI hipIpcOpenMemHandle(hipDeviceptr_t *pdptr, hipIpcMemHandle_t handle, unsigned int Flags);
  // HIP: hipError_t hipIpcOpenMemHandle(void** devPtr, hipIpcMemHandle_t handle, unsigned int flags);
  // CHECK: result = hipIpcOpenMemHandle(&deviceptr, ipcMemHandle, flags);
  result = hipIpcOpenMemHandle(&deviceptr, ipcMemHandle, flags);

  // CUDA: hipError_t CUDAAPI hipMalloc(hipDeviceptr_t *dptr, size_t bytesize);
  // HIP: hipError_t hipMalloc(void** ptr, size_t size);
  // CHECK: result = hipMalloc(&deviceptr, bytes);
  // CHECK-NEXT: result = hipMalloc(&deviceptr, bytes);
  result = hipMalloc(&deviceptr, bytes);
  result = hipMalloc(&deviceptr, bytes);

  // CUDA: hipError_t CUDAAPI hipMemAllocHost(void **pp, size_t bytesize);
  // HIP: DEPRECATED("use hipHostMalloc instead") hipError_t hipMemAllocHost(void** ptr, size_t size);
  // CHECK: result = hipMemAllocHost(&image, bytes);
  // CHECK-NEXT: result = hipMemAllocHost(&image, bytes);
  result = hipMemAllocHost(&image, bytes);
  result = hipMemAllocHost(&image, bytes);

  // CUDA: hipError_t CUDAAPI hipMallocManaged(hipDeviceptr_t *dptr, size_t bytesize, unsigned int flags);
  // HIP: hipError_t hipMallocManaged(void** dev_ptr, size_t size, unsigned int flags __dparm(hipMemAttachGlobal));
  // CHECK: result = hipMallocManaged(&deviceptr, bytes, flags);
  result = hipMallocManaged(&deviceptr, bytes, flags);

  size_t pitch = 0, width = 0, height = 0;
  // CUDA: hipError_t CUDAAPI hipMemAllocPitch(hipDeviceptr_t *dptr, size_t *pPitch, size_t WidthInBytes, size_t Height, unsigned int ElementSizeBytes);
  // HIP: hipError_t hipMemAllocPitch(hipDeviceptr_t* dptr, size_t* pitch, size_t widthInBytes, size_t height, unsigned int elementSizeBytes);
  // CHECK: result = hipMemAllocPitch(&deviceptr, &pitch, width, height, bytes);
  // CHECK-NEXT: result = hipMemAllocPitch(&deviceptr, &pitch, width, height, bytes);
  result = hipMemAllocPitch(&deviceptr, &pitch, width, height, bytes);
  result = hipMemAllocPitch(&deviceptr, &pitch, width, height, bytes);

  // CUDA: hipError_t CUDAAPI hipMemcpyParam2D(const hip_Memcpy2D *pCopy);
  // HIP: hipError_t hipMemcpyParam2D(const hip_Memcpy2D* pCopy);
  // CHECK: result = hipMemcpyParam2D(&MEMCPY2D);
  // CHECK-NEXT: result = hipMemcpyParam2D(&MEMCPY2D);
  result = hipMemcpyParam2D(&MEMCPY2D);
  result = hipMemcpyParam2D(&MEMCPY2D);

  // CUDA: hipError_t CUDAAPI hipMemcpyParam2DAsync(const hip_Memcpy2D *pCopy, hipStream_t hStream);
  // HIP: hipError_t hipMemcpyParam2DAsync(const hip_Memcpy2D* pCopy, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpyParam2DAsync(&MEMCPY2D, stream);
  // CHECK-NEXT: result = hipMemcpyParam2DAsync(&MEMCPY2D, stream);
  result = hipMemcpyParam2DAsync(&MEMCPY2D, stream);
  result = hipMemcpyParam2DAsync(&MEMCPY2D, stream);

  // CUDA: hipError_t CUDAAPI hipDrvMemcpy2DUnaligned(const hip_Memcpy2D *pCopy);
  // HIP: hipError_t hipDrvMemcpy2DUnaligned(const hip_Memcpy2D* pCopy);
  // CHECK: result = hipDrvMemcpy2DUnaligned(&MEMCPY2D);
  // CHECK-NEXT: result = hipDrvMemcpy2DUnaligned(&MEMCPY2D);
  result = hipDrvMemcpy2DUnaligned(&MEMCPY2D);
  result = hipDrvMemcpy2DUnaligned(&MEMCPY2D);

  // CUDA: hipError_t CUDAAPI hipDrvMemcpy3D(const HIP_MEMCPY3D *pCopy);
  // HIP: hipError_t hipDrvMemcpy3D(const HIP_MEMCPY3D* pCopy);
  // CHECK: result = hipDrvMemcpy3D(&MEMCPY3D);
  // CHECK-NEXT: result = hipDrvMemcpy3D(&MEMCPY3D);
  result = hipDrvMemcpy3D(&MEMCPY3D);
  result = hipDrvMemcpy3D(&MEMCPY3D);

  // CUDA: hipError_t CUDAAPI hipDrvMemcpy3DAsync(const HIP_MEMCPY3D *pCopy, hipStream_t hStream);
  // HIP: hipError_t hipDrvMemcpy3DAsync(const HIP_MEMCPY3D* pCopy, hipStream_t stream);
  // CHECK: result = hipDrvMemcpy3DAsync(&MEMCPY3D, stream);
  // CHECK-NEXT: result = hipDrvMemcpy3DAsync(&MEMCPY3D, stream);
  result = hipDrvMemcpy3DAsync(&MEMCPY3D, stream);
  result = hipDrvMemcpy3DAsync(&MEMCPY3D, stream);

  void* dsthost = nullptr;
  size_t offset = 0;
  // CUDA: hipError_t CUDAAPI hipMemcpyAtoH(void *dstHost, hipArray_t srcArray, size_t srcOffset, size_t ByteCount);
  // HIP: hipError_t hipMemcpyAtoH(void* dst, hipArray* srcArray, size_t srcOffset, size_t count);
  // CHECK: result = hipMemcpyAtoH(dsthost, array_, offset, bytes);
  // CHECK-NEXT: result = hipMemcpyAtoH(dsthost, array_, offset, bytes);
  result = hipMemcpyAtoH(dsthost, array_, offset, bytes);
  result = hipMemcpyAtoH(dsthost, array_, offset, bytes);

  // CUDA: hipError_t CUDAAPI hipMemcpyDtoD(hipDeviceptr_t dstDevice, hipDeviceptr_t srcDevice, size_t ByteCount);
  // HIP: hipError_t hipMemcpyDtoD(hipDeviceptr_t dst, hipDeviceptr_t src, size_t sizeBytes);
  // CHECK: result = hipMemcpyDtoD(deviceptr, deviceptr, bytes);
  // CHECK-NEXT: result = hipMemcpyDtoD(deviceptr, deviceptr, bytes);
  result = hipMemcpyDtoD(deviceptr, deviceptr, bytes);
  result = hipMemcpyDtoD(deviceptr, deviceptr, bytes);

  // CUDA: hipError_t CUDAAPI hipMemcpyDtoDAsync(hipDeviceptr_t dstDevice, hipDeviceptr_t srcDevice, size_t ByteCount, hipStream_t hStream);
  // HIP: hipError_t hipMemcpyDtoDAsync(hipDeviceptr_t dst, hipDeviceptr_t src, size_t sizeBytes, hipStream_t stream);
  // CHECK: result = hipMemcpyDtoDAsync(deviceptr, deviceptr, bytes, stream);
  // CHECK-NEXT: result = hipMemcpyDtoDAsync(deviceptr, deviceptr, bytes, stream);
  result = hipMemcpyDtoDAsync(deviceptr, deviceptr, bytes, stream);
  result = hipMemcpyDtoDAsync(deviceptr, deviceptr, bytes, stream);

  // CUDA: hipError_t CUDAAPI hipMemcpyDtoH(void *dstHost, hipDeviceptr_t srcDevice, size_t ByteCount);
  // HIP: hipError_t hipMemcpyDtoH(void* dst, hipDeviceptr_t src, size_t sizeBytes);
  // CHECK: result = hipMemcpyDtoH(dsthost, deviceptr, bytes);
  // CHECK-NEXT: result = hipMemcpyDtoH(dsthost, deviceptr, bytes);
  result = hipMemcpyDtoH(dsthost, deviceptr, bytes);
  result = hipMemcpyDtoH(dsthost, deviceptr, bytes);

  // CUDA: hipError_t CUDAAPI hipMemcpyDtoHAsync(void *dstHost, hipDeviceptr_t srcDevice, size_t ByteCount, hipStream_t hStream);
  // HIP: hipError_t hipMemcpyDtoHAsync(void* dst, hipDeviceptr_t src, size_t sizeBytes, hipStream_t stream);
  // CHECK: result = hipMemcpyDtoHAsync(dsthost, deviceptr, bytes, stream);
  // CHECK-NEXT: result = hipMemcpyDtoHAsync(dsthost, deviceptr, bytes, stream);
  result = hipMemcpyDtoHAsync(dsthost, deviceptr, bytes, stream);
  result = hipMemcpyDtoHAsync(dsthost, deviceptr, bytes, stream);

  // CUDA: hipError_t CUDAAPI hipMemcpyHtoA(hipArray_t dstArray, size_t dstOffset, const void *srcHost, size_t ByteCount);
  // HIP: hipError_t hipMemcpyHtoA(hipArray* dstArray, size_t dstOffset, const void* srcHost, size_t count);
  // CHECK: result = hipMemcpyHtoA(array_, offset, dsthost, bytes);
  // CHECK-NEXT: result = hipMemcpyHtoA(array_, offset, dsthost, bytes);
  result = hipMemcpyHtoA(array_, offset, dsthost, bytes);
  result = hipMemcpyHtoA(array_, offset, dsthost, bytes);

  // CUDA: hipError_t CUDAAPI hipMemcpyHtoD(hipDeviceptr_t dstDevice, const void *srcHost, size_t ByteCount);
  // HIP: hipError_t hipMemcpyHtoD(hipDeviceptr_t dst, void* src, size_t sizeBytes);
  // CHECK: result = hipMemcpyHtoD(deviceptr, dsthost, bytes);
  // CHECK-NEXT: result = hipMemcpyHtoD(deviceptr, dsthost, bytes);
  result = hipMemcpyHtoD(deviceptr, dsthost, bytes);
  result = hipMemcpyHtoD(deviceptr, dsthost, bytes);

  // CUDA: hipError_t CUDAAPI hipMemcpyHtoDAsync(hipDeviceptr_t dstDevice, const void *srcHost, size_t ByteCount, hipStream_t hStream);
  // HIP: hipError_t hipMemcpyHtoDAsync(hipDeviceptr_t dst, void* src, size_t sizeBytes, hipStream_t stream);
  // CHECK: result = hipMemcpyHtoDAsync(deviceptr, dsthost, bytes, stream);
  // CHECK-NEXT: result = hipMemcpyHtoDAsync(deviceptr, dsthost, bytes, stream);
  result = hipMemcpyHtoDAsync(deviceptr, dsthost, bytes, stream);
  result = hipMemcpyHtoDAsync(deviceptr, dsthost, bytes, stream);

  // CUDA: hipError_t CUDAAPI hipFree(hipDeviceptr_t dptr);
  // HIP: hipError_t hipFree(void* ptr);
  // CHECK: result = hipFree(deviceptr);
  // CHECK-NEXT: result = hipFree(deviceptr);
  result = hipFree(deviceptr);
  result = hipFree(deviceptr);

  // CUDA: hipError_t CUDAAPI hipHostFree(void *p);
  // HIP: hipError_t hipHostFree(void* ptr);
  // CHECK: result = hipHostFree(image);
  result = hipHostFree(image);

  // CUDA: hipError_t CUDAAPI hipMemGetAddressRange(hipDeviceptr_t *pbase, size_t *psize, hipDeviceptr_t dptr);
  // HIP: hipError_t hipMemGetAddressRange(hipDeviceptr_t* pbase, size_t* psize, hipDeviceptr_t dptr);
  // CHECK: result = hipMemGetAddressRange(&deviceptr, &bytes, deviceptr_2);
  // CHECK-NEXT: result = hipMemGetAddressRange(&deviceptr, &bytes, deviceptr_2);
  result = hipMemGetAddressRange(&deviceptr, &bytes, deviceptr_2);
  result = hipMemGetAddressRange(&deviceptr, &bytes, deviceptr_2);

  // CUDA: hipError_t CUDAAPI hipMemGetInfo(size_t *free, size_t *total);
  // HIP: hipError_t hipMemGetInfo(size_t* free, size_t* total);
  // CHECK: result = hipMemGetInfo(&bytes, &bytes_2);
  // CHECK-NEXT: result = hipMemGetInfo(&bytes, &bytes_2);
  result = hipMemGetInfo(&bytes, &bytes_2);
  result = hipMemGetInfo(&bytes, &bytes_2);

  // CUDA: hipError_t CUDAAPI hipHostAlloc(void **pp, size_t bytesize, unsigned int Flags);
  // HIP: DEPRECATED("use hipHostMalloc instead") hipError_t hipHostAlloc(void** ptr, size_t size, unsigned int flags);
  // CHECK: result = hipHostAlloc(&image, bytes, flags);
  result = hipHostAlloc(&image, bytes, flags);

  // CUDA: hipError_t CUDAAPI hipHostGetDevicePointer(hipDeviceptr_t *pdptr, void *p, unsigned int Flags);
  // HIP: hipError_t hipHostGetDevicePointer(void** devPtr, void* hstPtr, unsigned int flags);
  // CHECK: result = hipHostGetDevicePointer(&deviceptr, image, flags);
  // CHECK-NEXT: result = hipHostGetDevicePointer(&deviceptr, image, flags);
  result = hipHostGetDevicePointer(&deviceptr, image, flags);
  result = hipHostGetDevicePointer(&deviceptr, image, flags);

  // CUDA: hipError_t CUDAAPI hipHostGetFlags(unsigned int *pFlags, void *p);
  // HIP: hipError_t hipHostGetFlags(&flags, image);
  // CHECK: result = hipHostGetFlags(&flags, image);
  result = hipHostGetFlags(&flags, image);

  // CUDA: hipError_t CUDAAPI hipHostRegister(void *p, size_t bytesize, unsigned int Flags);
  // HIP: hipError_t hipHostRegister(void* hostPtr, size_t sizeBytes, unsigned int flags);
  // CHECK: result = hipHostRegister(image, bytes, flags);
  // CHECK-NEXT: result = hipHostRegister(image, bytes, flags);
  result = hipHostRegister(image, bytes, flags);
  result = hipHostRegister(image, bytes, flags);

  // CUDA: hipError_t CUDAAPI hipHostUnregister(void *p);
  // HIP: hipError_t hipHostUnregister(void* hostPtr);
  // CHECK: result = hipHostUnregister(image);
  result = hipHostUnregister(image);

  unsigned short us = 0;
  // CUDA: hipError_t CUDAAPI hipMemsetD16(hipDeviceptr_t dstDevice, unsigned short us, size_t N);
  // HIP: hipError_t hipMemsetD16(hipDeviceptr_t dest, unsigned short value, size_t count);
  // CHECK: result = hipMemsetD16(deviceptr, us, bytes);
  // CHECK-NEXT: result = hipMemsetD16(deviceptr, us, bytes);
  result = hipMemsetD16(deviceptr, us, bytes);
  result = hipMemsetD16(deviceptr, us, bytes);

  // CUDA: hipError_t CUDAAPI hipMemsetD16Async(hipDeviceptr_t dstDevice, unsigned short us, size_t N, hipStream_t hStream);
  // HIP: hipError_t hipMemsetD16Async(hipDeviceptr_t dest, unsigned short value, size_t count, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemsetD16Async(deviceptr, us, bytes, stream);
  result = hipMemsetD16Async(deviceptr, us, bytes, stream);

  // CUDA: hipError_t CUDAAPI hipMemsetD32(hipDeviceptr_t dstDevice, unsigned int ui, size_t N)
  // HIP: hipError_t hipMemsetD32(hipDeviceptr_t dest, int value, size_t count);
  // CHECK: result = hipMemsetD32(deviceptr, flags, bytes);
  // CHECK-NEXT: result = hipMemsetD32(deviceptr, flags, bytes);
  result = hipMemsetD32(deviceptr, flags, bytes);
  result = hipMemsetD32(deviceptr, flags, bytes);

  // CUDA: hipError_t CUDAAPI hipMemsetD32Async(hipDeviceptr_t dstDevice, unsigned int ui, size_t N, hipStream_t hStream);
  // HIP: hipError_t hipMemsetD32Async(hipDeviceptr_t dst, int value, size_t count, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemsetD32Async(deviceptr, flags, bytes, stream);
  result = hipMemsetD32Async(deviceptr, flags, bytes, stream);

  unsigned char uc = 0;
  // CUDA: hipError_t CUDAAPI hipMemsetD8(hipDeviceptr_t dstDevice, unsigned char uc, size_t N);
  // HIP: hipError_t hipMemsetD8(hipDeviceptr_t dest, unsigned char value, size_t count);
  // CHECK: result = hipMemsetD8(deviceptr, uc, bytes);
  // CHECK-NEXT: result = hipMemsetD8(deviceptr, uc, bytes);
  result = hipMemsetD8(deviceptr, uc, bytes);
  result = hipMemsetD8(deviceptr, uc, bytes);

  // CUDA: hipError_t CUDAAPI hipMemsetD8Async(hipDeviceptr_t dstDevice, unsigned char uc, size_t N, hipStream_t hStream);
  // HIP: hipError_t hipMemsetD8Async(hipDeviceptr_t dest, unsigned char value, size_t count, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemsetD8Async(deviceptr, uc, bytes, stream);
  result = hipMemsetD8Async(deviceptr, uc, bytes, stream);

  // CUDA: hipError_t CUDAAPI hipMipmappedArrayCreate(hipMipmappedArray_t *pHandle, const HIP_ARRAY3D_DESCRIPTOR *pMipmappedArrayDesc, unsigned int numMipmapLevels);
  // HIP: hipError_t hipMipmappedArrayCreate(hipMipmappedArray_t* pHandle, HIP_ARRAY3D_DESCRIPTOR* pMipmappedArrayDesc, unsigned int numMipmapLevels);
  // CHECK: result = hipMipmappedArrayCreate(&mipmappedArray, &ARRAY3D_DESCRIPTOR, flags);
  result = hipMipmappedArrayCreate(&mipmappedArray, &ARRAY3D_DESCRIPTOR, flags);

  // CUDA: hipError_t CUDAAPI hipMipmappedArrayDestroy(hipMipmappedArray_t hMipmappedArray);
  // HIP: hipError_t hipMipmappedArrayDestroy(hipMipmappedArray_t hMipmappedArray);
  // CHECK: result = hipMipmappedArrayDestroy(mipmappedArray);
  result = hipMipmappedArrayDestroy(mipmappedArray);

  // CUDA: hipError_t CUDAAPI hipMipmappedArrayGetLevel(hipArray_t *pLevelArray, hipMipmappedArray_t hMipmappedArray, unsigned int level);
  // HIP: hipError_t hipMipmappedArrayGetLevel(hipArray_t* pLevelArray, hipMipmappedArray_t hMipMappedArray, unsigned int level);
  // CHECK: result = hipMipmappedArrayGetLevel(&array_, mipmappedArray, flags);
  result = hipMipmappedArrayGetLevel(&array_, mipmappedArray, flags);

  // CUDA: hipError_t CUDAAPI hipPointerGetAttribute(void *data, hipPointer_attribute attribute, hipDeviceptr_t ptr);
  // HIP: hipError_t hipPointerGetAttribute(void* data, hipPointer_attribute attribute, hipDeviceptr_t ptr);
  // CHECK: result = hipPointerGetAttribute(image, pointer_attribute, deviceptr);
  result = hipPointerGetAttribute(image, pointer_attribute, deviceptr);

  // CUDA: hipError_t CUDAAPI hipDrvPointerGetAttributes(unsigned int numAttributes, hipPointer_attribute *attributes, void **data, hipDeviceptr_t ptr);
  // HIP: hipError_t hipDrvPointerGetAttributes(unsigned int numAttributes, hipPointer_attribute* attributes, void** data, hipDeviceptr_t ptr);
  // CHECK: result = hipDrvPointerGetAttributes(flags, &pointer_attribute, &image, deviceptr);
  result = hipDrvPointerGetAttributes(flags, &pointer_attribute, &image, deviceptr);

  // CUDA: hipError_t CUDAAPI hipStreamAddCallback(hipStream_t hStream, hipStreamCallback_t callback, void *userData, unsigned int flags);
  // HIP: hipError_t hipStreamAddCallback(hipStream_t stream, hipStreamCallback_t callback, void* userData, unsigned int flags);
  // CHECK: result = hipStreamAddCallback(stream, streamCallback, image, flags);
  result = hipStreamAddCallback(stream, streamCallback, image, flags);

  // CUDA: hipError_t CUDAAPI hipStreamAttachMemAsync(hipStream_t hStream, hipDeviceptr_t dptr, size_t length, unsigned int flags);
  // HIP: hipError_t hipStreamAttachMemAsync(hipStream_t stream, void* dev_ptr, size_t length __dparm(0), unsigned int flags __dparm(hipMemAttachSingle));
  // CHECK: result = hipStreamAttachMemAsync(stream, deviceptr, bytes, flags);
  result = hipStreamAttachMemAsync(stream, deviceptr, bytes, flags);

  // CUDA: hipError_t CUDAAPI hipStreamCreateWithFlags(hipStream_t *phStream, unsigned int Flags);
  // HIP: hipError_t hipStreamCreateWithFlags(hipStream_t* stream, unsigned int flags);
  // CHECK: result = hipStreamCreateWithFlags(&stream, flags);
  result = hipStreamCreateWithFlags(&stream, flags);

  // CUDA: hipError_t CUDAAPI hipStreamCreateWithPriority(hipStream_t *phStream, unsigned int flags, int priority);
  // HIP: hipError_t hipStreamCreateWithPriority(hipStream_t* stream, unsigned int flags, int priority);
  // CHECK: result = hipStreamCreateWithPriority(&stream, flags, leastPriority);
  result = hipStreamCreateWithPriority(&stream, flags, leastPriority);

  // CUDA: hipError_t CUDAAPI hipStreamDestroy(hipStream_t hStream);
  // HIP: hipError_t hipStreamDestroy(hipStream_t stream);
  // CHECK: result = hipStreamDestroy(stream);
  // CHECK-NEXT: result = hipStreamDestroy(stream);
  result = hipStreamDestroy(stream);
  result = hipStreamDestroy(stream);

  // CUDA: hipError_t CUDAAPI hipStreamGetFlags(hipStream_t hStream, unsigned int *flags);
  // HIP: hipError_t hipStreamGetFlags(hipStream_t stream, unsigned int* flags);
  // CHECK: result = hipStreamGetFlags(stream, &flags);
  result = hipStreamGetFlags(stream, &flags);

  // CUDA: hipError_t CUDAAPI hipStreamGetPriority(hipStream_t hStream, int *priority);
  // HIP: hipError_t hipStreamGetPriority(hipStream_t stream, int* priority);
  // CHECK: result = hipStreamGetPriority(stream, &leastPriority);
  result = hipStreamGetPriority(stream, &leastPriority);

  // CUDA: hipError_t CUDAAPI hipStreamQuery(hipStream_t hStream);
  // HIP: hipError_t hipStreamQuery(hipStream_t stream);
  // CHECK: result = hipStreamQuery(stream);
  result = hipStreamQuery(stream);

  // CUDA: hipError_t CUDAAPI hipStreamSynchronize(hipStream_t hStream);
  // HIP: hipError_t hipStreamSynchronize(hipStream_t stream);
  // CHECK: result = hipStreamSynchronize(stream);
  result = hipStreamSynchronize(stream);

  // CUDA: hipError_t CUDAAPI hipStreamWaitEvent(hipStream_t hStream, hipEvent_t hEvent, unsigned int Flags);
  // HIP: hipError_t hipStreamWaitEvent(hipStream_t stream, hipEvent_t event, unsigned int flags);
  // CHECK: result = hipStreamWaitEvent(stream, event_, flags);
  result = hipStreamWaitEvent(stream, event_, flags);

  // CUDA: hipError_t CUDAAPI hipEventCreateWithFlags(hipEvent_t *phEvent, unsigned int Flags);
  // HIP: hipError_t hipEventCreateWithFlags(hipEvent_t* event, unsigned flags);
  // CHECK: result = hipEventCreateWithFlags(&event_, flags);
  result = hipEventCreateWithFlags(&event_, flags);

  // CUDA: hipError_t CUDAAPI hipEventDestroy(hipEvent_t hEvent);
  // HIP: hipError_t hipEventDestroy(hipEvent_t event);
  // CHECK: result = hipEventDestroy(event_);
  // CHECK-NEXT: result = hipEventDestroy(event_);
  result = hipEventDestroy(event_);
  result = hipEventDestroy(event_);

  // CUDA: hipError_t CUDAAPI hipEventElapsedTime(float *pMilliseconds, hipEvent_t hStart, hipEvent_t hEnd);
  // HIP: hipError_t hipEventElapsedTime(float* ms, hipEvent_t start, hipEvent_t stop);
  // CHECK: result = hipEventElapsedTime(&ms, event_start, event_end);
  result = hipEventElapsedTime(&ms, event_start, event_end);

  // CUDA: hipError_t CUDAAPI hipEventRecord(hipEvent_t hEvent, hipStream_t hStream);
  // HIP: hipError_t hipEventRecord(hipEvent_t event, hipStream_t stream);
  // CHECK: result = hipEventRecord(event_, stream);
  result = hipEventRecord(event_, stream);

  // CUDA: hipError_t CUDAAPI hipEventSynchronize(hipEvent_t hEvent);
  // HIP: hipError_t hipEventSynchronize(hipEvent_t event);
  // CHECK: result = hipEventSynchronize(event_);
  result = hipEventSynchronize(event_);

  // CUDA: hipError_t CUDAAPI hipFuncGetAttribute(int *pi, hipFunction_attribute attrib, hipFunction_t hfunc);
  // HIP: hipError_t hipFuncGetAttribute(int* value, hipFunction_attribute attrib, hipFunction_t hfunc);
  // CHECK: result = hipFuncGetAttribute(value, function_attribute, function);
  result = hipFuncGetAttribute(value, function_attribute, function);

  // CUDA: hipError_t CUDAAPI hipModuleLaunchKernel(hipFunction_t f, unsigned int gridDimX, unsigned int gridDimY, unsigned int gridDimZ, unsigned int blockDimX, unsigned int blockDimY, unsigned int blockDimZ, unsigned int sharedMemBytes, hipStream_t hStream, void **kernelParams, void **extra);
  // HIP: hipError_t hipModuleLaunchKernel(hipFunction_t f, unsigned int gridDimX, unsigned int gridDimY, unsigned int gridDimZ, unsigned int blockDimX, unsigned int blockDimY, unsigned int blockDimZ, unsigned int sharedMemBytes, hipStream_t stream, void** kernelParams, void** extra);
  // CHECK: result = hipModuleLaunchKernel(function, gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, sharedMemBytes, stream, &kernelParams, &extra);
  result = hipModuleLaunchKernel(function, gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, sharedMemBytes, stream, &kernelParams, &extra);

  // CUDA: hipError_t CUDAAPI hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(int *numBlocks, hipFunction_t func, int blockSize, size_t dynamicSMemSize);
  // HIP: hipError_t hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(int* numBlocks, hipFunction_t f, int blockSize, size_t dynSharedMemPerBlk);
  // CHECK: result = hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(value, function, iBlockSize, bytes);
  result = hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(value, function, iBlockSize, bytes);

  // CUDA: hipError_t CUDAAPI hipModuleOccupancyMaxPotentialBlockSize(int *minGridSize, int *blockSize, hipFunction_t func, size_t dynamicSMemSize, int blockSizeLimit);
  // HIP: hipError_t hipModuleOccupancyMaxPotentialBlockSize(int* gridSize, int* blockSize, hipFunction_t f, int blockSizeLimit);
  // CHECK: result = hipModuleOccupancyMaxPotentialBlockSize(value, value_2, function, iBlockSize);
  result = hipModuleOccupancyMaxPotentialBlockSize(value, value_2, function, bytes, iBlockSize);

  // CUDA: hipError_t CUDAAPI hipModuleOccupancyMaxPotentialBlockSizeWithFlags(int *minGridSize, int *blockSize, hipFunction_t func, int blockSizeLimit, unsigned int flags);
  // HIP: hipError_t hipModuleOccupancyMaxPotentialBlockSizeWithFlags(int* gridSize, int* blockSize, hipFunction_t f, unsigned int flags);
  // CHECK: result = hipModuleOccupancyMaxPotentialBlockSizeWithFlags(value, value_2, function, iBlockSize_2);
  result = hipModuleOccupancyMaxPotentialBlockSizeWithFlags(value, value_2, function, iBlockSize, iBlockSize_2);

  // CUDA: hipError_t CUDAAPI hipGraphicsGLRegisterImage(hipGraphicsResource_t *pCudaResource, GLuint image, GLenum target, unsigned int Flags);
  // HIP: hipError_t hipGraphicsGLRegisterImage(hipGraphicsResource** resource, GLuint image, GLenum target, unsigned int flags);
  // CHECK: result = hipGraphicsGLRegisterImage(&graphicsResource, gl_uint, gl_enum, flags);
  result = hipGraphicsGLRegisterImage(&graphicsResource, gl_uint, gl_enum, flags);

  // CUDA: hipError_t CUDAAPI hipGraphicsSubResourceGetMappedArray(hipArray_t *pArray, hipGraphicsResource_t resource, unsigned int arrayIndex, unsigned int mipLevel);
  // HIP: hipError_t hipGraphicsSubResourceGetMappedArray(hipArray_t* array, hipGraphicsResource_t resource, unsigned int arrayIndex, unsigned int mipLevel);
  // CHECK: result = hipGraphicsSubResourceGetMappedArray(&array_, graphicsResource, flags, flags_2);
  result = hipGraphicsSubResourceGetMappedArray(&array_, graphicsResource, flags, flags_2);

  // CUDA: hipError_t CUDAAPI hipModuleOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(int *numBlocks, hipFunction_t func, int blockSize, size_t dynamicSMemSize, unsigned int flags);
  // HIP: hipError_t hipModuleOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(int* numBlocks, hipFunction_t f, int blockSize, size_t dynSharedMemPerBlk, unsigned int flags);
  // CHECK: result = hipModuleOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(&iBlockSize, function, iBlockSize_2, bytes, flags);
  result = hipModuleOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(&iBlockSize, function, iBlockSize_2, bytes, flags);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefGetAddress(hipDeviceptr_t *pdptr, hipTexRef hTexRef);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipTexRefGetAddress(hipDeviceptr_t* dev_ptr, const textureReference* texRef);
  // CHECK: result = hipTexRefGetAddress(&deviceptr, texref);
  // CHECK-NEXT: result = hipTexRefGetAddress(&deviceptr, texref);
  result = hipTexRefGetAddress(&deviceptr, texref);
  result = hipTexRefGetAddress(&deviceptr, texref);

  // CHECK: HIPaddress_mode address_mode;
  HIPaddress_mode address_mode;

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefGetAddressMode(HIPaddress_mode *pam, hipTexRef hTexRef, int dim);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipTexRefGetAddressMode(enum hipTextureAddressMode* pam, const textureReference* texRef, int dim);
  // CHECK: result = hipTexRefGetAddressMode(&address_mode, texref, dim);
  result = hipTexRefGetAddressMode(&address_mode, texref, dim);

  // CHECK: HIPfilter_mode filter_mode;
  HIPfilter_mode filter_mode;

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefGetFilterMode(HIPfilter_mode *pfm, hipTexRef hTexRef);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipTexRefGetFilterMode(enum hipTextureFilterMode* pfm, const textureReference* texRef);
  // CHECK: result = hipTexRefGetFilterMode(&filter_mode, texref);
  result = hipTexRefGetFilterMode(&filter_mode, texref);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefGetFlags(unsigned int *pFlags, hipTexRef hTexRef);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipTexRefGetFlags(unsigned int* pFlags, const textureReference* texRef);
  // CHECK: result = hipTexRefGetFlags(&flags, texref);
  result = hipTexRefGetFlags(&flags, texref);

  // CHECK: hipArray_Format array_format;
  hipArray_Format array_format;

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefGetFormat(hipArray_Format *pFormat, int *pNumChannels, hipTexRef hTexRef);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipTexRefGetFormat(hipArray_Format* pFormat, int* pNumChannels, const textureReference* texRef);
  // CHECK: result = hipTexRefGetFormat(&array_format, &iBlockSize, texref);
  result = hipTexRefGetFormat(&array_format, &iBlockSize, texref);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefGetMaxAnisotropy(int *pmaxAniso, hipTexRef hTexRef);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipTexRefGetMaxAnisotropy(int* pmaxAnsio, const textureReference* texRef);
  // CHECK: result = hipTexRefGetMaxAnisotropy(&iBlockSize, texref);
  result = hipTexRefGetMaxAnisotropy(&iBlockSize, texref);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefGetMipmapFilterMode(HIPfilter_mode *pfm, hipTexRef hTexRef);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipTexRefGetMipmapFilterMode(enum hipTextureFilterMode* pfm, const textureReference* texRef);
  // CHECK: result = hipTexRefGetMipmapFilterMode(&filter_mode, texref);
  result = hipTexRefGetMipmapFilterMode(&filter_mode, texref);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefGetMipmapLevelBias(float *pbias, hipTexRef hTexRef);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipTexRefGetMipmapLevelBias(float* pbias, const textureReference* texRef);
  // CHECK: result = hipTexRefGetMipmapLevelBias(&ms, texref);
  result = hipTexRefGetMipmapLevelBias(&ms, texref);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefGetMipmapLevelClamp(float *pminMipmapLevelClamp, float *pmaxMipmapLevelClamp, hipTexRef hTexRef);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipTexRefGetMipmapLevelClamp(float* pminMipmapLevelClamp, float* pmaxMipmapLevelClamp, const textureReference* texRef);
  // CHECK: result = hipTexRefGetMipmapLevelClamp(&ms, &ms_2, texref);
  result = hipTexRefGetMipmapLevelClamp(&ms, &ms_2, texref);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefGetMipMappedArray(hipMipmappedArray_t *phMipmappedArray, hipTexRef hTexRef);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipTexRefGetMipMappedArray(hipMipmappedArray_t* pArray, const textureReference* texRef);
  // CHECK: result = hipTexRefGetMipMappedArray(&mipmappedArray, texref);
  result = hipTexRefGetMipMappedArray(&mipmappedArray, texref);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefSetAddress(size_t *ByteOffset, hipTexRef hTexRef, hipDeviceptr_t dptr, size_t bytes);
  // HIP: hipError_t hipTexRefSetAddress(size_t* ByteOffset, textureReference* texRef, hipDeviceptr_t dptr, size_t bytes);
  // CHECK: result = hipTexRefSetAddress(&bytes, texref, deviceptr, bytes_2);
  // CHECK-NEXT: result = hipTexRefSetAddress(&bytes, texref, deviceptr, bytes_2);
  result = hipTexRefSetAddress(&bytes, texref, deviceptr, bytes_2);
  result = hipTexRefSetAddress(&bytes, texref, deviceptr, bytes_2);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefSetAddress2D(hipTexRef hTexRef, const HIP_ARRAY_DESCRIPTOR *desc, hipDeviceptr_t dptr, size_t Pitch);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipTexRefSetAddress2D(textureReference* texRef, const HIP_ARRAY_DESCRIPTOR* desc, hipDeviceptr_t dptr, size_t Pitch);
  // CHECK: result = hipTexRefSetAddress2D(texref, &ARRAY_DESCRIPTOR, deviceptr, bytes);
  // CHECK-NEXT: result = hipTexRefSetAddress2D(texref, &ARRAY_DESCRIPTOR, deviceptr, bytes);
  result = hipTexRefSetAddress2D(texref, &ARRAY_DESCRIPTOR, deviceptr, bytes);
  result = hipTexRefSetAddress2D(texref, &ARRAY_DESCRIPTOR, deviceptr, bytes);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefSetAddressMode(hipTexRef hTexRef, int dim, HIPaddress_mode am);
  // HIP: hipError_t hipTexRefSetAddressMode(textureReference* texRef, int dim, enum hipTextureAddressMode am);
  // CHECK: result = hipTexRefSetAddressMode(texref, dim, address_mode);
  result = hipTexRefSetAddressMode(texref, dim, address_mode);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefSetArray(hipTexRef hTexRef, hipArray_t hArray, unsigned int Flags);
  // HIP: hipError_t hipTexRefSetArray(textureReference* tex, hipArray_const_t array, unsigned int flags);
  // CHECK: result = hipTexRefSetArray(texref, array_, flags);
  result = hipTexRefSetArray(texref, array_, flags);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefSetFilterMode(hipTexRef hTexRef, HIPfilter_mode fm);
  // HIP: hipError_t hipError_t hipTexRefSetFilterMode(textureReference* texRef, enum hipTextureFilterMode fm);
  // CHECK: result = hipTexRefSetFilterMode(texref, filter_mode);
  result = hipTexRefSetFilterMode(texref, filter_mode);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefSetFlags(hipTexRef hTexRef, unsigned int Flags);
  // HIP: hipError_t hipTexRefSetFlags(textureReference* texRef, unsigned int Flags);
  // CHECK: result = hipTexRefSetFlags(texref, flags);
  result = hipTexRefSetFlags(texref, flags);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefSetFormat(hipTexRef hTexRef, hipArray_Format fmt, int NumPackedComponents);
  // HIP: hipError_t hipTexRefSetFormat(textureReference* texRef, hipArray_Format fmt, int NumPackedComponents);
  // CHECK: result = hipTexRefSetFormat(texref, array_format, iBlockSize);
  result = hipTexRefSetFormat(texref, array_format, iBlockSize);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefSetMaxAnisotropy(hipTexRef hTexRef, unsigned int maxAniso);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipTexRefSetMaxAnisotropy(textureReference* texRef, unsigned int maxAniso);
  // CHECK: result = hipTexRefSetMaxAnisotropy(texref, flags);
  result = hipTexRefSetMaxAnisotropy(texref, flags);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefSetMipmapFilterMode(hipTexRef hTexRef, HIPfilter_mode fm);
  // HIP: hipError_t hipTexRefSetMipmapFilterMode(textureReference* texRef, enum hipTextureFilterMode fm);
  // CHECK: result = hipTexRefSetMipmapFilterMode(texref, filter_mode);
  result = hipTexRefSetMipmapFilterMode(texref, filter_mode);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefSetMipmapLevelBias(hipTexRef hTexRef, float bias);
  // HIP: hipError_t hipTexRefSetMipmapLevelBias(textureReference* texRef, float bias);
  // CHECK: result = hipTexRefSetMipmapLevelBias(texref, ms);
  result = hipTexRefSetMipmapLevelBias(texref, ms);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefSetMipmapLevelClamp(hipTexRef hTexRef, float minMipmapLevelClamp, float maxMipmapLevelClamp);
  // HIP: hipError_t hipTexRefSetMipmapLevelClamp(textureReference* texRef, float minMipMapLevelClamp, float maxMipMapLevelClamp);
  // CHECK: result = hipTexRefSetMipmapLevelClamp(texref, ms, ms_2);
  result = hipTexRefSetMipmapLevelClamp(texref, ms, ms_2);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefSetMipmappedArray(hipTexRef hTexRef, hipMipmappedArray_t hMipmappedArray, unsigned int Flags);
  // HIP: hipError_t hipTexRefSetMipmappedArray(textureReference* texRef, struct hipMipmappedArray* mipmappedArray, unsigned int Flags);
  // CHECK: result = hipTexRefSetMipmappedArray(texref, mipmappedArray, flags);
  result = hipTexRefSetMipmappedArray(texref, mipmappedArray, flags);

  // CHECK: hipTextureObject_t texObject;
  hipTextureObject_t texObject;

  // CHECK: HIP_RESOURCE_DESC res_descr;
  HIP_RESOURCE_DESC res_descr;

  // CHECK: HIP_TEXTURE_DESC tex_descr;
  HIP_TEXTURE_DESC tex_descr;

  // CHECK: HIP_RESOURCE_VIEW_DESC res_view_descr;
  HIP_RESOURCE_VIEW_DESC res_view_descr;

  // CUDA: hipError_t CUDAAPI hipTexObjectCreate(hipTextureObject_t *pTexObject, const HIP_RESOURCE_DESC *pResDesc, const HIP_TEXTURE_DESC *pTexDesc, const HIP_RESOURCE_VIEW_DESC *pResViewDesc);
  // HIP: hipError_t hipTexObjectCreate(hipTextureObject_t* pTexObject, const HIP_RESOURCE_DESC* pResDesc, const HIP_TEXTURE_DESC* pTexDesc, const HIP_RESOURCE_VIEW_DESC* pResViewDesc);
  // CHECK: result = hipTexObjectCreate(&texObject, &res_descr, &tex_descr, &res_view_descr);
  result = hipTexObjectCreate(&texObject, &res_descr, &tex_descr, &res_view_descr);

  // CUDA: hipError_t CUDAAPI hipTexObjectDestroy(hipTextureObject_t texObject);
  // HIP: hipError_t hipTexObjectDestroy(hipTextureObject_t texObject);
  // CHECK: result = hipTexObjectDestroy(texObject);
  result = hipTexObjectDestroy(texObject);

  // CUDA: hipError_t CUDAAPI hipTexObjectGetResourceDesc(HIP_RESOURCE_DESC *pResDesc, hipTextureObject_t texObject);
  // HIP: hipError_t hipTexObjectGetResourceDesc(HIP_RESOURCE_DESC* pResDesc, hipTextureObject_t texObject);
  // CHECK: result = hipTexObjectGetResourceDesc(&res_descr, texObject);
  result = hipTexObjectGetResourceDesc(&res_descr, texObject);

  // CUDA: hipError_t CUDAAPI hipTexObjectGetResourceViewDesc(HIP_RESOURCE_VIEW_DESC *pResViewDesc, hipTextureObject_t texObject);
  // HIP: hipError_t hipTexObjectGetResourceViewDesc(HIP_RESOURCE_VIEW_DESC* pResViewDesc, hipTextureObject_t texObject);
  // CHECK: result = hipTexObjectGetResourceViewDesc(&res_view_descr, texObject);
  result = hipTexObjectGetResourceViewDesc(&res_view_descr, texObject);

  // CUDA: hipError_t CUDAAPI hipTexObjectGetTextureDesc(HIP_TEXTURE_DESC *pTexDesc, hipTextureObject_t texObject);
  // HIP: hipError_t hipTexObjectGetTextureDesc(HIP_TEXTURE_DESC* pTexDesc, hipTextureObject_t texObject);
  // CHECK: result = hipTexObjectGetTextureDesc(&tex_descr, texObject);
  result = hipTexObjectGetTextureDesc(&tex_descr, texObject);

  // CUDA: hipError_t CUDAAPI hipCtxEnablePeerAccess(hipCtx_t peerContext, unsigned int Flags);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxEnablePeerAccess(hipCtx_t peerCtx, unsigned int flags);
  // CHECK: result = hipCtxEnablePeerAccess(context, flags);
  result = hipCtxEnablePeerAccess(context, flags);

  // CUDA: hipError_t CUDAAPI hipCtxDisablePeerAccess(hipCtx_t peerContext);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxDisablePeerAccess(hipCtx_t peerCtx);
  // CHECK: result = hipCtxDisablePeerAccess(context);
  result = hipCtxDisablePeerAccess(context);

  // CUDA: hipError_t CUDAAPI hipDeviceCanAccessPeer(int *canAccessPeer, hipDevice_t dev, hipDevice_t peerDev);
  // HIP: hipError_t hipDeviceCanAccessPeer(int* canAccessPeer, int deviceId, int peerDeviceId);
  // CHECK: result = hipDeviceCanAccessPeer(value, device, dim);
  result = hipDeviceCanAccessPeer(value, device, dim);

  // CUDA: hipError_t CUDAAPI hipGraphicsMapResources(unsigned int count, hipGraphicsResource_t *resources, hipStream_t hStream);
  // HIP: hipError_t hipGraphicsMapResources(int count, hipGraphicsResource_t* resources, hipStream_t stream __dparm(0));
  // CHECK: result = hipGraphicsMapResources(iBlockSize, &graphicsResource, stream);
  result = hipGraphicsMapResources(iBlockSize, &graphicsResource, stream);

  // CUDA: hipError_t CUDAAPI hipGraphicsResourceGetMappedPointer(hipDeviceptr_t *pDevPtr, size_t *pSize, hipGraphicsResource_t resource);
  // HIP: hipError_t hipGraphicsResourceGetMappedPointer(void** devPtr, size_t* size, hipGraphicsResource_t resource);
  // CHECK: result = hipGraphicsResourceGetMappedPointer(&deviceptr, &bytes, graphicsResource);
  // CHECK-NEXT: result = hipGraphicsResourceGetMappedPointer(&deviceptr, &bytes, graphicsResource);
  result = hipGraphicsResourceGetMappedPointer(&deviceptr, &bytes, graphicsResource);
  result = hipGraphicsResourceGetMappedPointer(&deviceptr, &bytes, graphicsResource);

  // CUDA: hipError_t CUDAAPI hipGraphicsSubResourceGetMappedArray(hipArray_t *pArray, hipGraphicsResource_t resource, unsigned int arrayIndex, unsigned int mipLevel);
  // HIP: hipError_t hipGraphicsSubResourceGetMappedArray(hipArray_t* array, hipGraphicsResource_t resource, unsigned int arrayIndex, unsigned int mipLevel);
  // CHECK: result = hipGraphicsSubResourceGetMappedArray(&array_, graphicsResource, flags, flags_2);
  result = hipGraphicsSubResourceGetMappedArray(&array_, graphicsResource, flags, flags_2);

  // CUDA: hipError_t CUDAAPI hipGraphicsUnmapResources(unsigned int count, hipGraphicsResource_t *resources, hipStream_t hStream);
  // HIP: hipError_t hipGraphicsUnmapResources(int count, hipGraphicsResource_t* resources, hipStream_t stream __dparm(0));
  // CHECK: result = hipGraphicsUnmapResources(iBlockSize, &graphicsResource, stream);
  result = hipGraphicsUnmapResources(iBlockSize, &graphicsResource, stream);

  // CUDA: hipError_t CUDAAPI hipGraphicsUnregisterResource(hipGraphicsResource_t resource);
  // HIP: hipError_t hipGraphicsUnregisterResource(hipGraphicsResource_t resource);
  // CHECK: result = hipGraphicsUnregisterResource(graphicsResource);
  result = hipGraphicsUnregisterResource(graphicsResource);

  // CUDA: hipError_t CUDAAPI hipProfilerStart(void);
  // HIP: DEPRECATED("use roctracer/rocTX instead") hipError_t hipProfilerStart();
  // CHECK: result = hipProfilerStart();
  result = hipProfilerStart();

  // CUDA: hipError_t CUDAAPI hipProfilerStop(void);
  // HIP: DEPRECATED("use roctracer/rocTX instead") hipError_t hipProfilerStop();
  // CHECK: result = hipProfilerStop();
  result = hipProfilerStop();

  // CHECK: hipGLDeviceList GLDeviceList;
  hipGLDeviceList GLDeviceList;

  // CUDA: hipError_t CUDAAPI hipGLGetDevices(unsigned int *pCudaDeviceCount, hipDevice_t *pCudaDevices, unsigned int cudaDeviceCount, hipGLDeviceList deviceList);
  // HIP: hipError_t hipGLGetDevices(unsigned int* pHipDeviceCount, int* pHipDevices, unsigned int hipDeviceCount, hipGLDeviceList deviceList);
  // CHECK: result = hipGLGetDevices(&flags, &device, flags_2, GLDeviceList);
  result = hipGLGetDevices(&flags, &device, flags_2, GLDeviceList);

  // CUDA: hipError_t CUDAAPI hipGraphicsGLRegisterBuffer(hipGraphicsResource_t *pCudaResource, GLuint buffer, unsigned int Flags);
  // HIP: hipError_t hipGraphicsGLRegisterBuffer(hipGraphicsResource** resource, GLuint buffer, unsigned int flags);
  // CHECK: result = hipGraphicsGLRegisterBuffer(&graphicsResource, gl_uint, flags);
  result = hipGraphicsGLRegisterBuffer(&graphicsResource, gl_uint, flags);

  // CUDA: hipError_t CUDAAPI hipGraphicsGLRegisterImage(hipGraphicsResource_t *pCudaResource, GLuint image, GLenum target, unsigned int Flags);
  // HIP: hipError_t hipGraphicsGLRegisterImage(hipGraphicsResource** resource, GLuint image, GLenum target, unsigned int flags);
  // CHECK: result = hipGraphicsGLRegisterImage(&graphicsResource, gl_uint, gl_enum, flags);
  result = hipGraphicsGLRegisterImage(&graphicsResource, gl_uint, gl_enum, flags);

  // CUDA: hipError_t CUDAAPI hipDeviceSetLimit(hipLimit_t limit, size_t value);
  // HIP: hipError_t hipDeviceSetLimit(enum hipLimit_t limit, size_t value);
  // CHECK: result = hipDeviceSetLimit(limit, bytes);
  result = hipDeviceSetLimit(limit, bytes);

  // CHECK: hipError_t result_2;
  hipError_t result_2;
  const char* ret = NULL;

  // CUDA: hipError_t CUDAAPI hipDrvGetErrorName(hipError_t error, const char **pStr);
  // HIP: hipError_t hipDrvGetErrorName(hipError_t hipError, const char** errorString);
  // CHECK: result = hipDrvGetErrorName(result_2, &ret);
  result = hipDrvGetErrorName(result_2, &ret);

  // CUDA: hipError_t CUDAAPI hipDrvGetErrorString(hipError_t error, const char **pStr);
  // HIP: hipError_t hipDrvGetErrorString(hipError_t hipError, const char** errorString);
  // CHECK: result = hipDrvGetErrorString(result_2, &ret);
  result = hipDrvGetErrorString(result_2, &ret);

  // CUDA: hipError_t CUDAAPI hipPointerSetAttribute(const void *value, hipPointer_attribute attribute, hipDeviceptr_t ptr);
  // HIP: hipError_t hipPointerSetAttribute(const void* value, hipPointer_attribute attribute, hipDeviceptr_t ptr);
  // CHECK: result = hipPointerSetAttribute(image, pointer_attribute, deviceptr);
  result = hipPointerSetAttribute(image, pointer_attribute, deviceptr);

#if CUDA_VERSION >= 8000
  // CHECK: hipMemRangeAttribute MemoryRangeAttribute;
  // CHECK-NEXT: hipMemoryAdvise MemoryAdvise;
  hipMemRangeAttribute MemoryRangeAttribute;
  hipMemoryAdvise MemoryAdvise;

  // CUDA: hipError_t CUDAAPI hipMemAdvise(hipDeviceptr_t devPtr, size_t count, hipMemoryAdvise advice, hipDevice_t device);
  // HIP: hipError_t hipMemAdvise(const void* dev_ptr, size_t count, hipMemoryAdvise advice, int device);
  // CHECK: result = hipMemAdvise(deviceptr, bytes, MemoryAdvise, device);
  result = hipMemAdvise(deviceptr, bytes, MemoryAdvise, device);

  // CUDA: hipError_t CUDAAPI hipMemPrefetchAsync(hipDeviceptr_t devPtr, size_t count, hipDevice_t dstDevice, hipStream_t hStream);
  // HIP: hipError_t hipMemPrefetchAsync(const void* dev_ptr, size_t count, int device, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemPrefetchAsync(deviceptr, bytes, device, stream);
  result = hipMemPrefetchAsync(deviceptr, bytes, device, stream);

  // CUDA: hipError_t CUDAAPI hipMemRangeGetAttribute(void *data, size_t dataSize, hipMemRangeAttribute attribute, hipDeviceptr_t devPtr, size_t count);
  // HIP: hipError_t hipMemRangeGetAttribute(void* data, size_t data_size, hipMemRangeAttribute attribute, const void* dev_ptr, size_t count);
  // CHECK: result = hipMemRangeGetAttribute(image, bytes, MemoryRangeAttribute, deviceptr, bytes);
  result = hipMemRangeGetAttribute(image, bytes, MemoryRangeAttribute, deviceptr, bytes);

  // CUDA: hipError_t CUDAAPI hipMemRangeGetAttributes(void **data, size_t *dataSizes, hipMemRangeAttribute *attributes, size_t numAttributes, hipDeviceptr_t devPtr, size_t count);
  // HIP: hipError_t hipMemRangeGetAttributes(void** data, size_t* data_sizes, hipMemRangeAttribute* attributes, size_t num_attributes, const void* dev_ptr, size_t count);
  // CHECK: result = hipMemRangeGetAttributes(&image, &bytes, &MemoryRangeAttribute, bytes, deviceptr, bytes);
  result = hipMemRangeGetAttributes(&image, &bytes, &MemoryRangeAttribute, bytes, deviceptr, bytes);

  // CUDA: hipError_t CUDAAPI hipStreamWaitValue32(hipStream_t stream, hipDeviceptr_t addr, cuuint32_t value, unsigned int flags);
  // HIP: hipError_t hipStreamWaitValue32(hipStream_t stream, void* ptr, uint32_t value, unsigned int flags, uint32_t mask __dparm(0xFFFFFFFF));
  // CHECK: result = hipStreamWaitValue32(stream, deviceptr, u_value, flags);
  result = hipStreamWaitValue32(stream, deviceptr, u_value, flags);

  // CUDA: hipError_t CUDAAPI hipStreamWriteValue32(hipStream_t stream, hipDeviceptr_t addr, cuuint32_t value, unsigned int flags);
  // HIP: hipError_t hipStreamWriteValue32(hipStream_t stream, void* ptr, uint32_t value, unsigned int flags, uint32_t mask __dparm(0xFFFFFFFF));
  // CHECK: result = hipStreamWriteValue32(stream, deviceptr, u_value, flags);
  result = hipStreamWriteValue32(stream, deviceptr, u_value, flags);

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipTexRefSetBorderColor(hipTexRef hTexRef, float *pBorderColor);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipTexRefSetBorderColor(textureReference* texRef, float* pBorderColor);
  // CHECK: result = hipTexRefSetBorderColor(texref, &ms);
  result = hipTexRefSetBorderColor(texref, &ms);

  // CHECK: hipDeviceP2PAttr deviceP2PAttribute;
  hipDeviceP2PAttr deviceP2PAttribute;

  // CUDA: hipError_t CUDAAPI hipDeviceGetP2PAttribute(int* value, hipDeviceP2PAttr attrib, hipDevice_t srcDevice, hipDevice_t dstDevice);
  // HIP: hipError_t hipDeviceGetP2PAttribute(int* value, hipDeviceP2PAttr attr, int srcDevice, int dstDevice);
  // CHECK: result = hipDeviceGetP2PAttribute(value, deviceP2PAttribute, iBlockSize, iBlockSize_2);
  result = hipDeviceGetP2PAttribute(value, deviceP2PAttribute, iBlockSize, iBlockSize_2);
#endif

#if CUDA_VERSION >= 9000
  // CUDA: hipError_t CUDAAPI hipStreamWaitValue64(hipStream_t stream, hipDeviceptr_t addr, cuuint64_t value, unsigned int flags);
  // HIP: hipError_t hipStreamWaitValue64(hipStream_t stream, void* ptr, uint64_t value, unsigned int flags, uint64_t mask __dparm(0xFFFFFFFFFFFFFFFF));
  // CHECK: result = hipStreamWaitValue64(stream, deviceptr, u_value, flags);
  result = hipStreamWaitValue64(stream, deviceptr, u_value, flags);

  // CUDA: hipError_t CUDAAPI hipStreamWriteValue64(hipStream_t stream, hipDeviceptr_t addr, cuuint64_t value, unsigned int flags);
  // HIP: hipError_t hipStreamWriteValue64(hipStream_t stream, void* ptr, uint64_t value, unsigned int flags, uint64_t mask __dparm(0xFFFFFFFFFFFFFFFF));
  // CHECK: result = hipStreamWriteValue64(stream, deviceptr, u_value, flags);
  result = hipStreamWriteValue64(stream, deviceptr, u_value, flags);

  // CUDA: hipError_t CUDAAPI hipModuleLaunchCooperativeKernel(hipFunction_t f, unsigned int gridDimX, unsigned int gridDimY, unsigned int gridDimZ, unsigned int blockDimX, unsigned int blockDimY, unsigned int blockDimZ, unsigned int sharedMemBytes, hipStream_t hStream, void** kernelParams);
  // HIP: hipError_t hipModuleLaunchCooperativeKernel(hipFunction_t f, unsigned int gridDimX, unsigned int gridDimY, unsigned int gridDimZ, unsigned int blockDimX, unsigned int blockDimY, unsigned int blockDimZ, unsigned int sharedMemBytes, hipStream_t stream, void** kernelParams);
  // CHECK: result = hipModuleLaunchCooperativeKernel(function, gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, sharedMemBytes, stream, &kernelParams);
  result = hipModuleLaunchCooperativeKernel(function, gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, sharedMemBytes, stream, &kernelParams);

  // CHECK: hipFunctionLaunchParams_t LAUNCH_PARAMS_st;
  // CHECK-NEXT: hipFunctionLaunchParams LAUNCH_PARAMS;
  hipFunctionLaunchParams_t LAUNCH_PARAMS_st;
  hipFunctionLaunchParams LAUNCH_PARAMS;

  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipModuleLaunchCooperativeKernelMultiDevice(hipFunctionLaunchParams *launchParamsList, unsigned int numDevices, unsigned int flags);
  // HIP: hipError_t hipModuleLaunchCooperativeKernelMultiDevice(hipFunctionLaunchParams* launchParamsList, unsigned int numDevices, unsigned int flags);
  // CHECK: result = hipModuleLaunchCooperativeKernelMultiDevice(&LAUNCH_PARAMS, flags_2, flags);
  result = hipModuleLaunchCooperativeKernelMultiDevice(&LAUNCH_PARAMS, flags_2, flags);
#endif

#if CUDA_VERSION >= 10000
  // CHECK: hipGraph_t graph, graph2;
  // CHECK-NEXT: hipGraphNode_t graphNode, graphNode2;
  // CHECK-NEXT: const hipGraphNode_t *pGraphNode = nullptr;
  // CHECK-NEXT: hipKernelNodeParams KERNEL_NODE_PARAMS;
  // CHECK-NEXT: hipMemsetParams MEMSET_NODE_PARAMS;
  // CHECK-NEXT: hipGraphExec_t graphExec;
  // CHECK-NEXT: hipExternalMemory_t externalMemory;
  // CHECK-NEXT: hipExternalSemaphore_t externalSemaphore;
  // CHECK-NEXT: hipExternalMemoryBufferDesc EXTERNAL_MEMORY_BUFFER_DESC;
  // CHECK-NEXT: hipExternalMemoryHandleDesc EXTERNAL_MEMORY_HANDLE_DESC;
  // CHECK-NEXT: hipExternalSemaphoreHandleDesc EXTERNAL_SEMAPHORE_HANDLE_DESC;
  // CHECK-NEXT: hipExternalSemaphoreSignalParams EXTERNAL_SEMAPHORE_SIGNAL_PARAMS;
  // CHECK-NEXT: hipExternalSemaphoreWaitParams EXTERNAL_SEMAPHORE_WAIT_PARAMS;
  // CHECK-NEXT: hipStreamCaptureStatus streamCaptureStatus;
  // CHECK-NEXT: hipGraphNodeType graphNodeType;
  // CHECK-NEXT: hipHostNodeParams host_node_params;
  hipGraph_t graph, graph2;
  hipGraphNode_t graphNode, graphNode2;
  const hipGraphNode_t *pGraphNode = nullptr;
  hipKernelNodeParams KERNEL_NODE_PARAMS;
  hipMemsetParams MEMSET_NODE_PARAMS;
  hipGraphExec_t graphExec;
  hipExternalMemory_t externalMemory;
  hipExternalSemaphore_t externalSemaphore;
  hipExternalMemoryBufferDesc EXTERNAL_MEMORY_BUFFER_DESC;
  hipExternalMemoryHandleDesc EXTERNAL_MEMORY_HANDLE_DESC;
  hipExternalSemaphoreHandleDesc EXTERNAL_SEMAPHORE_HANDLE_DESC;
  hipExternalSemaphoreSignalParams EXTERNAL_SEMAPHORE_SIGNAL_PARAMS;
  hipExternalSemaphoreWaitParams EXTERNAL_SEMAPHORE_WAIT_PARAMS;
  hipStreamCaptureStatus streamCaptureStatus;
  hipGraphNodeType graphNodeType;
  hipHostNodeParams host_node_params;

  // CUDA: hipError_t CUDAAPI hipDeviceGetUuid(hipUUID *uuid, hipDevice_t dev);
  // HIP: hipError_t hipDeviceGetUuid(hipUUID* uuid, hipDevice_t device);
  // CHECK: result = hipDeviceGetUuid(&uuid, device);
  result = hipDeviceGetUuid(&uuid, device);

  // CHECK: hipHostFn_t hostFn;
  hipHostFn_t hostFn;

  // CUDA: hipError_t CUDAAPI hipLaunchHostFunc(hipStream_t hStream, hipHostFn_t fn, void *userData);
  // HIP: hipError_t hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void* userData);
  // CHECK: result = hipLaunchHostFunc(stream, hostFn, image);
  result = hipLaunchHostFunc(stream, hostFn, image);

  // CUDA: hipError_t CUDAAPI hipStreamEndCapture(hipStream_t hStream, hipGraph_t *phGraph);
  // HIP: hipError_t hipStreamEndCapture(hipStream_t stream, hipGraph_t* pGraph);
  // CHECK: result = hipStreamEndCapture(stream, &graph);
  result = hipStreamEndCapture(stream, &graph);

  // CUDA: hipError_t CUDAAPI hipDestroyExternalMemory(hipExternalMemory_t extMem);
  // HIP: hipError_t hipDestroyExternalMemory(hipExternalMemory_t extMem);
  // CHECK: result = hipDestroyExternalMemory(externalMemory);
  result = hipDestroyExternalMemory(externalMemory);

  // CUDA: hipError_t CUDAAPI hipDestroyExternalSemaphore(hipExternalSemaphore_t extSem);
  // HIP: hipError_t hipDestroyExternalSemaphore(hipExternalSemaphore_t extSem);
  // CHECK: result = hipDestroyExternalSemaphore(externalSemaphore);
  result = hipDestroyExternalSemaphore(externalSemaphore);

  // CUDA: hipError_t CUDAAPI hipExternalMemoryGetMappedBuffer(hipDeviceptr_t *devPtr, hipExternalMemory_t extMem, const hipExternalMemoryBufferDesc *bufferDesc);
  // HIP: hipError_t hipExternalMemoryGetMappedBuffer(void **devPtr, hipExternalMemory_t extMem, const hipExternalMemoryBufferDesc *bufferDesc);
  // CHECK: result = hipExternalMemoryGetMappedBuffer(&deviceptr, externalMemory, &EXTERNAL_MEMORY_BUFFER_DESC);
  result = hipExternalMemoryGetMappedBuffer(&deviceptr, externalMemory, &EXTERNAL_MEMORY_BUFFER_DESC);

  // CUDA: hipError_t CUDAAPI hipImportExternalMemory(hipExternalMemory_t *extMem_out, const hipExternalMemoryHandleDesc *memHandleDesc);
  // HIP: hipError_t hipImportExternalMemory(hipExternalMemory_t* extMem_out, const hipExternalMemoryHandleDesc* memHandleDesc);
  // CHECK: result = hipImportExternalMemory(&externalMemory, &EXTERNAL_MEMORY_HANDLE_DESC);
  result = hipImportExternalMemory(&externalMemory, &EXTERNAL_MEMORY_HANDLE_DESC);

  // CUDA: hipError_t CUDAAPI hipImportExternalSemaphore(hipExternalSemaphore_t *extSem_out, const hipExternalSemaphoreHandleDesc *semHandleDesc);
  // HIP: hipError_t hipImportExternalSemaphore(hipExternalSemaphore_t* extSem_out, const hipExternalSemaphoreHandleDesc* semHandleDesc);
  // CHECK: result = hipImportExternalSemaphore(&externalSemaphore, &EXTERNAL_SEMAPHORE_HANDLE_DESC);
  result = hipImportExternalSemaphore(&externalSemaphore, &EXTERNAL_SEMAPHORE_HANDLE_DESC);

  // CUDA: hipError_t CUDAAPI hipSignalExternalSemaphoresAsync(const hipExternalSemaphore_t *extSemArray, const hipExternalSemaphoreSignalParams *paramsArray, unsigned int numExtSems, hipStream_t stream);
  // HIP: hipError_t hipSignalExternalSemaphoresAsync(const hipExternalSemaphore_t* extSemArray, const hipExternalSemaphoreSignalParams* paramsArray, unsigned int numExtSems, hipStream_t stream);
  // CHECK: result = hipSignalExternalSemaphoresAsync(&externalSemaphore, &EXTERNAL_SEMAPHORE_SIGNAL_PARAMS, flags, stream);
  result = hipSignalExternalSemaphoresAsync(&externalSemaphore, &EXTERNAL_SEMAPHORE_SIGNAL_PARAMS, flags, stream);

  // CUDA: hipError_t CUDAAPI hipWaitExternalSemaphoresAsync(const hipExternalSemaphore_t *extSemArray, const hipExternalSemaphoreWaitParams *paramsArray, unsigned int numExtSems, hipStream_t stream);
  // HIP: hipError_t hipWaitExternalSemaphoresAsync(const hipExternalSemaphore_t* extSemArray, const hipExternalSemaphoreWaitParams* paramsArray, unsigned int numExtSems, hipStream_t stream);
  // CHECK: result = hipWaitExternalSemaphoresAsync(&externalSemaphore, &EXTERNAL_SEMAPHORE_WAIT_PARAMS, flags, stream);
  result = hipWaitExternalSemaphoresAsync(&externalSemaphore, &EXTERNAL_SEMAPHORE_WAIT_PARAMS, flags, stream);

  // CUDA: hipError_t CUDAAPI hipGraphAddDependencies(hipGraph_t hGraph, const hipGraphNode_t *from, const hipGraphNode_t *to, size_t numDependencies);
  // HIP: hipError_t hipGraphAddDependencies(hipGraph_t graph, const hipGraphNode_t* from, const hipGraphNode_t* to, size_t numDependencies);
  // CHECK: result = hipGraphAddDependencies(graph, &graphNode, &graphNode2, bytes);
  result = hipGraphAddDependencies(graph, &graphNode, &graphNode2, bytes);

  // CUDA: hipError_t CUDAAPI hipGraphAddEmptyNode(hipGraphNode_t *phGraphNode, hipGraph_t hGraph, const hipGraphNode_t *dependencies, size_t numDependencies);
  // HIP: hipError_t hipGraphAddEmptyNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies);
  // CHECK: result = hipGraphAddEmptyNode(&graphNode, graph, &graphNode2, bytes);
  result = hipGraphAddEmptyNode(&graphNode, graph, &graphNode2, bytes);

  // CUDA: hipError_t CUDAAPI hipGraphAddKernelNode(hipGraphNode_t *phGraphNode, hipGraph_t hGraph, const hipGraphNode_t *dependencies, size_t numDependencies, const hipKernelNodeParams *nodeParams);
  // HIP: hipError_t hipGraphAddKernelNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, const hipKernelNodeParams* pNodeParams);
  // CHECK: result = hipGraphAddKernelNode(&graphNode, graph, &graphNode2, bytes, &KERNEL_NODE_PARAMS);
  result = hipGraphAddKernelNode(&graphNode, graph, &graphNode2, bytes, &KERNEL_NODE_PARAMS);

  // CUDA: hipError_t CUDAAPI hipGraphCreate(hipGraph_t *phGraph, unsigned int flags);
  // HIP: hipError_t hipGraphCreate(hipGraph_t* pGraph, unsigned int flags);
  // CHECK: result = hipGraphCreate(&graph, flags);
  result = hipGraphCreate(&graph, flags);

  // CUDA: hipError_t CUDAAPI hipGraphDestroy(hipGraph_t hGraph);
  // HIP: hipError_t hipGraphDestroy(hipGraph_t graph);
  // CHECK: result = hipGraphDestroy(graph);
  result = hipGraphDestroy(graph);

  // CUDA: hipError_t CUDAAPI hipGraphExecDestroy(hipGraphExec_t hGraphExec);
  // HIP: hipError_t hipGraphExecDestroy(hipGraphExec_t pGraphExec);
  // CHECK: result = hipGraphExecDestroy(graphExec);
  result = hipGraphExecDestroy(graphExec);

  // CUDA: hipError_t CUDAAPI hipGraphGetNodes(hipGraph_t hGraph, hipGraphNode_t *nodes, size_t *numNodes);
  // HIP: hipError_t hipGraphGetNodes(hipGraph_t graph, hipGraphNode_t* nodes, size_t* numNodes);
  // CHECK: result = hipGraphGetNodes(graph, &graphNode, &bytes);
  result = hipGraphGetNodes(graph, &graphNode, &bytes);

  // CUDA: hipError_t CUDAAPI hipGraphGetRootNodes(hipGraph_t hGraph, hipGraphNode_t *rootNodes, size_t *numRootNodes);
  // HIP: hipError_t hipGraphGetRootNodes(hipGraph_t graph, hipGraphNode_t* pRootNodes, size_t* pNumRootNodes);
  // CHECK: result = hipGraphGetRootNodes(graph, &graphNode, &bytes);
  result = hipGraphGetRootNodes(graph, &graphNode, &bytes);

  // CUDA: hipError_t CUDAAPI hipGraphKernelNodeGetParams(hipGraphNode_t hNode, hipKernelNodeParams *nodeParams);
  // HIP: hipError_t hipGraphKernelNodeGetParams(hipGraphNode_t node, hipKernelNodeParams* pNodeParams);
  // CHECK: result = hipGraphKernelNodeGetParams(graphNode, &KERNEL_NODE_PARAMS);
  result = hipGraphKernelNodeGetParams(graphNode, &KERNEL_NODE_PARAMS);

  // CUDA: hipError_t CUDAAPI hipGraphKernelNodeSetParams(hipGraphNode_t hNode, const hipKernelNodeParams *nodeParams);
  // HIP: hipError_t hipGraphKernelNodeSetParams(hipGraphNode_t node, const hipKernelNodeParams* pNodeParams);
  // CHECK: result = hipGraphKernelNodeSetParams(graphNode, &KERNEL_NODE_PARAMS);
  result = hipGraphKernelNodeSetParams(graphNode, &KERNEL_NODE_PARAMS);

  // CUDA: hipError_t CUDAAPI hipGraphLaunch(hipGraphExec_t hGraphExec, hipStream_t hStream);
  // HIP: hipError_t hipGraphLaunch(hipGraphExec_t graphExec, hipStream_t stream);
  // CHECK: result = hipGraphLaunch(graphExec, stream);
  result = hipGraphLaunch(graphExec, stream);

  // CUDA: hipError_t CUDAAPI hipDrvGraphMemcpyNodeGetParams(hipGraphNode_t hNode, HIP_MEMCPY3D *nodeParams);
  // HIP: hipError_t hipDrvGraphMemcpyNodeGetParams(hipGraphNode_t node, hipMemcpy3DParms* pNodeParams);
  // CHECK: result = hipDrvGraphMemcpyNodeGetParams(graphNode, &MEMCPY3D);
  result = hipDrvGraphMemcpyNodeGetParams(graphNode, &MEMCPY3D);

  // CUDA: hipError_t CUDAAPI hipDrvGraphMemcpyNodeSetParams(hipGraphNode_t hNode, const HIP_MEMCPY3D *nodeParams);
  // HIP: hipError_t hipDrvGraphMemcpyNodeSetParams(hipGraphNode_t node, const hipMemcpy3DParms* pNodeParams);
  // CHECK: result = hipDrvGraphMemcpyNodeSetParams(graphNode, &MEMCPY3D);
  result = hipDrvGraphMemcpyNodeSetParams(graphNode, &MEMCPY3D);

  // CUDA: hipError_t CUDAAPI hipGraphMemsetNodeGetParams(hipGraphNode_t hNode, hipMemsetParams *nodeParams);
  // HIP: hipError_t hipGraphMemsetNodeGetParams(hipGraphNode_t node, hipMemsetParams* pNodeParams);
  // CHECK: result = hipGraphMemsetNodeGetParams(graphNode, &MEMSET_NODE_PARAMS);
  result = hipGraphMemsetNodeGetParams(graphNode, &MEMSET_NODE_PARAMS);

  // CUDA: hipError_t CUDAAPI hipGraphMemsetNodeSetParams(hipGraphNode_t hNode, const hipMemsetParams *nodeParams);
  // HIP: hipError_t hipGraphMemsetNodeSetParams(hipGraphNode_t node, const hipMemsetParams* pNodeParams);
  // CHECK: result = hipGraphMemsetNodeSetParams(graphNode, &MEMSET_NODE_PARAMS);
  result = hipGraphMemsetNodeSetParams(graphNode, &MEMSET_NODE_PARAMS);

  // CUDA: hipError_t CUDAAPI hipGraphGetEdges(hipGraph_t hGraph, hipGraphNode_t *from, hipGraphNode_t *to, size_t *numEdges);
  // HIP: hipError_t hipGraphGetEdges(hipGraph_t graph, hipGraphNode_t* from, hipGraphNode_t* to, size_t* numEdges);
  // CHECK: result = hipGraphGetEdges(graph, &graphNode, &graphNode2, &bytes);
  result = hipGraphGetEdges(graph, &graphNode, &graphNode2, &bytes);

  // CUDA: hipError_t CUDAAPI hipGraphNodeGetDependencies(hipGraphNode_t hNode, hipGraphNode_t *dependencies, size_t *numDependencies);
  // HIP: hipError_t hipGraphNodeGetDependencies(hipGraphNode_t node, hipGraphNode_t* pDependencies, size_t* pNumDependencies);
  // CHECK: result = hipGraphNodeGetDependencies(graphNode, &graphNode2, &bytes);
  result = hipGraphNodeGetDependencies(graphNode, &graphNode2, &bytes);

  // CUDA: hipError_t CUDAAPI hipGraphRemoveDependencies(hipGraph_t hGraph, const hipGraphNode_t *from, const hipGraphNode_t *to, size_t numDependencies);
  // HIP: hipError_t hipGraphRemoveDependencies(hipGraph_t graph, const hipGraphNode_t* from, const hipGraphNode_t* to, size_t numDependencies);
  // CHECK: result = hipGraphRemoveDependencies(graph, &graphNode, &graphNode2, bytes);
  result = hipGraphRemoveDependencies(graph, &graphNode, &graphNode2, bytes);

  // CUDA: hipError_t CUDAAPI hipGraphNodeGetDependentNodes(hipGraphNode_t hNode, hipGraphNode_t *dependentNodes, size_t *numDependentNodes);
  // HIP: hipError_t hipGraphNodeGetDependentNodes(hipGraphNode_t node, hipGraphNode_t* pDependentNodes, size_t* pNumDependentNodes);
  // CHECK: result = hipGraphNodeGetDependentNodes(graphNode, &graphNode2, &bytes);
  result = hipGraphNodeGetDependentNodes(graphNode, &graphNode2, &bytes);

  // CUDA: hipError_t CUDAAPI hipGraphNodeGetType(hipGraphNode_t hNode, hipGraphNodeType *type);
  // HIP: hipError_t hipGraphNodeGetType(hipGraphNode_t node, hipGraphNodeType* pType);
  // CHECK: result = hipGraphNodeGetType(graphNode, &graphNodeType);
  result = hipGraphNodeGetType(graphNode, &graphNodeType);

  // CUDA: hipError_t CUDAAPI hipGraphDestroyNode(hipGraphNode_t hNode);
  // HIP: hipError_t hipGraphDestroyNode(hipGraphNode_t node);
  // CHECK: result = hipGraphDestroyNode(graphNode);
  result = hipGraphDestroyNode(graphNode);

  // CUDA: hipError_t CUDAAPI hipGraphAddHostNode(hipGraphNode_t *phGraphNode, hipGraph_t hGraph, const hipGraphNode_t *dependencies, size_t numDependencies, const hipHostNodeParams *nodeParams);
  // HIP: hipError_t hipGraphAddHostNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, const hipHostNodeParams* pNodeParams);
  // CHECK: result = hipGraphAddHostNode(&graphNode, graph, &graphNode2, bytes, &host_node_params);
  result = hipGraphAddHostNode(&graphNode, graph, &graphNode2, bytes, &host_node_params);

  // CUDA: hipError_t CUDAAPI hipGraphNodeFindInClone(hipGraphNode_t *phNode, hipGraphNode_t hOriginalNode, hipGraph_t hClonedGraph);
  // HIP: hipError_t hipGraphNodeFindInClone(hipGraphNode_t* pNode, hipGraphNode_t originalNode, hipGraph_t clonedGraph);
  // CHECK: result = hipGraphNodeFindInClone(&graphNode2, graphNode, graph);
  result = hipGraphNodeFindInClone(&graphNode2, graphNode, graph);

  // CUDA: hipError_t CUDAAPI hipStreamIsCapturing(hipStream_t hStream, hipStreamCaptureStatus *captureStatus);
  // HIP: hipError_t hipStreamIsCapturing(hipStream_t stream, hipStreamCaptureStatus* pCaptureStatus);
  // CHECK: result = hipStreamIsCapturing(stream, &streamCaptureStatus);
  result = hipStreamIsCapturing(stream, &streamCaptureStatus);

  // CUDA: hipError_t CUDAAPI hipGraphHostNodeGetParams(hipGraphNode_t hNode, hipHostNodeParams *nodeParams);
  // HIP: hipError_t hipGraphHostNodeGetParams(hipGraphNode_t node, hipHostNodeParams* pNodeParams);
  // CHECK: result = hipGraphHostNodeGetParams(graphNode, &host_node_params);
  result = hipGraphHostNodeGetParams(graphNode, &host_node_params);

  // CUDA: hipError_t CUDAAPI hipGraphHostNodeSetParams(hipGraphNode_t hNode, const hipHostNodeParams *nodeParams);
  // HIP: hipError_t hipGraphHostNodeSetParams(hipGraphNode_t node, const hipHostNodeParams* pNodeParams);
  // CHECK: result = hipGraphHostNodeSetParams(graphNode, &host_node_params);
  result = hipGraphHostNodeSetParams(graphNode, &host_node_params);

  // CUDA: hipError_t CUDAAPI hipGraphAddChildGraphNode(hipGraphNode_t *phGraphNode, hipGraph_t hGraph, const hipGraphNode_t *dependencies, size_t numDependencies, hipGraph_t childGraph);
  // HIP: hipError_t hipGraphAddChildGraphNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, hipGraph_t childGraph);
  // CHECK: result = hipGraphAddChildGraphNode(&graphNode, graph, &graphNode2, bytes, graph2);
  result = hipGraphAddChildGraphNode(&graphNode, graph, &graphNode2, bytes, graph2);

  // CUDA: hipError_t CUDAAPI hipGraphChildGraphNodeGetGraph(hipGraphNode_t hNode, hipGraph_t *phGraph);
  // HIP: hipError_t hipGraphChildGraphNodeGetGraph(hipGraphNode_t node, hipGraph_t* pGraph);
  // CHECK: result = hipGraphChildGraphNodeGetGraph(graphNode, &graph);
  result = hipGraphChildGraphNodeGetGraph(graphNode, &graph);

  // CUDA: hipError_t CUDAAPI hipGraphClone(hipGraph_t *phGraphClone, hipGraph_t originalGraph);
  // HIP: hipError_t hipGraphClone(hipGraph_t* pGraphClone, hipGraph_t originalGraph);
  // CHECK: result = hipGraphClone(&graph, graph2);
  result = hipGraphClone(&graph, graph2);

  // CUDA: hipError_t CUDAAPI hipDrvGraphAddMemcpyNode(hipGraphNode_t *phGraphNode, hipGraph_t hGraph, const hipGraphNode_t *dependencies, size_t numDependencies, const HIP_MEMCPY3D *copyParams, hipCtx_t ctx);
  // HIP: hipError_t hipDrvGraphAddMemcpyNode(hipGraphNode_t* phGraphNode, hipGraph_t hGraph, const hipGraphNode_t* dependencies, size_t numDependencies, const HIP_MEMCPY3D* copyParams, hipCtx_t ctx);
  // CHECK: result = hipDrvGraphAddMemcpyNode(&graphNode, graph, &graphNode2, bytes, &MEMCPY3D, context);
  result = hipDrvGraphAddMemcpyNode(&graphNode, graph, &graphNode2, bytes, &MEMCPY3D, context);
#endif

#if CUDA_VERSION >= 10000 && CUDA_VERSION < 12000
  // CUDA: hipError_t CUDAAPI hipGraphInstantiate(hipGraphExec_t *phGraphExec, hipGraph_t hGraph, hipGraphNode_t *phErrorNode, char *logBuffer, size_t bufferSize);
  // HIP: hipError_t hipGraphInstantiate(hipGraphExec_t* pGraphExec, hipGraph_t graph, hipGraphNode_t* pErrorNode, char* pLogBuffer, size_t bufferSize);
  // CHECK: result = hipGraphInstantiate(&graphExec, graph, &graphNode, nullptr, bytes);
  result = hipGraphInstantiate(&graphExec, graph, &graphNode, nullptr, bytes);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipStreamCaptureMode streamCaptureMode;
  hipStreamCaptureMode streamCaptureMode;

  // CUDA: hipError_t CUDAAPI hipThreadExchangeStreamCaptureMode(hipStreamCaptureMode *mode);
  // HIP: hipError_t hipThreadExchangeStreamCaptureMode(hipStreamCaptureMode* mode);
  // CHECK: result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);
  result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);

  // CUDA: hipError_t CUDAAPI hipStreamBeginCapture(hipStream_t hStream, hipStreamCaptureMode mode);
  // HIP: hipError_t hipStreamBeginCapture(hipStream_t stream, hipStreamCaptureMode mode);
  // CHECK: result = hipStreamBeginCapture(stream, streamCaptureMode);
  // CHECK-NEXT: result = hipStreamBeginCapture(stream, streamCaptureMode);
  result = hipStreamBeginCapture(stream, streamCaptureMode);
  result = hipStreamBeginCapture(stream, streamCaptureMode);

  // CUDA: hipError_t CUDAAPI hipGraphExecKernelNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, const hipKernelNodeParams *nodeParams);
  // HIP: hipError_t hipGraphExecKernelNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t node, const hipKernelNodeParams* pNodeParams);
  // CHECK: result = hipGraphExecKernelNodeSetParams(graphExec, graphNode, &KERNEL_NODE_PARAMS);
  result = hipGraphExecKernelNodeSetParams(graphExec, graphNode, &KERNEL_NODE_PARAMS);
#endif

#if CUDA_VERSION >= 10010 && CUDA_VERSION < 12000
  // CUDA: hipError_t CUDAAPI hipStreamGetCaptureInfo(hipStream_t hStream, hipStreamCaptureStatus *captureStatus_out, cuuint64_t *id_out);
  // HIP: hipError_t hipStreamGetCaptureInfo(hipStream_t stream, hipStreamCaptureStatus* pCaptureStatus, unsigned long long* pId);
  // CHECK: result = hipStreamGetCaptureInfo(stream, &streamCaptureStatus, &ull);
  result = hipStreamGetCaptureInfo(stream, &streamCaptureStatus, &ull);
#endif

#if CUDA_VERSION >= 10020
  // CHECK: hipGraphExecUpdateResult graphExecUpdateResult;
  hipGraphExecUpdateResult graphExecUpdateResult;

  // CHECK: hipMemLocation memLocation_st;
  // CHECK-NEXT: hipMemLocation memLocation;
  // CHECK-NEXT: hipMemAllocationHandleType memAllocationHandleType;
  hipMemLocation memLocation_st;
  hipMemLocation memLocation;
  hipMemAllocationHandleType memAllocationHandleType;

  // CUDA: hipError_t CUDAAPI hipGraphExecHostNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, const hipHostNodeParams *nodeParams);
  // HIP: hipError_t hipError_t hipGraphExecHostNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t node, const hipHostNodeParams* pNodeParams);
  // CHECK: result = hipGraphExecHostNodeSetParams(graphExec, graphNode, &host_node_params);
  result = hipGraphExecHostNodeSetParams(graphExec, graphNode, &host_node_params);

  // CHECK: hipMemAllocationProp memAllocationProp;
  hipMemAllocationProp memAllocationProp;
  // CHECK: hipMemGenericAllocationHandle_t memGenericAllocationHandle_t;
  hipMemGenericAllocationHandle_t memGenericAllocationHandle_t;
  // CHECK: hipMemAllocationGranularity_flags memAllocationGranularity_flags;
  hipMemAllocationGranularity_flags memAllocationGranularity_flags;
  // CHECK: hipMemAccessDesc memAccessDesc;
  hipMemAccessDesc memAccessDesc;

  // CUDA: hipError_t CUDAAPI hipMemAddressFree(hipDeviceptr_t ptr, size_t size);
  // HIP: hipError_t hipMemAddressFree(void* devPtr, size_t size);
  // CHECK: result = hipMemAddressFree(deviceptr, bytes);
  result = hipMemAddressFree(deviceptr, bytes);

  // CUDA: hipError_t CUDAAPI hipMemAddressReserve(hipDeviceptr_t *ptr, size_t size, size_t alignment, hipDeviceptr_t addr, unsigned long long flags);
  // HIP: hipError_t hipMemAddressReserve(void** ptr, size_t size, size_t alignment, void* addr, unsigned long long flags);
  // CHECK: result = hipMemAddressReserve(&deviceptr, bytes, bytes_2, deviceptr_2, ull);
  result = hipMemAddressReserve(&deviceptr, bytes, bytes_2, deviceptr_2, ull);

  // CUDA: hipError_t CUDAAPI hipMemCreate(hipMemGenericAllocationHandle_t *handle, size_t size, const hipMemAllocationProp *prop, unsigned long long flags);
  // HIP: hipError_t hipMemCreate(hipMemGenericAllocationHandle_t* handle, size_t size, const hipMemAllocationProp* prop, unsigned long long flags);
  // CHECK: result = hipMemCreate(&memGenericAllocationHandle_t, bytes, &memAllocationProp, ull);
  result = hipMemCreate(&memGenericAllocationHandle_t, bytes, &memAllocationProp, ull);

  // CUDA: hipError_t CUDAAPI hipMemExportToShareableHandle(void *shareableHandle, hipMemGenericAllocationHandle_t handle, hipMemAllocationHandleType handleType, unsigned long long flags);
  // HIP: hipError_t hipMemExportToShareableHandle(void* shareableHandle, hipMemGenericAllocationHandle_t handle, hipMemAllocationHandleType handleType, unsigned long long flags);
  // CHECK: result = hipMemExportToShareableHandle(image, memGenericAllocationHandle_t, memAllocationHandleType, ull);
  result = hipMemExportToShareableHandle(image, memGenericAllocationHandle_t, memAllocationHandleType, ull);

  // CUDA: hipError_t CUDAAPI hipMemGetAccess(unsigned long long *flags, const hipMemLocation *location, hipDeviceptr_t ptr);
  // HIP: hipError_t hipMemGetAccess(unsigned long long* flags, const hipMemLocation* location, void* ptr);
  // CHECK: result = hipMemGetAccess(&ull_2, &memLocation, deviceptr);
  result = hipMemGetAccess(&ull_2, &memLocation, deviceptr);

  // CUDA: hipError_t CUDAAPI hipMemGetAllocationGranularity(size_t *granularity, const hipMemAllocationProp *prop, hipMemAllocationGranularity_flags option);
  // HIP: hipError_t hipMemGetAllocationGranularity(size_t* granularity, const hipMemAllocationProp* prop, hipMemAllocationGranularity_flags option);
  // CHECK: result = hipMemGetAllocationGranularity(&bytes, &memAllocationProp, memAllocationGranularity_flags);
  result = hipMemGetAllocationGranularity(&bytes, &memAllocationProp, memAllocationGranularity_flags);

  // CUDA: hipError_t CUDAAPI hipMemGetAllocationPropertiesFromHandle(hipMemAllocationProp *prop, hipMemGenericAllocationHandle_t handle);
  // HIP: hipError_t hipMemGetAllocationPropertiesFromHandle(hipMemAllocationProp* prop, hipMemGenericAllocationHandle_t handle);
  // CHECK: result = hipMemGetAllocationPropertiesFromHandle(&memAllocationProp, memGenericAllocationHandle_t);
  result = hipMemGetAllocationPropertiesFromHandle(&memAllocationProp, memGenericAllocationHandle_t);

  // CUDA: hipError_t CUDAAPI hipMemImportFromShareableHandle(hipMemGenericAllocationHandle_t *handle, void *osHandle, hipMemAllocationHandleType shHandleType);
  // HIP: hipError_t hipMemImportFromShareableHandle(hipMemGenericAllocationHandle_t* handle, void* osHandle, hipMemAllocationHandleType shHandleType);
  // CHECK: result = hipMemImportFromShareableHandle(&memGenericAllocationHandle_t, image, memAllocationHandleType);
  result = hipMemImportFromShareableHandle(&memGenericAllocationHandle_t, image, memAllocationHandleType);

  // CUDA: hipError_t CUDAAPI hipMemMap(hipDeviceptr_t ptr, size_t size, size_t offset, hipMemGenericAllocationHandle_t handle, unsigned long long flags);
  // HIP: hipError_t hipMemMap(void* ptr, size_t size, size_t offset, hipMemGenericAllocationHandle_t handle, unsigned long long flags);
  // CHECK: result = hipMemMap(deviceptr, bytes, bytes_2, memGenericAllocationHandle_t, ull);
  result = hipMemMap(deviceptr, bytes, bytes_2, memGenericAllocationHandle_t, ull);

  // CUDA: hipError_t CUDAAPI hipMemRelease(hipMemGenericAllocationHandle_t handle);
  // HIP: hipError_t hipMemRelease(hipMemGenericAllocationHandle_t handle);
  // CHECK: result = hipMemRelease(memGenericAllocationHandle_t);
  result = hipMemRelease(memGenericAllocationHandle_t);

  // CUDA: hipError_t CUDAAPI hipMemSetAccess(hipDeviceptr_t ptr, size_t size, const hipMemAccessDesc *desc, size_t count);
  // HIP: hipError_t hipMemSetAccess(void* ptr, size_t size, const hipMemAccessDesc* desc, size_t count);
  // CHECK: result = hipMemSetAccess(deviceptr, bytes, &memAccessDesc, bytes_2);
  result = hipMemSetAccess(deviceptr, bytes, &memAccessDesc, bytes_2);

  // CUDA: hipError_t CUDAAPI hipMemUnmap(hipDeviceptr_t ptr, size_t size);
  // HIP: hipError_t hipMemUnmap(void* ptr, size_t size);
  // CHECK: result = hipMemUnmap(deviceptr, bytes);
  result = hipMemUnmap(deviceptr, bytes);
#endif

#if CUDA_VERSION >= 10020 && CUDA_VERSION < 12000
  // CUDA: hipError_t CUDAAPI hipGraphExecUpdate(hipGraphExec_t hGraphExec, hipGraph_t hGraph, hipGraphNode_t *hErrorNode_out, hipGraphExecUpdateResult *updateResult_out);
  // HIP: hipError_t hipGraphExecUpdate(hipGraphExec_t hGraphExec, hipGraph_t hGraph, hipGraphNode_t* hErrorNode_out, hipGraphExecUpdateResult* updateResult_out);
  // CHECK: result = hipGraphExecUpdate(graphExec, graph, &graphNode, &graphExecUpdateResult);
  result = hipGraphExecUpdate(graphExec, graph, &graphNode, &graphExecUpdateResult);
#endif

#if CUDA_VERSION >= 11000
  // CHECK: result = hipDevicePrimaryCtxRelease(device);
  result = hipDevicePrimaryCtxRelease(device);

  // CHECK: result = hipDevicePrimaryCtxReset(device);
  result = hipDevicePrimaryCtxReset(device);

  // CHECK: result = hipDevicePrimaryCtxSetFlags(device, flags);
  result = hipDevicePrimaryCtxSetFlags(device, flags);

  // CUDA: hipError_t CUDAAPI hipMemRetainAllocationHandle(hipMemGenericAllocationHandle_t *handle, void *addr);
  // HIP: hipError_t hipMemRetainAllocationHandle(hipMemGenericAllocationHandle_t* handle, void* addr);
  // CHECK: result = hipMemRetainAllocationHandle(&memGenericAllocationHandle_t, image);
  result = hipMemRetainAllocationHandle(&memGenericAllocationHandle_t, image);

  // CHECK: hipKernelNodeAttrID kernelNodeAttrID;
  hipKernelNodeAttrID kernelNodeAttrID;
  // CHECK: hipKernelNodeAttrValue kernelNodeAttrValue;
  hipKernelNodeAttrValue kernelNodeAttrValue;

  // CUDA: hipError_t CUDAAPI hipGraphKernelNodeSetAttribute(hipGraphNode_t hNode, hipKernelNodeAttrID attr, const hipKernelNodeAttrValue* value);
  // HIP: hipError_t hipGraphKernelNodeSetAttribute(hipGraphNode_t hNode, hipKernelNodeAttrID attr, const hipKernelNodeAttrValue* value);
  // CHECK: result = hipGraphKernelNodeSetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);
  result = hipGraphKernelNodeSetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);

  // CUDA: hipError_t CUDAAPI hipGraphKernelNodeGetAttribute(hipGraphNode_t hNode, hipKernelNodeAttrID attr, hipKernelNodeAttrValue* value_out);
  // HIP: hipError_t hipGraphKernelNodeGetAttribute(hipGraphNode_t hNode, hipKernelNodeAttrID attr, hipKernelNodeAttrValue* value);
  // CHECK: result = hipGraphKernelNodeGetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);
  result = hipGraphKernelNodeGetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);

  // CUDA: hipError_t CUDAAPI hipGraphKernelNodeCopyAttributes(hipGraphNode_t dst, hipGraphNode_t src);
  // HIP: hipError_t hipGraphKernelNodeCopyAttributes(hipGraphNode_t hSrc, hipGraphNode_t hDst);
  // CHECK: result = hipGraphKernelNodeCopyAttributes(graphNode, graphNode2);
  result = hipGraphKernelNodeCopyAttributes(graphNode, graphNode2);
#endif

#if CUDA_VERSION >= 11000 && CUDA_VERSION < 12000
  // CHECK: result = hipGraphInstantiate(&graphExec, graph, &graphNode, nullptr, bytes);
  result = hipGraphInstantiate(&graphExec, graph, &graphNode, nullptr, bytes);
#endif

#if CUDA_VERSION >= 11010
  // CUDA: hipError_t CUDAAPI hipGraphExecChildGraphNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, hipGraph_t childGraph);
  // HIP: hipError_t hipGraphExecChildGraphNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t node, hipGraph_t childGraph);
  // CHECK: result = hipGraphExecChildGraphNodeSetParams(graphExec, graphNode, graph);
  result = hipGraphExecChildGraphNodeSetParams(graphExec, graphNode, graph);

  // CUDA: hipError_t CUDAAPI hipGraphAddEventRecordNode(hipGraphNode_t *phGraphNode, hipGraph_t hGraph, const hipGraphNode_t *dependencies, size_t numDependencies, hipEvent_t event);
  // HIP: hipError_t hipGraphAddEventRecordNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, hipEvent_t event);
  // CHECK: result = hipGraphAddEventRecordNode(&graphNode, graph, &graphNode2, bytes, event_);
  result = hipGraphAddEventRecordNode(&graphNode, graph, &graphNode2, bytes, event_);

  // CUDA: hipError_t CUDAAPI hipGraphEventRecordNodeGetEvent(hipGraphNode_t hNode, hipEvent_t *event_out);
  // HIP: hipError_t hipGraphEventRecordNodeGetEvent(hipGraphNode_t node, hipEvent_t* event_out);
  // CHECK: result = hipGraphEventRecordNodeGetEvent(graphNode, &event_);
  result = hipGraphEventRecordNodeGetEvent(graphNode, &event_);

  // CUDA: hipError_t CUDAAPI hipGraphEventRecordNodeSetEvent(hipGraphNode_t hNode, hipEvent_t event);
  // HIP: hipError_t hipGraphEventRecordNodeSetEvent(hipGraphNode_t node, hipEvent_t event);
  // CHECK: result = hipGraphEventRecordNodeSetEvent(graphNode, event_);
  result = hipGraphEventRecordNodeSetEvent(graphNode, event_);

  // CUDA: hipError_t CUDAAPI hipGraphExecEventRecordNodeSetEvent(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, hipEvent_t event);
  // HIP: hipError_t hipGraphExecEventRecordNodeSetEvent(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, hipEvent_t event);
  // CHECK: result = hipGraphExecEventRecordNodeSetEvent(graphExec, graphNode, event_);
  result = hipGraphExecEventRecordNodeSetEvent(graphExec, graphNode, event_);

  // CUDA: hipError_t CUDAAPI hipGraphAddEventWaitNode(hipGraphNode_t *phGraphNode, hipGraph_t hGraph, const hipGraphNode_t *dependencies, size_t numDependencies, hipEvent_t event);
  // HIP: hipError_t hipGraphAddEventWaitNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, hipEvent_t event);
  // CHECK: result = hipGraphAddEventWaitNode(&graphNode, graph, &graphNode2, bytes, event_);
  result = hipGraphAddEventWaitNode(&graphNode, graph, &graphNode2, bytes, event_);

  // CUDA: hipError_t CUDAAPI hipGraphEventWaitNodeGetEvent(hipGraphNode_t hNode, hipEvent_t *event_out);
  // HIP: hipError_t hipGraphEventWaitNodeGetEvent(hipGraphNode_t node, hipEvent_t* event_out);
  // CHECK: result = hipGraphEventWaitNodeGetEvent(graphNode, &event_);
  result = hipGraphEventWaitNodeGetEvent(graphNode, &event_);

  // CUDA: hipError_t CUDAAPI hipGraphEventWaitNodeSetEvent(hipGraphNode_t hNode, hipEvent_t event);
  // HIP: hipError_t hipGraphEventWaitNodeSetEvent(hipGraphNode_t node, hipEvent_t event);
  // CHECK: result = hipGraphEventWaitNodeSetEvent(graphNode, event_);
  result = hipGraphEventWaitNodeSetEvent(graphNode, event_);

  // CUDA: hipError_t CUDAAPI hipGraphExecEventWaitNodeSetEvent(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, hipEvent_t event);
  // HIP: hipError_t hipGraphExecEventWaitNodeSetEvent(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, hipEvent_t event);
  // CHECK: result = hipGraphExecEventWaitNodeSetEvent(graphExec, graphNode, event_);
  result = hipGraphExecEventWaitNodeSetEvent(graphExec, graphNode, event_);

  // CHECK: hipArrayMapInfo arrayMapInfo;
  hipArrayMapInfo arrayMapInfo;

  // CUDA: hipError_t CUDAAPI hipMemMapArrayAsync(hipArrayMapInfo *mapInfoList, unsigned int count, hipStream_t hStream);
  // HIP: hipError_t hipMemMapArrayAsync(hipArrayMapInfo* mapInfoList, unsigned int count, hipStream_t stream);
  // CHECK: result = hipMemMapArrayAsync(&arrayMapInfo, flags, stream);
  result = hipMemMapArrayAsync(&arrayMapInfo, flags, stream);

  // CUDA: hipError_t CUDAAPI hipGraphUpload(hipGraphExec_t hGraphExec, hipStream_t hStream);
  // HIP: hipError_t hipGraphUpload(hipGraphExec_t graphExec, hipStream_t stream);
  // CHECK: result = hipGraphUpload(graphExec, stream);
  result = hipGraphUpload(graphExec, stream);
#endif

#if CUDA_VERSION >= 11020
  // CHECK: hipMemPool_t memPool_t;
  // CHECK-NEXT: hipMemPoolProps memPoolProps_st;
  // CHECK-NEXT: hipMemPoolProps memPoolProps;
  hipMemPool_t memPool_t;
  hipMemPoolProps memPoolProps_st;
  hipMemPoolProps memPoolProps;

  // CUDA: hipError_t CUDAAPI hipDeviceGetDefaultMemPool(hipMemPool_t *pool_out, hipDevice_t dev);
  // HIP: hipError_t hipDeviceGetDefaultMemPool(hipMemPool_t* mem_pool, int device);
  // CHECK: result = hipDeviceGetDefaultMemPool(&memPool_t, device);
  result = hipDeviceGetDefaultMemPool(&memPool_t, device);

  // CUDA: hipError_t CUDAAPI hipDeviceSetMemPool(hipDevice_t dev, hipMemPool_t pool);
  // HIP: hipError_t hipDeviceSetMemPool(int device, hipMemPool_t mem_pool);
  // CHECK: result = hipDeviceSetMemPool(device, memPool_t);
  result = hipDeviceSetMemPool(device, memPool_t);

  // CUDA: hipError_t CUDAAPI hipDeviceGetMemPool(hipMemPool_t *pool, hipDevice_t dev);
  // HIP: hipError_t hipDeviceGetMemPool(hipMemPool_t* mem_pool, int device);
  // CHECK: result = hipDeviceGetMemPool(&memPool_t, device);
  result = hipDeviceGetMemPool(&memPool_t, device);

  // CUDA: hipError_t CUDAAPI hipMallocAsync(hipDeviceptr_t *dptr, size_t bytesize, hipStream_t hStream);
  // HIP: hipError_t hipMallocAsync(void** dev_ptr, size_t size, hipStream_t stream);
  // CHECK: result = hipMallocAsync(&deviceptr, bytes, stream);
  result = hipMallocAsync(&deviceptr, bytes, stream);

  // CUDA: hipError_t CUDAAPI hipFreeAsync(hipDeviceptr_t dptr, hipStream_t hStream);
  // HIP: hipError_t hipFreeAsync(void* dev_ptr, hipStream_t stream);
  // CHECK: result = hipFreeAsync(deviceptr, stream);
  result = hipFreeAsync(deviceptr, stream);

  // CUDA: hipError_t CUDAAPI hipMemPoolTrimTo(hipMemPool_t pool, size_t minBytesToKeep);
  // HIP: hipError_t hipMemPoolTrimTo(hipMemPool_t mem_pool, size_t min_bytes_to_hold);
  // CHECK: result = hipMemPoolTrimTo(memPool_t, bytes);
  result = hipMemPoolTrimTo(memPool_t, bytes);

  // CHECK: hipMemPoolAttr memPoolAttr;
  hipMemPoolAttr memPoolAttr;

  // CUDA: hipError_t CUDAAPI hipMemPoolSetAttribute(hipMemPool_t pool, hipMemPoolAttr attr, void *value);
  // HIP: hipError_t hipMemPoolSetAttribute(hipMemPool_t mem_pool, hipMemPoolAttr attr, void* value);
  // CHECK: result = hipMemPoolSetAttribute(memPool_t, memPoolAttr, image);
  result = hipMemPoolSetAttribute(memPool_t, memPoolAttr, image);

  // CUDA: hipError_t CUDAAPI hipMemPoolGetAttribute(hipMemPool_t pool, hipMemPoolAttr attr, void *value);
  // HIP: hipError_t hipMemPoolGetAttribute(hipMemPool_t mem_pool, hipMemPoolAttr attr, void* value);
  // CHECK: result = hipMemPoolGetAttribute(memPool_t, memPoolAttr, image);
  result = hipMemPoolGetAttribute(memPool_t, memPoolAttr, image);

  // CUDA: hipError_t CUDAAPI hipMemPoolSetAccess(hipMemPool_t pool, const hipMemAccessDesc *map, size_t count);
  // HIP: hipError_t hipMemPoolSetAccess(hipMemPool_t mem_pool, const hipMemAccessDesc* desc_list, size_t count);
  // CHECK: result = hipMemPoolSetAccess(memPool_t, &memAccessDesc, bytes);
  result = hipMemPoolSetAccess(memPool_t, &memAccessDesc, bytes);

  // CHECK: hipMemAccessFlags memAccessFlags;
  hipMemAccessFlags memAccessFlags;

  // CUDA: hipError_t CUDAAPI hipMemPoolGetAccess(hipMemAccessFlags *flags, hipMemPool_t memPool, hipMemLocation *location);
  // HIP: hipError_t hipMemPoolGetAccess(hipMemAccessFlags* flags, hipMemPool_t mem_pool, hipMemLocation* location);
  // CHECK: result = hipMemPoolGetAccess(&memAccessFlags, memPool_t, &memLocation);
  result = hipMemPoolGetAccess(&memAccessFlags, memPool_t, &memLocation);

  // CUDA: hipError_t CUDAAPI hipMemPoolCreate(hipMemPool_t *pool, const hipMemPoolProps *poolProps);
  // HIP: hipError_t hipMemPoolCreate(hipMemPool_t* mem_pool, const hipMemPoolProps* pool_props);
  // CHECK: result = hipMemPoolCreate(&memPool_t, &memPoolProps);
  result = hipMemPoolCreate(&memPool_t, &memPoolProps);

  // CUDA: hipError_t CUDAAPI hipMemPoolDestroy(hipMemPool_t pool);
  // HIP: hipError_t hipMemPoolDestroy(hipMemPool_t mem_pool);
  // CHECK: result = hipMemPoolDestroy(memPool_t);
  result = hipMemPoolDestroy(memPool_t);

  // CUDA: hipError_t CUDAAPI hipMemPoolExportToShareableHandle(void *handle_out, hipMemPool_t pool, hipMemAllocationHandleType handleType, unsigned long long flags);
  // HIP: hipError_t hipMemPoolExportToShareableHandle(void* shared_handle, hipMemPool_t mem_pool, hipMemAllocationHandleType handle_type, unsigned int flags);
  // CHECK: result = hipMemPoolExportToShareableHandle(image, memPool_t, memAllocationHandleType, ull);
  result = hipMemPoolExportToShareableHandle(image, memPool_t, memAllocationHandleType, ull);

  // CUDA: hipError_t CUDAAPI hipMemPoolImportFromShareableHandle(hipMemPool_t* pool_out, void* handle, hipMemAllocationHandleType handleType, unsigned long long flags);
  // HIP: hipError_t hipMemPoolImportFromShareableHandle(hipMemPool_t* mem_pool, void* shared_handle, hipMemAllocationHandleType handle_type, unsigned int flags);
  // CHECK: result = hipMemPoolImportFromShareableHandle(&memPool_t, image, memAllocationHandleType, ull);
  result = hipMemPoolImportFromShareableHandle(&memPool_t, image, memAllocationHandleType, ull);

  // CHECK: hipMemPoolPtrExportData memPoolPtrExportData;
  hipMemPoolPtrExportData memPoolPtrExportData;

  // CUDA: hipError_t CUDAAPI hipMemPoolExportPointer(hipMemPoolPtrExportData *shareData_out, hipDeviceptr_t ptr);
  // HIP: hipError_t hipMemPoolExportPointer(hipMemPoolPtrExportData* export_data, void* dev_ptr);
  // CHECK: result = hipMemPoolExportPointer(&memPoolPtrExportData, deviceptr);
  result = hipMemPoolExportPointer(&memPoolPtrExportData, deviceptr);

  // CUDA: hipError_t CUDAAPI hipMemPoolImportPointer(hipDeviceptr_t *ptr_out, hipMemPool_t pool, hipMemPoolPtrExportData *shareData);
  // HIP: hipError_t hipMemPoolImportPointer(void** dev_ptr, hipMemPool_t mem_pool, hipMemPoolPtrExportData* export_data);
  // CHECK: result = hipMemPoolImportPointer(&deviceptr, memPool_t, &memPoolPtrExportData);
  result = hipMemPoolImportPointer(&deviceptr, memPool_t, &memPoolPtrExportData);

  // CHECK: hipExternalSemaphoreSignalNodeParams EXT_SEM_SIGNAL_NODE_PARAMS;
  hipExternalSemaphoreSignalNodeParams EXT_SEM_SIGNAL_NODE_PARAMS;

  // CUDA: hipError_t CUDAAPI hipGraphAddExternalSemaphoresSignalNode(hipGraphNode_t *phGraphNode, hipGraph_t hGraph, const hipGraphNode_t *dependencies, size_t numDependencies, const hipExternalSemaphoreSignalNodeParams *nodeParams);
  // HIP: hipError_t hipGraphAddExternalSemaphoresSignalNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, const hipExternalSemaphoreSignalNodeParams* nodeParams);
  // CHECK: result = hipGraphAddExternalSemaphoresSignalNode(&graphNode, graph, &graphNode2, bytes, &EXT_SEM_SIGNAL_NODE_PARAMS);
  result = hipGraphAddExternalSemaphoresSignalNode(&graphNode, graph, &graphNode2, bytes, &EXT_SEM_SIGNAL_NODE_PARAMS);

  // CUDA: hipError_t CUDAAPI hipGraphExternalSemaphoresSignalNodeGetParams(hipGraphNode_t hNode, hipExternalSemaphoreSignalNodeParams *params_out);
  // HIP: hipError_t hipGraphExternalSemaphoresSignalNodeGetParams(hipGraphNode_t hNode, hipExternalSemaphoreSignalNodeParams* params_out);
  // CHECK: result = hipGraphExternalSemaphoresSignalNodeGetParams(graphNode, &EXT_SEM_SIGNAL_NODE_PARAMS);
  result = hipGraphExternalSemaphoresSignalNodeGetParams(graphNode, &EXT_SEM_SIGNAL_NODE_PARAMS);

  // CUDA: hipError_t CUDAAPI hipGraphExternalSemaphoresSignalNodeSetParams(hipGraphNode_t hNode, const hipExternalSemaphoreSignalNodeParams *nodeParams);
  // HIP: hipError_t hipGraphExternalSemaphoresSignalNodeSetParams(hipGraphNode_t hNode, const hipExternalSemaphoreSignalNodeParams* nodeParams);
  // CHECK: result = hipGraphExternalSemaphoresSignalNodeSetParams(graphNode, &EXT_SEM_SIGNAL_NODE_PARAMS);
  result = hipGraphExternalSemaphoresSignalNodeSetParams(graphNode, &EXT_SEM_SIGNAL_NODE_PARAMS);

  // CHECK: hipExternalSemaphoreWaitNodeParams EXT_SEM_WAIT_NODE_PARAMS;
  hipExternalSemaphoreWaitNodeParams EXT_SEM_WAIT_NODE_PARAMS;

  // CUDA: hipError_t CUDAAPI hipGraphAddExternalSemaphoresWaitNode(hipGraphNode_t *phGraphNode, hipGraph_t hGraph, const hipGraphNode_t *dependencies, size_t numDependencies, const hipExternalSemaphoreWaitNodeParams *nodeParams);
  // HIP: hipError_t hipGraphAddExternalSemaphoresWaitNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, const hipExternalSemaphoreWaitNodeParams* nodeParams);
  // CHECK: result = hipGraphAddExternalSemaphoresWaitNode(&graphNode, graph, &graphNode2, bytes, &EXT_SEM_WAIT_NODE_PARAMS);
  result = hipGraphAddExternalSemaphoresWaitNode(&graphNode, graph, &graphNode2, bytes, &EXT_SEM_WAIT_NODE_PARAMS);

  // CUDA: hipError_t CUDAAPI hipGraphExternalSemaphoresWaitNodeGetParams(hipGraphNode_t hNode, hipExternalSemaphoreWaitNodeParams *params_out);
  // HIP: hipError_t hipGraphExternalSemaphoresWaitNodeGetParams(hipGraphNode_t hNode, hipExternalSemaphoreWaitNodeParams* params_out);
  // CHECK: result = hipGraphExternalSemaphoresWaitNodeGetParams(graphNode, &EXT_SEM_WAIT_NODE_PARAMS);
  result = hipGraphExternalSemaphoresWaitNodeGetParams(graphNode, &EXT_SEM_WAIT_NODE_PARAMS);

  // CUDA: hipError_t CUDAAPI hipGraphExternalSemaphoresWaitNodeSetParams(hipGraphNode_t hNode, const hipExternalSemaphoreWaitNodeParams *nodeParams);
  // HIP: hipError_t hipGraphExternalSemaphoresWaitNodeSetParams(hipGraphNode_t hNode, const hipExternalSemaphoreWaitNodeParams* nodeParams);
  // CHECK: result = hipGraphExternalSemaphoresWaitNodeSetParams(graphNode, &EXT_SEM_WAIT_NODE_PARAMS);
  result = hipGraphExternalSemaphoresWaitNodeSetParams(graphNode, &EXT_SEM_WAIT_NODE_PARAMS);

  // CUDA: hipError_t CUDAAPI hipGraphExecExternalSemaphoresSignalNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, const hipExternalSemaphoreSignalNodeParams *nodeParams);
  // HIP: hipError_t hipGraphExecExternalSemaphoresSignalNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, const hipExternalSemaphoreSignalNodeParams* nodeParams);
  // CHECK: result = hipGraphExecExternalSemaphoresSignalNodeSetParams(graphExec, graphNode, &EXT_SEM_SIGNAL_NODE_PARAMS);
  result = hipGraphExecExternalSemaphoresSignalNodeSetParams(graphExec, graphNode, &EXT_SEM_SIGNAL_NODE_PARAMS);

  // CUDA: hipError_t CUDAAPI hipGraphExecExternalSemaphoresWaitNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, const hipExternalSemaphoreWaitNodeParams *nodeParams);
  // HIP: hipError_t hipGraphExecExternalSemaphoresWaitNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, const hipExternalSemaphoreWaitNodeParams* nodeParams);
  // CHECK: result = hipGraphExecExternalSemaphoresWaitNodeSetParams(graphExec, graphNode, &EXT_SEM_WAIT_NODE_PARAMS);
  result = hipGraphExecExternalSemaphoresWaitNodeSetParams(graphExec, graphNode, &EXT_SEM_WAIT_NODE_PARAMS);
#endif

#if CUDA_VERSION >= 11030
  // CUDA < 12000: hipError_t CUDAAPI hipStreamGetCaptureInfo(hipStream_t hStream, hipStreamCaptureStatus *captureStatus_out, cuuint64_t *id_out);
  // CUDA:         hipError_t CUDAAPI hipStreamGetCaptureInfo_v2(hipStream_t hStream, hipStreamCaptureStatus *captureStatus_out, cuuint64_t *id_out, hipGraph_t *graph_out, const hipGraphNode_t **dependencies_out, size_t *numDependencies_out);
  // HIP: hipError_t hipStreamGetCaptureInfo_v2(hipStream_t stream, hipStreamCaptureStatus* captureStatus_out, unsigned long long* id_out __dparm(0), hipGraph_t* graph_out __dparm(0), const hipGraphNode_t** dependencies_out __dparm(0), size_t* numDependencies_out __dparm(0));
  // CHECK: result = hipStreamGetCaptureInfo_v2(stream, &streamCaptureStatus, &ull, &graph, &pGraphNode, &bytes);
  result = hipStreamGetCaptureInfo_v2(stream, &streamCaptureStatus, &ull, &graph, &pGraphNode, &bytes);

  // CUDA: hipError_t CUDAAPI hipStreamUpdateCaptureDependencies(hipStream_t hStream, hipGraphNode_t *dependencies, size_t numDependencies, unsigned int flags);
  // HIP: hipError_t hipStreamUpdateCaptureDependencies(hipStream_t stream, hipGraphNode_t* dependencies, size_t numDependencies, unsigned int flags __dparm(0));
  // CHECK: result = hipStreamUpdateCaptureDependencies(stream, &graphNode, bytes, flags);
  result = hipStreamUpdateCaptureDependencies(stream, &graphNode, bytes, flags);

  // CHECK: hipUserObject_t userObject;
  hipUserObject_t userObject;

  // CUDA: hipError_t CUDAAPI hipUserObjectCreate(hipUserObject_t *object_out, void *ptr, hipHostFn_t destroy, unsigned int initialRefcount, unsigned int flags);
  // HIP: hipError_t hipUserObjectCreate(hipUserObject_t* object_out, void* ptr, hipHostFn_t destroy, unsigned int initialRefcount, unsigned int flags);
  // CHECK: result = hipUserObjectCreate(&userObject, image, hostFn, count, flags);
  result = hipUserObjectCreate(&userObject, image, hostFn, count, flags);

  // CUDA: hipError_t CUDAAPI hipUserObjectRelease(hipUserObject_t object, unsigned int count);
  // HIP: hipError_t hipUserObjectRelease(hipUserObject_t object, unsigned int count);
  // CHECK: result = hipUserObjectRelease(userObject, count);
  result = hipUserObjectRelease(userObject, count);

  // CUDA: hipError_t CUDAAPI hipUserObjectRetain(hipUserObject_t object, unsigned int count);
  // HIP: hipError_t hipUserObjectRetain(hipUserObject_t object, unsigned int count);
  // CHECK: result = hipUserObjectRetain(userObject, count);
  result = hipUserObjectRetain(userObject, count);

  // CUDA: hipError_t CUDAAPI hipGraphRetainUserObject(hipGraph_t graph, hipUserObject_t object, unsigned int count, unsigned int flags);
  // HIP: hipError_t hipGraphRetainUserObject(hipGraph_t graph, hipUserObject_t object, unsigned int count, unsigned int flags);
  // CHECK: result = hipGraphRetainUserObject(graph, userObject, count, flags);
  result = hipGraphRetainUserObject(graph, userObject, count, flags);

  // CUDA: hipError_t CUDAAPI hipGraphReleaseUserObject(hipGraph_t graph, hipUserObject_t object, unsigned int count);
  // HIP: hipError_t hipGraphReleaseUserObject(hipGraph_t graph, hipUserObject_t object, unsigned int count);
  // CHECK: result = hipGraphReleaseUserObject(graph, userObject, count);
  result = hipGraphReleaseUserObject(graph, userObject, count);

  // CUDA: hipError_t CUDAAPI hipGraphDebugDotPrint(hipGraph_t hGraph, const char *path, unsigned int flags);
  // HIP: hipError_t hipGraphDebugDotPrint(hipGraph_t graph, const char* path, unsigned int flags);
  // CHECK: result = hipGraphDebugDotPrint(graph, name.c_str(), flags);
  result = hipGraphDebugDotPrint(graph, name.c_str(), flags);
#endif

#if CUDA_VERSION >= 11040
  // CUDA: hipError_t CUDAAPI hipGraphInstantiateWithFlags(hipGraphExec_t *phGraphExec, hipGraph_t hGraph, unsigned long long flags);
  // HIP: hipError_t hipGraphInstantiateWithFlags(hipGraphExec_t* pGraphExec, hipGraph_t graph, unsigned long long flags);
  // CHECK: result = hipGraphInstantiateWithFlags(&graphExec, graph, ull);
  result = hipGraphInstantiateWithFlags(&graphExec, graph, ull);

  // CHECK: hipGraphMemAttributeType graphMem_attribute;
  hipGraphMemAttributeType graphMem_attribute;

  // CUDA: hipError_t CUDAAPI hipDeviceGetGraphMemAttribute(hipDevice_t device, hipGraphMemAttributeType attr, void* value);
  // HIP: hipError_t hipDeviceGetGraphMemAttribute(int device, hipGraphMemAttributeType attr, void* value);
  // CHECK: result = hipDeviceGetGraphMemAttribute(device, graphMem_attribute, image);
  result = hipDeviceGetGraphMemAttribute(device, graphMem_attribute, image);

  // CUDA: hipError_t CUDAAPI hipDeviceSetGraphMemAttribute(hipDevice_t device, hipGraphMemAttributeType attr, void* value);
  // HIP: hipError_t hipDeviceSetGraphMemAttribute(int device, hipGraphMemAttributeType attr, void* value);
  // CHECK: result = hipDeviceSetGraphMemAttribute(device, graphMem_attribute, image);
  result = hipDeviceSetGraphMemAttribute(device, graphMem_attribute, image);

  // CUDA: hipError_t CUDAAPI hipDeviceGraphMemTrim(hipDevice_t device);
  // HIP: hipError_t hipDeviceGraphMemTrim(int device);
  // CHECK: result = hipDeviceGraphMemTrim(device);
  result = hipDeviceGraphMemTrim(device);

  // CHECK: hipMemAllocNodeParams MEM_ALLOC_NODE_PARAMS;
  hipMemAllocNodeParams MEM_ALLOC_NODE_PARAMS;

  // CUDA: hipError_t CUDAAPI hipGraphAddMemAllocNode(hipGraphNode_t *phGraphNode, hipGraph_t hGraph, const hipGraphNode_t *dependencies, size_t numDependencies, hipMemAllocNodeParams *nodeParams);
  // HIP: hipError_t hipGraphAddMemAllocNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, hipMemAllocNodeParams* pNodeParams);
  // CHECK: result = hipGraphAddMemAllocNode(&graphNode, graph, &graphNode2, bytes, &MEM_ALLOC_NODE_PARAMS);
  result = hipGraphAddMemAllocNode(&graphNode, graph, &graphNode2, bytes, &MEM_ALLOC_NODE_PARAMS);

  // CUDA: hipError_t CUDAAPI hipGraphMemAllocNodeGetParams(hipGraphNode_t hNode, hipMemAllocNodeParams *params_out);
  // HIP: hipError_t hipGraphMemAllocNodeGetParams(hipGraphNode_t node, hipMemAllocNodeParams* pNodeParams);
  // CHECK: result = hipGraphMemAllocNodeGetParams(graphNode, &MEM_ALLOC_NODE_PARAMS);
  result = hipGraphMemAllocNodeGetParams(graphNode, &MEM_ALLOC_NODE_PARAMS);

  // CUDA: hipError_t CUDAAPI hipDrvGraphAddMemFreeNode(hipGraphNode_t *phGraphNode, hipGraph_t hGraph, const hipGraphNode_t *dependencies, size_t numDependencies, hipDeviceptr_t dptr);
  // HIP: hipError_t hipDrvGraphAddMemFreeNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, void* dev_ptr);
  // CHECK: result = hipDrvGraphAddMemFreeNode(&graphNode, graph, &graphNode2, bytes, deviceptr);
  result = hipDrvGraphAddMemFreeNode(&graphNode, graph, &graphNode2, bytes, deviceptr);

  // CUDA: hipError_t CUDAAPI hipGraphMemFreeNodeGetParams(hipGraphNode_t hNode, hipDeviceptr_t *dptr_out);
  // HIP: hipError_t hipGraphMemFreeNodeGetParams(hipGraphNode_t node, void* dev_ptr);
  // CHECK: result = hipGraphMemFreeNodeGetParams(graphNode, &deviceptr);
  result = hipGraphMemFreeNodeGetParams(graphNode, &deviceptr);
#endif

#if CUDA_VERSION >= 11040 && CUDA_VERSION < 12020
  // CHECK: hipMemAllocNodeParams MEM_ALLOC_NODE_PARAMS_st;
  hipMemAllocNodeParams MEM_ALLOC_NODE_PARAMS_st;
#endif

#if CUDA_VERSION >= 11060
  // CUDA: hipError_t CUDAAPI hipGraphNodeSetEnabled(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, unsigned int isEnabled);
  // HIP: hipError_t hipGraphNodeSetEnabled(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, unsigned int isEnabled);
  // CHECK: result = hipGraphNodeSetEnabled(graphExec, graphNode, flags);
  result = hipGraphNodeSetEnabled(graphExec, graphNode, flags);

  // CUDA: hipError_t CUDAAPI hipGraphNodeGetEnabled(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, unsigned int *isEnabled);
  // HIP: hipError_t hipGraphNodeGetEnabled(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, unsigned int* isEnabled);
  // CHECK: result = hipGraphNodeGetEnabled(graphExec, graphNode, &flags);
  result = hipGraphNodeGetEnabled(graphExec, graphNode, &flags);
#endif

#if CUDA_VERSION >= 11070
  // CUDA: hipError_t CUDAAPI hipStreamWaitValue32(hipStream_t stream, hipDeviceptr_t addr, cuuint32_t value, unsigned int flags);
  // HIP: hipError_t hipStreamWaitValue32(hipStream_t stream, void* ptr, uint32_t value, unsigned int flags, uint32_t mask __dparm(0xFFFFFFFF));
  // CHECK: result = hipStreamWaitValue32(stream, deviceptr, u_value, flags);
  result = hipStreamWaitValue32(stream, deviceptr, u_value, flags);

  // CUDA: hipError_t CUDAAPI hipStreamWaitValue64(hipStream_t stream, hipDeviceptr_t addr, cuuint64_t value, unsigned int flags);
  // HIP: hipError_t hipStreamWaitValue64(hipStream_t stream, void* ptr, uint64_t value, unsigned int flags, uint64_t mask __dparm(0xFFFFFFFFFFFFFFFF));
  // CHECK: result = hipStreamWaitValue64(stream, deviceptr, u_value, flags);
  result = hipStreamWaitValue64(stream, deviceptr, u_value, flags);

  // CUDA: hipError_t CUDAAPI hipStreamWriteValue32(hipStream_t stream, hipDeviceptr_t addr, cuuint32_t value, unsigned int flags);
  // HIP: hipError_t hipStreamWriteValue32(hipStream_t stream, void* ptr, uint32_t value, unsigned int flags, uint32_t mask __dparm(0xFFFFFFFF));
  // CHECK: result = hipStreamWriteValue32(stream, deviceptr, u_value, flags);
  result = hipStreamWriteValue32(stream, deviceptr, u_value, flags);

  // CUDA: hipError_t CUDAAPI hipStreamWriteValue64(hipStream_t stream, hipDeviceptr_t addr, cuuint64_t value, unsigned int flags);
  // HIP: hipError_t hipStreamWriteValue64(hipStream_t stream, void* ptr, uint64_t value, unsigned int flags, uint64_t mask __dparm(0xFFFFFFFFFFFFFFFF));
  // CHECK: result = hipStreamWriteValue64(stream, deviceptr, u_value, flags);
  result = hipStreamWriteValue64(stream, deviceptr, u_value, flags);
#endif

#if CUDA_VERSION >= 12000
  // TODO: https://github.com/ROCm-Developer-Tools/HIPIFY/issues/782 - Introduce 1-to-N conditional matcher
  //       Implement "conditional" matching in hipify-clang, based on CUDA_VERSION first;
  //       below the transformation hipStreamGetCaptureInfo -> hipStreamGetCaptureInfo_v2 should be applied for CUDA_VERSION >= 12000,
  //       otherwise, hipStreamGetCaptureInfo -> hipStreamGetCaptureInfo should be applied
  // CUDA < 12000: hipError_t CUDAAPI hipStreamGetCaptureInfo(hipStream_t hStream, hipStreamCaptureStatus *captureStatus_out, cuuint64_t *id_out);
  // CUDA:         hipError_t CUDAAPI hipStreamGetCaptureInfo(hipStream_t hStream, hipStreamCaptureStatus *captureStatus_out, cuuint64_t *id_out, hipGraph_t *graph_out, const hipGraphNode_t **dependencies_out, size_t *numDependencies_out);
  // HIP: hipError_t hipStreamGetCaptureInfo_v2(hipStream_t stream, hipStreamCaptureStatus* captureStatus_out, unsigned long long* id_out __dparm(0), hipGraph_t* graph_out __dparm(0), const hipGraphNode_t** dependencies_out __dparm(0), size_t* numDependencies_out __dparm(0));
  //
  result = hipStreamGetCaptureInfo(stream, &streamCaptureStatus, &ull, &graph, &pGraphNode, &bytes);

  // NOTE: not implemented yet in HIP
  // CUDA < 12000: hipError_t CUDAAPI hipGraphExecUpdate(hipGraphExec_t hGraphExec, hipGraph_t hGraph, hipGraphNode_t *hErrorNode_out, hipGraphExecUpdateResult *updateResult_out);
  // CUDA:         hipError_t CUDAAPI hipGraphExecUpdate(hipGraphExec_t hGraphExec, hipGraph_t hGraph, CUgraphExecUpdateResultInfo *resultInfo);
  // HIP:

  // NOTE: not implemented yet in HIP
  // CUDA < 12000: hipError_t CUDAAPI hipGraphInstantiate(hipGraphExec_t *phGraphExec, hipGraph_t hGraph, hipGraphNode_t *phErrorNode, char *logBuffer, size_t bufferSize);
  // CUDA:         hipError_t CUDAAPI hipGraphInstantiate(hipGraphExec_t *phGraphExec, hipGraph_t hGraph, unsigned long long flags);
  // HIP:
#endif

  return 0;
}
