// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

int main() {
  printf("07. CUDA Runtime API Typedefs synthetic test\n");

#if CUDA_VERSION >= 10000
  // CHECK: hipHostFn_t HostFn_t;
  hipHostFn_t HostFn_t;
#endif

  // CHECK: hipStreamCallback_t StreamCallback_t;
  // CHECK-NEXT: hipSurfaceObject_t SurfaceObject_t;
  // CHECK-NEXT: hipTextureObject_t TextureObject_t;
  hipStreamCallback_t StreamCallback_t;
  hipSurfaceObject_t SurfaceObject_t;
  hipTextureObject_t TextureObject_t;

  // CHECK: hipUUID uuid;
  hipUUID uuid;

  return 0;
}
