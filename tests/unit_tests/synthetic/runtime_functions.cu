// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>
#if defined(_WIN32)
  #include "windows.h"
  #include <GL/glew.h>
#endif
#include "cuda_gl_interop.h"

int main() {
  printf("12. CUDA Runtime API Functions synthetic test\n");

  size_t bytes = 0;
  size_t width = 0;
  size_t height = 0;
  size_t wOffset = 0;
  size_t hOffset = 0;
  size_t pitch = 0;
  size_t pitch_2 = 0;
  int device = 0;
  int deviceId = 0;
  int intVal = 0;
  int x = 0;
  int y = 0;
  int z = 0;
  int w = 0;
  unsigned int flags = 0;
  unsigned int levels = 0;
  unsigned int count = 0;
  float ms = 0;
  void* deviceptr = nullptr;
  void* deviceptr_2 = nullptr;
  void* image = nullptr;
  void* func = nullptr;
  char* ch = nullptr;
  const char* const_ch = nullptr;
  dim3 gridDim;
  dim3 blockDim;
  GLuint gl_uint = 0;
  GLenum gl_enum = 0;
  struct textureReference* texref = nullptr;

#if defined(_WIN32)
  unsigned long long ull = 0;
#else
  unsigned long ull = 0;
#endif
  unsigned long long ull_2 = 0;

  // CHECK: hipError_t result = hipSuccess;
  // CHECK-NEXT: hipError_t Error_t;
  // CHECK-NEXT: hipStream_t stream;
  hipError_t result = hipSuccess;
  hipError_t Error_t;
  hipStream_t stream;

#if CUDA_VERSION >= 8000
  // CHECK: hipDeviceP2PAttr DeviceP2PAttr;
  hipDeviceP2PAttr DeviceP2PAttr;

  // CHECK: hipMemoryAdvise MemoryAdvise;
  hipMemoryAdvise MemoryAdvise;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetP2PAttribute(int *value, enum hipDeviceP2PAttr attr, int srcDevice, int dstDevice);
  // HIP: hipError_t hipDeviceGetP2PAttribute(int* value, hipDeviceP2PAttr attr, int srcDevice, int dstDevice);
  // CHECK: result = hipDeviceGetP2PAttribute(&intVal, DeviceP2PAttr, device, deviceId);
  result = hipDeviceGetP2PAttribute(&intVal, DeviceP2PAttr, device, deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemAdvise(const void *devPtr, size_t count, enum hipMemoryAdvise advice, int device);
  // HIP: hipError_t hipMemAdvise(const void* dev_ptr, size_t count, hipMemoryAdvise advice, int device);
  // CHECK: result = hipMemAdvise(deviceptr, bytes, MemoryAdvise, device);
  result = hipMemAdvise(deviceptr, bytes, MemoryAdvise, device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPrefetchAsync(const void *devPtr, size_t count, int dstDevice, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemPrefetchAsync(const void* dev_ptr, size_t count, int device, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemPrefetchAsync(deviceptr, bytes, device, stream);
  result = hipMemPrefetchAsync(deviceptr, bytes, device, stream);

  // CHECK: hipMemRangeAttribute MemRangeAttribute;
  hipMemRangeAttribute MemRangeAttribute;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemRangeGetAttribute(void *data, size_t dataSize, enum hipMemRangeAttribute attribute, const void *devPtr, size_t count);
  // HIP: hipError_t hipMemRangeGetAttribute(void* data, size_t data_size, hipMemRangeAttribute attribute, const void* dev_ptr, size_t count);
  // CHECK: result = hipMemRangeGetAttribute(deviceptr, bytes, MemRangeAttribute, deviceptr_2, wOffset);
  result = hipMemRangeGetAttribute(deviceptr, bytes, MemRangeAttribute, deviceptr_2, wOffset);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemRangeGetAttributes(void **data, size_t *dataSizes, enum hipMemRangeAttribute *attributes, size_t numAttributes, const void *devPtr, size_t count);
  // HIP: hipError_t hipMemRangeGetAttributes(void** data, size_t* data_sizes, hipMemRangeAttribute* attributes, size_t num_attributes, const void* dev_ptr, size_t count);
  // CHECK: result = hipMemRangeGetAttributes(&deviceptr, &bytes, &MemRangeAttribute, wOffset, deviceptr_2, hOffset);
  result = hipMemRangeGetAttributes(&deviceptr, &bytes, &MemRangeAttribute, wOffset, deviceptr_2, hOffset);
#endif

#if CUDA_VERSION >= 9000
  // CHECK: hipFuncAttribute FuncAttribute;
  hipFuncAttribute FuncAttribute;

  // CHECK: hipLaunchParams LaunchParams;
  hipLaunchParams LaunchParams;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipFuncSetAttribute(reinterpret_cast<const void*>(const void *func), enum hipFuncAttribute attr, int value);
  // HIP: hipError_t hipFuncSetAttribute(reinterpret_cast<const void*>(const void* func), hipFuncAttribute attr, int value);
  // CHECK: result = hipFuncSetAttribute(reinterpret_cast<const void*>(func), FuncAttribute, intVal);
  result = hipFuncSetAttribute(reinterpret_cast<const void*>(func), FuncAttribute, intVal);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchCooperativeKernel(reinterpret_cast<const void*>(const void *func), dim3 gridDim, dim3 blockDim, void **args, size_t sharedMem, hipStream_t stream);
  // HIP: hipError_t hipLaunchCooperativeKernel(reinterpret_cast<const void*>(const void* f), dim3 gridDim, dim3 blockDimX, void** kernelParams, unsigned int sharedMemBytes, hipStream_t stream);
  // CHECK: result = hipLaunchCooperativeKernel(reinterpret_cast<const void*>(func), gridDim, blockDim, &image, flags, stream);
  result = hipLaunchCooperativeKernel(reinterpret_cast<const void*>(func), gridDim, blockDim, &image, flags, stream);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipLaunchCooperativeKernelMultiDevice(struct hipLaunchParams *launchParamsList, unsigned int numDevices, unsigned int flags  __dv(0));
  // HIP: hipError_t hipLaunchCooperativeKernelMultiDevice(hipLaunchParams* launchParamsList, int numDevices, unsigned int flags);
  // CHECK: result = hipLaunchCooperativeKernelMultiDevice(&LaunchParams, intVal, flags);
  result = hipLaunchCooperativeKernelMultiDevice(&LaunchParams, intVal, flags);
#endif

#if CUDA_VERSION >= 10000
  // CHECK: hipHostFn_t hostFn;
  hipHostFn_t hostFn;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void *userData);
  // HIP: hipError_t hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void* userData);
  // CHECK: result = hipLaunchHostFunc(stream, hostFn, image);
  result = hipLaunchHostFunc(stream, hostFn, image);

  // CHECK: hipStreamCaptureMode StreamCaptureMode;
  hipStreamCaptureMode StreamCaptureMode;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamBeginCapture(hipStream_t stream, enum hipStreamCaptureMode mode);
  // HIP: hipError_t hipStreamBeginCapture(hipStream_t stream, hipStreamCaptureMode mode);
  // CHECK: result = hipStreamBeginCapture(stream, StreamCaptureMode);
  result = hipStreamBeginCapture(stream, StreamCaptureMode);

  // CHECK: hipGraph_t Graph_t;
  hipGraph_t Graph_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamEndCapture(hipStream_t stream, hipGraph_t *pGraph);
  // HIP: hipError_t hipStreamEndCapture(hipStream_t stream, hipGraph_t* pGraph);
  // CHECK: result = hipStreamEndCapture(stream, &Graph_t);
  result = hipStreamEndCapture(stream, &Graph_t);

  // CHECK: hipStreamCaptureStatus StreamCaptureStatus;
  hipStreamCaptureStatus StreamCaptureStatus;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamIsCapturing(hipStream_t stream, enum hipStreamCaptureStatus *pCaptureStatus);
  // HIP: hipError_t hipStreamIsCapturing(hipStream_t stream, hipStreamCaptureStatus* pCaptureStatus);
  // CHECK: result = hipStreamIsCapturing(stream, &StreamCaptureStatus);
  result = hipStreamIsCapturing(stream, &StreamCaptureStatus);

  // CHECK: hipExternalMemory_t ExternalMemory_t;
  hipExternalMemory_t ExternalMemory_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDestroyExternalMemory(hipExternalMemory_t extMem);
  // HIP: hipError_t hipDestroyExternalMemory(hipExternalMemory_t extMem);
  // CHECK: result = hipDestroyExternalMemory(ExternalMemory_t);
  result = hipDestroyExternalMemory(ExternalMemory_t);

  // CHECK: hipExternalSemaphore_t ExternalSemaphore_t;
  hipExternalSemaphore_t ExternalSemaphore_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDestroyExternalSemaphore(hipExternalSemaphore_t extSem);
  // HIP: hipError_t hipDestroyExternalSemaphore(hipExternalSemaphore_t extSem);
  // CHECK: result = hipDestroyExternalSemaphore(ExternalSemaphore_t);
  result = hipDestroyExternalSemaphore(ExternalSemaphore_t);

  // CHECK: hipExternalMemoryBufferDesc ExternalMemoryBufferDesc;
  hipExternalMemoryBufferDesc ExternalMemoryBufferDesc;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipExternalMemoryGetMappedBuffer(void **devPtr, hipExternalMemory_t extMem, const struct hipExternalMemoryBufferDesc *bufferDesc);
  // HIP: hipError_t hipExternalMemoryGetMappedBuffer(void **devPtr, hipExternalMemory_t extMem, const hipExternalMemoryBufferDesc *bufferDesc);
  // CHECK: result = hipExternalMemoryGetMappedBuffer(&deviceptr, ExternalMemory_t, &ExternalMemoryBufferDesc);
  result = hipExternalMemoryGetMappedBuffer(&deviceptr, ExternalMemory_t, &ExternalMemoryBufferDesc);

  // CHECK: hipExternalMemoryHandleDesc ExternalMemoryHandleDesc;
  hipExternalMemoryHandleDesc ExternalMemoryHandleDesc;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipImportExternalMemory(hipExternalMemory_t *extMem_out, const struct hipExternalMemoryHandleDesc *memHandleDesc);
  // HIP: hipError_t hipImportExternalMemory(hipExternalMemory_t* extMem_out, const hipExternalMemoryHandleDesc* memHandleDesc);
  // CHECK: result = hipImportExternalMemory(&ExternalMemory_t, &ExternalMemoryHandleDesc);
  result = hipImportExternalMemory(&ExternalMemory_t, &ExternalMemoryHandleDesc);

  // CHECK: hipExternalSemaphoreHandleDesc ExternalSemaphoreHandleDesc;
  hipExternalSemaphoreHandleDesc ExternalSemaphoreHandleDesc;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipImportExternalSemaphore(hipExternalSemaphore_t *extSem_out, const struct hipExternalSemaphoreHandleDesc *semHandleDesc);
  // HIP: hipError_t hipImportExternalSemaphore(hipExternalSemaphore_t* extSem_out, const hipExternalSemaphoreHandleDesc* semHandleDesc);
  // CHECK: result = hipImportExternalSemaphore(&ExternalSemaphore_t, &ExternalSemaphoreHandleDesc);
  result = hipImportExternalSemaphore(&ExternalSemaphore_t, &ExternalSemaphoreHandleDesc);

  // CHECK: hipExternalSemaphoreSignalParams ExternalSemaphoreSignalParams;
  hipExternalSemaphoreSignalParams ExternalSemaphoreSignalParams;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipSignalExternalSemaphoresAsync(const hipExternalSemaphore_t *extSemArray, const struct hipExternalSemaphoreSignalParams *paramsArray, unsigned int numExtSems, hipStream_t stream __dv(0));
  // HIP: hipError_t hipSignalExternalSemaphoresAsync(const hipExternalSemaphore_t* extSemArray, const hipExternalSemaphoreSignalParams* paramsArray, unsigned int numExtSems, hipStream_t stream);
  // CHECK: result = hipSignalExternalSemaphoresAsync(&ExternalSemaphore_t, &ExternalSemaphoreSignalParams, flags, stream);
  result = hipSignalExternalSemaphoresAsync(&ExternalSemaphore_t, &ExternalSemaphoreSignalParams, flags, stream);

  // CHECK: hipExternalSemaphoreWaitParams ExternalSemaphoreWaitParams;
  hipExternalSemaphoreWaitParams ExternalSemaphoreWaitParams;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipWaitExternalSemaphoresAsync(const hipExternalSemaphore_t *extSemArray, const struct hipExternalSemaphoreWaitParams *paramsArray, unsigned int numExtSems, hipStream_t stream __dv(0));
  // HIP: hipError_t hipWaitExternalSemaphoresAsync(const hipExternalSemaphore_t* extSemArray, const hipExternalSemaphoreWaitParams* paramsArray, unsigned int numExtSems, hipStream_t stream);
  // CHECK: result = hipWaitExternalSemaphoresAsync(&ExternalSemaphore_t, &ExternalSemaphoreWaitParams, flags, stream);
  result = hipWaitExternalSemaphoresAsync(&ExternalSemaphore_t, &ExternalSemaphoreWaitParams, flags, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void *userData);
  // HIP: hipError_t hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void* userData);
  // CHECK: result = hipLaunchHostFunc(stream, hostFn, image);
  result = hipLaunchHostFunc(stream, hostFn, image);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipStreamCaptureMode streamCaptureMode;
  hipStreamCaptureMode streamCaptureMode;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipThreadExchangeStreamCaptureMode(enum hipStreamCaptureMode *mode);
  // HIP: hipError_t hipThreadExchangeStreamCaptureMode(hipStreamCaptureMode* mode);
  // CHECK: result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);
  result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamGetCaptureInfo(hipStream_t stream, enum hipStreamCaptureStatus *pCaptureStatus, unsigned long long *pId);
  // HIP: hipError_t hipStreamGetCaptureInfo(hipStream_t stream, hipStreamCaptureStatus* pCaptureStatus, unsigned long long* pId);
  // CHECK: result = hipStreamGetCaptureInfo(stream, &StreamCaptureStatus, &ull_2);
  result = hipStreamGetCaptureInfo(stream, &StreamCaptureStatus, &ull_2);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipThreadExchangeStreamCaptureMode(enum hipStreamCaptureMode *mode);
  // HIP: hipError_t hipThreadExchangeStreamCaptureMode(hipStreamCaptureMode* mode);
  // CHECK: result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);
  result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipKernelNodeAttrID kernelNodeAttrID;
  hipKernelNodeAttrID kernelNodeAttrID;
  // CHECK: hipKernelNodeAttrValue kernelNodeAttrValue;
  hipKernelNodeAttrValue kernelNodeAttrValue;
  // CHECK: hipGraphNode_t graphNode;
  hipGraphNode_t graphNode;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphKernelNodeSetAttribute(hipGraphNode_t hNode, enum hipKernelNodeAttrID attr, const union hipKernelNodeAttrValue* value);
  // HIP: hipError_t hipGraphKernelNodeSetAttribute(hipGraphNode_t hNode, hipKernelNodeAttrID attr, const hipKernelNodeAttrValue* value);
  // CHECK: result = hipGraphKernelNodeSetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);
  result = hipGraphKernelNodeSetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphKernelNodeGetAttribute(hipGraphNode_t hNode, enum hipKernelNodeAttrID attr, union hipKernelNodeAttrValue* value_out);
  // HIP: hipError_t hipGraphKernelNodeGetAttribute(hipGraphNode_t hNode, hipKernelNodeAttrID attr, hipKernelNodeAttrValue* value);
  // CHECK: result = hipGraphKernelNodeGetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);
  result = hipGraphKernelNodeGetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);
#endif

#if CUDA_VERSION >= 11020
  // CHECK: hipMemPoolAttr memPoolAttr;
  hipMemPoolAttr memPoolAttr;
  // CHECK: hipMemAccessDesc memAccessDesc;
  hipMemAccessDesc memAccessDesc;
  // CHECK: hipMemAccessFlags memAccessFlags;
  hipMemAccessFlags memAccessFlags;
  // CHECK: hipMemLocation memLocation;
  hipMemLocation memLocation;
  // CHECK: hipMemPoolProps memPoolProps;
  hipMemPoolProps memPoolProps;
  // CHECK: hipMemPool_t memPool_t;
  hipMemPool_t memPool_t;
  // CHECK: hipMemAllocationHandleType memAllocationHandleType;
  hipMemAllocationHandleType memAllocationHandleType;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetDefaultMemPool(hipMemPool_t *memPool, int device);
  // HIP: hipError_t hipDeviceGetDefaultMemPool(hipMemPool_t* mem_pool, int device);
  // CHECK: result = hipDeviceGetDefaultMemPool(&memPool_t, device);
  result = hipDeviceGetDefaultMemPool(&memPool_t, device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetMemPool(int device, hipMemPool_t memPool);
  // HIP: hipError_t hipDeviceSetMemPool(int device, hipMemPool_t mem_pool);
  // CHECK: result = hipDeviceSetMemPool(device, memPool_t);
  result = hipDeviceSetMemPool(device, memPool_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetMemPool(hipMemPool_t *memPool, int device);
  // HIP: hipError_t hipDeviceGetMemPool(hipMemPool_t* mem_pool, int device);
  // CHECK: result = hipDeviceGetMemPool(&memPool_t, device);
  result = hipDeviceGetMemPool(&memPool_t, device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocAsync(void **devPtr, size_t size, hipStream_t hStream);
  // HIP: hipError_t hipMallocAsync(void** dev_ptr, size_t size, hipStream_t stream);
  // CHECK: result = hipMallocAsync(&deviceptr, bytes, stream);
  result = hipMallocAsync(&deviceptr, bytes, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFreeAsync(void *devPtr, hipStream_t hStream);
  // HIP: hipError_t hipFreeAsync(void* dev_ptr, hipStream_t stream);
  // CHECK: result = hipFreeAsync(deviceptr, stream);
  result = hipFreeAsync(deviceptr, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolTrimTo(hipMemPool_t memPool, size_t minBytesToKeep);
  // HIP: hipError_t hipMemPoolTrimTo(hipMemPool_t mem_pool, size_t min_bytes_to_hold);
  // CHECK: result = hipMemPoolTrimTo(memPool_t, bytes);
  result = hipMemPoolTrimTo(memPool_t, bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolSetAttribute(hipMemPool_t memPool, enum hipMemPoolAttr attr, void *value );
  // HIP: hipError_t hipMemPoolSetAttribute(hipMemPool_t mem_pool, hipMemPoolAttr attr, void* value);
  // CHECK: result = hipMemPoolSetAttribute(memPool_t, memPoolAttr, image);
  result = hipMemPoolSetAttribute(memPool_t, memPoolAttr, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolGetAttribute(hipMemPool_t memPool, enum hipMemPoolAttr attr, void *value );
  // HIP: hipError_t hipMemPoolGetAttribute(hipMemPool_t mem_pool, hipMemPoolAttr attr, void* value);
  // CHECK: result = hipMemPoolGetAttribute(memPool_t, memPoolAttr, image);
  result = hipMemPoolGetAttribute(memPool_t, memPoolAttr, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolSetAccess(hipMemPool_t memPool, const struct hipMemAccessDesc *descList, size_t count);
  // HIP: hipError_t hipMemPoolSetAccess(hipMemPool_t mem_pool, const hipMemAccessDesc* desc_list, size_t count);
  // CHECK: result = hipMemPoolSetAccess(memPool_t, &memAccessDesc, bytes);
  result = hipMemPoolSetAccess(memPool_t, &memAccessDesc, bytes);

  // CUDA: hipError_t extern __host__ hipError_t CUDARTAPI hipMemPoolGetAccess(enum hipMemAccessFlags *flags, hipMemPool_t memPool, struct hipMemLocation *location);
  // HIP: hipError_t hipMemPoolGetAccess(hipMemAccessFlags* flags, hipMemPool_t mem_pool, hipMemLocation* location);
  // CHECK: result = hipMemPoolGetAccess(&memAccessFlags, memPool_t, &memLocation);
  result = hipMemPoolGetAccess(&memAccessFlags, memPool_t, &memLocation);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolCreate(hipMemPool_t *memPool, const struct hipMemPoolProps *poolProps);
  // HIP: hipError_t hipMemPoolCreate(hipMemPool_t* mem_pool, const hipMemPoolProps* pool_props);
  // CHECK: result = hipMemPoolCreate(&memPool_t, &memPoolProps);
  result = hipMemPoolCreate(&memPool_t, &memPoolProps);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolDestroy(hipMemPool_t memPool);
  // HIP: hipError_t hipMemPoolDestroy(hipMemPool_t mem_pool);
  // CHECK: result = hipMemPoolDestroy(memPool_t);
  result = hipMemPoolDestroy(memPool_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocFromPoolAsync(void **ptr, size_t size, hipMemPool_t memPool, hipStream_t stream);
  // HIP: hipError_t hipMallocFromPoolAsync(void** dev_ptr, size_t size, hipMemPool_t mem_pool, hipStream_t stream);
  // CHECK: result = hipMallocFromPoolAsync(&deviceptr, bytes, memPool_t, stream);
  result = hipMallocFromPoolAsync(&deviceptr, bytes, memPool_t, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolExportToShareableHandle(void* shareableHandle, hipMemPool_t memPool, enum hipMemAllocationHandleType handleType, unsigned int flags);
  // HIP: hipError_t hipMemPoolExportToShareableHandle(void* shared_handle, hipMemPool_t mem_pool, hipMemAllocationHandleType handle_type, unsigned int flags);
  // CHECK: result = hipMemPoolExportToShareableHandle(image, memPool_t, memAllocationHandleType, ull);
  result = hipMemPoolExportToShareableHandle(image, memPool_t, memAllocationHandleType, ull);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolImportFromShareableHandle(hipMemPool_t* memPool, void* shareableHandle, enum hipMemAllocationHandleType handleType, unsigned int flags);
  // HIP: hipError_t hipMemPoolImportFromShareableHandle(hipMemPool_t* mem_pool, void* shared_handle, hipMemAllocationHandleType handle_type, unsigned int flags);
  // CHECK: result = hipMemPoolImportFromShareableHandle(&memPool_t, image, memAllocationHandleType, ull);
  result = hipMemPoolImportFromShareableHandle(&memPool_t, image, memAllocationHandleType, ull);

  // CHECK: hipMemPoolPtrExportData memPoolPtrExportData;
  hipMemPoolPtrExportData memPoolPtrExportData;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolExportPointer(struct hipMemPoolPtrExportData *exportData, void *ptr);
  // HIP: hipError_t hipMemPoolExportPointer(hipMemPoolPtrExportData* export_data, void* dev_ptr);
  // CHECK: result = hipMemPoolExportPointer(&memPoolPtrExportData, deviceptr);
  result = hipMemPoolExportPointer(&memPoolPtrExportData, deviceptr);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolImportPointer(void **ptr, hipMemPool_t memPool, struct hipMemPoolPtrExportData *exportData);
  // HIP: hipError_t hipMemPoolImportPointer(void** dev_ptr, hipMemPool_t mem_pool, hipMemPoolPtrExportData* export_data);
  // CHECK: result = hipMemPoolImportPointer(&deviceptr, memPool_t, &memPoolPtrExportData);
  result = hipMemPoolImportPointer(&deviceptr, memPool_t, &memPoolPtrExportData);
#endif

  // CHECK: hipDeviceProp_t DeviceProp;
  hipDeviceProp_t DeviceProp;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipChooseDevice(int *device, const struct hipDeviceProp_t *prop);
  // HIP: hipError_t hipChooseDevice(int* device, const hipDeviceProp_t* prop);
  // CHECK: result = hipChooseDevice(&device, &DeviceProp);
  result = hipChooseDevice(&device, &DeviceProp);

  // CHECK: hipDeviceAttribute_t DeviceAttr;
  hipDeviceAttribute_t DeviceAttr;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetAttribute(int *value, enum hipDeviceAttribute_t attr, int device);
  // HIP: hipError_t hipDeviceGetAttribute(int* pi, hipDeviceAttribute_t attr, int deviceId);
  // CHECK: result = hipDeviceGetAttribute(&device, DeviceAttr, deviceId);
  result = hipDeviceGetAttribute(&device, DeviceAttr, deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetByPCIBusId(int *device, const char *pciBusId);
  // HIP: hipError_t hipDeviceGetByPCIBusId(int* device, const char* pciBusId);
  // CHECK: result = hipDeviceGetByPCIBusId(&device, ch);
  result = hipDeviceGetByPCIBusId(&device, ch);

  // CHECK: hipFuncCache_t FuncCache;
  hipFuncCache_t FuncCache;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetCacheConfig(enum hipFuncCache_t *pCacheConfig);
  // HIP: hipError_t hipDeviceGetCacheConfig(hipFuncCache_t* cacheConfig);
  // CHECK: result = hipDeviceGetCacheConfig(&FuncCache);
  result = hipDeviceGetCacheConfig(&FuncCache);

  // CHECK: hipLimit_t Limit;
  hipLimit_t Limit;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetLimit(size_t *pValue, enum hipLimit_t limit);
  // HIP: hipError_t hipDeviceGetLimit(size_t* pValue, enum hipLimit_t limit);
  // CHECK: result = hipDeviceGetLimit(&bytes, Limit);
  result = hipDeviceGetLimit(&bytes, Limit);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetPCIBusId(char *pciBusId, int len, int device);
  // HIP: hipError_t hipError_t hipDeviceGetPCIBusId(char* pciBusId, int len, int device);
  // CHECK: result = hipDeviceGetPCIBusId(ch, intVal, device);
  result = hipDeviceGetPCIBusId(ch, intVal, device);

  // CHECK: hipSharedMemConfig SharedMemConfig;
  hipSharedMemConfig SharedMemConfig;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetSharedMemConfig(enum hipSharedMemConfig *pConfig);
  // HIP: hipError_t hipDeviceGetSharedMemConfig(hipSharedMemConfig* pConfig);
  // CHECK: result = hipDeviceGetSharedMemConfig(&SharedMemConfig);
  result = hipDeviceGetSharedMemConfig(&SharedMemConfig);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetStreamPriorityRange(int *leastPriority, int *greatestPriority);
  // HIP: hipError_t hipDeviceGetStreamPriorityRange(int* leastPriority, int* greatestPriority);
  // CHECK: result = hipDeviceGetStreamPriorityRange(&deviceId, &intVal);
  result = hipDeviceGetStreamPriorityRange(&deviceId, &intVal);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceReset(void);
  // HIP: hipError_t hipError_t hipDeviceReset(void);
  // CHECK: result = hipDeviceReset();
  result = hipDeviceReset();

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetCacheConfig(enum hipFuncCache_t cacheConfig);
  // HIP: hipError_t hipDeviceSetCacheConfig(hipFuncCache_t cacheConfig);
  // CHECK: result = hipDeviceSetCacheConfig(FuncCache);
  result = hipDeviceSetCacheConfig(FuncCache);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetSharedMemConfig(enum hipSharedMemConfig config);
  // HIP: hipError_t hipDeviceSetSharedMemConfig(hipSharedMemConfig config);
  // CHECK: result = hipDeviceSetSharedMemConfig(SharedMemConfig);
  result = hipDeviceSetSharedMemConfig(SharedMemConfig);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceSynchronize(void);
  // HIP: hipError_t hipDeviceSynchronize(void);
  // CHECK: result = hipDeviceSynchronize();
  result = hipDeviceSynchronize();

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipGetDevice(int *device);
  // HIP: hipError_t hipGetDevice(int* deviceId);
  // CHECK: result = hipGetDevice(&deviceId);
  result = hipGetDevice(&deviceId);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipGetDeviceCount(int *count);
  // HIP: hipError_t hipGetDeviceCount(int* count);
  // CHECK: result = hipGetDeviceCount(&deviceId);
  result = hipGetDeviceCount(&deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetDeviceFlags( unsigned int *flags );
  // HIP: hipError_t hipGetDeviceFlags(unsigned int* flags);
  // CHECK: result = hipGetDeviceFlags(&flags);
  result = hipGetDeviceFlags(&flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipGetDeviceProperties(struct hipDeviceProp_t *prop, int device);
  // HIP: hipError_t hipGetDeviceProperties(hipDeviceProp_t* prop, int deviceId);
  // CHECK: result = hipGetDeviceProperties(&DeviceProp, deviceId);
  result = hipGetDeviceProperties(&DeviceProp, deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcCloseMemHandle(void *devPtr);
  // HIP: hipError_t hipError_t hipIpcCloseMemHandle(void* devPtr);
  // CHECK: result = hipIpcCloseMemHandle(deviceptr);
  result = hipIpcCloseMemHandle(deviceptr);

  // CHECK: hipIpcEventHandle_t IpcEventHandle_t;
  hipIpcEventHandle_t IpcEventHandle_t;

  // CHECK: hipEvent_t Event_t;
  // CHECK-Next: hipEvent_t Event_2;
  hipEvent_t Event_t;
  hipEvent_t Event_2;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcGetEventHandle(hipIpcEventHandle_t *handle, hipEvent_t event);
  // HIP: hipError_t hipIpcGetEventHandle(hipIpcEventHandle_t* handle, hipEvent_t event);
  // CHECK: result = hipIpcGetEventHandle(&IpcEventHandle_t, Event_t);
  result = hipIpcGetEventHandle(&IpcEventHandle_t, Event_t);

  // CHECK: hipIpcMemHandle_t IpcMemHandle_t;
  hipIpcMemHandle_t IpcMemHandle_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcGetMemHandle(hipIpcMemHandle_t *handle, void *devPtr);
  // HIP: hipError_t hipIpcGetMemHandle(hipIpcMemHandle_t* handle, void* devPtr);
  // CHECK: result = hipIpcGetMemHandle(&IpcMemHandle_t, deviceptr);
  result = hipIpcGetMemHandle(&IpcMemHandle_t, deviceptr);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcOpenEventHandle(hipEvent_t *event, hipIpcEventHandle_t handle);
  // HIP: hipError_t hipIpcOpenEventHandle(hipEvent_t* event, hipIpcEventHandle_t handle);
  // CHECK: result = hipIpcOpenEventHandle(&Event_t, IpcEventHandle_t);
  result = hipIpcOpenEventHandle(&Event_t, IpcEventHandle_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcOpenMemHandle(void **devPtr, hipIpcMemHandle_t handle, unsigned int flags);
  // HIP: hipError_t hipIpcOpenMemHandle(void** devPtr, hipIpcMemHandle_t handle, unsigned int flags);
  // CHECK: result = hipIpcOpenMemHandle(&deviceptr, IpcMemHandle_t, flags);
  result = hipIpcOpenMemHandle(&deviceptr, IpcMemHandle_t, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipSetDevice(int device);
  // HIP: hipError_t hipSetDevice(int deviceId);
  // CHECK: result = hipSetDevice(deviceId);
  result = hipSetDevice(deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipSetDeviceFlags( unsigned int flags );
  // HIP: hipError_t hipSetDeviceFlags(unsigned flags);
  // CHECK: result = hipSetDeviceFlags(flags);
  result = hipSetDeviceFlags(flags);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipDeviceReset(void);
  // HIP: hipError_t hipDeviceReset(void);
  // CHECK: result = hipDeviceReset();
  result = hipDeviceReset();

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipDeviceGetCacheConfig(enum hipFuncCache_t *pCacheConfig);
  // HIP: hipError_t hipDeviceGetCacheConfig(hipFuncCache_t* cacheConfig);
  // CHECK: result = hipDeviceGetCacheConfig(&FuncCache);
  result = hipDeviceGetCacheConfig(&FuncCache);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipDeviceSetCacheConfig(enum hipFuncCache_t cacheConfig);
  // HIP: hipError_t hipError_t hipDeviceSetCacheConfig(hipFuncCache_t cacheConfig);
  // CHECK: result = hipDeviceSetCacheConfig(FuncCache);
  result = hipDeviceSetCacheConfig(FuncCache);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipDeviceSynchronize(void);
  // HIP: hipError_t hipError_t hipDeviceSynchronize(void);
  // CHECK: result = hipDeviceSynchronize();
  result = hipDeviceSynchronize();

  // CUDA: extern __host__ __cudart_builtin__ const char* CUDARTAPI hipGetErrorName(hipError_t error);
  // HIP: const char* hipGetErrorName(hipError_t hip_error);
  // CHECK: const_ch = hipGetErrorName(Error_t);
  const_ch = hipGetErrorName(Error_t);

  // CUDA: extern __host__ __cudart_builtin__ const char* CUDARTAPI hipGetErrorString(hipError_t error);
  // HIP: const char* hipGetErrorString(hipError_t hipError);
  // CHECK: const_ch = hipGetErrorString(Error_t);
  const_ch = hipGetErrorString(Error_t);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipGetLastError(void);
  // HIP: hipError_t hipGetLastError(void);
  // CHECK: result = hipGetLastError();
  result = hipGetLastError();

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipPeekAtLastError(void);
  // HIP: hipError_t hipPeekAtLastError(void);
  // CHECK: result = hipPeekAtLastError();
  result = hipPeekAtLastError();

  // CHECK: hipStreamCallback_t StreamCallback_t;
  hipStreamCallback_t StreamCallback_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamAddCallback(hipStream_t stream, hipStreamCallback_t callback, void* userData, unsigned int flags);
  // HIP: hipError_t hipStreamAddCallback(hipStream_t stream, hipStreamCallback_t callback, void* userData, unsigned int flags);
  // CHECK: result = hipStreamAddCallback(stream, StreamCallback_t, image, flags);
  result = hipStreamAddCallback(stream, StreamCallback_t, image, flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamAttachMemAsync(hipStream_t stream, void *devPtr, size_t length __dv(0), unsigned int flags = hipMemAttachSingle);
  // HIP: hipError_t hipStreamAttachMemAsync(hipStream_t stream, void* dev_ptr, size_t length __dparm(0), unsigned int flags __dparm(hipMemAttachSingle));
  // CHECK: result = hipStreamAttachMemAsync(stream, deviceptr, bytes, flags);
  result = hipStreamAttachMemAsync(stream, deviceptr, bytes, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamCreate(hipStream_t *pStream);
  // HIP: hipError_t hipStreamCreate(hipStream_t* stream);
  // CHECK: result = hipStreamCreate(&stream);
  result = hipStreamCreate(&stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamCreateWithFlags(hipStream_t *pStream, unsigned int flags);
  // HIP: hipError_t hipStreamCreateWithFlags(hipStream_t* stream, unsigned int flags);
  // CHECK: result = hipStreamCreateWithFlags(&stream, flags);
  result = hipStreamCreateWithFlags(&stream, flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamCreateWithPriority(hipStream_t *pStream, unsigned int flags, int priority);
  // HIP: hipError_t hipStreamCreateWithPriority(hipStream_t* stream, unsigned int flags);
  // CHECK: result = hipStreamCreateWithPriority(&stream, flags, intVal);
  result = hipStreamCreateWithPriority(&stream, flags, intVal);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamDestroy(hipStream_t stream);
  // HIP: hipError_t hipStreamDestroy(hipStream_t stream);
  // CHECK: result = hipStreamDestroy(stream);
  result = hipStreamDestroy(stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamGetFlags(hipStream_t hStream, unsigned int *flags);
  // HIP: hipError_t hipStreamGetFlags(hipStream_t stream, unsigned int* flags);
  // CHECK: result = hipStreamGetFlags(stream, &flags);
  result = hipStreamGetFlags(stream, &flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamGetPriority(hipStream_t hStream, int *priority);
  // HIP: hipError_t hipStreamGetPriority(hipStream_t stream, int* priority);
  // CHECK: result = hipStreamGetPriority(stream, &intVal);
  result = hipStreamGetPriority(stream, &intVal);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamQuery(hipStream_t stream);
  // HIP: hipError_t hipStreamQuery(hipStream_t stream);
  // CHECK: result = hipStreamQuery(stream);
  result = hipStreamQuery(stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamSynchronize(hipStream_t stream);
  // HIP: hipError_t hipStreamSynchronize(hipStream_t stream);
  // CHECK: result = hipStreamSynchronize(stream);
  result = hipStreamSynchronize(stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamWaitEvent(hipStream_t stream, hipEvent_t event, unsigned int flags __dv(0));
  // HIP: hipError_t hipStreamWaitEvent(hipStream_t stream, hipEvent_t event, unsigned int flags);
  // CHECK: result = hipStreamWaitEvent(stream, Event_t, flags);
  result = hipStreamWaitEvent(stream, Event_t, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipEventCreate(hipEvent_t *event);
  // HIP: hipError_t hipEventCreate(hipEvent_t* event);
  // CHECK: result = hipEventCreate(&Event_t);
  result = hipEventCreate(&Event_t);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipEventCreateWithFlags(hipEvent_t *event, unsigned int flags);
  // HIP: hipError_t hipEventCreateWithFlags(hipEvent_t* event, unsigned flags);
  // CHECK: result = hipEventCreateWithFlags(&Event_t, flags);
  result = hipEventCreateWithFlags(&Event_t, flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipEventDestroy(hipEvent_t event);
  // HIP: hipError_t hipEventDestroy(hipEvent_t event);
  // CHECK: result = hipEventDestroy(Event_t);
  result = hipEventDestroy(Event_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipEventElapsedTime(float *ms, hipEvent_t start, hipEvent_t end);
  // HIP: hipError_t hipEventElapsedTime(float* ms, hipEvent_t start, hipEvent_t stop);
  // CHECK: result = hipEventElapsedTime(&ms, Event_t, Event_2);
  result = hipEventElapsedTime(&ms, Event_t, Event_2);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipEventQuery(hipEvent_t event);
  // HIP: hipError_t hipEventQuery(hipEvent_t event);
  // CHECK: result = hipEventQuery(Event_t);
  result = hipEventQuery(Event_t);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipEventRecord(hipEvent_t event, hipStream_t stream __dv(0));
  // HIP: hipError_t hipEventRecord(hipEvent_t event, hipStream_t stream);
  // CHECK: result = hipEventRecord(Event_t, stream);
  result = hipEventRecord(Event_t, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipEventSynchronize(hipEvent_t event);
  // HIP: hipError_t hipEventSynchronize(hipEvent_t event);
  // CHECK: result = hipEventSynchronize(Event_t);
  result = hipEventSynchronize(Event_t);

  // CHECK: hipFuncAttributes FuncAttributes;
  hipFuncAttributes FuncAttributes;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipFuncGetAttributes(struct hipFuncAttributes *attr, reinterpret_cast<const void*>(const void *func));
  // HIP: hipError_t hipFuncGetAttributes(struct hipFuncAttributes* attr, reinterpret_cast<const void*>(const void* func));
  // CHECK: result = hipFuncGetAttributes(&FuncAttributes, reinterpret_cast<const void*>(func));
  result = hipFuncGetAttributes(&FuncAttributes, reinterpret_cast<const void*>(func));

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFuncSetCacheConfig(reinterpret_cast<const void*>(const void *func), enum hipFuncCache_t cacheConfig);
  // HIP: hipError_t hipFuncSetCacheConfig(reinterpret_cast<const void*>(const void* func), hipFuncCache_t config);
  // CHECK: result = hipFuncSetCacheConfig(reinterpret_cast<const void*>(func), FuncCache);
  result = hipFuncSetCacheConfig(reinterpret_cast<const void*>(func), FuncCache);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(const void *func), enum hipSharedMemConfig config);
  // HIP: hipError_t hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(const void* func), hipSharedMemConfig config);
  // CHECK: result = hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(func), SharedMemConfig);
  result = hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(func), SharedMemConfig);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchKernel(reinterpret_cast<const void*>(const void *func), dim3 gridDim, dim3 blockDim, void **args, size_t sharedMem, hipStream_t stream);
  // HIP: hipError_t hipLaunchKernel(reinterpret_cast<const void*>(const void* function_address), dim3 numBlocks, dim3 dimBlocks, void** args, size_t sharedMemBytes __dparm(0), hipStream_t stream __dparm(0));
  // CHECK: result = hipLaunchKernel(reinterpret_cast<const void*>(func), gridDim, blockDim, &image, bytes, stream);
  result = hipLaunchKernel(reinterpret_cast<const void*>(func), gridDim, blockDim, &image, bytes, stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipOccupancyMaxActiveBlocksPerMultiprocessor(int *numBlocks, const void *func, int blockSize, size_t dynamicSMemSize);
  // HIP: hipError_t hipOccupancyMaxActiveBlocksPerMultiprocessor(int* numBlocks, const void* f, int blockSize, size_t dynSharedMemPerBlk);
  // CHECK: result = hipOccupancyMaxActiveBlocksPerMultiprocessor(&intVal, func, device, bytes);
  result = hipOccupancyMaxActiveBlocksPerMultiprocessor(&intVal, func, device, bytes);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(int *numBlocks, const void *func, int blockSize, size_t dynamicSMemSize, unsigned int flags);
  // HIP: hipError_t hipOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(int* numBlocks, const void* f, int blockSize, size_t dynSharedMemPerBlk, unsigned int flags __dparm(hipOccupancyDefault));
  // CHECK: result = hipOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(&intVal, func, intVal, bytes, flags);
  result = hipOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(&intVal, func, intVal, bytes, flags);

  // CUDA: template<class T> static __inline__ __host__ CUDART_DEVICE hipError_t hipOccupancyMaxPotentialBlockSize(int* minGridSize, int* blockSize, T func, size_t dynamicSMemSize = 0, int blockSizeLimit = 0);
  // HIP: template <typename T> static hipError_t __host__ inline hipOccupancyMaxPotentialBlockSize(int* gridSize, int* blockSize, T f, size_t dynSharedMemPerBlk = 0, int blockSizeLimit = 0);
  // CHECK: result = hipOccupancyMaxPotentialBlockSize(&intVal, &device, func, bytes, deviceId);
  result = hipOccupancyMaxPotentialBlockSize(&intVal, &device, func, bytes, deviceId);

  // CUDA: template<class T> static __inline__ __host__ CUDART_DEVICE hipError_t hipOccupancyMaxPotentialBlockSizeWithFlags(int* minGridSize, int* blockSize, T func, size_t dynamicSMemSize = 0, int blockSizeLimit = 0, unsigned int flags = 0);
  // HIP: template <typename T> static hipError_t __host__ inline hipOccupancyMaxPotentialBlockSizeWithFlags(int* gridSize, int* blockSize, T f, size_t dynSharedMemPerBlk = 0, int blockSizeLimit = 0, unsigned int  flags = 0);
  // CHECK: result = hipOccupancyMaxPotentialBlockSizeWithFlags(&intVal, &device, func, bytes, deviceId, flags);
  result = hipOccupancyMaxPotentialBlockSizeWithFlags(&intVal, &device, func, bytes, deviceId, flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipFree(void *devPtr);
  // HIP: hipError_t hipFree(void* ptr);
  // CHECK: result = hipFree(deviceptr);
  result = hipFree(deviceptr);

  // CHECK: hipArray* Array;
  // CHECK-NEXT: hipArray_t Array_t;
  // CHECK-NEXT: hipArray_const_t Array_const_t;
  hipArray* Array;
  hipArray_t Array_t;
  hipArray_const_t Array_const_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFreeArray(hipArray_t array);
  // HIP: hipError_t hipFreeArray(hipArray* array);
  // CHECK: result = hipFreeArray(Array_t);
  result = hipFreeArray(Array_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipHostFree(void *ptr);
  // HIP: hipError_t hipHostFree(void* ptr);
  // CHECK: result = hipHostFree(deviceptr);
  result = hipHostFree(deviceptr);

  // CHECK: hipMipmappedArray* MipmappedArray;
  // CHECK-NEXT: hipMipmappedArray_t MipmappedArray_t;
  // CHECK-NEXT: hipMipmappedArray_const_t MipmappedArray_const_t;
  hipMipmappedArray* MipmappedArray;
  hipMipmappedArray_t MipmappedArray_t;
  hipMipmappedArray_const_t MipmappedArray_const_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFreeMipmappedArray(hipMipmappedArray_t mipmappedArray);
  // HIP: hipError_t hipFreeMipmappedArray(hipMipmappedArray_t mipmappedArray);
  // CHECK: result = hipFreeMipmappedArray(MipmappedArray_t);
  result = hipFreeMipmappedArray(MipmappedArray_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetMipmappedArrayLevel(hipArray_t *levelArray, hipMipmappedArray_const_t mipmappedArray, unsigned int level);
  // HIP: hipError_t hipGetMipmappedArrayLevel(hipArray_t* levelArray, hipMipmappedArray_const_t mipmappedArray, unsigned int level);
  // CHECK: result = hipGetMipmappedArrayLevel(&Array_t, MipmappedArray_const_t, flags);
  result = hipGetMipmappedArrayLevel(&Array_t, MipmappedArray_const_t, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetSymbolAddress(void **devPtr, HIP_SYMBOL(const void *symbol));
  // HIP: hipError_t hipGetSymbolAddress(void** devPtr, HIP_SYMBOL(const void* symbol));
  // CHECK: result = hipGetSymbolAddress(&deviceptr, HIP_SYMBOL(HIP_SYMBOL(image)));
  result = hipGetSymbolAddress(&deviceptr, HIP_SYMBOL(image));

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetSymbolSize(size_t *size, HIP_SYMBOL(const void *symbol));
  // HIP: hipError_t hipGetSymbolSize(size_t* size, HIP_SYMBOL(const void* symbol));
  // CHECK: result = hipGetSymbolSize(&bytes, HIP_SYMBOL(HIP_SYMBOL(image)));
  result = hipGetSymbolSize(&bytes, HIP_SYMBOL(image));

  // CUDA: extern __host__ hipError_t CUDARTAPI hipHostAlloc(void **pHost, size_t size, unsigned int flags);
  // HIP: DEPRECATED("use hipHostMalloc instead") hipError_t hipHostAlloc(void** ptr, size_t size, unsigned int flags);
  // CHECK: result = hipHostAlloc(&deviceptr, bytes, flags);
  result = hipHostAlloc(&deviceptr, bytes, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipHostGetDevicePointer(void **pDevice, void *pHost, unsigned int flags);
  // HIP: hipError_t hipHostGetDevicePointer(void** devPtr, void* hstPtr, unsigned int flags);
  // CHECK: result = hipHostGetDevicePointer(&deviceptr, image, flags);
  result = hipHostGetDevicePointer(&deviceptr, image, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipHostGetFlags(unsigned int *pFlags, void *pHost);
  // HIP: hipError_t hipHostGetFlags(unsigned int* flagsPtr, void* hostPtr);
  // CHECK: result = hipHostGetFlags(&flags, image);
  result = hipHostGetFlags(&flags, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipHostRegister(void *ptr, size_t size, unsigned int flags);
  // HIP: hipError_t hipHostRegister(void* hostPtr, size_t sizeBytes, unsigned int flags);
  // CHECK: result = hipHostRegister(image, bytes, flags);
  result = hipHostRegister(image, bytes, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipHostUnregister(void *ptr);
  // HIP: hipError_t hipHostUnregister(void* hostPtr);
  // CHECK: result = hipHostUnregister(image);
  result = hipHostUnregister(image);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMalloc(void **devPtr, size_t size);
  // HIP: hipError_t hipMalloc(void** ptr, size_t size);
  // CHECK: result = hipMalloc(&deviceptr, bytes);
  result = hipMalloc(&deviceptr, bytes);

  // CHECK: hipPitchedPtr PitchedPtr;
  hipPitchedPtr PitchedPtr;

  // CHECK: hipExtent Extent;
  hipExtent Extent;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMalloc3D(struct hipPitchedPtr* pitchedDevPtr, struct hipExtent extent);
  // HIP: hipError_t hipMalloc3D(hipPitchedPtr* pitchedDevPtr, hipExtent extent);
  // CHECK: result = hipMalloc3D(&PitchedPtr, Extent);
  result = hipMalloc3D(&PitchedPtr, Extent);

  // CHECK: hipChannelFormatDesc ChannelFormatDesc;
  hipChannelFormatDesc ChannelFormatDesc;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMalloc3DArray(hipArray_t *array, const struct hipChannelFormatDesc* desc, struct hipExtent extent, unsigned int flags __dv(0));
  // HIP: hipError_t hipMalloc3DArray(hipArray** array, const struct hipChannelFormatDesc* desc, struct hipExtent extent, unsigned int flags);
  // CHECK: result = hipMalloc3DArray(&Array_t, &ChannelFormatDesc, Extent, flags);
  result = hipMalloc3DArray(&Array_t, &ChannelFormatDesc, Extent, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocArray(hipArray_t *array, const struct hipChannelFormatDesc *desc, size_t width, size_t height __dv(0), unsigned int flags __dv(0));
  // HIP: hipError_t hipMallocArray(hipArray** array, const hipChannelFormatDesc* desc, size_t width, size_t height __dparm(0), unsigned int flags __dparm(hipArrayDefault));
  // CHECK: result = hipMallocArray(&Array_t, &ChannelFormatDesc, width, height, flags);
  result = hipMallocArray(&Array_t, &ChannelFormatDesc, width, height, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipHostMalloc(void **ptr, size_t size);
  // HIP: hipError_t hipHostMalloc(void** ptr, size_t size, unsigned int flags);
  // CHECK: result = hipHostMalloc(&deviceptr, bytes);
  result = hipHostMalloc(&deviceptr, bytes);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMallocManaged(void **devPtr, size_t size, unsigned int flags = hipMemAttachGlobal);
  // HIP: hipError_t hipMallocManaged(void** dev_ptr, size_t size, unsigned int flags __dparm(hipMemAttachGlobal));
  // CHECK: result = hipMallocManaged(&deviceptr, bytes, flags);
  result = hipMallocManaged(&deviceptr, bytes, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocMipmappedArray(hipMipmappedArray_t *mipmappedArray, const struct hipChannelFormatDesc* desc, struct hipExtent extent, unsigned int numLevels, unsigned int flags __dv(0));
  // HIP: hipError_t hipMallocMipmappedArray(hipMipmappedArray_t* mipmappedArray, const struct hipChannelFormatDesc* desc, struct hipExtent extent, unsigned int numLevels, unsigned int flags __dparm(0));
  // CHECK: result = hipMallocMipmappedArray(&MipmappedArray_t, &ChannelFormatDesc, Extent, levels, flags);
  result = hipMallocMipmappedArray(&MipmappedArray_t, &ChannelFormatDesc, Extent, levels, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocPitch(void **devPtr, size_t *pitch, size_t width, size_t height);
  // HIP: hipError_t hipMallocPitch(void** ptr, size_t* pitch, size_t width, size_t height);
  // CHECK: result = hipMallocPitch(&deviceptr, &bytes, width, height);
  result = hipMallocPitch(&deviceptr, &bytes, width, height);

  // CHECK: hipMemcpyKind MemcpyKind;
  hipMemcpyKind MemcpyKind;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpy(void *dst, const void *src, size_t count, enum hipMemcpyKind kind);
  // HIP: hipError_t hipMemcpy(void* dst, const void* src, size_t sizeBytes, hipMemcpyKind kind);
  // CHECK: result = hipMemcpy(deviceptr, deviceptr_2, bytes, MemcpyKind);
  result = hipMemcpy(deviceptr, deviceptr_2, bytes, MemcpyKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpy2D(void *dst, size_t dpitch, const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind);
  // HIP: hipError_t hipMemcpy2D(void* dst, size_t dpitch, const void* src, size_t spitch, size_t width, size_t height, hipMemcpyKind kind);
  // CHECK: result = hipMemcpy2D(deviceptr, pitch, deviceptr_2, pitch_2, width, height, MemcpyKind);
  result = hipMemcpy2D(deviceptr, pitch, deviceptr_2, pitch_2, width, height, MemcpyKind);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMemcpy2DAsync(void *dst, size_t dpitch, const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpy2DAsync(void* dst, size_t dpitch, const void* src, size_t spitch, size_t width, size_t height, hipMemcpyKind kind, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpy2DAsync(deviceptr, pitch, deviceptr_2, pitch_2, width, height, MemcpyKind, stream);
  result = hipMemcpy2DAsync(deviceptr, pitch, deviceptr_2, pitch_2, width, height, MemcpyKind, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpy2DFromArray(void *dst, size_t dpitch, hipArray_const_t src, size_t wOffset, size_t hOffset, size_t width, size_t height, enum hipMemcpyKind kind);
  // HIP: hipError_t hipMemcpy2DFromArray( void* dst, size_t dpitch, hipArray_const_t src, size_t wOffset, size_t hOffset, size_t width, size_t height, hipMemcpyKind kind);
  // CHECK: result = hipMemcpy2DFromArray(deviceptr, pitch, Array_const_t, wOffset, hOffset, width, height, MemcpyKind);
  result = hipMemcpy2DFromArray(deviceptr, pitch, Array_const_t, wOffset, hOffset, width, height, MemcpyKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpy2DFromArrayAsync(void *dst, size_t dpitch, hipArray_const_t src, size_t wOffset, size_t hOffset, size_t width, size_t height, enum hipMemcpyKind kind, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpy2DFromArrayAsync( void* dst, size_t dpitch, hipArray_const_t src, size_t wOffset, size_t hOffset, size_t width, size_t height, hipMemcpyKind kind, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpy2DFromArrayAsync(deviceptr, pitch, Array_const_t, wOffset, hOffset, width, height, MemcpyKind, stream);
  result = hipMemcpy2DFromArrayAsync(deviceptr, pitch, Array_const_t, wOffset, hOffset, width, height, MemcpyKind, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpy2DToArray(hipArray_t dst, size_t wOffset, size_t hOffset, const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind);
  // HIP: hipError_t hipMemcpy2DToArray(hipArray* dst, size_t wOffset, size_t hOffset, const void* src, size_t spitch, size_t width, size_t height, hipMemcpyKind kind);
  // CHECK: result = hipMemcpy2DToArray(Array_t, wOffset, hOffset, deviceptr_2, pitch, width, height, MemcpyKind);
  result = hipMemcpy2DToArray(Array_t, wOffset, hOffset, deviceptr_2, pitch, width, height, MemcpyKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpy2DToArrayAsync(hipArray_t dst, size_t wOffset, size_t hOffset, const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpy2DToArrayAsync(hipArray* dst, size_t wOffset, size_t hOffset, const void* src, size_t spitch, size_t width, size_t height, hipMemcpyKind kind, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpy2DToArrayAsync(Array_t, wOffset, hOffset, deviceptr_2, pitch, width, height, MemcpyKind, stream);
  result = hipMemcpy2DToArrayAsync(Array_t, wOffset, hOffset, deviceptr_2, pitch, width, height, MemcpyKind, stream);

  // CHECK: hipMemcpy3DParms Memcpy3DParms;
  hipMemcpy3DParms Memcpy3DParms;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpy3D(const struct hipMemcpy3DParms *p);
  // HIP: hipError_t hipMemcpy3D(const struct hipMemcpy3DParms* p);
  // CHECK: result = hipMemcpy3D(&Memcpy3DParms);
  result = hipMemcpy3D(&Memcpy3DParms);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMemcpy3DAsync(const struct hipMemcpy3DParms *p, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpy3DAsync(const struct hipMemcpy3DParms* p, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpy3DAsync(&Memcpy3DParms, stream);
  result = hipMemcpy3DAsync(&Memcpy3DParms, stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMemcpyAsync(void *dst, const void *src, size_t count, enum hipMemcpyKind kind, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpyAsync(void* dst, const void* src, size_t sizeBytes, hipMemcpyKind kind, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpyAsync(deviceptr, deviceptr_2, bytes, MemcpyKind, stream);
  result = hipMemcpyAsync(deviceptr, deviceptr_2, bytes, MemcpyKind, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpyFromSymbol(void *dst, HIP_SYMBOL(const void *symbol), size_t count, size_t offset __dv(0), enum hipMemcpyKind kind __dv(hipMemcpyDeviceToHost));
  // HIP: hipError_t hipMemcpyFromSymbol(void* dst, HIP_SYMBOL(const void* symbol), size_t sizeBytes, size_t offset __dparm(0), hipMemcpyKind kind __dparm(hipMemcpyDeviceToHost));
  // CHECK: result = hipMemcpyFromSymbol(deviceptr, HIP_SYMBOL(HIP_SYMBOL(image)), bytes, wOffset, MemcpyKind);
  result = hipMemcpyFromSymbol(deviceptr, HIP_SYMBOL(image), bytes, wOffset, MemcpyKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpyFromSymbolAsync(void *dst, HIP_SYMBOL(const void *symbol), size_t count, size_t offset, enum hipMemcpyKind kind, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpyFromSymbolAsync(void* dst, HIP_SYMBOL(const void* symbol), size_t sizeBytes, size_t offset, hipMemcpyKind kind, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpyFromSymbolAsync(deviceptr, HIP_SYMBOL(HIP_SYMBOL(image)), bytes, wOffset, MemcpyKind, stream);
  result = hipMemcpyFromSymbolAsync(deviceptr, HIP_SYMBOL(image), bytes, wOffset, MemcpyKind, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpyPeer(void *dst, int dstDevice, const void *src, int srcDevice, size_t count);
  // HIP: hipError_t hipMemcpyPeer(void* dst, int dstDeviceId, const void* src, int srcDeviceId, size_t sizeBytes);
  // CHECK: result = hipMemcpyPeer(deviceptr, deviceId, deviceptr_2, device, bytes);
  result = hipMemcpyPeer(deviceptr, deviceId, deviceptr_2, device, bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpyPeerAsync(void *dst, int dstDevice, const void *src, int srcDevice, size_t count, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpyPeerAsync(void* dst, int dstDeviceId, const void* src, int srcDevice, size_t sizeBytes, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpyPeerAsync(deviceptr, deviceId, deviceptr_2, device, bytes, stream);
  result = hipMemcpyPeerAsync(deviceptr, deviceId, deviceptr_2, device, bytes, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpyToSymbol(HIP_SYMBOL(const void *symbol), const void *src, size_t count, size_t offset __dv(0), enum hipMemcpyKind kind __dv(hipMemcpyHostToDevice));
  // HIP: hipError_t hipMemcpyToSymbol(HIP_SYMBOL(const void* symbol), const void* src, size_t sizeBytes, size_t offset __dparm(0), hipMemcpyKind kind __dparm(hipMemcpyHostToDevice));
  // CHECK: result = hipMemcpyToSymbol(HIP_SYMBOL(image), deviceptr, bytes, wOffset, MemcpyKind);
  result = hipMemcpyToSymbol(HIP_SYMBOL(image), deviceptr, bytes, wOffset, MemcpyKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpyToSymbolAsync(HIP_SYMBOL(const void *symbol), const void *src, size_t count, size_t offset, enum hipMemcpyKind kind, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpyToSymbolAsync(HIP_SYMBOL(const void* symbol), const void* src, size_t sizeBytes, size_t offset, hipMemcpyKind kind, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpyToSymbolAsync(HIP_SYMBOL(image), deviceptr, bytes, wOffset, MemcpyKind, stream);
  result = hipMemcpyToSymbolAsync(HIP_SYMBOL(image), deviceptr, bytes, wOffset, MemcpyKind, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemGetInfo(size_t *free, size_t *total);
  // HIP: hipError_t hipMemGetInfo(size_t* free, size_t* total);
  // CHECK: result = hipMemGetInfo(&bytes, &wOffset);
  result = hipMemGetInfo(&bytes, &wOffset);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemset(void *devPtr, int value, size_t count);
  // HIP: hipError_t hipMemset(void* dst, int value, size_t sizeBytes);
  // CHECK: result = hipMemset(deviceptr, intVal, bytes);
  result = hipMemset(deviceptr, intVal, bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemset2D(void *devPtr, size_t pitch, int value, size_t width, size_t height);
  // HIP: hipError_t hipMemset2D(void* dst, size_t pitch, int value, size_t width, size_t height);
  // CHECK: result = hipMemset2D(deviceptr, pitch, intVal, width, height);
  result = hipMemset2D(deviceptr, pitch, intVal, width, height);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMemset2DAsync(void *devPtr, size_t pitch, int value, size_t width, size_t height, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemset2DAsync(void* dst, size_t pitch, int value, size_t width, size_t height,hipStream_t stream __dparm(0));
  // CHECK: result = hipMemset2DAsync(deviceptr, pitch, intVal, width, height, stream);
  result = hipMemset2DAsync(deviceptr, pitch, intVal, width, height, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemset3D(struct hipPitchedPtr pitchedDevPtr, int value, struct hipExtent extent);
  // HIP: hipError_t hipMemset3D(hipPitchedPtr pitchedDevPtr, int  value, hipExtent extent );
  // CHECK: result = hipMemset3D(PitchedPtr, intVal, Extent);
  result = hipMemset3D(PitchedPtr, intVal, Extent);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMemset3DAsync(struct hipPitchedPtr pitchedDevPtr, int value, struct hipExtent extent, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemset3DAsync(hipPitchedPtr pitchedDevPtr, int  value, hipExtent extent ,hipStream_t stream __dparm(0));
  // CHECK: result = hipMemset3DAsync(PitchedPtr, intVal, Extent, stream);
  result = hipMemset3DAsync(PitchedPtr, intVal, Extent, stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMemsetAsync(void *devPtr, int value, size_t count, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemsetAsync(void* dst, int value, size_t sizeBytes, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemsetAsync(deviceptr, intVal, bytes, stream);
  result = hipMemsetAsync(deviceptr, intVal, bytes, stream);

  // CUDA: static __inline__ __host__ struct hipExtent make_hipExtent(size_t w, size_t h, size_t d);
  // HIP: static inline struct hipExtent make_hipExtent(size_t w, size_t h, size_t d);
  // CHECK: Extent = make_hipExtent(width, height, bytes);
  Extent = make_hipExtent(width, height, bytes);

  // CUDA: static __inline__ __host__ struct hipPitchedPtr make_hipPitchedPtr(void *d, size_t p, size_t xsz, size_t ysz);
  // HIP: static inline struct hipPitchedPtr make_hipPitchedPtr(void* d, size_t p, size_t xsz, size_t ysz);
  // CHECK: PitchedPtr = make_hipPitchedPtr(image, pitch, width, height);
  PitchedPtr = make_hipPitchedPtr(image, pitch, width, height);

  // CHECK: hipPos Pos;
  hipPos Pos;

  // CUDA: static __inline__ __host__ struct hipPos make_hipPos(size_t x, size_t y, size_t z);
  // HIP: static inline struct hipPos make_hipPos(size_t x, size_t y, size_t z);
  // CHECK: Pos = make_hipPos(width, height, bytes);
  Pos = make_hipPos(width, height, bytes);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipMemcpyFromArray(void *dst, hipArray_const_t src, size_t wOffset, size_t hOffset, size_t count, enum hipMemcpyKind kind);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipMemcpyFromArray(void* dst, hipArray_const_t srcArray, size_t wOffset, size_t hOffset, size_t count, hipMemcpyKind kind);
  // CHECK: result = hipMemcpyFromArray(deviceptr, Array_const_t, wOffset, hOffset, bytes, MemcpyKind);
  result = hipMemcpyFromArray(deviceptr, Array_const_t, wOffset, hOffset, bytes, MemcpyKind);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipMemcpyToArray(hipArray_t dst, size_t wOffset, size_t hOffset, const void *src, size_t count, enum hipMemcpyKind kind);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipMemcpyToArray(hipArray* dst, size_t wOffset, size_t hOffset, const void* src, size_t count, hipMemcpyKind kind);
  // CHECK: result = hipMemcpyToArray(Array_t, wOffset, hOffset, deviceptr, bytes, MemcpyKind);
  result = hipMemcpyToArray(Array_t, wOffset, hOffset, deviceptr, bytes, MemcpyKind);

  // CHECK: hipPointerAttribute_t PointerAttributes;
  hipPointerAttribute_t PointerAttributes;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipPointerGetAttributes(struct hipPointerAttribute_t *attributes, const void *ptr);
  // HIP: hipError_t hipPointerGetAttributes(hipPointerAttribute_t* attributes, const void* ptr);
  // CHECK: result = hipPointerGetAttributes(&PointerAttributes, deviceptr);
  result = hipPointerGetAttributes(&PointerAttributes, deviceptr);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceCanAccessPeer(int *canAccessPeer, int device, int peerDevice);
  // HIP: hipError_t hipDeviceCanAccessPeer(int* canAccessPeer, int deviceId, int peerDeviceId);
  // CHECK: result = hipDeviceCanAccessPeer(&intVal, device, deviceId);
  result = hipDeviceCanAccessPeer(&intVal, device, deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceDisablePeerAccess(int peerDevice);
  // HIP: hipError_t hipDeviceDisablePeerAccess(int peerDeviceId);
  // CHECK: result = hipDeviceDisablePeerAccess(device);
  result = hipDeviceDisablePeerAccess(device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceEnablePeerAccess(int peerDevice, unsigned int flags);
  // HIP: hipError_t hipDeviceEnablePeerAccess(int peerDeviceId, unsigned int flags);
  // CHECK: result = hipDeviceEnablePeerAccess(device, flags);
  result = hipDeviceEnablePeerAccess(device, flags);

  // CHECK: hipGLDeviceList GLDeviceList;
  hipGLDeviceList GLDeviceList;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGLGetDevices(unsigned int *pCudaDeviceCount, int *pCudaDevices, unsigned int cudaDeviceCount, enum hipGLDeviceList deviceList);
  // HIP: hipError_t hipGLGetDevices(unsigned int* pHipDeviceCount, int* pHipDevices, unsigned int hipDeviceCount, hipGLDeviceList deviceList);
  // CHECK: result = hipGLGetDevices(&flags, &intVal, count, GLDeviceList);
  result = hipGLGetDevices(&flags, &intVal, count, GLDeviceList);

  // CHECK: hipGraphicsResource* GraphicsResource;
  // CHECK-NEXT: hipGraphicsResource_t GraphicsResource_t;
  hipGraphicsResource* GraphicsResource;
  hipGraphicsResource_t GraphicsResource_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphicsGLRegisterBuffer(struct hipGraphicsResource **resource, GLuint buffer, unsigned int flags);
  // HIP: hipError_t hipGraphicsGLRegisterBuffer(hipGraphicsResource** resource, GLuint buffer, unsigned int flags);
  // CHECK: result = hipGraphicsGLRegisterBuffer(&GraphicsResource, gl_uint, flags);
  result = hipGraphicsGLRegisterBuffer(&GraphicsResource, gl_uint, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphicsGLRegisterImage(struct hipGraphicsResource **resource, GLuint image, GLenum target, unsigned int flags);
  // HIP: hipError_t hipGraphicsGLRegisterImage(hipGraphicsResource** resource, GLuint image, GLenum target, unsigned int flags);
  // CHECK: result = hipGraphicsGLRegisterImage(&GraphicsResource, gl_uint, gl_enum, flags);
  result = hipGraphicsGLRegisterImage(&GraphicsResource, gl_uint, gl_enum, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphicsMapResources(int count, hipGraphicsResource_t *resources, hipStream_t stream __dv(0));
  // HIP: hipError_t hipGraphicsMapResources(int count, hipGraphicsResource_t* resources, hipStream_t stream  __dparm(0));
  // CHECK: result = hipGraphicsMapResources(intVal, &GraphicsResource, stream);
  result = hipGraphicsMapResources(intVal, &GraphicsResource, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphicsResourceGetMappedPointer(void **devPtr, size_t *size, hipGraphicsResource_t resource);
  // HIP: hipError_t hipGraphicsResourceGetMappedPointer(void** devPtr, size_t* size, hipGraphicsResource_t resource);
  // CHECK: result = hipGraphicsResourceGetMappedPointer(&deviceptr, &bytes, GraphicsResource);
  result = hipGraphicsResourceGetMappedPointer(&deviceptr, &bytes, GraphicsResource);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphicsUnmapResources(int count, hipGraphicsResource_t *resources, hipStream_t stream __dv(0));
  // HIP: hipError_t hipGraphicsUnmapResources(int count, hipGraphicsResource_t* resources, hipStream_t stream  __dparm(0));
  // CHECK: result = hipGraphicsUnmapResources(intVal, &GraphicsResource, stream);
  result = hipGraphicsUnmapResources(intVal, &GraphicsResource, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphicsUnregisterResource(hipGraphicsResource_t resource);
  // HIP: hipError_t hipGraphicsUnregisterResource(hipGraphicsResource_t resource);
  // CHECK: result = hipGraphicsUnregisterResource(GraphicsResource);
  result = hipGraphicsUnregisterResource(GraphicsResource);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipBindTexture(size_t *offset, const struct textureReference *texref, const void *devPtr, const struct hipChannelFormatDesc *desc, size_t size __dv(UINT_MAX));
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipBindTexture(size_t* offset, const textureReference* tex, const void* devPtr, const hipChannelFormatDesc* desc, size_t size __dparm(UINT_MAX));
  // CHECK: result = hipBindTexture(&wOffset, texref, deviceptr, &ChannelFormatDesc, bytes);
  result = hipBindTexture(&wOffset, texref, deviceptr, &ChannelFormatDesc, bytes);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipBindTexture2D(size_t *offset, const struct textureReference *texref, const void *devPtr, const struct hipChannelFormatDesc *desc, size_t width, size_t height, size_t pitch);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipBindTexture2D(size_t* offset, const textureReference* tex, const void* devPtr, const hipChannelFormatDesc* desc, size_t width, size_t height, size_t pitch);
  // CHECK: result = hipBindTexture2D(&wOffset, texref, deviceptr, &ChannelFormatDesc, width, height, pitch);
  result = hipBindTexture2D(&wOffset, texref, deviceptr, &ChannelFormatDesc, width, height, pitch);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipBindTextureToArray(const struct textureReference *texref, hipArray_const_t array, const struct hipChannelFormatDesc *desc);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipBindTextureToArray(const textureReference* tex, hipArray_const_t array, const hipChannelFormatDesc* desc);
  // CHECK: result = hipBindTextureToArray(texref, Array_const_t, &ChannelFormatDesc);
  result = hipBindTextureToArray(texref, Array_const_t, &ChannelFormatDesc);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipBindTextureToMipmappedArray(const struct textureReference *texref, hipMipmappedArray_const_t mipmappedArray, const struct hipChannelFormatDesc *desc);
  // HIP: hipError_t hipBindTextureToMipmappedArray(const textureReference* tex, hipMipmappedArray_const_t mipmappedArray, const hipChannelFormatDesc* desc);
  // CHECK: result = hipBindTextureToMipmappedArray(texref, MipmappedArray_const_t, &ChannelFormatDesc);
  result = hipBindTextureToMipmappedArray(texref, MipmappedArray_const_t, &ChannelFormatDesc);

  // CHECK: hipChannelFormatKind ChannelFormatKind;
  hipChannelFormatKind ChannelFormatKind;

  // CUDA: extern __host__ struct hipChannelFormatDesc CUDARTAPI hipCreateChannelDesc(int x, int y, int z, int w, enum hipChannelFormatKind f);
  // HIP: HIP_PUBLIC_API hipChannelFormatDesc hipCreateChannelDesc(int x, int y, int z, int w, hipChannelFormatKind f);
  // CHECK: ChannelFormatDesc = hipCreateChannelDesc(x, y, z, w, ChannelFormatKind);
  ChannelFormatDesc = hipCreateChannelDesc(x, y, z, w, ChannelFormatKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetChannelDesc(struct hipChannelFormatDesc *desc, hipArray_const_t array);
  // HIP: hipError_t hipGetChannelDesc(hipChannelFormatDesc* desc, hipArray_const_t array);
  // CHECK: result = hipGetChannelDesc(&ChannelFormatDesc, Array_const_t);
  result = hipGetChannelDesc(&ChannelFormatDesc, Array_const_t);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipGetTextureAlignmentOffset(size_t *offset, const struct textureReference *texref);
  // HIP: hipError_t hipGetTextureAlignmentOffset(size_t* offset, const textureReference* texref);
  // CHECK: result = hipGetTextureAlignmentOffset(&wOffset, texref);
  result = hipGetTextureAlignmentOffset(&wOffset, texref);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipGetTextureReference(const struct textureReference **texref, HIP_SYMBOL(const void *symbol));
  // HIP:  hipError_t hipGetTextureReference(const textureReference** texref, HIP_SYMBOL(const void* symbol));
  // CHECK: result = hipGetTextureReference(const_cast<const textureReference**>(&texref), HIP_SYMBOL(image));
  result = hipGetTextureReference(const_cast<const textureReference**>(&texref), image);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipUnbindTexture(const struct textureReference *texref);
  // HIP:  DEPRECATED(DEPRECATED_MSG) hipError_t hipUnbindTexture(const textureReference* tex);
  // CHECK: result = hipUnbindTexture(texref);
  result = hipUnbindTexture(texref);

  return 0;
}
