// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>
#if defined(_WIN32)
  #include "windows.h"
  #include <GL/glew.h>
#endif

#include "cuda_gl_interop.h"
// CHECK: #include "hip/hip_runtime_api.h"
#include "hip/hip_runtime_api.h"

int main() {
  printf("12. CUDA Runtime API Functions synthetic test\n");

  size_t bytes = 0;
  size_t width = 0;
  size_t height = 0;
  size_t wOffset = 0;
  size_t hOffset = 0;
  size_t pitch = 0;
  size_t pitch_2 = 0;
  int device = 0;
  int deviceId = 0;
  int intVal = 0;
  int x = 0;
  int y = 0;
  int z = 0;
  int w = 0;
  unsigned int flags = 0;
  unsigned int levels = 0;
  unsigned int count = 0;
  float ms = 0;
  void* deviceptr = nullptr;
  void* deviceptr_2 = nullptr;
  void* image = nullptr;
  void* func = nullptr;
  void* src = nullptr;
  void* dst = nullptr;
  char* ch = nullptr;
  const char* const_ch = nullptr;
  dim3 gridDim;
  dim3 blockDim;
  GLuint gl_uint = 0;
  GLenum gl_enum = 0;
  struct textureReference* texref = nullptr;
  std::string name = "str";

#if defined(_WIN32)
  unsigned long long ull = 0;
#else
  unsigned long ull = 0;
#endif
  unsigned long long ull_2 = 0;

  // CHECK: hipError_t result = hipSuccess;
  // CHECK-NEXT: hipError_t Error_t;
  // CHECK-NEXT: hipStream_t stream;
  hipError_t result = hipSuccess;
  hipError_t Error_t;
  hipStream_t stream;

  // CHECK: hipEvent_t Event_t;
  // CHECK-Next: hipEvent_t Event_2;
  hipEvent_t Event_t;
  hipEvent_t Event_2;

  // CHECK: hipMemcpy3DParms Memcpy3DParms;
  hipMemcpy3DParms Memcpy3DParms;

  // CHECK: hipMemcpyKind MemcpyKind;
  hipMemcpyKind MemcpyKind;

  // CHECK: hipChannelFormatDesc ChannelFormatDesc;
  hipChannelFormatDesc ChannelFormatDesc;

  // CHECK: hipMipmappedArray* MipmappedArray;
  // CHECK-NEXT: hipMipmappedArray_t MipmappedArray_t;
  // CHECK-NEXT: hipMipmappedArray_const_t MipmappedArray_const_t;
  hipMipmappedArray* MipmappedArray;
  hipMipmappedArray_t MipmappedArray_t;
  hipMipmappedArray_const_t MipmappedArray_const_t;

  // CHECK: hipArray* Array;
  // CHECK-NEXT: hipArray_t Array_t;
  // CHECK-NEXT: hipArray_const_t Array_const_t;
  hipArray* Array;
  hipArray_t Array_t;
  hipArray_const_t Array_const_t;

  // CHECK: hipDeviceProp_t DeviceProp;
  hipDeviceProp_t DeviceProp;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipChooseDevice(int *device, const struct hipDeviceProp_t *prop);
  // HIP: hipError_t hipChooseDevice(int* device, const hipDeviceProp_t* prop);
  // CHECK: result = hipChooseDevice(&device, &DeviceProp);
  result = hipChooseDevice(&device, &DeviceProp);

  // CHECK: hipDeviceAttribute_t DeviceAttr;
  hipDeviceAttribute_t DeviceAttr;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetAttribute(int *value, enum hipDeviceAttribute_t attr, int device);
  // HIP: hipError_t hipDeviceGetAttribute(int* pi, hipDeviceAttribute_t attr, int deviceId);
  // CHECK: result = hipDeviceGetAttribute(&device, DeviceAttr, deviceId);
  result = hipDeviceGetAttribute(&device, DeviceAttr, deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetByPCIBusId(int *device, const char *pciBusId);
  // HIP: hipError_t hipDeviceGetByPCIBusId(int* device, const char* pciBusId);
  // CHECK: result = hipDeviceGetByPCIBusId(&device, ch);
  result = hipDeviceGetByPCIBusId(&device, ch);

  // CHECK: hipFuncCache_t FuncCache;
  hipFuncCache_t FuncCache;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetCacheConfig(enum hipFuncCache_t *pCacheConfig);
  // HIP: hipError_t hipDeviceGetCacheConfig(hipFuncCache_t* cacheConfig);
  // CHECK: result = hipDeviceGetCacheConfig(&FuncCache);
  result = hipDeviceGetCacheConfig(&FuncCache);

  // CHECK: hipLimit_t Limit;
  hipLimit_t Limit;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetLimit(size_t *pValue, enum hipLimit_t limit);
  // HIP: hipError_t hipDeviceGetLimit(size_t* pValue, enum hipLimit_t limit);
  // CHECK: result = hipDeviceGetLimit(&bytes, Limit);
  result = hipDeviceGetLimit(&bytes, Limit);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetPCIBusId(char *pciBusId, int len, int device);
  // HIP: hipError_t hipError_t hipDeviceGetPCIBusId(char* pciBusId, int len, int device);
  // CHECK: result = hipDeviceGetPCIBusId(ch, intVal, device);
  result = hipDeviceGetPCIBusId(ch, intVal, device);

  // CHECK: hipSharedMemConfig SharedMemConfig;
  hipSharedMemConfig SharedMemConfig;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetSharedMemConfig(enum hipSharedMemConfig *pConfig);
  // HIP: hipError_t hipDeviceGetSharedMemConfig(hipSharedMemConfig* pConfig);
  // CHECK: result = hipDeviceGetSharedMemConfig(&SharedMemConfig);
  result = hipDeviceGetSharedMemConfig(&SharedMemConfig);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetStreamPriorityRange(int *leastPriority, int *greatestPriority);
  // HIP: hipError_t hipDeviceGetStreamPriorityRange(int* leastPriority, int* greatestPriority);
  // CHECK: result = hipDeviceGetStreamPriorityRange(&deviceId, &intVal);
  result = hipDeviceGetStreamPriorityRange(&deviceId, &intVal);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceReset(void);
  // HIP: hipError_t hipError_t hipDeviceReset(void);
  // CHECK: result = hipDeviceReset();
  result = hipDeviceReset();

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetCacheConfig(enum hipFuncCache_t cacheConfig);
  // HIP: hipError_t hipDeviceSetCacheConfig(hipFuncCache_t cacheConfig);
  // CHECK: result = hipDeviceSetCacheConfig(FuncCache);
  result = hipDeviceSetCacheConfig(FuncCache);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetSharedMemConfig(enum hipSharedMemConfig config);
  // HIP: hipError_t hipDeviceSetSharedMemConfig(hipSharedMemConfig config);
  // CHECK: result = hipDeviceSetSharedMemConfig(SharedMemConfig);
  result = hipDeviceSetSharedMemConfig(SharedMemConfig);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceSynchronize(void);
  // HIP: hipError_t hipDeviceSynchronize(void);
  // CHECK: result = hipDeviceSynchronize();
  result = hipDeviceSynchronize();

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipGetDevice(int *device);
  // HIP: hipError_t hipGetDevice(int* deviceId);
  // CHECK: result = hipGetDevice(&deviceId);
  result = hipGetDevice(&deviceId);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipGetDeviceCount(int *count);
  // HIP: hipError_t hipGetDeviceCount(int* count);
  // CHECK: result = hipGetDeviceCount(&deviceId);
  result = hipGetDeviceCount(&deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetDeviceFlags( unsigned int *flags );
  // HIP: hipError_t hipGetDeviceFlags(unsigned int* flags);
  // CHECK: result = hipGetDeviceFlags(&flags);
  result = hipGetDeviceFlags(&flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipGetDeviceProperties(struct hipDeviceProp_t *prop, int device);
  // HIP: hipError_t hipGetDeviceProperties(hipDeviceProp_t* prop, int deviceId);
  // CHECK: result = hipGetDeviceProperties(&DeviceProp, deviceId);
  result = hipGetDeviceProperties(&DeviceProp, deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcCloseMemHandle(void *devPtr);
  // HIP: hipError_t hipError_t hipIpcCloseMemHandle(void* devPtr);
  // CHECK: result = hipIpcCloseMemHandle(deviceptr);
  result = hipIpcCloseMemHandle(deviceptr);

  // CHECK: hipIpcEventHandle_t IpcEventHandle_t;
  hipIpcEventHandle_t IpcEventHandle_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcGetEventHandle(hipIpcEventHandle_t *handle, hipEvent_t event);
  // HIP: hipError_t hipIpcGetEventHandle(hipIpcEventHandle_t* handle, hipEvent_t event);
  // CHECK: result = hipIpcGetEventHandle(&IpcEventHandle_t, Event_t);
  result = hipIpcGetEventHandle(&IpcEventHandle_t, Event_t);

  // CHECK: hipIpcMemHandle_t IpcMemHandle_t;
  hipIpcMemHandle_t IpcMemHandle_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcGetMemHandle(hipIpcMemHandle_t *handle, void *devPtr);
  // HIP: hipError_t hipIpcGetMemHandle(hipIpcMemHandle_t* handle, void* devPtr);
  // CHECK: result = hipIpcGetMemHandle(&IpcMemHandle_t, deviceptr);
  result = hipIpcGetMemHandle(&IpcMemHandle_t, deviceptr);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcOpenEventHandle(hipEvent_t *event, hipIpcEventHandle_t handle);
  // HIP: hipError_t hipIpcOpenEventHandle(hipEvent_t* event, hipIpcEventHandle_t handle);
  // CHECK: result = hipIpcOpenEventHandle(&Event_t, IpcEventHandle_t);
  result = hipIpcOpenEventHandle(&Event_t, IpcEventHandle_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcOpenMemHandle(void **devPtr, hipIpcMemHandle_t handle, unsigned int flags);
  // HIP: hipError_t hipIpcOpenMemHandle(void** devPtr, hipIpcMemHandle_t handle, unsigned int flags);
  // CHECK: result = hipIpcOpenMemHandle(&deviceptr, IpcMemHandle_t, flags);
  result = hipIpcOpenMemHandle(&deviceptr, IpcMemHandle_t, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipSetDevice(int device);
  // HIP: hipError_t hipSetDevice(int deviceId);
  // CHECK: result = hipSetDevice(deviceId);
  result = hipSetDevice(deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipSetDeviceFlags( unsigned int flags );
  // HIP: hipError_t hipSetDeviceFlags(unsigned flags);
  // CHECK: result = hipSetDeviceFlags(flags);
  result = hipSetDeviceFlags(flags);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipDeviceReset(void);
  // HIP: hipError_t hipDeviceReset(void);
  // CHECK: result = hipDeviceReset();
  result = hipDeviceReset();

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipDeviceGetCacheConfig(enum hipFuncCache_t *pCacheConfig);
  // HIP: hipError_t hipDeviceGetCacheConfig(hipFuncCache_t* cacheConfig);
  // CHECK: result = hipDeviceGetCacheConfig(&FuncCache);
  result = hipDeviceGetCacheConfig(&FuncCache);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipDeviceSetCacheConfig(enum hipFuncCache_t cacheConfig);
  // HIP: hipError_t hipError_t hipDeviceSetCacheConfig(hipFuncCache_t cacheConfig);
  // CHECK: result = hipDeviceSetCacheConfig(FuncCache);
  result = hipDeviceSetCacheConfig(FuncCache);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipDeviceSynchronize(void);
  // HIP: hipError_t hipError_t hipDeviceSynchronize(void);
  // CHECK: result = hipDeviceSynchronize();
  result = hipDeviceSynchronize();

  // CUDA: extern __host__ __cudart_builtin__ const char* CUDARTAPI hipGetErrorName(hipError_t error);
  // HIP: const char* hipGetErrorName(hipError_t hip_error);
  // CHECK: const_ch = hipGetErrorName(Error_t);
  const_ch = hipGetErrorName(Error_t);

  // CUDA: extern __host__ __cudart_builtin__ const char* CUDARTAPI hipGetErrorString(hipError_t error);
  // HIP: const char* hipGetErrorString(hipError_t hipError);
  // CHECK: const_ch = hipGetErrorString(Error_t);
  const_ch = hipGetErrorString(Error_t);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipGetLastError(void);
  // HIP: hipError_t hipGetLastError(void);
  // CHECK: result = hipGetLastError();
  result = hipGetLastError();

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipPeekAtLastError(void);
  // HIP: hipError_t hipPeekAtLastError(void);
  // CHECK: result = hipPeekAtLastError();
  result = hipPeekAtLastError();

  // CHECK: hipStreamCallback_t StreamCallback_t;
  hipStreamCallback_t StreamCallback_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamAddCallback(hipStream_t stream, hipStreamCallback_t callback, void* userData, unsigned int flags);
  // HIP: hipError_t hipStreamAddCallback(hipStream_t stream, hipStreamCallback_t callback, void* userData, unsigned int flags);
  // CHECK: result = hipStreamAddCallback(stream, StreamCallback_t, image, flags);
  result = hipStreamAddCallback(stream, StreamCallback_t, image, flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamAttachMemAsync(hipStream_t stream, void *devPtr, size_t length __dv(0), unsigned int flags = hipMemAttachSingle);
  // HIP: hipError_t hipStreamAttachMemAsync(hipStream_t stream, void* dev_ptr, size_t length __dparm(0), unsigned int flags __dparm(hipMemAttachSingle));
  // CHECK: result = hipStreamAttachMemAsync(stream, deviceptr, bytes, flags);
  result = hipStreamAttachMemAsync(stream, deviceptr, bytes, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamCreate(hipStream_t *pStream);
  // HIP: hipError_t hipStreamCreate(hipStream_t* stream);
  // CHECK: result = hipStreamCreate(&stream);
  result = hipStreamCreate(&stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamCreateWithFlags(hipStream_t *pStream, unsigned int flags);
  // HIP: hipError_t hipStreamCreateWithFlags(hipStream_t* stream, unsigned int flags);
  // CHECK: result = hipStreamCreateWithFlags(&stream, flags);
  result = hipStreamCreateWithFlags(&stream, flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamCreateWithPriority(hipStream_t *pStream, unsigned int flags, int priority);
  // HIP: hipError_t hipStreamCreateWithPriority(hipStream_t* stream, unsigned int flags);
  // CHECK: result = hipStreamCreateWithPriority(&stream, flags, intVal);
  result = hipStreamCreateWithPriority(&stream, flags, intVal);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamDestroy(hipStream_t stream);
  // HIP: hipError_t hipStreamDestroy(hipStream_t stream);
  // CHECK: result = hipStreamDestroy(stream);
  result = hipStreamDestroy(stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamGetFlags(hipStream_t hStream, unsigned int *flags);
  // HIP: hipError_t hipStreamGetFlags(hipStream_t stream, unsigned int* flags);
  // CHECK: result = hipStreamGetFlags(stream, &flags);
  result = hipStreamGetFlags(stream, &flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamGetPriority(hipStream_t hStream, int *priority);
  // HIP: hipError_t hipStreamGetPriority(hipStream_t stream, int* priority);
  // CHECK: result = hipStreamGetPriority(stream, &intVal);
  result = hipStreamGetPriority(stream, &intVal);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamQuery(hipStream_t stream);
  // HIP: hipError_t hipStreamQuery(hipStream_t stream);
  // CHECK: result = hipStreamQuery(stream);
  result = hipStreamQuery(stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamSynchronize(hipStream_t stream);
  // HIP: hipError_t hipStreamSynchronize(hipStream_t stream);
  // CHECK: result = hipStreamSynchronize(stream);
  result = hipStreamSynchronize(stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamWaitEvent(hipStream_t stream, hipEvent_t event, unsigned int flags __dv(0));
  // HIP: hipError_t hipStreamWaitEvent(hipStream_t stream, hipEvent_t event, unsigned int flags);
  // CHECK: result = hipStreamWaitEvent(stream, Event_t, flags);
  result = hipStreamWaitEvent(stream, Event_t, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipEventCreate(hipEvent_t *event);
  // HIP: hipError_t hipEventCreate(hipEvent_t* event);
  // CHECK: result = hipEventCreate(&Event_t);
  result = hipEventCreate(&Event_t);

  // CUDA: static __inline__ __host__ hipError_t hipEventCreate(hipEvent_t* event, unsigned int flags);
  // HIP: hipError_t hipEventCreateWithFlags(hipEvent_t* event, unsigned flags);
  // CHECK: result = hipEventCreateWithFlags(&Event_t, flags);
  result = hipEventCreate(&Event_t, flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipEventCreateWithFlags(hipEvent_t *event, unsigned int flags);
  // HIP: hipError_t hipEventCreateWithFlags(hipEvent_t* event, unsigned flags);
  // CHECK: result = hipEventCreateWithFlags(&Event_t, flags);
  result = hipEventCreateWithFlags(&Event_t, flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipEventDestroy(hipEvent_t event);
  // HIP: hipError_t hipEventDestroy(hipEvent_t event);
  // CHECK: result = hipEventDestroy(Event_t);
  result = hipEventDestroy(Event_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipEventElapsedTime(float *ms, hipEvent_t start, hipEvent_t end);
  // HIP: hipError_t hipEventElapsedTime(float* ms, hipEvent_t start, hipEvent_t stop);
  // CHECK: result = hipEventElapsedTime(&ms, Event_t, Event_2);
  result = hipEventElapsedTime(&ms, Event_t, Event_2);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipEventQuery(hipEvent_t event);
  // HIP: hipError_t hipEventQuery(hipEvent_t event);
  // CHECK: result = hipEventQuery(Event_t);
  result = hipEventQuery(Event_t);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipEventRecord(hipEvent_t event, hipStream_t stream __dv(0));
  // HIP: hipError_t hipEventRecord(hipEvent_t event, hipStream_t stream);
  // CHECK: result = hipEventRecord(Event_t, stream);
  result = hipEventRecord(Event_t, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipEventSynchronize(hipEvent_t event);
  // HIP: hipError_t hipEventSynchronize(hipEvent_t event);
  // CHECK: result = hipEventSynchronize(Event_t);
  result = hipEventSynchronize(Event_t);

  // CHECK: hipFuncAttributes FuncAttributes;
  hipFuncAttributes FuncAttributes;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipFuncGetAttributes(struct hipFuncAttributes *attr, reinterpret_cast<const void*>(const void *func));
  // HIP: hipError_t hipFuncGetAttributes(struct hipFuncAttributes* attr, reinterpret_cast<const void*>(const void* func));
  // CHECK: result = hipFuncGetAttributes(&FuncAttributes, reinterpret_cast<const void*>(func));
  result = hipFuncGetAttributes(&FuncAttributes, reinterpret_cast<const void*>(func));

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFuncSetCacheConfig(reinterpret_cast<const void*>(const void *func), enum hipFuncCache_t cacheConfig);
  // HIP: hipError_t hipFuncSetCacheConfig(reinterpret_cast<const void*>(const void* func), hipFuncCache_t config);
  // CHECK: result = hipFuncSetCacheConfig(reinterpret_cast<const void*>(func), FuncCache);
  result = hipFuncSetCacheConfig(reinterpret_cast<const void*>(func), FuncCache);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(const void *func), enum hipSharedMemConfig config);
  // HIP: hipError_t hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(const void* func), hipSharedMemConfig config);
  // CHECK: result = hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(func), SharedMemConfig);
  result = hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(func), SharedMemConfig);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchKernel(reinterpret_cast<const void*>(const void *func), dim3 gridDim, dim3 blockDim, void **args, size_t sharedMem, hipStream_t stream);
  // HIP: hipError_t hipLaunchKernel(reinterpret_cast<const void*>(const void* function_address), dim3 numBlocks, dim3 dimBlocks, void** args, size_t sharedMemBytes __dparm(0), hipStream_t stream __dparm(0));
  // CHECK: result = hipLaunchKernel(reinterpret_cast<const void*>(func), gridDim, blockDim, &image, bytes, stream);
  result = hipLaunchKernel(reinterpret_cast<const void*>(func), gridDim, blockDim, &image, bytes, stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipOccupancyMaxActiveBlocksPerMultiprocessor(int *numBlocks, const void *func, int blockSize, size_t dynamicSMemSize);
  // HIP: hipError_t hipOccupancyMaxActiveBlocksPerMultiprocessor(int* numBlocks, const void* f, int blockSize, size_t dynSharedMemPerBlk);
  // CHECK: result = hipOccupancyMaxActiveBlocksPerMultiprocessor(&intVal, func, device, bytes);
  result = hipOccupancyMaxActiveBlocksPerMultiprocessor(&intVal, func, device, bytes);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(int *numBlocks, const void *func, int blockSize, size_t dynamicSMemSize, unsigned int flags);
  // HIP: hipError_t hipOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(int* numBlocks, const void* f, int blockSize, size_t dynSharedMemPerBlk, unsigned int flags __dparm(hipOccupancyDefault));
  // CHECK: result = hipOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(&intVal, func, intVal, bytes, flags);
  result = hipOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(&intVal, func, intVal, bytes, flags);

  // CUDA: template<class T> static __inline__ __host__ CUDART_DEVICE hipError_t hipOccupancyMaxPotentialBlockSize(int* minGridSize, int* blockSize, T func, size_t dynamicSMemSize = 0, int blockSizeLimit = 0);
  // HIP: template <typename T> static hipError_t __host__ inline hipOccupancyMaxPotentialBlockSize(int* gridSize, int* blockSize, T f, size_t dynSharedMemPerBlk = 0, int blockSizeLimit = 0);
  // CHECK: result = hipOccupancyMaxPotentialBlockSize(&intVal, &device, func, bytes, deviceId);
  result = hipOccupancyMaxPotentialBlockSize(&intVal, &device, func, bytes, deviceId);

  // CUDA: template<class T> static __inline__ __host__ CUDART_DEVICE hipError_t hipOccupancyMaxPotentialBlockSizeWithFlags(int* minGridSize, int* blockSize, T func, size_t dynamicSMemSize = 0, int blockSizeLimit = 0, unsigned int flags = 0);
  // HIP: template <typename T> static hipError_t __host__ inline hipOccupancyMaxPotentialBlockSizeWithFlags(int* gridSize, int* blockSize, T f, size_t dynSharedMemPerBlk = 0, int blockSizeLimit = 0, unsigned int flags = 0);
  // CHECK: result = hipOccupancyMaxPotentialBlockSizeWithFlags(&intVal, &device, func, bytes, deviceId, flags);
  result = hipOccupancyMaxPotentialBlockSizeWithFlags(&intVal, &device, func, bytes, deviceId, flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipFree(void *devPtr);
  // HIP: hipError_t hipFree(void* ptr);
  // CHECK: result = hipFree(deviceptr);
  result = hipFree(deviceptr);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFreeArray(hipArray_t array);
  // HIP: hipError_t hipFreeArray(hipArray* array);
  // CHECK: result = hipFreeArray(Array_t);
  result = hipFreeArray(Array_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipHostFree(void *ptr);
  // HIP: hipError_t hipHostFree(void* ptr);
  // CHECK: result = hipHostFree(deviceptr);
  result = hipHostFree(deviceptr);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFreeMipmappedArray(hipMipmappedArray_t mipmappedArray);
  // HIP: hipError_t hipFreeMipmappedArray(hipMipmappedArray_t mipmappedArray);
  // CHECK: result = hipFreeMipmappedArray(MipmappedArray_t);
  result = hipFreeMipmappedArray(MipmappedArray_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetMipmappedArrayLevel(hipArray_t *levelArray, hipMipmappedArray_const_t mipmappedArray, unsigned int level);
  // HIP: hipError_t hipGetMipmappedArrayLevel(hipArray_t* levelArray, hipMipmappedArray_const_t mipmappedArray, unsigned int level);
  // CHECK: result = hipGetMipmappedArrayLevel(&Array_t, MipmappedArray_const_t, flags);
  result = hipGetMipmappedArrayLevel(&Array_t, MipmappedArray_const_t, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetSymbolAddress(void **devPtr, HIP_SYMBOL(const void *symbol));
  // HIP: hipError_t hipGetSymbolAddress(void** devPtr, HIP_SYMBOL(const void* symbol));
  // CHECK: result = hipGetSymbolAddress(&deviceptr, HIP_SYMBOL(HIP_SYMBOL(image)));
  result = hipGetSymbolAddress(&deviceptr, HIP_SYMBOL(image));

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetSymbolSize(size_t *size, HIP_SYMBOL(const void *symbol));
  // HIP: hipError_t hipGetSymbolSize(size_t* size, HIP_SYMBOL(const void* symbol));
  // CHECK: result = hipGetSymbolSize(&bytes, HIP_SYMBOL(HIP_SYMBOL(image)));
  result = hipGetSymbolSize(&bytes, HIP_SYMBOL(image));

  // CUDA: extern __host__ hipError_t CUDARTAPI hipHostAlloc(void **pHost, size_t size, unsigned int flags);
  // HIP: DEPRECATED("use hipHostMalloc instead") hipError_t hipHostAlloc(void** ptr, size_t size, unsigned int flags);
  // CHECK: result = hipHostAlloc(&deviceptr, bytes, flags);
  result = hipHostAlloc(&deviceptr, bytes, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipHostGetDevicePointer(void **pDevice, void *pHost, unsigned int flags);
  // HIP: hipError_t hipHostGetDevicePointer(void** devPtr, void* hstPtr, unsigned int flags);
  // CHECK: result = hipHostGetDevicePointer(&deviceptr, image, flags);
  result = hipHostGetDevicePointer(&deviceptr, image, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipHostGetFlags(unsigned int *pFlags, void *pHost);
  // HIP: hipError_t hipHostGetFlags(unsigned int* flagsPtr, void* hostPtr);
  // CHECK: result = hipHostGetFlags(&flags, image);
  result = hipHostGetFlags(&flags, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipHostRegister(void *ptr, size_t size, unsigned int flags);
  // HIP: hipError_t hipHostRegister(void* hostPtr, size_t sizeBytes, unsigned int flags);
  // CHECK: result = hipHostRegister(image, bytes, flags);
  result = hipHostRegister(image, bytes, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipHostUnregister(void *ptr);
  // HIP: hipError_t hipHostUnregister(void* hostPtr);
  // CHECK: result = hipHostUnregister(image);
  result = hipHostUnregister(image);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMalloc(void **devPtr, size_t size);
  // HIP: hipError_t hipMalloc(void** ptr, size_t size);
  // CHECK: result = hipMalloc(&deviceptr, bytes);
  result = hipMalloc(&deviceptr, bytes);

  // CHECK: hipPitchedPtr PitchedPtr;
  hipPitchedPtr PitchedPtr;

  // CHECK: hipExtent Extent;
  hipExtent Extent;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMalloc3D(struct hipPitchedPtr* pitchedDevPtr, struct hipExtent extent);
  // HIP: hipError_t hipMalloc3D(hipPitchedPtr* pitchedDevPtr, hipExtent extent);
  // CHECK: result = hipMalloc3D(&PitchedPtr, Extent);
  result = hipMalloc3D(&PitchedPtr, Extent);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMalloc3DArray(hipArray_t *array, const struct hipChannelFormatDesc* desc, struct hipExtent extent, unsigned int flags __dv(0));
  // HIP: hipError_t hipMalloc3DArray(hipArray** array, const struct hipChannelFormatDesc* desc, struct hipExtent extent, unsigned int flags);
  // CHECK: result = hipMalloc3DArray(&Array_t, &ChannelFormatDesc, Extent, flags);
  result = hipMalloc3DArray(&Array_t, &ChannelFormatDesc, Extent, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocArray(hipArray_t *array, const struct hipChannelFormatDesc *desc, size_t width, size_t height __dv(0), unsigned int flags __dv(0));
  // HIP: hipError_t hipMallocArray(hipArray** array, const hipChannelFormatDesc* desc, size_t width, size_t height __dparm(0), unsigned int flags __dparm(hipArrayDefault));
  // CHECK: result = hipMallocArray(&Array_t, &ChannelFormatDesc, width, height, flags);
  result = hipMallocArray(&Array_t, &ChannelFormatDesc, width, height, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipHostMalloc(void **ptr, size_t size);
  // HIP: hipError_t hipHostMalloc(void** ptr, size_t size, unsigned int flags);
  // CHECK: result = hipHostMalloc(&deviceptr, bytes, hipHostMallocDefault);
  result = hipHostMalloc(&deviceptr, bytes);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMallocManaged(void **devPtr, size_t size, unsigned int flags = hipMemAttachGlobal);
  // HIP: hipError_t hipMallocManaged(void** dev_ptr, size_t size, unsigned int flags __dparm(hipMemAttachGlobal));
  // CHECK: result = hipMallocManaged(&deviceptr, bytes, flags);
  result = hipMallocManaged(&deviceptr, bytes, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocMipmappedArray(hipMipmappedArray_t *mipmappedArray, const struct hipChannelFormatDesc* desc, struct hipExtent extent, unsigned int numLevels, unsigned int flags __dv(0));
  // HIP: hipError_t hipMallocMipmappedArray(hipMipmappedArray_t* mipmappedArray, const struct hipChannelFormatDesc* desc, struct hipExtent extent, unsigned int numLevels, unsigned int flags __dparm(0));
  // CHECK: result = hipMallocMipmappedArray(&MipmappedArray_t, &ChannelFormatDesc, Extent, levels, flags);
  result = hipMallocMipmappedArray(&MipmappedArray_t, &ChannelFormatDesc, Extent, levels, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocPitch(void **devPtr, size_t *pitch, size_t width, size_t height);
  // HIP: hipError_t hipMallocPitch(void** ptr, size_t* pitch, size_t width, size_t height);
  // CHECK: result = hipMallocPitch(&deviceptr, &bytes, width, height);
  result = hipMallocPitch(&deviceptr, &bytes, width, height);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpy(void *dst, const void *src, size_t count, enum hipMemcpyKind kind);
  // HIP: hipError_t hipMemcpy(void* dst, const void* src, size_t sizeBytes, hipMemcpyKind kind);
  // CHECK: result = hipMemcpy(deviceptr, deviceptr_2, bytes, MemcpyKind);
  result = hipMemcpy(deviceptr, deviceptr_2, bytes, MemcpyKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpy2D(void *dst, size_t dpitch, const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind);
  // HIP: hipError_t hipMemcpy2D(void* dst, size_t dpitch, const void* src, size_t spitch, size_t width, size_t height, hipMemcpyKind kind);
  // CHECK: result = hipMemcpy2D(deviceptr, pitch, deviceptr_2, pitch_2, width, height, MemcpyKind);
  result = hipMemcpy2D(deviceptr, pitch, deviceptr_2, pitch_2, width, height, MemcpyKind);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMemcpy2DAsync(void *dst, size_t dpitch, const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpy2DAsync(void* dst, size_t dpitch, const void* src, size_t spitch, size_t width, size_t height, hipMemcpyKind kind, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpy2DAsync(deviceptr, pitch, deviceptr_2, pitch_2, width, height, MemcpyKind, stream);
  result = hipMemcpy2DAsync(deviceptr, pitch, deviceptr_2, pitch_2, width, height, MemcpyKind, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpy2DFromArray(void *dst, size_t dpitch, hipArray_const_t src, size_t wOffset, size_t hOffset, size_t width, size_t height, enum hipMemcpyKind kind);
  // HIP: hipError_t hipMemcpy2DFromArray( void* dst, size_t dpitch, hipArray_const_t src, size_t wOffset, size_t hOffset, size_t width, size_t height, hipMemcpyKind kind);
  // CHECK: result = hipMemcpy2DFromArray(deviceptr, pitch, Array_const_t, wOffset, hOffset, width, height, MemcpyKind);
  result = hipMemcpy2DFromArray(deviceptr, pitch, Array_const_t, wOffset, hOffset, width, height, MemcpyKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpy2DFromArrayAsync(void *dst, size_t dpitch, hipArray_const_t src, size_t wOffset, size_t hOffset, size_t width, size_t height, enum hipMemcpyKind kind, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpy2DFromArrayAsync( void* dst, size_t dpitch, hipArray_const_t src, size_t wOffset, size_t hOffset, size_t width, size_t height, hipMemcpyKind kind, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpy2DFromArrayAsync(deviceptr, pitch, Array_const_t, wOffset, hOffset, width, height, MemcpyKind, stream);
  result = hipMemcpy2DFromArrayAsync(deviceptr, pitch, Array_const_t, wOffset, hOffset, width, height, MemcpyKind, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpy2DToArray(hipArray_t dst, size_t wOffset, size_t hOffset, const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind);
  // HIP: hipError_t hipMemcpy2DToArray(hipArray* dst, size_t wOffset, size_t hOffset, const void* src, size_t spitch, size_t width, size_t height, hipMemcpyKind kind);
  // CHECK: result = hipMemcpy2DToArray(Array_t, wOffset, hOffset, deviceptr_2, pitch, width, height, MemcpyKind);
  result = hipMemcpy2DToArray(Array_t, wOffset, hOffset, deviceptr_2, pitch, width, height, MemcpyKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpy2DToArrayAsync(hipArray_t dst, size_t wOffset, size_t hOffset, const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpy2DToArrayAsync(hipArray* dst, size_t wOffset, size_t hOffset, const void* src, size_t spitch, size_t width, size_t height, hipMemcpyKind kind, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpy2DToArrayAsync(Array_t, wOffset, hOffset, deviceptr_2, pitch, width, height, MemcpyKind, stream);
  result = hipMemcpy2DToArrayAsync(Array_t, wOffset, hOffset, deviceptr_2, pitch, width, height, MemcpyKind, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpy3D(const struct hipMemcpy3DParms *p);
  // HIP: hipError_t hipMemcpy3D(const struct hipMemcpy3DParms* p);
  // CHECK: result = hipMemcpy3D(&Memcpy3DParms);
  result = hipMemcpy3D(&Memcpy3DParms);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMemcpy3DAsync(const struct hipMemcpy3DParms *p, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpy3DAsync(const struct hipMemcpy3DParms* p, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpy3DAsync(&Memcpy3DParms, stream);
  result = hipMemcpy3DAsync(&Memcpy3DParms, stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMemcpyAsync(void *dst, const void *src, size_t count, enum hipMemcpyKind kind, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpyAsync(void* dst, const void* src, size_t sizeBytes, hipMemcpyKind kind, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpyAsync(deviceptr, deviceptr_2, bytes, MemcpyKind, stream);
  result = hipMemcpyAsync(deviceptr, deviceptr_2, bytes, MemcpyKind, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpyFromSymbol(void *dst, HIP_SYMBOL(const void *symbol), size_t count, size_t offset __dv(0), enum hipMemcpyKind kind __dv(hipMemcpyDeviceToHost));
  // HIP: hipError_t hipMemcpyFromSymbol(void* dst, HIP_SYMBOL(const void* symbol), size_t sizeBytes, size_t offset __dparm(0), hipMemcpyKind kind __dparm(hipMemcpyDeviceToHost));
  // CHECK: result = hipMemcpyFromSymbol(deviceptr, HIP_SYMBOL(HIP_SYMBOL(image)), bytes, wOffset, MemcpyKind);
  result = hipMemcpyFromSymbol(deviceptr, HIP_SYMBOL(image), bytes, wOffset, MemcpyKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpyFromSymbolAsync(void *dst, HIP_SYMBOL(const void *symbol), size_t count, size_t offset, enum hipMemcpyKind kind, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpyFromSymbolAsync(void* dst, HIP_SYMBOL(const void* symbol), size_t sizeBytes, size_t offset, hipMemcpyKind kind, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpyFromSymbolAsync(deviceptr, HIP_SYMBOL(HIP_SYMBOL(image)), bytes, wOffset, MemcpyKind, stream);
  result = hipMemcpyFromSymbolAsync(deviceptr, HIP_SYMBOL(image), bytes, wOffset, MemcpyKind, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpyPeer(void *dst, int dstDevice, const void *src, int srcDevice, size_t count);
  // HIP: hipError_t hipMemcpyPeer(void* dst, int dstDeviceId, const void* src, int srcDeviceId, size_t sizeBytes);
  // CHECK: result = hipMemcpyPeer(deviceptr, deviceId, deviceptr_2, device, bytes);
  result = hipMemcpyPeer(deviceptr, deviceId, deviceptr_2, device, bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpyPeerAsync(void *dst, int dstDevice, const void *src, int srcDevice, size_t count, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpyPeerAsync(void* dst, int dstDeviceId, const void* src, int srcDevice, size_t sizeBytes, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpyPeerAsync(deviceptr, deviceId, deviceptr_2, device, bytes, stream);
  result = hipMemcpyPeerAsync(deviceptr, deviceId, deviceptr_2, device, bytes, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpyToSymbol(HIP_SYMBOL(const void *symbol), const void *src, size_t count, size_t offset __dv(0), enum hipMemcpyKind kind __dv(hipMemcpyHostToDevice));
  // HIP: hipError_t hipMemcpyToSymbol(HIP_SYMBOL(const void* symbol), const void* src, size_t sizeBytes, size_t offset __dparm(0), hipMemcpyKind kind __dparm(hipMemcpyHostToDevice));
  // CHECK: result = hipMemcpyToSymbol(HIP_SYMBOL(image), deviceptr, bytes, wOffset, MemcpyKind);
  result = hipMemcpyToSymbol(HIP_SYMBOL(image), deviceptr, bytes, wOffset, MemcpyKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemcpyToSymbolAsync(HIP_SYMBOL(const void *symbol), const void *src, size_t count, size_t offset, enum hipMemcpyKind kind, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemcpyToSymbolAsync(HIP_SYMBOL(const void* symbol), const void* src, size_t sizeBytes, size_t offset, hipMemcpyKind kind, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpyToSymbolAsync(HIP_SYMBOL(image), deviceptr, bytes, wOffset, MemcpyKind, stream);
  result = hipMemcpyToSymbolAsync(HIP_SYMBOL(image), deviceptr, bytes, wOffset, MemcpyKind, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemGetInfo(size_t *free, size_t *total);
  // HIP: hipError_t hipMemGetInfo(size_t* free, size_t* total);
  // CHECK: result = hipMemGetInfo(&bytes, &wOffset);
  result = hipMemGetInfo(&bytes, &wOffset);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemset(void *devPtr, int value, size_t count);
  // HIP: hipError_t hipMemset(void* dst, int value, size_t sizeBytes);
  // CHECK: result = hipMemset(deviceptr, intVal, bytes);
  result = hipMemset(deviceptr, intVal, bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemset2D(void *devPtr, size_t pitch, int value, size_t width, size_t height);
  // HIP: hipError_t hipMemset2D(void* dst, size_t pitch, int value, size_t width, size_t height);
  // CHECK: result = hipMemset2D(deviceptr, pitch, intVal, width, height);
  result = hipMemset2D(deviceptr, pitch, intVal, width, height);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMemset2DAsync(void *devPtr, size_t pitch, int value, size_t width, size_t height, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemset2DAsync(void* dst, size_t pitch, int value, size_t width, size_t height,hipStream_t stream __dparm(0));
  // CHECK: result = hipMemset2DAsync(deviceptr, pitch, intVal, width, height, stream);
  result = hipMemset2DAsync(deviceptr, pitch, intVal, width, height, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemset3D(struct hipPitchedPtr pitchedDevPtr, int value, struct hipExtent extent);
  // HIP: hipError_t hipMemset3D(hipPitchedPtr pitchedDevPtr, int value, hipExtent extent );
  // CHECK: result = hipMemset3D(PitchedPtr, intVal, Extent);
  result = hipMemset3D(PitchedPtr, intVal, Extent);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMemset3DAsync(struct hipPitchedPtr pitchedDevPtr, int value, struct hipExtent extent, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemset3DAsync(hipPitchedPtr pitchedDevPtr, int value, hipExtent extent ,hipStream_t stream __dparm(0));
  // CHECK: result = hipMemset3DAsync(PitchedPtr, intVal, Extent, stream);
  result = hipMemset3DAsync(PitchedPtr, intVal, Extent, stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipMemsetAsync(void *devPtr, int value, size_t count, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemsetAsync(void* dst, int value, size_t sizeBytes, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemsetAsync(deviceptr, intVal, bytes, stream);
  result = hipMemsetAsync(deviceptr, intVal, bytes, stream);

  // CUDA: static __inline__ __host__ struct hipExtent make_hipExtent(size_t w, size_t h, size_t d);
  // HIP: static inline struct hipExtent make_hipExtent(size_t w, size_t h, size_t d);
  // CHECK: Extent = make_hipExtent(width, height, bytes);
  Extent = make_hipExtent(width, height, bytes);

  // CUDA: static __inline__ __host__ struct hipPitchedPtr make_hipPitchedPtr(void *d, size_t p, size_t xsz, size_t ysz);
  // HIP: static inline struct hipPitchedPtr make_hipPitchedPtr(void* d, size_t p, size_t xsz, size_t ysz);
  // CHECK: PitchedPtr = make_hipPitchedPtr(image, pitch, width, height);
  PitchedPtr = make_hipPitchedPtr(image, pitch, width, height);

  // CHECK: hipPos Pos;
  hipPos Pos;

  // CUDA: static __inline__ __host__ struct hipPos make_hipPos(size_t x, size_t y, size_t z);
  // HIP: static inline struct hipPos make_hipPos(size_t x, size_t y, size_t z);
  // CHECK: Pos = make_hipPos(width, height, bytes);
  Pos = make_hipPos(width, height, bytes);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipMemcpyFromArray(void *dst, hipArray_const_t src, size_t wOffset, size_t hOffset, size_t count, enum hipMemcpyKind kind);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipMemcpyFromArray(void* dst, hipArray_const_t srcArray, size_t wOffset, size_t hOffset, size_t count, hipMemcpyKind kind);
  // CHECK: result = hipMemcpyFromArray(deviceptr, Array_const_t, wOffset, hOffset, bytes, MemcpyKind);
  result = hipMemcpyFromArray(deviceptr, Array_const_t, wOffset, hOffset, bytes, MemcpyKind);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipMemcpyToArray(hipArray_t dst, size_t wOffset, size_t hOffset, const void *src, size_t count, enum hipMemcpyKind kind);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipMemcpyToArray(hipArray* dst, size_t wOffset, size_t hOffset, const void* src, size_t count, hipMemcpyKind kind);
  // CHECK: result = hipMemcpyToArray(Array_t, wOffset, hOffset, deviceptr, bytes, MemcpyKind);
  result = hipMemcpyToArray(Array_t, wOffset, hOffset, deviceptr, bytes, MemcpyKind);

  // CHECK: hipPointerAttribute_t PointerAttributes;
  hipPointerAttribute_t PointerAttributes;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipPointerGetAttributes(struct hipPointerAttribute_t *attributes, const void *ptr);
  // HIP: hipError_t hipPointerGetAttributes(hipPointerAttribute_t* attributes, const void* ptr);
  // CHECK: result = hipPointerGetAttributes(&PointerAttributes, deviceptr);
  result = hipPointerGetAttributes(&PointerAttributes, deviceptr);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceCanAccessPeer(int *canAccessPeer, int device, int peerDevice);
  // HIP: hipError_t hipDeviceCanAccessPeer(int* canAccessPeer, int deviceId, int peerDeviceId);
  // CHECK: result = hipDeviceCanAccessPeer(&intVal, device, deviceId);
  result = hipDeviceCanAccessPeer(&intVal, device, deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceDisablePeerAccess(int peerDevice);
  // HIP: hipError_t hipDeviceDisablePeerAccess(int peerDeviceId);
  // CHECK: result = hipDeviceDisablePeerAccess(device);
  result = hipDeviceDisablePeerAccess(device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceEnablePeerAccess(int peerDevice, unsigned int flags);
  // HIP: hipError_t hipDeviceEnablePeerAccess(int peerDeviceId, unsigned int flags);
  // CHECK: result = hipDeviceEnablePeerAccess(device, flags);
  result = hipDeviceEnablePeerAccess(device, flags);

  // CHECK: hipGLDeviceList GLDeviceList;
  hipGLDeviceList GLDeviceList;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGLGetDevices(unsigned int *pCudaDeviceCount, int *pCudaDevices, unsigned int cudaDeviceCount, enum hipGLDeviceList deviceList);
  // HIP: hipError_t hipGLGetDevices(unsigned int* pHipDeviceCount, int* pHipDevices, unsigned int hipDeviceCount, hipGLDeviceList deviceList);
  // CHECK: result = hipGLGetDevices(&flags, &intVal, count, GLDeviceList);
  result = hipGLGetDevices(&flags, &intVal, count, GLDeviceList);

  // CHECK: hipGraphicsResource* GraphicsResource;
  // CHECK-NEXT: hipGraphicsResource_t GraphicsResource_t;
  hipGraphicsResource* GraphicsResource;
  hipGraphicsResource_t GraphicsResource_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphicsGLRegisterBuffer(struct hipGraphicsResource **resource, GLuint buffer, unsigned int flags);
  // HIP: hipError_t hipGraphicsGLRegisterBuffer(hipGraphicsResource** resource, GLuint buffer, unsigned int flags);
  // CHECK: result = hipGraphicsGLRegisterBuffer(&GraphicsResource, gl_uint, flags);
  result = hipGraphicsGLRegisterBuffer(&GraphicsResource, gl_uint, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphicsGLRegisterImage(struct hipGraphicsResource **resource, GLuint image, GLenum target, unsigned int flags);
  // HIP: hipError_t hipGraphicsGLRegisterImage(hipGraphicsResource** resource, GLuint image, GLenum target, unsigned int flags);
  // CHECK: result = hipGraphicsGLRegisterImage(&GraphicsResource, gl_uint, gl_enum, flags);
  result = hipGraphicsGLRegisterImage(&GraphicsResource, gl_uint, gl_enum, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphicsMapResources(int count, hipGraphicsResource_t *resources, hipStream_t stream __dv(0));
  // HIP: hipError_t hipGraphicsMapResources(int count, hipGraphicsResource_t* resources, hipStream_t stream __dparm(0));
  // CHECK: result = hipGraphicsMapResources(intVal, &GraphicsResource, stream);
  result = hipGraphicsMapResources(intVal, &GraphicsResource, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphicsResourceGetMappedPointer(void **devPtr, size_t *size, hipGraphicsResource_t resource);
  // HIP: hipError_t hipGraphicsResourceGetMappedPointer(void** devPtr, size_t* size, hipGraphicsResource_t resource);
  // CHECK: result = hipGraphicsResourceGetMappedPointer(&deviceptr, &bytes, GraphicsResource);
  result = hipGraphicsResourceGetMappedPointer(&deviceptr, &bytes, GraphicsResource);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphicsUnmapResources(int count, hipGraphicsResource_t *resources, hipStream_t stream __dv(0));
  // HIP: hipError_t hipGraphicsUnmapResources(int count, hipGraphicsResource_t* resources, hipStream_t stream __dparm(0));
  // CHECK: result = hipGraphicsUnmapResources(intVal, &GraphicsResource, stream);
  result = hipGraphicsUnmapResources(intVal, &GraphicsResource, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphicsUnregisterResource(hipGraphicsResource_t resource);
  // HIP: hipError_t hipGraphicsUnregisterResource(hipGraphicsResource_t resource);
  // CHECK: result = hipGraphicsUnregisterResource(GraphicsResource);
  result = hipGraphicsUnregisterResource(GraphicsResource);

  // CHECK: hipChannelFormatKind ChannelFormatKind;
  hipChannelFormatKind ChannelFormatKind;

  // CUDA: extern __host__ struct hipChannelFormatDesc CUDARTAPI hipCreateChannelDesc(int x, int y, int z, int w, enum hipChannelFormatKind f);
  // HIP: HIP_PUBLIC_API hipChannelFormatDesc hipCreateChannelDesc(int x, int y, int z, int w, hipChannelFormatKind f);
  // CHECK: ChannelFormatDesc = hipCreateChannelDesc(x, y, z, w, ChannelFormatKind);
  ChannelFormatDesc = hipCreateChannelDesc(x, y, z, w, ChannelFormatKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetChannelDesc(struct hipChannelFormatDesc *desc, hipArray_const_t array);
  // HIP: hipError_t hipGetChannelDesc(hipChannelFormatDesc* desc, hipArray_const_t array);
  // CHECK: result = hipGetChannelDesc(&ChannelFormatDesc, Array_const_t);
  result = hipGetChannelDesc(&ChannelFormatDesc, Array_const_t);

  // CHECK: hipTextureObject_t TextureObject_t;
  hipTextureObject_t TextureObject_t;

  // CHECK: hipResourceDesc ResourceDesc;
  hipResourceDesc ResourceDesc;

  // CHECK: hipTextureDesc TextureDesc;
  hipTextureDesc TextureDesc;

  // CHECK: hipResourceViewDesc ResourceViewDesc;
  hipResourceViewDesc ResourceViewDesc;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipCreateTextureObject(hipTextureObject_t *pTexObject, const struct hipResourceDesc *pResDesc, const struct hipTextureDesc *pTexDesc, const struct hipResourceViewDesc *pResViewDesc);
  // HIP: hipError_t hipCreateTextureObject(hipTextureObject_t* pTexObject, const hipResourceDesc* pResDesc, const hipTextureDesc* pTexDesc, const struct hipResourceViewDesc* pResViewDesc);
  // CHECK: result = hipCreateTextureObject(&TextureObject_t, &ResourceDesc, &TextureDesc, &ResourceViewDesc);
  result = hipCreateTextureObject(&TextureObject_t, &ResourceDesc, &TextureDesc, &ResourceViewDesc);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDestroyTextureObject(hipTextureObject_t texObject);
  // HIP: hipError_t hipDestroyTextureObject(hipTextureObject_t textureObject);
  // CHECK: result = hipDestroyTextureObject(TextureObject_t);
  result = hipDestroyTextureObject(TextureObject_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetTextureObjectResourceDesc(struct hipResourceDesc *pResDesc, hipTextureObject_t texObject);
  // HIP: hipError_t hipGetTextureObjectResourceDesc(hipResourceDesc* pResDesc, hipTextureObject_t textureObject);
  // CHECK: result = hipGetTextureObjectResourceDesc(&ResourceDesc, TextureObject_t);
  result = hipGetTextureObjectResourceDesc(&ResourceDesc, TextureObject_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetTextureObjectResourceViewDesc(struct hipResourceViewDesc *pResViewDesc, hipTextureObject_t texObject);
  // HIP: hipError_t hipGetTextureObjectResourceViewDesc(struct hipResourceViewDesc* pResViewDesc, hipTextureObject_t textureObject);
  // CHECK: result = hipGetTextureObjectResourceViewDesc(&ResourceViewDesc, TextureObject_t);
  result = hipGetTextureObjectResourceViewDesc(&ResourceViewDesc, TextureObject_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetTextureObjectTextureDesc(struct hipTextureDesc *pTexDesc, hipTextureObject_t texObject);
  // HIP: hipError_t hipGetTextureObjectTextureDesc(hipTextureDesc* pTexDesc, hipTextureObject_t textureObject);
  // CHECK: result = hipGetTextureObjectTextureDesc(&TextureDesc, TextureObject_t);
  result = hipGetTextureObjectTextureDesc(&TextureDesc, TextureObject_t);

  // CHECK: hipSurfaceObject_t SurfaceObject_t;
  hipSurfaceObject_t SurfaceObject_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipCreateSurfaceObject(hipSurfaceObject_t *pSurfObject, const struct hipResourceDesc *pResDesc);
  // HIP: hipError_t hipCreateSurfaceObject(hipSurfaceObject_t* pSurfObject, const hipResourceDesc* pResDesc);
  // CHECK: result = hipCreateSurfaceObject(&SurfaceObject_t, &ResourceDesc);
  result = hipCreateSurfaceObject(&SurfaceObject_t, &ResourceDesc);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDestroySurfaceObject(hipSurfaceObject_t surfObject);
  // HIP: hipError_t hipDestroySurfaceObject(hipSurfaceObject_t surfaceObject);
  // CHECK: result = hipDestroySurfaceObject(SurfaceObject_t);
  result = hipDestroySurfaceObject(SurfaceObject_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDriverGetVersion(int *driverVersion);
  // HIP: hipError_t hipDriverGetVersion(int* driverVersion);
  // CHECK: result = hipDriverGetVersion(&intVal);
  result = hipDriverGetVersion(&intVal);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipRuntimeGetVersion(int *runtimeVersion);
  // HIP: hipError_t hipRuntimeGetVersion(int* runtimeVersion);
  // CHECK: result = hipRuntimeGetVersion(&intVal);
  result = hipRuntimeGetVersion(&intVal);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipProfilerStart(void);
  // HIP: hipError_t hipProfilerStart();
  // CHECK: result = hipProfilerStart();
  result = hipProfilerStart();

  // CUDA: extern __host__ hipError_t CUDARTAPI hipProfilerStop(void);
  // HIP: hipError_t hipProfilerStop();
  // CHECK: result = hipProfilerStop();
  result = hipProfilerStop();

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetLimit(enum hipLimit_t limit, size_t value);
  // HIP: hipError_t hipDeviceSetLimit(enum hipLimit_t limit, size_t value);
  // CHECK: result = hipDeviceSetLimit(Limit, bytes);
  result = hipDeviceSetLimit(Limit, bytes);

  // TODO
  // CUDA: template<typename UnaryFunction, class T> static __inline__ __host__ CUDART_DEVICE hipError_t hipOccupancyMaxPotentialBlockSizeVariableSMem(int* minGridSize, int* blockSize, T func, UnaryFunction blockSizeToDynamicSMemSize, int blockSizeLimit = 0);
  // HIP: template<typename UnaryFunction, class T> static hipError_t __host__ inline hipOccupancyMaxPotentialBlockSizeVariableSMem(int* min_grid_size, int* block_size, T func, UnaryFunction block_size_to_dynamic_smem_size, int block_size_limit = 0);

  // TODO
  // CUDA: template<typename UnaryFunction, class T> static __inline__ __host__ CUDART_DEVICE hipError_t hipOccupancyMaxPotentialBlockSizeVariableSMemWithFlags(int* minGridSize, int* blockSize, T func, UnaryFunction blockSizeToDynamicSMemSize, int blockSizeLimit = 0, unsigned int flags = 0);
  // HIP:  template<typename UnaryFunction, class T> static hipError_t __host__ inline hipOccupancyMaxPotentialBlockSizeVariableSMemWithFlags(int* min_grid_size, int* block_size, T func, UnaryFunction block_size_to_dynamic_smem_size, int block_size_limit = 0, unsigned int flags = 0);

#if CUDA_VERSION >= 8000
  // CHECK: hipDeviceP2PAttr DeviceP2PAttr;
  hipDeviceP2PAttr DeviceP2PAttr;

  // CHECK: hipMemoryAdvise MemoryAdvise;
  hipMemoryAdvise MemoryAdvise;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetP2PAttribute(int *value, enum hipDeviceP2PAttr attr, int srcDevice, int dstDevice);
  // HIP: hipError_t hipDeviceGetP2PAttribute(int* value, hipDeviceP2PAttr attr, int srcDevice, int dstDevice);
  // CHECK: result = hipDeviceGetP2PAttribute(&intVal, DeviceP2PAttr, device, deviceId);
  result = hipDeviceGetP2PAttribute(&intVal, DeviceP2PAttr, device, deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemAdvise(const void *devPtr, size_t count, enum hipMemoryAdvise advice, int device);
  // HIP: hipError_t hipMemAdvise(const void* dev_ptr, size_t count, hipMemoryAdvise advice, int device);
  // CHECK: result = hipMemAdvise(deviceptr, bytes, MemoryAdvise, device);
  result = hipMemAdvise(deviceptr, bytes, MemoryAdvise, device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPrefetchAsync(const void *devPtr, size_t count, int dstDevice, hipStream_t stream __dv(0));
  // HIP: hipError_t hipMemPrefetchAsync(const void* dev_ptr, size_t count, int device, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemPrefetchAsync(deviceptr, bytes, device, stream);
  result = hipMemPrefetchAsync(deviceptr, bytes, device, stream);

  // CHECK: hipMemRangeAttribute MemRangeAttribute;
  hipMemRangeAttribute MemRangeAttribute;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemRangeGetAttribute(void *data, size_t dataSize, enum hipMemRangeAttribute attribute, const void *devPtr, size_t count);
  // HIP: hipError_t hipMemRangeGetAttribute(void* data, size_t data_size, hipMemRangeAttribute attribute, const void* dev_ptr, size_t count);
  // CHECK: result = hipMemRangeGetAttribute(deviceptr, bytes, MemRangeAttribute, deviceptr_2, wOffset);
  result = hipMemRangeGetAttribute(deviceptr, bytes, MemRangeAttribute, deviceptr_2, wOffset);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemRangeGetAttributes(void **data, size_t *dataSizes, enum hipMemRangeAttribute *attributes, size_t numAttributes, const void *devPtr, size_t count);
  // HIP: hipError_t hipMemRangeGetAttributes(void** data, size_t* data_sizes, hipMemRangeAttribute* attributes, size_t num_attributes, const void* dev_ptr, size_t count);
  // CHECK: result = hipMemRangeGetAttributes(&deviceptr, &bytes, &MemRangeAttribute, wOffset, deviceptr_2, hOffset);
  result = hipMemRangeGetAttributes(&deviceptr, &bytes, &MemRangeAttribute, wOffset, deviceptr_2, hOffset);
#endif

#if CUDA_VERSION >= 9000
  // CHECK: hipFuncAttribute FuncAttribute;
  hipFuncAttribute FuncAttribute;

  // CHECK: hipLaunchParams LaunchParams;
  hipLaunchParams LaunchParams;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipFuncSetAttribute(reinterpret_cast<const void*>(const void *func), enum hipFuncAttribute attr, int value);
  // HIP: hipError_t hipFuncSetAttribute(reinterpret_cast<const void*>(const void* func), hipFuncAttribute attr, int value);
  // CHECK: result = hipFuncSetAttribute(reinterpret_cast<const void*>(func), FuncAttribute, intVal);
  result = hipFuncSetAttribute(reinterpret_cast<const void*>(func), FuncAttribute, intVal);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchCooperativeKernel(reinterpret_cast<const void*>(const void *func), dim3 gridDim, dim3 blockDim, void **args, size_t sharedMem, hipStream_t stream);
  // HIP: hipError_t hipLaunchCooperativeKernel(reinterpret_cast<const void*>(const void* f), dim3 gridDim, dim3 blockDimX, void** kernelParams, unsigned int sharedMemBytes, hipStream_t stream);
  // CHECK: result = hipLaunchCooperativeKernel(reinterpret_cast<const void*>(func), gridDim, blockDim, &image, flags, stream);
  result = hipLaunchCooperativeKernel(reinterpret_cast<const void*>(func), gridDim, blockDim, &image, flags, stream);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipLaunchCooperativeKernelMultiDevice(struct hipLaunchParams *launchParamsList, unsigned int numDevices, unsigned int flags __dv(0));
  // HIP: hipError_t hipLaunchCooperativeKernelMultiDevice(hipLaunchParams* launchParamsList, int numDevices, unsigned int flags);
  // CHECK: result = hipLaunchCooperativeKernelMultiDevice(&LaunchParams, intVal, flags);
  result = hipLaunchCooperativeKernelMultiDevice(&LaunchParams, intVal, flags);
#endif

#if CUDA_VERSION <= 10000
  // CUDA: extern __host__ hipError_t CUDARTAPI hipConfigureCall(dim3 gridDim, dim3 blockDim, size_t sharedMem __dv(0), hipStream_t stream __dv(0));
  // HIP: hipError_t hipConfigureCall(dim3 gridDim, dim3 blockDim, size_t sharedMem __dparm(0), hipStream_t stream __dparm(0));
  // CHECK: result = hipConfigureCall(gridDim, blockDim, bytes, stream);
  result = hipConfigureCall(gridDim, blockDim, bytes, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchByPtr(const void *func);
  // HIP: hipError_t hipLaunchByPtr(const void* func);
  // CHECK: result = hipLaunchByPtr(deviceptr);
  result = hipLaunchByPtr(deviceptr);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipSetupArgument(const void *arg, size_t size, size_t offset);
  // HIP: hipError_t hipSetupArgument(const void* arg, size_t size, size_t offset);
  // CHECK: result = hipSetupArgument(deviceptr, bytes, wOffset);
  result = hipSetupArgument(deviceptr, bytes, wOffset);
#endif

#if CUDA_VERSION >= 10000
  // CHECK: hipHostFn_t hostFn;
  hipHostFn_t hostFn;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void *userData);
  // HIP: hipError_t hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void* userData);
  // CHECK: result = hipLaunchHostFunc(stream, hostFn, image);
  result = hipLaunchHostFunc(stream, hostFn, image);

  // CHECK: hipGraph_t Graph_t, Graph_t_2;
  hipGraph_t Graph_t, Graph_t_2;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamEndCapture(hipStream_t stream, hipGraph_t *pGraph);
  // HIP: hipError_t hipStreamEndCapture(hipStream_t stream, hipGraph_t* pGraph);
  // CHECK: result = hipStreamEndCapture(stream, &Graph_t);
  result = hipStreamEndCapture(stream, &Graph_t);

  // CHECK: hipStreamCaptureStatus StreamCaptureStatus;
  hipStreamCaptureStatus StreamCaptureStatus;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamIsCapturing(hipStream_t stream, enum hipStreamCaptureStatus *pCaptureStatus);
  // HIP: hipError_t hipStreamIsCapturing(hipStream_t stream, hipStreamCaptureStatus* pCaptureStatus);
  // CHECK: result = hipStreamIsCapturing(stream, &StreamCaptureStatus);
  result = hipStreamIsCapturing(stream, &StreamCaptureStatus);

  // CHECK: hipExternalMemory_t ExternalMemory_t;
  hipExternalMemory_t ExternalMemory_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDestroyExternalMemory(hipExternalMemory_t extMem);
  // HIP: hipError_t hipDestroyExternalMemory(hipExternalMemory_t extMem);
  // CHECK: result = hipDestroyExternalMemory(ExternalMemory_t);
  result = hipDestroyExternalMemory(ExternalMemory_t);

  // CHECK: hipExternalSemaphore_t ExternalSemaphore_t;
  hipExternalSemaphore_t ExternalSemaphore_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDestroyExternalSemaphore(hipExternalSemaphore_t extSem);
  // HIP: hipError_t hipDestroyExternalSemaphore(hipExternalSemaphore_t extSem);
  // CHECK: result = hipDestroyExternalSemaphore(ExternalSemaphore_t);
  result = hipDestroyExternalSemaphore(ExternalSemaphore_t);

  // CHECK: hipExternalMemoryBufferDesc ExternalMemoryBufferDesc;
  hipExternalMemoryBufferDesc ExternalMemoryBufferDesc;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipExternalMemoryGetMappedBuffer(void **devPtr, hipExternalMemory_t extMem, const struct hipExternalMemoryBufferDesc *bufferDesc);
  // HIP: hipError_t hipExternalMemoryGetMappedBuffer(void **devPtr, hipExternalMemory_t extMem, const hipExternalMemoryBufferDesc *bufferDesc);
  // CHECK: result = hipExternalMemoryGetMappedBuffer(&deviceptr, ExternalMemory_t, &ExternalMemoryBufferDesc);
  result = hipExternalMemoryGetMappedBuffer(&deviceptr, ExternalMemory_t, &ExternalMemoryBufferDesc);

  // CHECK: hipExternalMemoryHandleDesc ExternalMemoryHandleDesc;
  hipExternalMemoryHandleDesc ExternalMemoryHandleDesc;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipImportExternalMemory(hipExternalMemory_t *extMem_out, const struct hipExternalMemoryHandleDesc *memHandleDesc);
  // HIP: hipError_t hipImportExternalMemory(hipExternalMemory_t* extMem_out, const hipExternalMemoryHandleDesc* memHandleDesc);
  // CHECK: result = hipImportExternalMemory(&ExternalMemory_t, &ExternalMemoryHandleDesc);
  result = hipImportExternalMemory(&ExternalMemory_t, &ExternalMemoryHandleDesc);

  // CHECK: hipExternalSemaphoreHandleDesc ExternalSemaphoreHandleDesc;
  hipExternalSemaphoreHandleDesc ExternalSemaphoreHandleDesc;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipImportExternalSemaphore(hipExternalSemaphore_t *extSem_out, const struct hipExternalSemaphoreHandleDesc *semHandleDesc);
  // HIP: hipError_t hipImportExternalSemaphore(hipExternalSemaphore_t* extSem_out, const hipExternalSemaphoreHandleDesc* semHandleDesc);
  // CHECK: result = hipImportExternalSemaphore(&ExternalSemaphore_t, &ExternalSemaphoreHandleDesc);
  result = hipImportExternalSemaphore(&ExternalSemaphore_t, &ExternalSemaphoreHandleDesc);

  // CHECK: hipExternalSemaphoreSignalParams ExternalSemaphoreSignalParams;
  hipExternalSemaphoreSignalParams ExternalSemaphoreSignalParams;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipSignalExternalSemaphoresAsync(const hipExternalSemaphore_t *extSemArray, const struct hipExternalSemaphoreSignalParams *paramsArray, unsigned int numExtSems, hipStream_t stream __dv(0));
  // HIP: hipError_t hipSignalExternalSemaphoresAsync(const hipExternalSemaphore_t* extSemArray, const hipExternalSemaphoreSignalParams* paramsArray, unsigned int numExtSems, hipStream_t stream);
  // CHECK: result = hipSignalExternalSemaphoresAsync(&ExternalSemaphore_t, &ExternalSemaphoreSignalParams, flags, stream);
  result = hipSignalExternalSemaphoresAsync(&ExternalSemaphore_t, &ExternalSemaphoreSignalParams, flags, stream);

  // CHECK: hipExternalSemaphoreWaitParams ExternalSemaphoreWaitParams;
  hipExternalSemaphoreWaitParams ExternalSemaphoreWaitParams;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipWaitExternalSemaphoresAsync(const hipExternalSemaphore_t *extSemArray, const struct hipExternalSemaphoreWaitParams *paramsArray, unsigned int numExtSems, hipStream_t stream __dv(0));
  // HIP: hipError_t hipWaitExternalSemaphoresAsync(const hipExternalSemaphore_t* extSemArray, const hipExternalSemaphoreWaitParams* paramsArray, unsigned int numExtSems, hipStream_t stream);
  // CHECK: result = hipWaitExternalSemaphoresAsync(&ExternalSemaphore_t, &ExternalSemaphoreWaitParams, flags, stream);
  result = hipWaitExternalSemaphoresAsync(&ExternalSemaphore_t, &ExternalSemaphoreWaitParams, flags, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void *userData);
  // HIP: hipError_t hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void* userData);
  // CHECK: result = hipLaunchHostFunc(stream, hostFn, image);
  result = hipLaunchHostFunc(stream, hostFn, image);

  // CHECK: hipGraphNode_t graphNode, graphNode_2;
  hipGraphNode_t graphNode, graphNode_2;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphAddChildGraphNode(hipGraphNode_t *pGraphNode, hipGraph_t graph, const hipGraphNode_t *pDependencies, size_t numDependencies, hipGraph_t childGraph);
  // HIP: hipError_t hipGraphAddChildGraphNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, hipGraph_t childGraph);
  // CHECK: result = hipGraphAddChildGraphNode(&graphNode, Graph_t, &graphNode_2, bytes, Graph_t_2);
  result = hipGraphAddChildGraphNode(&graphNode, Graph_t, &graphNode_2, bytes, Graph_t_2);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphAddDependencies(hipGraph_t graph, const hipGraphNode_t *from, const hipGraphNode_t *to, size_t numDependencies);
  // HIP: hipError_t hipGraphAddDependencies(hipGraph_t graph, const hipGraphNode_t* from, const hipGraphNode_t* to, size_t numDependencies);
  // CHECK: result = hipGraphAddDependencies(Graph_t, &graphNode, &graphNode_2, bytes);
  result = hipGraphAddDependencies(Graph_t, &graphNode, &graphNode_2, bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphAddEmptyNode(hipGraphNode_t *pGraphNode, hipGraph_t graph, const hipGraphNode_t *pDependencies, size_t numDependencies);
  // HIP: hipError_t hipGraphAddEmptyNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies);
  // CHECK: result = hipGraphAddEmptyNode(&graphNode, Graph_t, &graphNode_2, bytes);
  result = hipGraphAddEmptyNode(&graphNode, Graph_t, &graphNode_2, bytes);

  // CHECK: hipHostNodeParams HostNodeParams;
  hipHostNodeParams HostNodeParams;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphAddHostNode(hipGraphNode_t *pGraphNode, hipGraph_t graph, const hipGraphNode_t *pDependencies, size_t numDependencies, const struct hipHostNodeParams *pNodeParams);
  // HIP: hipError_t hipGraphAddHostNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, const hipHostNodeParams* pNodeParams);
  // CHECK: result = hipGraphAddHostNode(&graphNode, Graph_t, &graphNode_2, bytes, &HostNodeParams);
  result = hipGraphAddHostNode(&graphNode, Graph_t, &graphNode_2, bytes, &HostNodeParams);

  // CHECK: hipKernelNodeParams KernelNodeParams;
  hipKernelNodeParams KernelNodeParams;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphAddKernelNode(hipGraphNode_t *pGraphNode, hipGraph_t graph, const hipGraphNode_t *pDependencies, size_t numDependencies, const struct hipKernelNodeParams *pNodeParams);
  // HIP: hipError_t hipGraphAddKernelNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, const hipKernelNodeParams* pNodeParams);
  // CHECK: result = hipGraphAddKernelNode(&graphNode, Graph_t, &graphNode_2, bytes, &KernelNodeParams);
  result = hipGraphAddKernelNode(&graphNode, Graph_t, &graphNode_2, bytes, &KernelNodeParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphAddMemcpyNode(hipGraphNode_t *pGraphNode, hipGraph_t graph, const hipGraphNode_t *pDependencies, size_t numDependencies, const struct hipMemcpy3DParms *pCopyParams);
  // HIP: hipError_t hipGraphAddMemcpyNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, const hipMemcpy3DParms* pCopyParams);
  // CHECK: result = hipGraphAddMemcpyNode(&graphNode, Graph_t, &graphNode_2, bytes, &Memcpy3DParms);
  result = hipGraphAddMemcpyNode(&graphNode, Graph_t, &graphNode_2, bytes, &Memcpy3DParms);

  // CHECK: hipMemsetParams MemsetParams;
  hipMemsetParams MemsetParams;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphAddMemsetNode(hipGraphNode_t *pGraphNode, hipGraph_t graph, const hipGraphNode_t *pDependencies, size_t numDependencies, const struct hipMemsetParams *pMemsetParams);
  // HIP: hipError_t hipGraphAddMemsetNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, const hipMemsetParams* pMemsetParams);
  // CHECK: result = hipGraphAddMemsetNode(&graphNode, Graph_t, &graphNode_2, bytes, &MemsetParams);
  result = hipGraphAddMemsetNode(&graphNode, Graph_t, &graphNode_2, bytes, &MemsetParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphChildGraphNodeGetGraph(hipGraphNode_t node, hipGraph_t *pGraph);
  // HIP: hipError_t hipGraphChildGraphNodeGetGraph(hipGraphNode_t node, hipGraph_t* pGraph);
  // CHECK: result = hipGraphChildGraphNodeGetGraph(graphNode, &Graph_t);
  result = hipGraphChildGraphNodeGetGraph(graphNode, &Graph_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphClone(hipGraph_t *pGraphClone, hipGraph_t originalGraph);
  // HIP: hipError_t hipGraphClone(hipGraph_t* pGraphClone, hipGraph_t originalGraph);
  // CHECK: result = hipGraphClone(&Graph_t, Graph_t_2);
  result = hipGraphClone(&Graph_t, Graph_t_2);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphCreate(hipGraph_t *pGraph, unsigned int flags);
  // HIP: hipError_t hipGraphCreate(hipGraph_t* pGraph, unsigned int flags);
  // CHECK: result = hipGraphCreate(&Graph_t, flags);
  result = hipGraphCreate(&Graph_t, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphDestroy(hipGraph_t graph);
  // HIP: hipError_t hipGraphDestroy(hipGraph_t graph);
  // CHECK: result = hipGraphDestroy(Graph_t);
  result = hipGraphDestroy(Graph_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphDestroyNode(hipGraphNode_t node);
  // HIP: hipError_t hipGraphDestroyNode(hipGraphNode_t node);
  // CHECK: result = hipGraphDestroyNode(graphNode);
  result = hipGraphDestroyNode(graphNode);

  // CHECK: hipGraphExec_t GraphExec_t;
  hipGraphExec_t GraphExec_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExecDestroy(hipGraphExec_t graphExec);
  // HIP: hipError_t hipGraphExecDestroy(hipGraphExec_t graphExec);
  // CHECK: result = hipGraphExecDestroy(GraphExec_t);
  result = hipGraphExecDestroy(GraphExec_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphGetEdges(hipGraph_t graph, hipGraphNode_t *from, hipGraphNode_t *to, size_t *numEdges);
  // HIP: hipError_t hipGraphGetEdges(hipGraph_t graph, hipGraphNode_t* from, hipGraphNode_t* to, size_t* numEdges);
  // CHECK: result = hipGraphGetEdges(Graph_t, &graphNode, &graphNode_2, &bytes);
  result = hipGraphGetEdges(Graph_t, &graphNode, &graphNode_2, &bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphGetNodes(hipGraph_t graph, hipGraphNode_t *nodes, size_t *numNodes);
  // HIP: hipError_t hipGraphGetNodes(hipGraph_t graph, hipGraphNode_t* nodes, size_t* numNodes);
  // CHECK: result = hipGraphGetNodes(Graph_t, &graphNode, &bytes);
  result = hipGraphGetNodes(Graph_t, &graphNode, &bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphGetRootNodes(hipGraph_t graph, hipGraphNode_t *pRootNodes, size_t *pNumRootNodes);
  // HIP: hipError_t hipGraphGetRootNodes(hipGraph_t graph, hipGraphNode_t* pRootNodes, size_t* pNumRootNodes);
  // CHECK: result = hipGraphGetRootNodes(Graph_t, &graphNode, &bytes);
  result = hipGraphGetRootNodes(Graph_t, &graphNode, &bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphHostNodeGetParams(hipGraphNode_t node, struct hipHostNodeParams *pNodeParams);
  // HIP: hipError_t hipGraphHostNodeGetParams(hipGraphNode_t node, hipHostNodeParams* pNodeParams);
  // CHECK: result = hipGraphHostNodeGetParams(graphNode, &HostNodeParams);
  result = hipGraphHostNodeGetParams(graphNode, &HostNodeParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphHostNodeSetParams(hipGraphNode_t node, const struct hipHostNodeParams *pNodeParams);
  // HIP: hipError_t hipGraphHostNodeSetParams(hipGraphNode_t node, const hipHostNodeParams* pNodeParams);
  // CHECK: result = hipGraphHostNodeSetParams(graphNode, &HostNodeParams);
  result = hipGraphHostNodeSetParams(graphNode, &HostNodeParams);

  char* name_ = const_cast<char*>(name.c_str());

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphInstantiate(hipGraphExec_t *pGraphExec, hipGraph_t graph, hipGraphNode_t *pErrorNode, char *pLogBuffer, size_t bufferSize);
  // HIP: hipError_t hipGraphInstantiate(hipGraphExec_t* pGraphExec, hipGraph_t graph, hipGraphNode_t* pErrorNode, char* pLogBuffer, size_t bufferSize);
  // CHECK: result = hipGraphInstantiate(&GraphExec_t, Graph_t, &graphNode, name_, bytes);
  result = hipGraphInstantiate(&GraphExec_t, Graph_t, &graphNode, name_, bytes);

  // CHECK: hipGraphNodeType GraphNodeType;
  hipGraphNodeType GraphNodeType;
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipStreamCaptureMode streamCaptureMode;
  hipStreamCaptureMode streamCaptureMode;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamBeginCapture(hipStream_t stream, enum hipStreamCaptureMode mode);
  // HIP: hipError_t hipStreamBeginCapture(hipStream_t stream, hipStreamCaptureMode mode);
  // CHECK: result = hipStreamBeginCapture(stream, streamCaptureMode);
  result = hipStreamBeginCapture(stream, streamCaptureMode);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipThreadExchangeStreamCaptureMode(enum hipStreamCaptureMode *mode);
  // HIP: hipError_t hipThreadExchangeStreamCaptureMode(hipStreamCaptureMode* mode);
  // CHECK: result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);
  result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamGetCaptureInfo(hipStream_t stream, enum hipStreamCaptureStatus *pCaptureStatus, unsigned long long *pId);
  // HIP: hipError_t hipStreamGetCaptureInfo(hipStream_t stream, hipStreamCaptureStatus* pCaptureStatus, unsigned long long* pId);
  // CHECK: result = hipStreamGetCaptureInfo(stream, &StreamCaptureStatus, &ull_2);
  result = hipStreamGetCaptureInfo(stream, &StreamCaptureStatus, &ull_2);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipThreadExchangeStreamCaptureMode(enum hipStreamCaptureMode *mode);
  // HIP: hipError_t hipThreadExchangeStreamCaptureMode(hipStreamCaptureMode* mode);
  // CHECK: result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);
  result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);
#endif

#if CUDA_VERSION >= 10020
  // CHECK: hipGraphExecUpdateResult GraphExecUpdateResult;
  hipGraphExecUpdateResult GraphExecUpdateResult;
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipKernelNodeAttrID kernelNodeAttrID;
  hipKernelNodeAttrID kernelNodeAttrID;
  // CHECK: hipKernelNodeAttrValue kernelNodeAttrValue;
  hipKernelNodeAttrValue kernelNodeAttrValue;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphKernelNodeSetAttribute(hipGraphNode_t hNode, enum hipKernelNodeAttrID attr, const union hipKernelNodeAttrValue* value);
  // HIP: hipError_t hipGraphKernelNodeSetAttribute(hipGraphNode_t hNode, hipKernelNodeAttrID attr, const hipKernelNodeAttrValue* value);
  // CHECK: result = hipGraphKernelNodeSetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);
  result = hipGraphKernelNodeSetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphKernelNodeGetAttribute(hipGraphNode_t hNode, enum hipKernelNodeAttrID attr, union hipKernelNodeAttrValue* value_out);
  // HIP: hipError_t hipGraphKernelNodeGetAttribute(hipGraphNode_t hNode, hipKernelNodeAttrID attr, hipKernelNodeAttrValue* value);
  // CHECK: result = hipGraphKernelNodeGetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);
  result = hipGraphKernelNodeGetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExecKernelNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t node, const struct hipKernelNodeParams *pNodeParams);
  // HIP: hipError_t hipGraphExecKernelNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t node, const hipKernelNodeParams* pNodeParams);
  // CHECK: result = hipGraphExecKernelNodeSetParams(GraphExec_t, graphNode, &KernelNodeParams);
  result = hipGraphExecKernelNodeSetParams(GraphExec_t, graphNode, &KernelNodeParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExecMemcpyNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t node, const struct hipMemcpy3DParms *pNodeParams);
  // HIP: hipError_t hipGraphExecMemcpyNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t node, hipMemcpy3DParms* pNodeParams);
  // CHECK: result = hipGraphExecMemcpyNodeSetParams(GraphExec_t, graphNode, &Memcpy3DParms);
  result = hipGraphExecMemcpyNodeSetParams(GraphExec_t, graphNode, &Memcpy3DParms);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExecMemsetNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t node, const struct hipMemsetParams *pNodeParams);
  // HIP: hipError_t hipGraphExecMemsetNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t node, const hipMemsetParams* pNodeParams);
  // CHECK: result = hipGraphExecMemsetNodeSetParams(GraphExec_t, graphNode, &MemsetParams);
  result = hipGraphExecMemsetNodeSetParams(GraphExec_t, graphNode, &MemsetParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExecHostNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t node, const struct hipHostNodeParams *pNodeParams);
  // HIP: hipError_t hipError_t hipGraphExecHostNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t node, const hipHostNodeParams* pNodeParams);
  // CHECK: result = hipGraphExecHostNodeSetParams(GraphExec_t, graphNode, &HostNodeParams);
  result = hipGraphExecHostNodeSetParams(GraphExec_t, graphNode, &HostNodeParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExecUpdate(hipGraphExec_t hGraphExec, hipGraph_t hGraph, hipGraphNode_t *hErrorNode_out, enum hipGraphExecUpdateResult *updateResult_out);
  // HIP: hipError_t hipGraphExecUpdate(hipGraphExec_t hGraphExec, hipGraph_t hGraph, hipGraphNode_t* hErrorNode_out, hipGraphExecUpdateResult* updateResult_out);
  // CHECK: result = hipGraphExecUpdate(GraphExec_t, Graph_t, &graphNode, &GraphExecUpdateResult);
  result = hipGraphExecUpdate(GraphExec_t, Graph_t, &graphNode, &GraphExecUpdateResult);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphKernelNodeGetParams(hipGraphNode_t node, struct hipKernelNodeParams *pNodeParams);
  // HIP: hipError_t hipGraphKernelNodeGetParams(hipGraphNode_t node, hipKernelNodeParams* pNodeParams);
  // CHECK: result = hipGraphKernelNodeGetParams(graphNode, &KernelNodeParams);
  result = hipGraphKernelNodeGetParams(graphNode, &KernelNodeParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphKernelNodeSetParams(hipGraphNode_t node, const struct hipKernelNodeParams *pNodeParams);
  // HIP: hipError_t hipGraphKernelNodeSetParams(hipGraphNode_t node, const hipKernelNodeParams* pNodeParams);
  // CHECK: result = hipGraphKernelNodeSetParams(graphNode, &KernelNodeParams);
  result = hipGraphKernelNodeSetParams(graphNode, &KernelNodeParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphLaunch(hipGraphExec_t graphExec, hipStream_t stream);
  // HIP: hipError_t hipGraphLaunch(hipGraphExec_t graphExec, hipStream_t stream);
  // CHECK: result = hipGraphLaunch(GraphExec_t, stream);
  result = hipGraphLaunch(GraphExec_t, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphMemcpyNodeGetParams(hipGraphNode_t node, struct hipMemcpy3DParms *pNodeParams);
  // HIP: hipError_t hipGraphMemcpyNodeGetParams(hipGraphNode_t node, hipMemcpy3DParms* pNodeParams);
  // CHECK: result = hipGraphMemcpyNodeGetParams(graphNode, &Memcpy3DParms);
  result = hipGraphMemcpyNodeGetParams(graphNode, &Memcpy3DParms);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphMemcpyNodeSetParams(hipGraphNode_t node, const struct hipMemcpy3DParms *pNodeParams);
  // HIP: hipError_t hipGraphMemcpyNodeSetParams(hipGraphNode_t node, const hipMemcpy3DParms* pNodeParams);
  // CHECK: result = hipGraphMemcpyNodeSetParams(graphNode, &Memcpy3DParms);
  result = hipGraphMemcpyNodeSetParams(graphNode, &Memcpy3DParms);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphMemsetNodeGetParams(hipGraphNode_t node, struct hipMemsetParams *pNodeParams);
  // HIP: hipError_t hipGraphMemsetNodeGetParams(hipGraphNode_t node, hipMemsetParams* pNodeParams);
  // CHECK: result = hipGraphMemsetNodeGetParams(graphNode, &MemsetParams);
  result = hipGraphMemsetNodeGetParams(graphNode, &MemsetParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphMemsetNodeSetParams(hipGraphNode_t node, const struct hipMemsetParams *pNodeParams);
  // HIP: hipError_t hipGraphMemsetNodeSetParams(hipGraphNode_t node, const hipMemsetParams* pNodeParams);
  // CHECK: result = hipGraphMemsetNodeSetParams(graphNode, &MemsetParams);
  result = hipGraphMemsetNodeSetParams(graphNode, &MemsetParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphNodeFindInClone(hipGraphNode_t *pNode, hipGraphNode_t originalNode, hipGraph_t clonedGraph);
  // HIP: hipError_t hipError_t hipGraphNodeFindInClone(hipGraphNode_t* pNode, hipGraphNode_t originalNode, hipGraph_t clonedGraph);
  // CHECK: result = hipGraphNodeFindInClone(&graphNode, graphNode_2, Graph_t);
  result = hipGraphNodeFindInClone(&graphNode, graphNode_2, Graph_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphNodeGetDependencies(hipGraphNode_t node, hipGraphNode_t *pDependencies, size_t *pNumDependencies);
  // HIP: hipError_t hipGraphNodeGetDependencies(hipGraphNode_t node, hipGraphNode_t* pDependencies, size_t* pNumDependencies);
  // CHECK: result = hipGraphNodeGetDependencies(graphNode, &graphNode_2, &bytes);
  result = hipGraphNodeGetDependencies(graphNode, &graphNode_2, &bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphNodeGetDependentNodes(hipGraphNode_t node, hipGraphNode_t *pDependentNodes, size_t *pNumDependentNodes);
  // HIP: hipError_t hipGraphNodeGetDependentNodes(hipGraphNode_t node, hipGraphNode_t* pDependentNodes, size_t* pNumDependentNodes);
  // CHECK: result = hipGraphNodeGetDependentNodes(graphNode, &graphNode_2, &bytes);
  result = hipGraphNodeGetDependentNodes(graphNode, &graphNode_2, &bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphNodeGetType(hipGraphNode_t node, enum hipGraphNodeType *pType);
  // HIP: hipError_t hipGraphNodeGetType(hipGraphNode_t node, hipGraphNodeType* pType);
  // CHECK: result = hipGraphNodeGetType(graphNode, &GraphNodeType);
  result = hipGraphNodeGetType(graphNode, &GraphNodeType);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphRemoveDependencies(hipGraph_t graph, const hipGraphNode_t *from, const hipGraphNode_t *to, size_t numDependencies);
  // HIP: hipError_t hipGraphRemoveDependencies(hipGraph_t graph, const hipGraphNode_t* from, const hipGraphNode_t* to, size_t numDependencies);
  // CHECK: result = hipGraphRemoveDependencies(Graph_t, &graphNode, &graphNode, bytes);
  result = hipGraphRemoveDependencies(Graph_t, &graphNode, &graphNode, bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphKernelNodeCopyAttributes(hipGraphNode_t hSrc, hipGraphNode_t hDst);
  // HIP: hipError_t hipGraphKernelNodeCopyAttributes(hipGraphNode_t hSrc, hipGraphNode_t hDst);
  // CHECK: result = hipGraphKernelNodeCopyAttributes(graphNode, graphNode_2);
  result = hipGraphKernelNodeCopyAttributes(graphNode, graphNode_2);
#endif

#if CUDA_VERSION >= 11010
  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphAddMemcpyNode1D(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, void* dst, const void* src, size_t count, enum hipMemcpyKind kind);
  // HIP: hipError_t hipGraphAddMemcpyNode1D(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, void* dst, const void* src, size_t count, hipMemcpyKind kind);
  // CHECK: result = hipGraphAddMemcpyNode1D(&graphNode, Graph_t, &graphNode_2, width, dst, src, bytes, MemcpyKind);
  result = hipGraphAddMemcpyNode1D(&graphNode, Graph_t, &graphNode_2, width, dst, src, bytes, MemcpyKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphMemcpyNodeSetParams1D(hipGraphNode_t node, void* dst, const void* src, size_t count, enum hipMemcpyKind kind);
  // HIP: hipError_t hipGraphMemcpyNodeSetParams1D(hipGraphNode_t node, void* dst, const void* src, size_t count, hipMemcpyKind kind);
  // CHECK: result = hipGraphMemcpyNodeSetParams1D(graphNode, dst, src, bytes, MemcpyKind);
  result = hipGraphMemcpyNodeSetParams1D(graphNode, dst, src, bytes, MemcpyKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphAddEventRecordNode(hipGraphNode_t *pGraphNode, hipGraph_t graph, const hipGraphNode_t *pDependencies, size_t numDependencies, hipEvent_t event);
  // HIP: hipError_t hipGraphAddEventRecordNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, hipEvent_t event);
  // CHECK: result = hipGraphAddEventRecordNode(&graphNode, Graph_t, &graphNode_2, bytes, Event_t);
  result = hipGraphAddEventRecordNode(&graphNode, Graph_t, &graphNode_2, bytes, Event_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphEventRecordNodeGetEvent(hipGraphNode_t node, hipEvent_t *event_out);
  // HIP: hipError_t hipGraphEventRecordNodeGetEvent(hipGraphNode_t node, hipEvent_t* event_out);
  // CHECK: result = hipGraphEventRecordNodeGetEvent(graphNode, &Event_t);
  result = hipGraphEventRecordNodeGetEvent(graphNode, &Event_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphEventRecordNodeSetEvent(hipGraphNode_t node, hipEvent_t event);
  // HIP: hipError_t hipGraphEventRecordNodeSetEvent(hipGraphNode_t node, hipEvent_t event);
  // CHECK: result = hipGraphEventRecordNodeSetEvent(graphNode, Event_t);
  result = hipGraphEventRecordNodeSetEvent(graphNode, Event_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphAddEventWaitNode(hipGraphNode_t *pGraphNode, hipGraph_t graph, const hipGraphNode_t *pDependencies, size_t numDependencies, hipEvent_t event);
  // HIP: hipError_t hipGraphAddEventWaitNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, hipEvent_t event);
  // CHECK: result = hipGraphAddEventWaitNode(&graphNode, Graph_t, &graphNode_2, bytes, Event_t);
  result = hipGraphAddEventWaitNode(&graphNode, Graph_t, &graphNode_2, bytes, Event_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphEventWaitNodeGetEvent(hipGraphNode_t node, hipEvent_t *event_out);
  // HIP: hipError_t hipGraphEventWaitNodeGetEvent(hipGraphNode_t node, hipEvent_t* event_out);
  // CHECK: result = hipGraphEventWaitNodeGetEvent(graphNode, &Event_t);
  result = hipGraphEventWaitNodeGetEvent(graphNode, &Event_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphEventWaitNodeSetEvent(hipGraphNode_t node, hipEvent_t event);
  // HIP: hipError_t hipGraphEventWaitNodeSetEvent(hipGraphNode_t node, hipEvent_t event);
  // CHECK: result = hipGraphEventWaitNodeSetEvent(graphNode, Event_t);
  result = hipGraphEventWaitNodeSetEvent(graphNode, Event_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExecMemcpyNodeSetParams1D(hipGraphExec_t hGraphExec, hipGraphNode_t node, void* dst, const void* src, size_t count, enum hipMemcpyKind kind);
  // HIP: hipError_t hipGraphExecMemcpyNodeSetParams1D(hipGraphExec_t hGraphExec, hipGraphNode_t node, void* dst, const void* src, size_t count, hipMemcpyKind kind);
  // CHECK: result = hipGraphExecMemcpyNodeSetParams1D(GraphExec_t, graphNode, dst, src, bytes, MemcpyKind);
  result = hipGraphExecMemcpyNodeSetParams1D(GraphExec_t, graphNode, dst, src, bytes, MemcpyKind);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExecChildGraphNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t node, hipGraph_t childGraph);
  // HIP: hipError_t hipGraphExecChildGraphNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t node, hipGraph_t childGraph);
  // CHECK: result = hipGraphExecChildGraphNodeSetParams(GraphExec_t, graphNode, Graph_t);
  result = hipGraphExecChildGraphNodeSetParams(GraphExec_t, graphNode, Graph_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExecEventRecordNodeSetEvent(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, hipEvent_t event);
  // HIP: hipError_t hipGraphExecEventRecordNodeSetEvent(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, hipEvent_t event);
  // CHECK: result = hipGraphExecEventRecordNodeSetEvent(GraphExec_t, graphNode, Event_t);
  result = hipGraphExecEventRecordNodeSetEvent(GraphExec_t, graphNode, Event_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExecEventWaitNodeSetEvent(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, hipEvent_t event);
  // HIP: hipError_t hipGraphExecEventWaitNodeSetEvent(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, hipEvent_t event);
  // CHECK: result = hipGraphExecEventWaitNodeSetEvent(GraphExec_t, graphNode, Event_t);
  result = hipGraphExecEventWaitNodeSetEvent(GraphExec_t, graphNode, Event_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphUpload(hipGraphExec_t graphExec, hipStream_t stream);
  // HIP: hipError_t hipGraphUpload(hipGraphExec_t graphExec, hipStream_t stream);
  // CHECK: result = hipGraphUpload(GraphExec_t, stream);
  result = hipGraphUpload(GraphExec_t, stream);
#endif

#if CUDA_VERSION >= 11020
  // CHECK: hipMemPoolAttr memPoolAttr;
  hipMemPoolAttr memPoolAttr;
  // CHECK: hipMemAccessDesc memAccessDesc;
  hipMemAccessDesc memAccessDesc;
  // CHECK: hipMemAccessFlags memAccessFlags;
  hipMemAccessFlags memAccessFlags;
  // CHECK: hipMemLocation memLocation;
  hipMemLocation memLocation;
  // CHECK: hipMemPoolProps memPoolProps;
  hipMemPoolProps memPoolProps;
  // CHECK: hipMemPool_t memPool_t;
  hipMemPool_t memPool_t;
  // CHECK: hipMemAllocationHandleType memAllocationHandleType;
  hipMemAllocationHandleType memAllocationHandleType;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetDefaultMemPool(hipMemPool_t *memPool, int device);
  // HIP: hipError_t hipDeviceGetDefaultMemPool(hipMemPool_t* mem_pool, int device);
  // CHECK: result = hipDeviceGetDefaultMemPool(&memPool_t, device);
  result = hipDeviceGetDefaultMemPool(&memPool_t, device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetMemPool(int device, hipMemPool_t memPool);
  // HIP: hipError_t hipDeviceSetMemPool(int device, hipMemPool_t mem_pool);
  // CHECK: result = hipDeviceSetMemPool(device, memPool_t);
  result = hipDeviceSetMemPool(device, memPool_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetMemPool(hipMemPool_t *memPool, int device);
  // HIP: hipError_t hipDeviceGetMemPool(hipMemPool_t* mem_pool, int device);
  // CHECK: result = hipDeviceGetMemPool(&memPool_t, device);
  result = hipDeviceGetMemPool(&memPool_t, device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocAsync(void **devPtr, size_t size, hipStream_t hStream);
  // HIP: hipError_t hipMallocAsync(void** dev_ptr, size_t size, hipStream_t stream);
  // CHECK: result = hipMallocAsync(&deviceptr, bytes, stream);
  result = hipMallocAsync(&deviceptr, bytes, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFreeAsync(void *devPtr, hipStream_t hStream);
  // HIP: hipError_t hipFreeAsync(void* dev_ptr, hipStream_t stream);
  // CHECK: result = hipFreeAsync(deviceptr, stream);
  result = hipFreeAsync(deviceptr, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolTrimTo(hipMemPool_t memPool, size_t minBytesToKeep);
  // HIP: hipError_t hipMemPoolTrimTo(hipMemPool_t mem_pool, size_t min_bytes_to_hold);
  // CHECK: result = hipMemPoolTrimTo(memPool_t, bytes);
  result = hipMemPoolTrimTo(memPool_t, bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolSetAttribute(hipMemPool_t memPool, enum hipMemPoolAttr attr, void *value );
  // HIP: hipError_t hipMemPoolSetAttribute(hipMemPool_t mem_pool, hipMemPoolAttr attr, void* value);
  // CHECK: result = hipMemPoolSetAttribute(memPool_t, memPoolAttr, image);
  result = hipMemPoolSetAttribute(memPool_t, memPoolAttr, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolGetAttribute(hipMemPool_t memPool, enum hipMemPoolAttr attr, void *value );
  // HIP: hipError_t hipMemPoolGetAttribute(hipMemPool_t mem_pool, hipMemPoolAttr attr, void* value);
  // CHECK: result = hipMemPoolGetAttribute(memPool_t, memPoolAttr, image);
  result = hipMemPoolGetAttribute(memPool_t, memPoolAttr, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolSetAccess(hipMemPool_t memPool, const struct hipMemAccessDesc *descList, size_t count);
  // HIP: hipError_t hipMemPoolSetAccess(hipMemPool_t mem_pool, const hipMemAccessDesc* desc_list, size_t count);
  // CHECK: result = hipMemPoolSetAccess(memPool_t, &memAccessDesc, bytes);
  result = hipMemPoolSetAccess(memPool_t, &memAccessDesc, bytes);

  // CUDA: hipError_t extern __host__ hipError_t CUDARTAPI hipMemPoolGetAccess(enum hipMemAccessFlags *flags, hipMemPool_t memPool, struct hipMemLocation *location);
  // HIP: hipError_t hipMemPoolGetAccess(hipMemAccessFlags* flags, hipMemPool_t mem_pool, hipMemLocation* location);
  // CHECK: result = hipMemPoolGetAccess(&memAccessFlags, memPool_t, &memLocation);
  result = hipMemPoolGetAccess(&memAccessFlags, memPool_t, &memLocation);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolCreate(hipMemPool_t *memPool, const struct hipMemPoolProps *poolProps);
  // HIP: hipError_t hipMemPoolCreate(hipMemPool_t* mem_pool, const hipMemPoolProps* pool_props);
  // CHECK: result = hipMemPoolCreate(&memPool_t, &memPoolProps);
  result = hipMemPoolCreate(&memPool_t, &memPoolProps);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolDestroy(hipMemPool_t memPool);
  // HIP: hipError_t hipMemPoolDestroy(hipMemPool_t mem_pool);
  // CHECK: result = hipMemPoolDestroy(memPool_t);
  result = hipMemPoolDestroy(memPool_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocFromPoolAsync(void **ptr, size_t size, hipMemPool_t memPool, hipStream_t stream);
  // HIP: hipError_t hipMallocFromPoolAsync(void** dev_ptr, size_t size, hipMemPool_t mem_pool, hipStream_t stream);
  // CHECK: result = hipMallocFromPoolAsync(&deviceptr, bytes, memPool_t, stream);
  result = hipMallocFromPoolAsync(&deviceptr, bytes, memPool_t, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolExportToShareableHandle(void* shareableHandle, hipMemPool_t memPool, enum hipMemAllocationHandleType handleType, unsigned int flags);
  // HIP: hipError_t hipMemPoolExportToShareableHandle(void* shared_handle, hipMemPool_t mem_pool, hipMemAllocationHandleType handle_type, unsigned int flags);
  // CHECK: result = hipMemPoolExportToShareableHandle(image, memPool_t, memAllocationHandleType, ull);
  result = hipMemPoolExportToShareableHandle(image, memPool_t, memAllocationHandleType, ull);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolImportFromShareableHandle(hipMemPool_t* memPool, void* shareableHandle, enum hipMemAllocationHandleType handleType, unsigned int flags);
  // HIP: hipError_t hipMemPoolImportFromShareableHandle(hipMemPool_t* mem_pool, void* shared_handle, hipMemAllocationHandleType handle_type, unsigned int flags);
  // CHECK: result = hipMemPoolImportFromShareableHandle(&memPool_t, image, memAllocationHandleType, ull);
  result = hipMemPoolImportFromShareableHandle(&memPool_t, image, memAllocationHandleType, ull);

  // CHECK: hipMemPoolPtrExportData memPoolPtrExportData;
  hipMemPoolPtrExportData memPoolPtrExportData;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolExportPointer(struct hipMemPoolPtrExportData *exportData, void *ptr);
  // HIP: hipError_t hipMemPoolExportPointer(hipMemPoolPtrExportData* export_data, void* dev_ptr);
  // CHECK: result = hipMemPoolExportPointer(&memPoolPtrExportData, deviceptr);
  result = hipMemPoolExportPointer(&memPoolPtrExportData, deviceptr);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolImportPointer(void **ptr, hipMemPool_t memPool, struct hipMemPoolPtrExportData *exportData);
  // HIP: hipError_t hipMemPoolImportPointer(void** dev_ptr, hipMemPool_t mem_pool, hipMemPoolPtrExportData* export_data);
  // CHECK: result = hipMemPoolImportPointer(&deviceptr, memPool_t, &memPoolPtrExportData);
  result = hipMemPoolImportPointer(&deviceptr, memPool_t, &memPoolPtrExportData);

  // CHECK: hipExternalSemaphoreSignalNodeParams ExternalSemaphoreSignalNodeParams;
  hipExternalSemaphoreSignalNodeParams ExternalSemaphoreSignalNodeParams;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphAddExternalSemaphoresSignalNode(hipGraphNode_t *pGraphNode, hipGraph_t graph, const hipGraphNode_t *pDependencies, size_t numDependencies, const struct hipExternalSemaphoreSignalNodeParams *nodeParams);
  // HIP: hipError_t hipGraphAddExternalSemaphoresSignalNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, const hipExternalSemaphoreSignalNodeParams* nodeParams);
  // CHECK: result = hipGraphAddExternalSemaphoresSignalNode(&graphNode, Graph_t, &graphNode_2, bytes, &ExternalSemaphoreSignalNodeParams);
  result = hipGraphAddExternalSemaphoresSignalNode(&graphNode, Graph_t, &graphNode_2, bytes, &ExternalSemaphoreSignalNodeParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExternalSemaphoresSignalNodeGetParams(hipGraphNode_t hNode, struct hipExternalSemaphoreSignalNodeParams *params_out);
  // HIP: hipError_t hipGraphExternalSemaphoresSignalNodeGetParams(hipGraphNode_t hNode, hipExternalSemaphoreSignalNodeParams* params_out);
  // CHECK: result = hipGraphExternalSemaphoresSignalNodeGetParams(graphNode, &ExternalSemaphoreSignalNodeParams);
  result = hipGraphExternalSemaphoresSignalNodeGetParams(graphNode, &ExternalSemaphoreSignalNodeParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExternalSemaphoresSignalNodeSetParams(hipGraphNode_t hNode, const struct hipExternalSemaphoreSignalNodeParams *nodeParams);
  // HIP: hipError_t hipGraphExternalSemaphoresSignalNodeSetParams(hipGraphNode_t hNode, const hipExternalSemaphoreSignalNodeParams* nodeParams);
  // CHECK: result = hipGraphExternalSemaphoresSignalNodeSetParams(graphNode, &ExternalSemaphoreSignalNodeParams);
  result = hipGraphExternalSemaphoresSignalNodeSetParams(graphNode, &ExternalSemaphoreSignalNodeParams);

  // CHECK: hipExternalSemaphoreWaitNodeParams ExternalSemaphoreWaitNodeParams;
  hipExternalSemaphoreWaitNodeParams ExternalSemaphoreWaitNodeParams;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphAddExternalSemaphoresWaitNode(hipGraphNode_t *pGraphNode, hipGraph_t graph, const hipGraphNode_t *pDependencies, size_t numDependencies, const struct hipExternalSemaphoreWaitNodeParams *nodeParams);
  // HIP: hipError_t hipGraphAddExternalSemaphoresWaitNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, const hipExternalSemaphoreWaitNodeParams* nodeParams);
  // CHECK: result = hipGraphAddExternalSemaphoresWaitNode(&graphNode, Graph_t, &graphNode_2, bytes, &ExternalSemaphoreWaitNodeParams);
  result = hipGraphAddExternalSemaphoresWaitNode(&graphNode, Graph_t, &graphNode_2, bytes, &ExternalSemaphoreWaitNodeParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExternalSemaphoresWaitNodeGetParams(hipGraphNode_t hNode, struct hipExternalSemaphoreWaitNodeParams *params_out);
  // HIP: hipError_t hipGraphExternalSemaphoresWaitNodeGetParams(hipGraphNode_t hNode, hipExternalSemaphoreWaitNodeParams* params_out);
  // CHECK: result = hipGraphExternalSemaphoresWaitNodeGetParams(graphNode, &ExternalSemaphoreWaitNodeParams);
  result = hipGraphExternalSemaphoresWaitNodeGetParams(graphNode, &ExternalSemaphoreWaitNodeParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExternalSemaphoresWaitNodeSetParams(hipGraphNode_t hNode, const struct hipExternalSemaphoreWaitNodeParams *nodeParams);
  // HIP: hipError_t hipGraphExternalSemaphoresWaitNodeSetParams(hipGraphNode_t hNode, const hipExternalSemaphoreWaitNodeParams* nodeParams);
  // CHECK: result = hipGraphExternalSemaphoresWaitNodeSetParams(graphNode, &ExternalSemaphoreWaitNodeParams);
  result = hipGraphExternalSemaphoresWaitNodeSetParams(graphNode, &ExternalSemaphoreWaitNodeParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExecExternalSemaphoresSignalNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, const struct hipExternalSemaphoreSignalNodeParams *nodeParams);
  // HIP: hipError_t hipGraphExecExternalSemaphoresSignalNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, const hipExternalSemaphoreSignalNodeParams* nodeParams);
  // CHECK: result = hipGraphExecExternalSemaphoresSignalNodeSetParams(GraphExec_t, graphNode, &ExternalSemaphoreSignalNodeParams);
  result = hipGraphExecExternalSemaphoresSignalNodeSetParams(GraphExec_t, graphNode, &ExternalSemaphoreSignalNodeParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphExecExternalSemaphoresWaitNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, const struct hipExternalSemaphoreWaitNodeParams *nodeParams);
  // HIP: hipError_t hipGraphExecExternalSemaphoresWaitNodeSetParams(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, const hipExternalSemaphoreWaitNodeParams* nodeParams);
  // CHECK: result = hipGraphExecExternalSemaphoresWaitNodeSetParams(GraphExec_t, graphNode, &ExternalSemaphoreWaitNodeParams);
  result = hipGraphExecExternalSemaphoresWaitNodeSetParams(GraphExec_t, graphNode, &ExternalSemaphoreWaitNodeParams);
#endif

#if CUDA_VERSION >= 11030
  // CHECK: hipUserObject_t userObject;
  hipUserObject_t userObject;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipUserObjectCreate(hipUserObject_t *object_out, void *ptr, hipHostFn_t destroy, unsigned int initialRefcount, unsigned int flags);
  // HIP: hipError_t hipUserObjectCreate(hipUserObject_t* object_out, void* ptr, hipHostFn_t destroy, unsigned int initialRefcount, unsigned int flags);
  // CHECK: result = hipUserObjectCreate(&userObject, image, hostFn, count, flags);
  result = hipUserObjectCreate(&userObject, image, hostFn, count, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipUserObjectRelease(hipUserObject_t object, unsigned int count __dv(1));
  // HIP: hipError_t hipUserObjectRelease(hipUserObject_t object, unsigned int count);
  // CHECK: result = hipUserObjectRelease(userObject, count);
  result = hipUserObjectRelease(userObject, count);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipUserObjectRetain(hipUserObject_t object, unsigned int count __dv(1));
  // HIP: hipError_t hipUserObjectRetain(hipUserObject_t object, unsigned int count);
  // CHECK: result = hipUserObjectRetain(userObject, count);
  result = hipUserObjectRetain(userObject, count);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphRetainUserObject(hipGraph_t graph, hipUserObject_t object, unsigned int count __dv(1), unsigned int flags __dv(0));
  // HIP: hipError_t hipGraphRetainUserObject(hipGraph_t graph, hipUserObject_t object, unsigned int count, unsigned int flags);
  // CHECK: result = hipGraphRetainUserObject(Graph_t, userObject, count, flags);
  result = hipGraphRetainUserObject(Graph_t, userObject, count, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphReleaseUserObject(hipGraph_t graph, hipUserObject_t object, unsigned int count __dv(1));
  // HIP: hipError_t hipGraphReleaseUserObject(hipGraph_t graph, hipUserObject_t object, unsigned int count);
  // CHECK: result = hipGraphReleaseUserObject(Graph_t, userObject, count);
  result = hipGraphReleaseUserObject(Graph_t, userObject, count);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphDebugDotPrint(hipGraph_t graph, const char *path, unsigned int flags);
  // HIP: hipError_t hipGraphDebugDotPrint(hipGraph_t graph, const char* path, unsigned int flags);
  // CHECK: result = hipGraphDebugDotPrint(Graph_t, name.c_str(), flags);
  result = hipGraphDebugDotPrint(Graph_t, name.c_str(), flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamUpdateCaptureDependencies(hipStream_t stream, hipGraphNode_t *dependencies, size_t numDependencies, unsigned int flags __dv(0));
  // HIP: hipError_t hipStreamUpdateCaptureDependencies(hipStream_t stream, hipGraphNode_t* dependencies, size_t numDependencies, unsigned int flags __dparm(0));
  // CHECK: result = hipStreamUpdateCaptureDependencies(stream, &graphNode, bytes, flags);
  result = hipStreamUpdateCaptureDependencies(stream, &graphNode, bytes, flags);
#endif

#if CUDA_VERSION >= 11040
  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphInstantiateWithFlags(hipGraphExec_t *pGraphExec, hipGraph_t graph, unsigned long long flags);
  // HIP: hipError_t hipGraphInstantiateWithFlags(hipGraphExec_t* pGraphExec, hipGraph_t graph, unsigned long long flags);
  // CHECK: result = hipGraphInstantiateWithFlags(&GraphExec_t, Graph_t, ull);
  result = hipGraphInstantiateWithFlags(&GraphExec_t, Graph_t, ull);

  // CHECK: hipGraphMemAttributeType GraphMemAttributeType;
  hipGraphMemAttributeType GraphMemAttributeType;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetGraphMemAttribute(int device, enum hipGraphMemAttributeType attr, void* value);
  // HIP: hipError_t hipDeviceGetGraphMemAttribute(int device, hipGraphMemAttributeType attr, void* value);
  // CHECK: result = hipDeviceGetGraphMemAttribute(device, GraphMemAttributeType, image);
  result = hipDeviceGetGraphMemAttribute(device, GraphMemAttributeType, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetGraphMemAttribute(int device, enum hipGraphMemAttributeType attr, void* value);
  // HIP: hipError_t hipDeviceSetGraphMemAttribute(int device, hipGraphMemAttributeType attr, void* value);
  // CHECK: result = hipDeviceSetGraphMemAttribute(device, GraphMemAttributeType, image);
  result = hipDeviceSetGraphMemAttribute(device, GraphMemAttributeType, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGraphMemTrim(int device);
  // HIP: hipError_t hipDeviceGraphMemTrim(int device);
  // CHECK: result = hipDeviceGraphMemTrim(device);
  result = hipDeviceGraphMemTrim(device);

  // CHECK: hipMemAllocNodeParams MemAllocNodeParams;
  hipMemAllocNodeParams MemAllocNodeParams;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphAddMemAllocNode(hipGraphNode_t *pGraphNode, hipGraph_t graph, const hipGraphNode_t *pDependencies, size_t numDependencies, struct hipMemAllocNodeParams *nodeParams);
  // HIP: hipError_t hipGraphAddMemAllocNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, hipMemAllocNodeParams* pNodeParams);
  // CHECK: result = hipGraphAddMemAllocNode(&graphNode, Graph_t, &graphNode_2, bytes, &MemAllocNodeParams);
  result = hipGraphAddMemAllocNode(&graphNode, Graph_t, &graphNode_2, bytes, &MemAllocNodeParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphMemAllocNodeGetParams(hipGraphNode_t node, struct hipMemAllocNodeParams *params_out);
  // HIP: hipError_t hipGraphMemAllocNodeGetParams(hipGraphNode_t node, hipMemAllocNodeParams* pNodeParams);
  // CHECK: result = hipGraphMemAllocNodeGetParams(graphNode, &MemAllocNodeParams);
  result = hipGraphMemAllocNodeGetParams(graphNode, &MemAllocNodeParams);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphAddMemFreeNode(hipGraphNode_t *pGraphNode, hipGraph_t graph, const hipGraphNode_t *pDependencies, size_t numDependencies, void *dptr);
  // HIP: hipError_t hipGraphAddMemFreeNode(hipGraphNode_t* pGraphNode, hipGraph_t graph, const hipGraphNode_t* pDependencies, size_t numDependencies, void* dev_ptr);
  // CHECK: result = hipGraphAddMemFreeNode(&graphNode, Graph_t, &graphNode_2, bytes, deviceptr);
  result = hipGraphAddMemFreeNode(&graphNode, Graph_t, &graphNode_2, bytes, deviceptr);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphMemFreeNodeGetParams(hipGraphNode_t node, void *dptr_out);
  // HIP: hipError_t hipGraphMemFreeNodeGetParams(hipGraphNode_t node, void* dev_ptr);
  // CHECK: result = hipGraphMemFreeNodeGetParams(graphNode, &deviceptr);
  result = hipGraphMemFreeNodeGetParams(graphNode, &deviceptr);
#endif

#if CUDA_VERSION >= 11060
  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphNodeSetEnabled(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, unsigned int isEnabled);
  // HIP: hipError_t hipGraphNodeSetEnabled(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, unsigned int isEnabled);
  // CHECK: result = hipGraphNodeSetEnabled(GraphExec_t, graphNode, flags);
  result = hipGraphNodeSetEnabled(GraphExec_t, graphNode, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphNodeGetEnabled(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, unsigned int *isEnabled);
  // HIP: hipError_t hipGraphNodeGetEnabled(hipGraphExec_t hGraphExec, hipGraphNode_t hNode, unsigned int* isEnabled);
  // CHECK: result = hipGraphNodeGetEnabled(GraphExec_t, graphNode, &flags);
  result = hipGraphNodeGetEnabled(GraphExec_t, graphNode, &flags);
#endif

#if CUDA_VERSION < 12000
  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipBindTexture(size_t *offset, const struct textureReference *texref, const void *devPtr, const struct hipChannelFormatDesc *desc, size_t size __dv(UINT_MAX));
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipBindTexture(size_t* offset, const textureReference* tex, const void* devPtr, const hipChannelFormatDesc* desc, size_t size __dparm(UINT_MAX));
  // CHECK: result = hipBindTexture(&wOffset, texref, deviceptr, &ChannelFormatDesc, bytes);
  result = hipBindTexture(&wOffset, texref, deviceptr, &ChannelFormatDesc, bytes);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipBindTexture2D(size_t *offset, const struct textureReference *texref, const void *devPtr, const struct hipChannelFormatDesc *desc, size_t width, size_t height, size_t pitch);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipBindTexture2D(size_t* offset, const textureReference* tex, const void* devPtr, const hipChannelFormatDesc* desc, size_t width, size_t height, size_t pitch);
  // CHECK: result = hipBindTexture2D(&wOffset, texref, deviceptr, &ChannelFormatDesc, width, height, pitch);
  result = hipBindTexture2D(&wOffset, texref, deviceptr, &ChannelFormatDesc, width, height, pitch);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipBindTextureToArray(const struct textureReference *texref, hipArray_const_t array, const struct hipChannelFormatDesc *desc);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipBindTextureToArray(const textureReference* tex, hipArray_const_t array, const hipChannelFormatDesc* desc);
  // CHECK: result = hipBindTextureToArray(texref, Array_const_t, &ChannelFormatDesc);
  result = hipBindTextureToArray(texref, Array_const_t, &ChannelFormatDesc);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipBindTextureToMipmappedArray(const struct textureReference *texref, hipMipmappedArray_const_t mipmappedArray, const struct hipChannelFormatDesc *desc);
  // HIP: hipError_t hipBindTextureToMipmappedArray(const textureReference* tex, hipMipmappedArray_const_t mipmappedArray, const hipChannelFormatDesc* desc);
  // CHECK: result = hipBindTextureToMipmappedArray(texref, MipmappedArray_const_t, &ChannelFormatDesc);
  result = hipBindTextureToMipmappedArray(texref, MipmappedArray_const_t, &ChannelFormatDesc);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipGetTextureAlignmentOffset(size_t *offset, const struct textureReference *texref);
  // HIP: hipError_t hipGetTextureAlignmentOffset(size_t* offset, const textureReference* texref);
  // CHECK: result = hipGetTextureAlignmentOffset(&wOffset, texref);
  result = hipGetTextureAlignmentOffset(&wOffset, texref);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipGetTextureReference(const struct textureReference **texref, HIP_SYMBOL(const void *symbol));
  // HIP: hipError_t hipGetTextureReference(const textureReference** texref, HIP_SYMBOL(const void* symbol));
  // CHECK: result = hipGetTextureReference(const_cast<const textureReference**>(&texref), {{(HIP_SYMBOL\()?}}image{{(\))?}});
  result = hipGetTextureReference(const_cast<const textureReference**>(&texref), image);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipUnbindTexture(const struct textureReference *texref);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipUnbindTexture(const textureReference* tex);
  // CHECK: result = hipUnbindTexture(texref);
  result = hipUnbindTexture(texref);
#endif

  return 0;
}
