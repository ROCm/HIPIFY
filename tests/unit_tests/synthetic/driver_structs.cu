// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
  printf("01. CUDA Driver API Structs synthetic test\n");

  // CHECK: HIP_ARRAY3D_DESCRIPTOR array3D_st;
  // CHECK-NEXT: HIP_ARRAY3D_DESCRIPTOR array3D;
  HIP_ARRAY3D_DESCRIPTOR array3D_st;
  HIP_ARRAY3D_DESCRIPTOR array3D;

  // CHECK: HIP_ARRAY_DESCRIPTOR array_descr_st;
  // CHECK-NEXT: HIP_ARRAY_DESCRIPTOR array_descr;
  HIP_ARRAY_DESCRIPTOR array_descr_st;
  HIP_ARRAY_DESCRIPTOR array_descr;
#define __CUDA_API_VERSION_INTERNAL
  // CHECK: HIP_ARRAY_DESCRIPTOR array_descr_v1_st;
  // CHECK-NEXT: HIP_ARRAY_DESCRIPTOR array_descr_v1;
  HIP_ARRAY_DESCRIPTOR array_descr_v1_st;
  HIP_ARRAY_DESCRIPTOR array_descr_v1;
#undef __CUDA_API_VERSION_INTERNAL

  // CHECK: hip_Memcpy2D memcpy2D_st;
  // CHECK-NEXT: hip_Memcpy2D memcpy2D_v1_st;
  // CHECK-NEXT: hip_Memcpy2D memcpy2D;
  // CHECK-NEXT: hip_Memcpy2D memcpy2D_v1;
  hip_Memcpy2D memcpy2D_st;
  hip_Memcpy2D memcpy2D_v1_st;
  hip_Memcpy2D memcpy2D;
  hip_Memcpy2D memcpy2D_v1;

  // CHECK: HIP_MEMCPY3D memcpy3D_st;
  // CHECK-NEXT: HIP_MEMCPY3D memcpy3D_v1_st;
  // CHECK-NEXT: HIP_MEMCPY3D memcpy3D;
  // CHECK-NEXT: HIP_MEMCPY3D memcpy3D_v1;
  HIP_MEMCPY3D memcpy3D_st;
  HIP_MEMCPY3D memcpy3D_v1_st;
  HIP_MEMCPY3D memcpy3D;
  HIP_MEMCPY3D memcpy3D_v1;

  // CHECK: HIP_RESOURCE_DESC_st res_descr_st;
  // CHECK-NEXT: HIP_RESOURCE_DESC res_descr;
  HIP_RESOURCE_DESC_st res_descr_st;
  HIP_RESOURCE_DESC res_descr;

  // CHECK: HIP_RESOURCE_VIEW_DESC_st res_view_descr_st;
  // CHECK-NEXT: HIP_RESOURCE_VIEW_DESC res_view_descr;
  HIP_RESOURCE_VIEW_DESC_st res_view_descr_st;
  HIP_RESOURCE_VIEW_DESC res_view_descr;

  // CHECK: HIP_TEXTURE_DESC_st tex_descr_st;
  // CHECK-NEXT: HIP_TEXTURE_DESC tex_descr;
  HIP_TEXTURE_DESC_st tex_descr_st;
  HIP_TEXTURE_DESC tex_descr;

  // CHECK: hipIpcMemHandle_st ipc_mem_handle_st;
  // CHECK-NEXT: hipIpcMemHandle_t ipc_mem_handle;
  hipIpcMemHandle_st ipc_mem_handle_st;
  hipIpcMemHandle_t ipc_mem_handle;

  // CHECK: hipArray* array_st_ptr;
  // CHECK-NEXT: hipArray_t array_ptr;
  hipArray* array_st_ptr;
  hipArray_t array_ptr;

  // CHECK: ihipCtx_t* ctx_st_ptr;
  // CHECK-NEXT: hipCtx_t ctx;
  ihipCtx_t* ctx_st_ptr;
  hipCtx_t ctx;

  // CHECK: ihipEvent_t* evnt_st_ptr;
  // CHECK-NEXT: hipEvent_t evnt;
  ihipEvent_t* evnt_st_ptr;
  hipEvent_t evnt;

  // CHECK: ihipModuleSymbol_t* func_st_ptr;
  // CHECK-NEXT: hipFunction_t func;
  ihipModuleSymbol_t* func_st_ptr;
  hipFunction_t func;

  // CHECK: hipMipmappedArray* mipmapped_array_st_ptr;
  // CHECK-NEXT: hipMipmappedArray_t mipmapped_array;
  hipMipmappedArray* mipmapped_array_st_ptr;
  hipMipmappedArray_t mipmapped_array;

  // CHECK: ihipStream_t* stream_st_ptr;
  // CHECK-NEXT: hipStream_t stream;
  ihipStream_t* stream_st_ptr;
  hipStream_t stream;

  // CHECK: textureReference* tex_ref_st_ptr;
  // CHECK-NEXT: hipTexRef tex_ref;
  textureReference* tex_ref_st_ptr;
  hipTexRef tex_ref;

  // CHECK: hipGraphicsResource* graphicsResource_st;
  // CHECK-NEXT: hipGraphicsResource_t graphicsResource;
  hipGraphicsResource* graphicsResource_st;
  hipGraphicsResource_t graphicsResource;

  // CHECK: hipUUID_t uuid_st;
  hipUUID_t uuid_st;

  // CHECK: ihiprtcLinkState* linkState_ptr;
  // CHECK-NEXT: hiprtcLinkState linkState;
  ihiprtcLinkState* linkState_ptr;
  hiprtcLinkState linkState;

#if CUDA_VERSION >= 9000
  // CHECK: hipFunctionLaunchParams_t LAUNCH_PARAMS_st;
  // CHECK-NEXT: hipFunctionLaunchParams LAUNCH_PARAMS;
  hipFunctionLaunchParams_t LAUNCH_PARAMS_st;
  hipFunctionLaunchParams LAUNCH_PARAMS;
#endif

#if CUDA_VERSION >= 10000
  // CHECK: hipExternalMemoryBufferDesc_st ext_mem_buff_st;
  // CHECK-NEXT: hipExternalMemoryBufferDesc ext_mem_buff;
  hipExternalMemoryBufferDesc_st ext_mem_buff_st;
  hipExternalMemoryBufferDesc ext_mem_buff;

  // CHECK: hipExternalMemoryHandleDesc_st ext_mem_handle_st;
  // CHECK-NEXT: hipExternalMemoryHandleDesc ext_mem_handle;
  hipExternalMemoryHandleDesc_st ext_mem_handle_st;
  hipExternalMemoryHandleDesc ext_mem_handle;

  // CHECK: hipExternalSemaphoreHandleDesc_st ext_sema_handle_st;
  // CHECK-NEXT: hipExternalSemaphoreHandleDesc ext_sema_handle;
  hipExternalSemaphoreHandleDesc_st ext_sema_handle_st;
  hipExternalSemaphoreHandleDesc ext_sema_handle;

  // CHECK: hipExternalSemaphoreSignalParams_st ext_sema_params_st;
  // CHECK-NEXT: hipExternalSemaphoreSignalParams ext_sema_params;
  hipExternalSemaphoreSignalParams_st ext_sema_params_st;
  hipExternalSemaphoreSignalParams ext_sema_params;

  // CHECK: hipHostNodeParams host_node_params_st;
  // CHECK-NEXT: hipHostNodeParams host_node_params;
  hipHostNodeParams host_node_params_st;
  hipHostNodeParams host_node_params;

  // CHECK: hipKernelNodeParams kern_node_params_st;
  // CHECK-NEXT: hipKernelNodeParams kern_node_params;
  hipKernelNodeParams kern_node_params_st;
  hipKernelNodeParams kern_node_params;

  // CHECK: hipExternalMemory_t ext_mem;
  hipExternalMemory_t ext_mem;

  // CHECK: hipExternalSemaphore_t ext_sema;
  hipExternalSemaphore_t ext_sema;

  // CHECK: ihipGraph* graph_st;
  // CHECK-NEXT: hipGraph_t graph;
  ihipGraph* graph_st;
  hipGraph_t graph;

  // CHECK: hipGraphExec* graphExec_st;
  // CHECK-NEXT: hipGraphExec_t graphExec;
  hipGraphExec* graphExec_st;
  hipGraphExec_t graphExec;
#endif

#if CUDA_VERSION >= 10020
  // CHECK: hipMemAccessDesc memAccessDesc_st;
  // CHECK-NEXT: hipMemAccessDesc memAccessDesc;
  hipMemAccessDesc memAccessDesc_st;
  hipMemAccessDesc memAccessDesc;

  // CHECK: hipMemAllocationProp memAllocationProp_st;
  // CHECK-NEXT: hipMemAllocationProp memAllocationProp;
  hipMemAllocationProp memAllocationProp_st;
  hipMemAllocationProp memAllocationProp;
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipAccessPolicyWindow accessPolicyWindow_st;
  // CHECK-NEXT: hipAccessPolicyWindow accessPolicyWindow;
  hipAccessPolicyWindow accessPolicyWindow_st;
  hipAccessPolicyWindow accessPolicyWindow;
#endif

#if CUDA_VERSION >= 11010
  // CHECK: hipArrayMapInfo arrayMapInfo_st;
  // CHECK-NEXT: hipArrayMapInfo arrayMapInfo;
  hipArrayMapInfo arrayMapInfo_st;
  hipArrayMapInfo arrayMapInfo;
#endif

#if CUDA_VERSION >= 11020
  // CHECK: ihipMemPoolHandle_t* memPoolHandle_st;
  // CHECK-NEXT: hipMemPool_t memPool_t;
  ihipMemPoolHandle_t* memPoolHandle_st;
  hipMemPool_t memPool_t;

  // CHECK: hipMemLocation memLocation_st;
  // CHECK-NEXT: hipMemLocation memLocation;
  hipMemLocation memLocation_st;
  hipMemLocation memLocation;

  // CHECK: hipMemPoolProps memPoolProps_st;
  // CHECK-NEXT: hipMemPoolProps memPoolProps;
  hipMemPoolProps memPoolProps_st;
  hipMemPoolProps memPoolProps;

  // CHECK: hipMemPoolPtrExportData memPoolPtrExportData_st;
  // CHECK-NEXT: hipMemPoolPtrExportData memPoolPtrExportData;
  hipMemPoolPtrExportData memPoolPtrExportData_st;
  hipMemPoolPtrExportData memPoolPtrExportData;

  // CHECK: hipExternalSemaphoreSignalNodeParams EXT_SEM_SIGNAL_NODE_PARAMS_st;
  // CHECK-NEXT: hipExternalSemaphoreSignalNodeParams EXT_SEM_SIGNAL_NODE_PARAMS;
  hipExternalSemaphoreSignalNodeParams EXT_SEM_SIGNAL_NODE_PARAMS_st;
  hipExternalSemaphoreSignalNodeParams EXT_SEM_SIGNAL_NODE_PARAMS;

  // CHECK: hipExternalSemaphoreWaitNodeParams EXT_SEM_WAIT_NODE_PARAMS_st;
  // CHECK-NEXT: hipExternalSemaphoreWaitNodeParams EXT_SEM_WAIT_NODE_PARAMS;
  hipExternalSemaphoreWaitNodeParams EXT_SEM_WAIT_NODE_PARAMS_st;
  hipExternalSemaphoreWaitNodeParams EXT_SEM_WAIT_NODE_PARAMS;
#endif

#if CUDA_VERSION >= 11030
  // CHECK: HIP_ARRAY3D_DESCRIPTOR array3D_v2;
  HIP_ARRAY3D_DESCRIPTOR array3D_v2;

  // CHECK: HIP_ARRAY_DESCRIPTOR array_descr_v2;
  HIP_ARRAY_DESCRIPTOR array_descr_v2;

  // CHECK: hipExternalMemoryBufferDesc ext_mem_buff_v1;
  hipExternalMemoryBufferDesc ext_mem_buff_v1;

  // CHECK: hipExternalMemoryHandleDesc ext_mem_handle_v1;
  hipExternalMemoryHandleDesc ext_mem_handle_v1;

  // CHECK: hipExternalSemaphoreHandleDesc ext_sema_handle_v1;
  hipExternalSemaphoreHandleDesc ext_sema_handle_v1;

  // CHECK: hipExternalSemaphoreSignalParams ext_sema_params_v1;
  hipExternalSemaphoreSignalParams ext_sema_params_v1;

  // CHECK: hipHostNodeParams host_node_params_v1;
  hipHostNodeParams host_node_params_v1;

  // CHECK: hipKernelNodeParams kern_node_params_v1;
  hipKernelNodeParams kern_node_params_v1;

  // CHECK: hip_Memcpy2D memcpy2D_v2;
  hip_Memcpy2D memcpy2D_v2;

  // CHECK: HIP_MEMCPY3D memcpy3D_v2;
  HIP_MEMCPY3D memcpy3D_v2;

  // CHECK: HIP_RESOURCE_DESC res_descr_v1;
  HIP_RESOURCE_DESC res_descr_v1;

  // CHECK: HIP_RESOURCE_VIEW_DESC res_view_descr_v1;
  HIP_RESOURCE_VIEW_DESC res_view_descr_v1;

  // CHECK: HIP_TEXTURE_DESC tex_descr_v1;
  HIP_TEXTURE_DESC tex_descr_v1;

  // CHECK: hipIpcMemHandle_t ipc_mem_handle_v1;
  hipIpcMemHandle_t ipc_mem_handle_v1;

  // CHECK: hipMemLocation memLocation_v1;
  hipMemLocation memLocation_v1;

  // CHECK: hipUserObject* userObject_st_ptr;
  // CHECK-NEXT: hipUserObject_t userObject;
  hipUserObject* userObject_st_ptr;
  hipUserObject_t userObject;

  // CHECK: hipMemAccessDesc memAccessDesc_v1;
  hipMemAccessDesc memAccessDesc_v1;

  // CHECK: hipMemPoolProps memPoolProps_v1;
  hipMemPoolProps memPoolProps_v1;

  // CHECK: hipMemPoolPtrExportData memPoolPtrExportData_v1;
  hipMemPoolPtrExportData memPoolPtrExportData_v1;

  // CHECK: hipMemAllocationProp memAllocationProp_v1;
  hipMemAllocationProp memAllocationProp_v1;

  // CHECK: hipArrayMapInfo arrayMapInfo_v1;
  hipArrayMapInfo arrayMapInfo_v1;

  // CHECK: hipFunctionLaunchParams LAUNCH_PARAMS_v1;
  hipFunctionLaunchParams LAUNCH_PARAMS_v1;

  // CHECK: hipExternalSemaphoreSignalNodeParams EXT_SEM_SIGNAL_NODE_PARAMS_v1;
  hipExternalSemaphoreSignalNodeParams EXT_SEM_SIGNAL_NODE_PARAMS_v1;

  // CHECK: hipExternalSemaphoreWaitNodeParams EXT_SEM_WAIT_NODE_PARAMS_v1;
  hipExternalSemaphoreWaitNodeParams EXT_SEM_WAIT_NODE_PARAMS_v1;
#endif

#if CUDA_VERSION >= 11040
  // CHECK: hipMemAllocNodeParams MEM_ALLOC_NODE_PARAMS;
  hipMemAllocNodeParams MEM_ALLOC_NODE_PARAMS;
#endif

#if CUDA_VERSION >= 11040 && CUDA_VERSION < 12020
  // CHECK: hipMemAllocNodeParams MEM_ALLOC_NODE_PARAMS_st;
  hipMemAllocNodeParams MEM_ALLOC_NODE_PARAMS_st;
#endif

#if CUDA_VERSION >= 12000
  // CHECK: hipGraphInstantiateParams GRAPH_INSTANTIATE_PARAMS_st;
  // CHECK-NEXT: hipGraphInstantiateParams GRAPH_INSTANTIATE_PARAMS;
  hipGraphInstantiateParams GRAPH_INSTANTIATE_PARAMS_st;
  hipGraphInstantiateParams GRAPH_INSTANTIATE_PARAMS;
#endif

#if CUDA_VERSION >= 12020
  // CHECK: hipMemAllocNodeParams MEM_ALLOC_NODE_PARAMS_v1_st;
  // CHECK-NEXT: hipMemAllocNodeParams MEM_ALLOC_NODE_PARAMS_v1;
  hipMemAllocNodeParams MEM_ALLOC_NODE_PARAMS_v1_st;
  hipMemAllocNodeParams MEM_ALLOC_NODE_PARAMS_v1;

  // CHECK: hipExternalSemaphoreSignalNodeParams EXT_SEM_SIGNAL_NODE_PARAMS_v2_st;
  // CHECK-NEXT: hipExternalSemaphoreSignalNodeParams EXT_SEM_SIGNAL_NODE_PARAMS_v2;
  hipExternalSemaphoreSignalNodeParams EXT_SEM_SIGNAL_NODE_PARAMS_v2_st;
  hipExternalSemaphoreSignalNodeParams EXT_SEM_SIGNAL_NODE_PARAMS_v2;

  // CHECK: hipExternalSemaphoreWaitNodeParams EXT_SEM_WAIT_NODE_PARAMS_v2_st;
  // CHECK-NEXT: hipExternalSemaphoreWaitNodeParams EXT_SEM_WAIT_NODE_PARAMS_v2;
  hipExternalSemaphoreWaitNodeParams EXT_SEM_WAIT_NODE_PARAMS_v2_st;
  hipExternalSemaphoreWaitNodeParams EXT_SEM_WAIT_NODE_PARAMS_v2;

  // CHECK: hipMemcpyNodeParams MEMCPY_NODE_PARAMS_st;
  // CHECK-NEXT: hipMemcpyNodeParams MEMCPY_NODE_PARAMS;
  hipMemcpyNodeParams MEMCPY_NODE_PARAMS_st;
  hipMemcpyNodeParams MEMCPY_NODE_PARAMS;

  // CHECK: hipChildGraphNodeParams CHILD_GRAPH_NODE_PARAMS_st;
  // CHECK-NEXT: hipChildGraphNodeParams CHILD_GRAPH_NODE_PARAMS;
  hipChildGraphNodeParams CHILD_GRAPH_NODE_PARAMS_st;
  hipChildGraphNodeParams CHILD_GRAPH_NODE_PARAMS;

  // CHECK: hipMemFreeNodeParams MEM_FREE_NODE_PARAMS_st;
  // CHECK-NEXT: hipMemFreeNodeParams MEM_FREE_NODE_PARAMS;
  hipMemFreeNodeParams MEM_FREE_NODE_PARAMS_st;
  hipMemFreeNodeParams MEM_FREE_NODE_PARAMS;

  // CHECK: hipEventRecordNodeParams EVENT_RECORD_NODE_PARAMS_st;
  // CHECK-NEXT: hipEventRecordNodeParams EVENT_RECORD_NODE_PARAMS;
  hipEventRecordNodeParams EVENT_RECORD_NODE_PARAMS_st;
  hipEventRecordNodeParams EVENT_RECORD_NODE_PARAMS;

  // CHECK: hipEventWaitNodeParams EVENT_WAIT_NODE_PARAMS_st;
  // CHECK-NEXT: hipEventWaitNodeParams EVENT_WAIT_NODE_PARAMS;
  hipEventWaitNodeParams EVENT_WAIT_NODE_PARAMS_st;
  hipEventWaitNodeParams EVENT_WAIT_NODE_PARAMS;

  // CHECK: hipGraphNodeParams graphNodeParams_st;
  // CHECK-NEXT: hipGraphNodeParams graphNodeParams;
  hipGraphNodeParams graphNodeParams_st;
  hipGraphNodeParams graphNodeParams;
#endif

  return 0;
}
