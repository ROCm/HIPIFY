// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
  printf("11. CUDA Runtime API Unions synthetic test\n");

#if CUDA_VERSION >= 11000
  // CHECK: hipKernelNodeAttrValue KernelNodeAttrValue;
  hipKernelNodeAttrValue KernelNodeAttrValue;
#endif

  return 0;
}
