// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
  printf("10. CUDA Driver API Unions synthetic test\n");

#if CUDA_VERSION >= 11000
  // CHECK: hipKernelNodeAttrValue kernelNodeAttrValue;
  hipKernelNodeAttrValue kernelNodeAttrValue;
#endif

#if CUDA_VERSION >= 11000 && CUDA_VERSION < 11080
  // CHECK: hipKernelNodeAttrValue kernelNodeAttrValue_union;
  CUkernelNodeAttrValue_union kernelNodeAttrValue_union;
#endif

#if CUDA_VERSION >= 11030
  // CHECK: hipKernelNodeAttrValue kernelNodeAttrValue_v1;
  hipKernelNodeAttrValue kernelNodeAttrValue_v1;
#endif

  return 0;
}
