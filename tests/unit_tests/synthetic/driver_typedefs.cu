// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
  printf("03. CUDA Driver API Typedefs synthetic test\n");

  // CHECK: hipDevice_t device;
  hipDevice_t device;

  // CHECK: hipDeviceptr_t deviceptr;
  // CHECK-NEXT: hipDeviceptr_t deviceptr_v1;
  hipDeviceptr_t deviceptr;
  hipDeviceptr_t deviceptr_v1;

  // CHECK: hipStreamCallback_t streamCallback;
  hipStreamCallback_t streamCallback;

  // CHECK: hipSurfaceObject_t surfObject;
  hipSurfaceObject_t surfObject;

  // CHECK: hipTextureObject_t texObject;
  hipTextureObject_t texObject;

  // CHECK: hipUUID uuid;
  hipUUID uuid;

#if CUDA_VERSION >= 10000
  // CHECK: hipHostFn_t hostFn;
  hipHostFn_t hostFn;
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipMemGenericAllocationHandle_t memGenericAllocationHandle_t;
  hipMemGenericAllocationHandle_t memGenericAllocationHandle_t;
#endif

#if CUDA_VERSION >= 11030
  // CHECK: hipDevice_t device_v1;
  hipDevice_t device_v1;

  // CHECK: hipDeviceptr_t deviceptr_v2;
  hipDeviceptr_t deviceptr_v2;

  // CHECK: hipSurfaceObject_t surfObject_v1;
  hipSurfaceObject_t surfObject_v1;

  // CHECK: hipTextureObject_t texObject_v1;
  hipTextureObject_t texObject_v1;

  // CHECK: hipMemGenericAllocationHandle_t memGenericAllocationHandle_v1;
  hipMemGenericAllocationHandle_t memGenericAllocationHandle_v1;
#endif

  return 0;
}
