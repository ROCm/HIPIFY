// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18.1. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: rocsparse_status status_t;
  hipsparseStatus_t status_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;

  // CHECK: rocsparse_action action_t;
  hipsparseAction_t action_t;

  // CHECK: rocsparse_index_base indexBase_t;
  hipsparseIndexBase_t indexBase_t;

  int m = 0;
  int n = 0;
  int innz = 0;
  int csrRowPtrA = 0;
  int csrRowPtrB = 0;
  int csrRowPtrC = 0;
  int cscRowIndA = 0;
  int csrColIndA = 0;
  int csrColIndB = 0;
  int csrColIndC = 0;
  int cscColPtrA = 0;
  size_t bufferSize = 0;
  void *pcsrVal = nullptr;
  void *pcscVal = nullptr;

#if CUDA_VERSION >= 8000
  // TODO: [#899] There should be rocsparse_datatype instead of hipDataType
  hipDataType dataType_t;
  hipDataType dataType;
#endif

#if CUDA_VERSION >= 10010
  // TODO: cusparseCsr2CscAlg_t has no analogue in rocSPARSE. The deletion of declaration and usage is needed to be implemented
  hipsparseCsr2CscAlg_t Csr2CscAlg_t;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCsr2cscEx2_bufferSize(cusparseHandle_t handle, int m, int n, int nnz, const void* csrVal, const int* csrRowPtr, const int* csrColInd, void* cscVal, int* cscColPtr, int* cscRowInd, cudaDataType valType, cusparseAction_t copyValues, cusparseIndexBase_t idxBase, cusparseCsr2CscAlg_t alg, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_csr2csc_buffer_size(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_action copy_values, size_t* buffer_size);
  // CHECK: status_t = rocsparse_csr2csc_buffer_size(handle_t, m, n, innz, &csrRowPtrA, &csrColIndA, action_t, &bufferSize);
  status_t = hipsparseCsr2cscEx2_bufferSize(handle_t, m, n, innz, pcsrVal, &csrRowPtrA, &csrColIndA, pcscVal, &cscColPtrA, &cscRowIndA, dataType, action_t, indexBase_t, Csr2CscAlg_t, &bufferSize);
#endif

  return 0;
}
