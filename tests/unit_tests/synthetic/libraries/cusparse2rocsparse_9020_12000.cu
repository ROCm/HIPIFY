// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18.1. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: rocsparse_status status_t;
  hipsparseStatus_t status_t;

  int batchCount = 0;
  int m = 0;
  int algo = 0;
  int nrhs = 0;
  int innz = 0;
  int ldb = 0;
  int csrRowPtrA = 0;
  int csrColIndA = 0;
  int iposition = 0;
  double dds = 0.f;
  double ddl = 0.f;
  double dd = 0.f;
  double ddu = 0.f;
  double ddw = 0.f;
  double dx = 0.f;
  double dA = 0.f;
  double dB = 0.f;
  double dcsrSortedVal = 0.f;
  float fA = 0.f;
  float fB = 0.f;
  float fds = 0.f;
  float fdl = 0.f;
  float fd = 0.f;
  float fdu = 0.f;
  float fdw = 0.f;
  float fx = 0.f;
  float csrSortedVal = 0.f;
  size_t bufferSize = 0;
  void *pBuffer = nullptr;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuDoubleComplex -> rocsparse_double_complex under a new option --sparse
  // CHECK: rocblas_double_complex dcomplex, dcomplexA, dcomplexB, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;
  hipDoubleComplex dcomplex, dcomplexA, dcomplexB, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuComplex -> rocsparse_float_complex under a new option --sparse
  // CHECK: rocblas_float_complex complex, complexA, complexB, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;
  hipComplex complex, complexA, complexB, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;

  // CHECK: rocsparse_operation opA, opB, opX;
  hipsparseOperation_t opA, opB, opX;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;

  // CHECK: rocsparse_solve_policy solvePolicy_t;
  // CHECK-NEXT: rocsparse_solve_policy SOLVE_POLICY_NO_LEVEL = rocsparse_solve_policy_auto;
  // CHECK-NEXT: rocsparse_solve_policy SOLVE_POLICY_USE_LEVEL = rocsparse_solve_policy_auto;
  hipsparseSolvePolicy_t solvePolicy_t;
  hipsparseSolvePolicy_t SOLVE_POLICY_NO_LEVEL = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
  hipsparseSolvePolicy_t SOLVE_POLICY_USE_LEVEL = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

#if CUDA_VERSION >= 9020 && CUDA_VERSION < 12000
  // CHECK: rocsparse_mat_info csrsm2_info;
  csrsm2Info_t csrsm2_info;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZcsrsm2_solve(cusparseHandle_t handle, int algo, cusparseOperation_t transA, cusparseOperation_t transB, int m, int nrhs, int nnz, const cuDoubleComplex* alpha, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, cuDoubleComplex* B, int ldb, csrsm2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsrsm_solve(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int nrhs, rocsparse_int nnz, const rocsparse_double_complex* alpha, const rocsparse_mat_descr descr, const rocsparse_double_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_double_complex* B, rocsparse_int ldb, rocsparse_mat_info info, rocsparse_solve_policy policy, void* temp_buffer);
  // CHECK: status_t = rocsparse_zcsrsm_solve(handle_t, algo, opA, opB, m, nrhs, innz, &dcomplexA, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, &dcomplexB, ldb, csrsm2_info, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseZcsrsm2_solve(handle_t, algo, opA, opB, m, nrhs, innz, &dcomplexA, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, &dcomplexB, ldb, csrsm2_info, solvePolicy_t, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCcsrsm2_solve(cusparseHandle_t handle, int algo, cusparseOperation_t transA, cusparseOperation_t transB, int m, int nrhs, int nnz, const cuComplex* alpha, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, cuComplex* B, int ldb, csrsm2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsrsm_solve(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int nrhs, rocsparse_int nnz, const rocsparse_float_complex* alpha, const rocsparse_mat_descr descr, const rocsparse_float_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_float_complex* B, rocsparse_int ldb, rocsparse_mat_info info, rocsparse_solve_policy policy, void* temp_buffer);
  // CHECK: status_t = rocsparse_ccsrsm_solve(handle_t, algo, opA, opB, m, nrhs, innz, &complexA, matDescr_A, &complex, &csrRowPtrA, &csrColIndA, &complexB, ldb, csrsm2_info, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseCcsrsm2_solve(handle_t, algo, opA, opB, m, nrhs, innz, &complexA, matDescr_A, &complex, &csrRowPtrA, &csrColIndA, &complexB, ldb, csrsm2_info, solvePolicy_t, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDcsrsm2_solve(cusparseHandle_t handle, int algo, cusparseOperation_t transA, cusparseOperation_t transB, int m, int nrhs, int nnz, const double* alpha, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, double* B, int ldb, csrsm2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsrsm_solve(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int nrhs, rocsparse_int nnz, const double* alpha, const rocsparse_mat_descr descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, double* B, rocsparse_int ldb, rocsparse_mat_info info, rocsparse_solve_policy policy, void* temp_buffer);
  // CHECK: status_t = rocsparse_dcsrsm_solve(handle_t, algo, opA, opB, m, nrhs, innz, &dA, matDescr_A, &dcsrSortedVal, &csrRowPtrA, &csrColIndA, &dB, ldb, csrsm2_info, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseDcsrsm2_solve(handle_t, algo, opA, opB, m, nrhs, innz, &dA, matDescr_A, &dcsrSortedVal, &csrRowPtrA, &csrColIndA, &dB, ldb, csrsm2_info, solvePolicy_t, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScsrsm2_solve(cusparseHandle_t handle, int algo, cusparseOperation_t transA, cusparseOperation_t transB, int m, int nrhs, int nnz, const float* alpha, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float* B, int ldb, csrsm2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsrsm_solve(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int nrhs, rocsparse_int nnz, const float* alpha, const rocsparse_mat_descr descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, float* B, rocsparse_int ldb, rocsparse_mat_info info, rocsparse_solve_policy policy, void* temp_buffer);
  // CHECK: status_t = rocsparse_scsrsm_solve(handle_t, algo, opA, opB, m, nrhs, innz, &fA, matDescr_A, &csrSortedVal, &csrRowPtrA, &csrColIndA, &fB, ldb, csrsm2_info, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseScsrsm2_solve(handle_t, algo, opA, opB, m, nrhs, innz, &fA, matDescr_A, &csrSortedVal, &csrRowPtrA, &csrColIndA, &fB, ldb, csrsm2_info, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSM) cusparseStatus_t CUSPARSEAPI cusparseZcsrsm2_analysis(cusparseHandle_t handle, int algo, cusparseOperation_t transA, cusparseOperation_t transB, int m, int nrhs, int nnz, const cuDoubleComplex* alpha, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cuDoubleComplex* B, int ldb, csrsm2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsrsm_analysis(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int nrhs, rocsparse_int nnz, const rocsparse_double_complex* alpha, const rocsparse_mat_descr descr, const rocsparse_double_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const rocsparse_double_complex* B, rocsparse_int ldb, rocsparse_mat_info info, rocsparse_analysis_policy analysis, rocsparse_solve_policy solve, void* temp_buffer);
  // CHECK: status_t = rocsparse_zcsrsm_analysis(handle_t, algo, opA, opB, m, nrhs, innz, &dcomplexA, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, &dcomplexB, ldb, csrsm2_info, rocsparse_analysis_policy_force, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseZcsrsm2_analysis(handle_t, algo, opA, opB, m, nrhs, innz, &dcomplexA, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, &dcomplexB, ldb, csrsm2_info, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSM) cusparseStatus_t CUSPARSEAPI cusparseCcsrsm2_analysis(cusparseHandle_t handle, int algo, cusparseOperation_t transA, cusparseOperation_t transB, int m, int nrhs, int nnz, const cuComplex* alpha, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cuComplex* B, int ldb, csrsm2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsrsm_analysis(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int nrhs, rocsparse_int nnz, const rocsparse_float_complex * alpha, const rocsparse_mat_descr descr, const rocsparse_float_complex * csr_val, const rocsparse_int * csr_row_ptr, const rocsparse_int * csr_col_ind, const rocsparse_float_complex * B, rocsparse_int ldb, rocsparse_mat_info info, rocsparse_analysis_policy analysis, rocsparse_solve_policy solve, void* temp_buffer);
  // CHECK: status_t = rocsparse_ccsrsm_analysis(handle_t, algo, opA, opB, m, nrhs, innz, &complexA, matDescr_A, &complex, &csrRowPtrA, &csrColIndA, &complexB, ldb, csrsm2_info, rocsparse_analysis_policy_force, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseCcsrsm2_analysis(handle_t, algo, opA, opB, m, nrhs, innz, &complexA, matDescr_A, &complex, &csrRowPtrA, &csrColIndA, &complexB, ldb, csrsm2_info, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSM) cusparseStatus_t CUSPARSEAPI cusparseDcsrsm2_analysis(cusparseHandle_t handle, int algo, cusparseOperation_t transA, cusparseOperation_t transB, int m, int nrhs, int nnz, const double* alpha, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* B, int ldb, csrsm2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsrsm_analysis(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int nrhs, rocsparse_int nnz, const double* alpha, const rocsparse_mat_descr descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const double* B, rocsparse_int ldb, rocsparse_mat_info info, rocsparse_analysis_policy analysis, rocsparse_solve_policy solve, void* temp_buffer);
  // CHECK: status_t = rocsparse_dcsrsm_analysis(handle_t, algo, opA, opB, m, nrhs, innz, &dA, matDescr_A, &dcsrSortedVal, &csrRowPtrA, &csrColIndA, &dB, ldb, csrsm2_info, rocsparse_analysis_policy_force, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseDcsrsm2_analysis(handle_t, algo, opA, opB, m, nrhs, innz, &dA, matDescr_A, &dcsrSortedVal, &csrRowPtrA, &csrColIndA, &dB, ldb, csrsm2_info, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSM) cusparseStatus_t CUSPARSEAPI cusparseScsrsm2_analysis(cusparseHandle_t handle, int algo, cusparseOperation_t transA, cusparseOperation_t transB, int m, int nrhs, int nnz, const float* alpha, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* B, int ldb, csrsm2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsrsm_analysis(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int nrhs, rocsparse_int nnz, const float* alpha, const rocsparse_mat_descr descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const float* B, rocsparse_int ldb, rocsparse_mat_info info, rocsparse_analysis_policy analysis, rocsparse_solve_policy solve, void* temp_buffer);
  // CHECK: status_t = rocsparse_scsrsm_analysis(handle_t, algo, opA, opB, m, nrhs, innz, &fA, matDescr_A, &csrSortedVal, &csrRowPtrA, &csrColIndA, &fB, ldb, csrsm2_info, rocsparse_analysis_policy_force, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseScsrsm2_analysis(handle_t, algo, opA, opB, m, nrhs, innz, &fA, matDescr_A, &csrSortedVal, &csrRowPtrA, &csrColIndA, &fB, ldb, csrsm2_info, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSM) cusparseStatus_t CUSPARSEAPI cusparseZcsrsm2_bufferSizeExt(cusparseHandle_t handle, int algo, cusparseOperation_t transA, cusparseOperation_t transB, int m, int nrhs, int nnz, const cuDoubleComplex* alpha, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cuDoubleComplex* B, int ldb, csrsm2Info_t info, cusparseSolvePolicy_t policy, size_t* pBufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsrsm_buffer_size(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int nrhs, rocsparse_int nnz, const rocsparse_double_complex* alpha, const rocsparse_mat_descr descr, const rocsparse_double_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const rocsparse_double_complex* B, rocsparse_int ldb, rocsparse_mat_info info, rocsparse_solve_policy policy, size_t* buffer_size);
  // CHECK: status_t = rocsparse_zcsrsm_buffer_size(handle_t, algo, opA, opB, m, nrhs, innz, &dcomplexA, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, &dcomplexB, ldb, csrsm2_info, rocsparse_solve_policy_auto, &bufferSize);
  status_t = cusparseZcsrsm2_bufferSizeExt(handle_t, algo, opA, opB, m, nrhs, innz, &dcomplexA, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, &dcomplexB, ldb, csrsm2_info, solvePolicy_t, &bufferSize);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSM) cusparseStatus_t CUSPARSEAPI cusparseCcsrsm2_bufferSizeExt(cusparseHandle_t handle, int algo, cusparseOperation_t transA, cusparseOperation_t transB, int m, int nrhs, int nnz, const cuComplex* alpha, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cuComplex* B, int ldb, csrsm2Info_t info, cusparseSolvePolicy_t policy, size_t* pBufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsrsm_buffer_size(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int nrhs, rocsparse_int nnz, const rocsparse_float_complex* alpha, const rocsparse_mat_descr descr, const rocsparse_float_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const rocsparse_float_complex* B, rocsparse_int ldb, rocsparse_mat_info info, rocsparse_solve_policy policy, size_t* buffer_size);
  // CHECK: status_t = rocsparse_ccsrsm_buffer_size(handle_t, algo, opA, opB, m, nrhs, innz, &complexA, matDescr_A, &complex, &csrRowPtrA, &csrColIndA, &complexB, ldb, csrsm2_info, rocsparse_solve_policy_auto, &bufferSize);
  status_t = cusparseCcsrsm2_bufferSizeExt(handle_t, algo, opA, opB, m, nrhs, innz, &complexA, matDescr_A, &complex, &csrRowPtrA, &csrColIndA, &complexB, ldb, csrsm2_info, solvePolicy_t, &bufferSize);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSM) cusparseStatus_t CUSPARSEAPI cusparseDcsrsm2_bufferSizeExt(cusparseHandle_t handle, int algo, cusparseOperation_t transA, cusparseOperation_t transB, int m, int nrhs, int nnz, const double* alpha, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* B, int ldb, csrsm2Info_t info, cusparseSolvePolicy_t policy, size_t* pBufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsrsm_buffer_size(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int nrhs, rocsparse_int nnz, const double* alpha, const rocsparse_mat_descr descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const double* B, rocsparse_int ldb, rocsparse_mat_info info, rocsparse_solve_policy policy, size_t* buffer_size);
  // CHECK: status_t = rocsparse_dcsrsm_buffer_size(handle_t, algo, opA, opB, m, nrhs, innz, &dA, matDescr_A, &dcsrSortedVal, &csrRowPtrA, &csrColIndA, &dB, ldb, csrsm2_info, rocsparse_solve_policy_auto, &bufferSize);
  status_t = cusparseDcsrsm2_bufferSizeExt(handle_t, algo, opA, opB, m, nrhs, innz, &dA, matDescr_A, &dcsrSortedVal, &csrRowPtrA, &csrColIndA, &dB, ldb, csrsm2_info, solvePolicy_t, &bufferSize);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSM) cusparseStatus_t CUSPARSEAPI cusparseScsrsm2_bufferSizeExt(cusparseHandle_t handle, int algo, cusparseOperation_t transA, cusparseOperation_t transB, int m, int nrhs, int nnz, const float* alpha, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* B, int ldb, csrsm2Info_t info, cusparseSolvePolicy_t policy, size_t* pBufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsrsm_buffer_size(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int nrhs, rocsparse_int nnz, const float* alpha, const rocsparse_mat_descr descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const float* B, rocsparse_int ldb, rocsparse_mat_info info, rocsparse_solve_policy policy, size_t* buffer_size);
  // CHECK: status_t = rocsparse_scsrsm_buffer_size(handle_t, algo, opA, opB, m, nrhs, innz, &fA, matDescr_A, &csrSortedVal, &csrRowPtrA, &csrColIndA, &fB, ldb, csrsm2_info, rocsparse_solve_policy_auto, &bufferSize);
  status_t = cusparseScsrsm2_bufferSizeExt(handle_t, algo, opA, opB, m, nrhs, innz, &fA, matDescr_A, &csrSortedVal, &csrRowPtrA, &csrColIndA, &fB, ldb, csrsm2_info, solvePolicy_t, &bufferSize);

  // TODO: rocsparse_csrsm_zero_pivot needs explicit synchronization because cusparseXcsrsm2_zeroPivot is blocking
  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSM) cusparseStatus_t CUSPARSEAPI cusparseXcsrsm2_zeroPivot(cusparseHandle_t handle, csrsm2Info_t info, int* position);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_csrsm_zero_pivot(rocsparse_handle handle, rocsparse_mat_info info, rocsparse_int* position);
  // CHECK: status_t = rocsparse_csrsm_zero_pivot(handle_t, csrsm2_info, &iposition);
  status_t = cusparseXcsrsm2_zeroPivot(handle_t, csrsm2_info, &iposition);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSM) cusparseStatus_t CUSPARSEAPI cusparseCreateCsrsm2Info(csrsm2Info_t* info);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_mat_info(rocsparse_mat_info* info);
  // CHECK: status_t = rocsparse_create_mat_info(&csrsm2_info);
  status_t = cusparseCreateCsrsm2Info(&csrsm2_info);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSM) cusparseStatus_t CUSPARSEAPI cusparseDestroyCsrsm2Info(csrsm2Info_t info);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_destroy_mat_info(rocsparse_mat_info info);
  // CHECK: status_t = rocsparse_destroy_mat_info(csrsm2_info);
  status_t = cusparseDestroyCsrsm2Info(csrsm2_info);
#endif

  return 0;
}
