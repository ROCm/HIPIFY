// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18.1. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: rocsparse_status status_t;
  hipsparseStatus_t status_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;

  // CHECK: rocsparse_action action_t;
  hipsparseAction_t action_t;

  // CHECK: rocsparse_index_base indexBase_t;
  hipsparseIndexBase_t indexBase_t;

  int m = 0;
  int n = 0;
  int innz = 0;
  int csrRowPtrA = 0;
  int csrRowPtrB = 0;
  int csrRowPtrC = 0;
  int cscRowIndA = 0;
  int csrColIndA = 0;
  int csrColIndB = 0;
  int csrColIndC = 0;
  int cscColPtrA = 0;
  size_t bufferSize = 0;
  void *pcsrVal = nullptr;
  void *pcscVal = nullptr;
  void *alpha = nullptr;
  void *beta = nullptr;
  void *tempBuffer = nullptr;
  void* result = nullptr;

  // CHECK: rocsparse_operation opA, opB, opX;
  hipsparseOperation_t opA, opB, opX;

#if CUDA_VERSION >= 8000
  // TODO: [#899] There should be rocsparse_datatype instead of hipDataType
  hipDataType dataType_t;
  hipDataType dataType;
#endif

#if CUDA_VERSION >= 10010 && CUSPARSE_VERSION >= 10200
  // TODO: cusparseCsr2CscAlg_t has no analogue in rocSPARSE. The deletion of declaration and usage is needed to be implemented
  hipsparseCsr2CscAlg_t Csr2CscAlg_t;

#if (CUDA_VERSION < 11000 && !defined(_WIN32)) || CUDA_VERSION >= 11000
  // CHECK: rocsparse_spmat_descr spMatDescr_t, spmatA, spmatB, spmatC;
  hipsparseSpMatDescr_t spMatDescr_t, spmatA, spmatB, spmatC;

  // CHECK: rocsparse_dnmat_descr dnMatDescr_t, dnmatA, dnmatB, dnmatC;
  hipsparseDnMatDescr_t dnMatDescr_t, dnmatA, dnmatB, dnmatC;

  // CHECK: rocsparse_spmm_alg spMMAlg_t;
  hipsparseSpMMAlg_t spMMAlg_t;

  // CHECK: _rocsparse_spvec_descr *spVecDescr = nullptr;
  // CHECK-NEXT: rocsparse_spvec_descr spVecDescr_t;
  cusparseSpVecDescr *spVecDescr = nullptr;
  hipsparseSpVecDescr_t spVecDescr_t;

  // CHECK: _rocsparse_dnvec_descr *dnVecDescr = nullptr;
  // CHECK-NEXT: rocsparse_dnvec_descr dnVecDescr_t, vecX, vecY;
  cusparseDnVecDescr *dnVecDescr = nullptr;
  hipsparseDnVecDescr_t dnVecDescr_t, vecX, vecY;
#endif
#endif

  return 0;
}
