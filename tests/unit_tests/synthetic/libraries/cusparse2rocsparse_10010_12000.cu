// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18.1. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: rocsparse_status status_t;
  hipsparseStatus_t status_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;

  // CHECK: rocsparse_action action_t;
  hipsparseAction_t action_t;

  // CHECK: rocsparse_index_base indexBase_t;
  hipsparseIndexBase_t indexBase_t;

  int m = 0;
  int n = 0;
  int innz = 0;
  int csrRowPtrA = 0;
  int csrRowPtrB = 0;
  int csrRowPtrC = 0;
  int cscRowIndA = 0;
  int csrColIndA = 0;
  int csrColIndB = 0;
  int csrColIndC = 0;
  int cscColPtrA = 0;
  size_t bufferSize = 0;
  void *pcsrVal = nullptr;
  void *pcscVal = nullptr;
  void *alpha = nullptr;
  void *beta = nullptr;

  // CHECK: rocsparse_operation opA, opB, opX;
  hipsparseOperation_t opA, opB, opX;

#if CUDA_VERSION >= 8000
  // TODO: [#899] There should be rocsparse_datatype instead of hipDataType
  hipDataType dataType_t;
  hipDataType dataType;
#endif

#if CUDA_VERSION >= 10010
  // TODO: cusparseCsr2CscAlg_t has no analogue in rocSPARSE. The deletion of declaration and usage is needed to be implemented
  hipsparseCsr2CscAlg_t Csr2CscAlg_t;

#if (CUDA_VERSION < 11000 && !defined(_WIN32)) || CUDA_VERSION >= 11000
  // CHECK: rocsparse_spmat_descr spMatDescr_t, spmatA, spmatB, spmatC;
  hipsparseSpMatDescr_t spMatDescr_t, spmatA, spmatB, spmatC;

  // CHECK: rocsparse_dnmat_descr dnMatDescr_t, dnmatA, dnmatB, dnmatC;
  hipsparseDnMatDescr_t dnMatDescr_t, dnmatA, dnmatB, dnmatC;

  // CHECK: rocsparse_spmm_alg spMMAlg_t;
  hipsparseSpMMAlg_t spMMAlg_t;

#if CUDA_VERSION < 12000
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMM_bufferSize(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, const cusparseSpMatDescr_t matA, const cusparseDnMatDescr_t matB, const void* beta, cusparseDnMatDescr_t matC, cudaDataType computeType, cusparseSpMMAlg_t alg, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, const void* alpha, const rocsparse_spmat_descr mat_A, const rocsparse_dnmat_descr mat_B, const void* beta, const rocsparse_dnmat_descr mat_C, rocsparse_datatype compute_type, rocsparse_spmm_alg alg, rocsparse_spmm_stage stage, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spmm(handle_t, opA, opB, alpha, spmatA, dnmatB, beta, dnmatC, dataType, spMMAlg_t, rocsparse_spmm_stage_compute, &bufferSize, nullptr);
  status_t = cusparseSpMM_bufferSize(handle_t, opA, opB, alpha, spmatA, dnmatB, beta, dnmatC, dataType, spMMAlg_t, &bufferSize);
#endif
#endif
#endif

  return 0;
}
