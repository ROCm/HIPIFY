// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18.1. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: rocsparse_status status_t;
  hipsparseStatus_t status_t;

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;

  // CHECK: rocsparse_operation opA, opB, opX;
  hipsparseOperation_t opA, opB, opX;

  // CHECK: rocsparse_solve_policy solvePolicy_t;
  hipsparseSolvePolicy_t solvePolicy_t;

  int m = 0;
  int n = 0;
  int k = 0;
  int innz = 0;
  int nnza = 0;
  int nnzb = 0;
  int nnzc = 0;
  int nnzd = 0;
  int csrRowPtrA = 0;
  int csrRowPtrB = 0;
  int csrRowPtrC = 0;
  int csrRowPtrD = 0;
  int csrColIndA = 0;
  int csrColIndB = 0;
  int csrColIndC = 0;
  int csrColIndD = 0;
  int bufferSizeInBytes = 0;
  size_t bufferSize = 0;
  double dA = 0.f;
  double dB = 0.f;
  double dAlpha = 0.f;
  double dF = 0.f;
  double dX = 0.f;
  double dcsrSortedValA = 0.f;
  double dcsrSortedValB = 0.f;
  double dcsrSortedValC = 0.f;
  double dcsrSortedValD = 0.f;
  float fAlpha = 0.f;
  float fA = 0.f;
  float fB = 0.f;
  float fF = 0.f;
  float fX = 0.f;
  float csrSortedValA = 0.f;
  float csrSortedValB = 0.f;
  float csrSortedValC = 0.f;
  float csrSortedValD = 0.f;
  void *pBuffer = nullptr;
  void *tempBuffer = nullptr;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuDoubleComplex -> rocsparse_double_complex under a new option --sparse
  // CHECK: rocblas_double_complex dcomplex, dcomplexA, dcomplexAlpha, dcomplexB, dcomplexBeta, dcomplexC, dcomplexF, dcomplexX, dcomplexY, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dComplexcsrSortedValD, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;
  hipDoubleComplex dcomplex, dcomplexA, dcomplexAlpha, dcomplexB, dcomplexBeta, dcomplexC, dcomplexF, dcomplexX, dcomplexY, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dComplexcsrSortedValD, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuComplex -> rocsparse_float_complex under a new option --sparse
  // CHECK: rocblas_float_complex complex, complexA, complexAlpha, complexB, complexBeta, complexC, complexF, complexX, complexY, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complexcsrSortedValD, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;
  hipComplex complex, complexA, complexAlpha, complexB, complexBeta, complexC, complexF, complexX, complexY, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complexcsrSortedValD, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;

#if (CUDA_VERSION >= 10010 && CUDA_VERSION < 11000 && !defined(_WIN32)) || CUDA_VERSION >= 11000
  // CHECK: rocsparse_spmat_descr spMatDescr_t, spmatA, spmatB, spmatC;
  hipsparseSpMatDescr_t spMatDescr_t, spmatA, spmatB, spmatC;

  // CHECK: rocsparse_dnmat_descr dnMatDescr_t, dnmatA, dnmatB, dnmatC;
  hipsparseDnMatDescr_t dnMatDescr_t, dnmatA, dnmatB, dnmatC;
#endif

#if CUDA_VERSION >= 11010 && CUSPARSE_VERSION >= 11300
  // CHECK: rocsparse_sparse_to_dense_alg sparseToDenseAlg_t;
  hipsparseSparseToDenseAlg_t sparseToDenseAlg_t;

  // CHECK: rocsparse_dense_to_sparse_alg denseToSparseAlg_t;
  hipsparseDenseToSparseAlg_t denseToSparseAlg_t;

#if CUDA_VERSION < 12000
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSparseToDense(cusparseHandle_t handle, cusparseSpMatDescr_t matA, cusparseDnMatDescr_t matB, cusparseSparseToDenseAlg_t alg, void* buffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sparse_to_dense(rocsparse_handle handle, const rocsparse_spmat_descr mat_A, rocsparse_dnmat_descr mat_B, rocsparse_sparse_to_dense_alg alg, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_sparse_to_dense(handle_t, spmatA, dnmatB, sparseToDenseAlg_t, nullptr, tempBuffer);
  status_t = cusparseSparseToDense(handle_t, spmatA, dnmatB, sparseToDenseAlg_t, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSparseToDense_bufferSize(cusparseHandle_t handle, cusparseSpMatDescr_t matA, cusparseDnMatDescr_t matB, cusparseSparseToDenseAlg_t alg, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sparse_to_dense(rocsparse_handle handle, const rocsparse_spmat_descr mat_A, rocsparse_dnmat_descr mat_B, rocsparse_sparse_to_dense_alg alg, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_sparse_to_dense(handle_t, spmatA, dnmatB, sparseToDenseAlg_t, &bufferSize, nullptr);
  status_t = cusparseSparseToDense_bufferSize(handle_t, spmatA, dnmatB, sparseToDenseAlg_t, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDenseToSparse_bufferSize(cusparseHandle_t handle, cusparseDnMatDescr_t matA, cusparseSpMatDescr_t matB, cusparseDenseToSparseAlg_t alg, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dense_to_sparse(rocsparse_handle handle, const rocsparse_dnmat_descr mat_A, rocsparse_spmat_descr mat_B, rocsparse_dense_to_sparse_alg alg, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_dense_to_sparse(handle_t, dnmatA, spmatB, denseToSparseAlg_t, &bufferSize, nullptr);
  status_t = cusparseDenseToSparse_bufferSize(handle_t, dnmatA, spmatB, denseToSparseAlg_t, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDenseToSparse_analysis(cusparseHandle_t handle, cusparseDnMatDescr_t matA, cusparseSpMatDescr_t matB, cusparseDenseToSparseAlg_t alg, void* buffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dense_to_sparse(rocsparse_handle handle, const rocsparse_dnmat_descr mat_A, rocsparse_spmat_descr mat_B, rocsparse_dense_to_sparse_alg alg, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_dense_to_sparse(handle_t, dnmatA, spmatB, denseToSparseAlg_t, nullptr, tempBuffer);
  status_t = cusparseDenseToSparse_analysis(handle_t, dnmatA, spmatB, denseToSparseAlg_t, tempBuffer);
#endif
#endif

  return 0;
}
