// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental --roc %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocsolver.h"
#include "hipsolver.h"

int main() {
  printf("20. cuSOLVER API to rocSOLVER API synthetic test\n");

  // CHECK: rocblas_handle handle;
  hipsolverHandle_t handle;

  // CHECK: rocblas_status status;
  // CHECK-NEXT: rocblas_status STATUS_SUCCESS = rocblas_status_success;
  // CHECK-NEXT: rocblas_status STATUS_NOT_INITIALIZED = rocblas_status_invalid_handle;
  // CHECK-NEXT: rocblas_status STATUS_ALLOC_FAILED = rocblas_status_memory_error;
  // CHECK-NEXT: rocblas_status STATUS_INVALID_VALUE = rocblas_status_invalid_value;
  // CHECK-NEXT: rocblas_status STATUS_ARCH_MISMATCH = rocblas_status_arch_mismatch;
  // CHECK-NEXT: rocblas_status STATUS_MAPPING_ERROR = rocblas_status_not_implemented;
  // CHECK-NEXT: rocblas_status STATUS_EXECUTION_FAILED = rocblas_status_not_implemented;
  // CHECK-NEXT: rocblas_status STATUS_INTERNAL_ERROR = rocblas_status_internal_error;
  // CHECK-NEXT: rocblas_status STATUS_NOT_SUPPORTED = rocblas_status_not_implemented;
  // CHECK-NEXT: rocblas_status STATUS_ZERO_PIVOT = rocblas_status_not_implemented;
  hipsolverStatus_t status;
  hipsolverStatus_t STATUS_SUCCESS = HIPSOLVER_STATUS_SUCCESS;
  hipsolverStatus_t STATUS_NOT_INITIALIZED = HIPSOLVER_STATUS_NOT_INITIALIZED;
  hipsolverStatus_t STATUS_ALLOC_FAILED = HIPSOLVER_STATUS_ALLOC_FAILED;
  hipsolverStatus_t STATUS_INVALID_VALUE = HIPSOLVER_STATUS_INVALID_VALUE;
  hipsolverStatus_t STATUS_ARCH_MISMATCH = HIPSOLVER_STATUS_ARCH_MISMATCH;
  hipsolverStatus_t STATUS_MAPPING_ERROR = HIPSOLVER_STATUS_MAPPING_ERROR;
  hipsolverStatus_t STATUS_EXECUTION_FAILED = HIPSOLVER_STATUS_EXECUTION_FAILED;
  hipsolverStatus_t STATUS_INTERNAL_ERROR = HIPSOLVER_STATUS_INTERNAL_ERROR;
  hipsolverStatus_t STATUS_NOT_SUPPORTED = HIPSOLVER_STATUS_NOT_SUPPORTED;
  hipsolverStatus_t STATUS_ZERO_PIVOT = HIPSOLVER_STATUS_ZERO_PIVOT;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCreate(cusolverDnHandle_t *handle);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_create_handle(rocblas_handle* handle);
  // CHECK: status = rocblas_create_handle(&handle);
  status = hipsolverDnCreate(&handle);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDestroy(cusolverDnHandle_t handle);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_destroy_handle(rocblas_handle handle);
  // CHECK: status = rocblas_destroy_handle(handle);
  status = hipsolverDnDestroy(handle);

#if CUDA_VERSION >= 8000
  // CHECK: rocblas_eform eigType;
  // CHECK-NEXT: rocblas_eform EIG_TYPE_1 = rocblas_eform_ax;
  // CHECK-NEXT: rocblas_eform EIG_TYPE_2 = rocblas_eform_abx;
  // CHECK-NEXT: rocblas_eform EIG_TYPE_3 = rocblas_eform_bax;
  hipsolverEigType_t eigType;
  hipsolverEigType_t EIG_TYPE_1 = HIPSOLVER_EIG_TYPE_1;
  hipsolverEigType_t EIG_TYPE_2 = HIPSOLVER_EIG_TYPE_2;
  hipsolverEigType_t EIG_TYPE_3 = HIPSOLVER_EIG_TYPE_3;

  // CHECK: rocblas_evect eigMode;
  // CHECK-NEXT: rocblas_evect SOLVER_EIG_MODE_NOVECTOR = rocblas_evect_none;
  // CHECK-NEXT: rocblas_evect SOLVER_EIG_MODE_VECTOR = rocblas_evect_original;
  hipsolverEigMode_t eigMode;
  hipsolverEigMode_t SOLVER_EIG_MODE_NOVECTOR = HIPSOLVER_EIG_MODE_NOVECTOR;
  hipsolverEigMode_t SOLVER_EIG_MODE_VECTOR = HIPSOLVER_EIG_MODE_VECTOR;
#endif

#if CUDA_VERSION >= 10010
  // CHECK: rocblas_erange eigRange;
  // CHECK-NEXT: rocblas_erange EIG_RANGE_ALL = rocblas_erange_all;
  // CHECK-NEXT: rocblas_erange EIG_RANGE_I = rocblas_erange_index;
  // CHECK-NEXT: rocblas_erange EIG_RANGE_V = rocblas_erange_value;
  hipsolverEigRange_t eigRange;
  hipsolverEigRange_t EIG_RANGE_ALL = HIPSOLVER_EIG_RANGE_ALL;
  hipsolverEigRange_t EIG_RANGE_I = HIPSOLVER_EIG_RANGE_I;
  hipsolverEigRange_t EIG_RANGE_V = HIPSOLVER_EIG_RANGE_V;
#endif

  return 0;
}
