// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18.1. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: rocsparse_status status_t;
  hipsparseStatus_t status_t;

  int batchCount = 0;
  int m = 0;
  int algo = 0;
  double dds = 0.f;
  double ddl = 0.f;
  double dd = 0.f;
  double ddu = 0.f;
  double ddw = 0.f;
  double dx = 0.f;
  float fds = 0.f;
  float fdl = 0.f;
  float fd = 0.f;
  float fdu = 0.f;
  float fdw = 0.f;
  float fx = 0.f;
  size_t bufferSize = 0;
  void *pBuffer = nullptr;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuDoubleComplex -> rocsparse_double_complex under a new option --sparse
  // CHECK: rocblas_double_complex dcomplex, dcomplexA, dcomplexB, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;
  hipDoubleComplex dcomplex, dcomplexA, dcomplexB, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuComplex -> rocsparse_float_complex under a new option --sparse
  // CHECK: rocblas_float_complex complex, complexA, complexB, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;
  hipComplex complex, complexA, complexB, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;

#if CUDA_VERSION >= 9020
  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_zgpsv_interleaved_batch function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseZgpsvInterleavedBatch calls rocsparse_zgpsv_interleaved_batch in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgpsvInterleavedBatch(cusparseHandle_t handle, int algo, int m, cuDoubleComplex* ds, cuDoubleComplex* dl, cuDoubleComplex* d, cuDoubleComplex* du, cuDoubleComplex* dw, cuDoubleComplex* x, int batchCount, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zgpsv_interleaved_batch(rocsparse_handle handle, rocsparse_gpsv_interleaved_alg alg, rocsparse_int m, rocsparse_double_complex* ds, rocsparse_double_complex* dl, rocsparse_double_complex* d, rocsparse_double_complex* du, rocsparse_double_complex* dw, rocsparse_double_complex* x, rocsparse_int batch_count, rocsparse_int batch_stride, void* temp_buffer);
  // CHECK: status_t = rocsparse_zgpsv_interleaved_batch(handle_t, algo, m, &dcomplexds, &dcomplexdl, &dcomplexd, &dcomplexdu, &dcomplexdw, &dcomplexx, batchCount, batchCount, pBuffer);
  status_t = hipsparseZgpsvInterleavedBatch(handle_t, algo, m, &dcomplexds, &dcomplexdl, &dcomplexd, &dcomplexdu, &dcomplexdw, &dcomplexx, batchCount, pBuffer);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_cgpsv_interleaved_batch function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseCgpsvInterleavedBatch calls rocsparse_cgpsv_interleaved_batch in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgpsvInterleavedBatch(cusparseHandle_t handle, int algo, int m, cuComplex* ds, cuComplex* dl, cuComplex* d, cuComplex* du, cuComplex* dw, cuComplex* x, int batchCount, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cgpsv_interleaved_batch(rocsparse_handle handle, rocsparse_gpsv_interleaved_alg alg, rocsparse_int m, rocsparse_float_complex* ds, rocsparse_float_complex* dl, rocsparse_float_complex* d, rocsparse_float_complex* du, rocsparse_float_complex* dw, rocsparse_float_complex* x, rocsparse_int batch_count, rocsparse_int batch_stride, void* temp_buffer);
  // CHECK: status_t = rocsparse_cgpsv_interleaved_batch(handle_t, algo, m, &complexds, &complexdl, &complexd, &complexdu, &complexdw, &complexx, batchCount, batchCount, pBuffer);
  status_t = hipsparseCgpsvInterleavedBatch(handle_t, algo, m, &complexds, &complexdl, &complexd, &complexdu, &complexdw, &complexx, batchCount, pBuffer);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_dgpsv_interleaved_batch function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseDgpsvInterleavedBatch calls rocsparse_dgpsv_interleaved_batch in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgpsvInterleavedBatch(cusparseHandle_t handle, int algo, int m, double* ds, double* dl, double* d, double* du, double* dw, double* x, int batchCount, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dgpsv_interleaved_batch(rocsparse_handle handle, rocsparse_gpsv_interleaved_alg alg, rocsparse_int m, double* ds, double* dl, double* d, double* du, double* dw, double* x, rocsparse_int batch_count, rocsparse_int batch_stride, void* temp_buffer);
  // CHECK: status_t = rocsparse_dgpsv_interleaved_batch(handle_t, algo, m, &dds, &ddl, &dd, &ddu, &ddw, &dx, batchCount, batchCount, pBuffer);
  status_t = hipsparseDgpsvInterleavedBatch(handle_t, algo, m, &dds, &ddl, &dd, &ddu, &ddw, &dx, batchCount, pBuffer);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_sgpsv_interleaved_batch function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseSgpsvInterleavedBatch calls rocsparse_sgpsv_interleaved_batch in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgpsvInterleavedBatch(cusparseHandle_t handle, int algo, int m, float* ds, float* dl, float* d, float* du, float* dw, float* x, int batchCount, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sgpsv_interleaved_batch(rocsparse_handle handle, rocsparse_gpsv_interleaved_alg alg, rocsparse_int m, float* ds, float* dl, float* d, float* du, float* dw, float* x, rocsparse_int batch_count, rocsparse_int batch_stride, void* temp_buffer);
  // CHECK: status_t = rocsparse_sgpsv_interleaved_batch(handle_t, algo, m, &fds, &fdl, &fd, &fdu, &fdw, &fx, batchCount, batchCount, pBuffer);
  status_t = hipsparseSgpsvInterleavedBatch(handle_t, algo, m, &fds, &fdl, &fd, &fdu, &fdw, &fx, batchCount, pBuffer);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_zgpsv_interleaved_batch_buffer_size function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseSgpsvInterleavedBatch calls rocsparse_zgpsv_interleaved_batch_buffer_size in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgpsvInterleavedBatch_bufferSizeExt(cusparseHandle_t handle, int algo, int m, const cuDoubleComplex* ds, const cuDoubleComplex* dl, const cuDoubleComplex* d, const cuDoubleComplex* du, const cuDoubleComplex* dw, const cuDoubleComplex* x, int batchCount, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zgpsv_interleaved_batch_buffer_size(rocsparse_handle handle, rocsparse_gpsv_interleaved_alg alg, rocsparse_int m, const rocsparse_double_complex* ds, const rocsparse_double_complex* dl, const rocsparse_double_complex* d, const rocsparse_double_complex* du, const rocsparse_double_complex* dw, const rocsparse_double_complex* x, rocsparse_int batch_count, rocsparse_int batch_stride, size_t* buffer_size);
  // CHECK: status_t = rocsparse_zgpsv_interleaved_batch_buffer_size(handle_t, algo, m, &dcomplexds, &dcomplexdl, &dcomplexd, &dcomplexdu, &dcomplexdw, &dcomplexx, batchCount, batchCount, &bufferSize);
  status_t = hipsparseZgpsvInterleavedBatch_bufferSizeExt(handle_t, algo, m, &dcomplexds, &dcomplexdl, &dcomplexd, &dcomplexdu, &dcomplexdw, &dcomplexx, batchCount, &bufferSize);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_cgpsv_interleaved_batch_buffer_size function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseCgpsvInterleavedBatch_bufferSizeExt calls rocsparse_cgpsv_interleaved_batch_buffer_size in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgpsvInterleavedBatch_bufferSizeExt(cusparseHandle_t handle, int algo, int m, const cuComplex* ds, const cuComplex* dl, const cuComplex* d, const cuComplex* du, const cuComplex* dw, const cuComplex* x, int batchCount, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cgpsv_interleaved_batch_buffer_size(rocsparse_handle handle, rocsparse_gpsv_interleaved_alg alg, rocsparse_int m, const rocsparse_float_complex* ds, const rocsparse_float_complex* dl, const rocsparse_float_complex* d, const rocsparse_float_complex* du, const rocsparse_float_complex* dw, const rocsparse_float_complex* x, rocsparse_int batch_count, rocsparse_int batch_stride, size_t* buffer_size);
  // CHECK: status_t = rocsparse_cgpsv_interleaved_batch_buffer_size(handle_t, algo, m, &complexds, &complexdl, &complexd, &complexdu, &complexdw, &complexx, batchCount, batchCount, &bufferSize);
  status_t = hipsparseCgpsvInterleavedBatch_bufferSizeExt(handle_t, algo, m, &complexds, &complexdl, &complexd, &complexdu, &complexdw, &complexx, batchCount, &bufferSize);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_dgpsv_interleaved_batch_buffer_size function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseDgpsvInterleavedBatch_bufferSizeExt calls rocsparse_dgpsv_interleaved_batch_buffer_size in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgpsvInterleavedBatch_bufferSizeExt(cusparseHandle_t handle, int algo, int m, const double* ds, const double* dl, const double* d, const double* du, const double* dw, const double* x, int batchCount, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dgpsv_interleaved_batch_buffer_size(rocsparse_handle handle, rocsparse_gpsv_interleaved_alg alg, rocsparse_int m, const double* ds, const double* dl, const double* d, const double* du, const double* dw, const double* x, rocsparse_int batch_count, rocsparse_int batch_stride, size_t* buffer_size);
  // CHECK: status_t = rocsparse_dgpsv_interleaved_batch_buffer_size(handle_t, algo, m, &dds, &ddl, &dd, &ddu, &ddw, &dx, batchCount, batchCount, &bufferSize);
  status_t = hipsparseDgpsvInterleavedBatch_bufferSizeExt(handle_t, algo, m, &dds, &ddl, &dd, &ddu, &ddw, &dx, batchCount, &bufferSize);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_dgpsv_interleaved_batch_buffer_size function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseSgpsvInterleavedBatch_bufferSizeExt calls rocsparse_dgpsv_interleaved_batch_buffer_size in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgpsvInterleavedBatch_bufferSizeExt(cusparseHandle_t handle, int algo, int m, const float* ds, const float* dl, const float* d, const float* du, const float* dw, const float* x, int batchCount, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sgpsv_interleaved_batch_buffer_size(rocsparse_handle handle, rocsparse_gpsv_interleaved_alg alg, rocsparse_int m, const float* ds, const float* dl, const float* d, const float* du, const float* dw, const float* x, rocsparse_int batch_count, rocsparse_int batch_stride, size_t* buffer_size);
  // CHECK: status_t = rocsparse_sgpsv_interleaved_batch_buffer_size(handle_t, algo, m, &fds, &fdl, &fd, &fdu, &fdw, &fx, batchCount, batchCount, &bufferSize);
  status_t = hipsparseSgpsvInterleavedBatch_bufferSizeExt(handle_t, algo, m, &fds, &fdl, &fd, &fdu, &fdw, &fx, batchCount, &bufferSize);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_zgtsv_interleaved_batch function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseZgtsvInterleavedBatch calls rocsparse_zgtsv_interleaved_batch in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgtsvInterleavedBatch(cusparseHandle_t handle, int algo, int m, cuDoubleComplex* dl, cuDoubleComplex* d, cuDoubleComplex* du, cuDoubleComplex* x, int batchCount, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zgtsv_interleaved_batch(rocsparse_handle handle, rocsparse_gtsv_interleaved_alg alg, rocsparse_int m, rocsparse_double_complex* dl, rocsparse_double_complex* d, rocsparse_double_complex* du, rocsparse_double_complex* x, rocsparse_int batch_count, rocsparse_int batch_stride, void* temp_buffer);
  // CHECK: status_t = rocsparse_zgtsv_interleaved_batch(handle_t, algo, m, &dcomplexdl, &dcomplexd, &dcomplexdu, &dcomplexx, batchCount, batchCount, pBuffer);
  status_t = hipsparseZgtsvInterleavedBatch(handle_t, algo, m, &dcomplexdl, &dcomplexd, &dcomplexdu, &dcomplexx, batchCount, pBuffer);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_zgtsv_interleaved_batch function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseCgtsvInterleavedBatch calls rocsparse_zgtsv_interleaved_batch in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgtsvInterleavedBatch(cusparseHandle_t handle, int algo, int m, cuComplex* dl, cuComplex* d, cuComplex* du, cuComplex* x, int batchCount, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cgtsv_interleaved_batch(rocsparse_handle handle, rocsparse_gtsv_interleaved_alg alg, rocsparse_int m, rocsparse_float_complex* dl, rocsparse_float_complex* d, rocsparse_float_complex* du, rocsparse_float_complex* x, rocsparse_int batch_count, rocsparse_int batch_stride, void* temp_buffer);
  // CHECK: status_t = rocsparse_cgtsv_interleaved_batch(handle_t, algo, m, &complexdl, &complexd, &complexdu, &complexx, batchCount, batchCount, pBuffer);
  status_t = hipsparseCgtsvInterleavedBatch(handle_t, algo, m, &complexdl, &complexd, &complexdu, &complexx, batchCount, pBuffer);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_dgtsv_interleaved_batch function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseDgtsvInterleavedBatch calls rocsparse_dgtsv_interleaved_batch in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgtsvInterleavedBatch(cusparseHandle_t handle, int algo, int m, double* dl, double* d, double* du, double* x, int batchCount, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dgtsv_interleaved_batch(rocsparse_handle handle, rocsparse_gtsv_interleaved_alg alg, rocsparse_int m, double* dl, double* d, double* du, double* x, rocsparse_int batch_count, rocsparse_int batch_stride, void* temp_buffer);
  // CHECK: status_t = rocsparse_dgtsv_interleaved_batch(handle_t, algo, m, &ddl, &dd, &ddu, &dx, batchCount, batchCount, pBuffer);
  status_t = hipsparseDgtsvInterleavedBatch(handle_t, algo, m, &ddl, &dd, &ddu, &dx, batchCount, pBuffer);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_sgtsv_interleaved_batch function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseSgtsvInterleavedBatch calls rocsparse_sgtsv_interleaved_batch in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgtsvInterleavedBatch(cusparseHandle_t handle, int algo, int m, float* dl, float* d, float* du, float* x, int batchCount, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sgtsv_interleaved_batch(rocsparse_handle handle, rocsparse_gtsv_interleaved_alg alg, rocsparse_int m, float* dl, float* d, float* du, float* x, rocsparse_int batch_count, rocsparse_int batch_stride, void* temp_buffer);
  // CHECK: status_t = rocsparse_sgtsv_interleaved_batch(handle_t, algo, m, &fdl, &fd, &fdu, &fx, batchCount, batchCount, pBuffer);
  status_t = hipsparseSgtsvInterleavedBatch(handle_t, algo, m, &fdl, &fd, &fdu, &fx, batchCount, pBuffer);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_zgtsv_interleaved_batch_buffer_size function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseZgtsvInterleavedBatch_bufferSizeExt calls rocsparse_zgtsv_interleaved_batch_buffer_size in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgtsvInterleavedBatch_bufferSizeExt(cusparseHandle_t handle, int algo, int m, const cuDoubleComplex* dl, const cuDoubleComplex* d, const cuDoubleComplex* du, const cuDoubleComplex* x, int batchCount, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zgtsv_interleaved_batch_buffer_size(rocsparse_handle handle, rocsparse_gtsv_interleaved_alg alg, rocsparse_int m, const rocsparse_double_complex* dl, const rocsparse_double_complex* d, const rocsparse_double_complex* du, const rocsparse_double_complex* x, rocsparse_int batch_count, rocsparse_int batch_stride, size_t* buffer_size);
  // CHECK: status_t = rocsparse_zgtsv_interleaved_batch_buffer_size(handle_t, algo, m, &dcomplexdl, &dcomplexd, &dcomplexdu, &dcomplexx, batchCount, batchCount, &bufferSize);
  status_t = hipsparseZgtsvInterleavedBatch_bufferSizeExt(handle_t, algo, m, &dcomplexdl, &dcomplexd, &dcomplexdu, &dcomplexx, batchCount, &bufferSize);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_cgtsv_interleaved_batch_buffer_size function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseCgtsvInterleavedBatch_bufferSizeExt calls rocsparse_cgtsv_interleaved_batch_buffer_size in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgtsvInterleavedBatch_bufferSizeExt(cusparseHandle_t handle, int algo, int m, const cuComplex* dl, const cuComplex* d, const cuComplex* du, const cuComplex* x, int batchCount, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cgtsv_interleaved_batch_buffer_size(rocsparse_handle handle, rocsparse_gtsv_interleaved_alg alg, rocsparse_int m, const rocsparse_float_complex* dl, const rocsparse_float_complex* d, const rocsparse_float_complex* du, const rocsparse_float_complex* x, rocsparse_int batch_count, rocsparse_int batch_stride, size_t* buffer_size);
  // CHECK: status_t = rocsparse_cgtsv_interleaved_batch_buffer_size(handle_t, algo, m, &complexdl, &complexd, &complexdu, &complexx, batchCount, batchCount, &bufferSize);
  status_t = hipsparseCgtsvInterleavedBatch_bufferSizeExt(handle_t, algo, m, &complexdl, &complexd, &complexdu, &complexx, batchCount, &bufferSize);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_cgtsv_interleaved_batch_buffer_size function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseDgtsvInterleavedBatch_bufferSizeExt calls rocsparse_cgtsv_interleaved_batch_buffer_size in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgtsvInterleavedBatch_bufferSizeExt(cusparseHandle_t handle, int algo, int m, const double* dl, const double* d, const double* du, const double* x, int batchCount, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dgtsv_interleaved_batch_buffer_size(rocsparse_handle handle, rocsparse_gtsv_interleaved_alg alg, rocsparse_int m, const double* dl, const double* d, const double* du, const double* x, rocsparse_int batch_count, rocsparse_int batch_stride, size_t* buffer_size);
  // CHECK: status_t = rocsparse_dgtsv_interleaved_batch_buffer_size(handle_t, algo, m, &ddl, &dd, &ddu, &dx, batchCount, batchCount, &bufferSize);
  status_t = hipsparseDgtsvInterleavedBatch_bufferSizeExt(handle_t, algo, m, &ddl, &dd, &ddu, &dx, batchCount, &bufferSize);

  // NOTE: An additional argument rocsparse_int batch_stride is added for the rocsparse_sgtsv_interleaved_batch_buffer_size function call: the argument is copied from the previous one: rocsparse_int batch_count. It is how hipsparseSgtsvInterleavedBatch_bufferSizeExt calls rocsparse_sgtsv_interleaved_batch_buffer_size in its implementation.
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgtsvInterleavedBatch_bufferSizeExt(cusparseHandle_t handle, int algo, int m, const float* dl, const float* d, const float* du, const float* x, int batchCount, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sgtsv_interleaved_batch_buffer_size(rocsparse_handle handle, rocsparse_gtsv_interleaved_alg alg, rocsparse_int m, const float* dl, const float* d, const float* du, const float* x, rocsparse_int batch_count, rocsparse_int batch_stride, size_t* buffer_size);
  // CHECK: status_t = rocsparse_sgtsv_interleaved_batch_buffer_size(handle_t, algo, m, &fdl, &fd, &fdu, &fx, batchCount, batchCount, &bufferSize);
  status_t = hipsparseSgtsvInterleavedBatch_bufferSizeExt(handle_t, algo, m, &fdl, &fd, &fdu, &fx, batchCount, &bufferSize);
#endif

  return 0;
}
