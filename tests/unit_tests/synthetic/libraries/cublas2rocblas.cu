// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental --roc %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocblas.h"
// CHECK-NOT: #include "hipblas.h"
#include "hipblas.h"
#include "hipblas.h"
// CHECK-NOT: #include "rocblas.h"

int main() {
  printf("16. cuBLAS API to hipBLAS API synthetic test\n");

  // CHECK: rocblas_operation blasOperation;
  // CHECK-NEXT: rocblas_operation BLAS_OP_N = rocblas_operation_none;
  // CHECK-NEXT: rocblas_operation BLAS_OP_T = rocblas_operation_transpose;
  // CHECK-NEXT: rocblas_operation BLAS_OP_C = rocblas_operation_conjugate_transpose;
  hipblasOperation_t blasOperation;
  hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;

  // CHECK: rocblas_status blasStatus;
  // CHECK-NEXT: rocblas_status blasStatus_t;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_SUCCESS = rocblas_status_success;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_NOT_INITIALIZED = rocblas_status_invalid_handle;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_ALLOC_FAILED = rocblas_status_not_implemented;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_INVALID_VALUE = rocblas_status_invalid_pointer;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_MAPPING_ERROR = rocblas_status_invalid_size;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_EXECUTION_FAILED = rocblas_status_memory_error;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_INTERNAL_ERROR = rocblas_status_internal_error;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_NOT_SUPPORTED = rocblas_status_perf_degraded;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_ARCH_MISMATCH = rocblas_status_size_query_mismatch;
  hipblasStatus_t blasStatus;
  hipblasStatus_t blasStatus_t;
  hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;

  // CHECK: rocblas_fill blasFillMode;
  // CHECK-NEXT: rocblas_fill BLAS_FILL_MODE_LOWER = rocblas_fill_lower;
  // CHECK-NEXT: rocblas_fill BLAS_FILL_MODE_UPPER = rocblas_fill_upper;
  hipblasFillMode_t blasFillMode;
  hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;

  // CHECK: rocblas_diagonal blasDiagType;
  // CHECK-NEXT: rocblas_diagonal BLAS_DIAG_NON_UNIT = rocblas_diagonal_non_unit;
  // CHECK-NEXT: rocblas_diagonal BLAS_DIAG_UNIT = rocblas_diagonal_unit;
  hipblasDiagType_t blasDiagType;
  hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;

  // CHECK: rocblas_side blasSideMode;
  // CHECK-NEXT: rocblas_side BLAS_SIDE_LEFT = rocblas_side_left;
  // CHECK-NEXT: rocblas_side BLAS_SIDE_RIGHT = rocblas_side_right;
  hipblasSideMode_t blasSideMode;
  hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;

  // CHECK: rocblas_pointer_mode blasPointerMode;
  // CHECK-NEXT: rocblas_pointer_mode BLAS_POINTER_MODE_HOST = rocblas_pointer_mode_host;
  // CHECK-NEXT: rocblas_pointer_mode BLAS_POINTER_MODE_DEVICE = rocblas_pointer_mode_device;
  hipblasPointerMode_t blasPointerMode;
  hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;

  // CHECK: rocblas_atomics_mode blasAtomicsMode;
  // CHECK-NEXT: rocblas_atomics_mode BLAS_ATOMICS_NOT_ALLOWED = rocblas_atomics_not_allowed;
  // CHECK-NEXT: rocblas_atomics_mode BLAS_ATOMICS_ALLOWED = rocblas_atomics_allowed;
  hipblasAtomicsMode_t blasAtomicsMode;
  hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;

  // CHECK: rocblas_gemm_algo blasGemmAlgo;
  // CHECK-NEXT: rocblas_gemm_algo BLAS_GEMM_DFALT = rocblas_gemm_algo_standard;
  hipblasGemmAlgo_t blasGemmAlgo;
  hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;

  // CHECK: rocblas_handle blasHandle;
  hipblasHandle_t blasHandle;

  // CUDA: hipblasStatus_t CUBLASWINAPI cublasInit(void);
  // ROC: ROCBLAS_EXPORT void rocblas_initialize(void);
  // CHECK: blasStatus = rocblas_initialize();
  blasStatus = cublasInit();

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasGetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t* mode);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_atomics_mode(rocblas_handle handle, rocblas_atomics_mode* atomics_mode);
  // CHECK: blasStatus = rocblas_get_atomics_mode(blasHandle, &blasAtomicsMode);
  blasStatus = hipblasGetAtomicsMode(blasHandle, &blasAtomicsMode);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t mode);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_atomics_mode(rocblas_handle handle, rocblas_atomics_mode atomics_mode);
  // CHECK: blasStatus = rocblas_set_atomics_mode(blasHandle, blasAtomicsMode);
  blasStatus = hipblasSetAtomicsMode(blasHandle, blasAtomicsMode);

  const char* const_ch = nullptr;

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCreate(hipblasHandle_t* handle);
  // CUDA: #define hipblasCreate hipblasCreate
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_create_handle(rocblas_handle* handle);
  // CHECK: blasStatus = rocblas_create_handle(&blasHandle);
  // CHECK-NEXT: blasStatus = rocblas_create_handle(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDestroy(hipblasHandle_t handle);
  // CUDA: #define hipblasDestroy hipblasDestroy
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_destroy_handle(rocblas_handle handle);
  // CHECK: blasStatus = rocblas_destroy_handle(blasHandle);
  // CHECK-NEXT: blasStatus = rocblas_destroy_handle(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);

  // CHECK: hipStream_t stream;
  hipStream_t stream;

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSetStream(hipblasHandle_t handle, hipStream_t streamId);
  // CUDA: #define hipblasSetStream hipblasSetStream
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_stream(rocblas_handle handle, hipStream_t stream);
  // CHECK: blasStatus = rocblas_set_stream(blasHandle, stream);
  // CHECK-NEXT: blasStatus = rocblas_set_stream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasGetStream(hipblasHandle_t handle, hipStream_t* streamId);
  // CUDA: #define hipblasGetStream hipblasGetStream
  // HIP: ROCBLAS_EXPORT rocblas_status rocblas_get_stream(rocblas_handle handle, hipStream_t* stream);
  // CHECK: blasStatus = rocblas_get_stream(blasHandle, &stream);
  // CHECK-NEXT: blasStatus = rocblas_get_stream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);

#if CUDA_VERSION >= 8000
  // CHECK: rocblas_datatype DataType;
  // CHECK-NEXT: rocblas_datatype_ DataType_t;
  // CHECK-NEXT: rocblas_datatype blasDataType;
  // CHECK-NEXT: rocblas_datatype R_16F = rocblas_datatype_f16_r;
  // CHECK-NEXT: rocblas_datatype C_16F = rocblas_datatype_f16_c;
  // CHECK-NEXT: rocblas_datatype R_32F = rocblas_datatype_f32_r;
  // CHECK-NEXT: rocblas_datatype C_32F = rocblas_datatype_f32_c;
  // CHECK-NEXT: rocblas_datatype R_64F = rocblas_datatype_f64_r;
  // CHECK-NEXT: rocblas_datatype C_64F = rocblas_datatype_f64_c;
  // CHECK-NEXT: rocblas_datatype R_8I = rocblas_datatype_i8_r;
  // CHECK-NEXT: rocblas_datatype C_8I = rocblas_datatype_i8_c;
  // CHECK-NEXT: rocblas_datatype R_8U = rocblas_datatype_u8_r;
  // CHECK-NEXT: rocblas_datatype C_8U = rocblas_datatype_u8_c;
  // CHECK-NEXT: rocblas_datatype R_32I = rocblas_datatype_i32_r;
  // CHECK-NEXT: rocblas_datatype C_32I = rocblas_datatype_i32_c;
  // CHECK-NEXT: rocblas_datatype R_32U = rocblas_datatype_u32_r;
  // CHECK-NEXT: rocblas_datatype C_32U = rocblas_datatype_u32_c;
  hipDataType DataType;
  hipDataType DataType_t;
  hipDataType blasDataType;
  hipDataType R_16F = HIP_R_16F;
  hipDataType C_16F = HIP_C_16F;
  hipDataType R_32F = HIP_R_32F;
  hipDataType C_32F = HIP_C_32F;
  hipDataType R_64F = HIP_R_64F;
  hipDataType C_64F = HIP_C_64F;
  hipDataType R_8I = HIP_R_8I;
  hipDataType C_8I = HIP_C_8I;
  hipDataType R_8U = HIP_R_8U;
  hipDataType C_8U = HIP_C_8U;
  hipDataType R_32I = HIP_R_32I;
  hipDataType C_32I = HIP_C_32I;
  hipDataType R_32U = HIP_R_32U;
  hipDataType C_32U = HIP_C_32U;
#endif

#if CUDA_VERSION >= 9000
  // CHECK: rocblas_gemm_algo BLAS_GEMM_DEFAULT = rocblas_gemm_algo_standard;
  hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;
#endif

#if CUDA_VERSION >= 10010
  // CHECK: rocblas_operation BLAS_OP_HERMITAN = rocblas_operation_conjugate_transpose;
  hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;

  // CHECK: rocblas_fill BLAS_FILL_MODE_FULL = rocblas_fill_full;
  hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;
#endif

#if CUDA_VERSION >= 11000
  // CHECK: rocblas_datatype R_16BF = rocblas_datatype_bf16_r;
  // CHECK-NEXT: rocblas_datatype C_16BF = rocblas_datatype_bf16_c;
  hipDataType R_16BF = HIP_R_16BF;
  hipDataType C_16BF = HIP_C_16BF;
#endif

#if CUDA_VERSION >= 11040
  // CUDA: CUBLASAPI const char* CUBLASWINAPI cublasGetStatusString(hipblasStatus_t status);
  // ROC: ROCBLAS_EXPORT const char* rocblas_status_to_string(rocblas_status status);
  // CHECK: const_ch = rocblas_status_to_string(blasStatus);
  const_ch = cublasGetStatusString(blasStatus);
#endif

  return 0;
}
