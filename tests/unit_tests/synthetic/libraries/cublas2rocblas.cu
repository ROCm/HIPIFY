// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental --roc %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocblas.h"
// CHECK-NOT: #include "hipblas.h"
#include "hipblas.h"
#include "hipblas.h"
// CHECK-NOT: #include "rocblas.h"

int main() {
  printf("16. cuBLAS API to hipBLAS API synthetic test\n");

  // CHECK: rocblas_operation blasOperation;
  // CHECK-NEXT: rocblas_operation BLAS_OP_N = rocblas_operation_none;
  // CHECK-NEXT: rocblas_operation BLAS_OP_T = rocblas_operation_transpose;
  // CHECK-NEXT: rocblas_operation BLAS_OP_C = rocblas_operation_conjugate_transpose;
  hipblasOperation_t blasOperation;
  hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;

  // CHECK: rocblas_status blasStatus;
  // CHECK-NEXT: rocblas_status blasStatus_t;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_SUCCESS = rocblas_status_success;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_NOT_INITIALIZED = rocblas_status_invalid_handle;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_ALLOC_FAILED = rocblas_status_not_implemented;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_INVALID_VALUE = rocblas_status_invalid_pointer;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_MAPPING_ERROR = rocblas_status_invalid_size;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_EXECUTION_FAILED = rocblas_status_memory_error;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_INTERNAL_ERROR = rocblas_status_internal_error;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_NOT_SUPPORTED = rocblas_status_perf_degraded;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_ARCH_MISMATCH = rocblas_status_size_query_mismatch;
  hipblasStatus_t blasStatus;
  hipblasStatus_t blasStatus_t;
  hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;

  // CHECK: rocblas_fill blasFillMode;
  // CHECK-NEXT: rocblas_fill BLAS_FILL_MODE_LOWER = rocblas_fill_lower;
  // CHECK-NEXT: rocblas_fill BLAS_FILL_MODE_UPPER = rocblas_fill_upper;
  hipblasFillMode_t blasFillMode;
  hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;

  // CHECK: rocblas_diagonal blasDiagType;
  // CHECK-NEXT: rocblas_diagonal BLAS_DIAG_NON_UNIT = rocblas_diagonal_non_unit;
  // CHECK-NEXT: rocblas_diagonal BLAS_DIAG_UNIT = rocblas_diagonal_unit;
  hipblasDiagType_t blasDiagType;
  hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;

  // CHECK: rocblas_side blasSideMode;
  // CHECK-NEXT: rocblas_side BLAS_SIDE_LEFT = rocblas_side_left;
  // CHECK-NEXT: rocblas_side BLAS_SIDE_RIGHT = rocblas_side_right;
  hipblasSideMode_t blasSideMode;
  hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;

  // CHECK: rocblas_pointer_mode blasPointerMode;
  // CHECK-NEXT: rocblas_pointer_mode BLAS_POINTER_MODE_HOST = rocblas_pointer_mode_host;
  // CHECK-NEXT: rocblas_pointer_mode BLAS_POINTER_MODE_DEVICE = rocblas_pointer_mode_device;
  hipblasPointerMode_t blasPointerMode;
  hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;

  // CHECK: rocblas_atomics_mode blasAtomicsMode;
  // CHECK-NEXT: rocblas_atomics_mode BLAS_ATOMICS_NOT_ALLOWED = rocblas_atomics_not_allowed;
  // CHECK-NEXT: rocblas_atomics_mode BLAS_ATOMICS_ALLOWED = rocblas_atomics_allowed;
  hipblasAtomicsMode_t blasAtomicsMode;
  hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;

  // CHECK: rocblas_gemm_algo blasGemmAlgo;
  // CHECK-NEXT: rocblas_gemm_algo BLAS_GEMM_DFALT = rocblas_gemm_algo_standard;
  hipblasGemmAlgo_t blasGemmAlgo;
  hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;

  // CHECK: rocblas_handle blasHandle;
  hipblasHandle_t blasHandle;

  // CUDA: hipblasStatus_t CUBLASWINAPI cublasInit(void);
  // ROC: ROCBLAS_EXPORT void rocblas_initialize(void);
  // CHECK: blasStatus = rocblas_initialize();
  blasStatus = cublasInit();

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasGetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t* mode);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_atomics_mode(rocblas_handle handle, rocblas_atomics_mode* atomics_mode);
  // CHECK: blasStatus = rocblas_get_atomics_mode(blasHandle, &blasAtomicsMode);
  blasStatus = hipblasGetAtomicsMode(blasHandle, &blasAtomicsMode);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t mode);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_atomics_mode(rocblas_handle handle, rocblas_atomics_mode atomics_mode);
  // CHECK: blasStatus = rocblas_set_atomics_mode(blasHandle, blasAtomicsMode);
  blasStatus = hipblasSetAtomicsMode(blasHandle, blasAtomicsMode);

  const char* const_ch = nullptr;

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCreate(hipblasHandle_t* handle);
  // CUDA: #define hipblasCreate hipblasCreate
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_create_handle(rocblas_handle* handle);
  // CHECK: blasStatus = rocblas_create_handle(&blasHandle);
  // CHECK-NEXT: blasStatus = rocblas_create_handle(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDestroy(hipblasHandle_t handle);
  // CUDA: #define hipblasDestroy hipblasDestroy
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_destroy_handle(rocblas_handle handle);
  // CHECK: blasStatus = rocblas_destroy_handle(blasHandle);
  // CHECK-NEXT: blasStatus = rocblas_destroy_handle(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);

  // CHECK: hipStream_t stream;
  hipStream_t stream;

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSetStream(hipblasHandle_t handle, hipStream_t streamId);
  // CUDA: #define hipblasSetStream hipblasSetStream
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_stream(rocblas_handle handle, hipStream_t stream);
  // CHECK: blasStatus = rocblas_set_stream(blasHandle, stream);
  // CHECK-NEXT: blasStatus = rocblas_set_stream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasGetStream(hipblasHandle_t handle, hipStream_t* streamId);
  // CUDA: #define hipblasGetStream hipblasGetStream
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_stream(rocblas_handle handle, hipStream_t* stream);
  // CHECK: blasStatus = rocblas_get_stream(blasHandle, &stream);
  // CHECK-NEXT: blasStatus = rocblas_get_stream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSetPointerMode(hipblasHandle_t handle, hipblasPointerMode_t mode);
  // CUDA: #define hipblasSetPointerMode hipblasSetPointerMode
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_pointer_mode(rocblas_handle handle, rocblas_pointer_mode pointer_mode);
  // CHECK: blasStatus = rocblas_set_pointer_mode(blasHandle, blasPointerMode);
  // CHECK-NEXT: blasStatus = rocblas_set_pointer_mode(blasHandle, blasPointerMode);
  blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);
  blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasGetPointerMode(hipblasHandle_t handle, hipblasPointerMode_t* mode);
  // CUDA: #define hipblasGetPointerMode hipblasGetPointerMode
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_pointer_mode(rocblas_handle handle, rocblas_pointer_mode* pointer_mode);
  // CHECK: blasStatus = rocblas_get_pointer_mode(blasHandle, &blasPointerMode);
  // CHECK-NEXT: blasStatus = rocblas_get_pointer_mode(blasHandle, &blasPointerMode);
  blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);
  blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);

  int n = 0;
  int num = 0;
  int incx = 0;
  int incy = 0;
  void* image = nullptr;
  void* image_2 = nullptr;

  // https://github.com/ROCmSoftwarePlatform/rocBLAS/issues/1281
  // TODO: Apply the chosen typecasting of int to rocblas_int arguments

  /*
  #if defined(rocblas_ILP64)
    typedef int64_t rocblas_int;
  #else
    typedef int32_t rocblas_int;
  #endif
  */

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasSetVector(int n, int elemSize, const void* x, int incx, void* devicePtr, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_vector(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_set_vector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasSetVector(n, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasGetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_vector(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_get_vector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasGetVector(n, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasSetVectorAsync(int n, int elemSize, const void* hostPtr, int incx, void* devicePtr, int incy, hipStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_vector_async(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy, hipStream_t stream);
  // CHECK: blasStatus = rocblas_set_vector_async(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetVectorAsync(n, num, image, incx, image_2, incy, stream);

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasGetVectorAsync(int n, int elemSize, const void* devicePtr, int incx, void* hostPtr, int incy, hipStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_vector_async(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy, hipStream_t stream);
  // CHECK: blasStatus = rocblas_get_vector_async(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetVectorAsync(n, num, image, incx, image_2, incy, stream);

  int rows = 0;
  int cols = 0;

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasSetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_matrix(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_set_matrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasSetMatrix(rows, cols, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasGetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_matrix(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_get_matrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasGetMatrix(rows, cols, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasSetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, hipStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_matrix_async(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb, hipStream_t stream);
  // CHECK: blasStatus = rocblas_set_matrix_async(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasGetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, hipStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_matrix_async(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb, hipStream_t stream);
  // CHECK: blasStatus = rocblas_get_matrix_async(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  float fa = 0;
  float fA = 0;
  float fb = 0;
  float fB = 0;
  float fx = 0;
  float fx1 = 0;
  float fy = 0;
  float fy1 = 0;
  float fc = 0;
  float fC = 0;
  float fs = 0;
  float fd1 = 0;
  float fd2 = 0;
  float fresult = 0;

  float** fAarray = 0;
  float** fBarray = 0;
  float** fCarray = 0;
  float** fTauarray = 0;

  double da = 0;
  double dA = 0;
  double db = 0;
  double dB = 0;
  double dx = 0;
  double dx1 = 0;
  double dy = 0;
  double dy1 = 0;
  double dc = 0;
  double dC = 0;
  double ds = 0;
  double dd1 = 0;
  double dd2 = 0;
  double dresult = 0;

  double** dAarray = 0;
  double** dBarray = 0;
  double** dCarray = 0;
  double** dTauarray = 0;

  void** voidAarray = nullptr;
  void** voidBarray = nullptr;
  void** voidCarray = nullptr;

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSnrm2(hipblasHandle_t handle, int n, const float* x, int incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_snrm2(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, float* result);
  // CHECK: blasStatus = rocblas_snrm2(blasHandle, n, &fx, incx, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_snrm2(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDnrm2(hipblasHandle_t handle, int n, const double* x, int incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dnrm2(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, double* result);
  // CHECK: blasStatus = rocblas_dnrm2(blasHandle, n, &dx, incx, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_dnrm2(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);

  // CHECK: rocblas_float_complex complex, complexa, complexA, complexB, complexC, complexx, complexy, complexs, complexb;
  hipComplex complex, complexa, complexA, complexB, complexC, complexx, complexy, complexs, complexb;
  // CHECK: rocblas_double_complex dcomplex, dcomplexa, dcomplexA, dcomplexB, dcomplexC, dcomplexx, dcomplexy, dcomplexs, dcomplexb;
  hipDoubleComplex dcomplex, dcomplexa, dcomplexA, dcomplexB, dcomplexC, dcomplexx, dcomplexy, dcomplexs, dcomplexb;

  // CHECK: rocblas_float_complex** complexAarray = 0;
  // CHECK-NEXT: rocblas_float_complex** complexBarray = 0;
  // CHECK-NEXT: rocblas_float_complex** complexCarray = 0;
  // CHECK-NEXT: rocblas_float_complex** complexTauarray = 0;
  hipComplex** complexAarray = 0;
  hipComplex** complexBarray = 0;
  hipComplex** complexCarray = 0;
  hipComplex** complexTauarray = 0;

  // CHECK: rocblas_double_complex** dcomplexAarray = 0;
  // CHECK-NEXT: rocblas_double_complex** dcomplexBarray = 0;
  // CHECK-NEXT: rocblas_double_complex** dcomplexCarray = 0;
  // CHECK-NEXT: rocblas_double_complex** dcomplexTauarray = 0;
  hipDoubleComplex** dcomplexAarray = 0;
  hipDoubleComplex** dcomplexBarray = 0;
  hipDoubleComplex** dcomplexCarray = 0;
  hipDoubleComplex** dcomplexTauarray = 0;

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasScnrm2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scnrm2(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, float* result);
  // CHECK: blasStatus = rocblas_scnrm2(blasHandle, n, &complex, incx, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_scnrm2(blasHandle, n, &complex, incx, &fresult);
  blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);
  blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDznrm2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dznrm2(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, double* result);
  // CHECK: blasStatus = rocblas_dznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_dznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);

#if CUDA_VERSION >= 8000
  // CHECK: rocblas_datatype DataType;
  // CHECK-NEXT: rocblas_datatype_ DataType_t;
  // CHECK-NEXT: rocblas_datatype blasDataType;
  // CHECK-NEXT: rocblas_datatype R_16F = rocblas_datatype_f16_r;
  // CHECK-NEXT: rocblas_datatype C_16F = rocblas_datatype_f16_c;
  // CHECK-NEXT: rocblas_datatype R_32F = rocblas_datatype_f32_r;
  // CHECK-NEXT: rocblas_datatype C_32F = rocblas_datatype_f32_c;
  // CHECK-NEXT: rocblas_datatype R_64F = rocblas_datatype_f64_r;
  // CHECK-NEXT: rocblas_datatype C_64F = rocblas_datatype_f64_c;
  // CHECK-NEXT: rocblas_datatype R_8I = rocblas_datatype_i8_r;
  // CHECK-NEXT: rocblas_datatype C_8I = rocblas_datatype_i8_c;
  // CHECK-NEXT: rocblas_datatype R_8U = rocblas_datatype_u8_r;
  // CHECK-NEXT: rocblas_datatype C_8U = rocblas_datatype_u8_c;
  // CHECK-NEXT: rocblas_datatype R_32I = rocblas_datatype_i32_r;
  // CHECK-NEXT: rocblas_datatype C_32I = rocblas_datatype_i32_c;
  // CHECK-NEXT: rocblas_datatype R_32U = rocblas_datatype_u32_r;
  // CHECK-NEXT: rocblas_datatype C_32U = rocblas_datatype_u32_c;
  hipDataType DataType;
  hipDataType DataType_t;
  hipDataType blasDataType;
  hipDataType R_16F = HIP_R_16F;
  hipDataType C_16F = HIP_C_16F;
  hipDataType R_32F = HIP_R_32F;
  hipDataType C_32F = HIP_C_32F;
  hipDataType R_64F = HIP_R_64F;
  hipDataType C_64F = HIP_C_64F;
  hipDataType R_8I = HIP_R_8I;
  hipDataType C_8I = HIP_C_8I;
  hipDataType R_8U = HIP_R_8U;
  hipDataType C_8U = HIP_C_8U;
  hipDataType R_32I = HIP_R_32I;
  hipDataType C_32I = HIP_C_32I;
  hipDataType R_32U = HIP_R_32U;
  hipDataType C_32U = HIP_C_32U;
#endif

#if CUDA_VERSION >= 9000
  // CHECK: rocblas_gemm_algo BLAS_GEMM_DEFAULT = rocblas_gemm_algo_standard;
  hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;
#endif

#if CUDA_VERSION >= 10010
  // CHECK: rocblas_operation BLAS_OP_HERMITAN = rocblas_operation_conjugate_transpose;
  hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;

  // CHECK: rocblas_fill BLAS_FILL_MODE_FULL = rocblas_fill_full;
  hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;
#endif

#if CUDA_VERSION >= 11000
  // CHECK: rocblas_datatype R_16BF = rocblas_datatype_bf16_r;
  // CHECK-NEXT: rocblas_datatype C_16BF = rocblas_datatype_bf16_c;
  hipDataType R_16BF = HIP_R_16BF;
  hipDataType C_16BF = HIP_C_16BF;
#endif

#if CUDA_VERSION >= 11040
  // CUDA: CUBLASAPI const char* CUBLASWINAPI cublasGetStatusString(hipblasStatus_t status);
  // ROC: ROCBLAS_EXPORT const char* rocblas_status_to_string(rocblas_status status);
  // CHECK: const_ch = rocblas_status_to_string(blasStatus);
  const_ch = cublasGetStatusString(blasStatus);
#endif

  return 0;
}
