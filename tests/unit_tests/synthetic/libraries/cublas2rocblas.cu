// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental --roc %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocblas.h"
// CHECK-NOT: #include "hipblas.h"
#include "hipblas.h"
#include "hipblas.h"
// CHECK-NOT: #include "rocblas.h"

int main() {
  printf("16. cuBLAS API to hipBLAS API synthetic test\n");

  // CHECK: rocblas_operation blasOperation;
  // CHECK-NEXT: rocblas_operation BLAS_OP_N = rocblas_operation_none;
  // CHECK-NEXT: rocblas_operation BLAS_OP_T = rocblas_operation_transpose;
  // CHECK-NEXT: rocblas_operation BLAS_OP_C = rocblas_operation_conjugate_transpose;
  hipblasOperation_t blasOperation;
  hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;

  // CHECK: rocblas_status blasStatus;
  // CHECK-NEXT: rocblas_status blasStatus_t;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_SUCCESS = rocblas_status_success;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_NOT_INITIALIZED = rocblas_status_invalid_handle;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_ALLOC_FAILED = rocblas_status_not_implemented;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_INVALID_VALUE = rocblas_status_invalid_pointer;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_MAPPING_ERROR = rocblas_status_invalid_size;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_EXECUTION_FAILED = rocblas_status_memory_error;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_INTERNAL_ERROR = rocblas_status_internal_error;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_NOT_SUPPORTED = rocblas_status_perf_degraded;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_ARCH_MISMATCH = rocblas_status_size_query_mismatch;
  hipblasStatus_t blasStatus;
  hipblasStatus_t blasStatus_t;
  hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;

  // CHECK: rocblas_fill blasFillMode;
  // CHECK-NEXT: rocblas_fill BLAS_FILL_MODE_LOWER = rocblas_fill_lower;
  // CHECK-NEXT: rocblas_fill BLAS_FILL_MODE_UPPER = rocblas_fill_upper;
  hipblasFillMode_t blasFillMode;
  hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;

  // CHECK: rocblas_diagonal blasDiagType;
  // CHECK-NEXT: rocblas_diagonal BLAS_DIAG_NON_UNIT = rocblas_diagonal_non_unit;
  // CHECK-NEXT: rocblas_diagonal BLAS_DIAG_UNIT = rocblas_diagonal_unit;
  hipblasDiagType_t blasDiagType;
  hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;

  // CHECK: rocblas_side blasSideMode;
  // CHECK-NEXT: rocblas_side BLAS_SIDE_LEFT = rocblas_side_left;
  // CHECK-NEXT: rocblas_side BLAS_SIDE_RIGHT = rocblas_side_right;
  hipblasSideMode_t blasSideMode;
  hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;

  // CHECK: rocblas_pointer_mode blasPointerMode;
  // CHECK-NEXT: rocblas_pointer_mode BLAS_POINTER_MODE_HOST = rocblas_pointer_mode_host;
  // CHECK-NEXT: rocblas_pointer_mode BLAS_POINTER_MODE_DEVICE = rocblas_pointer_mode_device;
  hipblasPointerMode_t blasPointerMode;
  hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;

  // CHECK: rocblas_atomics_mode blasAtomicsMode;
  // CHECK-NEXT: rocblas_atomics_mode BLAS_ATOMICS_NOT_ALLOWED = rocblas_atomics_not_allowed;
  // CHECK-NEXT: rocblas_atomics_mode BLAS_ATOMICS_ALLOWED = rocblas_atomics_allowed;
  hipblasAtomicsMode_t blasAtomicsMode;
  hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;

  // CHECK: rocblas_gemm_algo blasGemmAlgo;
  // CHECK-NEXT: rocblas_gemm_algo BLAS_GEMM_DFALT = rocblas_gemm_algo_standard;
  hipblasGemmAlgo_t blasGemmAlgo;
  hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;

#if CUDA_VERSION >= 9000
  // CHECK: rocblas_gemm_algo BLAS_GEMM_DEFAULT = rocblas_gemm_algo_standard;
  hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;
#endif

#if CUDA_VERSION >= 10010
  // CHECK: rocblas_operation BLAS_OP_HERMITAN = rocblas_operation_conjugate_transpose;
  hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;

  // CHECK: rocblas_fill BLAS_FILL_MODE_FULL = rocblas_fill_full;
  hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;
#endif

  return 0;
}
