// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --amap --skip-excluded-preprocessor-conditional-blocks --experimental --roc %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocblas.h"
#include "hipblas.h"
// CHECK-NOT: #include "rocblas.h"

#if defined(_WIN32) && CUDA_VERSION < 9000
  typedef signed   __int64 int64_t;
  typedef unsigned __int64 uint64_t;
#endif

int main() {
  printf("16. cuBLAS API to hipBLAS API synthetic test\n");

  // CHECK: rocblas_operation blasOperation;
  // CHECK-NEXT: rocblas_operation BLAS_OP_N = rocblas_operation_none;
  // CHECK-NEXT: rocblas_operation BLAS_OP_T = rocblas_operation_transpose;
  // CHECK-NEXT: rocblas_operation BLAS_OP_C = rocblas_operation_conjugate_transpose;
  hipblasOperation_t blasOperation;
  hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;

  // CHECK: rocblas_status blasStatus;
  // CHECK-NEXT: rocblas_status blasStatus_t;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_SUCCESS = rocblas_status_success;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_NOT_INITIALIZED = rocblas_status_invalid_handle;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_ALLOC_FAILED = rocblas_status_not_implemented;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_INVALID_VALUE = rocblas_status_invalid_value;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_MAPPING_ERROR = rocblas_status_invalid_size;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_EXECUTION_FAILED = rocblas_status_memory_error;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_INTERNAL_ERROR = rocblas_status_internal_error;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_NOT_SUPPORTED = rocblas_status_perf_degraded;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_ARCH_MISMATCH = rocblas_status_arch_mismatch;
  hipblasStatus_t blasStatus;
  hipblasStatus_t blasStatus_t;
  hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;

  // CHECK: rocblas_fill blasFillMode;
  // CHECK-NEXT: rocblas_fill BLAS_FILL_MODE_LOWER = rocblas_fill_lower;
  // CHECK-NEXT: rocblas_fill BLAS_FILL_MODE_UPPER = rocblas_fill_upper;
  hipblasFillMode_t blasFillMode;
  hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;

  // CHECK: rocblas_diagonal blasDiagType;
  // CHECK-NEXT: rocblas_diagonal BLAS_DIAG_NON_UNIT = rocblas_diagonal_non_unit;
  // CHECK-NEXT: rocblas_diagonal BLAS_DIAG_UNIT = rocblas_diagonal_unit;
  hipblasDiagType_t blasDiagType;
  hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;

  // CHECK: rocblas_side blasSideMode;
  // CHECK-NEXT: rocblas_side BLAS_SIDE_LEFT = rocblas_side_left;
  // CHECK-NEXT: rocblas_side BLAS_SIDE_RIGHT = rocblas_side_right;
  hipblasSideMode_t blasSideMode;
  hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;

  // CHECK: rocblas_pointer_mode blasPointerMode;
  // CHECK-NEXT: rocblas_pointer_mode BLAS_POINTER_MODE_HOST = rocblas_pointer_mode_host;
  // CHECK-NEXT: rocblas_pointer_mode BLAS_POINTER_MODE_DEVICE = rocblas_pointer_mode_device;
  hipblasPointerMode_t blasPointerMode;
  hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;

  // CHECK: rocblas_atomics_mode blasAtomicsMode;
  // CHECK-NEXT: rocblas_atomics_mode BLAS_ATOMICS_NOT_ALLOWED = rocblas_atomics_not_allowed;
  // CHECK-NEXT: rocblas_atomics_mode BLAS_ATOMICS_ALLOWED = rocblas_atomics_allowed;
  hipblasAtomicsMode_t blasAtomicsMode;
  hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;

  // CHECK: rocblas_handle blasHandle;
  hipblasHandle_t blasHandle;

  // CUDA: cublasStatus CUBLASWINAPI cublasInit(void);
  // ROC: ROCBLAS_EXPORT void rocblas_initialize(void);
  // CHECK: blasStatus = rocblas_initialize();
  blasStatus = cublasInit();

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetAtomicsMode(cublasHandle_t handle, cublasAtomicsMode_t* mode);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_atomics_mode(rocblas_handle handle, rocblas_atomics_mode* atomics_mode);
  // CHECK: blasStatus = rocblas_get_atomics_mode(blasHandle, &blasAtomicsMode);
  blasStatus = hipblasGetAtomicsMode(blasHandle, &blasAtomicsMode);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetAtomicsMode(cublasHandle_t handle, cublasAtomicsMode_t mode);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_atomics_mode(rocblas_handle handle, rocblas_atomics_mode atomics_mode);
  // CHECK: blasStatus = rocblas_set_atomics_mode(blasHandle, blasAtomicsMode);
  blasStatus = hipblasSetAtomicsMode(blasHandle, blasAtomicsMode);

  const char* const_ch = nullptr;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCreate_v2(cublasHandle_t* handle);
  // CUDA: #define cublasCreate cublasCreate_v2
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_create_handle(rocblas_handle* handle);
  // CHECK: blasStatus = rocblas_create_handle(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDestroy_v2(cublasHandle_t handle);
  // CUDA: #define cublasDestroy cublasDestroy_v2
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_destroy_handle(rocblas_handle handle);
  // CHECK: blasStatus = rocblas_destroy_handle(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);

  // CHECK: hipStream_t stream;
  hipStream_t stream;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetStream_v2(cublasHandle_t handle, cudaStream_t streamId);
  // CUDA: #define cublasSetStream cublasSetStream_v2
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_stream(rocblas_handle handle, hipStream_t stream);
  // CHECK: blasStatus = rocblas_set_stream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetStream_v2(cublasHandle_t handle, cudaStream_t* streamId);
  // CUDA: #define cublasGetStream cublasGetStream_v2
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_stream(rocblas_handle handle, hipStream_t* stream);
  // CHECK: blasStatus = rocblas_get_stream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetPointerMode_v2(cublasHandle_t handle, cublasPointerMode_t mode);
  // CUDA: #define cublasSetPointerMode cublasSetPointerMode_v2
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_pointer_mode(rocblas_handle handle, rocblas_pointer_mode pointer_mode);
  // CHECK: blasStatus = rocblas_set_pointer_mode(blasHandle, blasPointerMode);
  blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetPointerMode_v2(cublasHandle_t handle, cublasPointerMode_t* mode);
  // CUDA: #define cublasGetPointerMode cublasGetPointerMode_v2
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_pointer_mode(rocblas_handle handle, rocblas_pointer_mode* pointer_mode);
  // CHECK: blasStatus = rocblas_get_pointer_mode(blasHandle, &blasPointerMode);
  blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);

  int n = 0;
  int m = 0;
  int num = 0;
  int lda = 0;
  int ldb = 0;
  int ldc = 0;
  int res = 0;
  int incx = 0;
  int incy = 0;
  int k = 0;
  int kl = 0;
  int ku = 0;
  int batchCount = 0;
  void* image = nullptr;
  void* image_2 = nullptr;

  // https://github.com/ROCmSoftwarePlatform/rocBLAS/issues/1281
  // TODO: Apply the chosen typecasting of int to rocblas_int arguments

  /*
  #if defined(rocblas_ILP64)
    typedef int64_t rocblas_int;
  #else
    typedef int32_t rocblas_int;
  #endif
  */

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetVector(int n, int elemSize, const void* x, int incx, void* devicePtr, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_vector(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_set_vector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasSetVector(n, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_vector(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_get_vector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasGetVector(n, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetVectorAsync(int n, int elemSize, const void* hostPtr, int incx, void* devicePtr, int incy, cudaStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_vector_async(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy, hipStream_t stream);
  // CHECK: blasStatus = rocblas_set_vector_async(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetVectorAsync(n, num, image, incx, image_2, incy, stream);

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetVectorAsync(int n, int elemSize, const void* devicePtr, int incx, void* hostPtr, int incy, cudaStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_vector_async(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy, hipStream_t stream);
  // CHECK: blasStatus = rocblas_get_vector_async(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetVectorAsync(n, num, image, incx, image_2, incy, stream);

  int rows = 0;
  int cols = 0;

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_matrix(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_set_matrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasSetMatrix(rows, cols, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_matrix(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_get_matrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasGetMatrix(rows, cols, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, cudaStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_matrix_async(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb, hipStream_t stream);
  // CHECK: blasStatus = rocblas_set_matrix_async(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, cudaStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_matrix_async(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb, hipStream_t stream);
  // CHECK: blasStatus = rocblas_get_matrix_async(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  float fa = 0;
  float fA = 0;
  float fb = 0;
  float fB = 0;
  float fx = 0;
  float fx1 = 0;
  float fy = 0;
  float fy1 = 0;
  float fc = 0;
  float fC = 0;
  float fs = 0;
  float fd1 = 0;
  float fd2 = 0;
  float fresult = 0;

  float** fAarray = 0;
  const float** const fAarray_const = const_cast<const float**>(fAarray);
  float** fBarray = 0;
  const float** const fBarray_const = const_cast<const float**>(fBarray);
  float** fCarray = 0;
  float** fTauarray = 0;
  float** fyarray = 0;

  double da = 0;
  double dA = 0;
  double db = 0;
  double dB = 0;
  double dx = 0;
  double dx1 = 0;
  double dy = 0;
  double dy1 = 0;
  double dc = 0;
  double dC = 0;
  double ds = 0;
  double dd1 = 0;
  double dd2 = 0;
  double dresult = 0;

  double** dAarray = 0;
  const double** const dAarray_const = const_cast<const double**>(dAarray);
  double** dBarray = 0;
  const double** const dBarray_const = const_cast<const double**>(dBarray);
  double** dCarray = 0;
  double** dTauarray = 0;

  void** voidAarray = nullptr;
  const void** const voidAarray_const = const_cast<const void**>(voidAarray);
  void** voidBarray = nullptr;
  const void** const voidBarray_const = const_cast<const void**>(voidBarray);
  void** voidCarray = nullptr;

  // TODO: #1281
  // NOTE: float CUBLASWINAPI cublasScnrm2(int n, const cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSnrm2_v2(cublasHandle_t handle, int n, const float* x, int incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_snrm2(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, float* result);
  // CHECK: blasStatus = rocblas_snrm2(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);

  // TODO: #1281
  // NOTE: double CUBLASWINAPI cublasDnrm2(int n, const double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDnrm2_v2(cublasHandle_t handle, int n, const double* x, int incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dnrm2(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, double* result);
  // CHECK: blasStatus = rocblas_dnrm2(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);

  // CHECK: rocblas_float_complex complex, complexa, complexA, complexB, complexC, complexx, complexy, complexs, complexb;
  hipComplex complex, complexa, complexA, complexB, complexC, complexx, complexy, complexs, complexb;
  // CHECK: rocblas_double_complex dcomplex, dcomplexa, dcomplexA, dcomplexB, dcomplexC, dcomplexx, dcomplexy, dcomplexs, dcomplexb;
  hipDoubleComplex dcomplex, dcomplexa, dcomplexA, dcomplexB, dcomplexC, dcomplexx, dcomplexy, dcomplexs, dcomplexb;

  // CHECK: rocblas_float_complex** complexAarray = 0;
  // CHECK: const rocblas_float_complex** const complexAarray_const = const_cast<const rocblas_float_complex**>(complexAarray);
  // CHECK-NEXT: rocblas_float_complex** complexBarray = 0;
  // CHECK: const rocblas_float_complex** const complexBarray_const = const_cast<const rocblas_float_complex**>(complexBarray);
  // CHECK-NEXT: rocblas_float_complex** complexCarray = 0;
  // CHECK-NEXT: rocblas_float_complex** complexTauarray = 0;
  // CHECK-NEXT: rocblas_float_complex** complexXarray = 0;
  // CHECK: const rocblas_float_complex** const complexXarray_const = const_cast<const rocblas_float_complex**>(complexXarray);
  // CHECK-NEXT: rocblas_float_complex** complexYarray = 0;
  // CHECK: const rocblas_float_complex** const complexYarray_const = const_cast<const rocblas_float_complex**>(complexYarray);
  hipComplex** complexAarray = 0;
  const hipComplex** const complexAarray_const = const_cast<const hipComplex**>(complexAarray);
  hipComplex** complexBarray = 0;
  const hipComplex** const complexBarray_const = const_cast<const hipComplex**>(complexBarray);
  hipComplex** complexCarray = 0;
  hipComplex** complexTauarray = 0;
  hipComplex** complexXarray = 0;
  const hipComplex** const complexXarray_const = const_cast<const hipComplex**>(complexXarray);
  hipComplex** complexYarray = 0;
  const hipComplex** const complexYarray_const = const_cast<const hipComplex**>(complexYarray);

  // CHECK: rocblas_double_complex** dcomplexAarray = 0;
  // CHECK: const rocblas_double_complex** const dcomplexAarray_const = const_cast<const rocblas_double_complex**>(dcomplexAarray);
  // CHECK-NEXT: rocblas_double_complex** dcomplexBarray = 0;
  // CHECK: const rocblas_double_complex** const dcomplexBarray_const = const_cast<const rocblas_double_complex**>(dcomplexBarray);
  // CHECK-NEXT: rocblas_double_complex** dcomplexCarray = 0;
  // CHECK-NEXT: rocblas_double_complex** dcomplexTauarray = 0;
  // CHECK-NEXT: rocblas_double_complex** dcomplexXarray = 0;
  // CHECK: const rocblas_double_complex** const dcomplexXarray_const = const_cast<const rocblas_double_complex**>(dcomplexXarray);
  // CHECK-NEXT: rocblas_double_complex** dcomplexYarray = 0;
  // CHECK: const rocblas_double_complex** const dcomplexYarray_const = const_cast<const rocblas_double_complex**>(dcomplexYarray);
  hipDoubleComplex** dcomplexAarray = 0;
  const hipDoubleComplex** const dcomplexAarray_const = const_cast<const hipDoubleComplex**>(dcomplexAarray);
  hipDoubleComplex** dcomplexBarray = 0;
  const hipDoubleComplex** const dcomplexBarray_const = const_cast<const hipDoubleComplex**>(dcomplexBarray);
  hipDoubleComplex** dcomplexCarray = 0;
  hipDoubleComplex** dcomplexTauarray = 0;
  hipDoubleComplex** dcomplexXarray = 0;
  const hipDoubleComplex** const dcomplexXarray_const = const_cast<const hipDoubleComplex**>(dcomplexXarray);
  hipDoubleComplex** dcomplexYarray = 0;
  const hipDoubleComplex** const dcomplexYarray_const = const_cast<const hipDoubleComplex**>(dcomplexYarray);

  // TODO: #1281
  // NOTE: float CUBLASWINAPI cublasScnrm2(int n, const cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScnrm2_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scnrm2(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, float* result);
  // CHECK: blasStatus = rocblas_scnrm2(blasHandle, n, &complex, incx, &fresult);
  blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);

  // TODO: #1281
  // NOTE: double CUBLASWINAPI cublasDznrm2(int n, const cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDznrm2_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dznrm2(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, double* result);
  // CHECK: blasStatus = rocblas_dznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);

  // TODO: #1281
  // NOTE: float CUBLASWINAPI cublasSdot(int n, const float* x, int incx, const float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSdot_v2(cublasHandle_t handle, int n, const float* x, int incx, const float* y, int incy, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sdot(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, const float* y, rocblas_int incy, float* result);
  // CHECK: blasStatus = rocblas_sdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);

  // TODO: #1281
  // NOTE: double CUBLASWINAPI cublasDdot(int n, const double* x, int incx, const double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDdot_v2(cublasHandle_t handle, int n, const double* x, int incx, const double* y, int incy, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ddot(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, const double* y, rocblas_int incy, double* result);
  // CHECK: blasStatus = rocblas_ddot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);

  // TODO: #1281
  // NOTE: cuComplex CUBLASWINAPI cublasCdotu(int n, const cuComplex* x, int incx, const cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdotu_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cdotu(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* result);
  // CHECK: blasStatus = rocblas_cdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);

  // TODO: #1281
  // NOTE: cuComplex CUBLASWINAPI cublasCdotc(int n, const cuComplex* x, int incx, const cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdotc_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cdotc(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* result);
  // CHECK: blasStatus = rocblas_cdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);

  // TODO: #1281
  // NOTE: cuDoubleComplex CUBLASWINAPI cublasZdotu(int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdotu_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdotu(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* result);
  // CHECK: blasStatus = rocblas_zdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);

  // TODO: #1281
  // NOTE: cuDoubleComplex CUBLASWINAPI cublasZdotc(int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdotc_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdotc(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* result);
  // CHECK: blasStatus = rocblas_zdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSscal(int n, float alpha, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSscal_v2(cublasHandle_t handle, int n, const float* alpha, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sscal(rocblas_handle handle, rocblas_int n, const float* alpha, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_sscal(blasHandle, n, &fy, &fx, incx);
  blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDscal(int n, double alpha, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDscal_v2(cublasHandle_t handle, int n, const double* alpha, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dscal(rocblas_handle handle, rocblas_int n, const double* alpha, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dscal(blasHandle, n, &dx, &dy, incx);
  blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCscal(int n, cuComplex alpha, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCscal_v2(cublasHandle_t handle, int n, const cuComplex* alpha, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cscal(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* alpha, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_cscal(blasHandle, n, &complexa, &complexx, incx);
  blasStatus = hipblasCscal(blasHandle, n, &complexa, &complexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCsscal(int n, float alpha, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsscal_v2(cublasHandle_t handle, int n, const float* alpha, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csscal(rocblas_handle handle, rocblas_int n, const float* alpha, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_csscal(blasHandle, n, &fx, &complexx, incx);
  blasStatus = hipblasCsscal(blasHandle, n, &fx, &complexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZscal(int n, cuDoubleComplex alpha, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZscal_v2(cublasHandle_t handle, int n, const cuDoubleComplex* alpha, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zscal(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* alpha, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_zscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  blasStatus = hipblasZscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZdscal(int n, double alpha, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdscal_v2(cublasHandle_t handle, int n, const double* alpha, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdscal(rocblas_handle handle, rocblas_int n, const double* alpha, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_zdscal(blasHandle, n, &dx, &dcomplexx, incx);
  blasStatus = hipblasZdscal(blasHandle, n, &dx, &dcomplexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSaxpy(int n, float alpha, const float* x, int incx, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSaxpy_v2(cublasHandle_t handle, int n, const float* alpha, const float* x, int incx, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_saxpy(rocblas_handle handle, rocblas_int n, const float* alpha, const float* x, rocblas_int incx, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_saxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDaxpy(int n, double alpha, const double* x, int incx, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDaxpy_v2(cublasHandle_t handle, int n, const double* alpha, const double* x, int incx, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_daxpy(rocblas_handle handle, rocblas_int n, const double* alpha, const double* x, rocblas_int incx, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_daxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCaxpy(int n, cuComplex alpha, const cuComplex* x, int incx, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCaxpy_v2(cublasHandle_t handle, int n, const cuComplex* alpha, const cuComplex* x, int incx, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_caxpy(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_caxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCaxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZaxpy(int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZaxpy_v2(cublasHandle_t handle, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zaxpy(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasScopy(int n, const float* x, int incx, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScopy_v2(cublasHandle_t handle, int n, const float* x, int incx, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scopy(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_scopy(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDcopy(int n, const double* x, int incx, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDcopy_v2(cublasHandle_t handle, int n, const double* x, int incx, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dcopy(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dcopy(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCcopy(int n, const cuComplex* x, int incx, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCcopy_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ccopy(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_ccopy(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCcopy(blasHandle, n, &complexx, incx, &complexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZcopy(int n, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZcopy_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zcopy(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSswap(int n, float* x, int incx, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSswap_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sswap(rocblas_handle handle, rocblas_int n, float* x, rocblas_int incx, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_sswap(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDswap(int n, double* x, int incx, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDswap_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dswap(rocblas_handle handle, rocblas_int n, double* x, rocblas_int incx, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dswap(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCswap(int n, cuComplex* x, int incx, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCswap_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cswap(rocblas_handle handle, rocblas_int n, rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_cswap(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCswap(blasHandle, n, &complexx, incx, &complexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZswap(int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZswap_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zswap(rocblas_handle handle, rocblas_int n, rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);

  // TODO: #1281
  // NOTE: int CUBLASWINAPI cublasIsamax(int n, const float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIsamax_v2(cublasHandle_t handle, int n, const float* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_isamax(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_isamax(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);

  // TODO: #1281
  // NOTE: int CUBLASWINAPI cublasIdamax(int n, const double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIdamax_v2(cublasHandle_t handle, int n, const double* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_idamax(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_idamax(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);

  // TODO: #1281
  // NOTE: int CUBLASWINAPI cublasIcamax(int n, const cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIcamax_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_icamax(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_icamax(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamax(blasHandle, n, &complexx, incx, &res);

  // TODO: #1281
  // NOTE: int CUBLASWINAPI cublasIzamax(int n, const cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIzamax_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_izamax(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_izamax(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamax(blasHandle, n, &dcomplexx, incx, &res);

  // TODO: #1281
  // NOTE: int CUBLASWINAPI cublasIsamin(int n, const float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIsamin_v2(cublasHandle_t handle, int n, const float* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_isamin(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_isamin(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);

  // TODO: #1281
  // NOTE: int CUBLASWINAPI cublasIdamin(int n, const double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIdamin_v2(cublasHandle_t handle, int n, const double* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_idamin(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_idamin(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);

  // TODO: #1281
  // NOTE: int CUBLASWINAPI cublasIcamin(int n, const cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIcamin_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_icamin(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_icamin(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamin(blasHandle, n, &complexx, incx, &res);

  // TODO: #1281
  // NOTE: int CUBLASWINAPI cublasIzamin(int n, const cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIzamin_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_izamin(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_izamin(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamin(blasHandle, n, &dcomplexx, incx, &res);

  // TODO: #1281
  // NOTE: float CUBLASWINAPI cublasSasum(int n, const float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSasum_v2(cublasHandle_t handle, int n, const float* x, int incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sasum(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, float* result);
  // CHECK: blasStatus = rocblas_sasum(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);

  // TODO: #1281
  // NOTE: double CUBLASWINAPI cublasDasum(int n, const double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDasum_v2(cublasHandle_t handle, int n, const double* x, int incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dasum(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, double* result);
  // CHECK: blasStatus = rocblas_dasum(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);

  // TODO: #1281
  // NOTE: float CUBLASWINAPI cublasScasum(int n, const cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScasum_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scasum(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, float* result);
  // CHECK: blasStatus = rocblas_scasum(blasHandle, n, &complexx, incx, &fresult);
  blasStatus = hipblasScasum(blasHandle, n, &complexx, incx, &fresult);

  // TODO: #1281
  // NOTE: double CUBLASWINAPI cublasDzasum(int n, const cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDzasum_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dzasum(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, double* result);
  // CHECK: blasStatus = rocblas_dzasum(blasHandle, n, &dcomplexx, incx, &dresult);
  blasStatus = hipblasDzasum(blasHandle, n, &dcomplexx, incx, &dresult);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSrot(int n, float* x, int incx, float* y, int incy, float sc, float ss); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrot_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* c, const float* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_srot(rocblas_handle handle, rocblas_int n, float* x, rocblas_int incx, float* y, rocblas_int incy, const float* c, const float* s);
  // CHECK: blasStatus = rocblas_srot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);
  blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDrot(int n, double* x, int incx, double* y, int incy, double sc, double ss); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrot_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* c, const double* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_drot(rocblas_handle handle, rocblas_int n, double* x, rocblas_int incx, double* y, rocblas_int incy, const double* c, const double* s);
  // CHECK: blasStatus = rocblas_drot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);
  blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCrot(int n, cuComplex* x, int incx, cuComplex* y, int incy, float c, cuComplex s); is not supported by HIP
  // CUDA: CUBLASAPI CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCrot_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy, const float* c, const cuComplex* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_crot(rocblas_handle handle, rocblas_int n, rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* y, rocblas_int incy, const float* c, const rocblas_float_complex* s);
  // CHECK: blasStatus = rocblas_crot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);
  blasStatus = hipblasCrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCsrot(int n, cuComplex* x, int incx, cuComplex* y, int incy, float c, float s); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsrot_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy, const float* c, const float* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csrot(rocblas_handle handle, rocblas_int n, rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* y, rocblas_int incy, const float* c, const float* s);
  // CHECK: blasStatus = rocblas_csrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);
  blasStatus = hipblasCsrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZrot(int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, double sc, cuDoubleComplex cs); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZrot_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, const double* c, const cuDoubleComplex* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zrot(rocblas_handle handle, rocblas_int n, rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* y, rocblas_int incy, const double* c, const rocblas_double_complex* s);
  // CHECK: blasStatus = rocblas_zrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);
  blasStatus = hipblasZrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZdrot(int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, double c, double s); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdrot_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, const double* c, const double* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdrot(rocblas_handle handle, rocblas_int n, rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* y, rocblas_int incy, const double* c, const double* s);
  // CHECK: blasStatus = rocblas_zdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);
  blasStatus = hipblasZdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);

  // NOTE: void CUBLASWINAPI cublasSrotg(float* sa, float* sb, float* sc, float* ss); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotg_v2(cublasHandle_t handle, float* a, float* b, float* c, float* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_srotg(rocblas_handle handle, float* a, float* b, float* c, float* s);
  // CHECK: blasStatus = rocblas_srotg(blasHandle, &fa, &fb, &fc, &fs);
  blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);

  // NOTE: void CUBLASWINAPI cublasDrotg(double* sa, double* sb, double* sc, double* ss); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotg_v2(cublasHandle_t handle, double* a, double* b, double* c, double* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_drotg(rocblas_handle handle, double* a, double* b, double* c, double* s);
  // CHECK: blasStatus = rocblas_drotg(blasHandle, &da, &db, &dc, &ds);
  blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);

  // NOTE: void CUBLASWINAPI cublasCrotg(cuComplex* ca, cuComplex cb, float* sc, cuComplex* cs); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCrotg_v2(cublasHandle_t handle, cuComplex* a, cuComplex* b, float* c, cuComplex* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_crotg(rocblas_handle handle, rocblas_float_complex* a, rocblas_float_complex* b, float* c, rocblas_float_complex* s);
  // CHECK: blasStatus = rocblas_crotg(blasHandle, &complexa, &complexb, &fc, &complexs);
  blasStatus = hipblasCrotg(blasHandle, &complexa, &complexb, &fc, &complexs);

  // NOTE: void CUBLASWINAPI cublasZrotg(cuDoubleComplex* ca, cuDoubleComplex cb, double* sc, cuDoubleComplex* cs); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZrotg_v2(cublasHandle_t handle, cuDoubleComplex* a, cuDoubleComplex* b, double* c, cuDoubleComplex* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zrotg(rocblas_handle handle, rocblas_double_complex* a, rocblas_double_complex* b, double* c, rocblas_double_complex* s);
  // CHECK: blasStatus = rocblas_zrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);
  blasStatus = hipblasZrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSrotm(int n, float* x, int incx, float* y, int incy, const float* sparam); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotm_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* param);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_srotm(rocblas_handle handle, rocblas_int n, float* x, rocblas_int incx, float* y, rocblas_int incy, const float* param);
  // CHECK: blasStatus = rocblas_srotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDrotm(int n, double* x, int incx, double* y, int incy, const double* sparam); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotm_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* param);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_drotm(rocblas_handle handle, rocblas_int n, double* x, rocblas_int incx, double* y, rocblas_int incy, const double* param);
  // CHECK: blasStatus = rocblas_drotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);

  // NOTE: void CUBLASWINAPI cublasSrotmg(float* sd1, float* sd2, float* sx1, const float* sy1, float* sparam); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotmg_v2(cublasHandle_t handle, float* d1, float* d2, float* x1, const float* y1, float* param);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_srotmg(rocblas_handle handle, float* d1, float* d2, float* x1, const float* y1, float* param);
  // CHECK: blasStatus = rocblas_srotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);
  blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);

  // NOTE: void CUBLASWINAPI cublasDrotmg(double* sd1, double* sd2, double* sx1, const double* sy1, double* sparam); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotmg_v2(cublasHandle_t handle, double* d1, double* d2, double* x1, const double* y1, double* param);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_drotmg(rocblas_handle handle, double* d1, double* d2, double* x1, const double* y1, double* param);
  // CHECK: blasStatus = rocblas_drotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);
  blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSgemv(char trans, int m, int n, float alpha, const float* A, int lda, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sgemv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, const float* alpha, const float* A, rocblas_int lda, const float* x, rocblas_int incx, const float* beta, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_sgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDgemv(char trans, int m, int n, double alpha, const double* A, int lda, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dgemv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, const double* alpha, const double* A, rocblas_int lda, const double* x, rocblas_int incx, const double* beta, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDgemv(char trans, int m, int n, double alpha, const double* A, int lda, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgemv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* beta, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_cgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZgemv(char trans, int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgemv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* beta, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSgbmv(char trans, int m, int n, int kl, int ku, float alpha, const float* A, int lda, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sgbmv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, rocblas_int kl, rocblas_int ku, const float* alpha, const float* A, rocblas_int lda, const float* x, rocblas_int incx, const float* beta, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_sgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDgbmv(char trans, int m, int n, int kl, int ku, double alpha, const double* A, int lda, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dgbmv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, rocblas_int kl, rocblas_int ku, const double* alpha, const double* A, rocblas_int lda, const double* x, rocblas_int incx, const double* beta, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCgbmv(char trans, int m, int n, int kl, int ku, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex beta, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgbmv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, rocblas_int kl, rocblas_int ku, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* beta, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_cgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZgbmv(char trans, int m, int n, int kl, int ku, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgbmv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, rocblas_int kl, rocblas_int ku, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* beta, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasStrmv(char uplo, char trans, char diag, int n, const float* A, int lda, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* A, int lda, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_strmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const float* A, rocblas_int lda, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_strmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDtrmv(char uplo, char trans, char diag, int n, const double* A, int lda, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* A, int lda, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtrmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const double* A, rocblas_int lda, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCtrmv(char uplo, char trans, char diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctrmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const rocblas_float_complex* A, rocblas_int lda, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ctrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZtrmv(char uplo, char trans, char diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztrmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const rocblas_double_complex* A, rocblas_int lda, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ztrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasStbmv(char uplo, char trans, char diag, int n, int k, const float* A, int lda, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const float* A, int lda, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_stbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_diagonal diag, rocblas_int m, rocblas_int k, const float* A, rocblas_int lda, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_stbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDtbmv(char uplo, char trans, char diag, int n, int k, const double* A, int lda, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const double* A, int lda, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_diagonal diag, rocblas_int m, rocblas_int k, const double* A, rocblas_int lda, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCtbmv(char uplo, char trans, char diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_diagonal diag, rocblas_int m, rocblas_int k, const rocblas_float_complex* A, rocblas_int lda, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ctbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZtbmv(char uplo, char trans, char diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_diagonal diag, rocblas_int m, rocblas_int k, const rocblas_double_complex* A, rocblas_int lda, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ztbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasStpmv(char uplo, char trans, char diag, int n, const float* AP, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* AP, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_stpmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const float* A, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_stpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDtpmv(char uplo, char trans, char diag, int n, const double* AP, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* AP, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtpmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const double* A, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCtpmv(char uplo, char trans, char diag, int n, const cuComplex* AP, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* AP, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctpmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const rocblas_float_complex* A, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ctpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZtpmv(char uplo, char trans, char diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztpmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const rocblas_double_complex* A, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ztpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasStrsv(char uplo, char trans, char diag, int n, const float* A, int lda, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* A, int lda, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_strsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const float* A, rocblas_int lda, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_strsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDtrsv(char uplo, char trans, char diag, int n, const double* A, int lda, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* A, int lda, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtrsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const double* A, rocblas_int lda, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCtrsv(char uplo, char trans, char diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctrsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const rocblas_float_complex* A, rocblas_int lda, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ctrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZtrsv(char uplo, char trans, char diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztrsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const rocblas_double_complex* A, rocblas_int lda, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ztrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasStpsv(char uplo, char trans, char diag, int n, const float* AP, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* AP, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_stpsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, const float* AP, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_stpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDtpsv(char uplo, char trans, char diag, int n, const double* AP, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* AP, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtpsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, const double* AP, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCtpsv(char uplo, char trans, char diag, int n, const cuComplex* AP, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* AP, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctpsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, const rocblas_float_complex* AP, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ctpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZtpsv(char uplo, char trans, char diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztpsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, const rocblas_double_complex* AP, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ztpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasStbsv(char uplo, char trans, char diag, int n, int k, const float* A, int lda, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const float* A, int lda, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_stbsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, rocblas_int k, const float* A, rocblas_int lda, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_stbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDtbsv(char uplo, char trans, char diag, int n, int k, const double* A, int lda, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const double* A, int lda, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtbsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, rocblas_int k, const double* A, rocblas_int lda, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCtbsv(char uplo, char trans, char diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctbsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, rocblas_int k, const rocblas_float_complex* A, rocblas_int lda, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ctbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZtbsv(char uplo, char trans, char diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztbsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, rocblas_int k, const rocblas_double_complex* A, rocblas_int lda, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ztbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSsymv(char uplo, int n, float alpha, const float* A, int lda, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssymv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const float* A, rocblas_int lda, const float* x, rocblas_int incx, const float* beta, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_ssymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDsymv(char uplo, int n, double alpha, const double* A, int lda, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsymv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const double* A, rocblas_int lda, const double* x, rocblas_int incx, const double* beta, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csymv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* beta, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_csymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCsymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zsymv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* beta, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasChemv(char uplo, int n, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex beta, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChemv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_chemv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* beta, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_chemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZhemv(char uplo, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhemv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zhemv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* beta, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSsbmv(char uplo, int n, int k, float alpha, const float* A, int lda, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, rocblas_int k, const float* alpha, const float* A, rocblas_int lda, const float* x, rocblas_int incx, const float* beta, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_ssbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDsbmv(char uplo, int n, int k, double alpha, const double* A, int lda, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, rocblas_int k, const double* alpha, const double* A, rocblas_int lda, const double* x, rocblas_int incx, const double* beta, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasChbmv(char uplo, int n, int k, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex beta, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_chbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* beta, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_chbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZhbmv(char uplo, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zhbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* beta, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSspmv(char uplo, int n, float alpha, const float* AP, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* AP, const float* x, int incx, const float* beta, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sspmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const float* A, const float* x, rocblas_int incx, const float* beta, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_sspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDspmv(char uplo, int n, double alpha, const double* AP, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* AP, const double* x, int incx, const double* beta, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dspmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const double* A, const double* x, rocblas_int incx, const double* beta, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasChpmv(char uplo, int n, cuComplex alpha, const cuComplex* AP, const cuComplex* x, int incx, cuComplex beta, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* AP, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_chpmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* AP, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* beta, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_chpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZhpmv(char uplo, int n, cuDoubleComplex alpha, const cuDoubleComplex* AP, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* AP, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zhpmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* AP, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* beta, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSger(int m, int n, float alpha, const float* x, int incx, const float* y, int incy, float* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSger_v2(cublasHandle_t handle, int m, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sger(rocblas_handle handle, rocblas_int m, rocblas_int n, const float* alpha, const float* x, rocblas_int incx, const float* y, rocblas_int incy, float* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_sger(blasHandle, m, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  blasStatus = hipblasSger(blasHandle, m, n, &fa, &fx, incx, &fy, incy, &fA, lda);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDger(int m, int n, double alpha, const double* x, int incx, const double* y, int incy, double* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDger_v2(cublasHandle_t handle, int m, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dger(rocblas_handle handle, rocblas_int m, rocblas_int n, const double* alpha, const double* x, rocblas_int incx, const double* y, rocblas_int incy, double* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_dger(blasHandle, m, n, &da, &dx, incx, &dy, incy, &dA, lda);
  blasStatus = hipblasDger(blasHandle, m, n, &da, &dx, incx, &dy, incy, &dA, lda);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCgeru(int m, int n, cuComplex alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgeru_v2(cublasHandle_t handle, int m, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgeru(rocblas_handle handle, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_cgeru(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCgeru(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCgerc(int m, int n, cuComplex alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgerc_v2(cublasHandle_t handle, int m, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgerc(rocblas_handle handle, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_cgerc(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCgerc(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZgeru(int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgeru_v2(cublasHandle_t handle, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgeru(rocblas_handle handle, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_zgeru(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZgeru(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZgerc(int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgerc_v2(cublasHandle_t handle, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgerc(rocblas_handle handle, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_zgerc(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZgerc(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSsyr(char uplo, int n, float alpha, const float* x, int incx, float* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssyr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const float* x, rocblas_int incx, float* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_ssyr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA, lda);
  blasStatus = hipblasSsyr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA, lda);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDsyr(char uplo, int n, double alpha, const double* x, int incx, double* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsyr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const double* x, rocblas_int incx, double* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_dsyr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA, lda);
  blasStatus = hipblasDsyr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, cuComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csyr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_csyr(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexA, lda);
  blasStatus = hipblasCsyr(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zsyr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_zsyr(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexA, lda);
  blasStatus = hipblasZsyr(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexA, lda);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCher(char uplo, int n, float alpha, const cuComplex* x, int incx, cuComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const cuComplex* x, int incx, cuComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cher(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_cher(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA, lda);
  blasStatus = hipblasCher(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA, lda);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZher(char uplo, int n, double alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zher(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_zher(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA, lda);
  blasStatus = hipblasZher(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA, lda);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSspr(char uplo, int n, float alpha, const float* x, int incx, float* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sspr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const float* x, rocblas_int incx, float* AP);
  // CHECK: blasStatus = rocblas_sspr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA);
  blasStatus = hipblasSspr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDspr(char uplo, int n, double alpha, const double* x, int incx, double* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, double* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dspr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const double* x, rocblas_int incx, double* AP);
  // CHECK: blasStatus = rocblas_dspr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA);
  blasStatus = hipblasDspr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasChpr(char uplo, int n, float alpha, const cuComplex* x, int incx, cuComplex* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const cuComplex* x, int incx, cuComplex* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_chpr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* AP);
  // CHECK: blasStatus = rocblas_chpr(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA);
  blasStatus = hipblasChpr(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZhpr(char uplo, int n, double alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zhpr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* AP);
  // CHECK: blasStatus = rocblas_zhpr(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA);
  blasStatus = hipblasZhpr(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSsyr2(char uplo, int n, float alpha, const float* x, int incx, const float* y, int incy, float* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssyr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const float* x, rocblas_int incx, const float* y, rocblas_int incy, float* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_ssyr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  blasStatus = hipblasSsyr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA, lda);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDsyr2(char uplo, int n, double alpha, const double* x, int incx, const double* y, int incy, double* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsyr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const double* x, rocblas_int incx, const double* y, rocblas_int incy, double* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_dsyr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA, lda);
  blasStatus = hipblasDsyr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csyr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_csyr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCsyr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zsyr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_zsyr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZsyr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCher2(char uplo, int n, cuComplex alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cher2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_cher2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCher2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZher2(char uplo, int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zher2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_zher2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZher2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSspr2(char uplo, int n, float alpha, const float* x, int incx, const float* y, int incy, float* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sspr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const float* x, rocblas_int incx, const float* y, rocblas_int incy, float* AP);
  // CHECK: blasStatus = rocblas_sspr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA);
  blasStatus = hipblasSspr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDspr2(char uplo, int n, double alpha, const double* x, int incx, const double* y, int incy, double* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dspr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const double* x, rocblas_int incx, const double* y, rocblas_int incy, double* AP);
  // CHECK: blasStatus = rocblas_dspr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA);
  blasStatus = hipblasDspr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasChpr2(char uplo, int n, cuComplex alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_chpr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* AP);
  // CHECK: blasStatus = rocblas_chpr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA);
  blasStatus = hipblasChpr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZhpr2(char uplo, int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zhpr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* AP);
  // CHECK: blasStatus = rocblas_zhpr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA);
  blasStatus = hipblasZhpr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA);

  // CHECK rocblas_operation transa, transb;
  hipblasOperation_t transa, transb;

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSgemm(char transa, char transb, int m, int n, int k, float alpha, const float* A, int lda, const float* B, int ldb, float beta, float* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sgemm(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const float* alpha, const float* A, rocblas_int lda, const float* B, rocblas_int ldb, const float* beta, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_sgemm(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSgemm(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDgemm(char transa, char transb, int m, int n, int k, double alpha, const double* A, int lda, const double* B, int ldb, double beta, double* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dgemm(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const double* alpha, const double* A, rocblas_int lda, const double* B, rocblas_int ldb, const double* beta, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_dgemm(blasHandle, transa, transb, m, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDgemm(blasHandle, transa, transb, m, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCgemm(char transa, char transb, int m, int n, int k, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgemm(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, const rocblas_float_complex* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_cgemm(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCgemm(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZgemm(char transa, char transb, int m, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgemm(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, const rocblas_double_complex* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zgemm(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZgemm(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* const Aarray[], int lda, const float* const Barray[], int ldb, const float* beta, float* const Carray[], int ldc, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sgemm_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const float* alpha, const float* const A[], rocblas_int lda, const float* const B[], rocblas_int ldb, const float* beta, float* const C[], rocblas_int ldc, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_sgemm_batched(blasHandle, transa, transb, m, n, k, &fa, fAarray_const, lda, fBarray_const, ldb, &fb, fCarray, ldc, batchCount);
  blasStatus = hipblasSgemmBatched(blasHandle, transa, transb, m, n, k, &fa, fAarray_const, lda, fBarray_const, ldb, &fb, fCarray, ldc, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* const Aarray[], int lda, const double* const Barray[], int ldb, const double* beta, double* const Carray[], int ldc, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dgemm_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const double* alpha, const double* const A[], rocblas_int lda, const double* const B[], rocblas_int ldb, const double* beta, double* const C[], rocblas_int ldc, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_dgemm_batched(blasHandle, transa, transb, m, n, k, &da, dAarray_const, lda, dBarray_const, ldb, &db, dCarray, ldc, batchCount);
  blasStatus = hipblasDgemmBatched(blasHandle, transa, transb, m, n, k, &da, dAarray_const, lda, dBarray_const, ldb, &db, dCarray, ldc, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* const Aarray[], int lda, const cuComplex* const Barray[], int ldb, const cuComplex* beta, cuComplex* const Carray[], int ldc, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgemm_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* const A[], rocblas_int lda, const rocblas_float_complex* const B[], rocblas_int ldb, const rocblas_float_complex* beta, rocblas_float_complex* const C[], rocblas_int ldc, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_cgemm_batched(blasHandle, transa, transb, m, n, k, &complexa, complexAarray_const, lda, complexBarray_const, ldb, &complexb, complexCarray, ldc, batchCount);
  blasStatus = hipblasCgemmBatched(blasHandle, transa, transb, m, n, k, &complexa, complexAarray_const, lda, complexBarray_const, ldb, &complexb, complexCarray, ldc, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* const Aarray[], int lda, const cuDoubleComplex* const Barray[], int ldb, const cuDoubleComplex* beta, cuDoubleComplex* const Carray[], int ldc, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgemm_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* const A[], rocblas_int lda, const rocblas_double_complex* const B[], rocblas_int ldb, const rocblas_double_complex* beta, rocblas_double_complex* const C[], rocblas_int ldc, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_zgemm_batched(blasHandle, transa, transb, m, n, k, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray_const, ldb, &dcomplexb, dcomplexCarray, ldc, batchCount);
  blasStatus = hipblasZgemmBatched(blasHandle, transa, transb, m, n, k, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray_const, ldb, &dcomplexb, dcomplexCarray, ldc, batchCount);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSsyrk(char uplo, char trans, int n, int k, float alpha, const float* A, int lda, float beta, float* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* beta, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssyrk(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_int n, rocblas_int k, const float* alpha, const float* A, rocblas_int lda, const float* beta, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_ssyrk(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, &fC, ldc);
  blasStatus = hipblasSsyrk(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, &fC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDsyrk(char uplo, char trans, int n, int k, double alpha, const double* A, int lda, double beta, double* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* beta, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsyrk(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_int n, rocblas_int k, const double* alpha, const double* A, rocblas_int lda, const double* beta, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_dsyrk(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, &dC, ldc);
  blasStatus = hipblasDsyrk(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, &dC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCsyrk(char uplo, char trans, int n, int k, cuComplex alpha, const cuComplex* A, int lda, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csyrk(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_csyrk(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyrk(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, &complexC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZsyrk(char uplo, char trans, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zsyrk(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zsyrk(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyrk(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCherk(char uplo, char trans, int n, int k, float alpha, const cuComplex* A, int lda, float beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCherk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const cuComplex* A, int lda, const float* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cherk(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_int n, rocblas_int k, const float* alpha, const rocblas_float_complex* A, rocblas_int lda, const float* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_cherk(blasHandle, blasFillMode, transa, n, k, &fa, &complexA, lda, &fb, &complexC, ldc);
  blasStatus = hipblasCherk(blasHandle, blasFillMode, transa, n, k, &fa, &complexA, lda, &fb, &complexC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZherk(char uplo, char trans, int n, int k, double alpha, const cuDoubleComplex* A, int lda, double beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZherk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const cuDoubleComplex* A, int lda, const double* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zherk(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_int n, rocblas_int k, const double* alpha, const rocblas_double_complex* A, rocblas_int lda, const double* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zherk(blasHandle, blasFillMode, transa, n, k, &da, &dcomplexA, lda, &db, &dcomplexC, ldc);
  blasStatus = hipblasZherk(blasHandle, blasFillMode, transa, n, k, &da, &dcomplexA, lda, &db, &dcomplexC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSsyr2k(char uplo, char trans, int n, int k, float alpha, const float* A, int lda, const float* B, int ldb, float beta, float* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssyr2k(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const float* alpha, const float* A, rocblas_int lda, const float* B, rocblas_int ldb, const float* beta, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_ssyr2k(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsyr2k(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, ldb, &fb, &fC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDsyr2k(char uplo, char trans, int n, int k, double alpha, const double* A, int lda, const double* B, int ldb, double beta, double* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsyr2k(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const double* alpha, const double* A, rocblas_int lda, const double* B, rocblas_int ldb, const double* beta, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_dsyr2k(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsyr2k(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, ldb, &db, &dC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCsyr2k(char uplo, char trans, int n, int k, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csyr2k(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, const rocblas_float_complex* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_csyr2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyr2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &complexb, &complexC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZsyr2k(char uplo, char trans, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zsyr2k(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, const rocblas_double_complex* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zsyr2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyr2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &dcomplexb, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssyrkx(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const float* alpha, const float* A, rocblas_int lda, const float* B, rocblas_int ldb, const float* beta, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_ssyrkx(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsyrkx(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsyrkx(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const double* alpha, const double* A, rocblas_int lda, const double* B, rocblas_int ldb, const double* beta, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_dsyrkx(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsyrkx(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csyrkx(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, const rocblas_float_complex* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_csyrkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyrkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zsyrkx(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, const rocblas_double_complex* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zsyrkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyrkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCher2k(char uplo, char trans, int n, int k, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, float beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const float* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cher2k(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, const float* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_cher2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &fb, &complexC, ldc);
  blasStatus = hipblasCher2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &fb, &complexC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZher2k(char uplo, char trans, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, double beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const double* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zher2k(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, const double* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zher2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &db, &dcomplexC, ldc);
  blasStatus = hipblasZher2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &db, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCherkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const float* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cherkx(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, const float* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_cherkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &fb, &complexC, ldc);
  blasStatus = hipblasCherkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &fb, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZherkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const double* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zherkx(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, const double* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zherkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &db, &dcomplexC, ldc);
  blasStatus = hipblasZherkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &db, &dcomplexC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasSsymm(char side, char uplo, int m, int n, float alpha, const float* A, int lda, const float* B, int ldb, float beta, float* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssymm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_int m, rocblas_int n, const float* alpha, const float* A, rocblas_int lda, const float* B, rocblas_int ldb, const float* beta, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_ssymm(blasHandle, blasSideMode, blasFillMode, m, n, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsymm(blasHandle, blasSideMode, blasFillMode, m, n, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDsymm(char side, char uplo, int m, int n, double alpha, const double* A, int lda, const double* B, int ldb, double beta, double* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsymm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_int m, rocblas_int n, const double* alpha, const double* A, rocblas_int lda, const double* B, rocblas_int ldb, const double* beta, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_dsymm(blasHandle, blasSideMode, blasFillMode, m, n, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsymm(blasHandle, blasSideMode, blasFillMode, m, n, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCsymm(char side, char uplo, int m, int n, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csymm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, const rocblas_float_complex* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_csymm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsymm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZsymm(char side, char uplo, int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zsymm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, const rocblas_double_complex* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zsymm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsymm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasChemm(char side, char uplo, int m, int n, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChemm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_chemm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, const rocblas_float_complex* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_chemm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasChemm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZhemm(char side, char uplo, int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhemm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zhemm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, const rocblas_double_complex* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zhemm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZhemm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasStrsm(char side, char uplo, char transa, char diag, int m, int n, float alpha, const float* A, int lda, float* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const float* alpha, const float* A, int lda, float* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_strsm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const float* alpha, const float* A, rocblas_int lda, float* B, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_strsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb);
  blasStatus = hipblasStrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDtrsm(char side, char uplo, char transa, char diag, int m, int n, double alpha, const double* A, int lda, double* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const double* alpha, const double* A, int lda, double* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtrsm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const double* alpha, const double* A, rocblas_int lda, double* B, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_dtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb);
  blasStatus = hipblasDtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCtrsm(char side, char uplo, char transa, char diag, int m, int n, cuComplex alpha, const cuComplex* A, int lda, cuComplex* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, cuComplex* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctrsm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, rocblas_float_complex* B, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_ctrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb);
  blasStatus = hipblasCtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZtrsm(char side, char uplo, char transa, char diag, int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, cuDoubleComplex* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, cuDoubleComplex* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztrsm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, rocblas_double_complex* B, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_ztrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb);
  blasStatus = hipblasZtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasStrmm(char side, char uplo, char transa, char diag, int m, int n, float alpha, const float* A, int lda, float* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const float* alpha, const float* A, int lda, const float* B, int ldb, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_strmm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const float* alpha, const float* A, rocblas_int lda, const float* B, rocblas_int ldb, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_strmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb, &fC, ldc);
  blasStatus = hipblasStrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb, &fC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasDtrmm(char side, char uplo, char transa, char diag, int m, int n, double alpha, const double* A, int lda, double* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const double* alpha, const double* A, int lda, const double* B, int ldb, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtrmm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const double* alpha, const double* A, rocblas_int lda, const double* B, rocblas_int ldb, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_dtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb, &dC, ldc);
  blasStatus = hipblasDtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb, &dC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasCtrmm(char side, char uplo, char transa, char diag, int m, int n, cuComplex alpha, const cuComplex* A, int lda, cuComplex* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctrmm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_ctrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexC, ldc);
  blasStatus = hipblasCtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexC, ldc);

  // TODO: #1281
  // NOTE: void CUBLASWINAPI cublasZtrmm(char side, char uplo, char transa, char diag, int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, cuDoubleComplex* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztrmm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_ztrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexC, ldc);
  blasStatus = hipblasZtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const float* alpha, const float* A, int lda, const float* beta, const float* B, int ldb, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sgeam(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, const float* alpha, const float* A, rocblas_int lda, const float* beta, const float* B, rocblas_int ldb, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_sgeam(blasHandle, transa, transb, m, n, &fa, &fA, lda, &fb, &fB, ldb, &fC, ldc);
  blasStatus = hipblasSgeam(blasHandle, transa, transb, m, n, &fa, &fA, lda, &fb, &fB, ldb, &fC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const double* alpha, const double* A, int lda, const double* beta, const double* B, int ldb, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dgeam(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, const double* alpha, const double* A, rocblas_int lda, const double* beta, const double* B, rocblas_int ldb, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_dgeam(blasHandle, transa, transb, m, n, &da, &dA, lda, &db, &dB, ldb, &dC, ldc);
  blasStatus = hipblasDgeam(blasHandle, transa, transb, m, n, &da, &dA, lda, &db, &dB, ldb, &dC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* beta, const cuComplex* B, int ldb, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgeam(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* beta, const rocblas_float_complex* B, rocblas_int ldb, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_cgeam(blasHandle, transa, transb, m, n, &complexa, &complexA, lda, &complexb, &complexB, ldb, &complexC, ldc);
  blasStatus = hipblasCgeam(blasHandle, transa, transb, m, n, &complexa, &complexA, lda, &complexb, &complexB, ldb, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* beta, const cuDoubleComplex* B, int ldb, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgeam(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* beta, const rocblas_double_complex* B, rocblas_int ldb, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zgeam(blasHandle, transa, transb, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexB, ldb, &dcomplexC, ldc);
  blasStatus = hipblasZgeam(blasHandle, transa, transb, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexB, ldb, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const float* alpha, const float* const A[], int lda, float* const B[], int ldb, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_strsm_batched(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const float* alpha, const float* const A[], rocblas_int lda, float* const B[], rocblas_int ldb, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_strsm_batched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, fAarray_const, lda, fBarray, ldb, batchCount);
  blasStatus = hipblasStrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, fAarray_const, lda, fBarray, ldb, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const double* alpha, const double* const A[], int lda, double* const B[], int ldb, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtrsm_batched(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const double* alpha, const double* const A[], rocblas_int lda, double* const B[], rocblas_int ldb, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_dtrsm_batched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, dAarray_const, lda, dBarray, ldb, batchCount);
  blasStatus = hipblasDtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, dAarray_const, lda, dBarray, ldb, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuComplex* alpha, const cuComplex* const A[], int lda, cuComplex* const B[], int ldb, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctrsm_batched(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* const A[], rocblas_int lda, rocblas_float_complex* const B[], rocblas_int ldb, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_ctrsm_batched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, complexAarray_const, lda, complexBarray, ldb, batchCount);
  blasStatus = hipblasCtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, complexAarray_const, lda, complexBarray, ldb, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* const A[], int lda, cuDoubleComplex* const B[], int ldb, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztrsm_batched(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* const A[], rocblas_int lda, rocblas_double_complex* const B[], rocblas_int ldb, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_ztrsm_batched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray, ldb, batchCount);
  blasStatus = hipblasZtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray, ldb, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const float* A, int lda, const float* x, int incx, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sdgmm(rocblas_handle handle, rocblas_side side, rocblas_int m, rocblas_int n, const float* A, rocblas_int lda, const float* x, rocblas_int incx, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_sdgmm(blasHandle, blasSideMode, m, n, &fa, lda, &fx, incx, &fC, ldc);
  blasStatus = hipblasSdgmm(blasHandle, blasSideMode, m, n, &fa, lda, &fx, incx, &fC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const double* A, int lda, const double* x, int incx, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ddgmm(rocblas_handle handle, rocblas_side side, rocblas_int m, rocblas_int n, const double* A, rocblas_int lda, const double* x, rocblas_int incx, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_ddgmm(blasHandle, blasSideMode, m, n, &da, lda, &dx, incx, &dC, ldc);
  blasStatus = hipblasDdgmm(blasHandle, blasSideMode, m, n, &da, lda, &dx, incx, &dC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cdgmm(rocblas_handle handle, rocblas_side side, rocblas_int m, rocblas_int n, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_cdgmm(blasHandle, blasSideMode, m, n, &complexa, lda, &complexx, incx, &complexC, ldc);
  blasStatus = hipblasCdgmm(blasHandle, blasSideMode, m, n, &complexa, lda, &complexx, incx, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdgmm(rocblas_handle handle, rocblas_side side, rocblas_int m, rocblas_int n, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zdgmm(blasHandle, blasSideMode, m, n, &dcomplexa, lda, &dcomplexx, incx, &dcomplexC, ldc);
  blasStatus = hipblasZdgmm(blasHandle, blasSideMode, m, n, &dcomplexa, lda, &dcomplexx, incx, &dcomplexC, ldc);

  long long int strideA = 0;
  long long int strideB = 0;
  long long int strideC = 0;
  long long int stridex = 0;
  long long int stridey = 0;

#if CUDA_VERSION >= 7050
  // CHECK: rocblas_half* ha = 0;
  __half* ha = 0;
  // CHECK: rocblas_half* hA = 0;
  __half* hA = 0;
  // CHECK: rocblas_half* hb = 0;
  __half* hb = 0;
  // CHECK: rocblas_half* hB = 0;
  __half* hB = 0;
  // CHECK: rocblas_half* hc = 0;
  __half* hc = 0;
  // CHECK: rocblas_half* hC = 0;
  __half* hC = 0;
  // CHECK: rocblas_half* hx = 0;
  __half* hx = 0;
  // CHECK: rocblas_half* hy = 0;
  __half* hy = 0;

  // CHECK: rocblas_half** hAarray = 0;
  __half** hAarray = 0;
  // CHECK: const rocblas_half** const hAarray_const = const_cast<const rocblas_half**>(hAarray);
  const __half** const hAarray_const = const_cast<const __half**>(hAarray);
  // CHECK: rocblas_half** hBarray = 0;
  __half** hBarray = 0;
  // CHECK: const rocblas_half** const hBarray_const = const_cast<const rocblas_half**>(hBarray);
  const __half** const hBarray_const = const_cast<const __half**>(hBarray);
  // CHECK: rocblas_half** hCarray = 0;
  __half** hCarray = 0;
  // CHECK: const rocblas_half** const hCarray_const = const_cast<const rocblas_half**>(hCarray);
  const __half** const hCarray_const = const_cast<const __half**>(hCarray);
  // CHECK: rocblas_half** hxarray = 0;
  __half** hxarray = 0;
  // CHECK: const rocblas_half** const hxarray_const = const_cast<const rocblas_half**>(hxarray_const);
  const __half** const hxarray_const = const_cast<const __half**>(hxarray_const);
  // CHECK: rocblas_half** hyarray = 0;
  __half** hyarray = 0;

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemm(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* A, int lda, const __half* B, int ldb, const __half* beta, __half* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_hgemm(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_half* alpha, const rocblas_half* A, rocblas_int lda, const rocblas_half* B, rocblas_int ldb, const rocblas_half* beta, rocblas_half* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_hgemm(blasHandle, transa, transb, m, n, k, ha, hA, lda, hB, ldb, hb, hC, ldc);
  blasStatus = hipblasHgemm(blasHandle, transa, transb, m, n, k, ha, hA, lda, hB, ldb, hb, hC, ldc);
#endif

#if CUDA_VERSION >= 8000
  // CHECK: rocblas_datatype DataType;
  // CHECK-NEXT: rocblas_datatype_ DataType_t;
  // CHECK-NEXT: rocblas_datatype blasDataType;
  // CHECK-NEXT: rocblas_datatype R_16F = rocblas_datatype_f16_r;
  // CHECK-NEXT: rocblas_datatype C_16F = rocblas_datatype_f16_c;
  // CHECK-NEXT: rocblas_datatype R_32F = rocblas_datatype_f32_r;
  // CHECK-NEXT: rocblas_datatype C_32F = rocblas_datatype_f32_c;
  // CHECK-NEXT: rocblas_datatype R_64F = rocblas_datatype_f64_r;
  // CHECK-NEXT: rocblas_datatype C_64F = rocblas_datatype_f64_c;
  // CHECK-NEXT: rocblas_datatype R_8I = rocblas_datatype_i8_r;
  // CHECK-NEXT: rocblas_datatype C_8I = rocblas_datatype_i8_c;
  // CHECK-NEXT: rocblas_datatype R_8U = rocblas_datatype_u8_r;
  // CHECK-NEXT: rocblas_datatype C_8U = rocblas_datatype_u8_c;
  // CHECK-NEXT: rocblas_datatype R_32I = rocblas_datatype_i32_r;
  // CHECK-NEXT: rocblas_datatype C_32I = rocblas_datatype_i32_c;
  // CHECK-NEXT: rocblas_datatype R_32U = rocblas_datatype_u32_r;
  // CHECK-NEXT: rocblas_datatype C_32U = rocblas_datatype_u32_c;
  hipDataType DataType;
  hipDataType DataType_t;
  hipDataType blasDataType;
  hipDataType R_16F = HIP_R_16F;
  hipDataType C_16F = HIP_C_16F;
  hipDataType R_32F = HIP_R_32F;
  hipDataType C_32F = HIP_C_32F;
  hipDataType R_64F = HIP_R_64F;
  hipDataType C_64F = HIP_C_64F;
  hipDataType R_8I = HIP_R_8I;
  hipDataType C_8I = HIP_C_8I;
  hipDataType R_8U = HIP_R_8U;
  hipDataType C_8U = HIP_C_8U;
  hipDataType R_32I = HIP_R_32I;
  hipDataType C_32I = HIP_C_32I;
  hipDataType R_32U = HIP_R_32U;
  hipDataType C_32U = HIP_C_32U;

  // CHECK: rocblas_datatype DataType_2, DataType_3;
  hipDataType DataType_2, DataType_3;

  // CHECK: rocblas_gemm_algo blasGemmAlgo;
  // CHECK-NEXT: rocblas_gemm_algo BLAS_GEMM_DFALT = rocblas_gemm_algo_standard;
  hipblasGemmAlgo_t blasGemmAlgo;
  hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasNrm2Ex(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, void* result, cudaDataType resultType, cudaDataType executionType);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_nrm2_ex(rocblas_handle handle, rocblas_int n, const void* x, rocblas_datatype x_type, rocblas_int incx, void* results, rocblas_datatype result_type, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_nrm2_ex(blasHandle, n, image, DataType, incx, image_2, DataType_2, DataType_3);
  blasStatus = hipblasNrm2Ex(blasHandle, n, image, DataType, incx, image_2, DataType_2, DataType_3);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* A, int lda, long long int strideA, const float* B, int ldb, long long int strideB, const float* beta, float* C, int ldc, long long int strideC, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sgemm_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const float* alpha, const float* A, rocblas_int lda, rocblas_stride stride_a, const float* B, rocblas_int ldb, rocblas_stride stride_b, const float* beta, float* C, rocblas_int ldc, rocblas_stride stride_c, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_sgemm_strided_batched(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, strideA, &fB, ldb, strideB, &fb, &fC, ldc, strideC, batchCount);
  blasStatus = hipblasSgemmStridedBatched(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, strideA, &fB, ldb, strideB, &fb, &fC, ldc, strideC, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* A, int lda, long long int strideA, const double* B, int ldb, long long int strideB, const double* beta, double* C, int ldc, long long int strideC, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dgemm_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const double* alpha, const double* A, rocblas_int lda, rocblas_stride stride_a, const double* B, rocblas_int ldb, rocblas_stride stride_b, const double* beta, double* C, rocblas_int ldc, rocblas_stride stride_c, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_dgemm_strided_batched(blasHandle, transa, transb, m, n, k, &da, &dA, lda, strideA, &dB, ldb, strideB, &db, &dC, ldc, strideC, batchCount);
  blasStatus = hipblasDgemmStridedBatched(blasHandle, transa, transb, m, n, k, &da, &dA, lda, strideA, &dB, ldb, strideB, &db, &dC, ldc, strideC, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, long long int strideA, const cuComplex* B, int ldb, long long int strideB, const cuComplex* beta, cuComplex* C, int ldc, long long int strideC, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgemm_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, rocblas_stride stride_a, const rocblas_float_complex* B, rocblas_int ldb, rocblas_stride stride_b, const rocblas_float_complex* beta, rocblas_float_complex* C, rocblas_int ldc, rocblas_stride stride_c, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_cgemm_strided_batched(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, strideA, &complexB, ldb, strideB, &complexb, &complexC, ldc, strideC, batchCount);
  blasStatus = hipblasCgemmStridedBatched(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, strideA, &complexB, ldb, strideB, &complexb, &complexC, ldc, strideC, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, long long int strideA, const cuDoubleComplex* B, int ldb, long long int strideB, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc, long long int strideC, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgemm_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, rocblas_stride stride_a, const rocblas_double_complex* B, rocblas_int ldb, rocblas_stride stride_b, const rocblas_double_complex* beta, rocblas_double_complex* C, rocblas_int ldc, rocblas_stride stride_c, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_zgemm_strided_batched(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, strideA, &dcomplexB, ldb, strideB, &dcomplexb, &dcomplexC, ldc, strideC, batchCount);
  blasStatus = hipblasZgemmStridedBatched(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, strideA, &dcomplexB, ldb, strideB, &dcomplexb, &dcomplexC, ldc, strideC, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* A, int lda, long long int strideA, const __half* B, int ldb, long long int strideB, const __half* beta, __half* C, int ldc, long long int strideC, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_hgemm_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_half* alpha, const rocblas_half* A, rocblas_int lda, rocblas_stride stride_a, const rocblas_half* B, rocblas_int ldb, rocblas_stride stride_b, const rocblas_half* beta, rocblas_half* C, rocblas_int ldc, rocblas_stride stride_c, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_hgemm_strided_batched(blasHandle, transa, transb, m, n, k, ha, hA, lda, strideA, hB, ldb, strideB, hb, hC, ldc, strideC, batchCount);
  blasStatus = hipblasHgemmStridedBatched(blasHandle, transa, transb, m, n, k, ha, hA, lda, strideA, hB, ldb, strideB, hb, hC, ldc, strideC, batchCount);

  void* aptr = nullptr;
  void* Aptr = nullptr;
  void* bptr = nullptr;
  void* Bptr = nullptr;
  void* cptr = nullptr;
  void* Cptr = nullptr;
  void* xptr = nullptr;
  void* yptr = nullptr;
  void* sptr = nullptr;

  // CHECK: rocblas_datatype Atype;
  // CHECK-NEXT: rocblas_datatype Btype;
  // CHECK-NEXT: rocblas_datatype Ctype;
  // CHECK-NEXT: rocblas_datatype Xtype;
  // CHECK-NEXT: rocblas_datatype Ytype;
  // CHECK-NEXT: rocblas_datatype CStype;
  // CHECK-NEXT: rocblas_datatype Executiontype;
  hipDataType Atype;
  hipDataType Btype;
  hipDataType Ctype;
  hipDataType Xtype;
  hipDataType Ytype;
  hipDataType CStype;
  hipDataType Executiontype;

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScalEx(cublasHandle_t handle, int n, const void* alpha, cudaDataType alphaType, void* x, cudaDataType xType, int incx, cudaDataType executionType);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scal_ex(rocblas_handle handle, rocblas_int n, const void* alpha, rocblas_datatype alpha_type, void* x, rocblas_datatype x_type, rocblas_int incx, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_scal_ex(blasHandle, n, aptr, Atype, xptr, Xtype, incx, Executiontype);
  blasStatus = hipblasScalEx(blasHandle, n, aptr, Atype, xptr, Xtype, incx, Executiontype);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasAxpyEx(cublasHandle_t handle, int n, const void* alpha, cudaDataType alphaType, const void* x, cudaDataType xType, int incx, void* y, cudaDataType yType, int incy, cudaDataType executiontype);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_axpy_ex(rocblas_handle handle, rocblas_int n, const void* alpha, rocblas_datatype alpha_type, const void* x, rocblas_datatype x_type, rocblas_int incx, void* y, rocblas_datatype y_type, rocblas_int incy, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_axpy_ex(blasHandle, n, aptr, Atype, xptr, Xtype, incx, yptr, Ytype, incy, Executiontype);
  blasStatus = hipblasAxpyEx(blasHandle, n, aptr, Atype, xptr, Xtype, incx, yptr, Ytype, incy, Executiontype);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDotEx(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, const void* y, cudaDataType yType, int incy, void* result, cudaDataType resultType, cudaDataType executionType);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dot_ex(rocblas_handle handle, rocblas_int n, const void* x, rocblas_datatype x_type, rocblas_int incx, const void* y, rocblas_datatype y_type, rocblas_int incy, void* result, rocblas_datatype result_type, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_dot_ex(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
  blasStatus = hipblasDotEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDotcEx(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, const void* y, cudaDataType yType, int incy, void* result, cudaDataType resultType, cudaDataType executionType);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dotc_ex(rocblas_handle handle, rocblas_int n, const void* x, rocblas_datatype x_type, rocblas_int incx, const void* y, rocblas_datatype y_type, rocblas_int incy, void* result, rocblas_datatype result_type, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_dotc_ex(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
  blasStatus = hipblasDotcEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
#endif

#if CUDA_VERSION >= 8000 && CUDA_VERSION < 11000
  // CHECK: rocblas_datatype computeType;
  cudaDataType computeType;

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, const void* B, cudaDataType Btype, int ldb, const void* beta, void* C, cudaDataType Ctype, int ldc, cudaDataType computeType, cublasGemmAlgo_t algo);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_gemm_ex(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const void* alpha, const void* a, rocblas_datatype a_type, rocblas_int lda, const void* b, rocblas_datatype b_type, rocblas_int ldb, const void* beta, const void* c, rocblas_datatype c_type, rocblas_int ldc, void* d, rocblas_datatype d_type, rocblas_int ldd, rocblas_datatype compute_type, rocblas_gemm_algo algo, int32_t solution_index, uint32_t flags);
  // CHECK: blasStatus = rocblas_gemm_ex(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, computeType, blasGemmAlgo);
  blasStatus = cublasGemmEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, computeType, blasGemmAlgo);
#endif

#if CUDA_VERSION >= 9000
  // CHECK: rocblas_gemm_algo BLAS_GEMM_DEFAULT = rocblas_gemm_algo_standard;
  hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;

  // CHECK: rocblas_math_mode blasMath;
  // CHECK-NEXT: rocblas_math_mode BLAS_DEFAULT_MATH = rocblas_default_math;
  hipblasMath_t blasMath;
  hipblasMath_t BLAS_DEFAULT_MATH = HIPBLAS_DEFAULT_MATH;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetMathMode(cublasHandle_t handle, cublasMath_t* mode);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_math_mode(rocblas_handle handle, rocblas_math_mode* math_mode);
  // CHECK: blasStatus = rocblas_get_math_mode(blasHandle, &blasMath);
  blasStatus = hipblasGetMathMode(blasHandle, &blasMath);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetMathMode(cublasHandle_t handle, cublasMath_t mode);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_math_mode(rocblas_handle handle, rocblas_math_mode math_mode);
  // CHECK: blasStatus = rocblas_set_math_mode(blasHandle, blasMath);
  blasStatus = hipblasSetMathMode(blasHandle, blasMath);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* const Aarray[], int lda, const __half* const Barray[], int ldb, const __half* beta, __half* const Carray[], int ldc, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_hgemm_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_half* alpha, const rocblas_half* const A[], rocblas_int lda, const rocblas_half* const B[], rocblas_int ldb, const rocblas_half* beta, rocblas_half* const C[], rocblas_int ldc, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_hgemm_batched(blasHandle, transa, transb, m, n, k, ha, hAarray_const, lda, hBarray_const, ldb, hb, hCarray, ldc, batchCount);
  blasStatus = hipblasHgemmBatched(blasHandle, transa, transb, m, n, k, ha, hAarray_const, lda, hBarray_const, ldb, hb, hCarray, ldc, batchCount);
#endif

#if CUDA_VERSION >= 9010 && CUDA_VERSION < 11000
  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* const Aarray[], cudaDataType Atype, int lda, const void* const Barray[], cudaDataType Btype, int ldb, const void* beta, void* const Carray[], cudaDataType Ctype, int ldc, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_gemm_batched_ex(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const void* alpha, const void* a, rocblas_datatype a_type, rocblas_int lda, const void* b, rocblas_datatype b_type, rocblas_int ldb, const void* beta, const void* c, rocblas_datatype c_type, rocblas_int ldc, void* d, rocblas_datatype d_type, rocblas_int ldd, rocblas_int batch_count, rocblas_datatype compute_type, rocblas_gemm_algo algo, int32_t solution_index, uint32_t flags);
  // CHECK: blasStatus = rocblas_gemm_batched_ex(blasHandle, transa, transb, m, n, k, aptr, voidAarray_const, Atype, lda, voidBarray_const, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, computeType, blasGemmAlgo);
  blasStatus = cublasGemmBatchedEx(blasHandle, transa, transb, m, n, k, aptr, voidAarray_const, Atype, lda, voidBarray_const, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, computeType, blasGemmAlgo);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmStridedBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, long long int strideA, const void* B, cudaDataType Btype, int ldb, long long int strideB, const void* beta, void* C, cudaDataType Ctype, int ldc, long long int strideC, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_gemm_strided_batched_ex(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const void* alpha, const void* a, rocblas_datatype a_type, rocblas_int lda, rocblas_stride stride_a, const void* b, rocblas_datatype b_type, rocblas_int ldb, rocblas_stride stride_b, const void* beta, const void* c, rocblas_datatype c_type, rocblas_int ldc, rocblas_stride stride_c, void* d, rocblas_datatype d_type, rocblas_int ldd, rocblas_stride stride_d, rocblas_int batch_count, rocblas_datatype compute_type, rocblas_gemm_algo algo, int32_t solution_index, uint32_t flags);
  // CHECK: blasStatus = rocblas_gemm_strided_batched_ex(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, computeType, blasGemmAlgo);
  blasStatus = cublasGemmStridedBatchedEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, computeType, blasGemmAlgo);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: rocblas_operation BLAS_OP_HERMITAN = rocblas_operation_conjugate_transpose;
  hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;

  // CHECK: rocblas_fill BLAS_FILL_MODE_FULL = rocblas_fill_full;
  hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasRotEx(cublasHandle_t handle, int n, void* x, cudaDataType xType, int incx, void* y, cudaDataType yType, int incy, const void* c, const void* s, cudaDataType csType, cudaDataType executiontype);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_rot_ex(rocblas_handle handle, rocblas_int n, void* x, rocblas_datatype x_type, rocblas_int incx, void* y, rocblas_datatype y_type, rocblas_int incy, const void* c, const void* s, rocblas_datatype cs_type, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_rot_ex(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, cptr, sptr, CStype, Executiontype);
  blasStatus = hipblasRotEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, cptr, sptr, CStype, Executiontype);
#endif

#if CUDA_VERSION >= 11000
  // CHECK: rocblas_datatype R_16BF = rocblas_datatype_bf16_r;
  // CHECK-NEXT: rocblas_datatype C_16BF = rocblas_datatype_bf16_c;
  hipDataType R_16BF = HIP_R_16BF;
  hipDataType C_16BF = HIP_C_16BF;

  // CHECK: rocblas_bfloat16* bf16A = 0;
  __hip_bfloat16* bf16A = 0;
  // CHECK: rocblas_bfloat16* bf16x = 0;
  __hip_bfloat16* bf16x = 0;
  // CHECK: rocblas_bfloat16* bf16y = 0;
  __hip_bfloat16* bf16y = 0;

  // CHECK: rocblas_bfloat16** bf16Aarray = 0;
  __hip_bfloat16** bf16Aarray = 0;
  // CHECK: const rocblas_bfloat16** const bf16Aarray_const = const_cast<const rocblas_bfloat16**>(bf16Aarray);
  const __hip_bfloat16** const bf16Aarray_const = const_cast<const __hip_bfloat16**>(bf16Aarray);
  // CHECK: rocblas_bfloat16** bf16xarray = 0;
  __hip_bfloat16** bf16xarray = 0;
  // CHECK: const rocblas_bfloat16** const bf16xarray_const = const_cast<const rocblas_bfloat16**>(bf16xarray_const);
  const __hip_bfloat16** const bf16xarray_const = const_cast<const __hip_bfloat16**>(bf16xarray_const);
  // CHECK: rocblas_bfloat16** bf16yarray = 0;
  __hip_bfloat16** bf16yarray = 0;

  // CHECK: rocblas_computetype blasComputeType;
  // CHECK-NEXT: rocblas_computetype BLAS_COMPUTE_32F = rocblas_compute_type_f32;
  hipblasComputeType_t blasComputeType;
  hipblasComputeType_t BLAS_COMPUTE_32F = HIPBLAS_COMPUTE_32F;
#endif

#if CUDA_VERSION >= 11040 && CUBLAS_VERSION >= 11600
  // CUDA: CUBLASAPI const char* CUBLASWINAPI cublasGetStatusString(cublasStatus_t status);
  // ROC: ROCBLAS_EXPORT const char* rocblas_status_to_string(rocblas_status status);
  // CHECK: const_ch = rocblas_status_to_string(blasStatus);
  const_ch = cublasGetStatusString(blasStatus);
#endif

#if CUDA_VERSION >= 11060 && CUBLAS_VERSION >= 110902 // CUDA 11.6.2
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemvBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuComplex* alpha, const cuComplex* const Aarray[], int lda, const cuComplex* const xarray[], int incx, const cuComplex* beta, cuComplex* const yarray[], int incy, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgemv_batched(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* const A[], rocblas_int lda, const rocblas_float_complex* const x[], rocblas_int incx, const rocblas_float_complex* beta, rocblas_float_complex* const y[], rocblas_int incy, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_cgemv_batched(blasHandle, blasOperation, m, n, &complexa, complexAarray_const, lda, complexXarray_const, incx, &complexb, complexYarray, incy, batchCount);
  blasStatus = hipblasCgemvBatched(blasHandle, blasOperation, m, n, &complexa, complexAarray_const, lda, complexXarray_const, incx, &complexb, complexYarray, incy, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemvBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* const Aarray[], int lda, const cuDoubleComplex* const xarray[], int incx, const cuDoubleComplex* beta, cuDoubleComplex* const yarray[], int incy, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgemv_batched(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* const A[], rocblas_int lda, const rocblas_double_complex* const x[], rocblas_int incx, const rocblas_double_complex* beta, rocblas_double_complex* const y[], rocblas_int incy, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_zgemv_batched(blasHandle, blasOperation, m, n, &dcomplexa, dcomplexAarray_const, lda, dcomplexXarray_const, incx, &dcomplexb, dcomplexYarray, incy, batchCount);
  blasStatus = hipblasZgemvBatched(blasHandle, blasOperation, m, n, &dcomplexa, dcomplexAarray_const, lda, dcomplexXarray_const, incx, &dcomplexb, dcomplexYarray, incy, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemvStridedBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, long long int strideA, const cuComplex* x, int incx, long long int stridex, const cuComplex* beta, cuComplex* y, int incy, long long int stridey, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgemv_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, rocblas_stride strideA, const rocblas_float_complex* x, rocblas_int incx, rocblas_stride stridex, const rocblas_float_complex* beta, rocblas_float_complex* y, rocblas_int incy, rocblas_stride stridey, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_cgemv_strided_batched(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, strideA, &complexx, incx, stridex, &complexb, &complexy, incy, stridey, batchCount);
  blasStatus = hipblasCgemvStridedBatched(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, strideA, &complexx, incx, stridex, &complexb, &complexy, incy, stridey, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemvStridedBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, long long int strideA, const cuDoubleComplex* x, int incx, long long int stridex, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy, long long int stridey, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgemv_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, rocblas_stride strideA, const rocblas_double_complex* x, rocblas_int incx, rocblas_stride stridex, const rocblas_double_complex* beta, rocblas_double_complex* y, rocblas_int incy, rocblas_stride stridey, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_zgemv_strided_batched(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, strideA, &dcomplexx, incx, stridex, &dcomplexb, &dcomplexy, incy, stridey, batchCount);
  blasStatus = hipblasZgemvStridedBatched(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, strideA, &dcomplexx, incx, stridex, &dcomplexb, &dcomplexy, incy, stridey, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHSHgemvBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const float* alpha, const __half* const Aarray[], int lda, const __half* const xarray[], int incx, const float* beta, __half* const yarray[], int incy, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_hshgemv_batched(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, const float* alpha, const rocblas_half* const A[], rocblas_int lda, const rocblas_half* const x[], rocblas_int incx, const float* beta, rocblas_half* const y[], rocblas_int incy, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_hshgemv_batched(blasHandle, blasOperation, m, n, &fa, hAarray_const, lda, hxarray_const, incx, &fb, hyarray, incy, batchCount);
  blasStatus = cublasHSHgemvBatched(blasHandle, blasOperation, m, n, &fa, hAarray_const, lda, hxarray_const, incx, &fb, hyarray, incy, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHSSgemvBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const float* alpha, const __half* const Aarray[], int lda, const __half* const xarray[], int incx, const float* beta, float* const yarray[], int incy, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_hssgemv_batched(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, const float* alpha, const rocblas_half* const A[], rocblas_int lda, const rocblas_half* const x[], rocblas_int incx, const float* beta, float* const y[], rocblas_int incy, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_hssgemv_batched(blasHandle, blasOperation, m, n, &fa, hAarray_const, lda, hxarray_const, incx, &fb, fyarray, incy, batchCount);
  blasStatus = cublasHSSgemvBatched(blasHandle, blasOperation, m, n, &fa, hAarray_const, lda, hxarray_const, incx, &fb, fyarray, incy, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasTSTgemvBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const float* alpha, const __nv_bfloat16* const Aarray[], int lda, const __nv_bfloat16* const xarray[], int incx, const float* beta, __nv_bfloat16* const yarray[], int incy, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_tstgemv_batched(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, const float* alpha, const rocblas_bfloat16* const A[], rocblas_int lda, const rocblas_bfloat16* const x[], rocblas_int incx, const float* beta, rocblas_bfloat16* const y[], rocblas_int incy, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_tstgemv_batched(blasHandle, blasOperation, m, n, &fa, bf16Aarray_const, lda, bf16xarray_const, incx, &fb, bf16yarray, incy, batchCount);
  blasStatus = cublasTSTgemvBatched(blasHandle, blasOperation, m, n, &fa, bf16Aarray_const, lda, bf16xarray_const, incx, &fb, bf16yarray, incy, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasTSSgemvBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const float* alpha, const __nv_bfloat16* const Aarray[], int lda, const __nv_bfloat16* const xarray[], int incx, const float* beta, float* const yarray[], int incy, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_tssgemv_batched(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, const float* alpha, const rocblas_bfloat16* const A[], rocblas_int lda, const rocblas_bfloat16* const x[], rocblas_int incx, const float* beta, float* const y[], rocblas_int incy, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_tssgemv_batched(blasHandle, blasOperation, m, n, &fa, bf16Aarray_const, lda, bf16xarray_const, incx, &fb, fyarray, incy, batchCount);
  blasStatus = cublasTSSgemvBatched(blasHandle, blasOperation, m, n, &fa, bf16Aarray_const, lda, bf16xarray_const, incx, &fb, fyarray, incy, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHSHgemvStridedBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const float* alpha, const __half* A, int lda, long long int strideA, const __half* x, int incx, long long int stridex, const float* beta, __half* y, int incy, long long int stridey, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_hshgemv_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_int m, rocblas_int n, const float* alpha, const rocblas_half* A, rocblas_int lda, rocblas_stride strideA, const rocblas_half* x, rocblas_int incx, rocblas_stride stridex, const float* beta, rocblas_half* y, rocblas_int incy, rocblas_stride stridey, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_hshgemv_strided_batched(blasHandle, blasOperation, m, n, &fa, hA, lda, strideA, hx, incx, stridex, &fb, hy, incy, stridey, batchCount);
  blasStatus = cublasHSHgemvStridedBatched(blasHandle, blasOperation, m, n, &fa, hA, lda, strideA, hx, incx, stridex, &fb, hy, incy, stridey, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHSSgemvStridedBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const float* alpha, const __half* A, int lda, long long int strideA, const __half* x, int incx, long long int stridex, const float* beta, float* y, int incy, long long int stridey, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_hssgemv_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_int m, rocblas_int n, const float* alpha, const rocblas_half* A, rocblas_int lda, rocblas_stride strideA, const rocblas_half* x, rocblas_int incx, rocblas_stride stridex, const float* beta, float* y, rocblas_int incy, rocblas_stride stridey, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_hssgemv_strided_batched(blasHandle, blasOperation, m, n, &fa, hA, lda, strideA, hx, incx, stridex, &fb, &fy, incy, stridey, batchCount);
  blasStatus = cublasHSSgemvStridedBatched(blasHandle, blasOperation, m, n, &fa, hA, lda, strideA, hx, incx, stridex, &fb, &fy, incy, stridey, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasTSTgemvStridedBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const float* alpha, const __nv_bfloat16* A, int lda, long long int strideA, const __nv_bfloat16* x, int incx, long long int stridex, const float* beta, __nv_bfloat16* y, int incy, long long int stridey, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_tstgemv_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_int m, rocblas_int n, const float* alpha, const rocblas_bfloat16* A, rocblas_int lda, rocblas_stride strideA, const rocblas_bfloat16* x, rocblas_int incx, rocblas_stride stridex, const float* beta, rocblas_bfloat16* y, rocblas_int incy, rocblas_stride stridey, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_tstgemv_strided_batched(blasHandle, blasOperation, m, n, &fa, bf16A, lda, strideA, bf16x, incx, stridex, &fb, bf16y, incy, stridey, batchCount);
  blasStatus = cublasTSTgemvStridedBatched(blasHandle, blasOperation, m, n, &fa, bf16A, lda, strideA, bf16x, incx, stridex, &fb, bf16y, incy, stridey, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasTSSgemvStridedBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const float* alpha, const __nv_bfloat16* A, int lda, long long int strideA, const __nv_bfloat16* x, int incx, long long int stridex, const float* beta, float* y, int incy, long long int stridey, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_tssgemv_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_int m, rocblas_int n, const float* alpha, const rocblas_bfloat16* A, rocblas_int lda, rocblas_stride strideA, const rocblas_bfloat16* x, rocblas_int incx, rocblas_stride stridex, const float* beta, float* y, rocblas_int incy, rocblas_stride stridey, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_tssgemv_strided_batched(blasHandle, blasOperation, m, n, &fa, bf16A, lda, strideA, bf16x, incx, stridex, &fb, &fy, incy, stridey, batchCount);
  blasStatus = cublasTSSgemvStridedBatched(blasHandle, blasOperation, m, n, &fa, bf16A, lda, strideA, bf16x, incx, stridex, &fb, &fy, incy, stridey, batchCount);
#endif

  return 0;
}
