// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental --roc %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocblas.h"
// CHECK-NOT: #include "hipblas.h"
#include "hipblas.h"
#include "hipblas.h"
// CHECK-NOT: #include "rocblas.h"

int main() {
  printf("16. cuBLAS API to hipBLAS API synthetic test\n");

  // CHECK: rocblas_operation blasOperation;
  // CHECK-NEXT: rocblas_operation BLAS_OP_N = rocblas_operation_none;
  // CHECK-NEXT: rocblas_operation BLAS_OP_T = rocblas_operation_transpose;
  // CHECK-NEXT: rocblas_operation BLAS_OP_C = rocblas_operation_conjugate_transpose;
  hipblasOperation_t blasOperation;
  hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;

  // CHECK: rocblas_status blasStatus;
  // CHECK-NEXT: rocblas_status blasStatus_t;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_SUCCESS = rocblas_status_success;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_NOT_INITIALIZED = rocblas_status_invalid_handle;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_ALLOC_FAILED = rocblas_status_not_implemented;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_INVALID_VALUE = rocblas_status_invalid_pointer;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_MAPPING_ERROR = rocblas_status_invalid_size;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_EXECUTION_FAILED = rocblas_status_memory_error;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_INTERNAL_ERROR = rocblas_status_internal_error;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_NOT_SUPPORTED = rocblas_status_perf_degraded;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_ARCH_MISMATCH = rocblas_status_size_query_mismatch;
  hipblasStatus_t blasStatus;
  hipblasStatus_t blasStatus_t;
  hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;

  // CHECK: rocblas_fill blasFillMode;
  // CHECK-NEXT: rocblas_fill BLAS_FILL_MODE_LOWER = rocblas_fill_lower;
  // CHECK-NEXT: rocblas_fill BLAS_FILL_MODE_UPPER = rocblas_fill_upper;
  hipblasFillMode_t blasFillMode;
  hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;

  // CHECK: rocblas_diagonal blasDiagType;
  // CHECK-NEXT: rocblas_diagonal BLAS_DIAG_NON_UNIT = rocblas_diagonal_non_unit;
  // CHECK-NEXT: rocblas_diagonal BLAS_DIAG_UNIT = rocblas_diagonal_unit;
  hipblasDiagType_t blasDiagType;
  hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;

  // CHECK: rocblas_side blasSideMode;
  // CHECK-NEXT: rocblas_side BLAS_SIDE_LEFT = rocblas_side_left;
  // CHECK-NEXT: rocblas_side BLAS_SIDE_RIGHT = rocblas_side_right;
  hipblasSideMode_t blasSideMode;
  hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;

  // CHECK: rocblas_pointer_mode blasPointerMode;
  // CHECK-NEXT: rocblas_pointer_mode BLAS_POINTER_MODE_HOST = rocblas_pointer_mode_host;
  // CHECK-NEXT: rocblas_pointer_mode BLAS_POINTER_MODE_DEVICE = rocblas_pointer_mode_device;
  hipblasPointerMode_t blasPointerMode;
  hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;

  // CHECK: rocblas_atomics_mode blasAtomicsMode;
  // CHECK-NEXT: rocblas_atomics_mode BLAS_ATOMICS_NOT_ALLOWED = rocblas_atomics_not_allowed;
  // CHECK-NEXT: rocblas_atomics_mode BLAS_ATOMICS_ALLOWED = rocblas_atomics_allowed;
  hipblasAtomicsMode_t blasAtomicsMode;
  hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;

  // CHECK: rocblas_gemm_algo blasGemmAlgo;
  // CHECK-NEXT: rocblas_gemm_algo BLAS_GEMM_DFALT = rocblas_gemm_algo_standard;
  hipblasGemmAlgo_t blasGemmAlgo;
  hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;

  // CHECK: rocblas_handle blasHandle;
  hipblasHandle_t blasHandle;

  // CUDA: hipblasStatus_t CUBLASWINAPI cublasInit(void);
  // ROC: ROCBLAS_EXPORT void rocblas_initialize(void);
  // CHECK: blasStatus = rocblas_initialize();
  blasStatus = cublasInit();

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasGetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t* mode);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_atomics_mode(rocblas_handle handle, rocblas_atomics_mode* atomics_mode);
  // CHECK: blasStatus = rocblas_get_atomics_mode(blasHandle, &blasAtomicsMode);
  blasStatus = hipblasGetAtomicsMode(blasHandle, &blasAtomicsMode);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t mode);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_atomics_mode(rocblas_handle handle, rocblas_atomics_mode atomics_mode);
  // CHECK: blasStatus = rocblas_set_atomics_mode(blasHandle, blasAtomicsMode);
  blasStatus = hipblasSetAtomicsMode(blasHandle, blasAtomicsMode);

  const char* const_ch = nullptr;

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCreate(hipblasHandle_t* handle);
  // CUDA: #define hipblasCreate hipblasCreate
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_create_handle(rocblas_handle* handle);
  // CHECK: blasStatus = rocblas_create_handle(&blasHandle);
  // CHECK-NEXT: blasStatus = rocblas_create_handle(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDestroy(hipblasHandle_t handle);
  // CUDA: #define hipblasDestroy hipblasDestroy
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_destroy_handle(rocblas_handle handle);
  // CHECK: blasStatus = rocblas_destroy_handle(blasHandle);
  // CHECK-NEXT: blasStatus = rocblas_destroy_handle(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);

  // CHECK: hipStream_t stream;
  hipStream_t stream;

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSetStream(hipblasHandle_t handle, hipStream_t streamId);
  // CUDA: #define hipblasSetStream hipblasSetStream
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_stream(rocblas_handle handle, hipStream_t stream);
  // CHECK: blasStatus = rocblas_set_stream(blasHandle, stream);
  // CHECK-NEXT: blasStatus = rocblas_set_stream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasGetStream(hipblasHandle_t handle, hipStream_t* streamId);
  // CUDA: #define hipblasGetStream hipblasGetStream
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_stream(rocblas_handle handle, hipStream_t* stream);
  // CHECK: blasStatus = rocblas_get_stream(blasHandle, &stream);
  // CHECK-NEXT: blasStatus = rocblas_get_stream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSetPointerMode(hipblasHandle_t handle, hipblasPointerMode_t mode);
  // CUDA: #define hipblasSetPointerMode hipblasSetPointerMode
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_pointer_mode(rocblas_handle handle, rocblas_pointer_mode pointer_mode);
  // CHECK: blasStatus = rocblas_set_pointer_mode(blasHandle, blasPointerMode);
  // CHECK-NEXT: blasStatus = rocblas_set_pointer_mode(blasHandle, blasPointerMode);
  blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);
  blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasGetPointerMode(hipblasHandle_t handle, hipblasPointerMode_t* mode);
  // CUDA: #define hipblasGetPointerMode hipblasGetPointerMode
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_pointer_mode(rocblas_handle handle, rocblas_pointer_mode* pointer_mode);
  // CHECK: blasStatus = rocblas_get_pointer_mode(blasHandle, &blasPointerMode);
  // CHECK-NEXT: blasStatus = rocblas_get_pointer_mode(blasHandle, &blasPointerMode);
  blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);
  blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);

  int n = 0;
  int num = 0;
  int res = 0;
  int incx = 0;
  int incy = 0;
  void* image = nullptr;
  void* image_2 = nullptr;

  // https://github.com/ROCmSoftwarePlatform/rocBLAS/issues/1281
  // TODO: Apply the chosen typecasting of int to rocblas_int arguments

  /*
  #if defined(rocblas_ILP64)
    typedef int64_t rocblas_int;
  #else
    typedef int32_t rocblas_int;
  #endif
  */

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasSetVector(int n, int elemSize, const void* x, int incx, void* devicePtr, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_vector(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_set_vector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasSetVector(n, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasGetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_vector(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_get_vector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasGetVector(n, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasSetVectorAsync(int n, int elemSize, const void* hostPtr, int incx, void* devicePtr, int incy, hipStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_vector_async(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy, hipStream_t stream);
  // CHECK: blasStatus = rocblas_set_vector_async(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetVectorAsync(n, num, image, incx, image_2, incy, stream);

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasGetVectorAsync(int n, int elemSize, const void* devicePtr, int incx, void* hostPtr, int incy, hipStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_vector_async(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy, hipStream_t stream);
  // CHECK: blasStatus = rocblas_get_vector_async(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetVectorAsync(n, num, image, incx, image_2, incy, stream);

  int rows = 0;
  int cols = 0;

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasSetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_matrix(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_set_matrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasSetMatrix(rows, cols, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasGetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_matrix(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_get_matrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasGetMatrix(rows, cols, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasSetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, hipStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_matrix_async(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb, hipStream_t stream);
  // CHECK: blasStatus = rocblas_set_matrix_async(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  // TODO: #1281
  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasGetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, hipStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_matrix_async(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb, hipStream_t stream);
  // CHECK: blasStatus = rocblas_get_matrix_async(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  float fa = 0;
  float fA = 0;
  float fb = 0;
  float fB = 0;
  float fx = 0;
  float fx1 = 0;
  float fy = 0;
  float fy1 = 0;
  float fc = 0;
  float fC = 0;
  float fs = 0;
  float fd1 = 0;
  float fd2 = 0;
  float fresult = 0;

  float** fAarray = 0;
  float** fBarray = 0;
  float** fCarray = 0;
  float** fTauarray = 0;

  double da = 0;
  double dA = 0;
  double db = 0;
  double dB = 0;
  double dx = 0;
  double dx1 = 0;
  double dy = 0;
  double dy1 = 0;
  double dc = 0;
  double dC = 0;
  double ds = 0;
  double dd1 = 0;
  double dd2 = 0;
  double dresult = 0;

  double** dAarray = 0;
  double** dBarray = 0;
  double** dCarray = 0;
  double** dTauarray = 0;

  void** voidAarray = nullptr;
  void** voidBarray = nullptr;
  void** voidCarray = nullptr;

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSnrm2(hipblasHandle_t handle, int n, const float* x, int incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_snrm2(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, float* result);
  // CHECK: blasStatus = rocblas_snrm2(blasHandle, n, &fx, incx, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_snrm2(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDnrm2(hipblasHandle_t handle, int n, const double* x, int incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dnrm2(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, double* result);
  // CHECK: blasStatus = rocblas_dnrm2(blasHandle, n, &dx, incx, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_dnrm2(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);

  // CHECK: rocblas_float_complex complex, complexa, complexA, complexB, complexC, complexx, complexy, complexs, complexb;
  hipComplex complex, complexa, complexA, complexB, complexC, complexx, complexy, complexs, complexb;
  // CHECK: rocblas_double_complex dcomplex, dcomplexa, dcomplexA, dcomplexB, dcomplexC, dcomplexx, dcomplexy, dcomplexs, dcomplexb;
  hipDoubleComplex dcomplex, dcomplexa, dcomplexA, dcomplexB, dcomplexC, dcomplexx, dcomplexy, dcomplexs, dcomplexb;

  // CHECK: rocblas_float_complex** complexAarray = 0;
  // CHECK-NEXT: rocblas_float_complex** complexBarray = 0;
  // CHECK-NEXT: rocblas_float_complex** complexCarray = 0;
  // CHECK-NEXT: rocblas_float_complex** complexTauarray = 0;
  hipComplex** complexAarray = 0;
  hipComplex** complexBarray = 0;
  hipComplex** complexCarray = 0;
  hipComplex** complexTauarray = 0;

  // CHECK: rocblas_double_complex** dcomplexAarray = 0;
  // CHECK-NEXT: rocblas_double_complex** dcomplexBarray = 0;
  // CHECK-NEXT: rocblas_double_complex** dcomplexCarray = 0;
  // CHECK-NEXT: rocblas_double_complex** dcomplexTauarray = 0;
  hipDoubleComplex** dcomplexAarray = 0;
  hipDoubleComplex** dcomplexBarray = 0;
  hipDoubleComplex** dcomplexCarray = 0;
  hipDoubleComplex** dcomplexTauarray = 0;

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasScnrm2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scnrm2(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, float* result);
  // CHECK: blasStatus = rocblas_scnrm2(blasHandle, n, &complex, incx, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_scnrm2(blasHandle, n, &complex, incx, &fresult);
  blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);
  blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDznrm2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dznrm2(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, double* result);
  // CHECK: blasStatus = rocblas_dznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_dznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSdot(hipblasHandle_t handle, int n, const float* x, int incx, const float* y, int incy, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sdot(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, const float* y, rocblas_int incy, float* result);
  // CHECK: blasStatus = rocblas_sdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_sdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDdot(hipblasHandle_t handle, int n, const double* x, int incx, const double* y, int incy, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ddot(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, const double* y, rocblas_int incy, double* result);
  // CHECK: blasStatus = rocblas_ddot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_ddot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCdotu(hipblasHandle_t handle, int n, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cdotu(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* result);
  // CHECK: blasStatus = rocblas_cdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  // CHECK-NEXT: blasStatus = rocblas_cdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCdotc(hipblasHandle_t handle, int n, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cdotc(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* result);
  // CHECK: blasStatus = rocblas_cdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  // CHECK-NEXT: blasStatus = rocblas_cdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZdotu(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdotu(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* result);
  // CHECK: blasStatus = rocblas_zdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  // CHECK-NEXT: blasStatus = rocblas_zdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZdotc(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdotc(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* result);
  // CHECK: blasStatus = rocblas_zdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  // CHECK-NEXT: blasStatus = rocblas_zdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSscal(hipblasHandle_t handle, int n, const float* alpha, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sscal(rocblas_handle handle, rocblas_int n, const float* alpha, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_sscal(blasHandle, n, &fy, &fx, incx);
  // CHECK-NEXT: blasStatus = rocblas_sscal(blasHandle, n, &fy, &fx, incx);
  blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);
  blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDscal(hipblasHandle_t handle, int n, const double* alpha, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dscal(rocblas_handle handle, rocblas_int n, const double* alpha, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dscal(blasHandle, n, &dx, &dy, incx);
  // CHECK-NEXT: blasStatus = rocblas_dscal(blasHandle, n, &dx, &dy, incx);
  blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);
  blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCscal(hipblasHandle_t handle, int n, const hipComplex* alpha, hipComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cscal(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* alpha, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_cscal(blasHandle, n, &complexa, &complexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_cscal(blasHandle, n, &complexa, &complexx, incx);
  blasStatus = hipblasCscal(blasHandle, n, &complexa, &complexx, incx);
  blasStatus = hipblasCscal(blasHandle, n, &complexa, &complexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCsscal(hipblasHandle_t handle, int n, const float* alpha, hipComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csscal(rocblas_handle handle, rocblas_int n, const float* alpha, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_csscal(blasHandle, n, &fx, &complexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_csscal(blasHandle, n, &fx, &complexx, incx);
  blasStatus = hipblasCsscal(blasHandle, n, &fx, &complexx, incx);
  blasStatus = hipblasCsscal(blasHandle, n, &fx, &complexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZscal(hipblasHandle_t handle, int n, const hipDoubleComplex* alpha, hipDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zscal(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* alpha, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_zscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_zscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  blasStatus = hipblasZscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  blasStatus = hipblasZscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZdscal(hipblasHandle_t handle, int n, const double* alpha, hipDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdscal(rocblas_handle handle, rocblas_int n, const double* alpha, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_zdscal(blasHandle, n, &dx, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_zdscal(blasHandle, n, &dx, &dcomplexx, incx);
  blasStatus = hipblasZdscal(blasHandle, n, &dx, &dcomplexx, incx);
  blasStatus = hipblasZdscal(blasHandle, n, &dx, &dcomplexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSaxpy(hipblasHandle_t handle, int n, const float* alpha, const float* x, int incx, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_saxpy(rocblas_handle handle, rocblas_int n, const float* alpha, const float* x, rocblas_int incx, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_saxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  // CHECK-NEXT: blasStatus = rocblas_saxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDaxpy(hipblasHandle_t handle, int n, const double* alpha, const double* x, int incx, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_daxpy(rocblas_handle handle, rocblas_int n, const double* alpha, const double* x, rocblas_int incx, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_daxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  // CHECK-NEXT: blasStatus = rocblas_daxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCaxpy(hipblasHandle_t handle, int n, const hipComplex* alpha, const hipComplex* x, int incx, hipComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_caxpy(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_caxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_caxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCaxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCaxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZaxpy(hipblasHandle_t handle, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zaxpy(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_zaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasScopy(hipblasHandle_t handle, int n, const float* x, int incx, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scopy(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_scopy(blasHandle, n, &fx, incx, &fy, incy);
  // CHECK-NEXT: blasStatus = rocblas_scopy(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDcopy(hipblasHandle_t handle, int n, const double* x, int incx, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dcopy(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dcopy(blasHandle, n, &dx, incx, &dy, incy);
  // CHECK-NEXT: blasStatus = rocblas_dcopy(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCcopy(hipblasHandle_t handle, int n, const hipComplex* x, int incx, hipComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ccopy(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_ccopy(blasHandle, n, &complexx, incx, &complexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_ccopy(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCcopy(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCcopy(blasHandle, n, &complexx, incx, &complexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZcopy(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zcopy(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_zcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSswap(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sswap(rocblas_handle handle, rocblas_int n, float* x, rocblas_int incx, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_sswap(blasHandle, n, &fx, incx, &fy, incy);
  // CHECK-NEXT: blasStatus = rocblas_sswap(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDswap(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dswap(rocblas_handle handle, rocblas_int n, double* x, rocblas_int incx, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dswap(blasHandle, n, &dx, incx, &dy, incy);
  // CHECK-NEXT: blasStatus = rocblas_dswap(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCswap(hipblasHandle_t handle, int n, hipComplex* x, int incx, hipComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cswap(rocblas_handle handle, rocblas_int n, rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_cswap(blasHandle, n, &complexx, incx, &complexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_cswap(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCswap(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCswap(blasHandle, n, &complexx, incx, &complexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZswap(hipblasHandle_t handle, int n, hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zswap(rocblas_handle handle, rocblas_int n, rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_zswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIsamax(hipblasHandle_t handle, int n, const float* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_isamax(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_isamax(blasHandle, n, &fx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_isamax(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIdamax(hipblasHandle_t handle, int n, const double* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_idamax(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_idamax(blasHandle, n, &dx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_idamax(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIcamax(hipblasHandle_t handle, int n, const hipComplex* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_icamax(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_icamax(blasHandle, n, &complexx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_icamax(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamax(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamax(blasHandle, n, &complexx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIzamax(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_izamax(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_izamax(blasHandle, n, &dcomplexx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_izamax(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamax(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamax(blasHandle, n, &dcomplexx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIsamin(hipblasHandle_t handle, int n, const float* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_isamin(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_isamin(blasHandle, n, &fx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_isamin(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIdamin(hipblasHandle_t handle, int n, const double* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_idamin(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_idamin(blasHandle, n, &dx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_idamin(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIcamin(hipblasHandle_t handle, int n, const hipComplex* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_icamin(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_icamin(blasHandle, n, &complexx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_icamin(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamin(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamin(blasHandle, n, &complexx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIzamin(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_izamin(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_izamin(blasHandle, n, &dcomplexx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_izamin(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamin(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamin(blasHandle, n, &dcomplexx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSasum(hipblasHandle_t handle, int n, const float* x, int incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sasum(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, float* result);
  // CHECK: blasStatus = rocblas_sasum(blasHandle, n, &fx, incx, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_sasum(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDasum(hipblasHandle_t handle, int n, const double* x, int incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dasum(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, double* result);
  // CHECK: blasStatus = rocblas_dasum(blasHandle, n, &dx, incx, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_dasum(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasScasum(hipblasHandle_t handle, int n, const hipComplex* x, int incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scasum(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, float* result);
  // CHECK: blasStatus = rocblas_scasum(blasHandle, n, &complexx, incx, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_scasum(blasHandle, n, &complexx, incx, &fresult);
  blasStatus = hipblasScasum(blasHandle, n, &complexx, incx, &fresult);
  blasStatus = hipblasScasum(blasHandle, n, &complexx, incx, &fresult);

  // TODO: #1281
  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDzasum(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dzasum(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, double* result);
  // CHECK: blasStatus = rocblas_dzasum(blasHandle, n, &dcomplexx, incx, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_dzasum(blasHandle, n, &dcomplexx, incx, &dresult);
  blasStatus = hipblasDzasum(blasHandle, n, &dcomplexx, incx, &dresult);
  blasStatus = hipblasDzasum(blasHandle, n, &dcomplexx, incx, &dresult);

#if CUDA_VERSION >= 8000
  // CHECK: rocblas_datatype DataType;
  // CHECK-NEXT: rocblas_datatype_ DataType_t;
  // CHECK-NEXT: rocblas_datatype blasDataType;
  // CHECK-NEXT: rocblas_datatype R_16F = rocblas_datatype_f16_r;
  // CHECK-NEXT: rocblas_datatype C_16F = rocblas_datatype_f16_c;
  // CHECK-NEXT: rocblas_datatype R_32F = rocblas_datatype_f32_r;
  // CHECK-NEXT: rocblas_datatype C_32F = rocblas_datatype_f32_c;
  // CHECK-NEXT: rocblas_datatype R_64F = rocblas_datatype_f64_r;
  // CHECK-NEXT: rocblas_datatype C_64F = rocblas_datatype_f64_c;
  // CHECK-NEXT: rocblas_datatype R_8I = rocblas_datatype_i8_r;
  // CHECK-NEXT: rocblas_datatype C_8I = rocblas_datatype_i8_c;
  // CHECK-NEXT: rocblas_datatype R_8U = rocblas_datatype_u8_r;
  // CHECK-NEXT: rocblas_datatype C_8U = rocblas_datatype_u8_c;
  // CHECK-NEXT: rocblas_datatype R_32I = rocblas_datatype_i32_r;
  // CHECK-NEXT: rocblas_datatype C_32I = rocblas_datatype_i32_c;
  // CHECK-NEXT: rocblas_datatype R_32U = rocblas_datatype_u32_r;
  // CHECK-NEXT: rocblas_datatype C_32U = rocblas_datatype_u32_c;
  hipDataType DataType;
  hipDataType DataType_t;
  hipDataType blasDataType;
  hipDataType R_16F = HIP_R_16F;
  hipDataType C_16F = HIP_C_16F;
  hipDataType R_32F = HIP_R_32F;
  hipDataType C_32F = HIP_C_32F;
  hipDataType R_64F = HIP_R_64F;
  hipDataType C_64F = HIP_C_64F;
  hipDataType R_8I = HIP_R_8I;
  hipDataType C_8I = HIP_C_8I;
  hipDataType R_8U = HIP_R_8U;
  hipDataType C_8U = HIP_C_8U;
  hipDataType R_32I = HIP_R_32I;
  hipDataType C_32I = HIP_C_32I;
  hipDataType R_32U = HIP_R_32U;
  hipDataType C_32U = HIP_C_32U;

  // CHECK: rocblas_datatype DataType_2, DataType_3;
  hipDataType DataType_2, DataType_3;

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasNrm2Ex(hipblasHandle_t handle, int n, const void* x, hipDataType xType, int incx, void* result, hipDataType resultType, hipDataType executionType);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_nrm2_ex(rocblas_handle handle, rocblas_int n, const void* x, rocblas_datatype x_type, rocblas_int incx, void* results, rocblas_datatype result_type, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_nrm2_ex(blasHandle, n, image, DataType, incx, image_2, DataType_2, DataType_3);
  blasStatus = hipblasNrm2Ex(blasHandle, n, image, DataType, incx, image_2, DataType_2, DataType_3);
#endif

#if CUDA_VERSION >= 9000
  // CHECK: rocblas_gemm_algo BLAS_GEMM_DEFAULT = rocblas_gemm_algo_standard;
  hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;
#endif

#if CUDA_VERSION >= 10010
  // CHECK: rocblas_operation BLAS_OP_HERMITAN = rocblas_operation_conjugate_transpose;
  hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;

  // CHECK: rocblas_fill BLAS_FILL_MODE_FULL = rocblas_fill_full;
  hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;
#endif

#if CUDA_VERSION >= 11000
  // CHECK: rocblas_datatype R_16BF = rocblas_datatype_bf16_r;
  // CHECK-NEXT: rocblas_datatype C_16BF = rocblas_datatype_bf16_c;
  hipDataType R_16BF = HIP_R_16BF;
  hipDataType C_16BF = HIP_C_16BF;
#endif

#if CUDA_VERSION >= 11040
  // CUDA: CUBLASAPI const char* CUBLASWINAPI cublasGetStatusString(hipblasStatus_t status);
  // ROC: ROCBLAS_EXPORT const char* rocblas_status_to_string(rocblas_status status);
  // CHECK: const_ch = rocblas_status_to_string(blasStatus);
  const_ch = cublasGetStatusString(blasStatus);
#endif

  return 0;
}
