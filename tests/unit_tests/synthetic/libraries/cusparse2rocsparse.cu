// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental --roc %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t, matDescr_t_2;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2;

  // CHECK: _rocsparse_color_info *colorInfo = nullptr;
  // CHECK-NEXT: rocsparse_color_info colorInfo_t;
  cusparseColorInfo *colorInfo = nullptr;
  hipsparseColorInfo_t colorInfo_t;

  // CHECK: rocsparse_operation sparseOperation_t;
  // CHECK-NEXT: rocsparse_operation OPERATION_NON_TRANSPOSE = rocsparse_operation_none;
  // CHECK-NEXT: rocsparse_operation OPERATION_TRANSPOSE = rocsparse_operation_transpose;
  // CHECK-NEXT: rocsparse_operation OPERATION_CONJUGATE_TRANSPOSE = rocsparse_operation_conjugate_transpose;
  hipsparseOperation_t sparseOperation_t;
  hipsparseOperation_t OPERATION_NON_TRANSPOSE = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipsparseOperation_t OPERATION_TRANSPOSE = HIPSPARSE_OPERATION_TRANSPOSE;
  hipsparseOperation_t OPERATION_CONJUGATE_TRANSPOSE = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;

  // CHECK: rocsparse_index_base indexBase_t;
  // CHECK-NEXT: rocsparse_index_base INDEX_BASE_ZERO = rocsparse_index_base_zero;
  // CHECK-NEXT: rocsparse_index_base INDEX_BASE_ONE = rocsparse_index_base_one;
  hipsparseIndexBase_t indexBase_t;
  hipsparseIndexBase_t INDEX_BASE_ZERO = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseIndexBase_t INDEX_BASE_ONE = HIPSPARSE_INDEX_BASE_ONE;

  // CHECK: rocsparse_matrix_type matrixType_t;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_GENERAL = rocsparse_matrix_type_general;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_SYMMETRIC = rocsparse_matrix_type_symmetric;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_HERMITIAN = rocsparse_matrix_type_hermitian;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_TRIANGULAR = rocsparse_matrix_type_triangular;
  hipsparseMatrixType_t matrixType_t;
  hipsparseMatrixType_t MATRIX_TYPE_GENERAL = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseMatrixType_t MATRIX_TYPE_SYMMETRIC = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  hipsparseMatrixType_t MATRIX_TYPE_HERMITIAN = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  hipsparseMatrixType_t MATRIX_TYPE_TRIANGULAR = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;

  // CHECK: rocsparse_diag_type diagType_t;
  // CHECK-NEXT: rocsparse_diag_type DIAG_TYPE_NON_UNIT = rocsparse_diag_type_non_unit;
  // CHECK-NEXT: rocsparse_diag_type DIAG_TYPE_UNIT = rocsparse_diag_type_unit;
  hipsparseDiagType_t diagType_t;
  hipsparseDiagType_t DIAG_TYPE_NON_UNIT = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseDiagType_t DIAG_TYPE_UNIT = HIPSPARSE_DIAG_TYPE_UNIT;

  // CHECK: rocsparse_fill_mode fillMode_t;
  // CHECK-NEXT: rocsparse_fill_mode FILL_MODE_LOWER = rocsparse_fill_mode_lower;
  // CHECK-NEXT: rocsparse_fill_mode FILL_MODE_UPPER = rocsparse_fill_mode_upper;
  hipsparseFillMode_t fillMode_t;
  hipsparseFillMode_t FILL_MODE_LOWER = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseFillMode_t FILL_MODE_UPPER = HIPSPARSE_FILL_MODE_UPPER;

  // CHECK: rocsparse_action action_t;
  // CHECK-NEXT: rocsparse_action ACTION_SYMBOLIC = rocsparse_action_symbolic;
  // CHECK-NEXT: rocsparse_action ACTION_NUMERIC = rocsparse_action_numeric;
  hipsparseAction_t action_t;
  hipsparseAction_t ACTION_SYMBOLIC = HIPSPARSE_ACTION_SYMBOLIC;
  hipsparseAction_t ACTION_NUMERIC = HIPSPARSE_ACTION_NUMERIC;

  // CHECK: rocsparse_direction direction_t;
  // CHECK-NEXT: rocsparse_direction DIRECTION_ROW = rocsparse_direction_row;
  // CHECK-NEXT: rocsparse_direction DIRECTION_COLUMN = rocsparse_direction_column;
  hipsparseDirection_t direction_t;
  hipsparseDirection_t DIRECTION_ROW = HIPSPARSE_DIRECTION_ROW;
  hipsparseDirection_t DIRECTION_COLUMN = HIPSPARSE_DIRECTION_COLUMN;

  // CHECK: rocsparse_solve_policy solvePolicy_t;
  // CHECK-NEXT: rocsparse_solve_policy SOLVE_POLICY_NO_LEVEL = rocsparse_solve_policy_auto;
  // CHECK-NEXT: rocsparse_solve_policy SOLVE_POLICY_USE_LEVEL = rocsparse_solve_policy_auto;
  hipsparseSolvePolicy_t solvePolicy_t;
  hipsparseSolvePolicy_t SOLVE_POLICY_NO_LEVEL = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
  hipsparseSolvePolicy_t SOLVE_POLICY_USE_LEVEL = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

  // CHECK: rocsparse_pointer_mode pointerMode_t;
  // CHECK-NEXT: rocsparse_pointer_mode POINTER_MODE_HOST = rocsparse_pointer_mode_host;
  // CHECK-NEXT: rocsparse_pointer_mode POINTER_MODE_DEVICE = rocsparse_pointer_mode_device;
  hipsparsePointerMode_t pointerMode_t;
  hipsparsePointerMode_t POINTER_MODE_HOST = HIPSPARSE_POINTER_MODE_HOST;
  hipsparsePointerMode_t POINTER_MODE_DEVICE = HIPSPARSE_POINTER_MODE_DEVICE;

  // CHECK: rocsparse_status status_t;
  // CHECK-NEXT: rocsparse_status STATUS_SUCCESS = rocsparse_status_success;
  // CHECK-NEXT: rocsparse_status STATUS_NOT_INITIALIZED = rocsparse_status_not_initialized;
  // CHECK-NEXT: rocsparse_status STATUS_ALLOC_FAILED = rocsparse_status_memory_error;
  // CHECK-NEXT: rocsparse_status STATUS_INVALID_VALUE = rocsparse_status_invalid_value;
  // CHECK-NEXT: rocsparse_status STATUS_ARCH_MISMATCH = rocsparse_status_arch_mismatch;
  // CHECK-NEXT: rocsparse_status STATUS_INTERNAL_ERROR = rocsparse_status_internal_error;
  // CHECK-NEXT: rocsparse_status STATUS_ZERO_PIVOT = rocsparse_status_zero_pivot;
  hipsparseStatus_t status_t;
  hipsparseStatus_t STATUS_SUCCESS = HIPSPARSE_STATUS_SUCCESS;
  hipsparseStatus_t STATUS_NOT_INITIALIZED = HIPSPARSE_STATUS_NOT_INITIALIZED;
  hipsparseStatus_t STATUS_ALLOC_FAILED = HIPSPARSE_STATUS_ALLOC_FAILED;
  hipsparseStatus_t STATUS_INVALID_VALUE = HIPSPARSE_STATUS_INVALID_VALUE;
  hipsparseStatus_t STATUS_ARCH_MISMATCH = HIPSPARSE_STATUS_ARCH_MISMATCH;
  hipsparseStatus_t STATUS_INTERNAL_ERROR = HIPSPARSE_STATUS_INTERNAL_ERROR;
  hipsparseStatus_t STATUS_ZERO_PIVOT = HIPSPARSE_STATUS_ZERO_PIVOT;

  // CHECK: hipStream_t stream_t;
  hipStream_t stream_t;

  int iVal = 0;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreate(cusparseHandle_t* handle);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_handle(rocsparse_handle* handle);
  // CHECK: status_t = rocsparse_create_handle(&handle_t);
  status_t = hipsparseCreate(&handle_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroy(cusparseHandle_t handle);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_destroy_handle(rocsparse_handle handle);
  // CHECK: status_t = rocsparse_destroy_handle(handle_t);
  status_t = hipsparseDestroy(handle_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetStream(cusparseHandle_t handle, cudaStream_t streamId);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_set_stream(rocsparse_handle handle, hipStream_t stream);
  // CHECK: status_t = rocsparse_set_stream(handle_t, stream_t);
  status_t = hipsparseSetStream(handle_t, stream_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetStream(cusparseHandle_t handle, cudaStream_t* streamId);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_get_stream(rocsparse_handle handle, hipStream_t* stream);
  // CHECK: status_t = rocsparse_get_stream(handle_t, &stream_t);
  status_t = hipsparseGetStream(handle_t, &stream_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetPointerMode(cusparseHandle_t handle, cusparsePointerMode_t mode);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_set_pointer_mode(rocsparse_handle handle, rocsparse_pointer_mode pointer_mode);
  // CHECK: status_t = rocsparse_set_pointer_mode(handle_t, pointerMode_t);
  status_t = hipsparseSetPointerMode(handle_t, pointerMode_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetPointerMode(cusparseHandle_t handle, cusparsePointerMode_t* mode);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_get_pointer_mode(rocsparse_handle handle, rocsparse_pointer_mode* pointer_mode);
  // CHECK: status_t = rocsparse_get_pointer_mode(handle_t, &pointerMode_t);
  status_t = hipsparseGetPointerMode(handle_t, &pointerMode_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetVersion(cusparseHandle_t handle, int* version);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_get_version(rocsparse_handle handle, int* version);
  // CHECK: status_t = rocsparse_get_version(handle_t, &iVal);
  status_t = hipsparseGetVersion(handle_t, &iVal);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateMatDescr(cusparseMatDescr_t* descrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_mat_descr(rocsparse_mat_descr* descr);
  // CHECK: status_t = rocsparse_create_mat_descr(&matDescr_t);
  status_t = hipsparseCreateMatDescr(&matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyMatDescr(cusparseMatDescr_t descrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_destroy_mat_descr(rocsparse_mat_descr descr);
  // CHECK: status_t = rocsparse_destroy_mat_descr(matDescr_t);
  status_t = hipsparseDestroyMatDescr(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatIndexBase(cusparseMatDescr_t descrA, cusparseIndexBase_t base);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_set_mat_index_base(rocsparse_mat_descr descr, rocsparse_index_base base);
  // CHECK: status_t = rocsparse_set_mat_index_base(matDescr_t, indexBase_t);
  status_t = hipsparseSetMatIndexBase(matDescr_t, indexBase_t);

  // CUDA: cusparseIndexBase_t CUSPARSEAPI cusparseGetMatIndexBase(const cusparseMatDescr_t descrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_index_base rocsparse_get_mat_index_base(const rocsparse_mat_descr descr);
  // CHECK: indexBase_t = rocsparse_get_mat_index_base(matDescr_t);
  indexBase_t = hipsparseGetMatIndexBase(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatType(cusparseMatDescr_t descrA, cusparseMatrixType_t type);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_set_mat_type(rocsparse_mat_descr descr, rocsparse_matrix_type type);
  // CHECK: status_t = rocsparse_set_mat_type(matDescr_t, matrixType_t);
  status_t = hipsparseSetMatType(matDescr_t, matrixType_t);

  // CUDA: cusparseMatrixType_t CUSPARSEAPI cusparseGetMatType(const cusparseMatDescr_t descrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_matrix_type rocsparse_get_mat_type(const rocsparse_mat_descr descr);
  // CHECK: matrixType_t = rocsparse_get_mat_type(matDescr_t);
  matrixType_t = hipsparseGetMatType(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatFillMode(cusparseMatDescr_t descrA, cusparseFillMode_t fillMode);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_set_mat_fill_mode(rocsparse_mat_descr descr, rocsparse_fill_mode fill_mode);
  // CHECK: status_t = rocsparse_set_mat_fill_mode(matDescr_t, fillMode_t);
  status_t = hipsparseSetMatFillMode(matDescr_t, fillMode_t);

  // CUDA: cusparseFillMode_t CUSPARSEAPI cusparseGetMatFillMode(const cusparseMatDescr_t descrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_fill_mode rocsparse_get_mat_fill_mode(const rocsparse_mat_descr descr);
  // CHECK: fillMode_t = rocsparse_get_mat_fill_mode(matDescr_t);
  fillMode_t = hipsparseGetMatFillMode(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatDiagType(cusparseMatDescr_t descrA, cusparseDiagType_t diagType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_set_mat_diag_type(rocsparse_mat_descr descr, rocsparse_diag_type diag_type);
  // CHECK: status_t = rocsparse_set_mat_diag_type(matDescr_t, diagType_t);
  status_t = hipsparseSetMatDiagType(matDescr_t, diagType_t);

  // CUDA: cusparseDiagType_t CUSPARSEAPI cusparseGetMatDiagType(const cusparseMatDescr_t descrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_diag_type rocsparse_get_mat_diag_type(const rocsparse_mat_descr descr);
  // CHECK: diagType_t = rocsparse_get_mat_diag_type(matDescr_t);
  diagType_t = hipsparseGetMatDiagType(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateColorInfo(cusparseColorInfo_t* info);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_color_info(rocsparse_color_info* info);
  // CHECK: status_t = rocsparse_create_color_info(&colorInfo_t);
  status_t = hipsparseCreateColorInfo(&colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyColorInfo(cusparseColorInfo_t info);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_destroy_color_info(rocsparse_color_info info);
  // CHECK: status_t = rocsparse_destroy_color_info(colorInfo_t);
  status_t = hipsparseDestroyColorInfo(colorInfo_t);

#if CUDA_VERSION >= 8000 && CUDA_VERSION < 12000
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCopyMatDescr(cusparseMatDescr_t dest, const cusparseMatDescr_t src);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_copy_mat_descr(rocsparse_mat_descr dest, const rocsparse_mat_descr src);
  // CHECK: status_t = rocsparse_copy_mat_descr(matDescr_t, matDescr_t_2);
  status_t = cusparseCopyMatDescr(matDescr_t, matDescr_t_2);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: _rocsparse_spmat_descr *spMatDescr = nullptr;
  // CHECK-NEXT: rocsparse_spmat_descr spMatDescr_t;
  cusparseSpMatDescr *spMatDescr = nullptr;
  hipsparseSpMatDescr_t spMatDescr_t;

  // CHECK: _rocsparse_dnmat_descr *dnMatDescr = nullptr;
  // CHECK-NEXT: rocsparse_dnmat_descr dnMatDescr_t;
  cusparseDnMatDescr *dnMatDescr = nullptr;
  hipsparseDnMatDescr_t dnMatDescr_t;

  // CHECK: rocsparse_indextype indexType_t;
  // CHECK-NEXT: rocsparse_indextype INDEX_16U = rocsparse_indextype_u16;
  // CHECK-NEXT: rocsparse_indextype INDEX_32I = rocsparse_indextype_i32;
  // CHECK-NEXT: rocsparse_indextype INDEX_64I = rocsparse_indextype_i64;
  hipsparseIndexType_t indexType_t;
  hipsparseIndexType_t INDEX_16U = HIPSPARSE_INDEX_16U;
  hipsparseIndexType_t INDEX_32I = HIPSPARSE_INDEX_32I;
  hipsparseIndexType_t INDEX_64I = HIPSPARSE_INDEX_64I;

  // CHECK: rocsparse_format format_t;
  // CHECK-NEXT: rocsparse_format FORMAT_CSR = rocsparse_format_csr;
  // CHECK-NEXT: rocsparse_format FORMAT_CSC = rocsparse_format_csc;
  // CHECK-NEXT: rocsparse_format FORMAT_CSO = rocsparse_format_coo;
  hipsparseFormat_t format_t;
  hipsparseFormat_t FORMAT_CSR = HIPSPARSE_FORMAT_CSR;
  hipsparseFormat_t FORMAT_CSC = HIPSPARSE_FORMAT_CSC;
  hipsparseFormat_t FORMAT_CSO = HIPSPARSE_FORMAT_COO;

  // CHECK: rocsparse_order order_t;
  // CHECK-NEXT: rocsparse_order ORDER_COL = rocsparse_order_row;
  // CHECK-NEXT: rocsparse_order ORDER_ROW = rocsparse_order_column;
  hipsparseOrder_t order_t;
  hipsparseOrder_t ORDER_COL = HIPSPARSE_ORDER_COL;
  hipsparseOrder_t ORDER_ROW = HIPSPARSE_ORDER_ROW;

  // CHECK: rocsparse_spmm_alg spMMAlg_t;
  hipsparseSpMMAlg_t spMMAlg_t;
#endif

#if CUDA_VERSION >= 10020
  // CHECK: _rocsparse_spvec_descr *spVecDescr = nullptr;
  // CHECK-NEXT: rocsparse_spvec_descr spVecDescr_t;
  cusparseSpVecDescr *spVecDescr = nullptr;
  hipsparseSpVecDescr_t spVecDescr_t;

  // CHECK: _rocsparse_dnvec_descr *dnVecDescr = nullptr;
  // CHECK-NEXT: rocsparse_dnvec_descr dnVecDescr_t;
  cusparseDnVecDescr *dnVecDescr = nullptr;
  hipsparseDnVecDescr_t dnVecDescr_t;

  // CHECK: rocsparse_status STATUS_NOT_SUPPORTED = rocsparse_status_not_implemented;
  hipsparseStatus_t STATUS_NOT_SUPPORTED = HIPSPARSE_STATUS_NOT_SUPPORTED;

  // CHECK: rocsparse_spmv_alg spMVAlg_t;
  hipsparseSpMVAlg_t spMVAlg_t;
#endif

#if CUDA_VERSION >= 10020 && CUDA_VERSION < 12000
  // CHECK: rocsparse_format FORMAT_COO_AOS = rocsparse_format_coo_aos;
  cusparseFormat_t FORMAT_COO_AOS = CUSPARSE_FORMAT_COO_AOS;
#endif

#if CUDA_VERSION < 11000
  // CHECK: _rocsparse_hyb_mat *hybMat = nullptr;
  // CHECK-NEXT: rocsparse_hyb_mat hybMat_t;
  cusparseHybMat *hybMat = nullptr;
  cusparseHybMat_t hybMat_t;

  // CHECK: rocsparse_hyb_partition hybPartition_t;
  // CHECK-NEXT: rocsparse_hyb_partition HYB_PARTITION_AUTO = rocsparse_hyb_partition_auto;
  // CHECK-NEXT: rocsparse_hyb_partition HYB_PARTITION_USER = rocsparse_hyb_partition_user;
  // CHECK-NEXT: rocsparse_hyb_partition HYB_PARTITION_MAX = rocsparse_hyb_partition_max;
  cusparseHybPartition_t hybPartition_t;
  cusparseHybPartition_t HYB_PARTITION_AUTO = CUSPARSE_HYB_PARTITION_AUTO;
  cusparseHybPartition_t HYB_PARTITION_USER = CUSPARSE_HYB_PARTITION_USER;
  cusparseHybPartition_t HYB_PARTITION_MAX = CUSPARSE_HYB_PARTITION_MAX;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateHybMat(cusparseHybMat_t* hybA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_hyb_mat(rocsparse_hyb_mat* hyb);
  // CHECK: status_t = rocsparse_create_hyb_mat(&hybMat_t);
  status_t = cusparseCreateHybMat(&hybMat_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyHybMat(cusparseHybMat_t hybA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_destroy_hyb_mat(rocsparse_hyb_mat hyb);
  // CHECK: status_t = rocsparse_destroy_hyb_mat(hybMat_t);
  status_t = cusparseDestroyHybMat(hybMat_t);
#endif

#if CUDA_VERSION >= 11000
  // CHECK: rocsparse_spmm_alg SPMM_ALG_DEFAULT = rocsparse_spmm_alg_default;
  // CHECK-NEXT: rocsparse_spmm_alg SPMM_COO_ALG1 = rocsparse_spmm_alg_coo_segmented;
  // CHECK-NEXT: rocsparse_spmm_alg SPMM_COO_ALG2 = rocsparse_spmm_alg_coo_atomic;
  // CHECK-NEXT: rocsparse_spmm_alg SPMM_COO_ALG3 = rocsparse_spmm_alg_coo_segmented_atomic;
  // CHECK-NEXT: rocsparse_spmm_alg SPMM_CSR_ALG1 = rocsparse_spmm_alg_csr;
  // CHECK-NEXT: rocsparse_spmm_alg SPMM_CSR_ALG2 = rocsparse_spmm_alg_csr_row_split;
  hipsparseSpMMAlg_t SPMM_ALG_DEFAULT = HIPSPARSE_SPMM_ALG_DEFAULT;
  hipsparseSpMMAlg_t SPMM_COO_ALG1 = HIPSPARSE_SPMM_COO_ALG1;
  hipsparseSpMMAlg_t SPMM_COO_ALG2 = HIPSPARSE_SPMM_COO_ALG2;
  hipsparseSpMMAlg_t SPMM_COO_ALG3 = HIPSPARSE_SPMM_COO_ALG3;
  hipsparseSpMMAlg_t SPMM_CSR_ALG1 = HIPSPARSE_SPMM_CSR_ALG1;
  hipsparseSpMMAlg_t SPMM_CSR_ALG2 = HIPSPARSE_SPMM_CSR_ALG2;

  // CHECK: rocsparse_spgemm_alg spGEMMAlg_t;
  // CHECK-NEXT: rocsparse_spgemm_alg SPGEMM_DEFAULT = rocsparse_spgemm_alg_default;
  hipsparseSpGEMMAlg_t spGEMMAlg_t;
  hipsparseSpGEMMAlg_t SPGEMM_DEFAULT = HIPSPARSE_SPGEMM_DEFAULT;
#endif

#if CUDA_VERSION >= 11010
  // CHECK: rocsparse_sparse_to_dense_alg sparseToDenseAlg_t;
  // CHECK-NEXT: rocsparse_sparse_to_dense_alg SPARSETODENSE_ALG_DEFAULT = rocsparse_sparse_to_dense_alg_default;
  hipsparseSparseToDenseAlg_t sparseToDenseAlg_t;
  hipsparseSparseToDenseAlg_t SPARSETODENSE_ALG_DEFAULT = HIPSPARSE_SPARSETODENSE_ALG_DEFAULT;

  // CHECK: rocsparse_dense_to_sparse_alg denseToSparseAlg_t;
  // CHECK-NEXT: rocsparse_dense_to_sparse_alg DENSETOSPARSE_ALG_DEFAULT = rocsparse_dense_to_sparse_alg_default;
  hipsparseDenseToSparseAlg_t denseToSparseAlg_t;
  hipsparseDenseToSparseAlg_t DENSETOSPARSE_ALG_DEFAULT = HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT;
#endif

#if CUDA_VERSION >= 11020
  // CHECK: rocsparse_format FORMAT_BLOCKED_ELL = rocsparse_format_bell;
  hipsparseFormat_t FORMAT_BLOCKED_ELL = HIPSPARSE_FORMAT_BLOCKED_ELL;

  // CHECK: rocsparse_spmv_alg SPMV_ALG_DEFAULT = rocsparse_spmv_alg_default;
  // CHECK-NEXT: rocsparse_spmv_alg SPMV_COO_ALG1 = rocsparse_spmv_alg_coo;
  // CHECK-NEXT: rocsparse_spmv_alg SPMV_COO_ALG2 = rocsparse_spmv_alg_coo_atomic;
  // CHECK-NEXT: rocsparse_spmv_alg SPMV_CSR_ALG1 = rocsparse_spmv_alg_csr_adaptive;
  // CHECK-NEXT: rocsparse_spmv_alg SPMV_CSR_ALG2 = rocsparse_spmv_alg_csr_stream;
  hipsparseSpMVAlg_t SPMV_ALG_DEFAULT = HIPSPARSE_SPMV_ALG_DEFAULT;
  hipsparseSpMVAlg_t SPMV_COO_ALG1 = HIPSPARSE_SPMV_COO_ALG1;
  hipsparseSpMVAlg_t SPMV_COO_ALG2 = HIPSPARSE_SPMV_COO_ALG2;
  hipsparseSpMVAlg_t SPMV_CSR_ALG1 = HIPSPARSE_SPMV_CSR_ALG1;
  hipsparseSpMVAlg_t SPMV_CSR_ALG2 = HIPSPARSE_SPMV_CSR_ALG2;

  // CHECK: rocsparse_spmm_alg SPMM_CSR_ALG3 = rocsparse_spmm_alg_csr_merge;
  // CHECK-NEXT: rocsparse_spmm_alg SPMM_BLOCKED_ELL_ALG1 = rocsparse_spmm_alg_bell;
  hipsparseSpMMAlg_t SPMM_CSR_ALG3 = HIPSPARSE_SPMM_CSR_ALG3;
  hipsparseSpMMAlg_t SPMM_BLOCKED_ELL_ALG1 = HIPSPARSE_SPMM_BLOCKED_ELL_ALG1;

  // CHECK: rocsparse_sddmm_alg sDDMMAlg_t;
  // CHECK-NEXT: rocsparse_sddmm_alg SDDMM_ALG_DEFAULT = rocsparse_sddmm_alg_default;
  hipsparseSDDMMAlg_t sDDMMAlg_t;
  hipsparseSDDMMAlg_t SDDMM_ALG_DEFAULT = HIPSPARSE_SDDMM_ALG_DEFAULT;
#endif

#if CUDA_VERSION >= 11030
  // CHECK: rocsparse_spmat_attribute spMatAttribute_t;
  // CHECK-NEXT: rocsparse_spmat_attribute SPMAT_FILL_MODE = rocsparse_spmat_fill_mode;
  // CHECK-NEXT: rocsparse_spmat_attribute SPMAT_DIAG_TYPE = rocsparse_spmat_diag_type;
  hipsparseSpMatAttribute_t spMatAttribute_t;
  hipsparseSpMatAttribute_t SPMAT_FILL_MODE = HIPSPARSE_SPMAT_FILL_MODE;
  hipsparseSpMatAttribute_t SPMAT_DIAG_TYPE = HIPSPARSE_SPMAT_DIAG_TYPE;

  // CHECK: rocsparse_spsv_alg spSVAlg_t;
  // CHECK-NEXT: rocsparse_spsv_alg SPSV_ALG_DEFAULT = rocsparse_spsv_alg_default;
  hipsparseSpSVAlg_t spSVAlg_t;
  hipsparseSpSVAlg_t SPSV_ALG_DEFAULT = HIPSPARSE_SPSV_ALG_DEFAULT;

  // CHECK: rocsparse_spsm_alg spSMAlg_t;
  // CHECK-NEXT: rocsparse_spsm_alg SPSM_ALG_DEFAULT = rocsparse_spsm_alg_default;
  hipsparseSpSMAlg_t spSMAlg_t;
  hipsparseSpSMAlg_t SPSM_ALG_DEFAULT = HIPSPARSE_SPSM_ALG_DEFAULT;
#endif

#if CUDA_VERSION >= 12010
  // CHECK: rocsparse_format FORMAT_BSR = rocsparse_format_bsr;
  // CHECK-NEXT: rocsparse_format FORMAT_SLICED_ELLPACK = rocsparse_format_ell;
  hipsparseFormat_t FORMAT_BSR = CUSPARSE_FORMAT_BSR;
  hipsparseFormat_t FORMAT_SLICED_ELLPACK = CUSPARSE_FORMAT_SLICED_ELLPACK;
#endif

#if CUDA_VERSION >= 12010 && CUSPARSE_VERSION >= 12100
  // CHECK: rocsparse_spmv_alg SPMV_SELL_ALG1 = rocsparse_spmv_alg_ell;
  hipsparseSpMVAlg_t SPMV_SELL_ALG1 = CUSPARSE_SPMV_SELL_ALG1;
#endif

  return 0;
}
