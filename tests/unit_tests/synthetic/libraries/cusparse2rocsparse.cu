// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental --roc %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t;

  // CHECK: _rocsparse_color_info *colorInfo = nullptr;
  // CHECK-NEXT: rocsparse_color_info colorInfo_t;
  cusparseColorInfo *colorInfo = nullptr;
  hipsparseColorInfo_t colorInfo_t;

  // CHECK: rocsparse_operation sparseOperation_t;
  // CHECK-NEXT: rocsparse_operation OPERATION_NON_TRANSPOSE = rocsparse_operation_none;
  // CHECK-NEXT: rocsparse_operation OPERATION_TRANSPOSE = rocsparse_operation_transpose;
  // CHECK-NEXT: rocsparse_operation OPERATION_CONJUGATE_TRANSPOSE = rocsparse_operation_conjugate_transpose;
  hipsparseOperation_t sparseOperation_t;
  hipsparseOperation_t OPERATION_NON_TRANSPOSE = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipsparseOperation_t OPERATION_TRANSPOSE = HIPSPARSE_OPERATION_TRANSPOSE;
  hipsparseOperation_t OPERATION_CONJUGATE_TRANSPOSE = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;

  // CHECK: rocsparse_index_base indexBase_t;
  // CHECK-NEXT: rocsparse_index_base INDEX_BASE_ZERO = rocsparse_index_base_zero;
  // CHECK-NEXT: rocsparse_index_base INDEX_BASE_ONE = rocsparse_index_base_one;
  hipsparseIndexBase_t indexBase_t;
  hipsparseIndexBase_t INDEX_BASE_ZERO = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseIndexBase_t INDEX_BASE_ONE = HIPSPARSE_INDEX_BASE_ONE;

  // CHECK: rocsparse_matrix_type matrixType_t;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_GENERAL = rocsparse_matrix_type_general;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_SYMMETRIC = rocsparse_matrix_type_symmetric;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_HERMITIAN = rocsparse_matrix_type_hermitian;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_TRIANGULAR = rocsparse_matrix_type_triangular;
  hipsparseMatrixType_t matrixType_t;
  hipsparseMatrixType_t MATRIX_TYPE_GENERAL = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseMatrixType_t MATRIX_TYPE_SYMMETRIC = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  hipsparseMatrixType_t MATRIX_TYPE_HERMITIAN = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  hipsparseMatrixType_t MATRIX_TYPE_TRIANGULAR = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;

  // CHECK: rocsparse_diag_type diagType_t;
  // CHECK-NEXT: rocsparse_diag_type DIAG_TYPE_NON_UNIT = rocsparse_diag_type_non_unit;
  // CHECK-NEXT: rocsparse_diag_type DIAG_TYPE_UNIT = rocsparse_diag_type_unit;
  hipsparseDiagType_t diagType_t;
  hipsparseDiagType_t DIAG_TYPE_NON_UNIT = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseDiagType_t DIAG_TYPE_UNIT = HIPSPARSE_DIAG_TYPE_UNIT;

  // CHECK: rocsparse_fill_mode fillMode_t;
  // CHECK-NEXT: rocsparse_fill_mode FILL_MODE_LOWER = rocsparse_fill_mode_lower;
  // CHECK-NEXT: rocsparse_fill_mode FILL_MODE_UPPER = rocsparse_fill_mode_upper;
  hipsparseFillMode_t fillMode_t;
  hipsparseFillMode_t FILL_MODE_LOWER = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseFillMode_t FILL_MODE_UPPER = HIPSPARSE_FILL_MODE_UPPER;

  // CHECK: rocsparse_action action_t;
  // CHECK-NEXT: rocsparse_action ACTION_SYMBOLIC = rocsparse_action_symbolic;
  // CHECK-NEXT: rocsparse_action ACTION_NUMERIC = rocsparse_action_numeric;
  hipsparseAction_t action_t;
  hipsparseAction_t ACTION_SYMBOLIC = HIPSPARSE_ACTION_SYMBOLIC;
  hipsparseAction_t ACTION_NUMERIC = HIPSPARSE_ACTION_NUMERIC;

  // CHECK: rocsparse_direction direction_t;
  // CHECK-NEXT: rocsparse_direction DIRECTION_ROW = rocsparse_direction_row;
  // CHECK-NEXT: rocsparse_direction DIRECTION_COLUMN = rocsparse_direction_column;
  hipsparseDirection_t direction_t;
  hipsparseDirection_t DIRECTION_ROW = HIPSPARSE_DIRECTION_ROW;
  hipsparseDirection_t DIRECTION_COLUMN = HIPSPARSE_DIRECTION_COLUMN;

  // CHECK: rocsparse_solve_policy solvePolicy_t;
  // CHECK-NEXT: rocsparse_solve_policy SOLVE_POLICY_NO_LEVEL = rocsparse_solve_policy_auto;
  // CHECK-NEXT: rocsparse_solve_policy SOLVE_POLICY_USE_LEVEL = rocsparse_solve_policy_auto;
  hipsparseSolvePolicy_t solvePolicy_t;
  hipsparseSolvePolicy_t SOLVE_POLICY_NO_LEVEL = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
  hipsparseSolvePolicy_t SOLVE_POLICY_USE_LEVEL = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

  // CHECK: rocsparse_pointer_mode pointerMode_t;
  // CHECK-NEXT: rocsparse_pointer_mode POINTER_MODE_HOST = rocsparse_pointer_mode_host;
  // CHECK-NEXT: rocsparse_pointer_mode POINTER_MODE_DEVICE = rocsparse_pointer_mode_device;
  hipsparsePointerMode_t pointerMode_t;
  hipsparsePointerMode_t POINTER_MODE_HOST = HIPSPARSE_POINTER_MODE_HOST;
  hipsparsePointerMode_t POINTER_MODE_DEVICE = HIPSPARSE_POINTER_MODE_DEVICE;

  // CHECK: rocsparse_status status_t;
  // CHECK-NEXT: rocsparse_status STATUS_SUCCESS = rocsparse_status_success;
  // CHECK-NEXT: rocsparse_status STATUS_NOT_INITIALIZED = rocsparse_status_not_initialized;
  // CHECK-NEXT: rocsparse_status STATUS_ALLOC_FAILED = rocsparse_status_memory_error;
  // CHECK-NEXT: rocsparse_status STATUS_INVALID_VALUE = rocsparse_status_invalid_value;
  // CHECK-NEXT: rocsparse_status STATUS_ARCH_MISMATCH = rocsparse_status_arch_mismatch;
  // CHECK-NEXT: rocsparse_status STATUS_INTERNAL_ERROR = rocsparse_status_internal_error;
  // CHECK-NEXT: rocsparse_status STATUS_ZERO_PIVOT = rocsparse_status_zero_pivot;
  hipsparseStatus_t status_t;
  hipsparseStatus_t STATUS_SUCCESS = HIPSPARSE_STATUS_SUCCESS;
  hipsparseStatus_t STATUS_NOT_INITIALIZED = HIPSPARSE_STATUS_NOT_INITIALIZED;
  hipsparseStatus_t STATUS_ALLOC_FAILED = HIPSPARSE_STATUS_ALLOC_FAILED;
  hipsparseStatus_t STATUS_INVALID_VALUE = HIPSPARSE_STATUS_INVALID_VALUE;
  hipsparseStatus_t STATUS_ARCH_MISMATCH = HIPSPARSE_STATUS_ARCH_MISMATCH;
  hipsparseStatus_t STATUS_INTERNAL_ERROR = HIPSPARSE_STATUS_INTERNAL_ERROR;
  hipsparseStatus_t STATUS_ZERO_PIVOT = HIPSPARSE_STATUS_ZERO_PIVOT;

#if CUDA_VERSION >= 10010
  // CHECK: _rocsparse_spmat_descr *spMatDescr = nullptr;
  // CHECK-NEXT: rocsparse_spmat_descr spMatDescr_t;
  cusparseSpMatDescr *spMatDescr = nullptr;
  hipsparseSpMatDescr_t spMatDescr_t;

  // CHECK: _rocsparse_dnmat_descr *dnMatDescr = nullptr;
  // CHECK-NEXT: rocsparse_dnmat_descr dnMatDescr_t;
  cusparseDnMatDescr *dnMatDescr = nullptr;
  hipsparseDnMatDescr_t dnMatDescr_t;

  // CHECK: rocsparse_indextype indexType_t;
  // CHECK-NEXT: rocsparse_indextype INDEX_16U = rocsparse_indextype_u16;
  // CHECK-NEXT: rocsparse_indextype INDEX_32I = rocsparse_indextype_i32;
  // CHECK-NEXT: rocsparse_indextype INDEX_64I = rocsparse_indextype_i64;
  hipsparseIndexType_t indexType_t;
  hipsparseIndexType_t INDEX_16U = HIPSPARSE_INDEX_16U;
  hipsparseIndexType_t INDEX_32I = HIPSPARSE_INDEX_32I;
  hipsparseIndexType_t INDEX_64I = HIPSPARSE_INDEX_64I;

  // CHECK: rocsparse_format format_t;
  // CHECK-NEXT: rocsparse_format FORMAT_CSR = rocsparse_format_csr;
  // CHECK-NEXT: rocsparse_format FORMAT_CSC = rocsparse_format_csc;
  // CHECK-NEXT: rocsparse_format FORMAT_CSO = rocsparse_format_coo;
  hipsparseFormat_t format_t;
  hipsparseFormat_t FORMAT_CSR = HIPSPARSE_FORMAT_CSR;
  hipsparseFormat_t FORMAT_CSC = HIPSPARSE_FORMAT_CSC;
  hipsparseFormat_t FORMAT_CSO = HIPSPARSE_FORMAT_COO;
#endif

#if CUDA_VERSION >= 10020
  // CHECK: _rocsparse_spvec_descr *spVecDescr = nullptr;
  // CHECK-NEXT: rocsparse_spvec_descr spVecDescr_t;
  cusparseSpVecDescr *spVecDescr = nullptr;
  hipsparseSpVecDescr_t spVecDescr_t;

  // CHECK: _rocsparse_dnvec_descr *dnVecDescr = nullptr;
  // CHECK-NEXT: rocsparse_dnvec_descr dnVecDescr_t;
  cusparseDnVecDescr *dnVecDescr = nullptr;
  hipsparseDnVecDescr_t dnVecDescr_t;

  // CHECK: rocsparse_status STATUS_NOT_SUPPORTED = rocsparse_status_not_implemented;
  hipsparseStatus_t STATUS_NOT_SUPPORTED = HIPSPARSE_STATUS_NOT_SUPPORTED;
#endif

#if CUDA_VERSION >= 10020 && CUDA_VERSION < 12000
  // CHECK: rocsparse_format FORMAT_COO_AOS = rocsparse_format_coo_aos;
  cusparseFormat_t FORMAT_COO_AOS = CUSPARSE_FORMAT_COO_AOS;
#endif

#if CUDA_VERSION < 11000
  // CHECK: _rocsparse_hyb_mat *hybMat = nullptr;
  // CHECK-NEXT: rocsparse_hyb_mat hybMat_t;
  cusparseHybMat *hybMat = nullptr;
  cusparseHybMat_t hybMat_t;

  // CHECK: rocsparse_hyb_partition hybPartition_t;
  // CHECK-NEXT: rocsparse_hyb_partition HYB_PARTITION_AUTO = rocsparse_hyb_partition_auto;
  // CHECK-NEXT: rocsparse_hyb_partition HYB_PARTITION_USER = rocsparse_hyb_partition_user;
  // CHECK-NEXT: rocsparse_hyb_partition HYB_PARTITION_MAX = rocsparse_hyb_partition_max;
  cusparseHybPartition_t hybPartition_t;
  cusparseHybPartition_t HYB_PARTITION_AUTO = CUSPARSE_HYB_PARTITION_AUTO;
  cusparseHybPartition_t HYB_PARTITION_USER = CUSPARSE_HYB_PARTITION_USER;
  cusparseHybPartition_t HYB_PARTITION_MAX = CUSPARSE_HYB_PARTITION_MAX;
#endif

#if CUDA_VERSION >= 11020
  // CHECK: rocsparse_format FORMAT_BLOCKED_ELL = rocsparse_format_bell;
  hipsparseFormat_t FORMAT_BLOCKED_ELL = HIPSPARSE_FORMAT_BLOCKED_ELL;
#endif

#if CUDA_VERSION >= 12010
  // CHECK: rocsparse_format FORMAT_BSR = rocsparse_format_bsr;
  // CHECK-NEXT: rocsparse_format FORMAT_SLICED_ELLPACK = rocsparse_format_ell;
  hipsparseFormat_t FORMAT_BSR = CUSPARSE_FORMAT_BSR;
  hipsparseFormat_t FORMAT_SLICED_ELLPACK = CUSPARSE_FORMAT_SLICED_ELLPACK;
#endif

  return 0;
}
