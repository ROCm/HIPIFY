// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipblas.h"
// CHECK-NOT: #include "cublas_v2.h"
#include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"

#if defined(_WIN32) && CUDA_VERSION < 9000
  typedef signed   __int64 int64_t;
  typedef unsigned __int64 uint64_t;
#endif

int main() {
  printf("14.v2. cuBLAS API to hipBLAS API synthetic test\n");

  // CHECK: hipblasOperation_t blasOperation;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;
  hipblasOperation_t blasOperation;
  hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;

  // CHECK: hipblasStatus_t blasStatus;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;
  hipblasStatus_t blasStatus;
  hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;

  // CHECK: hipblasFillMode_t blasFillMode;
  // CHECK-NEXT: hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  // CHECK-NEXT: hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;
  hipblasFillMode_t blasFillMode;
  hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;

  // CHECK: hipblasDiagType_t blasDiagType;
  // CHECK-NEXT: hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  // CHECK-NEXT: hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;
  hipblasDiagType_t blasDiagType;
  hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;

  // CHECK: hipblasSideMode_t blasSideMode;
  // CHECK-NEXT: hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  // CHECK-NEXT: hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;
  hipblasSideMode_t blasSideMode;
  hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;

  // CHECK: hipblasPointerMode_t blasPointerMode;
  // CHECK-NEXT: hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  // CHECK-NEXT: hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;
  hipblasPointerMode_t blasPointerMode;
  hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;

  // CHECK: hipblasAtomicsMode_t blasAtomicsMode;
  // CHECK-NEXT: hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  // CHECK-NEXT: hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;
  hipblasAtomicsMode_t blasAtomicsMode;
  hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;

  // CHECK: hipblasHandle_t blasHandle;
  hipblasHandle_t blasHandle;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetAtomicsMode(cublasHandle_t handle, cublasAtomicsMode_t* mode);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t* atomics_mode);
  // CHECK: blasStatus = hipblasGetAtomicsMode(blasHandle, &blasAtomicsMode);
  blasStatus = hipblasGetAtomicsMode(blasHandle, &blasAtomicsMode);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetAtomicsMode(cublasHandle_t handle, cublasAtomicsMode_t mode);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t atomics_mode);
  // CHECK: blasStatus = hipblasSetAtomicsMode(blasHandle, blasAtomicsMode);
  blasStatus = hipblasSetAtomicsMode(blasHandle, blasAtomicsMode);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCreate_v2(cublasHandle_t* handle);
  // CUDA: #define cublasCreate cublasCreate_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCreate(hipblasHandle_t* handle);
  // CHECK: blasStatus = hipblasCreate(&blasHandle);
  // CHECK-NEXT: blasStatus = hipblasCreate(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDestroy_v2(cublasHandle_t handle);
  // CUDA: #define cublasDestroy cublasDestroy_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDestroy(hipblasHandle_t handle);
  // CHECK: blasStatus = hipblasDestroy(blasHandle);
  // CHECK-NEXT: blasStatus = hipblasDestroy(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);

  // CHECK: hipStream_t stream;
  hipStream_t stream;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetStream_v2(cublasHandle_t handle, cudaStream_t streamId);
  // CUDA: #define cublasSetStream cublasSetStream_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetStream(hipblasHandle_t handle, hipStream_t streamId);
  // CHECK: blasStatus = hipblasSetStream(blasHandle, stream);
  // CHECK-NEXT: blasStatus = hipblasSetStream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetStream_v2(cublasHandle_t handle, cudaStream_t* streamId);
  // CUDA: #define cublasGetStream cublasGetStream_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetStream(hipblasHandle_t handle, hipStream_t* streamId);
  // CHECK: blasStatus = hipblasGetStream(blasHandle, &stream);
  // CHECK-NEXT: blasStatus = hipblasGetStream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetPointerMode_v2(cublasHandle_t handle, cublasPointerMode_t mode);
  // CUDA: #define cublasSetPointerMode cublasSetPointerMode_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetPointerMode(hipblasHandle_t handle, hipblasPointerMode_t mode);
  // CHECK: blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);
  // CHECK-NEXT: blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);
  blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);
  blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetPointerMode_v2(cublasHandle_t handle, cublasPointerMode_t* mode);
  // CUDA: #define cublasGetPointerMode cublasGetPointerMode_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetPointerMode(hipblasHandle_t handle, hipblasPointerMode_t* mode);
  // CHECK: blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);
  // CHECK-NEXT: blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);
  blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);
  blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);

  int n = 0;
  int64_t n_64 = 0;
  int nrhs = 0;
  int m = 0;
  int num = 0;
  int lda = 0;
  int ldb = 0;
  int ldc = 0;
  int res = 0;
  int64_t res_64 = 0;
  int incx = 0;
  int64_t incx_64 = 0;
  int incy = 0;
  int64_t incy_64 = 0;
  int k = 0;
  int kl = 0;
  int ku = 0;
  int batchCount = 0;
  int P = 0;
  int info = 0;
  void* image = nullptr;
  void* image_2 = nullptr;
  void* deviceptr = nullptr;

  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetVector(int n, int elemSize, const void* x, int incx, void* devicePtr, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // CHECK: blasStatus = hipblasSetVector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasSetVector(n, num, image, incx, image_2, incy);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // CHECK: blasStatus = hipblasGetVector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasGetVector(n, num, image, incx, image_2, incy);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetVectorAsync(int n, int elemSize, const void* hostPtr, int incx, void* devicePtr, int incy, cudaStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetVectorAsync(int n, int elemSize, const void* x, int incx, void* y, int incy, hipStream_t stream);
  // CHECK: blasStatus = hipblasSetVectorAsync(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetVectorAsync(n, num, image, incx, image_2, incy, stream);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetVectorAsync(int n, int elemSize, const void* devicePtr, int incx, void* hostPtr, int incy, cudaStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetVectorAsync(int n, int elemSize, const void* x, int incx, void* y, int incy, hipStream_t stream);
  // CHECK: blasStatus = hipblasGetVectorAsync(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetVectorAsync(n, num, image, incx, image_2, incy, stream);

  int rows = 0;
  int cols = 0;

  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetMatrix(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb);
  // CHECK: blasStatus = hipblasSetMatrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasSetMatrix(rows, cols, num, image, incx, image_2, incy);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetMatrix(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb);
  // CHECK: blasStatus = hipblasGetMatrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasGetMatrix(rows, cols, num, image, incx, image_2, incy);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, cudaStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetMatrixAsync(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb, hipStream_t stream);
  // CHECK: blasStatus = hipblasSetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, cudaStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetMatrixAsync(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb, hipStream_t stream);
  // CHECK: blasStatus = hipblasGetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  float fa = 0;
  float fA = 0;
  float fb = 0;
  float fB = 0;
  float fx = 0;
  float fx1 = 0;
  float fy = 0;
  float fy1 = 0;
  float fc = 0;
  float fC = 0;
  float fs = 0;
  float fd1 = 0;
  float fd2 = 0;
  float fresult = 0;
  float fparam = 0;

  float** fAarray = 0;
  const float** const fAarray_const = const_cast<const float**>(fAarray);
  float** fBarray = 0;
  const float** const fBarray_const = const_cast<const float**>(fBarray);
  float** fCarray = 0;
  float** fTauarray = 0;

  double da = 0;
  double dA = 0;
  double db = 0;
  double dB = 0;
  double dx = 0;
  double dx1 = 0;
  double dy = 0;
  double dy1 = 0;
  double dc = 0;
  double dC = 0;
  double ds = 0;
  double dd1 = 0;
  double dd2 = 0;
  double dresult = 0;
  double dparam = 0;

  double** dAarray = 0;
  const double** const dAarray_const = const_cast<const double**>(dAarray);
  double** dBarray = 0;
  const double** const dBarray_const = const_cast<const double**>(dBarray);
  double** dCarray = 0;
  double** dTauarray = 0;

  void** voidAarray = nullptr;
  const void** const voidAarray_const = const_cast<const void**>(voidAarray);
  void** voidBarray = nullptr;
  const void** const voidBarray_const = const_cast<const void**>(voidBarray);
  void** voidCarray = nullptr;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSnrm2_v2(cublasHandle_t handle, int n, const float* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSnrm2(hipblasHandle_t handle, int n, const float* x, int incx, float* result);
  // CHECK: blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);
  // CHECK-NEXT: blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDnrm2_v2(cublasHandle_t handle, int n, const double* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDnrm2(hipblasHandle_t handle, int n, const double* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);

  // CHECK: hipComplex complex, complexa, complexA, complexB, complexC, complexx, complexy, complexs, complexb, complexresult;
  hipComplex complex, complexa, complexA, complexB, complexC, complexx, complexy, complexs, complexb, complexresult;
  // CHECK: hipDoubleComplex dcomplex, dcomplexa, dcomplexA, dcomplexB, dcomplexC, dcomplexx, dcomplexy, dcomplexs, dcomplexb, dcomplexresult;
  hipDoubleComplex dcomplex, dcomplexa, dcomplexA, dcomplexB, dcomplexC, dcomplexx, dcomplexy, dcomplexs, dcomplexb, dcomplexresult;

  // CHECK: hipComplex** complexAarray = 0;
  // CHECK: const hipComplex** const complexAarray_const = const_cast<const hipComplex**>(complexAarray);
  // CHECK-NEXT: hipComplex** complexBarray = 0;
  // CHECK: const hipComplex** const complexBarray_const = const_cast<const hipComplex**>(complexBarray);
  // CHECK-NEXT: hipComplex** complexCarray = 0;
  // CHECK-NEXT: hipComplex** complexTauarray = 0;
  hipComplex** complexAarray = 0;
  const hipComplex** const complexAarray_const = const_cast<const hipComplex**>(complexAarray);
  hipComplex** complexBarray = 0;
  const hipComplex** const complexBarray_const = const_cast<const hipComplex**>(complexBarray);
  hipComplex** complexCarray = 0;
  hipComplex** complexTauarray = 0;

  // CHECK: hipDoubleComplex** dcomplexAarray = 0;
  // CHECK: const hipDoubleComplex** const dcomplexAarray_const = const_cast<const hipDoubleComplex**>(dcomplexAarray);
  // CHECK-NEXT: hipDoubleComplex** dcomplexBarray = 0;
  // CHECK: const hipDoubleComplex** const dcomplexBarray_const = const_cast<const hipDoubleComplex**>(dcomplexBarray);
  // CHECK-NEXT: hipDoubleComplex** dcomplexCarray = 0;
  // CHECK-NEXT: hipDoubleComplex** dcomplexTauarray = 0;
  hipDoubleComplex** dcomplexAarray = 0;
  const hipDoubleComplex** const dcomplexAarray_const = const_cast<const hipDoubleComplex**>(dcomplexAarray);
  hipDoubleComplex** dcomplexBarray = 0;
  const hipDoubleComplex** const dcomplexBarray_const = const_cast<const hipDoubleComplex**>(dcomplexBarray);
  hipDoubleComplex** dcomplexCarray = 0;
  hipDoubleComplex** dcomplexTauarray = 0;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScnrm2_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScnrm2_v2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, float* result);
  // CHECK: blasStatus = hipblasScnrm2_v2(blasHandle, n, &complex, incx, &fresult);
  // CHECK-NEXT: blasStatus = hipblasScnrm2_v2(blasHandle, n, &complex, incx, &fresult);
  blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);
  blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDznrm2_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDznrm2_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDznrm2_v2(blasHandle, n, &dcomplex, incx, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDznrm2_v2(blasHandle, n, &dcomplex, incx, &dresult);
  blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSdot_v2(cublasHandle_t handle, int n, const float* x, int incx, const float* y, int incy, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSdot(hipblasHandle_t handle, int n, const float* x, int incx, const float* y, int incy, float* result);
  // CHECK: blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  // CHECK-NEXT: blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDdot_v2(cublasHandle_t handle, int n, const double* x, int incx, const double* y, int incy, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDdot(hipblasHandle_t handle, int n, const double* x, int incx, const double* y, int incy, double* result);
  // CHECK: blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdotu_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCdotu_v2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* result);
  // CHECK: blasStatus = hipblasCdotu_v2(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  // CHECK-NEXT: blasStatus = hipblasCdotu_v2(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdotc_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCdotc_v2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* result);
  // CHECK: blasStatus = hipblasCdotc_v2(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  // CHECK-NEXT: blasStatus = hipblasCdotc_v2(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdotu_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdotu_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* result);
  // CHECK: blasStatus = hipblasZdotu_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  // CHECK-NEXT: blasStatus = hipblasZdotu_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdotc_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdotc_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* result);
  // CHECK: blasStatus = hipblasZdotc_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  // CHECK-NEXT: blasStatus = hipblasZdotc_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSscal_v2(cublasHandle_t handle, int n, const float* alpha, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSscal(hipblasHandle_t handle, int n, const float* alpha, float* x, int incx);
  // CHECK: blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);
  // CHECK-NEXT: blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);
  blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);
  blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDscal_v2(cublasHandle_t handle, int n, const double* alpha, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDscal(hipblasHandle_t handle, int n, const double* alpha, double* x, int incx);
  // CHECK: blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);
  // CHECK-NEXT: blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);
  blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);
  blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCscal_v2(cublasHandle_t handle, int n, const cuComplex* alpha, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCscal_v2(hipblasHandle_t handle, int n, const hipComplex* alpha, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCscal_v2(blasHandle, n, &complexa, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCscal_v2(blasHandle, n, &complexa, &complexx, incx);
  blasStatus = hipblasCscal(blasHandle, n, &complexa, &complexx, incx);
  blasStatus = hipblasCscal(blasHandle, n, &complexa, &complexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsscal_v2(cublasHandle_t handle, int n, const float* alpha, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsscal_v2(hipblasHandle_t handle, int n, const float* alpha, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCsscal_v2(blasHandle, n, &fx, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCsscal_v2(blasHandle, n, &fx, &complexx, incx);
  blasStatus = hipblasCsscal(blasHandle, n, &fx, &complexx, incx);
  blasStatus = hipblasCsscal(blasHandle, n, &fx, &complexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZscal_v2(cublasHandle_t handle, int n, const cuDoubleComplex* alpha, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZscal_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* alpha, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZscal_v2(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZscal_v2(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  blasStatus = hipblasZscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  blasStatus = hipblasZscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdscal_v2(cublasHandle_t handle, int n, const double* alpha, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdscal_v2(hipblasHandle_t handle, int n, const double* alpha, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZdscal_v2(blasHandle, n, &dx, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZdscal_v2(blasHandle, n, &dx, &dcomplexx, incx);
  blasStatus = hipblasZdscal(blasHandle, n, &dx, &dcomplexx, incx);
  blasStatus = hipblasZdscal(blasHandle, n, &dx, &dcomplexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSaxpy_v2(cublasHandle_t handle, int n, const float* alpha, const float* x, int incx, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSaxpy(hipblasHandle_t handle, int n, const float* alpha, const float* x, int incx, float* y, int incy);
  // CHECK: blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDaxpy_v2(cublasHandle_t handle, int n, const double* alpha, const double* x, int incx, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDaxpy(hipblasHandle_t handle, int n, const double* alpha, const double* x, int incx, double* y, int incy);
  // CHECK: blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCaxpy_v2(cublasHandle_t handle, int n, const cuComplex* alpha, const cuComplex* x, int incx, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCaxpy_v2(hipblasHandle_t handle, int n, const hipComplex* alpha, const hipComplex* x, int incx, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasCaxpy_v2(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasCaxpy_v2(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCaxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCaxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZaxpy_v2(cublasHandle_t handle, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZaxpy_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZaxpy_v2(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZaxpy_v2(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScopy_v2(cublasHandle_t handle, int n, const float* x, int incx, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScopy(hipblasHandle_t handle, int n, const float* x, int incx, float* y, int incy);
  // CHECK: blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDcopy_v2(cublasHandle_t handle, int n, const double* x, int incx, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDcopy(hipblasHandle_t handle, int n, const double* x, int incx, double* y, int incy);
  // CHECK: blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCcopy_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCcopy_v2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasCcopy_v2(blasHandle, n, &complexx, incx, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasCcopy_v2(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCcopy(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCcopy(blasHandle, n, &complexx, incx, &complexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZcopy_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZcopy_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZcopy_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZcopy_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSswap_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSswap(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy);
  // CHECK: blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDswap_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDswap(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy);
  // CHECK: blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCswap_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCswap_v2(hipblasHandle_t handle, int n, hipComplex* x, int incx, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasCswap_v2(blasHandle, n, &complexx, incx, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasCswap_v2(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCswap(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCswap(blasHandle, n, &complexx, incx, &complexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZswap_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZswap_v2(hipblasHandle_t handle, int n, hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZswap_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZswap_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIsamax_v2(cublasHandle_t handle, int n, const float* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIsamax(hipblasHandle_t handle, int n, const float* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIdamax_v2(cublasHandle_t handle, int n, const double* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIdamax(hipblasHandle_t handle, int n, const double* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIcamax_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIcamax_v2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIcamax_v2(blasHandle, n, &complexx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIcamax_v2(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamax(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamax(blasHandle, n, &complexx, incx, &res);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIzamax_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIzamax_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIzamax_v2(blasHandle, n, &dcomplexx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIzamax_v2(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamax(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamax(blasHandle, n, &dcomplexx, incx, &res);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIsamin_v2(cublasHandle_t handle, int n, const float* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIsamin(hipblasHandle_t handle, int n, const float* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIdamin_v2(cublasHandle_t handle, int n, const double* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIdamin(hipblasHandle_t handle, int n, const double* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIcamin_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIcamin_v2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIcamin_v2(blasHandle, n, &complexx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIcamin_v2(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamin(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamin(blasHandle, n, &complexx, incx, &res);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIzamin_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIzamin_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIzamin_v2(blasHandle, n, &dcomplexx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIzamin_v2(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamin(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamin(blasHandle, n, &dcomplexx, incx, &res);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSasum_v2(cublasHandle_t handle, int n, const float* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSasum(hipblasHandle_t handle, int n, const float* x, int incx, float* result);
  // CHECK: blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);
  // CHECK-NEXT: blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDasum_v2(cublasHandle_t handle, int n, const double* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDasum(hipblasHandle_t handle, int n, const double* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScasum_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScasum_v2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, float* result);
  // CHECK: blasStatus = hipblasScasum_v2(blasHandle, n, &complexx, incx, &fresult);
  // CHECK-NEXT: blasStatus = hipblasScasum_v2(blasHandle, n, &complexx, incx, &fresult);
  blasStatus = hipblasScasum(blasHandle, n, &complexx, incx, &fresult);
  blasStatus = hipblasScasum(blasHandle, n, &complexx, incx, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDzasum_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDzasum_v2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDzasum_v2(blasHandle, n, &dcomplexx, incx, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDzasum_v2(blasHandle, n, &dcomplexx, incx, &dresult);
  blasStatus = hipblasDzasum(blasHandle, n, &dcomplexx, incx, &dresult);
  blasStatus = hipblasDzasum(blasHandle, n, &dcomplexx, incx, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrot_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* c, const float* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrot(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* c, const float* s);
  // CHECK: blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);
  // CHECK-NEXT: blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);
  blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);
  blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrot_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* c, const double* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrot(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* c, const double* s);
  // CHECK: blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);
  // CHECK-NEXT: blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);
  blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);
  blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);

  // CUDA: CUBLASAPI CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCrot_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy, const float* c, const cuComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCrot_v2(hipblasHandle_t handle, int n, hipComplex* x, int incx, hipComplex* y, int incy, const float* c, const hipComplex* s);
  // CHECK: blasStatus = hipblasCrot_v2(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);
  // CHECK-NEXT: blasStatus = hipblasCrot_v2(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);
  blasStatus = hipblasCrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);
  blasStatus = hipblasCrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsrot_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy, const float* c, const float* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsrot_v2(hipblasHandle_t handle, int n, hipComplex* x, int incx, hipComplex* y, int incy, const float* c, const float* s);
  // CHECK: blasStatus = hipblasCsrot_v2(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);
  // CHECK-NEXT: blasStatus = hipblasCsrot_v2(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);
  blasStatus = hipblasCsrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);
  blasStatus = hipblasCsrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZrot_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, const double* c, const cuDoubleComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZrot_v2(hipblasHandle_t handle, int n, hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy, const double* c, const hipDoubleComplex* s);
  // CHECK: blasStatus = hipblasZrot_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);
  // CHECK-NEXT: blasStatus = hipblasZrot_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);
  blasStatus = hipblasZrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);
  blasStatus = hipblasZrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdrot_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, const double* c, const double* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdrot_v2(hipblasHandle_t handle, int n, hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy, const double* c, const double* s);
  // CHECK: blasStatus = hipblasZdrot_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);
  // CHECK-NEXT: blasStatus = hipblasZdrot_v2(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);
  blasStatus = hipblasZdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);
  blasStatus = hipblasZdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotg_v2(cublasHandle_t handle, float* a, float* b, float* c, float* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrotg(hipblasHandle_t handle, float* a, float* b, float* c, float* s);
  // CHECK: blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);
  // CHECK-NEXT: blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);
  blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);
  blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotg_v2(cublasHandle_t handle, double* a, double* b, double* c, double* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrotg(hipblasHandle_t handle, double* a, double* b, double* c, double* s);
  // CHECK: blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);
  // CHECK-NEXT: blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);
  blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);
  blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCrotg_v2(cublasHandle_t handle, cuComplex* a, cuComplex* b, float* c, cuComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCrotg_v2(hipblasHandle_t handle, hipComplex* a, hipComplex* b, float* c, hipComplex* s);
  // CHECK: blasStatus = hipblasCrotg_v2(blasHandle, &complexa, &complexb, &fc, &complexs);
  // CHECK-NEXT: blasStatus = hipblasCrotg_v2(blasHandle, &complexa, &complexb, &fc, &complexs);
  blasStatus = hipblasCrotg(blasHandle, &complexa, &complexb, &fc, &complexs);
  blasStatus = hipblasCrotg(blasHandle, &complexa, &complexb, &fc, &complexs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZrotg_v2(cublasHandle_t handle, cuDoubleComplex* a, cuDoubleComplex* b, double* c, cuDoubleComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZrotg_v2(hipblasHandle_t handle, hipDoubleComplex* a, hipDoubleComplex* b, double* c, hipDoubleComplex* s);
  // CHECK: blasStatus = hipblasZrotg_v2(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);
  // CHECK-NEXT: blasStatus = hipblasZrotg_v2(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);
  blasStatus = hipblasZrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);
  blasStatus = hipblasZrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotm_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrotm(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* param);
  // CHECK: blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  // CHECK-NEXT: blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotm_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrotm(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* param);
  // CHECK: blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotmg_v2(cublasHandle_t handle, float* d1, float* d2, float* x1, const float* y1, float* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrotmg(hipblasHandle_t handle, float* d1, float* d2, float* x1, const float* y1, float* param);
  // CHECK: blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);
  // CHECK-NEXT: blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);
  blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);
  blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotmg_v2(cublasHandle_t handle, double* d1, double* d2, double* x1, const double* y1, double* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrotmg(hipblasHandle_t handle, double* d1, double* d2, double* x1, const double* y1, double* param);
  // CHECK: blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);
  blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);
  blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemv_v2(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasCgemv_v2(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasCgemv_v2(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemv_v2(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZgemv_v2(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZgemv_v2(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgbmv_v2(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasCgbmv_v2(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasCgbmv_v2(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgbmv_v2(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZgbmv_v2(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZgbmv_v2(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  // CHECK-NEXT: blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  // CHECK-NEXT: blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipComplex* AP, int lda, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtrmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCtrmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipDoubleComplex* AP, int lda, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtrmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZtrmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  // CHECK-NEXT: blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  // CHECK-NEXT: blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtbmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const hipComplex* AP, int lda, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtbmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCtbmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtbmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const hipDoubleComplex* AP, int lda, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtbmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZtbmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* AP, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, float* x, int incx);
  // CHECK: blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  // CHECK-NEXT: blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* AP, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, double* x, int incx);
  // CHECK: blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  // CHECK-NEXT: blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* AP, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtpmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipComplex* AP, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtpmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCtpmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtpmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipDoubleComplex* AP, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtpmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZtpmv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  // CHECK-NEXT: blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  // CHECK-NEXT: blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrsv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipComplex* AP, int lda, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtrsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCtrsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrsv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipDoubleComplex* AP, int lda, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtrsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZtrsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* AP, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, float* x, int incx);
  // CHECK: blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  // CHECK-NEXT: blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* AP, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, double* x, int incx);
  // CHECK: blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  // CHECK-NEXT: blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* AP, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtpsv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipComplex* AP, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtpsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCtpsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtpsv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipDoubleComplex* AP, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtpsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZtpsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  // CHECK-NEXT: blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  // CHECK-NEXT: blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtbsv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const hipComplex* AP, int lda, hipComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtbsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCtbsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtbsv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const hipDoubleComplex* AP, int lda, hipDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtbsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZtbsv_v2(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsymv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasCsymv_v2(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasCsymv_v2(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCsymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCsymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsymv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZsymv_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZsymv_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChemv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChemv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasChemv_v2(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasChemv_v2(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhemv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhemv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZhemv_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZhemv_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChbmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasChbmv_v2(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasChbmv_v2(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhbmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZhbmv_v2(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZhbmv_v2(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* AP, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSspmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* AP, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* AP, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDspmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* AP, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* AP, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChpmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* AP, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // CHECK: blasStatus = hipblasChpmv_v2(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasChpmv_v2(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* AP, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhpmv_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZhpmv_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZhpmv_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSger_v2(cublasHandle_t handle, int m, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSger(hipblasHandle_t handle, int m, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* AP, int lda);
  // CHECK: blasStatus = hipblasSger(blasHandle, m, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  // CHECK-NEXT: blasStatus = hipblasSger(blasHandle, m, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  blasStatus = hipblasSger(blasHandle, m, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  blasStatus = hipblasSger(blasHandle, m, n, &fa, &fx, incx, &fy, incy, &fA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDger_v2(cublasHandle_t handle, int m, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDger(hipblasHandle_t handle, int m, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* AP, int lda);
  // CHECK: blasStatus = hipblasDger(blasHandle, m, n, &da, &dx, incx, &dy, incy, &dA, lda);
  // CHECK-NEXT: blasStatus = hipblasDger(blasHandle, m, n, &da, &dx, incx, &dy, incy, &dA, lda);
  blasStatus = hipblasDger(blasHandle, m, n, &da, &dx, incx, &dy, incy, &dA, lda);
  blasStatus = hipblasDger(blasHandle, m, n, &da, &dx, incx, &dy, incy, &dA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgeru_v2(cublasHandle_t handle, int m, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgeru_v2(hipblasHandle_t handle, int m, int n, const hipComplex* alpha, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCgeru_v2(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  // CHECK-NEXT: blasStatus = hipblasCgeru_v2(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCgeru(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCgeru(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgerc_v2(cublasHandle_t handle, int m, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgerc_v2(hipblasHandle_t handle, int m, int n, const hipComplex* alpha, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCgerc_v2(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  // CHECK-NEXT: blasStatus = hipblasCgerc_v2(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCgerc(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCgerc(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgeru_v2(cublasHandle_t handle, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgeru_v2(hipblasHandle_t handle, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZgeru_v2(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  // CHECK-NEXT: blasStatus = hipblasZgeru_v2(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZgeru(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZgeru(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgerc_v2(cublasHandle_t handle, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgerc_v2(hipblasHandle_t handle, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZgerc_v2(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  // CHECK-NEXT: blasStatus = hipblasZgerc_v2(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZgerc(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZgerc(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* AP, int lda);
  // CHECK: blasStatus = hipblasSsyr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA, lda);
  // CHECK-NEXT: blasStatus = hipblasSsyr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA, lda);
  blasStatus = hipblasSsyr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA, lda);
  blasStatus = hipblasSsyr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* AP, int lda);
  // CHECK: blasStatus = hipblasDsyr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA, lda);
  // CHECK-NEXT: blasStatus = hipblasDsyr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA, lda);
  blasStatus = hipblasDsyr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA, lda);
  blasStatus = hipblasDsyr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyr_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* x, int incx, hipComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCsyr_v2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexA, lda);
  // CHECK-NEXT: blasStatus = hipblasCsyr_v2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexA, lda);
  blasStatus = hipblasCsyr(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexA, lda);
  blasStatus = hipblasCsyr(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyr_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, hipDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZsyr_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexA, lda);
  // CHECK-NEXT: blasStatus = hipblasZsyr_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexA, lda);
  blasStatus = hipblasZsyr(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexA, lda);
  blasStatus = hipblasZsyr(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const cuComplex* x, int incx, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCher_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const hipComplex* x, int incx, hipComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCher_v2(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA, lda);
  // CHECK-NEXT: blasStatus = hipblasCher_v2(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA, lda);
  blasStatus = hipblasCher(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA, lda);
  blasStatus = hipblasCher(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZher_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const hipDoubleComplex* x, int incx, hipDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZher_v2(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA, lda);
  // CHECK-NEXT: blasStatus = hipblasZher_v2(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA, lda);
  blasStatus = hipblasZher(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA, lda);
  blasStatus = hipblasZher(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSspr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* AP);
  // CHECK: blasStatus = hipblasSspr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA);
  // CHECK-NEXT: blasStatus = hipblasSspr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA);
  blasStatus = hipblasSspr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA);
  blasStatus = hipblasSspr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, double* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDspr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, double* AP);
  // CHECK: blasStatus = hipblasDspr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA);
  // CHECK-NEXT: blasStatus = hipblasDspr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA);
  blasStatus = hipblasDspr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA);
  blasStatus = hipblasDspr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const cuComplex* x, int incx, cuComplex* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChpr_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const hipComplex* x, int incx, hipComplex* AP);
  // CHECK: blasStatus = hipblasChpr_v2(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA);
  // CHECK-NEXT: blasStatus = hipblasChpr_v2(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA);
  blasStatus = hipblasChpr(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA);
  blasStatus = hipblasChpr(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhpr_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const hipDoubleComplex* x, int incx, hipDoubleComplex* AP);
  // CHECK: blasStatus = hipblasZhpr_v2(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA);
  // CHECK-NEXT: blasStatus = hipblasZhpr_v2(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA);
  blasStatus = hipblasZhpr(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA);
  blasStatus = hipblasZhpr(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* AP, int lda);
  // CHECK: blasStatus = hipblasSsyr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  // CHECK-NEXT: blasStatus = hipblasSsyr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  blasStatus = hipblasSsyr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  blasStatus = hipblasSsyr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* AP, int lda);
  // CHECK: blasStatus = hipblasDsyr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA, lda);
  // CHECK-NEXT: blasStatus = hipblasDsyr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA, lda);
  blasStatus = hipblasDsyr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA, lda);
  blasStatus = hipblasDsyr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyr2_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCsyr2_v2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  // CHECK-NEXT: blasStatus = hipblasCsyr2_v2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCsyr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCsyr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyr2_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZsyr2_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  // CHECK-NEXT: blasStatus = hipblasZsyr2_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZsyr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZsyr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCher2_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCher2_v2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  // CHECK-NEXT: blasStatus = hipblasCher2_v2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCher2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCher2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZher2_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZher2_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  // CHECK-NEXT: blasStatus = hipblasZher2_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZher2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZher2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSspr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* AP);
  // CHECK: blasStatus = hipblasSspr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA);
  // CHECK-NEXT: blasStatus = hipblasSspr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA);
  blasStatus = hipblasSspr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA);
  blasStatus = hipblasSspr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDspr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* AP);
  // CHECK: blasStatus = hipblasDspr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA);
  // CHECK-NEXT: blasStatus = hipblasDspr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA);
  blasStatus = hipblasDspr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA);
  blasStatus = hipblasDspr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChpr2_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* AP);
  // CHECK: blasStatus = hipblasChpr2_v2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA);
  // CHECK-NEXT: blasStatus = hipblasChpr2_v2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA);
  blasStatus = hipblasChpr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA);
  blasStatus = hipblasChpr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhpr2_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* AP);
  // CHECK: blasStatus = hipblasZhpr2_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA);
  // CHECK-NEXT: blasStatus = hipblasZhpr2_v2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA);
  blasStatus = hipblasZhpr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA);
  blasStatus = hipblasZhpr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA);

  hipblasOperation_t transa, transb;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgemm(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const float* alpha, const float* AP, int lda, const float* BP, int ldb, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSgemm(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  // CHECK-NEXT: blasStatus = hipblasSgemm(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSgemm(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSgemm(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgemm(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const double* alpha, const double* AP, int lda, const double* BP, int ldb, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDgemm(blasHandle, transa, transb, m, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  // CHECK-NEXT: blasStatus = hipblasDgemm(blasHandle, transa, transb, m, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDgemm(blasHandle, transa, transb, m, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDgemm(blasHandle, transa, transb, m, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemm_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* BP, int ldb, const hipComplex* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCgemm_v2(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasCgemm_v2(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCgemm(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCgemm(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemm_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* BP, int ldb, const hipDoubleComplex* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZgemm_v2(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasZgemm_v2(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZgemm(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZgemm(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* const Aarray[], int lda, const float* const Barray[], int ldb, const float* beta, float* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgemmBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const float* alpha, const float* const AP[], int lda, const float* const BP[], int ldb, const float* beta, float* const CP[], int ldc, int batchCount);
  // CHECK: blasStatus = hipblasSgemmBatched(blasHandle, transa, transb, m, n, k, &fa, fAarray_const, lda, fBarray_const, ldb, &fb, fCarray, ldc, batchCount);
  blasStatus = hipblasSgemmBatched(blasHandle, transa, transb, m, n, k, &fa, fAarray_const, lda, fBarray_const, ldb, &fb, fCarray, ldc, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* const Aarray[], int lda, const double* const Barray[], int ldb, const double* beta, double* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgemmBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const double* alpha, const double* const AP[], int lda, const double* const BP[], int ldb, const double* beta, double* const CP[], int ldc, int batchCount);
  // CHECK: blasStatus = hipblasDgemmBatched(blasHandle, transa, transb, m, n, k, &da, dAarray_const, lda, dBarray_const, ldb, &db, dCarray, ldc, batchCount);
  blasStatus = hipblasDgemmBatched(blasHandle, transa, transb, m, n, k, &da, dAarray_const, lda, dBarray_const, ldb, &db, dCarray, ldc, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* const Aarray[], int lda, const cuComplex* const Barray[], int ldb, const cuComplex* beta, cuComplex* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemmBatched_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipComplex* alpha, const hipComplex* const AP[], int lda, const hipComplex* const BP[], int ldb, const hipComplex* beta, hipComplex* const CP[], int ldc, int batchCount);
  // CHECK: blasStatus = hipblasCgemmBatched_v2(blasHandle, transa, transb, m, n, k, &complexa, complexAarray_const, lda, complexBarray_const, ldb, &complexb, complexCarray, ldc, batchCount);
  blasStatus = hipblasCgemmBatched(blasHandle, transa, transb, m, n, k, &complexa, complexAarray_const, lda, complexBarray_const, ldb, &complexb, complexCarray, ldc, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* const Aarray[], int lda, const cuDoubleComplex* const Barray[], int ldb, const cuDoubleComplex* beta, cuDoubleComplex* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemmBatched_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* const AP[], int lda, const hipDoubleComplex* const BP[], int ldb, const hipDoubleComplex* beta, hipDoubleComplex* const CP[], int ldc, int batchCount);
  // CHECK: blasStatus = hipblasZgemmBatched_v2(blasHandle, transa, transb, m, n, k, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray_const, ldb, &dcomplexb, dcomplexCarray, ldc, batchCount);
  blasStatus = hipblasZgemmBatched(blasHandle, transa, transb, m, n, k, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray_const, ldb, &dcomplexb, dcomplexCarray, ldc, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyrk(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const float* alpha, const float* AP, int lda, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSsyrk(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, &fC, ldc);
  // CHECK-NEXT: blasStatus = hipblasSsyrk(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, &fC, ldc);
  blasStatus = hipblasSsyrk(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, &fC, ldc);
  blasStatus = hipblasSsyrk(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, &fC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyrk(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const double* alpha, const double* AP, int lda, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDsyrk(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, &dC, ldc);
  // CHECK-NEXT: blasStatus = hipblasDsyrk(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, &dC, ldc);
  blasStatus = hipblasDsyrk(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, &dC, ldc);
  blasStatus = hipblasDsyrk(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, &dC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyrk_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCsyrk_v2(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasCsyrk_v2(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyrk(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyrk(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyrk_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZsyrk_v2(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasZsyrk_v2(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyrk(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyrk(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCherk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const cuComplex* A, int lda, const float* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCherk_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const float* alpha, const hipComplex* AP, int lda, const float* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCherk_v2(blasHandle, blasFillMode, transa, n, k, &fa, &complexA, lda, &fb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasCherk_v2(blasHandle, blasFillMode, transa, n, k, &fa, &complexA, lda, &fb, &complexC, ldc);
  blasStatus = hipblasCherk(blasHandle, blasFillMode, transa, n, k, &fa, &complexA, lda, &fb, &complexC, ldc);
  blasStatus = hipblasCherk(blasHandle, blasFillMode, transa, n, k, &fa, &complexA, lda, &fb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZherk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const cuDoubleComplex* A, int lda, const double* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZherk_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const double* alpha, const hipDoubleComplex* AP, int lda, const double* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZherk_v2(blasHandle, blasFillMode, transa, n, k, &da, &dcomplexA, lda, &db, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasZherk_v2(blasHandle, blasFillMode, transa, n, k, &da, &dcomplexA, lda, &db, &dcomplexC, ldc);
  blasStatus = hipblasZherk(blasHandle, blasFillMode, transa, n, k, &da, &dcomplexA, lda, &db, &dcomplexC, ldc);
  blasStatus = hipblasZherk(blasHandle, blasFillMode, transa, n, k, &da, &dcomplexA, lda, &db, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyr2k(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const float* alpha, const float* AP, int lda, const float* BP, int ldb, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSsyr2k(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, ldb, &fb, &fC, ldc);
  // CHECK-NEXT: blasStatus = hipblasSsyr2k(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsyr2k(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsyr2k(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, ldb, &fb, &fC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyr2k(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const double* alpha, const double* AP, int lda, const double* BP, int ldb, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDsyr2k(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, ldb, &db, &dC, ldc);
  // CHECK-NEXT: blasStatus = hipblasDsyr2k(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsyr2k(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsyr2k(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, ldb, &db, &dC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyr2k_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* BP, int ldb, const hipComplex* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCsyr2k_v2(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &complexb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasCsyr2k_v2(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyr2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyr2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &complexb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyr2k_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* BP, int ldb, const hipDoubleComplex* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZsyr2k_v2(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &dcomplexb, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasZsyr2k_v2(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyr2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyr2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &dcomplexb, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyrkx(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const float* alpha, const float* AP, int lda, const float* BP, int ldb, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSsyrkx(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsyrkx(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyrkx(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const double* alpha, const double* AP, int lda, const double* BP, int ldb, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDsyrkx(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsyrkx(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyrkx_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* BP, int ldb, const hipComplex* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCsyrkx_v2(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyrkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyrkx_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* BP, int ldb, const hipDoubleComplex* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZsyrkx_v2(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyrkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const float* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCher2k_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* BP, int ldb, const float* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCher2k_v2(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &fb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasCher2k_v2(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &fb, &complexC, ldc);
  blasStatus = hipblasCher2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &fb, &complexC, ldc);
  blasStatus = hipblasCher2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &fb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const double* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZher2k_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* BP, int ldb, const double* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZher2k_v2(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &db, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasZher2k_v2(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &db, &dcomplexC, ldc);
  blasStatus = hipblasZher2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &db, &dcomplexC, ldc);
  blasStatus = hipblasZher2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &db, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCherkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const float* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCherkx_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* BP, int ldb, const float* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCherkx_v2(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &fb, &complexC, ldc);
  blasStatus = hipblasCherkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &fb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZherkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const double* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZherkx_v2(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* BP, int ldb, const double* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZherkx_v2(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &db, &dcomplexC, ldc);
  blasStatus = hipblasZherkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &db, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsymm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int m, int n, const float* alpha, const float* AP, int lda, const float* BP, int ldb, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSsymm(blasHandle, blasSideMode, blasFillMode, m, n, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  // CHECK-NEXT: blasStatus = hipblasSsymm(blasHandle, blasSideMode, blasFillMode, m, n, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsymm(blasHandle, blasSideMode, blasFillMode, m, n, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsymm(blasHandle, blasSideMode, blasFillMode, m, n, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsymm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int m, int n, const double* alpha, const double* AP, int lda, const double* BP, int ldb, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDsymm(blasHandle, blasSideMode, blasFillMode, m, n, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  // CHECK-NEXT: blasStatus = hipblasDsymm(blasHandle, blasSideMode, blasFillMode, m, n, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsymm(blasHandle, blasSideMode, blasFillMode, m, n, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsymm(blasHandle, blasSideMode, blasFillMode, m, n, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsymm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int m, int n, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* BP, int ldb, const hipComplex* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCsymm_v2(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasCsymm_v2(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsymm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsymm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsymm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* BP, int ldb, const hipDoubleComplex* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZsymm_v2(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasZsymm_v2(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsymm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsymm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChemm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChemm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* BP, int ldb, const hipComplex* beta, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasChemm_v2(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasChemm_v2(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasChemm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasChemm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhemm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhemm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* BP, int ldb, const hipDoubleComplex* beta, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZhemm_v2(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasZhemm_v2(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZhemm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZhemm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const float* alpha, const float* A, int lda, float* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrsm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const float* alpha, float* AP, int lda, float* BP, int ldb);
  // CHECK: blasStatus = hipblasStrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb);
  // CHECK-NEXT: blasStatus = hipblasStrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb);
  blasStatus = hipblasStrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb);
  blasStatus = hipblasStrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const double* alpha, const double* A, int lda, double* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrsm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const double* alpha, double* AP, int lda, double* BP, int ldb);
  // CHECK: blasStatus = hipblasDtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb);
  // CHECK-NEXT: blasStatus = hipblasDtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb);
  blasStatus = hipblasDtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb);
  blasStatus = hipblasDtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, cuComplex* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrsm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const hipComplex* alpha, const hipComplex* AP, int lda, hipComplex* BP, int ldb);
  // CHECK: blasStatus = hipblasCtrsm_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb);
  // CHECK-NEXT: blasStatus = hipblasCtrsm_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb);
  blasStatus = hipblasCtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb);
  blasStatus = hipblasCtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, cuDoubleComplex* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrsm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, hipDoubleComplex* BP, int ldb);
  // CHECK: blasStatus = hipblasZtrsm_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb);
  // CHECK-NEXT: blasStatus = hipblasZtrsm_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb);
  blasStatus = hipblasZtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb);
  blasStatus = hipblasZtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const float* alpha, const float* A, int lda, const float* beta, const float* B, int ldb, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgeam(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, const float* alpha, const float* AP, int lda, const float* beta, const float* BP, int ldb, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSgeam(blasHandle, transa, transb, m, n, &fa, &fA, lda, &fb, &fB, ldb, &fC, ldc);
  blasStatus = hipblasSgeam(blasHandle, transa, transb, m, n, &fa, &fA, lda, &fb, &fB, ldb, &fC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const double* alpha, const double* A, int lda, const double* beta, const double* B, int ldb, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgeam(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, const double* alpha, const double* AP, int lda, const double* beta, const double* BP, int ldb, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDgeam(blasHandle, transa, transb, m, n, &da, &dA, lda, &db, &dB, ldb, &dC, ldc);
  blasStatus = hipblasDgeam(blasHandle, transa, transb, m, n, &da, &dA, lda, &db, &dB, ldb, &dC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* beta, const cuComplex* B, int ldb, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgeam_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, const hipComplex* alpha, const hipComplex* AP, int lda, const hipComplex* beta, const hipComplex* BP, int ldb, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCgeam_v2(blasHandle, transa, transb, m, n, &complexa, &complexA, lda, &complexb, &complexB, ldb, &complexC, ldc);
  blasStatus = hipblasCgeam(blasHandle, transa, transb, m, n, &complexa, &complexA, lda, &complexb, &complexB, ldb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* beta, const cuDoubleComplex* B, int ldb, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgeam_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* beta, const hipDoubleComplex* BP, int ldb, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZgeam_v2(blasHandle, transa, transb, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexB, ldb, &dcomplexC, ldc);
  blasStatus = hipblasZgeam(blasHandle, transa, transb, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexB, ldb, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgetrfBatched(cublasHandle_t handle, int n, float* const A[], int lda, int* P, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgetrfBatched(hipblasHandle_t handle, const int n, float* const A[], const int lda, int* ipiv, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasSgetrfBatched(blasHandle, n, fAarray, lda, &P, &info, batchCount);
  blasStatus = hipblasSgetrfBatched(blasHandle, n, fAarray, lda, &P, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgetrfBatched(cublasHandle_t handle, int n, double* const A[], int lda, int* P, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgetrfBatched(hipblasHandle_t handle, const int n, double* const A[], const int lda, int* ipiv, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasDgetrfBatched(blasHandle, n, dAarray, lda, &P, &info, batchCount);
  blasStatus = hipblasDgetrfBatched(blasHandle, n, dAarray, lda, &P, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgetrfBatched(cublasHandle_t handle, int n, cuComplex* const A[], int lda, int* P, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgetrfBatched_v2(hipblasHandle_t handle, const int n, hipComplex* const A[], const int lda, int* ipiv, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasCgetrfBatched_v2(blasHandle, n, complexAarray, lda, &P, &info, batchCount);
  blasStatus = hipblasCgetrfBatched(blasHandle, n, complexAarray, lda, &P, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgetrfBatched(cublasHandle_t handle, int n, cuDoubleComplex* const A[], int lda, int* P, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgetrfBatched_v2(hipblasHandle_t handle, const int n, hipDoubleComplex* const A[], const int lda, int* ipiv, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasZgetrfBatched_v2(blasHandle, n, dcomplexAarray, lda, &P, &info, batchCount);
  blasStatus = hipblasZgetrfBatched(blasHandle, n, dcomplexAarray, lda, &P, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgetriBatched(cublasHandle_t handle, int n, const float* const A[], int lda, const int* P, float* const C[], int ldc, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgetriBatched(hipblasHandle_t handle, const int n, float* const A[], const int lda, int* ipiv, float* const C[], const int ldc, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasSgetriBatched(blasHandle, n, fAarray_const, lda, &P, fCarray, ldc, &info, batchCount);
  blasStatus = hipblasSgetriBatched(blasHandle, n, fAarray_const, lda, &P, fCarray, ldc, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgetriBatched(cublasHandle_t handle, int n, const double* const A[], int lda, const int* P, double* const C[], int ldc, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgetriBatched(hipblasHandle_t handle, const int n, double* const A[], const int lda, int* ipiv, double* const C[], const int ldc, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasDgetriBatched(blasHandle, n, dAarray_const, lda, &P, dCarray, ldc, &info, batchCount);
  blasStatus = hipblasDgetriBatched(blasHandle, n, dAarray_const, lda, &P, dCarray, ldc, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgetriBatched(cublasHandle_t handle, int n, const cuComplex* const A[], int lda, const int* P, cuComplex* const C[], int ldc, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgetriBatched_v2(hipblasHandle_t handle, const int n, hipComplex* const A[], const int lda, int* ipiv, hipComplex* const C[], const int ldc, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasCgetriBatched_v2(blasHandle, n, complexAarray_const, lda, &P, complexCarray, ldc, &info, batchCount);
  blasStatus = hipblasCgetriBatched(blasHandle, n, complexAarray_const, lda, &P, complexCarray, ldc, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgetriBatched(cublasHandle_t handle, int n, const cuDoubleComplex* const A[], int lda, const int* P, cuDoubleComplex* const C[], int ldc, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgetriBatched_v2(hipblasHandle_t handle, const int n, hipDoubleComplex* const A[], const int lda, int* ipiv, hipDoubleComplex* const C[], const int ldc, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasZgetriBatched_v2(blasHandle, n, dcomplexAarray_const, lda, &P, dcomplexCarray, ldc, &info, batchCount);
  blasStatus = hipblasZgetriBatched(blasHandle, n, dcomplexAarray_const, lda, &P, dcomplexCarray, ldc, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgetrsBatched(cublasHandle_t handle, cublasOperation_t trans, int n, int nrhs, const float* const Aarray[], int lda, const int* devIpiv, float* const Barray[], int ldb, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgetrsBatched(hipblasHandle_t handle, const hipblasOperation_t trans, const int n, const int nrhs, float* const A[], const int lda, const int* ipiv, float* const B[], const int ldb, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasSgetrsBatched(blasHandle, transa, n, nrhs, fAarray_const, lda, &P, fBarray, ldb, &info, batchCount);
  blasStatus = hipblasSgetrsBatched(blasHandle, transa, n, nrhs, fAarray_const, lda, &P, fBarray, ldb, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgetrsBatched(cublasHandle_t handle, cublasOperation_t trans, int n, int nrhs, const double* const Aarray[], int lda, const int* devIpiv, double* const Barray[], int ldb, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgetrsBatched(hipblasHandle_t handle, const hipblasOperation_t trans, const int n, const int nrhs, double* const A[], const int lda, const int* ipiv, double* const B[], const int ldb, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasDgetrsBatched(blasHandle, transa, n, nrhs, dAarray_const, lda, &P, dBarray, ldb, &info, batchCount);
  blasStatus = hipblasDgetrsBatched(blasHandle, transa, n, nrhs, dAarray_const, lda, &P, dBarray, ldb, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgetrsBatched(cublasHandle_t handle, cublasOperation_t trans, int n, int nrhs, const cuComplex* const Aarray[], int lda, const int* devIpiv, cuComplex* const Barray[], int ldb, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgetrsBatched_v2(hipblasHandle_t handle, const hipblasOperation_t trans, const int n, const int nrhs, hipComplex* const A[], const int lda, const int* ipiv, hipComplex* const B[], const int ldb, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasCgetrsBatched_v2(blasHandle, transa, n, nrhs, complexAarray_const, lda, &P, complexBarray, ldb, &info, batchCount);
  blasStatus = hipblasCgetrsBatched(blasHandle, transa, n, nrhs, complexAarray_const, lda, &P, complexBarray, ldb, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgetrsBatched(cublasHandle_t handle, cublasOperation_t trans, int n, int nrhs, const cuDoubleComplex* const Aarray[], int lda, const int* devIpiv, cuDoubleComplex* const Barray[], int ldb, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgetrsBatched_v2(hipblasHandle_t handle, const hipblasOperation_t trans, const int n, const int nrhs, hipDoubleComplex* const A[], const int lda, const int* ipiv, hipDoubleComplex* const B[], const int ldb, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasZgetrsBatched_v2(blasHandle, transa, n, nrhs, dcomplexAarray_const, lda, &P, dcomplexBarray, ldb, &info, batchCount);
  blasStatus = hipblasZgetrsBatched(blasHandle, transa, n, nrhs, dcomplexAarray_const, lda, &P, dcomplexBarray, ldb, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const float* alpha, const float* const A[], int lda, float* const B[], int ldb, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrsmBatched(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const float* alpha, float* const AP[], int lda, float* BP[], int ldb, int batchCount);
  // CHECK: blasStatus = hipblasStrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, fAarray_const, lda, fBarray, ldb, batchCount);
  blasStatus = hipblasStrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, fAarray_const, lda, fBarray, ldb, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const double* alpha, const double* const A[], int lda, double* const B[], int ldb, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrsmBatched(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const double* alpha, double* const AP[], int lda, double* BP[], int ldb, int batchCount);
  // CHECK: blasStatus = hipblasDtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, dAarray_const, lda, dBarray, ldb, batchCount);
  blasStatus = hipblasDtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, dAarray_const, lda, dBarray, ldb, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuComplex* alpha, const cuComplex* const A[], int lda, cuComplex* const B[], int ldb, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrsmBatched_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA,hipblasDiagType_t diag, int m, int n, const hipComplex* alpha, const hipComplex* const AP[], int lda, hipComplex* const BP[], int ldb, int batchCount);
  // CHECK: blasStatus = hipblasCtrsmBatched_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, complexAarray_const, lda, complexBarray, ldb, batchCount);
  blasStatus = hipblasCtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, complexAarray_const, lda, complexBarray, ldb, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* const A[], int lda, cuDoubleComplex* const B[], int ldb, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrsmBatched_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* const AP[], int lda, hipDoubleComplex* const BP[], int ldb, int batchCount);
  // CHECK: blasStatus = hipblasZtrsmBatched_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray, ldb, batchCount);
  blasStatus = hipblasZtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray, ldb, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgeqrfBatched(cublasHandle_t handle, int m, int n, float* const Aarray[], int lda, float* const TauArray[], int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgeqrfBatched(hipblasHandle_t handle, const int m, const int n, float* const A[], const int lda, float* const ipiv[], int* info, const int batchCount);
  // CHECK: blasStatus = hipblasSgeqrfBatched(blasHandle, m, n, fAarray, lda, fTauarray, &info, batchCount);
  blasStatus = hipblasSgeqrfBatched(blasHandle, m, n, fAarray, lda, fTauarray, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgeqrfBatched(cublasHandle_t handle, int m, int n, double* const Aarray[], int lda, double* const TauArray[], int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgeqrfBatched(hipblasHandle_t handle, const int m, const int n, double* const A[], const int lda, double* const ipiv[], int* info, const int batchCount);
  // CHECK: blasStatus = hipblasDgeqrfBatched(blasHandle, m, n, dAarray, lda, dTauarray, &info, batchCount);
  blasStatus = hipblasDgeqrfBatched(blasHandle, m, n, dAarray, lda, dTauarray, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgeqrfBatched(cublasHandle_t handle, int m, int n, cuComplex* const Aarray[], int lda, cuComplex* const TauArray[], int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgeqrfBatched_v2(hipblasHandle_t handle, const int m, const int n, hipComplex* const A[], const int lda, hipComplex* const ipiv[], int* info, const int batchCount);
  // CHECK: blasStatus = hipblasCgeqrfBatched_v2(blasHandle, m, n, complexAarray, lda, complexTauarray, &info, batchCount);
  blasStatus = hipblasCgeqrfBatched(blasHandle, m, n, complexAarray, lda, complexTauarray, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgeqrfBatched(cublasHandle_t handle, int m, int n, cuDoubleComplex* const Aarray[], int lda, cuDoubleComplex* const TauArray[], int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgeqrfBatched_v2(hipblasHandle_t handle, const int m, const int n, hipDoubleComplex* const A[], const int lda, hipDoubleComplex* const ipiv[], int* info, const int batchCount);
  // CHECK: blasStatus = hipblasZgeqrfBatched_v2(blasHandle, m, n, dcomplexAarray, lda, dcomplexTauarray, &info, batchCount);
  blasStatus = hipblasZgeqrfBatched(blasHandle, m, n, dcomplexAarray, lda, dcomplexTauarray, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const float* A, int lda, const float* x, int incx, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSdgmm(hipblasHandle_t handle, hipblasSideMode_t side, int m, int n, const float* AP, int lda, const float* x, int incx, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSdgmm(blasHandle, blasSideMode, m, n, &fa, lda, &fx, incx, &fC, ldc);
  blasStatus = hipblasSdgmm(blasHandle, blasSideMode, m, n, &fa, lda, &fx, incx, &fC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const double* A, int lda, const double* x, int incx, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDdgmm(hipblasHandle_t handle, hipblasSideMode_t side, int m, int n, const double* AP, int lda, const double* x, int incx, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDdgmm(blasHandle, blasSideMode, m, n, &da, lda, &dx, incx, &dC, ldc);
  blasStatus = hipblasDdgmm(blasHandle, blasSideMode, m, n, &da, lda, &dx, incx, &dC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCdgmm_v2(hipblasHandle_t handle, hipblasSideMode_t side, int m, int n, const hipComplex* AP, int lda, const hipComplex* x, int incx, hipComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCdgmm_v2(blasHandle, blasSideMode, m, n, &complexa, lda, &complexx, incx, &complexC, ldc);
  blasStatus = hipblasCdgmm(blasHandle, blasSideMode, m, n, &complexa, lda, &complexx, incx, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdgmm_v2(hipblasHandle_t handle, hipblasSideMode_t side, int m, int n, const hipDoubleComplex* AP, int lda, const hipDoubleComplex* x, int incx, hipDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZdgmm_v2(blasHandle, blasSideMode, m, n, &dcomplexa, lda, &dcomplexx, incx, &dcomplexC, ldc);
  blasStatus = hipblasZdgmm(blasHandle, blasSideMode, m, n, &dcomplexa, lda, &dcomplexx, incx, &dcomplexC, ldc);

  int deviceInfo = 0;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgelsBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int nrhs, float* const Aarray[], int lda, float* const Carray[], int ldc, int* info, int* devInfoArray, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgelsBatched(hipblasHandle_t handle, hipblasOperation_t trans, const int m, const int n, const int nrhs, float* const A[], const int lda, float* const B[], const int ldb, int* info, int* deviceInfo, const int batchCount);
  // CHECK: blasStatus = hipblasSgelsBatched(blasHandle, blasOperation, m, n, nrhs, fAarray, lda, fCarray, ldc, &info, &deviceInfo, batchCount);
  blasStatus = hipblasSgelsBatched(blasHandle, blasOperation, m, n, nrhs, fAarray, lda, fCarray, ldc, &info, &deviceInfo, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgelsBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int nrhs, double* const Aarray[], int lda, double* const Carray[], int ldc, int* info, int* devInfoArray, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgelsBatched(hipblasHandle_t handle, hipblasOperation_t trans, const int m, const int n, const int nrhs, double* const A[], const int lda, double* const B[], const int ldb, int* info, int* deviceInfo, const int batchCount);
  // CHECK: blasStatus = hipblasDgelsBatched(blasHandle, blasOperation, m, n, nrhs, dAarray, lda, dCarray, ldc, &info, &deviceInfo, batchCount);
  blasStatus = hipblasDgelsBatched(blasHandle, blasOperation, m, n, nrhs, dAarray, lda, dCarray, ldc, &info, &deviceInfo, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgelsBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int nrhs, cuComplex* const Aarray[], int lda, cuComplex* const Carray[], int ldc, int* info, int* devInfoArray, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgelsBatched_v2(hipblasHandle_t handle, hipblasOperation_t trans, const int m, const int n, const int nrhs, hipComplex* const A[], const int lda, hipComplex* const B[], const int ldb, int* info, int* deviceInfo, const int batchCount);
  // CHECK: blasStatus = hipblasCgelsBatched_v2(blasHandle, blasOperation, m, n, nrhs, complexAarray, lda, complexCarray, ldc, &info, &deviceInfo, batchCount);
  blasStatus = hipblasCgelsBatched(blasHandle, blasOperation, m, n, nrhs, complexAarray, lda, complexCarray, ldc, &info, &deviceInfo, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgelsBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int nrhs, cuDoubleComplex* const Aarray[], int lda, cuDoubleComplex* const Carray[], int ldc, int* info, int* devInfoArray, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgelsBatched_v2(hipblasHandle_t handle, hipblasOperation_t trans, const int m, const int n, const int nrhs, hipDoubleComplex* const A[], const int lda, hipDoubleComplex* const B[], const int ldb, int* info, int* deviceInfo, const int batchCount);
  // CHECK: blasStatus = hipblasZgelsBatched_v2(blasHandle, blasOperation, m, n, nrhs, dcomplexAarray, lda, dcomplexCarray, ldc, &info, &deviceInfo, batchCount);
  blasStatus = hipblasZgelsBatched(blasHandle, blasOperation, m, n, nrhs, dcomplexAarray, lda, dcomplexCarray, ldc, &info, &deviceInfo, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const float* alpha, const float* A, int lda, const float* B, int ldb, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrmm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const float* alpha, const float* A, int lda, const float* B, int ldb, float* C, int ldc);
  // CHECK: blasStatus = hipblasStrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb, &fC, ldc);
  // CHECK-NEXT: blasStatus = hipblasStrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb, &fC, ldc);
  blasStatus = hipblasStrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb, &fC, ldc);
  blasStatus = hipblasStrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb, &fC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const double* alpha, const double* A, int lda, const double* B, int ldb, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrmm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const double* alpha, const double* A, int lda, const double* B, int ldb, double* C, int ldc);
  // CHECK: blasStatus = hipblasDtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb, &dC, ldc);
  // CHECK-NEXT: blasStatus = hipblasDtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb, &dC, ldc);
  blasStatus = hipblasDtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb, &dC, ldc);
  blasStatus = hipblasDtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb, &dC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrmm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const hipComplex* alpha, const hipComplex* A, int lda, const hipComplex* B, int ldb, hipComplex* C, int ldc);
  // CHECK: blasStatus = hipblasCtrmm_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasCtrmm_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexC, ldc);
  blasStatus = hipblasCtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexC, ldc);
  blasStatus = hipblasCtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrmm_v2(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* A, int lda, const hipDoubleComplex* B, int ldb, hipDoubleComplex* C, int ldc);
  // CHECK: blasStatus = hipblasZtrmm_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = hipblasZtrmm_v2(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexC, ldc);
  blasStatus = hipblasZtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexC, ldc);
  blasStatus = hipblasZtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexC, ldc);

  long long int strideA = 0;
  long long int strideB = 0;
  long long int strideC = 0;

#if CUDA_VERSION >= 7050
  // CHECK: __half* ha = 0;
  __half* ha = 0;
  // CHECK: __half* hA = 0;
  __half* hA = 0;
  // CHECK: __half* hb = 0;
  __half* hb = 0;
  // CHECK: __half* hB = 0;
  __half* hB = 0;
  // CHECK: __half* hc = 0;
  __half* hc = 0;
  // CHECK: __half* hC = 0;
  __half* hC = 0;

  // CHECK: __half** hAarray = 0;
  __half** hAarray = 0;
  // CHECK: const __half** const hAarray_const = const_cast<const __half**>(hAarray);
  const __half** const hAarray_const = const_cast<const __half**>(hAarray);
  // CHECK: __half** hBarray = 0;
  __half** hBarray = 0;
  // CHECK: const __half** const hBarray_const = const_cast<const __half**>(hBarray);
  const __half** const hBarray_const = const_cast<const __half**>(hBarray);
  // CHECK: __half** hCarray = 0;
  __half** hCarray = 0;
  // CHECK: const __half** const hCarray_const = const_cast<const __half**>(hCarray);
  const __half** const hCarray_const = const_cast<const __half**>(hCarray);
  // CHECK: __half** hxarray = 0;
  __half** hxarray = 0;
  // CHECK: const __half** const hxarray_const = const_cast<const __half**>(hxarray_const);
  const __half** const hxarray_const = const_cast<const __half**>(hxarray_const);
  // CHECK: __half** hyarray = 0;
  __half** hyarray = 0;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemm(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* A, int lda, const __half* B, int ldb, const __half* beta, __half* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasHgemm(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasHalf* alpha, const hipblasHalf* AP, int lda, const hipblasHalf* BP, int ldb, const hipblasHalf* beta, hipblasHalf* CP, int ldc);
  // CHECK: blasStatus = hipblasHgemm(blasHandle, transa, transb, m, n, k, ha, hA, lda, hB, ldb, hb, hC, ldc);
  blasStatus = hipblasHgemm(blasHandle, transa, transb, m, n, k, ha, hA, lda, hB, ldb, hb, hC, ldc);
#endif

#if CUDA_VERSION >= 8000
  // CHECK: hipDataType DataType;
  // CHECK-NEXT: hipDataType DataType_t;
  // CHECK-NEXT: hipDataType blasDataType;
  // CHECK-NEXT: hipDataType R_16F = HIP_R_16F;
  // CHECK-NEXT: hipDataType C_16F = HIP_C_16F;
  // CHECK-NEXT: hipDataType R_32F = HIP_R_32F;
  // CHECK-NEXT: hipDataType C_32F = HIP_C_32F;
  // CHECK-NEXT: hipDataType R_64F = HIP_R_64F;
  // CHECK-NEXT: hipDataType C_64F = HIP_C_64F;
  // CHECK-NEXT: hipDataType R_8I = HIP_R_8I;
  // CHECK-NEXT: hipDataType C_8I = HIP_C_8I;
  // CHECK-NEXT: hipDataType R_8U = HIP_R_8U;
  // CHECK-NEXT: hipDataType C_8U = HIP_C_8U;
  // CHECK-NEXT: hipDataType R_32I = HIP_R_32I;
  // CHECK-NEXT: hipDataType C_32I = HIP_C_32I;
  // CHECK-NEXT: hipDataType R_32U = HIP_R_32U;
  // CHECK-NEXT: hipDataType C_32U = HIP_C_32U;
  hipDataType DataType;
  hipDataType DataType_t;
  hipDataType blasDataType;
  hipDataType R_16F = HIP_R_16F;
  hipDataType C_16F = HIP_C_16F;
  hipDataType R_32F = HIP_R_32F;
  hipDataType C_32F = HIP_C_32F;
  hipDataType R_64F = HIP_R_64F;
  hipDataType C_64F = HIP_C_64F;
  hipDataType R_8I = HIP_R_8I;
  hipDataType C_8I = HIP_C_8I;
  hipDataType R_8U = HIP_R_8U;
  hipDataType C_8U = HIP_C_8U;
  hipDataType R_32I = HIP_R_32I;
  hipDataType C_32I = HIP_C_32I;
  hipDataType R_32U = HIP_R_32U;
  hipDataType C_32U = HIP_C_32U;

  // CHECK: hipDataType DataType_2, DataType_3;
  hipDataType DataType_2, DataType_3;

  // CHECK: hipblasGemmAlgo_t blasGemmAlgo;
  // CHECK-NEXT: hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;
  hipblasGemmAlgo_t blasGemmAlgo;
  hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasNrm2Ex(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, void* result, cudaDataType resultType, cudaDataType executionType);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasNrm2Ex_v2(hipblasHandle_t handle, int n, const void* x, hipDataType xType, int incx, void* result, hipDataType resultType, hipDataType executionType);
  // CHECK: blasStatus = hipblasNrm2Ex_v2(blasHandle, n, image, DataType, incx, image_2, DataType_2, DataType_3);
  blasStatus = hipblasNrm2Ex(blasHandle, n, image, DataType, incx, image_2, DataType_2, DataType_3);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* A, int lda, long long int strideA, const float* B, int ldb, long long int strideB, const float* beta, float* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgemmStridedBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const float* alpha, const float* AP, int lda, long long strideA, const float* BP, int ldb, long long strideB, const float* beta, float* CP, int ldc, long long strideC, int batchCount);
  // CHECK: blasStatus = hipblasSgemmStridedBatched(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, strideA, &fB, ldb, strideB, &fb, &fC, ldc, strideC, batchCount);
  blasStatus = hipblasSgemmStridedBatched(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, strideA, &fB, ldb, strideB, &fb, &fC, ldc, strideC, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* A, int lda, long long int strideA, const double* B, int ldb, long long int strideB, const double* beta, double* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgemmStridedBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const double* alpha, const double* AP, int lda, long long strideA, const double* BP, int ldb, long long strideB, const double* beta, double* CP, int ldc, long long strideC, int batchCount);
  // CHECK: blasStatus = hipblasDgemmStridedBatched(blasHandle, transa, transb, m, n, k, &da, &dA, lda, strideA, &dB, ldb, strideB, &db, &dC, ldc, strideC, batchCount);
  blasStatus = hipblasDgemmStridedBatched(blasHandle, transa, transb, m, n, k, &da, &dA, lda, strideA, &dB, ldb, strideB, &db, &dC, ldc, strideC, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, long long int strideA, const cuComplex* B, int ldb, long long int strideB, const cuComplex* beta, cuComplex* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemmStridedBatched_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipComplex* alpha, const hipComplex* AP, int lda, long long strideA, const hipComplex* BP, int ldb, long long strideB, const hipComplex* beta, hipComplex* CP, int ldc, long long strideC, int batchCount);
  // CHECK: blasStatus = hipblasCgemmStridedBatched_v2(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, strideA, &complexB, ldb, strideB, &complexb, &complexC, ldc, strideC, batchCount);
  blasStatus = hipblasCgemmStridedBatched(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, strideA, &complexB, ldb, strideB, &complexb, &complexC, ldc, strideC, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, long long int strideA, const cuDoubleComplex* B, int ldb, long long int strideB, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemmStridedBatched_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, int lda, long long strideA, const hipDoubleComplex* BP, int ldb, long long strideB, const hipDoubleComplex* beta, hipDoubleComplex* CP, int ldc, long long strideC, int batchCount);
  // CHECK: blasStatus = hipblasZgemmStridedBatched_v2(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, strideA, &dcomplexB, ldb, strideB, &dcomplexb, &dcomplexC, ldc, strideC, batchCount);
  blasStatus = hipblasZgemmStridedBatched(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, strideA, &dcomplexB, ldb, strideB, &dcomplexb, &dcomplexC, ldc, strideC, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* A, int lda, long long int strideA, const __half* B, int ldb, long long int strideB, const __half* beta, __half* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasHgemmStridedBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasHalf* alpha, const hipblasHalf* AP, int lda, long long strideA, const hipblasHalf* BP, int ldb, long long strideB, const hipblasHalf* beta, hipblasHalf* CP, int ldc, long long strideC, int batchCount);
  // CHECK: blasStatus = hipblasHgemmStridedBatched(blasHandle, transa, transb, m, n, k, ha, hA, lda, strideA, hB, ldb, strideB, hb, hC, ldc, strideC, batchCount);
  blasStatus = hipblasHgemmStridedBatched(blasHandle, transa, transb, m, n, k, ha, hA, lda, strideA, hB, ldb, strideB, hb, hC, ldc, strideC, batchCount);

  void* aptr = nullptr;
  void* Aptr = nullptr;
  void* bptr = nullptr;
  void* Bptr = nullptr;
  void* cptr = nullptr;
  void* Cptr = nullptr;
  void* xptr = nullptr;
  void* yptr = nullptr;
  void* sptr = nullptr;

  // CHECK: hipDataType Atype;
  // CHECK-NEXT: hipDataType Btype;
  // CHECK-NEXT: hipDataType Ctype;
  // CHECK-NEXT: hipDataType Xtype;
  // CHECK-NEXT: hipDataType Ytype;
  // CHECK-NEXT: hipDataType CStype;
  // CHECK-NEXT: hipDataType Executiontype;
  hipDataType Atype;
  hipDataType Btype;
  hipDataType Ctype;
  hipDataType Xtype;
  hipDataType Ytype;
  hipDataType CStype;
  hipDataType Executiontype;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScalEx(cublasHandle_t handle, int n, const void* alpha, cudaDataType alphaType, void* x, cudaDataType xType, int incx, cudaDataType executionType);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScalEx_v2(hipblasHandle_t handle, int n, const void* alpha, hipDataType alphaType, void* x, hipDataType xType, int incx, hipDataType executionType);
  // CHECK: blasStatus = hipblasScalEx_v2(blasHandle, n, aptr, Atype, xptr, Xtype, incx, Executiontype);
  blasStatus = hipblasScalEx(blasHandle, n, aptr, Atype, xptr, Xtype, incx, Executiontype);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasAxpyEx(cublasHandle_t handle, int n, const void* alpha, cudaDataType alphaType, const void* x, cudaDataType xType, int incx, void* y, cudaDataType yType, int incy, cudaDataType executiontype);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasAxpyEx_v2(hipblasHandle_t handle, int n, const void* alpha, hipDataType alphaType, const void* x, hipDataType xType, int incx, void* y, hipDataType yType, int incy, hipDataType executionType);
  // CHECK: blasStatus = hipblasAxpyEx_v2(blasHandle, n, aptr, Atype, xptr, Xtype, incx, yptr, Ytype, incy, Executiontype);
  blasStatus = hipblasAxpyEx(blasHandle, n, aptr, Atype, xptr, Xtype, incx, yptr, Ytype, incy, Executiontype);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDotEx(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, const void* y, cudaDataType yType, int incy, void* result, cudaDataType resultType, cudaDataType executionType);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDotEx_v2(hipblasHandle_t handle, int n, const void* x, hipDataType xType, int incx, const void* y, hipDataType yType, int incy, void* result, hipDataType resultType, hipDataType executionType);
  // CHECK: blasStatus = hipblasDotEx_v2(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
  blasStatus = hipblasDotEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDotcEx(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, const void* y, cudaDataType yType, int incy, void* result, cudaDataType resultType, cudaDataType executionType);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDotcEx_v2(hipblasHandle_t handle, int n, const void* x, hipDataType xType, int incx, const void* y, hipDataType yType, int incy, void* result, hipDataType resultType, hipDataType executionType);
  // CHECK: blasStatus = hipblasDotcEx_v2(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
  blasStatus = hipblasDotcEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
#endif

#if CUDA_VERSION >= 8000 && CUDA_VERSION < 11000
  // CHECK: hipDataType computeType;
  cudaDataType computeType;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, const void* B, cudaDataType Btype, int ldb, const void* beta, void* C, cudaDataType Ctype, int ldc, cudaDataType computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmEx_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A, hipDataType aType, int lda, const void* B, hipDataType bType, int ldb, const void* beta, void* C, hipDataType cType, int ldc, hipblasComputeType_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmEx_v2(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, computeType, blasGemmAlgo);
  blasStatus = cublasGemmEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, computeType, blasGemmAlgo);
#endif

#if CUDA_VERSION >= 9000
  // CHECK: hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;
  hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* const Aarray[], int lda, const __half* const Barray[], int ldb, const __half* beta, __half* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasHgemmBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasHalf* alpha, const hipblasHalf* const AP[], int lda, const hipblasHalf* const BP[], int ldb, const hipblasHalf* beta, hipblasHalf* const CP[], int ldc, int batchCount);
  // CHECK: blasStatus = hipblasHgemmBatched(blasHandle, transa, transb, m, n, k, ha, hAarray_const, lda, hBarray_const, ldb, hb, hCarray, ldc, batchCount);
  blasStatus = hipblasHgemmBatched(blasHandle, transa, transb, m, n, k, ha, hAarray_const, lda, hBarray_const, ldb, hb, hCarray, ldc, batchCount);
#endif

#if CUDA_VERSION >= 9010 && CUDA_VERSION < 11000
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* const Aarray[], cudaDataType Atype, int lda, const void* const Barray[], cudaDataType Btype, int ldb, const void* beta, void* const Carray[], cudaDataType Ctype, int ldc, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmBatchedEx_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A[], hipDataType aType, int lda, const void* B[], hipDataType bType, int ldb, const void* beta, void* C[], hipDataType cType, int ldc, int batchCount, hipblasComputeType_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmBatchedEx_v2(blasHandle, transa, transb, m, n, k, aptr, voidAarray_const, Atype, lda, voidBarray_const, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, computeType, blasGemmAlgo);
  blasStatus = cublasGemmBatchedEx(blasHandle, transa, transb, m, n, k, aptr, voidAarray_const, Atype, lda, voidBarray_const, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, computeType, blasGemmAlgo);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmStridedBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, long long int strideA, const void* B, cudaDataType Btype, int ldb, long long int strideB, const void* beta, void* C, cudaDataType Ctype, int ldc, long long int strideC, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmStridedBatchedEx_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A, hipDataType aType, int lda, hipblasStride strideA, const void* B, hipDataType bType, int ldb, hipblasStride strideB, const void* beta, void* C, hipDataType cType, int ldc, hipblasStride strideC, int batchCount, hipblasComputeType_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmStridedBatchedEx_v2(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, computeType, blasGemmAlgo);
  blasStatus = cublasGemmStridedBatchedEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, computeType, blasGemmAlgo);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;
  hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;

  // CHECK: hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;
  hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasRotEx(cublasHandle_t handle, int n, void* x, cudaDataType xType, int incx, void* y, cudaDataType yType, int incy, const void* c, const void* s, cudaDataType csType, cudaDataType executiontype);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasRotEx_v2(hipblasHandle_t handle, int n, void* x, hipDataType xType, int incx, void* y, hipDataType yType, int incy, const void* c, const void* s, hipDataType csType, hipDataType executionType);
  // CHECK: blasStatus = hipblasRotEx_v2(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, cptr, sptr, CStype, Executiontype);
  blasStatus = hipblasRotEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, cptr, sptr, CStype, Executiontype);
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipDataType R_16BF = HIP_R_16BF;
  // CHECK-NEXT: hipDataType C_16BF = HIP_C_16BF;
  hipDataType R_16BF = HIP_R_16BF;
  hipDataType C_16BF = HIP_C_16BF;

  // CHECK: hip_bfloat16** bf16Aarray = 0;
  __hip_bfloat16** bf16Aarray = 0;
  // CHECK: const hip_bfloat16** const bf16Aarray_const = const_cast<const hip_bfloat16**>(bf16Aarray);
  const __hip_bfloat16** const bf16Aarray_const = const_cast<const __hip_bfloat16**>(bf16Aarray);
  // CHECK: hip_bfloat16** bf16xarray = 0;
  __hip_bfloat16** bf16xarray = 0;
  // CHECK: const hip_bfloat16** const bf16xarray_const = const_cast<const hip_bfloat16**>(bf16xarray_const);
  const __hip_bfloat16** const bf16xarray_const = const_cast<const __hip_bfloat16**>(bf16xarray_const);
  // CHECK: hip_bfloat16** bf16yarray = 0;
  __hip_bfloat16** bf16yarray = 0;

  // CHECK: hipblasComputeType_t blasComputeType;
  hipblasComputeType_t blasComputeType;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, const void* B, cudaDataType Btype, int ldb, const void* beta, void* C, cudaDataType Ctype, int ldc, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmEx_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A, hipDataType aType, int lda, const void* B, hipDataType bType, int ldb, const void* beta, void* C, hipDataType cType, int ldc, hipblasComputeType_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmEx_v2(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, blasComputeType, blasGemmAlgo);
  blasStatus = hipblasGemmEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, blasComputeType, blasGemmAlgo);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* const Aarray[], cudaDataType Atype, int lda, const void* const Barray[], cudaDataType Btype, int ldb, const void* beta, void* const Carray[], cudaDataType Ctype, int ldc, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmBatchedEx_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A[], hipDataType aType, int lda, const void* B[], hipDataType bType, int ldb, const void* beta, void* C[], hipDataType cType, int ldc, int batchCount, hipblasComputeType_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmBatchedEx_v2(blasHandle, transa, transb, m, n, k, aptr, voidAarray, Atype, lda, voidBarray, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, blasComputeType, blasGemmAlgo);
  blasStatus = hipblasGemmBatchedEx(blasHandle, transa, transb, m, n, k, aptr, voidAarray, Atype, lda, voidBarray, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, blasComputeType, blasGemmAlgo);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmStridedBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, long long int strideA, const void* B, cudaDataType Btype, int ldb, long long int strideB, const void* beta, void* C, cudaDataType Ctype, int ldc, long long int strideC, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmStridedBatchedEx_v2(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A, hipDataType aType, int lda, hipblasStride strideA, const void* B, hipDataType bType, int ldb, hipblasStride strideB, const void* beta, void* C, hipDataType cType, int ldc, hipblasStride strideC, int batchCount, hipblasComputeType_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmStridedBatchedEx_v2(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, blasComputeType, blasGemmAlgo);
  blasStatus = hipblasGemmStridedBatchedEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, blasComputeType, blasGemmAlgo);
#endif

#if CUDA_VERSION >= 12000
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIsamax_v2_64(cublasHandle_t handle, int64_t n, const float* x, int64_t incx, int64_t* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIsamax_64(hipblasHandle_t handle, int64_t n, const float* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = hipblasIsamax_64(blasHandle, n_64, &fx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = hipblasIsamax_64(blasHandle, n_64, &fx, incx_64, &res_64);
  blasStatus = hipblasIsamax_64(blasHandle, n_64, &fx, incx_64, &res_64);
  blasStatus = hipblasIsamax_64(blasHandle, n_64, &fx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIdamax_v2_64(cublasHandle_t handle, int64_t n, const double* x, int64_t incx, int64_t* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIdamax_64(hipblasHandle_t handle, int64_t n, const double* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = hipblasIdamax_64(blasHandle, n_64, &dx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = hipblasIdamax_64(blasHandle, n_64, &dx, incx_64, &res_64);
  blasStatus = hipblasIdamax_64(blasHandle, n_64, &dx, incx_64, &res_64);
  blasStatus = hipblasIdamax_64(blasHandle, n_64, &dx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIcamax_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* x, int64_t incx, int64_t* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIcamax_v2_64(hipblasHandle_t handle, int64_t n, const hipComplex* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = hipblasIcamax_v2_64(blasHandle, n_64, &complexx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = hipblasIcamax_v2_64(blasHandle, n_64, &complexx, incx_64, &res_64);
  blasStatus = hipblasIcamax_64(blasHandle, n_64, &complexx, incx_64, &res_64);
  blasStatus = hipblasIcamax_64(blasHandle, n_64, &complexx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIzamax_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* x, int64_t incx, int64_t* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIzamax_v2_64(hipblasHandle_t handle, int64_t n, const hipDoubleComplex* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = hipblasIzamax_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = hipblasIzamax_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);
  blasStatus = hipblasIzamax_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);
  blasStatus = hipblasIzamax_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIsamin_v2_64(cublasHandle_t handle, int64_t n, const float* x, int64_t incx, int64_t* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIsamin_64(hipblasHandle_t handle, int64_t n, const float* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = hipblasIsamin_64(blasHandle, n_64, &fx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = hipblasIsamin_64(blasHandle, n_64, &fx, incx_64, &res_64);
  blasStatus = hipblasIsamin_64(blasHandle, n_64, &fx, incx_64, &res_64);
  blasStatus = hipblasIsamin_64(blasHandle, n_64, &fx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIdamin_v2_64(cublasHandle_t handle, int64_t n, const double* x, int64_t incx, int64_t* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIdamin_64(hipblasHandle_t handle, int64_t n, const double* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = hipblasIdamin_64(blasHandle, n_64, &dx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = hipblasIdamin_64(blasHandle, n_64, &dx, incx_64, &res_64);
  blasStatus = hipblasIdamin_64(blasHandle, n_64, &dx, incx_64, &res_64);
  blasStatus = hipblasIdamin_64(blasHandle, n_64, &dx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIcamin_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* x, int64_t incx, int64_t* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIcamin_v2_64(hipblasHandle_t handle, int64_t n, const hipComplex* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = hipblasIcamin_v2_64(blasHandle, n_64, &complexx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = hipblasIcamin_v2_64(blasHandle, n_64, &complexx, incx_64, &res_64);
  blasStatus = hipblasIcamin_64(blasHandle, n_64, &complexx, incx_64, &res_64);
  blasStatus = hipblasIcamin_64(blasHandle, n_64, &complexx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIzamin_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* x, int64_t incx, int64_t* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIzamin_v2_64(hipblasHandle_t handle, int64_t n, const hipDoubleComplex* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = hipblasIzamin_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = hipblasIzamin_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);
  blasStatus = hipblasIzamin_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);
  blasStatus = hipblasIzamin_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSasum_v2_64(cublasHandle_t handle, int64_t n, const float* x, int64_t incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSasum_64(hipblasHandle_t handle, int64_t n, const float* x, int64_t incx, float* result);
  // CHECK: blasStatus = hipblasSasum_64(blasHandle, n_64, &fx, incx_64, &fresult);
  // CHECK-NEXT: blasStatus = hipblasSasum_64(blasHandle, n_64, &fx, incx_64, &fresult);
  blasStatus = hipblasSasum_64(blasHandle, n_64, &fx, incx_64, &fresult);
  blasStatus = hipblasSasum_64(blasHandle, n_64, &fx, incx_64, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDasum_v2_64(cublasHandle_t handle, int64_t n, const double* x, int64_t incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDasum_64(hipblasHandle_t handle, int64_t n, const double* x, int64_t incx, double* result);
  // CHECK: blasStatus = hipblasDasum_64(blasHandle, n_64, &dx, incx_64, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDasum_64(blasHandle, n_64, &dx, incx_64, &dresult);
  blasStatus = hipblasDasum_64(blasHandle, n_64, &dx, incx_64, &dresult);
  blasStatus = hipblasDasum_64(blasHandle, n_64, &dx, incx_64, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScasum_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* x, int64_t incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScasum_v2_64(hipblasHandle_t handle, int64_t n, const hipComplex* x, int64_t incx, float* result);
  // CHECK: blasStatus = hipblasScasum_v2_64(blasHandle, n_64, &complexx, incx_64, &fresult);
  // CHECK-NEXT: blasStatus = hipblasScasum_v2_64(blasHandle, n_64, &complexx, incx_64, &fresult);
  blasStatus = hipblasScasum_64(blasHandle, n_64, &complexx, incx_64, &fresult);
  blasStatus = hipblasScasum_64(blasHandle, n_64, &complexx, incx_64, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDzasum_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* x, int64_t incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDzasum_v2_64(hipblasHandle_t handle, int64_t n, const hipDoubleComplex* x, int64_t incx, double* result);
  // CHECK: blasStatus = hipblasDzasum_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDzasum_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);
  blasStatus = hipblasDzasum_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);
  blasStatus = hipblasDzasum_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSaxpy_v2_64(cublasHandle_t handle, int64_t n, const float* alpha, const float* x, int64_t incx, float* y, int64_t incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSaxpy_64(hipblasHandle_t handle, int64_t n, const float* alpha, const float* x, int64_t incx, float* y, int64_t incy);
  // CHECK: blasStatus = hipblasSaxpy_64(blasHandle, n_64, &fa, &fx, incx_64, &fy, incy_64);
  // CHECK-NEXT: blasStatus = hipblasSaxpy_64(blasHandle, n_64, &fa, &fx, incx_64, &fy, incy_64);
  blasStatus = hipblasSaxpy_64(blasHandle, n_64, &fa, &fx, incx_64, &fy, incy_64);
  blasStatus = hipblasSaxpy_64(blasHandle, n_64, &fa, &fx, incx_64, &fy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDaxpy_v2_64(cublasHandle_t handle, int64_t n, const double* alpha, const double* x, int64_t incx, double* y, int64_t incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDaxpy_64(hipblasHandle_t handle, int64_t n, const double* alpha, const double* x, int64_t incx, double* y, int64_t incy);
  // CHECK: blasStatus = hipblasDaxpy_64(blasHandle, n_64, &da, &dx, incx_64, &dy, incy_64);
  // CHECK-NEXT: blasStatus = hipblasDaxpy_64(blasHandle, n_64, &da, &dx, incx_64, &dy, incy_64);
  blasStatus = hipblasDaxpy_64(blasHandle, n_64, &da, &dx, incx_64, &dy, incy_64);
  blasStatus = hipblasDaxpy_64(blasHandle, n_64, &da, &dx, incx_64, &dy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCaxpy_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* alpha, const cuComplex* x, int64_t incx, cuComplex* y, int64_t incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCaxpy_v2_64(hipblasHandle_t handle, int64_t n, const hipComplex* alpha, const hipComplex* x, int64_t incx, hipComplex* y, int64_t incy);
  // CHECK: blasStatus = hipblasCaxpy_v2_64(blasHandle, n_64, &complexa, &complexx, incx_64, &complexy, incy_64);
  // CHECK-NEXT: blasStatus = hipblasCaxpy_v2_64(blasHandle, n_64, &complexa, &complexx, incx_64, &complexy, incy_64);
  blasStatus = hipblasCaxpy_64(blasHandle, n_64, &complexa, &complexx, incx_64, &complexy, incy_64);
  blasStatus = hipblasCaxpy_64(blasHandle, n_64, &complexa, &complexx, incx_64, &complexy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZaxpy_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int64_t incx, cuDoubleComplex* y, int64_t incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZaxpy_v2_64(hipblasHandle_t handle, int64_t n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int64_t incx, hipDoubleComplex* y, int64_t incy);
  // CHECK: blasStatus = hipblasZaxpy_v2_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64, &dcomplexy, incy_64);
  // CHECK-NEXT: blasStatus = hipblasZaxpy_v2_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64, &dcomplexy, incy_64);
  blasStatus = hipblasZaxpy_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64, &dcomplexy, incy_64);
  blasStatus = hipblasZaxpy_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64, &dcomplexy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScopy_v2_64(cublasHandle_t handle, int64_t n, const float* x, int64_t incx, float* y, int64_t incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScopy_64(hipblasHandle_t handle, int64_t n, const float* x, int64_t incx, float* y, int64_t incy);
  // CHECK: blasStatus = hipblasScopy_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);
  // CHECK-NEXT: blasStatus = hipblasScopy_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);
  blasStatus = hipblasScopy_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);
  blasStatus = hipblasScopy_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDcopy_v2_64(cublasHandle_t handle, int64_t n, const double* x, int64_t incx, double* y, int64_t incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDcopy_64(hipblasHandle_t handle, int64_t n, const double* x, int64_t incx, double* y, int64_t incy);
  // CHECK: blasStatus = hipblasDcopy_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);
  // CHECK-NEXT: blasStatus = hipblasDcopy_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);
  blasStatus = hipblasDcopy_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);
  blasStatus = hipblasDcopy_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCcopy_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* x, int64_t incx, cuComplex* y, int64_t incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCcopy_v2_64(hipblasHandle_t handle, int64_t n, const hipComplex* x, int64_t incx, hipComplex* y, int64_t incy);
  // CHECK: blasStatus = hipblasCcopy_v2_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);
  // CHECK-NEXT: blasStatus = hipblasCcopy_v2_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);
  blasStatus = hipblasCcopy_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);
  blasStatus = hipblasCcopy_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZcopy_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* x, int64_t incx, cuDoubleComplex* y, int64_t incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZcopy_v2_64(hipblasHandle_t handle, int64_t n, const hipDoubleComplex* x, int64_t incx, hipDoubleComplex* y, int64_t incy);
  // CHECK: blasStatus = hipblasZcopy_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);
  // CHECK-NEXT: blasStatus = hipblasZcopy_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);
  blasStatus = hipblasZcopy_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);
  blasStatus = hipblasZcopy_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSdot_v2_64(cublasHandle_t handle, int64_t n, const float* x, int64_t incx, const float* y, int64_t incy, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSdot_64(hipblasHandle_t handle, int64_t n, const float* x, int64_t incx, const float* y, int64_t incy, float* result);
  // CHECK: blasStatus = hipblasSdot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fresult);
  // CHECK-NEXT: blasStatus = hipblasSdot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fresult);
  blasStatus = hipblasSdot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fresult);
  blasStatus = hipblasSdot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDdot_v2_64(cublasHandle_t handle, int64_t n, const double* x, int64_t incx, const double* y, int64_t incy, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDdot_64(hipblasHandle_t handle, int64_t n, const double* x, int64_t incx, const double* y, int64_t incy, double* result);
  // CHECK: blasStatus = hipblasDdot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDdot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dresult);
  blasStatus = hipblasDdot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dresult);
  blasStatus = hipblasDdot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdotc_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* x, int64_t incx, const cuComplex* y, int64_t incy, cuComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCdotc_v2_64(hipblasHandle_t handle, int64_t n, const hipComplex* x, int64_t incx, const hipComplex* y, int64_t incy, hipComplex* result);
  // CHECK: blasStatus = hipblasCdotc_v2_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);
  // CHECK-NEXT: blasStatus = hipblasCdotc_v2_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);
  blasStatus = hipblasCdotc_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);
  blasStatus = hipblasCdotc_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdotu_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* x, int64_t incx, const cuComplex* y, int64_t incy, cuComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCdotu_v2_64(hipblasHandle_t handle, int64_t n, const hipComplex* x, int64_t incx, const hipComplex* y, int64_t incy, hipComplex* result);
  // CHECK: blasStatus = hipblasCdotu_v2_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);
  // CHECK-NEXT: blasStatus = hipblasCdotu_v2_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);
  blasStatus = hipblasCdotu_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);
  blasStatus = hipblasCdotu_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdotc_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* x, int64_t incx, const cuDoubleComplex* y, int64_t incy, cuDoubleComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdotc_v2_64(hipblasHandle_t handle, int64_t n, const hipDoubleComplex* x, int64_t incx, const hipDoubleComplex* y, int64_t incy, hipDoubleComplex* result);
  // CHECK: blasStatus = hipblasZdotc_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);
  // CHECK-NEXT: blasStatus = hipblasZdotc_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);
  blasStatus = hipblasZdotc_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);
  blasStatus = hipblasZdotc_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdotu_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* x, int64_t incx, const cuDoubleComplex* y, int64_t incy, cuDoubleComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdotu_v2_64(hipblasHandle_t handle, int64_t n, const hipDoubleComplex* x, int64_t incx, const hipDoubleComplex* y, int64_t incy, hipDoubleComplex* result);
  // CHECK: blasStatus = hipblasZdotu_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);
  // CHECK-NEXT: blasStatus = hipblasZdotu_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);
  blasStatus = hipblasZdotu_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);
  blasStatus = hipblasZdotu_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSnrm2_v2_64(cublasHandle_t handle, int64_t n, const float* x, int64_t incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSnrm2_64(hipblasHandle_t handle, int64_t n, const float* x, int64_t incx, float* result);
  // CHECK: blasStatus = hipblasSnrm2_64(blasHandle, n_64, &fx, incx_64, &fresult);
  // CHECK-NEXT: blasStatus = hipblasSnrm2_64(blasHandle, n_64, &fx, incx_64, &fresult);
  blasStatus = hipblasSnrm2_64(blasHandle, n_64, &fx, incx_64, &fresult);
  blasStatus = hipblasSnrm2_64(blasHandle, n_64, &fx, incx_64, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDnrm2_v2_64(cublasHandle_t handle, int64_t n, const double* x, int64_t incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDnrm2_64(hipblasHandle_t handle, int64_t n, const double* x, int64_t incx, double* result);
  // CHECK: blasStatus = hipblasDnrm2_64(blasHandle, n_64, &dx, incx_64, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDnrm2_64(blasHandle, n_64, &dx, incx_64, &dresult);
  blasStatus = hipblasDnrm2_64(blasHandle, n_64, &dx, incx_64, &dresult);
  blasStatus = hipblasDnrm2_64(blasHandle, n_64, &dx, incx_64, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScnrm2_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* x, int64_t incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScnrm2_v2_64(hipblasHandle_t handle, int64_t n, const hipComplex* x, int64_t incx, float* result);
  // CHECK: blasStatus = hipblasScnrm2_v2_64(blasHandle, n_64, &complexx, incx_64, &fresult);
  // CHECK-NEXT: blasStatus = hipblasScnrm2_v2_64(blasHandle, n_64, &complexx, incx_64, &fresult);
  blasStatus = hipblasScnrm2_64(blasHandle, n_64, &complexx, incx_64, &fresult);
  blasStatus = hipblasScnrm2_64(blasHandle, n_64, &complexx, incx_64, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDznrm2_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* x, int64_t incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDznrm2_v2_64(hipblasHandle_t handle, int64_t n, const hipDoubleComplex* x, int64_t incx, double* result);
  // CHECK: blasStatus = hipblasDznrm2_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDznrm2_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);
  blasStatus = hipblasDznrm2_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);
  blasStatus = hipblasDznrm2_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrot_v2_64(cublasHandle_t handle, int64_t n, float* x, int64_t incx, float* y, int64_t incy, const float* c, const float* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrot_64(hipblasHandle_t handle, int64_t n,float* x, int64_t incx, float* y, int64_t incy, const float* c, const float* s);
  // CHECK: blasStatus = hipblasSrot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fc, &fs);
  // CHECK-NEXT: blasStatus = hipblasSrot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fc, &fs);
  blasStatus = hipblasSrot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fc, &fs);
  blasStatus = hipblasSrot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fc, &fs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrot_v2_64(cublasHandle_t handle, int64_t n, double* x, int64_t incx, double* y, int64_t incy, const double* c, const double* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrot_64(hipblasHandle_t handle, int64_t n, double* x, int64_t incx, double* y, int64_t incy, const double* c, const double* s);
  // CHECK: blasStatus = hipblasDrot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dc, &ds);
  // CHECK-NEXT: blasStatus = hipblasDrot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dc, &ds);
  blasStatus = hipblasDrot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dc, &ds);
  blasStatus = hipblasDrot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dc, &ds);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCrot_v2_64(cublasHandle_t handle, int64_t n, cuComplex* x, int64_t incx, cuComplex* y, int64_t incy, const float* c, const cuComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCrot_v2_64(hipblasHandle_t handle, int64_t n, hipComplex* x, int64_t incx, hipComplex* y, int64_t incy, const float* c, const hipComplex* s);
  // CHECK: blasStatus = hipblasCrot_v2_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &complexs);
  // CHECK-NEXT: blasStatus = hipblasCrot_v2_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &complexs);
  blasStatus = hipblasCrot_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &complexs);
  blasStatus = hipblasCrot_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &complexs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsrot_v2_64(cublasHandle_t handle, int64_t n, cuComplex* x, int64_t incx, cuComplex* y, int64_t incy, const float* c, const float* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsrot_v2_64(hipblasHandle_t handle, int64_t n, hipComplex* x, int64_t incx, hipComplex* y, int64_t incy, const float* c, const float* s);
  // CHECK: blasStatus = hipblasCsrot_v2_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &fs);
  // CHECK-NEXT: blasStatus = hipblasCsrot_v2_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &fs);
  blasStatus = hipblasCsrot_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &fs);
  blasStatus = hipblasCsrot_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &fs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZrot_v2_64(cublasHandle_t handle, int64_t n, cuDoubleComplex* x, int64_t incx, cuDoubleComplex* y, int64_t incy, const double* c, const cuDoubleComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZrot_v2_64(hipblasHandle_t handle, int64_t n, hipDoubleComplex* x, int64_t incx, hipDoubleComplex* y, int64_t incy, const double* c, const hipDoubleComplex* s);
  // CHECK: blasStatus = hipblasZrot_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &dcomplexs);
  // CHECK-NEXT: blasStatus = hipblasZrot_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &dcomplexs);
  blasStatus = hipblasZrot_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &dcomplexs);
  blasStatus = hipblasZrot_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &dcomplexs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdrot_v2_64(cublasHandle_t handle, int64_t n, cuDoubleComplex* x, int64_t incx, cuDoubleComplex* y, int64_t incy, const double* c, const double* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdrot_v2_64(hipblasHandle_t handle, int64_t n, hipDoubleComplex* x, int64_t incx, hipDoubleComplex* y, int64_t incy, const double* c, const double* s);
  // CHECK: blasStatus = hipblasZdrot_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &ds);
  // CHECK-NEXT: blasStatus = hipblasZdrot_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &ds);
  blasStatus = hipblasZdrot_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &ds);
  blasStatus = hipblasZdrot_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &ds);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotm_v2_64(cublasHandle_t handle, int64_t n, float* x, int64_t incx, float* y, int64_t incy, const float* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrotm_64(hipblasHandle_t handle, int64_t n, float* x, int64_t incx, float* y, int64_t incy, const float* param);
  // CHECK: blasStatus = hipblasSrotm_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fparam);
  // CHECK-NEXT: blasStatus = hipblasSrotm_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fparam);
  blasStatus = hipblasSrotm_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fparam);
  blasStatus = hipblasSrotm_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fparam);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotm_v2_64(cublasHandle_t handle, int64_t n, double* x, int64_t incx, double* y, int64_t incy, const double* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrotm_64(hipblasHandle_t handle, int64_t n, double* x, int64_t incx, double* y, int64_t incy, const double* param);
  // CHECK: blasStatus = hipblasDrotm_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dparam);
  // CHECK-NEXT: blasStatus = hipblasDrotm_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dparam);
  blasStatus = hipblasDrotm_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dparam);
  blasStatus = hipblasDrotm_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dparam);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSscal_v2_64(cublasHandle_t handle, int64_t n, const float* alpha, float* x, int64_t incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSscal_64(hipblasHandle_t handle, int64_t n, const float* alpha, float* x, int64_t incx);
  // CHECK: blasStatus = hipblasSscal_64(blasHandle, n_64, &fa, &fx, incx_64);
  // CHECK-NEXT: blasStatus = hipblasSscal_64(blasHandle, n_64, &fa, &fx, incx_64);
  blasStatus = hipblasSscal_64(blasHandle, n_64, &fa, &fx, incx_64);
  blasStatus = hipblasSscal_64(blasHandle, n_64, &fa, &fx, incx_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDscal_v2_64(cublasHandle_t handle, int64_t n, const double* alpha, double* x, int64_t incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDscal_64(hipblasHandle_t handle, int64_t n, const double* alpha, double* x, int64_t incx);
  // CHECK: blasStatus = hipblasDscal_64(blasHandle, n_64, &da, &dx, incx_64);
  // CHECK-NEXT: blasStatus = hipblasDscal_64(blasHandle, n_64, &da, &dx, incx_64);
  blasStatus = hipblasDscal_64(blasHandle, n_64, &da, &dx, incx_64);
  blasStatus = hipblasDscal_64(blasHandle, n_64, &da, &dx, incx_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCscal_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* alpha, cuComplex* x, int64_t incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCscal_v2_64(hipblasHandle_t handle, int64_t n, const hipComplex* alpha, hipComplex* x, int64_t incx);
  // CHECK: blasStatus = hipblasCscal_v2_64(blasHandle, n_64, &complexa, &complexx, incx_64);
  // CHECK-NEXT: blasStatus = hipblasCscal_v2_64(blasHandle, n_64, &complexa, &complexx, incx_64);
  blasStatus = hipblasCscal_64(blasHandle, n_64, &complexa, &complexx, incx_64);
  blasStatus = hipblasCscal_64(blasHandle, n_64, &complexa, &complexx, incx_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsscal_v2_64(cublasHandle_t handle, int64_t n, const float* alpha, cuComplex* x, int64_t incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsscal_v2_64(hipblasHandle_t handle, int64_t n, const float* alpha, hipComplex* x, int64_t incx);
  // CHECK: blasStatus = hipblasCsscal_v2_64(blasHandle, n_64, &fa, &complexx, incx_64);
  // CHECK-NEXT: blasStatus = hipblasCsscal_v2_64(blasHandle, n_64, &fa, &complexx, incx_64);
  blasStatus = hipblasCsscal_64(blasHandle, n_64, &fa, &complexx, incx_64);
  blasStatus = hipblasCsscal_64(blasHandle, n_64, &fa, &complexx, incx_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZscal_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* alpha, cuDoubleComplex* x, int64_t incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZscal_v2_64(hipblasHandle_t handle, int64_t n, const hipDoubleComplex* alpha, hipDoubleComplex* x, int64_t incx);
  // CHECK: blasStatus = hipblasZscal_v2_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64);
  // CHECK-NEXT: blasStatus = hipblasZscal_v2_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64);
  blasStatus = hipblasZscal_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64);
  blasStatus = hipblasZscal_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdscal_v2_64(cublasHandle_t handle, int64_t n, const double* alpha, cuDoubleComplex* x, int64_t incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdscal_v2_64(hipblasHandle_t handle, int64_t n, const double* alpha, hipDoubleComplex* x, int64_t incx);
  // CHECK: blasStatus = hipblasZdscal_v2_64(blasHandle, n_64, &da, &dcomplexx, incx_64);
  // CHECK-NEXT: blasStatus = hipblasZdscal_v2_64(blasHandle, n_64, &da, &dcomplexx, incx_64);
  blasStatus = hipblasZdscal_64(blasHandle, n_64, &da, &dcomplexx, incx_64);
  blasStatus = hipblasZdscal_64(blasHandle, n_64, &da, &dcomplexx, incx_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSswap_v2_64(cublasHandle_t handle, int64_t n, float* x, int64_t incx, float* y, int64_t incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSswap_64(hipblasHandle_t handle, int64_t n, float* x, int64_t incx, float* y, int64_t incy);
  // CHECK: blasStatus = hipblasSswap_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);
  // CHECK-NEXT: blasStatus = hipblasSswap_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);
  blasStatus = hipblasSswap_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);
  blasStatus = hipblasSswap_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDswap_v2_64(cublasHandle_t handle, int64_t n, double* x, int64_t incx, double* y, int64_t incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDswap_64(hipblasHandle_t handle, int64_t n, double* x, int64_t incx, double* y, int64_t incy);
  // CHECK: blasStatus = hipblasDswap_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);
  // CHECK-NEXT: blasStatus = hipblasDswap_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);
  blasStatus = hipblasDswap_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);
  blasStatus = hipblasDswap_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCswap_v2_64(cublasHandle_t handle, int64_t n, cuComplex* x, int64_t incx, cuComplex* y, int64_t incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCswap_v2_64(hipblasHandle_t handle, int64_t n, hipComplex* x, int64_t incx, hipComplex* y, int64_t incy);
  // CHECK: blasStatus = hipblasCswap_v2_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);
  // CHECK-NEXT: blasStatus = hipblasCswap_v2_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);
  blasStatus = hipblasCswap_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);
  blasStatus = hipblasCswap_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZswap_v2_64(cublasHandle_t handle, int64_t n, cuDoubleComplex* x, int64_t incx, cuDoubleComplex* y, int64_t incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZswap_v2_64(hipblasHandle_t handle, int64_t n, hipDoubleComplex* x, int64_t incx, hipDoubleComplex* y, int64_t incy);
  // CHECK: blasStatus = hipblasZswap_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);
  // CHECK-NEXT: blasStatus = hipblasZswap_v2_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);
  blasStatus = hipblasZswap_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);
  blasStatus = hipblasZswap_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);
#endif

  return 0;
}
