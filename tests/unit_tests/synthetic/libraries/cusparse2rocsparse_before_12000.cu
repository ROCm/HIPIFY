// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18.1. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: rocsparse_status status_t;
  hipsparseStatus_t status_t;

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;

  // CHECK: rocsparse_operation opA, opB, opX;
  hipsparseOperation_t opA, opB, opX;

  // CHECK: rocsparse_solve_policy solvePolicy_t;
  hipsparseSolvePolicy_t solvePolicy_t;

  int m = 0;
  int innz = 0;
  int csrRowPtrA = 0;
  int csrColIndA = 0;
  double dAlpha = 0.f;
  double dF = 0.f;
  double dX = 0.f;
  double dcsrSortedValA = 0.f;
  float fAlpha = 0.f;
  float fF = 0.f;
  float fX = 0.f;
  float csrSortedValA = 0.f;
  void *pBuffer = nullptr;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuDoubleComplex -> rocsparse_double_complex under a new option --sparse
  // CHECK: rocblas_double_complex dcomplex, dcomplexA, dcomplexAlpha, dcomplexB, dcomplexBeta, dcomplexC, dcomplexF, dcomplexX, dcomplexY, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;
  hipDoubleComplex dcomplex, dcomplexA, dcomplexAlpha, dcomplexB, dcomplexBeta, dcomplexC, dcomplexF, dcomplexX, dcomplexY, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuComplex -> rocsparse_float_complex under a new option --sparse
  // CHECK: rocblas_float_complex complex, complexA, complexAlpha, complexB, complexBeta, complexC, complexF, complexX, complexY, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;
  hipComplex complex, complexA, complexAlpha, complexB, complexBeta, complexC, complexF, complexX, complexY, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;

#if CUDA_VERSION < 12000
  // CHECK: rocsparse_mat_descr csrsv2_info;
  csrsv2Info_t csrsv2_info;

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseZcsrsv2_solve(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cuDoubleComplex* alpha, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, const cuDoubleComplex* f, cuDoubleComplex* x, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsrsv_solve(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_double_complex* alpha, const rocsparse_mat_descr descr, const rocsparse_double_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, const rocsparse_double_complex* x, rocsparse_double_complex* y, rocsparse_solve_policy policy, void* temp_buffer);
  // CHECK: status_t = rocsparse_zcsrsv_solve(handle_t, opA, m, innz, &dcomplexAlpha, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &dcomplexF, &dcomplexX, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseZcsrsv2_solve(handle_t, opA, m, innz, &dcomplexAlpha, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &dcomplexF, &dcomplexX, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseCcsrsv2_solve(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cuComplex* alpha, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, const cuComplex* f, cuComplex* x,cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsrsv_solve(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_float_complex* alpha, const rocsparse_mat_descr descr, const rocsparse_float_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, const rocsparse_float_complex* x, rocsparse_float_complex* y, rocsparse_solve_policy policy, void* temp_buffer);
  // CHECK: status_t = rocsparse_ccsrsv_solve(handle_t, opA, m, innz, &complexAlpha, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &complexF, &complexX, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseCcsrsv2_solve(handle_t, opA, m, innz, &complexAlpha, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &complexF, &complexX, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseDcsrsv2_solve(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const double* alpha, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, const double* f, double* x, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsrsv_solve(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const double* alpha, const rocsparse_mat_descr descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, const double* x, double* y, rocsparse_solve_policy policy, void* temp_buffer);
  // CHECK: status_t = rocsparse_dcsrsv_solve(handle_t, opA, m, innz, &dAlpha, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &dF, &dX, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseDcsrsv2_solve(handle_t, opA, m, innz, &dAlpha, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &dF, &dX, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseScsrsv2_solve(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const float* alpha, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, const float* f, float* x, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsrsv_solve(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const float* alpha, const rocsparse_mat_descr descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, const float* x, float* y, rocsparse_solve_policy policy, void* temp_buffer);
  // CHECK: status_t = rocsparse_scsrsv_solve(handle_t, opA, m, innz, &fAlpha, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &fF, &fX, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseScsrsv2_solve(handle_t, opA, m, innz, &fAlpha, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &fF, &fX, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseZcsrsv2_analysis(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsrsv_analysis(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const rocsparse_double_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, rocsparse_analysis_policy analysis, rocsparse_solve_policy solve, void* temp_buffer);
  // CHECK: status_t = rocsparse_zcsrsv_analysis(handle_t, opA, m, innz, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, rocsparse_analysis_policy_force, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseZcsrsv2_analysis(handle_t, opA, m, innz, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseCcsrsv2_analysis(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsrsv_analysis(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const rocsparse_float_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, rocsparse_analysis_policy analysis, rocsparse_solve_policy solve, void* temp_buffer);
  // CHECK: status_t = rocsparse_ccsrsv_analysis(handle_t, opA, m, innz, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, rocsparse_analysis_policy_force, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseCcsrsv2_analysis(handle_t, opA, m, innz, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseDcsrsv2_analysis(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsrsv_analysis(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, rocsparse_analysis_policy analysis, rocsparse_solve_policy solve, void* temp_buffer);
  // CHECK: status_t = rocsparse_dcsrsv_analysis(handle_t, opA, m, innz, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, rocsparse_analysis_policy_force, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseDcsrsv2_analysis(handle_t, opA, m, innz, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseScsrsv2_analysis(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsrsv_analysis(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, rocsparse_analysis_policy analysis, rocsparse_solve_policy solve, void* temp_buffer);
  // CHECK: status_t = rocsparse_scsrsv_analysis(handle_t, opA, m, innz, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, rocsparse_analysis_policy_force, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseScsrsv2_analysis(handle_t, opA, m, innz, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, solvePolicy_t, pBuffer);
#endif

  return 0;
}
