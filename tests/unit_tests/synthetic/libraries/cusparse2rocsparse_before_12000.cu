// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18.1. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: rocsparse_status status_t;
  hipsparseStatus_t status_t;

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;

  // CHECK: rocsparse_operation opA, opB, opX;
  hipsparseOperation_t opA, opB, opX;

  // CHECK: rocsparse_solve_policy solvePolicy_t;
  hipsparseSolvePolicy_t solvePolicy_t;

  int m = 0;
  int n = 0;
  int k = 0;
  int innz = 0;
  int nnza = 0;
  int nnzb = 0;
  int nnzc = 0;
  int nnzd = 0;
  int csrRowPtrA = 0;
  int csrRowPtrB = 0;
  int csrRowPtrC = 0;
  int csrRowPtrD = 0;
  int csrColIndA = 0;
  int csrColIndB = 0;
  int csrColIndC = 0;
  int csrColIndD = 0;
  int bufferSizeInBytes = 0;
  size_t bufferSize = 0;
  double dA = 0.f;
  double dB = 0.f;
  double dAlpha = 0.f;
  double dF = 0.f;
  double dX = 0.f;
  double dcsrSortedValA = 0.f;
  double dcsrSortedValB = 0.f;
  double dcsrSortedValC = 0.f;
  double dcsrSortedValD = 0.f;
  float fAlpha = 0.f;
  float fA = 0.f;
  float fB = 0.f;
  float fF = 0.f;
  float fX = 0.f;
  float csrSortedValA = 0.f;
  float csrSortedValB = 0.f;
  float csrSortedValC = 0.f;
  float csrSortedValD = 0.f;
  void *pBuffer = nullptr;
  void *tempBuffer = nullptr;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuDoubleComplex -> rocsparse_double_complex under a new option --sparse
  // CHECK: rocblas_double_complex dcomplex, dcomplexA, dcomplexAlpha, dcomplexB, dcomplexBeta, dcomplexC, dcomplexF, dcomplexX, dcomplexY, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dComplexcsrSortedValD, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;
  hipDoubleComplex dcomplex, dcomplexA, dcomplexAlpha, dcomplexB, dcomplexBeta, dcomplexC, dcomplexF, dcomplexX, dcomplexY, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dComplexcsrSortedValD, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuComplex -> rocsparse_float_complex under a new option --sparse
  // CHECK: rocblas_float_complex complex, complexA, complexAlpha, complexB, complexBeta, complexC, complexF, complexX, complexY, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complexcsrSortedValD, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;
  hipComplex complex, complexA, complexAlpha, complexB, complexBeta, complexC, complexF, complexX, complexY, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complexcsrSortedValD, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;

#if CUDA_VERSION < 12000
  // CHECK: rocsparse_mat_descr csrsv2_info;
  csrsv2Info_t csrsv2_info;
  // CHECK: rocsparse_mat_info csrgemm2_info;
  csrgemm2Info_t csrgemm2_info;

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseZcsrsv2_solve(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cuDoubleComplex* alpha, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, const cuDoubleComplex* f, cuDoubleComplex* x, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsrsv_solve(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_double_complex* alpha, const rocsparse_mat_descr descr, const rocsparse_double_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, const rocsparse_double_complex* x, rocsparse_double_complex* y, rocsparse_solve_policy policy, void* temp_buffer);
  // CHECK: status_t = rocsparse_zcsrsv_solve(handle_t, opA, m, innz, &dcomplexAlpha, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &dcomplexF, &dcomplexX, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseZcsrsv2_solve(handle_t, opA, m, innz, &dcomplexAlpha, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &dcomplexF, &dcomplexX, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseCcsrsv2_solve(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cuComplex* alpha, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, const cuComplex* f, cuComplex* x,cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsrsv_solve(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_float_complex* alpha, const rocsparse_mat_descr descr, const rocsparse_float_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, const rocsparse_float_complex* x, rocsparse_float_complex* y, rocsparse_solve_policy policy, void* temp_buffer);
  // CHECK: status_t = rocsparse_ccsrsv_solve(handle_t, opA, m, innz, &complexAlpha, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &complexF, &complexX, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseCcsrsv2_solve(handle_t, opA, m, innz, &complexAlpha, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &complexF, &complexX, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseDcsrsv2_solve(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const double* alpha, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, const double* f, double* x, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsrsv_solve(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const double* alpha, const rocsparse_mat_descr descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, const double* x, double* y, rocsparse_solve_policy policy, void* temp_buffer);
  // CHECK: status_t = rocsparse_dcsrsv_solve(handle_t, opA, m, innz, &dAlpha, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &dF, &dX, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseDcsrsv2_solve(handle_t, opA, m, innz, &dAlpha, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &dF, &dX, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseScsrsv2_solve(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const float* alpha, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, const float* f, float* x, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsrsv_solve(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const float* alpha, const rocsparse_mat_descr descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, const float* x, float* y, rocsparse_solve_policy policy, void* temp_buffer);
  // CHECK: status_t = rocsparse_scsrsv_solve(handle_t, opA, m, innz, &fAlpha, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &fF, &fX, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseScsrsv2_solve(handle_t, opA, m, innz, &fAlpha, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &fF, &fX, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseZcsrsv2_analysis(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsrsv_analysis(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const rocsparse_double_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, rocsparse_analysis_policy analysis, rocsparse_solve_policy solve, void* temp_buffer);
  // CHECK: status_t = rocsparse_zcsrsv_analysis(handle_t, opA, m, innz, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, rocsparse_analysis_policy_force, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseZcsrsv2_analysis(handle_t, opA, m, innz, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseCcsrsv2_analysis(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsrsv_analysis(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const rocsparse_float_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, rocsparse_analysis_policy analysis, rocsparse_solve_policy solve, void* temp_buffer);
  // CHECK: status_t = rocsparse_ccsrsv_analysis(handle_t, opA, m, innz, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, rocsparse_analysis_policy_force, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseCcsrsv2_analysis(handle_t, opA, m, innz, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseDcsrsv2_analysis(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsrsv_analysis(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, rocsparse_analysis_policy analysis, rocsparse_solve_policy solve, void* temp_buffer);
  // CHECK: status_t = rocsparse_dcsrsv_analysis(handle_t, opA, m, innz, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, rocsparse_analysis_policy_force, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseDcsrsv2_analysis(handle_t, opA, m, innz, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseScsrsv2_analysis(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, cusparseSolvePolicy_t policy, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsrsv_analysis(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, rocsparse_analysis_policy analysis, rocsparse_solve_policy solve, void* temp_buffer);
  // CHECK: status_t = rocsparse_scsrsv_analysis(handle_t, opA, m, innz, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, rocsparse_analysis_policy_force, rocsparse_solve_policy_auto, pBuffer);
  status_t = cusparseScsrsv2_analysis(handle_t, opA, m, innz, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, solvePolicy_t, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseZcsrsv2_bufferSize(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cusparseMatDescr_t descrA, cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsrsv_buffer_size(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const rocsparse_double_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, size_t* buffer_size);
  // CHECK: status_t = rocsparse_zcsrsv_buffer_size(handle_t, opA, m, innz, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, reinterpret_cast<size_t*>(&bufferSizeInBytes));
  status_t = cusparseZcsrsv2_bufferSize(handle_t, opA, m, innz, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &bufferSizeInBytes);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseCcsrsv2_bufferSize(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cusparseMatDescr_t descrA, cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsrsv_buffer_size(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const rocsparse_float_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, size_t* buffer_size);
  // CHECK: status_t = rocsparse_ccsrsv_buffer_size(handle_t, opA, m, innz, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, reinterpret_cast<size_t*>(&bufferSizeInBytes));
  status_t = cusparseCcsrsv2_bufferSize(handle_t, opA, m, innz, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &bufferSizeInBytes);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseDcsrsv2_bufferSize(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cusparseMatDescr_t descrA, double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsrsv_buffer_size(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, size_t* buffer_size);
  // CHECK: status_t = rocsparse_dcsrsv_buffer_size(handle_t, opA, m, innz, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, reinterpret_cast<size_t*>(&bufferSizeInBytes));
  status_t = cusparseDcsrsv2_bufferSize(handle_t, opA, m, innz, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &bufferSizeInBytes);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpSV) cusparseStatus_t CUSPARSEAPI cusparseScsrsv2_bufferSize(cusparseHandle_t handle, cusparseOperation_t transA, int m, int nnz, const cusparseMatDescr_t descrA, float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, csrsv2Info_t info, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsrsv_buffer_size(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, size_t* buffer_size);
  // CHECK: status_t = rocsparse_scsrsv_buffer_size(handle_t, opA, m, innz, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, reinterpret_cast<size_t*>(&bufferSizeInBytes));
  status_t = cusparseScsrsv2_bufferSize(handle_t, opA, m, innz, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, csrsv2_info, &bufferSizeInBytes);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpGEMM) cusparseStatus_t CUSPARSEAPI cusparseZcsrgemm2(cusparseHandle_t handle, int m, int n, int k, const cuDoubleComplex* alpha, const cusparseMatDescr_t descrA, int nnzA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrB, int nnzB, const cuDoubleComplex* csrSortedValB, const int* csrSortedRowPtrB, const int* csrSortedColIndB, const cuDoubleComplex* beta, const cusparseMatDescr_t descrD, int nnzD, const cuDoubleComplex* csrSortedValD, const int* csrSortedRowPtrD, const int* csrSortedColIndD, const cusparseMatDescr_t descrC, cuDoubleComplex* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, const csrgemm2Info_t info, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsrgemm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int n, rocsparse_int k, const rocsparse_double_complex* alpha, const rocsparse_mat_descr descr_A, rocsparse_int nnz_A, const rocsparse_double_complex* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const rocsparse_mat_descr descr_B, rocsparse_int nnz_B, const rocsparse_double_complex* csr_val_B, const rocsparse_int* csr_row_ptr_B, const rocsparse_int* csr_col_ind_B, const rocsparse_double_complex* beta, const rocsparse_mat_descr descr_D, rocsparse_int nnz_D, const rocsparse_double_complex* csr_val_D, const rocsparse_int* csr_row_ptr_D, const rocsparse_int* csr_col_ind_D, const rocsparse_mat_descr descr_C, rocsparse_double_complex* csr_val_C, const rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C, const rocsparse_mat_info info_C, void* temp_buffer);
  // CHECK: status_t = rocsparse_zcsrgemm(handle_t, rocsparse_operation_none, rocsparse_operation_none, m, n, k, &dcomplexA, matDescr_A, nnza, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_B, nnzb, &dComplexcsrSortedValB, &csrRowPtrB, &csrColIndB, &dcomplexB, matDescr_D, nnzd, &dComplexcsrSortedValD, &csrRowPtrD, &csrColIndD, matDescr_C, &dComplexcsrSortedValC, &csrRowPtrC, &csrColIndC, csrgemm2_info, pBuffer);
  status_t = cusparseZcsrgemm2(handle_t, m, n, k, &dcomplexA, matDescr_A, nnza, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_B, nnzb, &dComplexcsrSortedValB, &csrRowPtrB, &csrColIndB, &dcomplexB, matDescr_D, nnzd, &dComplexcsrSortedValD, &csrRowPtrD, &csrColIndD, matDescr_C, &dComplexcsrSortedValC, &csrRowPtrC, &csrColIndC, csrgemm2_info, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpGEMM) cusparseStatus_t CUSPARSEAPI cusparseCcsrgemm2(cusparseHandle_t handle, int m, int n, int k, const cuComplex* alpha, const cusparseMatDescr_t descrA, int nnzA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrB, int nnzB, const cuComplex* csrSortedValB, const int* csrSortedRowPtrB, const int* csrSortedColIndB, const cuComplex* beta, const cusparseMatDescr_t descrD, int nnzD, const cuComplex* csrSortedValD, const int* csrSortedRowPtrD, const int* csrSortedColIndD, const cusparseMatDescr_t descrC, cuComplex* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, const csrgemm2Info_t info, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsrgemm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int n, rocsparse_int k, const rocsparse_float_complex* alpha, const rocsparse_mat_descr descr_A, rocsparse_int nnz_A, const rocsparse_float_complex* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const rocsparse_mat_descr descr_B, rocsparse_int nnz_B, const rocsparse_float_complex* csr_val_B, const rocsparse_int* csr_row_ptr_B, const rocsparse_int* csr_col_ind_B, const rocsparse_float_complex* beta, const rocsparse_mat_descr descr_D, rocsparse_int nnz_D, const rocsparse_float_complex* csr_val_D, const rocsparse_int* csr_row_ptr_D, const rocsparse_int* csr_col_ind_D, const rocsparse_mat_descr descr_C, rocsparse_float_complex* csr_val_C, const rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C, const rocsparse_mat_info info_C, void* temp_buffer);
  // CHECK: status_t = rocsparse_ccsrgemm(handle_t, rocsparse_operation_none, rocsparse_operation_none, m, n, k, &complexA, matDescr_A, nnza, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_B, nnzb, &complexcsrSortedValB, &csrRowPtrB, &csrColIndB, &complexB, matDescr_D, nnzd, &complexcsrSortedValD, &csrRowPtrD, &csrColIndD, matDescr_C, &complexcsrSortedValC, &csrRowPtrC, &csrColIndC, csrgemm2_info, pBuffer);
  status_t = cusparseCcsrgemm2(handle_t, m, n, k, &complexA, matDescr_A, nnza, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_B, nnzb, &complexcsrSortedValB, &csrRowPtrB, &csrColIndB, &complexB, matDescr_D, nnzd, &complexcsrSortedValD, &csrRowPtrD, &csrColIndD, matDescr_C, &complexcsrSortedValC, &csrRowPtrC, &csrColIndC, csrgemm2_info, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpGEMM) cusparseStatus_t CUSPARSEAPI cusparseDcsrgemm2(cusparseHandle_t handle, int m, int n, int k, const double* alpha, const cusparseMatDescr_t descrA, int nnzA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrB, int nnzB, const double* csrSortedValB, const int* csrSortedRowPtrB, const int* csrSortedColIndB, const double* beta, const cusparseMatDescr_t descrD, int nnzD, const double* csrSortedValD, const int* csrSortedRowPtrD, const int* csrSortedColIndD, const cusparseMatDescr_t descrC, double* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, const csrgemm2Info_t info, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsrgemm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int n, rocsparse_int k, const double* alpha, const rocsparse_mat_descr descr_A, rocsparse_int nnz_A, const double* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const rocsparse_mat_descr descr_B, rocsparse_int nnz_B, const double* csr_val_B, const rocsparse_int* csr_row_ptr_B, const rocsparse_int* csr_col_ind_B, const double* beta, const rocsparse_mat_descr descr_D, rocsparse_int nnz_D, const double* csr_val_D, const rocsparse_int* csr_row_ptr_D, const rocsparse_int* csr_col_ind_D, const rocsparse_mat_descr descr_C, double* csr_val_C, const rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C, const rocsparse_mat_info info_C, void* temp_buffer);
  // CHECK: status_t = rocsparse_dcsrgemm(handle_t, rocsparse_operation_none, rocsparse_operation_none, m, n, k, &dA, matDescr_A, nnza, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_B, nnzb, &dcsrSortedValB, &csrRowPtrB, &csrColIndB, &dB, matDescr_D, nnzd, &dcsrSortedValD, &csrRowPtrD, &csrColIndD, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, csrgemm2_info, pBuffer);
  status_t = cusparseDcsrgemm2(handle_t, m, n, k, &dA, matDescr_A, nnza, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_B, nnzb, &dcsrSortedValB, &csrRowPtrB, &csrColIndB, &dB, matDescr_D, nnzd, &dcsrSortedValD, &csrRowPtrD, &csrColIndD, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, csrgemm2_info, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSpGEMM) cusparseStatus_t CUSPARSEAPI cusparseScsrgemm2(cusparseHandle_t handle, int m, int n, int k, const float* alpha, const cusparseMatDescr_t descrA, int nnzA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrB, int nnzB, const float* csrSortedValB, const int* csrSortedRowPtrB, const int* csrSortedColIndB, const float* beta, const cusparseMatDescr_t descrD, int nnzD, const float* csrSortedValD, const int* csrSortedRowPtrD, const int* csrSortedColIndD, const cusparseMatDescr_t descrC, float* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, const csrgemm2Info_t info, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsrgemm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, rocsparse_int m, rocsparse_int n, rocsparse_int k, const float* alpha, const rocsparse_mat_descr descr_A, rocsparse_int nnz_A, const float* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const rocsparse_mat_descr descr_B, rocsparse_int nnz_B, const float* csr_val_B, const rocsparse_int* csr_row_ptr_B, const rocsparse_int* csr_col_ind_B, const float* beta, const rocsparse_mat_descr descr_D, rocsparse_int nnz_D, const float* csr_val_D, const rocsparse_int* csr_row_ptr_D, const rocsparse_int* csr_col_ind_D, const rocsparse_mat_descr descr_C, float* csr_val_C, const rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C, const rocsparse_mat_info info_C, void* temp_buffer);
  // CHECK: status_t = rocsparse_scsrgemm(handle_t, rocsparse_operation_none, rocsparse_operation_none, m, n, k, &fA, matDescr_A, nnza, &csrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_B, nnzb, &csrSortedValB, &csrRowPtrB, &csrColIndB, &fB, matDescr_D, nnzd, &csrSortedValD, &csrRowPtrD, &csrColIndD, matDescr_C, &csrSortedValC, &csrRowPtrC, &csrColIndC, csrgemm2_info, pBuffer);
  status_t = cusparseScsrgemm2(handle_t, m, n, k, &fA, matDescr_A, nnza, &csrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_B, nnzb, &csrSortedValB, &csrRowPtrB, &csrColIndB, &fB, matDescr_D, nnzd, &csrSortedValD, &csrRowPtrD, &csrColIndD, matDescr_C, &csrSortedValC, &csrRowPtrC, &csrColIndC, csrgemm2_info, pBuffer);
#endif

  return 0;
}
