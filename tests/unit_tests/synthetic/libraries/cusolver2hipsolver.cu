// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipsolver.h"
#include "hipsolver.h"

int main() {
  printf("19. cuSOLVER API to hipSOLVER API synthetic test\n");

  int m = 0;
  int n = 0;
  int k = 0;
  int nrhs = 0;
  int lda = 0;
  int ldb = 0;
  int ldc = 0;
  int Lwork = 0;
  int devIpiv = 0;
  int devInfo = 0;
  int info = 0;
  int infoArray = 0;
  int batchSize = 0;
  float fA = 0.f;
  float fB = 0.f;
  float fC = 0.f;
  float fD = 0.f;
  float fE = 0.f;
  float fX = 0.f;
  float fTAU = 0.f;
  float fTAUQ = 0.f;
  float fTAUP = 0.f;
  double dA = 0.f;
  double dB = 0.f;
  double dC = 0.f;
  double dD = 0.f;
  double dE = 0.f;
  double dX = 0.f;
  double dTAU = 0.f;
  double dTAUQ = 0.f;
  double dTAUP = 0.f;
  float fWorkspace = 0.f;
  double dWorkspace = 0.f;
  void *Workspace = nullptr;
  size_t lwork_bytes = 0;

  float** fAarray = 0;
  float** fBarray = 0;
  double** dAarray = 0;
  double** dBarray = 0;

  // CHECK: hipDoubleComplex dComplexA, dComplexB, dComplexC, dComplexD, dComplexE, dComplexX, dComplexWorkspace, dComplexTAU, dComplexTAUQ, dComplexTAUP;
  hipDoubleComplex dComplexA, dComplexB, dComplexC, dComplexD, dComplexE, dComplexX, dComplexWorkspace, dComplexTAU, dComplexTAUQ, dComplexTAUP;

  // CHECK: hipComplex complexA, complexB, complexC, complexD, complexE, complexX, complexWorkspace, complexTAU, complexTAUQ, complexTAUP;
  hipComplex complexA, complexB, complexC, complexD, complexE, complexX, complexWorkspace, complexTAU, complexTAUQ, complexTAUP;

  // CHECK: hipDoubleComplex** dcomplexAarray = 0;
  // CHECK-NEXT: hipDoubleComplex** dcomplexBarray = 0;
  hipDoubleComplex** dcomplexAarray = 0;
  hipDoubleComplex** dcomplexBarray = 0;

  // CHECK: hipComplex** complexAarray = 0;
  // CHECK-NEXT: hipComplex** complexBarray = 0;
  hipComplex** complexAarray = 0;
  hipComplex** complexBarray = 0;

  // CHECK: hipsolverHandle_t handle;
  hipsolverHandle_t handle;

  // CHECK: hipblasFillMode_t fillMode;
  hipblasFillMode_t fillMode;

  // CHECK: hipsolverStatus_t status;
  // CHECK-NEXT: hipsolverStatus_t STATUS_SUCCESS = HIPSOLVER_STATUS_SUCCESS;
  // CHECK-NEXT: hipsolverStatus_t STATUS_NOT_INITIALIZED = HIPSOLVER_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT: hipsolverStatus_t STATUS_ALLOC_FAILED = HIPSOLVER_STATUS_ALLOC_FAILED;
  // CHECK-NEXT: hipsolverStatus_t STATUS_INVALID_VALUE = HIPSOLVER_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipsolverStatus_t STATUS_ARCH_MISMATCH = HIPSOLVER_STATUS_ARCH_MISMATCH;
  // CHECK-NEXT: hipsolverStatus_t STATUS_MAPPING_ERROR = HIPSOLVER_STATUS_MAPPING_ERROR;
  // CHECK-NEXT: hipsolverStatus_t STATUS_EXECUTION_FAILED = HIPSOLVER_STATUS_EXECUTION_FAILED;
  // CHECK-NEXT: hipsolverStatus_t STATUS_INTERNAL_ERROR = HIPSOLVER_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipsolverStatus_t STATUS_NOT_SUPPORTED = HIPSOLVER_STATUS_NOT_SUPPORTED;
  // CHECK-NEXT: hipsolverStatus_t STATUS_ZERO_PIVOT = HIPSOLVER_STATUS_ZERO_PIVOT;
  hipsolverStatus_t status;
  hipsolverStatus_t STATUS_SUCCESS = HIPSOLVER_STATUS_SUCCESS;
  hipsolverStatus_t STATUS_NOT_INITIALIZED = HIPSOLVER_STATUS_NOT_INITIALIZED;
  hipsolverStatus_t STATUS_ALLOC_FAILED = HIPSOLVER_STATUS_ALLOC_FAILED;
  hipsolverStatus_t STATUS_INVALID_VALUE = HIPSOLVER_STATUS_INVALID_VALUE;
  hipsolverStatus_t STATUS_ARCH_MISMATCH = HIPSOLVER_STATUS_ARCH_MISMATCH;
  hipsolverStatus_t STATUS_MAPPING_ERROR = HIPSOLVER_STATUS_MAPPING_ERROR;
  hipsolverStatus_t STATUS_EXECUTION_FAILED = HIPSOLVER_STATUS_EXECUTION_FAILED;
  hipsolverStatus_t STATUS_INTERNAL_ERROR = HIPSOLVER_STATUS_INTERNAL_ERROR;
  hipsolverStatus_t STATUS_NOT_SUPPORTED = HIPSOLVER_STATUS_NOT_SUPPORTED;
  hipsolverStatus_t STATUS_ZERO_PIVOT = HIPSOLVER_STATUS_ZERO_PIVOT;

  // CHECK: hipblasOperation_t blasOperation;
  hipblasOperation_t blasOperation;

  // CHECK: hipblasSideMode_t blasSideMode;
  hipblasSideMode_t blasSideMode;

  // CHECK: hipStream_t stream_t;
  hipStream_t stream_t;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCreate(cusolverDnHandle_t *handle);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCreate(hipsolverHandle_t* handle);
  // CHECK: status = hipsolverDnCreate(&handle);
  status = hipsolverDnCreate(&handle);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDestroy(cusolverDnHandle_t handle);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDestroy(hipsolverHandle_t handle);
  // CHECK: status = hipsolverDnDestroy(handle);
  status = hipsolverDnDestroy(handle);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgetrf(cusolverDnHandle_t handle, int m, int n, float* A, int lda, float* Workspace, int* devIpiv, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgetrf(hipsolverHandle_t handle, int m, int n, float* A, int lda, float* work, int* devIpiv, int* devInfo);
  // CHECK: status = hipsolverDnSgetrf(handle, m, n, &fA, lda, &fWorkspace, &devIpiv, &devInfo);
  status = hipsolverDnSgetrf(handle, m, n, &fA, lda, &fWorkspace, &devIpiv, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgetrf(cusolverDnHandle_t handle, int m, int n, double* A, int lda, double* Workspace, int* devIpiv, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgetrf(hipsolverHandle_t handle, int m, int n,double* A, int lda, double* work, int* devIpiv, int* devInfo);
  // CHECK: status = hipsolverDnDgetrf(handle, m, n, &dA, lda, &dWorkspace, &devIpiv, &devInfo);
  status = hipsolverDnDgetrf(handle, m, n, &dA, lda, &dWorkspace, &devIpiv, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgetrf(cusolverDnHandle_t handle, int m, int n, cuComplex * A, int lda, cuComplex * Workspace, int * devIpiv, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgetrf(hipsolverHandle_t handle, int m, int n, hipFloatComplex* A, int lda, hipFloatComplex* work, int* devIpiv, int* devInfo);
  // CHECK: status = hipsolverDnCgetrf(handle, m, n, &complexA, lda, &complexWorkspace, &devIpiv, &devInfo);
  status = hipsolverDnCgetrf(handle, m, n, &complexA, lda, &complexWorkspace, &devIpiv, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgetrf(cusolverDnHandle_t handle, int m, int n, cuDoubleComplex * A, int lda, cuDoubleComplex * Workspace, int * devIpiv, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgetrf(hipsolverHandle_t handle, int m, int n, hipDoubleComplex* A, int lda, hipDoubleComplex* work, int* devIpiv, int* devInfo);
  // CHECK: status = hipsolverDnZgetrf(handle, m, n, &dComplexA, lda, &dComplexWorkspace, &devIpiv, &devInfo);
  status = hipsolverDnZgetrf(handle, m, n, &dComplexA, lda, &dComplexWorkspace, &devIpiv, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgetrf_bufferSize(cusolverDnHandle_t handle, int m, int n, float* A, int lda, int* Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgetrf_bufferSize(hipsolverHandle_t handle, int m, int n, float* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnSgetrf_bufferSize(handle, m, n, &fA, lda, &Lwork);
  status = hipsolverDnSgetrf_bufferSize(handle, m, n, &fA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgetrf_bufferSize(cusolverDnHandle_t handle, int m, int n, double* A, int lda, int* Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgetrf_bufferSize(hipsolverHandle_t handle, int m, int n, double* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnDgetrf_bufferSize(handle, m, n, &dA, lda, &Lwork);
  status = hipsolverDnDgetrf_bufferSize(handle, m, n, &dA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgetrf_bufferSize(cusolverDnHandle_t handle, int m, int n, cuComplex * A, int lda, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgetrf_bufferSize(hipsolverHandle_t handle, int m, int n, hipFloatComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnCgetrf_bufferSize(handle, m, n, &complexA, lda, &Lwork);
  status = hipsolverDnCgetrf_bufferSize(handle, m, n, &complexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgetrf_bufferSize(cusolverDnHandle_t handle, int m, int n, cuDoubleComplex * A, int lda, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgetrf_bufferSize(hipsolverHandle_t handle, int m, int n, hipDoubleComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnZgetrf_bufferSize(handle, m, n, &dComplexA, lda, &Lwork);
  status = hipsolverDnZgetrf_bufferSize(handle, m, n, &dComplexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgetrs(cusolverDnHandle_t handle, cublasOperation_t trans, int n, int nrhs, const float* A, int lda, const int* devIpiv, float* B, int ldb, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgetrs(hipsolverHandle_t handle, hipsolverOperation_t trans, int n, int nrhs, const float* A, int lda, const int* devIpiv, float* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnSgetrs(handle, blasOperation, n, nrhs , &fA, lda, &devIpiv, &fB, ldb, &devInfo);
  status = hipsolverDnSgetrs(handle, blasOperation, n, nrhs , &fA, lda, &devIpiv, &fB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgetrs(cusolverDnHandle_t handle, cublasOperation_t trans, int n, int nrhs,const double* A, int lda, const int* devIpiv, double* B, int ldb, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgetrs(hipsolverHandle_t handle, hipsolverOperation_t trans, int n, int nrhs, const double* A, int lda, const int* devIpiv, double* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnDgetrs(handle, blasOperation, n, nrhs , &dA, lda, &devIpiv, &dB, ldb, &devInfo);
  status = hipsolverDnDgetrs(handle, blasOperation, n, nrhs , &dA, lda, &devIpiv, &dB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgetrs(cusolverDnHandle_t handle, cublasOperation_t trans, int n, int nrhs, const cuComplex * A, int lda, const int * devIpiv, cuComplex * B, int ldb, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgetrs(hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const hipFloatComplex* A, int lda, const int* devIpiv, hipFloatComplex* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnCgetrs(handle, blasOperation, n, nrhs , &complexA, lda, &devIpiv, &complexB, ldb, &devInfo);
  status = hipsolverDnCgetrs(handle, blasOperation, n, nrhs , &complexA, lda, &devIpiv, &complexB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgetrs(cusolverDnHandle_t handle, cublasOperation_t trans, int n, int nrhs, const cuDoubleComplex *A, int lda, const int * devIpiv, cuDoubleComplex * B, int ldb, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgetrs(hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const hipDoubleComplex* A, int lda, const int* devIpiv, hipDoubleComplex* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnZgetrs(handle, blasOperation, n, nrhs , &dComplexA, lda, &devIpiv, &dComplexB, ldb, &devInfo);
  status = hipsolverDnZgetrs(handle, blasOperation, n, nrhs , &dComplexA, lda, &devIpiv, &dComplexB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSetStream(cusolverDnHandle_t handle, cudaStream_t streamId);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverSetStream(hipsolverHandle_t handle, hipStream_t streamId);
  // CHECK: status = hipsolverSetStream(handle, stream_t);
  status = hipsolverSetStream(handle, stream_t);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnGetStream(cusolverDnHandle_t handle, cudaStream_t *streamId);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverGetStream(hipsolverHandle_t handle, hipStream_t* streamId);
  // CHECK: status = hipsolverGetStream(handle, &stream_t);
  status = hipsolverGetStream(handle, &stream_t);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSpotrf_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, float * A, int lda, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSpotrf_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnSpotrf_bufferSize(handle, fillMode, n, &fA, lda, &Lwork);
  status = hipsolverDnSpotrf_bufferSize(handle, fillMode, n, &fA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDpotrf_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, double * A, int lda, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDpotrf_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnDpotrf_bufferSize(handle, fillMode, n, &dA, lda, &Lwork);
  status = hipsolverDnDpotrf_bufferSize(handle, fillMode, n, &dA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCpotrf_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuComplex * A, int lda, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCpotrf_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnCpotrf_bufferSize(handle, fillMode, n, &complexA, lda, &Lwork);
  status = hipsolverDnCpotrf_bufferSize(handle, fillMode, n, &complexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZpotrf_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZpotrf_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnZpotrf_bufferSize(handle, fillMode, n, &dComplexA, lda, &Lwork);
  status = hipsolverDnZpotrf_bufferSize(handle, fillMode, n, &dComplexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSpotrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, float * A, int lda, float * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSpotrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* A, int lda, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSpotrf(handle, fillMode, n, &fA, lda, &fWorkspace, Lwork, &devInfo);
  status = hipsolverDnSpotrf(handle, fillMode, n, &fA, lda, &fWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDpotrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, double * A, int lda, double * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDpotrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* A, int lda, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDpotrf(handle, fillMode, n, &dA, lda, &dWorkspace, Lwork, &devInfo);
  status = hipsolverDnDpotrf(handle, fillMode, n, &dA, lda, &dWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCpotrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuComplex * A, int lda, cuComplex * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCpotrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCpotrf(handle, fillMode, n, &complexA, lda, &complexWorkspace, Lwork, &devInfo);
  status = hipsolverDnCpotrf(handle, fillMode, n, &complexA, lda, &complexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZpotrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, cuDoubleComplex * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZpotrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZpotrf(handle, fillMode, n, &dComplexA, lda, &dComplexWorkspace, Lwork, &devInfo);
  status = hipsolverDnZpotrf(handle, fillMode, n, &dComplexA, lda, &dComplexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSpotrs(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, const float * A, int lda, float * B, int ldb, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSpotrs(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, const float* A, int lda, float* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnSpotrs(handle, fillMode, n, nrhs, &fA, lda, &fB, ldb, &devInfo);
  status = hipsolverDnSpotrs(handle, fillMode, n, nrhs, &fA, lda, &fB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDpotrs(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, const double * A, int lda, double * B, int ldb, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDpotrs(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, const double* A, int lda, double* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnDpotrs(handle, fillMode, n, nrhs, &dA, lda, &dB, ldb, &devInfo);
  status = hipsolverDnDpotrs(handle, fillMode, n, nrhs, &dA, lda, &dB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCpotrs(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, const cuComplex * A, int lda, cuComplex * B, int ldb, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCpotrs(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, const hipFloatComplex* A, int lda, hipFloatComplex* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnCpotrs(handle, fillMode, n, nrhs, &complexA, lda, &complexB, ldb, &devInfo);
  status = hipsolverDnCpotrs(handle, fillMode, n, nrhs, &complexA, lda, &complexB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZpotrs(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, const cuDoubleComplex *A, int lda, cuDoubleComplex * B, int ldb, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZpotrs(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, const hipDoubleComplex* A, int lda, hipDoubleComplex* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnZpotrs(handle, fillMode, n, nrhs, &dComplexA, lda, &dComplexB, ldb, &devInfo);
  status = hipsolverDnZpotrs(handle, fillMode, n, nrhs, &dComplexA, lda, &dComplexB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgeqrf_bufferSize(cusolverDnHandle_t handle, int m, int n, float * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgeqrf_bufferSize(hipsolverHandle_t handle, int m, int n, float* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnSgeqrf_bufferSize(handle, m, n, &fA, lda, &Lwork);
  status = hipsolverDnSgeqrf_bufferSize(handle, m, n, &fA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgeqrf_bufferSize(cusolverDnHandle_t handle, int m, int n, double * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgeqrf_bufferSize(hipsolverHandle_t handle, int m, int n, double* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnDgeqrf_bufferSize(handle, m, n, &dA, lda, &Lwork);
  status = hipsolverDnDgeqrf_bufferSize(handle, m, n, &dA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgeqrf_bufferSize(cusolverDnHandle_t handle, int m, int n, cuComplex * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgeqrf_bufferSize(hipsolverHandle_t handle, int m, int n, hipFloatComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnCgeqrf_bufferSize(handle, m, n, &complexA, lda, &Lwork);
  status = hipsolverDnCgeqrf_bufferSize(handle, m, n, &complexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgeqrf_bufferSize(cusolverDnHandle_t handle, int m, int n, cuDoubleComplex * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgeqrf_bufferSize(hipsolverHandle_t handle, int m, int n, hipDoubleComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnZgeqrf_bufferSize(handle, m, n, &dComplexA, lda, &Lwork);
  status = hipsolverDnZgeqrf_bufferSize(handle, m, n, &dComplexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgeqrf(cusolverDnHandle_t handle, int m, int n, float * A, int lda, float * TAU, float * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgeqrf(hipsolverHandle_t handle, int m, int n, float* A, int lda, float* tau, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSgeqrf(handle, m, n, &fA, lda, &fTAU, &fWorkspace, Lwork, &devInfo);
  status = hipsolverDnSgeqrf(handle, m, n, &fA, lda, &fTAU, &fWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgeqrf(cusolverDnHandle_t handle, int m, int n, double * A, int lda, double * TAU, double * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgeqrf(hipsolverHandle_t handle, int m, int n, double* A, int lda, double* tau, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDgeqrf(handle, m, n, &dA, lda, &dTAU, &dWorkspace, Lwork, &devInfo);
  status = hipsolverDnDgeqrf(handle, m, n, &dA, lda, &dTAU, &dWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgeqrf(cusolverDnHandle_t handle, int m, int n, cuComplex * A, int lda, cuComplex * TAU, cuComplex * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgeqrf(hipsolverHandle_t handle, int m, int n, hipFloatComplex* A, int lda, hipFloatComplex* tau, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCgeqrf(handle, m, n, &complexA, lda, &complexTAU, &complexWorkspace, Lwork, &devInfo);
  status = hipsolverDnCgeqrf(handle, m, n, &complexA, lda, &complexTAU, &complexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgeqrf(cusolverDnHandle_t handle, int m, int n, cuDoubleComplex * A, int lda, cuDoubleComplex * TAU, cuDoubleComplex * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgeqrf(hipsolverHandle_t handle, int m, int n, hipDoubleComplex* A, int lda, hipDoubleComplex* tau, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZgeqrf(handle, m, n, &dComplexA, lda, &dComplexTAU, &dComplexWorkspace, Lwork, &devInfo);
  status = hipsolverDnZgeqrf(handle, m, n, &dComplexA, lda, &dComplexTAU, &dComplexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSsytrf_bufferSize(cusolverDnHandle_t handle, int n, float * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSsytrf_bufferSize(hipsolverHandle_t handle, int n, float* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnSsytrf_bufferSize(handle, n, &fA, lda, &Lwork);
  status = hipsolverDnSsytrf_bufferSize(handle, n, &fA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDsytrf_bufferSize(cusolverDnHandle_t handle, int n, double * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDsytrf_bufferSize(hipsolverHandle_t handle, int n, double* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnDsytrf_bufferSize(handle, n, &dA, lda, &Lwork);
  status = hipsolverDnDsytrf_bufferSize(handle, n, &dA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCsytrf_bufferSize(cusolverDnHandle_t handle, int n, cuComplex * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCsytrf_bufferSize(hipsolverHandle_t handle, int n, hipFloatComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnCsytrf_bufferSize(handle, n, &complexA, lda, &Lwork);
  status = hipsolverDnCsytrf_bufferSize(handle, n, &complexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZsytrf_bufferSize(cusolverDnHandle_t handle, int n, cuDoubleComplex * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZsytrf_bufferSize(hipsolverHandle_t handle, int n, hipDoubleComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnZsytrf_bufferSize(handle, n, &dComplexA, lda, &Lwork);
  status = hipsolverDnZsytrf_bufferSize(handle, n, &dComplexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSsytrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, float * A, int lda, int * ipiv, float * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSsytrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* A, int lda, int* ipiv, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSsytrf(handle, fillMode, n, &fA, lda, &devIpiv, &fWorkspace, Lwork, &devInfo);
  status = hipsolverDnSsytrf(handle, fillMode, n, &fA, lda, &devIpiv, &fWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDsytrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, double * A, int lda, int * ipiv, double * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDsytrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* A, int lda, int* ipiv, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDsytrf(handle, fillMode, n, &dA, lda, &devIpiv, &dWorkspace, Lwork, &devInfo);
  status = hipsolverDnDsytrf(handle, fillMode, n, &dA, lda, &devIpiv, &dWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCsytrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuComplex * A, int lda, int * ipiv, cuComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCsytrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, int* ipiv, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCsytrf(handle, fillMode, n, &complexA, lda, &devIpiv, &complexWorkspace, Lwork, &devInfo);
  status = hipsolverDnCsytrf(handle, fillMode, n, &complexA, lda, &devIpiv, &complexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZsytrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, int * ipiv, cuDoubleComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZsytrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, int* ipiv, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZsytrf(handle, fillMode, n, &dComplexA, lda, &devIpiv, &dComplexWorkspace, Lwork, &devInfo);
  status = hipsolverDnZsytrf(handle, fillMode, n, &dComplexA, lda, &devIpiv, &dComplexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgebrd_bufferSize(cusolverDnHandle_t handle, int m, int n, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgebrd_bufferSize(hipsolverHandle_t handle, int m, int n, int* lwork);
  // CHECK: status = hipsolverDnSgebrd_bufferSize(handle, m, n, &Lwork);
  status = hipsolverDnSgebrd_bufferSize(handle, m, n, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgebrd_bufferSize(cusolverDnHandle_t handle, int m, int n, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgebrd_bufferSize(hipsolverHandle_t handle, int m, int n, int* lwork);
  // CHECK: status = hipsolverDnDgebrd_bufferSize(handle, m, n, &Lwork);
  status = hipsolverDnDgebrd_bufferSize(handle, m, n, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgebrd_bufferSize(cusolverDnHandle_t handle, int m, int n, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgebrd_bufferSize(hipsolverHandle_t handle, int m, int n, int* lwork);
  // CHECK: status = hipsolverDnCgebrd_bufferSize(handle, m, n, &Lwork);
  status = hipsolverDnCgebrd_bufferSize(handle, m, n, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgebrd_bufferSize(cusolverDnHandle_t handle, int m, int n, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgebrd_bufferSize(hipsolverHandle_t handle, int m, int n, int* lwork);
  // CHECK: status = hipsolverDnZgebrd_bufferSize(handle, m, n, &Lwork);
  status = hipsolverDnZgebrd_bufferSize(handle, m, n, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgebrd(cusolverDnHandle_t handle, int m, int n, float * A, int lda, float * D, float * E, float * TAUQ, float * TAUP, float * Work, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgebrd(hipsolverHandle_t handle, int m, int n, float* A, int lda, float* D, float* E, float* tauq, float* taup, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSgebrd(handle, m, n, &fA, lda, &fD, &fE, &fTAUQ, &fTAUP, &fWorkspace, Lwork, &devInfo);
  status = hipsolverDnSgebrd(handle, m, n, &fA, lda, &fD, &fE, &fTAUQ, &fTAUP, &fWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgebrd(cusolverDnHandle_t handle, int m, int n, double * A, int lda, double * D, double * E, double * TAUQ, double * TAUP, double * Work, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgebrd(hipsolverHandle_t handle, int m, int n, double* A, int lda, double* D, double* E, double* tauq, double* taup, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDgebrd(handle, m, n, &dA, lda, &dD, &dE, &dTAUQ, &dTAUP, &dWorkspace, Lwork, &devInfo);
  status = hipsolverDnDgebrd(handle, m, n, &dA, lda, &dD, &dE, &dTAUQ, &dTAUP, &dWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgebrd(cusolverDnHandle_t handle, int m, int n, cuComplex * A, int lda, float * D, float * E, cuComplex * TAUQ, cuComplex * TAUP, cuComplex * Work, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgebrd(hipsolverHandle_t handle, int m, int n, hipFloatComplex* A, int lda, float* D, float* E, hipFloatComplex* tauq, hipFloatComplex* taup, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCgebrd(handle, m, n, &complexA, lda, &fD, &fE, &complexTAUQ, &complexTAUP, &complexWorkspace, Lwork, &devInfo);
  status = hipsolverDnCgebrd(handle, m, n, &complexA, lda, &fD, &fE, &complexTAUQ, &complexTAUP, &complexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgebrd(cusolverDnHandle_t handle, int m, int n, cuDoubleComplex * A, int lda, double * D, double * E, cuDoubleComplex * TAUQ, cuDoubleComplex * TAUP, cuDoubleComplex * Work, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgebrd(hipsolverHandle_t handle, int m, int n, hipDoubleComplex* A, int lda, double* D, double* E, hipDoubleComplex* tauq, hipDoubleComplex* taup, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZgebrd(handle, m, n, &dComplexA, lda, &dD, &dE, &dComplexTAUQ, &dComplexTAUP, &dComplexWorkspace, Lwork, &devInfo);
  status = hipsolverDnZgebrd(handle, m, n, &dComplexA, lda, &dD, &dE, &dComplexTAUQ, &dComplexTAUP, &dComplexWorkspace, Lwork, &devInfo);

#if CUDA_VERSION >= 8000
  // CHECK: hipsolverEigType_t eigType;
  // CHECK-NEXT: hipsolverEigType_t EIG_TYPE_1 = HIPSOLVER_EIG_TYPE_1;
  // CHECK-NEXT: hipsolverEigType_t EIG_TYPE_2 = HIPSOLVER_EIG_TYPE_2;
  // CHECK-NEXT: hipsolverEigType_t EIG_TYPE_3 = HIPSOLVER_EIG_TYPE_3;
  hipsolverEigType_t eigType;
  hipsolverEigType_t EIG_TYPE_1 = HIPSOLVER_EIG_TYPE_1;
  hipsolverEigType_t EIG_TYPE_2 = HIPSOLVER_EIG_TYPE_2;
  hipsolverEigType_t EIG_TYPE_3 = HIPSOLVER_EIG_TYPE_3;

  // CHECK: hipsolverEigMode_t eigMode;
  // CHECK-NEXT: hipsolverEigMode_t SOLVER_EIG_MODE_NOVECTOR = HIPSOLVER_EIG_MODE_NOVECTOR;
  // CHECK-NEXT: hipsolverEigMode_t SOLVER_EIG_MODE_VECTOR = HIPSOLVER_EIG_MODE_VECTOR;
  hipsolverEigMode_t eigMode;
  hipsolverEigMode_t SOLVER_EIG_MODE_NOVECTOR = HIPSOLVER_EIG_MODE_NOVECTOR;
  hipsolverEigMode_t SOLVER_EIG_MODE_VECTOR = HIPSOLVER_EIG_MODE_VECTOR;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSorgqr_bufferSize(cusolverDnHandle_t handle, int m, int n, int k, const float * A, int lda, const float * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSorgqr_bufferSize(hipsolverHandle_t handle, int m, int n, int k, const float* A, int lda, const float* tau, int* lwork);
  // CHECK: status = hipsolverDnSorgqr_bufferSize(handle, m, n, k, &fA, lda, &fTAU, &Lwork);
  status = hipsolverDnSorgqr_bufferSize(handle, m, n, k, &fA, lda, &fTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDorgqr_bufferSize(cusolverDnHandle_t handle, int m, int n, int k, const double * A, int lda, const double * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDorgqr_bufferSize(hipsolverHandle_t handle, int m, int n, int k, const double* A, int lda, const double* tau, int* lwork);
  // CHECK: status = hipsolverDnDorgqr_bufferSize(handle, m, n, k, &dA, lda, &dTAU, &Lwork);
  status = hipsolverDnDorgqr_bufferSize(handle, m, n, k, &dA, lda, &dTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCungqr_bufferSize(cusolverDnHandle_t handle, int m, int n, int k, const cuComplex * A, int lda, const cuComplex * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCungqr_bufferSize(hipsolverHandle_t handle, int m, int n, int k, const hipFloatComplex* A, int lda, const hipFloatComplex* tau, int* lwork);
  // CHECK: status = hipsolverDnCungqr_bufferSize(handle, m, n, k, &complexA, lda, &complexTAU, &Lwork);
  status = hipsolverDnCungqr_bufferSize(handle, m, n, k, &complexA, lda, &complexTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZungqr_bufferSize(cusolverDnHandle_t handle, int m, int n, int k, const cuDoubleComplex *A, int lda, const cuDoubleComplex *tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZungqr_bufferSize(hipsolverHandle_t handle, int m, int n, int k, const hipDoubleComplex* A, int lda, const hipDoubleComplex* tau, int* lwork);
  // CHECK: status = hipsolverDnZungqr_bufferSize(handle, m, n, k, &dComplexA, lda, &dComplexTAU, &Lwork);
  status = hipsolverDnZungqr_bufferSize(handle, m, n, k, &dComplexA, lda, &dComplexTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSorgqr(cusolverDnHandle_t handle, int m, int n, int k, float * A, int lda, const float * tau, float * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSorgqr(hipsolverHandle_t handle, int m, int n, int k, float* A, int lda, const float* tau, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSorgqr(handle, m, n, k, &fA, lda, &fTAU, &fWorkspace, Lwork, &info);
  status = hipsolverDnSorgqr(handle, m, n, k, &fA, lda, &fTAU, &fWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDorgqr(cusolverDnHandle_t handle, int m, int n, int k, double * A, int lda, const double * tau, double * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDorgqr(hipsolverHandle_t handle, int m, int n, int k, double* A, int lda, const double* tau, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDorgqr(handle, m, n, k, &dA, lda, &dTAU, &dWorkspace, Lwork, &info);
  status = hipsolverDnDorgqr(handle, m, n, k, &dA, lda, &dTAU, &dWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCungqr(cusolverDnHandle_t handle, int m, int n, int k, cuComplex * A, int lda, const cuComplex * tau, cuComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCungqr(hipsolverHandle_t handle, int m, int n, int k, hipFloatComplex* A, int lda, const hipFloatComplex* tau, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCungqr(handle, m, n, k, &complexA, lda, &complexTAU, &complexWorkspace, Lwork, &info);
  status = hipsolverDnCungqr(handle, m, n, k, &complexA, lda, &complexTAU, &complexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZungqr(cusolverDnHandle_t handle, int m, int n, int k, cuDoubleComplex * A, int lda, const cuDoubleComplex *tau, cuDoubleComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZungqr(hipsolverHandle_t handle, int m, int n, int k, hipDoubleComplex* A, int lda, const hipDoubleComplex* tau, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZungqr(handle, m, n, k, &dComplexA, lda, &dComplexTAU, &dComplexWorkspace, Lwork, &info);
  status = hipsolverDnZungqr(handle, m, n, k, &dComplexA, lda, &dComplexTAU, &dComplexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSormqr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const float * A, int lda, const float * tau, const float * C, int ldc, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSormqr_bufferSize(hipsolverHandle_t  handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const float* A, int lda, const float* tau, const float* C, int ldc, int* lwork);
  // CHECK: status = hipsolverDnSormqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &fA, lda, &fTAU, &fC, ldc, &Lwork);
  status = hipsolverDnSormqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &fA, lda, &fTAU, &fC, ldc, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDormqr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const double * A, int lda, const double * tau, const double * C, int ldc, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDormqr_bufferSize(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const double* A, int lda, const double* tau, const double* C, int ldc, int* lwork);
  // CHECK: status = hipsolverDnDormqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &dA, lda, &dTAU, &dC, ldc, &Lwork);
  status = hipsolverDnDormqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &dA, lda, &dTAU, &dC, ldc, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCunmqr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const cuComplex * A, int lda, const cuComplex * tau, const cuComplex * C, int ldc, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCunmqr_bufferSize(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const hipFloatComplex* A, int lda, const hipFloatComplex* tau, const hipFloatComplex* C, int ldc, int* lwork);
  // CHECK: status = hipsolverDnCunmqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &complexA, lda, &complexTAU, &complexC, ldc, &Lwork);
  status = hipsolverDnCunmqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &complexA, lda, &complexTAU, &complexC, ldc, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZunmqr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const cuDoubleComplex *A, int lda, const cuDoubleComplex *tau, const cuDoubleComplex *C, int ldc, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZunmqr_bufferSize(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const hipDoubleComplex* A, int lda, const hipDoubleComplex* tau, const hipDoubleComplex* C, int ldc, int* lwork);
  // CHECK: status = hipsolverDnZunmqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &dComplexA, lda, &dComplexTAU, &dComplexC, ldc, &Lwork);
  status = hipsolverDnZunmqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &dComplexA, lda, &dComplexTAU, &dComplexC, ldc, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSormqr(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const float * A, int lda, const float * tau, float * C, int ldc, float * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSormqr(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const float* A, int lda, const float* tau, float* C, int ldc, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSormqr(handle, blasSideMode, blasOperation, m, n, k, &fA, lda, &fTAU, &fC, ldc, &fWorkspace, Lwork, &devInfo);
  status = hipsolverDnSormqr(handle, blasSideMode, blasOperation, m, n, k, &fA, lda, &fTAU, &fC, ldc, &fWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDormqr(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const double * A, int lda, const double * tau, double * C, int ldc, double * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDormqr(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const double* A, int lda, const double* tau, double* C, int ldc, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDormqr(handle, blasSideMode, blasOperation, m, n, k, &dA, lda, &dTAU, &dC, ldc, &dWorkspace, Lwork, &devInfo);
  status = hipsolverDnDormqr(handle, blasSideMode, blasOperation, m, n, k, &dA, lda, &dTAU, &dC, ldc, &dWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCunmqr(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const cuComplex * A, int lda, const cuComplex * tau, cuComplex * C, int ldc, cuComplex * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCunmqr(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const hipFloatComplex* A, int lda, const hipFloatComplex* tau, hipFloatComplex* C, int ldc, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCunmqr(handle, blasSideMode, blasOperation, m, n, k, &complexA, lda, &complexTAU, &complexC, ldc, &complexWorkspace, Lwork, &devInfo);
  status = hipsolverDnCunmqr(handle, blasSideMode, blasOperation, m, n, k, &complexA, lda, &complexTAU, &complexC, ldc, &complexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZunmqr(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const cuDoubleComplex *A, int lda, const cuDoubleComplex *tau, cuDoubleComplex * C, int ldc, cuDoubleComplex * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZunmqr(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const hipDoubleComplex* A, int lda, const hipDoubleComplex* tau, hipDoubleComplex* C, int ldc, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZunmqr(handle, blasSideMode, blasOperation, m, n, k, &dComplexA, lda, &dComplexTAU, &dComplexC, ldc, &dComplexWorkspace, Lwork, &devInfo);
  status = hipsolverDnZunmqr(handle, blasSideMode, blasOperation, m, n, k, &dComplexA, lda, &dComplexTAU, &dComplexC, ldc, &dComplexWorkspace, Lwork, &devInfo);
#endif

#if CUDA_VERSION >= 9000
  // CHECK: hipsolverSyevjInfo_t syevj_info;
  hipsolverSyevjInfo_t syevj_info;

  // CHECK: hipsolverGesvdjInfo_t gesvdj_info;
  hipsolverGesvdjInfo_t gesvdj_info;
#endif

#if CUDA_VERSION >= 9010
  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSpotrfBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, float * Aarray[], int lda, int * infoArray, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSpotrfBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* A[], int lda, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnSpotrfBatched(handle, fillMode, n, fAarray, lda, &infoArray, batchSize);
  status = hipsolverDnSpotrfBatched(handle, fillMode, n, fAarray, lda, &infoArray, batchSize);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDpotrfBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, double * Aarray[], int lda, int * infoArray, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDpotrfBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* A[], int lda, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnDpotrfBatched(handle, fillMode, n, dAarray, lda, &infoArray, batchSize);
  status = hipsolverDnDpotrfBatched(handle, fillMode, n, dAarray, lda, &infoArray, batchSize);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCpotrfBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuComplex * Aarray[], int lda, int * infoArray, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCpotrfBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipFloatComplex* A[], int lda, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnCpotrfBatched(handle, fillMode, n, complexAarray, lda, &infoArray, batchSize);
  status = hipsolverDnCpotrfBatched(handle, fillMode, n, complexAarray, lda, &infoArray, batchSize);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZpotrfBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuDoubleComplex * Aarray[], int lda, int * infoArray, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZpotrfBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipDoubleComplex* A[], int lda, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnZpotrfBatched(handle, fillMode, n, dcomplexAarray, lda, &infoArray, batchSize);
  status = hipsolverDnZpotrfBatched(handle, fillMode, n, dcomplexAarray, lda, &infoArray, batchSize);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSpotrsBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, float * A[], int lda, float * B[], int ldb, int * d_info, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSpotrsBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, float* A[], int lda, float* B[], int ldb, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnSpotrsBatched(handle, fillMode, n, nrhs, fAarray, lda, fBarray, ldb, &infoArray, batchSize);
  status = hipsolverDnSpotrsBatched(handle, fillMode, n, nrhs, fAarray, lda, fBarray, ldb, &infoArray, batchSize);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDpotrsBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, double * A[], int lda, double * B[], int ldb, int * d_info, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDpotrsBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, double* A[], int lda, double* B[], int ldb, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnDpotrsBatched(handle, fillMode, n, nrhs, dAarray, lda, dBarray, ldb, &infoArray, batchSize);
  status = hipsolverDnDpotrsBatched(handle, fillMode, n, nrhs, dAarray, lda, dBarray, ldb, &infoArray, batchSize);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCpotrsBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, cuComplex * A[], int lda, cuComplex * B[], int ldb, int * d_info, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCpotrsBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, hipFloatComplex* A[], int lda, hipFloatComplex* B[], int ldb, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnCpotrsBatched(handle, fillMode, n, nrhs, complexAarray, lda, complexBarray, ldb, &infoArray, batchSize);
  status = hipsolverDnCpotrsBatched(handle, fillMode, n, nrhs, complexAarray, lda, complexBarray, ldb, &infoArray, batchSize);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZpotrsBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, cuDoubleComplex * A[], int lda, cuDoubleComplex * B[], int ldb, int * d_info, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZpotrsBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, hipDoubleComplex* A[], int lda, hipDoubleComplex* B[], int ldb, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnZpotrsBatched(handle, fillMode, n, nrhs, dcomplexAarray, lda, dcomplexBarray, ldb, &infoArray, batchSize);
  status = hipsolverDnZpotrsBatched(handle, fillMode, n, nrhs, dcomplexAarray, lda, dcomplexBarray, ldb, &infoArray, batchSize);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipsolverEigRange_t eigRange;
  // CHECK-NEXT: hipsolverEigRange_t EIG_RANGE_ALL = HIPSOLVER_EIG_RANGE_ALL;
  // CHECK-NEXT: hipsolverEigRange_t EIG_RANGE_I = HIPSOLVER_EIG_RANGE_I;
  // CHECK-NEXT: hipsolverEigRange_t EIG_RANGE_V = HIPSOLVER_EIG_RANGE_V;
  hipsolverEigRange_t eigRange;
  hipsolverEigRange_t EIG_RANGE_ALL = HIPSOLVER_EIG_RANGE_ALL;
  hipsolverEigRange_t EIG_RANGE_I = HIPSOLVER_EIG_RANGE_I;
  hipsolverEigRange_t EIG_RANGE_V = HIPSOLVER_EIG_RANGE_V;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSpotri_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, float * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSpotri_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnSpotri_bufferSize(handle, fillMode, n, &fA, lda, &Lwork);
  status = hipsolverDnSpotri_bufferSize(handle, fillMode, n, &fA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDpotri_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, double * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDpotri_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnDpotri_bufferSize(handle, fillMode, n, &dA, lda, &Lwork);
  status = hipsolverDnDpotri_bufferSize(handle, fillMode, n, &dA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCpotri_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuComplex * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCpotri_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnCpotri_bufferSize(handle, fillMode, n, &complexA, lda, &Lwork);
  status = hipsolverDnCpotri_bufferSize(handle, fillMode, n, &complexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZpotri_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZpotri_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnZpotri_bufferSize(handle, fillMode, n, &dComplexA, lda, &Lwork);
  status = hipsolverDnZpotri_bufferSize(handle, fillMode, n, &dComplexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSpotri(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, float * A, int lda, float * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSpotri(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* A, int lda, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSpotri(handle, fillMode, n, &fA, lda, &fWorkspace, Lwork, &infoArray);
  status = hipsolverDnSpotri(handle, fillMode, n, &fA, lda, &fWorkspace, Lwork, &infoArray);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDpotri(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, double * A, int lda, double * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDpotri(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* A, int lda, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDpotri(handle, fillMode, n, &dA, lda, &dWorkspace, Lwork, &infoArray);
  status = hipsolverDnDpotri(handle, fillMode, n, &dA, lda, &dWorkspace, Lwork, &infoArray);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCpotri(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuComplex * A, int lda, cuComplex * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCpotri(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCpotri(handle, fillMode, n, &complexA, lda, &complexWorkspace, Lwork, &infoArray);
  status = hipsolverDnCpotri(handle, fillMode, n, &complexA, lda, &complexWorkspace, Lwork, &infoArray);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZpotri(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, cuDoubleComplex * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZpotri(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZpotri(handle, fillMode, n, &dComplexA, lda, &dComplexWorkspace, Lwork, &infoArray);
  status = hipsolverDnZpotri(handle, fillMode, n, &dComplexA, lda, &dComplexWorkspace, Lwork, &infoArray);
#endif

#if CUDA_VERSION >= 10020
  // CHECK: int solver_int = 0;
  // CHECK: int lm = 0;
  // CHECK: int ln = 0;
  // CHECK: int lnrhs = 0;
  // CHECK: int ldda = 0;
  // CHECK: int lddb = 0;
  // CHECK: int lddx = 0;
  // CHECK: int dipiv = 0;
  // CHECK: int iter = 0;
  // CHECK: int d_info = 0;
  int solver_int = 0;
  int lm = 0;
  int ln = 0;
  int lnrhs = 0;
  int ldda = 0;
  int lddb = 0;
  int lddx = 0;
  int dipiv = 0;
  int iter = 0;
  int d_info = 0;

  // CHECK: hipsolverStatus_t STATUS_IRS_PARAMS_INVALID = HIPSOLVER_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipsolverStatus_t STATUS_IRS_INTERNAL_ERROR = HIPSOLVER_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipsolverStatus_t STATUS_IRS_NOT_SUPPORTED = HIPSOLVER_STATUS_NOT_SUPPORTED;
  hipsolverStatus_t STATUS_IRS_PARAMS_INVALID = HIPSOLVER_STATUS_INVALID_VALUE;
  hipsolverStatus_t STATUS_IRS_INTERNAL_ERROR = HIPSOLVER_STATUS_INTERNAL_ERROR;
  hipsolverStatus_t STATUS_IRS_NOT_SUPPORTED = HIPSOLVER_STATUS_NOT_SUPPORTED;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZZgesv(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, cuDoubleComplex * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, cuDoubleComplex * dB, cusolver_int_t lddb, cuDoubleComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZZgesv(hipsolverHandle_t handle, int n, int nrhs, hipDoubleComplex* A, int lda, int* devIpiv, hipDoubleComplex* B, int ldb, hipDoubleComplex* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnZZgesv(handle, ln, lnrhs, &dComplexA, ldda, &dipiv, &dComplexB, lddb, &dComplexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnZZgesv(handle, ln, lnrhs, &dComplexA, ldda, &dipiv, &dComplexB, lddb, &dComplexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCCgesv(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, cuComplex * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, cuComplex * dB, cusolver_int_t lddb, cuComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCCgesv(hipsolverHandle_t handle, int n, int nrhs, hipFloatComplex* A, int lda, int* devIpiv, hipFloatComplex* B, int ldb, hipFloatComplex* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnCCgesv(handle, ln, lnrhs, &complexA, ldda, &dipiv, &complexB, lddb, &complexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnCCgesv(handle, ln, lnrhs, &complexA, ldda, &dipiv, &complexB, lddb, &complexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDDgesv(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, double * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, double * dB, cusolver_int_t lddb, double * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDDgesv(hipsolverHandle_t handle, int n, int nrhs, double* A, int lda, int* devIpiv, double* B, int ldb, double* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnDDgesv(handle, ln, lnrhs, &dA, ldda, &dipiv, &dB, lddb, &dX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnDDgesv(handle, ln, lnrhs, &dA, ldda, &dipiv, &dB, lddb, &dX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSSgesv(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, float * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, float * dB, cusolver_int_t lddb, float * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSSgesv(hipsolverHandle_t handle, int n, int nrhs, float* A, int lda, int* devIpiv, float* B, int ldb, float* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnSSgesv(handle, ln, lnrhs, &fA, ldda, &dipiv, &fB, lddb, &fX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnSSgesv(handle, ln, lnrhs, &fA, ldda, &dipiv, &fB, lddb, &fX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZZgesv_bufferSize(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, cuDoubleComplex * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, cuDoubleComplex * dB, cusolver_int_t lddb, cuDoubleComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZZgesv_bufferSize(hipsolverHandle_t handle, int n, int nrhs, hipDoubleComplex* A, int lda, int* devIpiv, hipDoubleComplex* B, int ldb, hipDoubleComplex* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnZZgesv_bufferSize(handle, ln, lnrhs, &dComplexA, ldda, &dipiv, &dComplexB, lddb, &dComplexX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnZZgesv_bufferSize(handle, ln, lnrhs, &dComplexA, ldda, &dipiv, &dComplexB, lddb, &dComplexX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCCgesv_bufferSize(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, cuComplex * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, cuComplex * dB, cusolver_int_t lddb, cuComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCCgesv_bufferSize(hipsolverHandle_t handle, int n, int nrhs, hipFloatComplex* A, int lda, int* devIpiv, hipFloatComplex* B, int ldb, hipFloatComplex* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnCCgesv_bufferSize(handle, ln, lnrhs, &complexA, ldda, &dipiv, &complexB, lddb, &complexX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnCCgesv_bufferSize(handle, ln, lnrhs, &complexA, ldda, &dipiv, &complexB, lddb, &complexX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDDgesv_bufferSize(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, double * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, double * dB, cusolver_int_t lddb, double * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDDgesv_bufferSize(hipsolverHandle_t handle, int n, int nrhs, double* A, int lda, int* devIpiv, double* B, int ldb, double* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnDDgesv_bufferSize(handle, ln, lnrhs, &dA, ldda, &dipiv, &dB, lddb, &dX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnDDgesv_bufferSize(handle, ln, lnrhs, &dA, ldda, &dipiv, &dB, lddb, &dX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSSgesv_bufferSize(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, float * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, float * dB, cusolver_int_t lddb, float * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSSgesv_bufferSize(hipsolverHandle_t handle, int n, int nrhs, float* A, int lda, int* devIpiv, float* B, int ldb, float* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnSSgesv_bufferSize(handle, ln, lnrhs, &fA, ldda, &dipiv, &fB, lddb, &fX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnSSgesv_bufferSize(handle, ln, lnrhs, &fA, ldda, &dipiv, &fB, lddb, &fX, lddx, &Workspace, &lwork_bytes);
#endif

#if CUDA_VERSION >= 11000
  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZZgels(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, cuDoubleComplex * dA, cusolver_int_t ldda, cuDoubleComplex * dB, cusolver_int_t lddb, cuDoubleComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZZgels(hipsolverHandle_t handle, int m, int n, int nrhs, hipDoubleComplex* A, int lda, hipDoubleComplex* B, int ldb, hipDoubleComplex* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnZZgels(handle, lm, ln, lnrhs, &dComplexA, ldda, &dComplexB, lddb, &dComplexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnZZgels(handle, lm, ln, lnrhs, &dComplexA, ldda, &dComplexB, lddb, &dComplexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCCgels(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, cuComplex * dA, cusolver_int_t ldda, cuComplex * dB, cusolver_int_t lddb, cuComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCCgels(hipsolverHandle_t handle, int m, int n, int nrhs, hipFloatComplex* A, int lda, hipFloatComplex* B, int ldb, hipFloatComplex* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnCCgels(handle, lm, ln, lnrhs, &complexA, ldda, &complexB, lddb, &complexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnCCgels(handle, lm, ln, lnrhs, &complexA, ldda, &complexB, lddb, &complexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDDgels(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, double * dA, cusolver_int_t ldda, double * dB, cusolver_int_t lddb, double * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDDgels(hipsolverHandle_t handle, int m, int n, int nrhs, double* A, int lda, double* B, int ldb, double* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnDDgels(handle, lm, ln, lnrhs, &dA, ldda, &dB, lddb, &dX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnDDgels(handle, lm, ln, lnrhs, &dA, ldda, &dB, lddb, &dX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSSgels(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, float * dA, cusolver_int_t ldda, float * dB, cusolver_int_t lddb, float * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSSgels(hipsolverHandle_t handle, int m, int n, int nrhs, float* A, int lda, float* B, int ldb, float* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnSSgels(handle, lm, ln, lnrhs, &fA, ldda, &fB, lddb, &fX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnSSgels(handle, lm, ln, lnrhs, &fA, ldda, &fB, lddb, &fX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZZgels_bufferSize(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, cuDoubleComplex * dA, cusolver_int_t ldda, cuDoubleComplex * dB, cusolver_int_t lddb, cuDoubleComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZZgels_bufferSize(hipsolverHandle_t handle, int m, int n, int nrhs, hipDoubleComplex* A, int lda, hipDoubleComplex* B, int ldb, hipDoubleComplex* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnZZgels_bufferSize(handle, lm, ln, lnrhs, &dComplexA, ldda, &dComplexB, lddb, &dComplexX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnZZgels_bufferSize(handle, lm, ln, lnrhs, &dComplexA, ldda, &dComplexB, lddb, &dComplexX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCCgels_bufferSize(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, cuComplex * dA, cusolver_int_t ldda, cuComplex * dB, cusolver_int_t lddb, cuComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCCgels_bufferSize(hipsolverHandle_t handle, int m, int n, int nrhs, hipFloatComplex* A, int lda, hipFloatComplex* B, int ldb, hipFloatComplex* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnCCgels_bufferSize(handle, lm, ln, lnrhs, &complexA, ldda, &complexB, lddb, &complexX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnCCgels_bufferSize(handle, lm, ln, lnrhs, &complexA, ldda, &complexB, lddb, &complexX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDDgels_bufferSize(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, double * dA, cusolver_int_t ldda, double * dB, cusolver_int_t lddb, double * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDDgels_bufferSize(hipsolverHandle_t handle, int m, int n, int nrhs, double* A, int lda, double* B, int ldb, double* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnDDgels_bufferSize(handle, lm, ln, lnrhs, &dA, ldda, &dB, lddb, &dX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnDDgels_bufferSize(handle, lm, ln, lnrhs, &dA, ldda, &dB, lddb, &dX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSSgels_bufferSize(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, float * dA, cusolver_int_t ldda, float * dB, cusolver_int_t lddb, float * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSSgels_bufferSize(hipsolverHandle_t handle, int m, int n, int nrhs, float* A, int lda, float* B, int ldb, float* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnSSgels_bufferSize(handle, lm, ln, lnrhs, &fA, ldda, &fB, lddb, &fX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnSSgels_bufferSize(handle, lm, ln, lnrhs, &fA, ldda, &fB, lddb, &fX, lddx, &Workspace, &lwork_bytes);
#endif

  return 0;
}
