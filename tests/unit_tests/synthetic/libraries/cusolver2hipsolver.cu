// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipsolver.h"
#include "hipsolver.h"

int main() {
  printf("19. cuSOLVER API to hipSOLVER API synthetic test\n");

  int m = 0;
  int n = 0;
  int k = 0;
  int il = 0;
  int iu = 0;
  int imeig = 0;
  int nrhs = 0;
  int lda = 0;
  int ldb = 0;
  int ldc = 0;
  int ldu = 0;
  int ldvt = 0;
  int Lwork = 0;
  int devIpiv = 0;
  int devInfo = 0;
  int info = 0;
  int infoArray = 0;
  int batchSize = 0;
  float fA = 0.f;
  float fB = 0.f;
  float fC = 0.f;
  float fD = 0.f;
  float fE = 0.f;
  float fS = 0.f;
  float fU = 0.f;
  float fvl = 0.f;
  float fvu = 0.f;
  float fVT = 0.f;
  float fX = 0.f;
  float fW = 0.f;
  float fTAU = 0.f;
  float fTAUQ = 0.f;
  float fTAUP = 0.f;
  double dA = 0.f;
  double dB = 0.f;
  double dC = 0.f;
  double dD = 0.f;
  double dE = 0.f;
  double dS = 0.f;
  double dU = 0.f;
  double dvl = 0.f;
  double dvu = 0.f;
  double dVT = 0.f;
  double dX = 0.f;
  double dW = 0.f;
  double dTAU = 0.f;
  double dTAUQ = 0.f;
  double dTAUP = 0.f;
  float fWorkspace = 0.f;
  float frWork = 0.f;
  double dWorkspace = 0.f;
  double drWork = 0.f;
  void *Workspace = nullptr;
  size_t lwork_bytes = 0;

  signed char jobu = 0;
  signed char jobvt = 0;

  float** fAarray = 0;
  float** fBarray = 0;
  double** dAarray = 0;
  double** dBarray = 0;

  // CHECK: hipDoubleComplex dComplexA, dComplexB, dComplexC, dComplexD, dComplexE, dComplexS, dComplexU, dComplexVT, dComplexX, dComplexWorkspace, dComplexrWork, dComplexTAU, dComplexTAUQ, dComplexTAUP;
  hipDoubleComplex dComplexA, dComplexB, dComplexC, dComplexD, dComplexE, dComplexS, dComplexU, dComplexVT, dComplexX, dComplexWorkspace, dComplexrWork, dComplexTAU, dComplexTAUQ, dComplexTAUP;

  // CHECK: hipComplex complexA, complexB, complexC, complexD, complexE, complexS, complexU, complexVT, complexX, complexWorkspace, complexrWork, complexTAU, complexTAUQ, complexTAUP;
  hipComplex complexA, complexB, complexC, complexD, complexE, complexS, complexU, complexVT, complexX, complexWorkspace, complexrWork, complexTAU, complexTAUQ, complexTAUP;

  // CHECK: hipDoubleComplex** dcomplexAarray = 0;
  // CHECK-NEXT: hipDoubleComplex** dcomplexBarray = 0;
  hipDoubleComplex** dcomplexAarray = 0;
  hipDoubleComplex** dcomplexBarray = 0;

  // CHECK: hipComplex** complexAarray = 0;
  // CHECK-NEXT: hipComplex** complexBarray = 0;
  hipComplex** complexAarray = 0;
  hipComplex** complexBarray = 0;

  // CHECK: hipsolverHandle_t handle;
  hipsolverHandle_t handle;

  // CHECK: hipblasFillMode_t fillMode;
  hipblasFillMode_t fillMode;

  // CHECK: hipsolverStatus_t status;
  // CHECK-NEXT: hipsolverStatus_t STATUS_SUCCESS = HIPSOLVER_STATUS_SUCCESS;
  // CHECK-NEXT: hipsolverStatus_t STATUS_NOT_INITIALIZED = HIPSOLVER_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT: hipsolverStatus_t STATUS_ALLOC_FAILED = HIPSOLVER_STATUS_ALLOC_FAILED;
  // CHECK-NEXT: hipsolverStatus_t STATUS_INVALID_VALUE = HIPSOLVER_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipsolverStatus_t STATUS_ARCH_MISMATCH = HIPSOLVER_STATUS_ARCH_MISMATCH;
  // CHECK-NEXT: hipsolverStatus_t STATUS_MAPPING_ERROR = HIPSOLVER_STATUS_MAPPING_ERROR;
  // CHECK-NEXT: hipsolverStatus_t STATUS_EXECUTION_FAILED = HIPSOLVER_STATUS_EXECUTION_FAILED;
  // CHECK-NEXT: hipsolverStatus_t STATUS_INTERNAL_ERROR = HIPSOLVER_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipsolverStatus_t STATUS_NOT_SUPPORTED = HIPSOLVER_STATUS_NOT_SUPPORTED;
  // CHECK-NEXT: hipsolverStatus_t STATUS_ZERO_PIVOT = HIPSOLVER_STATUS_ZERO_PIVOT;
  hipsolverStatus_t status;
  hipsolverStatus_t STATUS_SUCCESS = HIPSOLVER_STATUS_SUCCESS;
  hipsolverStatus_t STATUS_NOT_INITIALIZED = HIPSOLVER_STATUS_NOT_INITIALIZED;
  hipsolverStatus_t STATUS_ALLOC_FAILED = HIPSOLVER_STATUS_ALLOC_FAILED;
  hipsolverStatus_t STATUS_INVALID_VALUE = HIPSOLVER_STATUS_INVALID_VALUE;
  hipsolverStatus_t STATUS_ARCH_MISMATCH = HIPSOLVER_STATUS_ARCH_MISMATCH;
  hipsolverStatus_t STATUS_MAPPING_ERROR = HIPSOLVER_STATUS_MAPPING_ERROR;
  hipsolverStatus_t STATUS_EXECUTION_FAILED = HIPSOLVER_STATUS_EXECUTION_FAILED;
  hipsolverStatus_t STATUS_INTERNAL_ERROR = HIPSOLVER_STATUS_INTERNAL_ERROR;
  hipsolverStatus_t STATUS_NOT_SUPPORTED = HIPSOLVER_STATUS_NOT_SUPPORTED;
  hipsolverStatus_t STATUS_ZERO_PIVOT = HIPSOLVER_STATUS_ZERO_PIVOT;

  // CHECK: hipblasOperation_t blasOperation;
  hipblasOperation_t blasOperation;

  // CHECK: hipblasSideMode_t blasSideMode;
  hipblasSideMode_t blasSideMode;

  // CHECK: hipStream_t stream_t;
  hipStream_t stream_t;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCreate(cusolverDnHandle_t *handle);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCreate(hipsolverHandle_t* handle);
  // CHECK: status = hipsolverDnCreate(&handle);
  status = hipsolverDnCreate(&handle);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDestroy(cusolverDnHandle_t handle);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDestroy(hipsolverHandle_t handle);
  // CHECK: status = hipsolverDnDestroy(handle);
  status = hipsolverDnDestroy(handle);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgetrf(cusolverDnHandle_t handle, int m, int n, float* A, int lda, float* Workspace, int* devIpiv, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgetrf(hipsolverHandle_t handle, int m, int n, float* A, int lda, float* work, int* devIpiv, int* devInfo);
  // CHECK: status = hipsolverDnSgetrf(handle, m, n, &fA, lda, &fWorkspace, &devIpiv, &devInfo);
  status = hipsolverDnSgetrf(handle, m, n, &fA, lda, &fWorkspace, &devIpiv, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgetrf(cusolverDnHandle_t handle, int m, int n, double* A, int lda, double* Workspace, int* devIpiv, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgetrf(hipsolverHandle_t handle, int m, int n,double* A, int lda, double* work, int* devIpiv, int* devInfo);
  // CHECK: status = hipsolverDnDgetrf(handle, m, n, &dA, lda, &dWorkspace, &devIpiv, &devInfo);
  status = hipsolverDnDgetrf(handle, m, n, &dA, lda, &dWorkspace, &devIpiv, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgetrf(cusolverDnHandle_t handle, int m, int n, cuComplex * A, int lda, cuComplex * Workspace, int * devIpiv, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgetrf(hipsolverHandle_t handle, int m, int n, hipFloatComplex* A, int lda, hipFloatComplex* work, int* devIpiv, int* devInfo);
  // CHECK: status = hipsolverDnCgetrf(handle, m, n, &complexA, lda, &complexWorkspace, &devIpiv, &devInfo);
  status = hipsolverDnCgetrf(handle, m, n, &complexA, lda, &complexWorkspace, &devIpiv, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgetrf(cusolverDnHandle_t handle, int m, int n, cuDoubleComplex * A, int lda, cuDoubleComplex * Workspace, int * devIpiv, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgetrf(hipsolverHandle_t handle, int m, int n, hipDoubleComplex* A, int lda, hipDoubleComplex* work, int* devIpiv, int* devInfo);
  // CHECK: status = hipsolverDnZgetrf(handle, m, n, &dComplexA, lda, &dComplexWorkspace, &devIpiv, &devInfo);
  status = hipsolverDnZgetrf(handle, m, n, &dComplexA, lda, &dComplexWorkspace, &devIpiv, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgetrf_bufferSize(cusolverDnHandle_t handle, int m, int n, float* A, int lda, int* Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgetrf_bufferSize(hipsolverHandle_t handle, int m, int n, float* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnSgetrf_bufferSize(handle, m, n, &fA, lda, &Lwork);
  status = hipsolverDnSgetrf_bufferSize(handle, m, n, &fA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgetrf_bufferSize(cusolverDnHandle_t handle, int m, int n, double* A, int lda, int* Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgetrf_bufferSize(hipsolverHandle_t handle, int m, int n, double* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnDgetrf_bufferSize(handle, m, n, &dA, lda, &Lwork);
  status = hipsolverDnDgetrf_bufferSize(handle, m, n, &dA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgetrf_bufferSize(cusolverDnHandle_t handle, int m, int n, cuComplex * A, int lda, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgetrf_bufferSize(hipsolverHandle_t handle, int m, int n, hipFloatComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnCgetrf_bufferSize(handle, m, n, &complexA, lda, &Lwork);
  status = hipsolverDnCgetrf_bufferSize(handle, m, n, &complexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgetrf_bufferSize(cusolverDnHandle_t handle, int m, int n, cuDoubleComplex * A, int lda, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgetrf_bufferSize(hipsolverHandle_t handle, int m, int n, hipDoubleComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnZgetrf_bufferSize(handle, m, n, &dComplexA, lda, &Lwork);
  status = hipsolverDnZgetrf_bufferSize(handle, m, n, &dComplexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgetrs(cusolverDnHandle_t handle, cublasOperation_t trans, int n, int nrhs, const float* A, int lda, const int* devIpiv, float* B, int ldb, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgetrs(hipsolverHandle_t handle, hipsolverOperation_t trans, int n, int nrhs, const float* A, int lda, const int* devIpiv, float* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnSgetrs(handle, blasOperation, n, nrhs , &fA, lda, &devIpiv, &fB, ldb, &devInfo);
  status = hipsolverDnSgetrs(handle, blasOperation, n, nrhs , &fA, lda, &devIpiv, &fB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgetrs(cusolverDnHandle_t handle, cublasOperation_t trans, int n, int nrhs,const double* A, int lda, const int* devIpiv, double* B, int ldb, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgetrs(hipsolverHandle_t handle, hipsolverOperation_t trans, int n, int nrhs, const double* A, int lda, const int* devIpiv, double* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnDgetrs(handle, blasOperation, n, nrhs , &dA, lda, &devIpiv, &dB, ldb, &devInfo);
  status = hipsolverDnDgetrs(handle, blasOperation, n, nrhs , &dA, lda, &devIpiv, &dB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgetrs(cusolverDnHandle_t handle, cublasOperation_t trans, int n, int nrhs, const cuComplex * A, int lda, const int * devIpiv, cuComplex * B, int ldb, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgetrs(hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const hipFloatComplex* A, int lda, const int* devIpiv, hipFloatComplex* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnCgetrs(handle, blasOperation, n, nrhs , &complexA, lda, &devIpiv, &complexB, ldb, &devInfo);
  status = hipsolverDnCgetrs(handle, blasOperation, n, nrhs , &complexA, lda, &devIpiv, &complexB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgetrs(cusolverDnHandle_t handle, cublasOperation_t trans, int n, int nrhs, const cuDoubleComplex *A, int lda, const int * devIpiv, cuDoubleComplex * B, int ldb, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgetrs(hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const hipDoubleComplex* A, int lda, const int* devIpiv, hipDoubleComplex* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnZgetrs(handle, blasOperation, n, nrhs , &dComplexA, lda, &devIpiv, &dComplexB, ldb, &devInfo);
  status = hipsolverDnZgetrs(handle, blasOperation, n, nrhs , &dComplexA, lda, &devIpiv, &dComplexB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSetStream(cusolverDnHandle_t handle, cudaStream_t streamId);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverSetStream(hipsolverHandle_t handle, hipStream_t streamId);
  // CHECK: status = hipsolverSetStream(handle, stream_t);
  status = hipsolverSetStream(handle, stream_t);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnGetStream(cusolverDnHandle_t handle, cudaStream_t *streamId);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverGetStream(hipsolverHandle_t handle, hipStream_t* streamId);
  // CHECK: status = hipsolverGetStream(handle, &stream_t);
  status = hipsolverGetStream(handle, &stream_t);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSpotrf_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, float * A, int lda, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSpotrf_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnSpotrf_bufferSize(handle, fillMode, n, &fA, lda, &Lwork);
  status = hipsolverDnSpotrf_bufferSize(handle, fillMode, n, &fA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDpotrf_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, double * A, int lda, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDpotrf_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnDpotrf_bufferSize(handle, fillMode, n, &dA, lda, &Lwork);
  status = hipsolverDnDpotrf_bufferSize(handle, fillMode, n, &dA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCpotrf_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuComplex * A, int lda, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCpotrf_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnCpotrf_bufferSize(handle, fillMode, n, &complexA, lda, &Lwork);
  status = hipsolverDnCpotrf_bufferSize(handle, fillMode, n, &complexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZpotrf_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZpotrf_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnZpotrf_bufferSize(handle, fillMode, n, &dComplexA, lda, &Lwork);
  status = hipsolverDnZpotrf_bufferSize(handle, fillMode, n, &dComplexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSpotrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, float * A, int lda, float * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSpotrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* A, int lda, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSpotrf(handle, fillMode, n, &fA, lda, &fWorkspace, Lwork, &devInfo);
  status = hipsolverDnSpotrf(handle, fillMode, n, &fA, lda, &fWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDpotrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, double * A, int lda, double * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDpotrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* A, int lda, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDpotrf(handle, fillMode, n, &dA, lda, &dWorkspace, Lwork, &devInfo);
  status = hipsolverDnDpotrf(handle, fillMode, n, &dA, lda, &dWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCpotrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuComplex * A, int lda, cuComplex * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCpotrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCpotrf(handle, fillMode, n, &complexA, lda, &complexWorkspace, Lwork, &devInfo);
  status = hipsolverDnCpotrf(handle, fillMode, n, &complexA, lda, &complexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZpotrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, cuDoubleComplex * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZpotrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZpotrf(handle, fillMode, n, &dComplexA, lda, &dComplexWorkspace, Lwork, &devInfo);
  status = hipsolverDnZpotrf(handle, fillMode, n, &dComplexA, lda, &dComplexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSpotrs(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, const float * A, int lda, float * B, int ldb, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSpotrs(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, const float* A, int lda, float* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnSpotrs(handle, fillMode, n, nrhs, &fA, lda, &fB, ldb, &devInfo);
  status = hipsolverDnSpotrs(handle, fillMode, n, nrhs, &fA, lda, &fB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDpotrs(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, const double * A, int lda, double * B, int ldb, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDpotrs(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, const double* A, int lda, double* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnDpotrs(handle, fillMode, n, nrhs, &dA, lda, &dB, ldb, &devInfo);
  status = hipsolverDnDpotrs(handle, fillMode, n, nrhs, &dA, lda, &dB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCpotrs(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, const cuComplex * A, int lda, cuComplex * B, int ldb, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCpotrs(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, const hipFloatComplex* A, int lda, hipFloatComplex* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnCpotrs(handle, fillMode, n, nrhs, &complexA, lda, &complexB, ldb, &devInfo);
  status = hipsolverDnCpotrs(handle, fillMode, n, nrhs, &complexA, lda, &complexB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZpotrs(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, const cuDoubleComplex *A, int lda, cuDoubleComplex * B, int ldb, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZpotrs(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, const hipDoubleComplex* A, int lda, hipDoubleComplex* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnZpotrs(handle, fillMode, n, nrhs, &dComplexA, lda, &dComplexB, ldb, &devInfo);
  status = hipsolverDnZpotrs(handle, fillMode, n, nrhs, &dComplexA, lda, &dComplexB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgeqrf_bufferSize(cusolverDnHandle_t handle, int m, int n, float * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgeqrf_bufferSize(hipsolverHandle_t handle, int m, int n, float* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnSgeqrf_bufferSize(handle, m, n, &fA, lda, &Lwork);
  status = hipsolverDnSgeqrf_bufferSize(handle, m, n, &fA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgeqrf_bufferSize(cusolverDnHandle_t handle, int m, int n, double * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgeqrf_bufferSize(hipsolverHandle_t handle, int m, int n, double* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnDgeqrf_bufferSize(handle, m, n, &dA, lda, &Lwork);
  status = hipsolverDnDgeqrf_bufferSize(handle, m, n, &dA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgeqrf_bufferSize(cusolverDnHandle_t handle, int m, int n, cuComplex * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgeqrf_bufferSize(hipsolverHandle_t handle, int m, int n, hipFloatComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnCgeqrf_bufferSize(handle, m, n, &complexA, lda, &Lwork);
  status = hipsolverDnCgeqrf_bufferSize(handle, m, n, &complexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgeqrf_bufferSize(cusolverDnHandle_t handle, int m, int n, cuDoubleComplex * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgeqrf_bufferSize(hipsolverHandle_t handle, int m, int n, hipDoubleComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnZgeqrf_bufferSize(handle, m, n, &dComplexA, lda, &Lwork);
  status = hipsolverDnZgeqrf_bufferSize(handle, m, n, &dComplexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgeqrf(cusolverDnHandle_t handle, int m, int n, float * A, int lda, float * TAU, float * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgeqrf(hipsolverHandle_t handle, int m, int n, float* A, int lda, float* tau, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSgeqrf(handle, m, n, &fA, lda, &fTAU, &fWorkspace, Lwork, &devInfo);
  status = hipsolverDnSgeqrf(handle, m, n, &fA, lda, &fTAU, &fWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgeqrf(cusolverDnHandle_t handle, int m, int n, double * A, int lda, double * TAU, double * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgeqrf(hipsolverHandle_t handle, int m, int n, double* A, int lda, double* tau, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDgeqrf(handle, m, n, &dA, lda, &dTAU, &dWorkspace, Lwork, &devInfo);
  status = hipsolverDnDgeqrf(handle, m, n, &dA, lda, &dTAU, &dWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgeqrf(cusolverDnHandle_t handle, int m, int n, cuComplex * A, int lda, cuComplex * TAU, cuComplex * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgeqrf(hipsolverHandle_t handle, int m, int n, hipFloatComplex* A, int lda, hipFloatComplex* tau, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCgeqrf(handle, m, n, &complexA, lda, &complexTAU, &complexWorkspace, Lwork, &devInfo);
  status = hipsolverDnCgeqrf(handle, m, n, &complexA, lda, &complexTAU, &complexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgeqrf(cusolverDnHandle_t handle, int m, int n, cuDoubleComplex * A, int lda, cuDoubleComplex * TAU, cuDoubleComplex * Workspace, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgeqrf(hipsolverHandle_t handle, int m, int n, hipDoubleComplex* A, int lda, hipDoubleComplex* tau, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZgeqrf(handle, m, n, &dComplexA, lda, &dComplexTAU, &dComplexWorkspace, Lwork, &devInfo);
  status = hipsolverDnZgeqrf(handle, m, n, &dComplexA, lda, &dComplexTAU, &dComplexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSsytrf_bufferSize(cusolverDnHandle_t handle, int n, float * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSsytrf_bufferSize(hipsolverHandle_t handle, int n, float* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnSsytrf_bufferSize(handle, n, &fA, lda, &Lwork);
  status = hipsolverDnSsytrf_bufferSize(handle, n, &fA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDsytrf_bufferSize(cusolverDnHandle_t handle, int n, double * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDsytrf_bufferSize(hipsolverHandle_t handle, int n, double* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnDsytrf_bufferSize(handle, n, &dA, lda, &Lwork);
  status = hipsolverDnDsytrf_bufferSize(handle, n, &dA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCsytrf_bufferSize(cusolverDnHandle_t handle, int n, cuComplex * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCsytrf_bufferSize(hipsolverHandle_t handle, int n, hipFloatComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnCsytrf_bufferSize(handle, n, &complexA, lda, &Lwork);
  status = hipsolverDnCsytrf_bufferSize(handle, n, &complexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZsytrf_bufferSize(cusolverDnHandle_t handle, int n, cuDoubleComplex * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZsytrf_bufferSize(hipsolverHandle_t handle, int n, hipDoubleComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnZsytrf_bufferSize(handle, n, &dComplexA, lda, &Lwork);
  status = hipsolverDnZsytrf_bufferSize(handle, n, &dComplexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSsytrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, float * A, int lda, int * ipiv, float * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSsytrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* A, int lda, int* ipiv, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSsytrf(handle, fillMode, n, &fA, lda, &devIpiv, &fWorkspace, Lwork, &devInfo);
  status = hipsolverDnSsytrf(handle, fillMode, n, &fA, lda, &devIpiv, &fWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDsytrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, double * A, int lda, int * ipiv, double * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDsytrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* A, int lda, int* ipiv, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDsytrf(handle, fillMode, n, &dA, lda, &devIpiv, &dWorkspace, Lwork, &devInfo);
  status = hipsolverDnDsytrf(handle, fillMode, n, &dA, lda, &devIpiv, &dWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCsytrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuComplex * A, int lda, int * ipiv, cuComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCsytrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, int* ipiv, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCsytrf(handle, fillMode, n, &complexA, lda, &devIpiv, &complexWorkspace, Lwork, &devInfo);
  status = hipsolverDnCsytrf(handle, fillMode, n, &complexA, lda, &devIpiv, &complexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZsytrf(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, int * ipiv, cuDoubleComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZsytrf(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, int* ipiv, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZsytrf(handle, fillMode, n, &dComplexA, lda, &devIpiv, &dComplexWorkspace, Lwork, &devInfo);
  status = hipsolverDnZsytrf(handle, fillMode, n, &dComplexA, lda, &devIpiv, &dComplexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgebrd_bufferSize(cusolverDnHandle_t handle, int m, int n, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgebrd_bufferSize(hipsolverHandle_t handle, int m, int n, int* lwork);
  // CHECK: status = hipsolverDnSgebrd_bufferSize(handle, m, n, &Lwork);
  status = hipsolverDnSgebrd_bufferSize(handle, m, n, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgebrd_bufferSize(cusolverDnHandle_t handle, int m, int n, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgebrd_bufferSize(hipsolverHandle_t handle, int m, int n, int* lwork);
  // CHECK: status = hipsolverDnDgebrd_bufferSize(handle, m, n, &Lwork);
  status = hipsolverDnDgebrd_bufferSize(handle, m, n, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgebrd_bufferSize(cusolverDnHandle_t handle, int m, int n, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgebrd_bufferSize(hipsolverHandle_t handle, int m, int n, int* lwork);
  // CHECK: status = hipsolverDnCgebrd_bufferSize(handle, m, n, &Lwork);
  status = hipsolverDnCgebrd_bufferSize(handle, m, n, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgebrd_bufferSize(cusolverDnHandle_t handle, int m, int n, int * Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgebrd_bufferSize(hipsolverHandle_t handle, int m, int n, int* lwork);
  // CHECK: status = hipsolverDnZgebrd_bufferSize(handle, m, n, &Lwork);
  status = hipsolverDnZgebrd_bufferSize(handle, m, n, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgebrd(cusolverDnHandle_t handle, int m, int n, float * A, int lda, float * D, float * E, float * TAUQ, float * TAUP, float * Work, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgebrd(hipsolverHandle_t handle, int m, int n, float* A, int lda, float* D, float* E, float* tauq, float* taup, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSgebrd(handle, m, n, &fA, lda, &fD, &fE, &fTAUQ, &fTAUP, &fWorkspace, Lwork, &devInfo);
  status = hipsolverDnSgebrd(handle, m, n, &fA, lda, &fD, &fE, &fTAUQ, &fTAUP, &fWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgebrd(cusolverDnHandle_t handle, int m, int n, double * A, int lda, double * D, double * E, double * TAUQ, double * TAUP, double * Work, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgebrd(hipsolverHandle_t handle, int m, int n, double* A, int lda, double* D, double* E, double* tauq, double* taup, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDgebrd(handle, m, n, &dA, lda, &dD, &dE, &dTAUQ, &dTAUP, &dWorkspace, Lwork, &devInfo);
  status = hipsolverDnDgebrd(handle, m, n, &dA, lda, &dD, &dE, &dTAUQ, &dTAUP, &dWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgebrd(cusolverDnHandle_t handle, int m, int n, cuComplex * A, int lda, float * D, float * E, cuComplex * TAUQ, cuComplex * TAUP, cuComplex * Work, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgebrd(hipsolverHandle_t handle, int m, int n, hipFloatComplex* A, int lda, float* D, float* E, hipFloatComplex* tauq, hipFloatComplex* taup, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCgebrd(handle, m, n, &complexA, lda, &fD, &fE, &complexTAUQ, &complexTAUP, &complexWorkspace, Lwork, &devInfo);
  status = hipsolverDnCgebrd(handle, m, n, &complexA, lda, &fD, &fE, &complexTAUQ, &complexTAUP, &complexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgebrd(cusolverDnHandle_t handle, int m, int n, cuDoubleComplex * A, int lda, double * D, double * E, cuDoubleComplex * TAUQ, cuDoubleComplex * TAUP, cuDoubleComplex * Work, int Lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgebrd(hipsolverHandle_t handle, int m, int n, hipDoubleComplex* A, int lda, double* D, double* E, hipDoubleComplex* tauq, hipDoubleComplex* taup, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZgebrd(handle, m, n, &dComplexA, lda, &dD, &dE, &dComplexTAUQ, &dComplexTAUP, &dComplexWorkspace, Lwork, &devInfo);
  status = hipsolverDnZgebrd(handle, m, n, &dComplexA, lda, &dD, &dE, &dComplexTAUQ, &dComplexTAUP, &dComplexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSsytrd(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, float * A, int lda, float * d, float * e, float * tau, float * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSsytrd(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* A, int lda, float* D, float* E, float* tau, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSsytrd(handle, fillMode, n, &fA, lda, &fD, &fE, &fTAU, &fWorkspace, Lwork, &info);
  status = hipsolverDnSsytrd(handle, fillMode, n, &fA, lda, &fD, &fE, &fTAU, &fWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDsytrd(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, double * A,int lda, double * d, double * e, double * tau, double * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDsytrd(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* A, int lda, double* D, double* E, double* tau, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDsytrd(handle, fillMode, n, &dA, lda, &dD, &dE, &dTAU, &dWorkspace, Lwork, &info);
  status = hipsolverDnDsytrd(handle, fillMode, n, &dA, lda, &dD, &dE, &dTAU, &dWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgesvd_bufferSize(cusolverDnHandle_t handle, int m, int n, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgesvd_bufferSize(hipsolverHandle_t handle, int m, int n, int* lwork);
  // CHECK: status = hipsolverDnSgesvd_bufferSize(handle, m, n, &Lwork);
  status = hipsolverDnSgesvd_bufferSize(handle, m, n, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgesvd_bufferSize(cusolverDnHandle_t handle, int m, int n, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgesvd_bufferSize(hipsolverHandle_t handle, int m, int n, int* lwork);
  // CHECK: status = hipsolverDnDgesvd_bufferSize(handle, m, n, &Lwork);
  status = hipsolverDnDgesvd_bufferSize(handle, m, n, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgesvd_bufferSize(cusolverDnHandle_t handle, int m, int n, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgesvd_bufferSize(hipsolverHandle_t handle, int m, int n, int* lwork);
  // CHECK: status = hipsolverDnCgesvd_bufferSize(handle, m, n, &Lwork);
  status = hipsolverDnCgesvd_bufferSize(handle, m, n, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgesvd_bufferSize(cusolverDnHandle_t handle, int m, int n, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgesvd_bufferSize(hipsolverHandle_t handle, int m, int n, int* lwork);
  // CHECK: status = hipsolverDnZgesvd_bufferSize(handle, m, n, &Lwork);
  status = hipsolverDnZgesvd_bufferSize(handle, m, n, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgesvd(cusolverDnHandle_t handle, signed char jobu, signed char jobvt, int m, int n, float * A, int lda, float * S, float * U, int ldu, float * VT, int ldvt, float * work, int lwork, float * rwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgesvd(hipsolverHandle_t handle, signed char jobu, signed char jobv, int m, int n, float* A, int lda, float* S, float* U, int ldu, float* V, int ldv, float* work, int lwork, float* rwork, int* devInfo);
  // CHECK: status = hipsolverDnSgesvd(handle, jobu, jobvt, m, n, &fA, lda, &fS, &fU, ldu, &fVT, ldvt, &fWorkspace, Lwork, &frWork, &info);
  status = hipsolverDnSgesvd(handle, jobu, jobvt, m, n, &fA, lda, &fS, &fU, ldu, &fVT, ldvt, &fWorkspace, Lwork, &frWork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgesvd(cusolverDnHandle_t handle, signed char jobu, signed char jobvt, int m, int n, double * A, int lda, double * S, double * U, int ldu, double * VT, int ldvt, double * work, int lwork, double * rwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgesvd(hipsolverHandle_t handle, signed char jobu, signed char jobv, int m, int n, double* A, int lda, double* S, double* U, int ldu, double* V, int ldv, double* work, int lwork, double* rwork, int* devInfo);
  // CHECK: status = hipsolverDnDgesvd(handle, jobu, jobvt, m, n, &dA, lda, &dS, &dU, ldu, &dVT, ldvt, &dWorkspace, Lwork, &drWork, &info);
  status = hipsolverDnDgesvd(handle, jobu, jobvt, m, n, &dA, lda, &dS, &dU, ldu, &dVT, ldvt, &dWorkspace, Lwork, &drWork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCgesvd(cusolverDnHandle_t handle, signed char jobu, signed char jobvt, int m, int n, cuComplex * A, int lda, float * S, cuComplex * U, int ldu, cuComplex * VT, int ldvt, cuComplex * work, int lwork, float * rwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCgesvd(hipsolverHandle_t handle, signed char jobu, signed char jobv, int m, int n, hipFloatComplex* A, int lda, float* S, hipFloatComplex* U, int ldu, hipFloatComplex* V, int ldv, hipFloatComplex* work, int lwork, float* rwork, int* devInfo);
  // CHECK: status = hipsolverDnCgesvd(handle, jobu, jobvt, m, n, &complexA, lda, &fS, &complexU, ldu, &complexVT, ldvt, &complexWorkspace, Lwork, &frWork, &info);
  status = hipsolverDnCgesvd(handle, jobu, jobvt, m, n, &complexA, lda, &fS, &complexU, ldu, &complexVT, ldvt, &complexWorkspace, Lwork, &frWork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZgesvd(cusolverDnHandle_t handle, signed char jobu, signed char jobvt, int m, int n, cuDoubleComplex * A, int lda, double * S, cuDoubleComplex * U, int ldu, cuDoubleComplex * VT, int ldvt, cuDoubleComplex * work, int lwork, double * rwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZgesvd(hipsolverHandle_t handle, signed char jobu, signed char jobv, int m, int n, hipDoubleComplex* A, int lda, double* S, hipDoubleComplex* U, int ldu, hipDoubleComplex* V, int ldv, hipDoubleComplex* work, int lwork, double* rwork, int* devInfo);
  // CHECK: status = hipsolverDnZgesvd(handle, jobu, jobvt, m, n, &dComplexA, lda, &dS, &dComplexU, ldu, &dComplexVT, ldvt, &dComplexWorkspace, Lwork, &drWork, &info);
  status = hipsolverDnZgesvd(handle, jobu, jobvt, m, n, &dComplexA, lda, &dS, &dComplexU, ldu, &dComplexVT, ldvt, &dComplexWorkspace, Lwork, &drWork, &info);

#if CUDA_VERSION >= 8000
  // CHECK: hipsolverEigType_t eigType;
  // CHECK-NEXT: hipsolverEigType_t EIG_TYPE_1 = HIPSOLVER_EIG_TYPE_1;
  // CHECK-NEXT: hipsolverEigType_t EIG_TYPE_2 = HIPSOLVER_EIG_TYPE_2;
  // CHECK-NEXT: hipsolverEigType_t EIG_TYPE_3 = HIPSOLVER_EIG_TYPE_3;
  hipsolverEigType_t eigType;
  hipsolverEigType_t EIG_TYPE_1 = HIPSOLVER_EIG_TYPE_1;
  hipsolverEigType_t EIG_TYPE_2 = HIPSOLVER_EIG_TYPE_2;
  hipsolverEigType_t EIG_TYPE_3 = HIPSOLVER_EIG_TYPE_3;

  // CHECK: hipsolverEigMode_t eigMode, jobz;
  // CHECK-NEXT: hipsolverEigMode_t SOLVER_EIG_MODE_NOVECTOR = HIPSOLVER_EIG_MODE_NOVECTOR;
  // CHECK-NEXT: hipsolverEigMode_t SOLVER_EIG_MODE_VECTOR = HIPSOLVER_EIG_MODE_VECTOR;
  hipsolverEigMode_t eigMode, jobz;
  hipsolverEigMode_t SOLVER_EIG_MODE_NOVECTOR = HIPSOLVER_EIG_MODE_NOVECTOR;
  hipsolverEigMode_t SOLVER_EIG_MODE_VECTOR = HIPSOLVER_EIG_MODE_VECTOR;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSorgqr_bufferSize(cusolverDnHandle_t handle, int m, int n, int k, const float * A, int lda, const float * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSorgqr_bufferSize(hipsolverHandle_t handle, int m, int n, int k, const float* A, int lda, const float* tau, int* lwork);
  // CHECK: status = hipsolverDnSorgqr_bufferSize(handle, m, n, k, &fA, lda, &fTAU, &Lwork);
  status = hipsolverDnSorgqr_bufferSize(handle, m, n, k, &fA, lda, &fTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDorgqr_bufferSize(cusolverDnHandle_t handle, int m, int n, int k, const double * A, int lda, const double * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDorgqr_bufferSize(hipsolverHandle_t handle, int m, int n, int k, const double* A, int lda, const double* tau, int* lwork);
  // CHECK: status = hipsolverDnDorgqr_bufferSize(handle, m, n, k, &dA, lda, &dTAU, &Lwork);
  status = hipsolverDnDorgqr_bufferSize(handle, m, n, k, &dA, lda, &dTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCungqr_bufferSize(cusolverDnHandle_t handle, int m, int n, int k, const cuComplex * A, int lda, const cuComplex * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCungqr_bufferSize(hipsolverHandle_t handle, int m, int n, int k, const hipFloatComplex* A, int lda, const hipFloatComplex* tau, int* lwork);
  // CHECK: status = hipsolverDnCungqr_bufferSize(handle, m, n, k, &complexA, lda, &complexTAU, &Lwork);
  status = hipsolverDnCungqr_bufferSize(handle, m, n, k, &complexA, lda, &complexTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZungqr_bufferSize(cusolverDnHandle_t handle, int m, int n, int k, const cuDoubleComplex *A, int lda, const cuDoubleComplex *tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZungqr_bufferSize(hipsolverHandle_t handle, int m, int n, int k, const hipDoubleComplex* A, int lda, const hipDoubleComplex* tau, int* lwork);
  // CHECK: status = hipsolverDnZungqr_bufferSize(handle, m, n, k, &dComplexA, lda, &dComplexTAU, &Lwork);
  status = hipsolverDnZungqr_bufferSize(handle, m, n, k, &dComplexA, lda, &dComplexTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSorgqr(cusolverDnHandle_t handle, int m, int n, int k, float * A, int lda, const float * tau, float * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSorgqr(hipsolverHandle_t handle, int m, int n, int k, float* A, int lda, const float* tau, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSorgqr(handle, m, n, k, &fA, lda, &fTAU, &fWorkspace, Lwork, &info);
  status = hipsolverDnSorgqr(handle, m, n, k, &fA, lda, &fTAU, &fWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDorgqr(cusolverDnHandle_t handle, int m, int n, int k, double * A, int lda, const double * tau, double * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDorgqr(hipsolverHandle_t handle, int m, int n, int k, double* A, int lda, const double* tau, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDorgqr(handle, m, n, k, &dA, lda, &dTAU, &dWorkspace, Lwork, &info);
  status = hipsolverDnDorgqr(handle, m, n, k, &dA, lda, &dTAU, &dWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCungqr(cusolverDnHandle_t handle, int m, int n, int k, cuComplex * A, int lda, const cuComplex * tau, cuComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCungqr(hipsolverHandle_t handle, int m, int n, int k, hipFloatComplex* A, int lda, const hipFloatComplex* tau, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCungqr(handle, m, n, k, &complexA, lda, &complexTAU, &complexWorkspace, Lwork, &info);
  status = hipsolverDnCungqr(handle, m, n, k, &complexA, lda, &complexTAU, &complexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZungqr(cusolverDnHandle_t handle, int m, int n, int k, cuDoubleComplex * A, int lda, const cuDoubleComplex *tau, cuDoubleComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZungqr(hipsolverHandle_t handle, int m, int n, int k, hipDoubleComplex* A, int lda, const hipDoubleComplex* tau, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZungqr(handle, m, n, k, &dComplexA, lda, &dComplexTAU, &dComplexWorkspace, Lwork, &info);
  status = hipsolverDnZungqr(handle, m, n, k, &dComplexA, lda, &dComplexTAU, &dComplexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSormqr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const float * A, int lda, const float * tau, const float * C, int ldc, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSormqr_bufferSize(hipsolverHandle_t  handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const float* A, int lda, const float* tau, const float* C, int ldc, int* lwork);
  // CHECK: status = hipsolverDnSormqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &fA, lda, &fTAU, &fC, ldc, &Lwork);
  status = hipsolverDnSormqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &fA, lda, &fTAU, &fC, ldc, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDormqr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const double * A, int lda, const double * tau, const double * C, int ldc, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDormqr_bufferSize(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const double* A, int lda, const double* tau, const double* C, int ldc, int* lwork);
  // CHECK: status = hipsolverDnDormqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &dA, lda, &dTAU, &dC, ldc, &Lwork);
  status = hipsolverDnDormqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &dA, lda, &dTAU, &dC, ldc, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCunmqr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const cuComplex * A, int lda, const cuComplex * tau, const cuComplex * C, int ldc, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCunmqr_bufferSize(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const hipFloatComplex* A, int lda, const hipFloatComplex* tau, const hipFloatComplex* C, int ldc, int* lwork);
  // CHECK: status = hipsolverDnCunmqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &complexA, lda, &complexTAU, &complexC, ldc, &Lwork);
  status = hipsolverDnCunmqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &complexA, lda, &complexTAU, &complexC, ldc, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZunmqr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const cuDoubleComplex *A, int lda, const cuDoubleComplex *tau, const cuDoubleComplex *C, int ldc, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZunmqr_bufferSize(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const hipDoubleComplex* A, int lda, const hipDoubleComplex* tau, const hipDoubleComplex* C, int ldc, int* lwork);
  // CHECK: status = hipsolverDnZunmqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &dComplexA, lda, &dComplexTAU, &dComplexC, ldc, &Lwork);
  status = hipsolverDnZunmqr_bufferSize(handle, blasSideMode, blasOperation, m, n, k, &dComplexA, lda, &dComplexTAU, &dComplexC, ldc, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSormqr(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const float * A, int lda, const float * tau, float * C, int ldc, float * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSormqr(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const float* A, int lda, const float* tau, float* C, int ldc, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSormqr(handle, blasSideMode, blasOperation, m, n, k, &fA, lda, &fTAU, &fC, ldc, &fWorkspace, Lwork, &devInfo);
  status = hipsolverDnSormqr(handle, blasSideMode, blasOperation, m, n, k, &fA, lda, &fTAU, &fC, ldc, &fWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDormqr(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const double * A, int lda, const double * tau, double * C, int ldc, double * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDormqr(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const double* A, int lda, const double* tau, double* C, int ldc, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDormqr(handle, blasSideMode, blasOperation, m, n, k, &dA, lda, &dTAU, &dC, ldc, &dWorkspace, Lwork, &devInfo);
  status = hipsolverDnDormqr(handle, blasSideMode, blasOperation, m, n, k, &dA, lda, &dTAU, &dC, ldc, &dWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCunmqr(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const cuComplex * A, int lda, const cuComplex * tau, cuComplex * C, int ldc, cuComplex * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCunmqr(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const hipFloatComplex* A, int lda, const hipFloatComplex* tau, hipFloatComplex* C, int ldc, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCunmqr(handle, blasSideMode, blasOperation, m, n, k, &complexA, lda, &complexTAU, &complexC, ldc, &complexWorkspace, Lwork, &devInfo);
  status = hipsolverDnCunmqr(handle, blasSideMode, blasOperation, m, n, k, &complexA, lda, &complexTAU, &complexC, ldc, &complexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZunmqr(cusolverDnHandle_t handle, cublasSideMode_t side, cublasOperation_t trans, int m, int n, int k, const cuDoubleComplex *A, int lda, const cuDoubleComplex *tau, cuDoubleComplex * C, int ldc, cuDoubleComplex * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZunmqr(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans, int m, int n, int k, const hipDoubleComplex* A, int lda, const hipDoubleComplex* tau, hipDoubleComplex* C, int ldc, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZunmqr(handle, blasSideMode, blasOperation, m, n, k, &dComplexA, lda, &dComplexTAU, &dComplexC, ldc, &dComplexWorkspace, Lwork, &devInfo);
  status = hipsolverDnZunmqr(handle, blasSideMode, blasOperation, m, n, k, &dComplexA, lda, &dComplexTAU, &dComplexC, ldc, &dComplexWorkspace, Lwork, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSorgbr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, int m, int n, int k, const float * A, int lda, const float * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSorgbr_bufferSize(hipsolverHandle_t handle, hipblasSideMode_t side, int m, int n, int k, const float* A, int lda, const float* tau, int* lwork);
  // CHECK: status = hipsolverDnSorgbr_bufferSize(handle, blasSideMode, m, n, k, &fA, lda, &fTAU, &Lwork);
  status = hipsolverDnSorgbr_bufferSize(handle, blasSideMode, m, n, k, &fA, lda, &fTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDorgbr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, int m, int n, int k, const double * A, int lda, const double * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDorgbr_bufferSize(hipsolverHandle_t handle, hipblasSideMode_t side, int m, int n, int k, const double* A, int lda, const double* tau, int* lwork);
  // CHECK: status = hipsolverDnDorgbr_bufferSize(handle, blasSideMode, m, n, k, &dA, lda, &dTAU, &Lwork);
  status = hipsolverDnDorgbr_bufferSize(handle, blasSideMode, m, n, k, &dA, lda, &dTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCungbr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, int m, int n, int k, const cuComplex * A, int lda, const cuComplex * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCungbr_bufferSize(hipsolverHandle_t handle, hipblasSideMode_t side, int m, int n, int k, const hipFloatComplex* A, int lda, const hipFloatComplex* tau, int* lwork);
  // CHECK: status = hipsolverDnCungbr_bufferSize(handle, blasSideMode, m, n, k, &complexA, lda, &complexTAU, &Lwork);
  status = hipsolverDnCungbr_bufferSize(handle, blasSideMode, m, n, k, &complexA, lda, &complexTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZungbr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, int m, int n, int k, const cuDoubleComplex *A, int lda, const cuDoubleComplex *tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZungbr_bufferSize(hipsolverHandle_t handle, hipblasSideMode_t side, int m, int n, int k, const hipDoubleComplex* A, int lda, const hipDoubleComplex* tau, int* lwork);
  // CHECK: status = hipsolverDnZungbr_bufferSize(handle, blasSideMode, m, n, k, &dComplexA, lda, &dComplexTAU, &Lwork);
  status = hipsolverDnZungbr_bufferSize(handle, blasSideMode, m, n, k, &dComplexA, lda, &dComplexTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSorgbr(cusolverDnHandle_t handle, cublasSideMode_t side, int m, int n, int k, float * A, int lda, const float * tau, float * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSorgbr(hipsolverHandle_t handle, hipblasSideMode_t side, int m, int n, int k, float* A, int lda, const float* tau, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSorgbr(handle, blasSideMode, m, n, k, &fA, lda, &fTAU, &fWorkspace, Lwork, &info);
  status = hipsolverDnSorgbr(handle, blasSideMode, m, n, k, &fA, lda, &fTAU, &fWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDorgbr(cusolverDnHandle_t handle, cublasSideMode_t side, int m, int n, int k, double * A, int lda, const double * tau, double * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDorgbr(hipsolverHandle_t handle, hipblasSideMode_t side, int m, int n, int k, double* A, int lda, const double* tau, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDorgbr(handle, blasSideMode, m, n, k, &dA, lda, &dTAU, &dWorkspace, Lwork, &info);
  status = hipsolverDnDorgbr(handle, blasSideMode, m, n, k, &dA, lda, &dTAU, &dWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCungbr(cusolverDnHandle_t handle, cublasSideMode_t side, int m, int n, int k, cuComplex * A, int lda, const cuComplex * tau, cuComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCungbr(hipsolverHandle_t handle, hipblasSideMode_t side, int m, int n, int k, hipFloatComplex* A, int lda, const hipFloatComplex* tau, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCungbr(handle, blasSideMode, m, n, k, &complexA, lda, &complexTAU, &complexWorkspace, Lwork, &info);
  status = hipsolverDnCungbr(handle, blasSideMode, m, n, k, &complexA, lda, &complexTAU, &complexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZungbr(cusolverDnHandle_t handle, cublasSideMode_t side, int m, int n, int k, cuDoubleComplex * A, int lda, const cuDoubleComplex *tau, cuDoubleComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZungbr(hipsolverHandle_t handle, hipblasSideMode_t side, int m, int n, int k, hipDoubleComplex* A, int lda, const hipDoubleComplex* tau, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZungbr(handle, blasSideMode, m, n, k, &dComplexA, lda, &dComplexTAU, &dComplexWorkspace, Lwork, &info);
  status = hipsolverDnZungbr(handle, blasSideMode, m, n, k, &dComplexA, lda, &dComplexTAU, &dComplexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSsytrd_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, const float * A, int lda, const float * d, const float * e, const float * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSsytrd_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, const float* A, int lda, const float* D, const float* E, const float* tau, int* lwork);
  // CHECK: status = hipsolverDnSsytrd_bufferSize(handle, fillMode, n, &fA, lda, &fD, &fE, &fTAU, &Lwork);
  status = hipsolverDnSsytrd_bufferSize(handle, fillMode, n, &fA, lda, &fD, &fE, &fTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDsytrd_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, const double * A, int lda, const double * d, const double * e, const double * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDsytrd_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, const double* A, int lda, const double* D, const double* E, const double* tau, int* lwork);
  // CHECK: status = hipsolverDnDsytrd_bufferSize(handle, fillMode, n, &dA, lda, &dD, &dE, &dTAU, &Lwork);
  status = hipsolverDnDsytrd_bufferSize(handle, fillMode, n, &dA, lda, &dD, &dE, &dTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnChetrd_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex * A, int lda, const float * d, const float * e, const cuComplex * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnChetrd_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, const hipFloatComplex* A, int lda, const float* D, const float* E, const hipFloatComplex* tau, int* lwork);
  // CHECK: status = hipsolverDnChetrd_bufferSize(handle, fillMode, n, &complexA, lda, &fD, &fE, &complexTAU, &Lwork);
  status = hipsolverDnChetrd_bufferSize(handle, fillMode, n, &complexA, lda, &fD, &fE, &complexTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZhetrd_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex *A, int lda, const double * d, const double * e, const cuDoubleComplex *tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZhetrd_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* A, int lda, const double* D, const double* E, const hipDoubleComplex* tau, int* lwork);
  // CHECK: status = hipsolverDnZhetrd_bufferSize(handle, fillMode, n, &dComplexA, lda, &dD, &dE, &dComplexTAU, &Lwork);
  status = hipsolverDnZhetrd_bufferSize(handle, fillMode, n, &dComplexA, lda, &dD, &dE, &dComplexTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnChetrd(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuComplex * A, int lda, float * d, float * e, cuComplex * tau, cuComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnChetrd(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, float* D, float* E, hipFloatComplex* tau, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnChetrd(handle, fillMode, n, &complexA, lda, &fD, &fE, &complexTAU, &complexWorkspace, Lwork, &info);
  status = hipsolverDnChetrd(handle, fillMode, n, &complexA, lda, &fD, &fE, &complexTAU, &complexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZhetrd(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, double * d, double * e, cuDoubleComplex * tau, cuDoubleComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZhetrd(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, double* D, double* E, hipDoubleComplex* tau, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZhetrd(handle, fillMode, n, &dComplexA, lda, &dD, &dE, &dComplexTAU, &dComplexWorkspace, Lwork, &info);
  status = hipsolverDnZhetrd(handle, fillMode, n, &dComplexA, lda, &dD, &dE, &dComplexTAU, &dComplexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSorgtr_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, const float * A, int lda, const float * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSorgtr_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, const float* A, int lda, const float* tau, int* lwork);
  // CHECK: status = hipsolverDnSorgtr_bufferSize(handle, fillMode, n, &fA, lda, &fTAU, &Lwork);
  status = hipsolverDnSorgtr_bufferSize(handle, fillMode, n, &fA, lda, &fTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDorgtr_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, const double * A, int lda, const double * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDorgtr_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, const double* A, int lda, const double* tau, int* lwork);
  // CHECK: status = hipsolverDnDorgtr_bufferSize(handle, fillMode, n, &dA, lda, &dTAU, &Lwork);
  status = hipsolverDnDorgtr_bufferSize(handle, fillMode, n, &dA, lda, &dTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCungtr_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex * A, int lda, const cuComplex * tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCungtr_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, const hipFloatComplex* A, int lda, const hipFloatComplex* tau, int* lwork);
  // CHECK: status = hipsolverDnCungtr_bufferSize(handle, fillMode, n, &complexA, lda, &complexTAU, &Lwork);
  status = hipsolverDnCungtr_bufferSize(handle, fillMode, n, &complexA, lda, &complexTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZungtr_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex *A, int lda, const cuDoubleComplex *tau, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZungtr_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* A, int lda, const hipDoubleComplex* tau, int* lwork);
  // CHECK: status = hipsolverDnZungtr_bufferSize(handle, fillMode, n, &dComplexA, lda, &dComplexTAU, &Lwork);
  status = hipsolverDnZungtr_bufferSize(handle, fillMode, n, &dComplexA, lda, &dComplexTAU, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSorgtr(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, float * A, int lda, const float * tau, float * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSorgtr(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* A, int lda, const float* tau, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSorgtr(handle, fillMode, n, &fA, lda, &fTAU, &fWorkspace, Lwork, &info);
  status = hipsolverDnSorgtr(handle, fillMode, n, &fA, lda, &fTAU, &fWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDorgtr(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, double * A, int lda, const double * tau, double * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDorgtr(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* A, int lda, const double* tau, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDorgtr(handle, fillMode, n, &dA, lda, &dTAU, &dWorkspace, Lwork, &info);
  status = hipsolverDnDorgtr(handle, fillMode, n, &dA, lda, &dTAU, &dWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCungtr(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuComplex * A, int lda, const cuComplex * tau, cuComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCungtr(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, const hipFloatComplex* tau, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCungtr(handle, fillMode, n, &complexA, lda, &complexTAU, &complexWorkspace, Lwork, &info);
  status = hipsolverDnCungtr(handle, fillMode, n, &complexA, lda, &complexTAU, &complexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZungtr(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, const cuDoubleComplex *tau, cuDoubleComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZungtr(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, const hipDoubleComplex* tau, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZungtr(handle, fillMode, n, &dComplexA, lda, &dComplexTAU, &dComplexWorkspace, Lwork, &info);
  status = hipsolverDnZungtr(handle, fillMode, n, &dComplexA, lda, &dComplexTAU, &dComplexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSormtr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, int m, int n, const float * A, int lda, const float * tau, const float * C, int ldc, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSormtr_bufferSize(hipsolverHandle_t  handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t trans, int m, int n, const float* A, int lda, const float* tau, const float* C, int ldc, int* lwork);
  // CHECK: status = hipsolverDnSormtr_bufferSize(handle, blasSideMode, fillMode, blasOperation, m, n, &fA, lda, &fTAU, &fC, ldc, &Lwork);
  status = hipsolverDnSormtr_bufferSize(handle, blasSideMode, fillMode, blasOperation, m, n, &fA, lda, &fTAU, &fC, ldc, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDormtr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, int m, int n, const double * A, int lda, const double * tau, const double * C, int ldc, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDormtr_bufferSize(hipsolverHandle_t  handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t trans, int m, int n, const double* A, int lda, const double* tau, const double* C, int ldc, int* lwork);
  // CHECK: status = hipsolverDnDormtr_bufferSize(handle, blasSideMode, fillMode, blasOperation, m, n, &dA, lda, &dTAU, &dC, ldc, &Lwork);
  status = hipsolverDnDormtr_bufferSize(handle, blasSideMode, fillMode, blasOperation, m, n, &dA, lda, &dTAU, &dC, ldc, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCunmtr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, int m, int n, const cuComplex * A, int lda, const cuComplex * tau, const cuComplex * C, int ldc, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCunmtr_bufferSize(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t trans, int m, int n, const hipFloatComplex* A, int lda, const hipFloatComplex* tau, const hipFloatComplex* C, int ldc, int* lwork);
  // CHECK: status = hipsolverDnCunmtr_bufferSize(handle, blasSideMode, fillMode, blasOperation, m, n, &complexA, lda, &complexTAU, &complexC, ldc, &Lwork);
  status = hipsolverDnCunmtr_bufferSize(handle, blasSideMode, fillMode, blasOperation, m, n, &complexA, lda, &complexTAU, &complexC, ldc, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZunmtr_bufferSize(cusolverDnHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, int m, int n, const cuDoubleComplex *A, int lda, const cuDoubleComplex *tau, const cuDoubleComplex *C, int ldc, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZunmtr_bufferSize(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t trans, int m, int n, const hipDoubleComplex* A, int lda, const hipDoubleComplex* tau, const hipDoubleComplex* C, int ldc, int* lwork);
  // CHECK: status = hipsolverDnZunmtr_bufferSize(handle, blasSideMode, fillMode, blasOperation, m, n, &dComplexA, lda, &dComplexTAU, &dComplexC, ldc, &Lwork);
  status = hipsolverDnZunmtr_bufferSize(handle, blasSideMode, fillMode, blasOperation, m, n, &dComplexA, lda, &dComplexTAU, &dComplexC, ldc, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSormtr(cusolverDnHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, int m, int n, float * A, int lda, float * tau, float * C, int ldc, float * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSormtr(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t trans, int m, int n, float* A, int lda, float* tau, float* C, int ldc, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSormtr(handle, blasSideMode, fillMode, blasOperation, m, n, &fA, lda, &fTAU, &fC, ldc, &fWorkspace, Lwork, &info);
  status = hipsolverDnSormtr(handle, blasSideMode, fillMode, blasOperation, m, n, &fA, lda, &fTAU, &fC, ldc, &fWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDormtr(cusolverDnHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, int m, int n, double * A, int lda, double * tau, double * C, int ldc, double * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDormtr(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t trans, int m, int n, double* A, int lda, double* tau, double* C, int ldc, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDormtr(handle, blasSideMode, fillMode, blasOperation, m, n, &dA, lda, &dTAU, &dC, ldc, &dWorkspace, Lwork, &info);
  status = hipsolverDnDormtr(handle, blasSideMode, fillMode, blasOperation, m, n, &dA, lda, &dTAU, &dC, ldc, &dWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCunmtr(cusolverDnHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, int m, int n, cuComplex * A, int lda, cuComplex * tau, cuComplex * C, int ldc, cuComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCunmtr(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t trans, int m, int n, hipFloatComplex* A, int lda, hipFloatComplex* tau, hipFloatComplex* C, int ldc, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCunmtr(handle, blasSideMode, fillMode, blasOperation, m, n, &complexA, lda, &complexTAU, &complexC, ldc, &complexWorkspace, Lwork, &info);
  status = hipsolverDnCunmtr(handle, blasSideMode, fillMode, blasOperation, m, n, &complexA, lda, &complexTAU, &complexC, ldc, &complexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZunmtr(cusolverDnHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, int m, int n, cuDoubleComplex * A, int lda, cuDoubleComplex * tau, cuDoubleComplex * C, int ldc, cuDoubleComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZunmtr(hipsolverHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t trans, int m, int n, hipDoubleComplex* A, int lda, hipDoubleComplex* tau, hipDoubleComplex* C, int ldc, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZunmtr(handle, blasSideMode, fillMode, blasOperation, m, n, &dComplexA, lda, &dComplexTAU, &dComplexC, ldc, &dComplexWorkspace, Lwork, &info);
  status = hipsolverDnZunmtr(handle, blasSideMode, fillMode, blasOperation, m, n, &dComplexA, lda, &dComplexTAU, &dComplexC, ldc, &dComplexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSsyevd_bufferSize(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cublasFillMode_t uplo, int n, const float * A, int lda, const float * W, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSsyevd_bufferSize(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, const float* A, int lda, const float* W, int* lwork);
  // CHECK: status = hipsolverDnSsyevd_bufferSize(handle, eigMode, fillMode, n, &fA, lda, &fW, &Lwork);
  status = hipsolverDnSsyevd_bufferSize(handle, eigMode, fillMode, n, &fA, lda, &fW, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDsyevd_bufferSize(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cublasFillMode_t uplo, int n, const double * A, int lda, const double * W, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDsyevd_bufferSize(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, const double* A, int lda, const double* W, int* lwork);
  // CHECK: status = hipsolverDnDsyevd_bufferSize(handle, eigMode, fillMode, n, &dA, lda, &dW, &Lwork);
  status = hipsolverDnDsyevd_bufferSize(handle, eigMode, fillMode, n, &dA, lda, &dW, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCheevd_bufferSize(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cublasFillMode_t uplo, int n, const cuComplex * A, int lda, const float * W, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCheevd_bufferSize(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, const hipFloatComplex* A, int lda, const float* W, int* lwork);
  // CHECK: status = hipsolverDnCheevd_bufferSize(handle, eigMode, fillMode, n, &complexA, lda, &fW, &Lwork);
  status = hipsolverDnCheevd_bufferSize(handle, eigMode, fillMode, n, &complexA, lda, &fW, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZheevd_bufferSize(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cublasFillMode_t uplo, int n, const cuDoubleComplex *A, int lda, const double * W, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZheevd_bufferSize(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, const hipDoubleComplex* A, int lda, const double* W, int* lwork);
  // CHECK: status = hipsolverDnZheevd_bufferSize(handle, eigMode, fillMode, n, &dComplexA, lda, &dW, &Lwork);
  status = hipsolverDnZheevd_bufferSize(handle, eigMode, fillMode, n, &dComplexA, lda, &dW, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSsyevd(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cublasFillMode_t uplo, int n, float * A, int lda, float * W, float * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSsyevd(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, float* A, int lda, float* W, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSsyevd(handle, eigMode, fillMode, n, &fA, lda, &fW, &fWorkspace, Lwork, &info);
  status = hipsolverDnSsyevd(handle, eigMode, fillMode, n, &fA, lda, &fW, &fWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDsyevd(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cublasFillMode_t uplo, int n, double * A, int lda, double * W, double * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDsyevd(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, double* A, int lda, double* W, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDsyevd(handle, eigMode, fillMode, n, &dA, lda, &dW, &dWorkspace, Lwork, &info);
  status = hipsolverDnDsyevd(handle, eigMode, fillMode, n, &dA, lda, &dW, &dWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCheevd(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cublasFillMode_t uplo, int n, cuComplex * A, int lda, float * W, cuComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCheevd(hipsolverHandle_t  handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, float* W, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCheevd(handle, eigMode, fillMode, n, &complexA, lda, &fW, &complexWorkspace, Lwork, &info);
  status = hipsolverDnCheevd(handle, eigMode, fillMode, n, &complexA, lda, &fW, &complexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZheevd(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, double * W, cuDoubleComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZheevd(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, double* W, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZheevd(handle, eigMode, fillMode, n, &dComplexA, lda, &dW, &dComplexWorkspace, Lwork, &info);
  status = hipsolverDnZheevd(handle, eigMode, fillMode, n, &dComplexA, lda, &dW, &dComplexWorkspace, Lwork, &info);
#endif

#if CUDA_VERSION >= 9000
  // CHECK: hipsolverSyevjInfo_t syevj_info;
  hipsolverSyevjInfo_t syevj_info;

  // CHECK: hipsolverGesvdjInfo_t gesvdj_info;
  hipsolverGesvdjInfo_t gesvdj_info;
#endif

#if CUDA_VERSION >= 9010
  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSpotrfBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, float * Aarray[], int lda, int * infoArray, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSpotrfBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* A[], int lda, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnSpotrfBatched(handle, fillMode, n, fAarray, lda, &infoArray, batchSize);
  status = hipsolverDnSpotrfBatched(handle, fillMode, n, fAarray, lda, &infoArray, batchSize);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDpotrfBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, double * Aarray[], int lda, int * infoArray, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDpotrfBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* A[], int lda, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnDpotrfBatched(handle, fillMode, n, dAarray, lda, &infoArray, batchSize);
  status = hipsolverDnDpotrfBatched(handle, fillMode, n, dAarray, lda, &infoArray, batchSize);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCpotrfBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuComplex * Aarray[], int lda, int * infoArray, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCpotrfBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipFloatComplex* A[], int lda, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnCpotrfBatched(handle, fillMode, n, complexAarray, lda, &infoArray, batchSize);
  status = hipsolverDnCpotrfBatched(handle, fillMode, n, complexAarray, lda, &infoArray, batchSize);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZpotrfBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuDoubleComplex * Aarray[], int lda, int * infoArray, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZpotrfBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipDoubleComplex* A[], int lda, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnZpotrfBatched(handle, fillMode, n, dcomplexAarray, lda, &infoArray, batchSize);
  status = hipsolverDnZpotrfBatched(handle, fillMode, n, dcomplexAarray, lda, &infoArray, batchSize);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSpotrsBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, float * A[], int lda, float * B[], int ldb, int * d_info, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSpotrsBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, float* A[], int lda, float* B[], int ldb, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnSpotrsBatched(handle, fillMode, n, nrhs, fAarray, lda, fBarray, ldb, &infoArray, batchSize);
  status = hipsolverDnSpotrsBatched(handle, fillMode, n, nrhs, fAarray, lda, fBarray, ldb, &infoArray, batchSize);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDpotrsBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, double * A[], int lda, double * B[], int ldb, int * d_info, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDpotrsBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, double* A[], int lda, double* B[], int ldb, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnDpotrsBatched(handle, fillMode, n, nrhs, dAarray, lda, dBarray, ldb, &infoArray, batchSize);
  status = hipsolverDnDpotrsBatched(handle, fillMode, n, nrhs, dAarray, lda, dBarray, ldb, &infoArray, batchSize);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCpotrsBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, cuComplex * A[], int lda, cuComplex * B[], int ldb, int * d_info, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCpotrsBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, hipFloatComplex* A[], int lda, hipFloatComplex* B[], int ldb, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnCpotrsBatched(handle, fillMode, n, nrhs, complexAarray, lda, complexBarray, ldb, &infoArray, batchSize);
  status = hipsolverDnCpotrsBatched(handle, fillMode, n, nrhs, complexAarray, lda, complexBarray, ldb, &infoArray, batchSize);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZpotrsBatched(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, int nrhs, cuDoubleComplex * A[], int lda, cuDoubleComplex * B[], int ldb, int * d_info, int batchSize);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZpotrsBatched(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs, hipDoubleComplex* A[], int lda, hipDoubleComplex* B[], int ldb, int* devInfo, int batch_count);
  // CHECK: status = hipsolverDnZpotrsBatched(handle, fillMode, n, nrhs, dcomplexAarray, lda, dcomplexBarray, ldb, &infoArray, batchSize);
  status = hipsolverDnZpotrsBatched(handle, fillMode, n, nrhs, dcomplexAarray, lda, dcomplexBarray, ldb, &infoArray, batchSize);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipsolverEigRange_t eigRange;
  // CHECK-NEXT: hipsolverEigRange_t EIG_RANGE_ALL = HIPSOLVER_EIG_RANGE_ALL;
  // CHECK-NEXT: hipsolverEigRange_t EIG_RANGE_I = HIPSOLVER_EIG_RANGE_I;
  // CHECK-NEXT: hipsolverEigRange_t EIG_RANGE_V = HIPSOLVER_EIG_RANGE_V;
  hipsolverEigRange_t eigRange;
  hipsolverEigRange_t EIG_RANGE_ALL = HIPSOLVER_EIG_RANGE_ALL;
  hipsolverEigRange_t EIG_RANGE_I = HIPSOLVER_EIG_RANGE_I;
  hipsolverEigRange_t EIG_RANGE_V = HIPSOLVER_EIG_RANGE_V;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSpotri_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, float * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSpotri_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnSpotri_bufferSize(handle, fillMode, n, &fA, lda, &Lwork);
  status = hipsolverDnSpotri_bufferSize(handle, fillMode, n, &fA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDpotri_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, double * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDpotri_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnDpotri_bufferSize(handle, fillMode, n, &dA, lda, &Lwork);
  status = hipsolverDnDpotri_bufferSize(handle, fillMode, n, &dA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCpotri_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuComplex * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCpotri_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnCpotri_bufferSize(handle, fillMode, n, &complexA, lda, &Lwork);
  status = hipsolverDnCpotri_bufferSize(handle, fillMode, n, &complexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZpotri_bufferSize(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZpotri_bufferSize(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnZpotri_bufferSize(handle, fillMode, n, &dComplexA, lda, &Lwork);
  status = hipsolverDnZpotri_bufferSize(handle, fillMode, n, &dComplexA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSpotri(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, float * A, int lda, float * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSpotri(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* A, int lda, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSpotri(handle, fillMode, n, &fA, lda, &fWorkspace, Lwork, &infoArray);
  status = hipsolverDnSpotri(handle, fillMode, n, &fA, lda, &fWorkspace, Lwork, &infoArray);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDpotri(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, double * A, int lda, double * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDpotri(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* A, int lda, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDpotri(handle, fillMode, n, &dA, lda, &dWorkspace, Lwork, &infoArray);
  status = hipsolverDnDpotri(handle, fillMode, n, &dA, lda, &dWorkspace, Lwork, &infoArray);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCpotri(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuComplex * A, int lda, cuComplex * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCpotri(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCpotri(handle, fillMode, n, &complexA, lda, &complexWorkspace, Lwork, &infoArray);
  status = hipsolverDnCpotri(handle, fillMode, n, &complexA, lda, &complexWorkspace, Lwork, &infoArray);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZpotri(cusolverDnHandle_t handle, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, cuDoubleComplex * work, int lwork, int * devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZpotri(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZpotri(handle, fillMode, n, &dComplexA, lda, &dComplexWorkspace, Lwork, &infoArray);
  status = hipsolverDnZpotri(handle, fillMode, n, &dComplexA, lda, &dComplexWorkspace, Lwork, &infoArray);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSsyevdx_bufferSize(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, const float * A, int lda, float vl, float vu, int il, int iu, int * meig, const float * W, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSsyevdx_bufferSize(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, const float* A, int lda, float vl, float vu, int il, int iu, int* nev, const float* W, int* lwork);
  // CHECK: status = hipsolverDnSsyevdx_bufferSize(handle, jobz, eigRange, fillMode, n, &fA, lda, fvl, fvu, il, iu, &imeig, &fW, &Lwork);
  status = hipsolverDnSsyevdx_bufferSize(handle, jobz, eigRange, fillMode, n, &fA, lda, fvl, fvu, il, iu, &imeig, &fW, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDsyevdx_bufferSize(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, const double * A, int lda, double vl, double vu, int il, int iu, int * meig, const double * W, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDsyevdx_bufferSize(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, const double* A, int lda, double vl, double vu, int il, int iu, int* nev, const double* W, int* lwork);
  // CHECK: status = hipsolverDnDsyevdx_bufferSize(handle, jobz, eigRange, fillMode, n, &dA, lda, dvl, dvu, il, iu, &imeig, &dW, &Lwork);
  status = hipsolverDnDsyevdx_bufferSize(handle, jobz, eigRange, fillMode, n, &dA, lda, dvl, dvu, il, iu, &imeig, &dW, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCheevdx_bufferSize(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, const cuComplex * A, int lda, float vl, float vu, int il, int iu, int * meig, const float * W, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCheevdx_bufferSize(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, const hipFloatComplex* A, int lda, float vl, float vu, int il, int iu, int* nev, const float* W, int* lwork);
  // CHECK: status = hipsolverDnCheevdx_bufferSize(handle, jobz, eigRange, fillMode, n, &complexA, lda, fvl, fvu, il, iu, &imeig, &fW, &Lwork);
  status = hipsolverDnCheevdx_bufferSize(handle, jobz, eigRange, fillMode, n, &complexA, lda, fvl, fvu, il, iu, &imeig, &fW, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZheevdx_bufferSize(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, const cuDoubleComplex *A, int lda, double vl, double vu, int il, int iu, int * meig, const double * W, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZheevdx_bufferSize(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, const hipDoubleComplex* A, int lda, double vl, double vu, int il, int iu, int* nev, const double* W, int* lwork);
  // CHECK: status = hipsolverDnZheevdx_bufferSize(handle, jobz, eigRange, fillMode, n, &dComplexA, lda, dvl, dvu, il, iu, &imeig, &dW, &Lwork);
  status = hipsolverDnZheevdx_bufferSize(handle, jobz, eigRange, fillMode, n, &dComplexA, lda, dvl, dvu, il, iu, &imeig, &dW, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSsyevdx(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, float * A, int lda, float vl, float vu, int il, int iu, int * meig, float * W, float * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSsyevdx(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, float* A, int lda, float vl, float vu, int il, int iu, int* nev, float* W, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSsyevdx(handle, jobz, eigRange, fillMode, n, &fA, lda, fvl, fvu, il, iu, &imeig, &fW, &fWorkspace, Lwork, &info);
  status = hipsolverDnSsyevdx(handle, jobz, eigRange, fillMode, n, &fA, lda, fvl, fvu, il, iu, &imeig, &fW, &fWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDsyevdx(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, double * A, int lda, double vl, double vu, int il, int iu, int * meig, double * W, double * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDsyevdx(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, double* A, int lda, double vl, double vu, int il, int iu, int* nev, double* W, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDsyevdx(handle, jobz, eigRange, fillMode, n, &dA, lda, dvl, dvu, il, iu, &imeig, &dW, &dWorkspace, Lwork, &info);
  status = hipsolverDnDsyevdx(handle, jobz, eigRange, fillMode, n, &dA, lda, dvl, dvu, il, iu, &imeig, &dW, &dWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCheevdx(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, cuComplex * A, int lda, float vl, float vu, int il, int iu, int * meig, float * W, cuComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCheevdx(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, float vl, float vu, int il, int iu, int* nev, float* W, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnCheevdx(handle, jobz, eigRange, fillMode, n, &complexA, lda, fvl, fvu, il, iu, &imeig, &fW, &complexWorkspace, Lwork, &info);
  status = hipsolverDnCheevdx(handle, jobz, eigRange, fillMode, n, &complexA, lda, fvl, fvu, il, iu, &imeig, &fW, &complexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZheevdx(cusolverDnHandle_t handle, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, double vl, double vu, int il, int iu, int * meig, double * W, cuDoubleComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZheevdx(hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, double vl, double vu, int il, int iu, int* nev, double* W, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZheevdx(handle, jobz, eigRange, fillMode, n, &dComplexA, lda, dvl, dvu, il, iu, &imeig, &dW, &dComplexWorkspace, Lwork, &info);
  status = hipsolverDnZheevdx(handle, jobz, eigRange, fillMode, n, &dComplexA, lda, dvl, dvu, il, iu, &imeig, &dW, &dComplexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSsygvdx_bufferSize(cusolverDnHandle_t handle, cusolverEigType_t itype, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, const float * A, int lda, const float * B, int ldb, float vl, float vu, int il, int iu, int * meig, const float * W, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSsygvdx_bufferSize(hipsolverHandle_t handle, hipsolverEigType_t itype, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, const float* A, int lda, const float* B, int ldb, float vl, float vu, int il, int iu, int* nev, const float* W, int* lwork);
  // CHECK: status = hipsolverDnSsygvdx_bufferSize(handle, eigType, jobz, eigRange, fillMode, n, &fA, lda, &fB, ldb, fvl, fvu, il, iu, &imeig, &fW, &Lwork);
  status = hipsolverDnSsygvdx_bufferSize(handle, eigType, jobz, eigRange, fillMode, n, &fA, lda, &fB, ldb, fvl, fvu, il, iu, &imeig, &fW, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDsygvdx_bufferSize(cusolverDnHandle_t handle, cusolverEigType_t itype, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, const double * A, int lda, const double * B, int ldb, double vl, double vu, int il, int iu, int * meig, const double * W, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDsygvdx_bufferSize(hipsolverHandle_t handle, hipsolverEigType_t itype, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, const double* A, int lda, const double* B, int ldb, double vl, double vu, int il, int iu, int* nev, const double* W, int* lwork);
  // CHECK: status = hipsolverDnDsygvdx_bufferSize(handle, eigType, jobz, eigRange, fillMode, n, &dA, lda, &dB, ldb, dvl, dvu, il, iu, &imeig, &dW, &Lwork);
  status = hipsolverDnDsygvdx_bufferSize(handle, eigType, jobz, eigRange, fillMode, n, &dA, lda, &dB, ldb, dvl, dvu, il, iu, &imeig, &dW, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnChegvdx_bufferSize(cusolverDnHandle_t handle, cusolverEigType_t itype, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, const cuComplex * A, int lda, const cuComplex * B, int ldb, float vl, float vu, int il, int iu, int * meig, const float * W, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnChegvdx_bufferSize(hipsolverHandle_t handle, hipsolverEigType_t itype, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, const hipFloatComplex* A, int lda, const hipFloatComplex* B, int ldb, float vl, float vu, int il, int iu, int* nev, const float* W, int* lwork);
  // CHECK: status = hipsolverDnChegvdx_bufferSize(handle, eigType, jobz, eigRange, fillMode, n, &complexA, lda, &complexB, ldb, fvl, fvu, il, iu, &imeig, &fW, &Lwork);
  status = hipsolverDnChegvdx_bufferSize(handle, eigType, jobz, eigRange, fillMode, n, &complexA, lda, &complexB, ldb, fvl, fvu, il, iu, &imeig, &fW, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZhegvdx_bufferSize(cusolverDnHandle_t handle, cusolverEigType_t itype, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, const cuDoubleComplex *A, int lda, const cuDoubleComplex *B, int ldb, double vl, double vu, int il, int iu, int * meig, const double * W, int * lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZhegvdx_bufferSize(hipsolverHandle_t handle, hipsolverEigType_t itype, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, const hipDoubleComplex* A, int lda, const hipDoubleComplex* B, int ldb, double vl, double vu, int il, int iu, int* nev, const double* W, int* lwork);
  // CHECK: status = hipsolverDnZhegvdx_bufferSize(handle, eigType, jobz, eigRange, fillMode, n, &dComplexA, lda, &dComplexB, ldb, dvl, dvu, il, iu, &imeig, &dW, &Lwork);
  status = hipsolverDnZhegvdx_bufferSize(handle, eigType, jobz, eigRange, fillMode, n, &dComplexA, lda, &dComplexB, ldb, dvl, dvu, il, iu, &imeig, &dW, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSsygvdx(cusolverDnHandle_t handle, cusolverEigType_t itype, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, float * A, int lda, float * B, int ldb, float vl, float vu, int il, int iu, int * meig, float * W, float * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSsygvdx(hipsolverHandle_t handle, hipsolverEigType_t itype, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, float* A, int lda, float* B, int ldb, float vl, float vu, int il, int iu, int* nev, float* W, float* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnSsygvdx(handle, eigType, jobz, eigRange, fillMode, n, &fA, lda, &fB, ldb, fvl, fvu, il, iu, &imeig, &fW, &fWorkspace, Lwork, &info);
  status = hipsolverDnSsygvdx(handle, eigType, jobz, eigRange, fillMode, n, &fA, lda, &fB, ldb, fvl, fvu, il, iu, &imeig, &fW, &fWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDsygvdx(cusolverDnHandle_t handle, cusolverEigType_t itype, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, double * A, int lda, double * B, int ldb, double vl, double vu, int il, int iu, int * meig, double * W, double * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDsygvdx(hipsolverHandle_t handle, hipsolverEigType_t itype, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, double* A, int lda, double* B, int ldb, double vl, double vu, int il, int iu, int* nev, double* W, double* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnDsygvdx(handle, eigType, jobz, eigRange, fillMode, n, &dA, lda, &dB, ldb, dvl, dvu, il, iu, &imeig, &dW, &dWorkspace, Lwork, &info);
  status = hipsolverDnDsygvdx(handle, eigType, jobz, eigRange, fillMode, n, &dA, lda, &dB, ldb, dvl, dvu, il, iu, &imeig, &dW, &dWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnChegvdx(cusolverDnHandle_t handle, cusolverEigType_t itype, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, cuComplex * A, int lda, cuComplex * B, int ldb, float vl, float vu, int il, int iu, int * meig, float * W, cuComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnChegvdx(hipsolverHandle_t handle, hipsolverEigType_t itype, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, hipFloatComplex* A, int lda, hipFloatComplex* B, int ldb, float vl, float vu, int il, int iu, int* nev, float* W, hipFloatComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnChegvdx(handle, eigType, jobz, eigRange, fillMode, n, &complexA, lda, &complexB, ldb, fvl, fvu, il, iu, &imeig, &fW, &complexWorkspace, Lwork, &info);
  status = hipsolverDnChegvdx(handle, eigType, jobz, eigRange, fillMode, n, &complexA, lda, &complexB, ldb, fvl, fvu, il, iu, &imeig, &fW, &complexWorkspace, Lwork, &info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZhegvdx(cusolverDnHandle_t handle, cusolverEigType_t itype, cusolverEigMode_t jobz, cusolverEigRange_t range, cublasFillMode_t uplo, int n, cuDoubleComplex * A, int lda, cuDoubleComplex * B, int ldb, double vl, double vu, int il, int iu, int * meig, double * W, cuDoubleComplex * work, int lwork, int * info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZhegvdx(hipsolverHandle_t handle, hipsolverEigType_t itype, hipsolverEigMode_t jobz, hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, hipDoubleComplex* A, int lda, hipDoubleComplex* B, int ldb, double vl, double vu, int il, int iu, int* nev, double* W, hipDoubleComplex* work, int lwork, int* devInfo);
  // CHECK: status = hipsolverDnZhegvdx(handle, eigType, jobz, eigRange, fillMode, n, &dComplexA, lda, &dComplexB, ldb, dvl, dvu, il, iu, &imeig, &dW, &dComplexWorkspace, Lwork, &info);
  status = hipsolverDnZhegvdx(handle, eigType, jobz, eigRange, fillMode, n, &dComplexA, lda, &dComplexB, ldb, dvl, dvu, il, iu, &imeig, &dW, &dComplexWorkspace, Lwork, &info);
#endif

#if CUDA_VERSION >= 10020
  // CHECK: int solver_int = 0;
  // CHECK: int lm = 0;
  // CHECK: int ln = 0;
  // CHECK: int lnrhs = 0;
  // CHECK: int ldda = 0;
  // CHECK: int lddb = 0;
  // CHECK: int lddx = 0;
  // CHECK: int dipiv = 0;
  // CHECK: int iter = 0;
  // CHECK: int d_info = 0;
  int solver_int = 0;
  int lm = 0;
  int ln = 0;
  int lnrhs = 0;
  int ldda = 0;
  int lddb = 0;
  int lddx = 0;
  int dipiv = 0;
  int iter = 0;
  int d_info = 0;

  // CHECK: hipsolverStatus_t STATUS_IRS_PARAMS_INVALID = HIPSOLVER_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipsolverStatus_t STATUS_IRS_INTERNAL_ERROR = HIPSOLVER_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipsolverStatus_t STATUS_IRS_NOT_SUPPORTED = HIPSOLVER_STATUS_NOT_SUPPORTED;
  hipsolverStatus_t STATUS_IRS_PARAMS_INVALID = HIPSOLVER_STATUS_INVALID_VALUE;
  hipsolverStatus_t STATUS_IRS_INTERNAL_ERROR = HIPSOLVER_STATUS_INTERNAL_ERROR;
  hipsolverStatus_t STATUS_IRS_NOT_SUPPORTED = HIPSOLVER_STATUS_NOT_SUPPORTED;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZZgesv(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, cuDoubleComplex * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, cuDoubleComplex * dB, cusolver_int_t lddb, cuDoubleComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZZgesv(hipsolverHandle_t handle, int n, int nrhs, hipDoubleComplex* A, int lda, int* devIpiv, hipDoubleComplex* B, int ldb, hipDoubleComplex* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnZZgesv(handle, ln, lnrhs, &dComplexA, ldda, &dipiv, &dComplexB, lddb, &dComplexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnZZgesv(handle, ln, lnrhs, &dComplexA, ldda, &dipiv, &dComplexB, lddb, &dComplexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCCgesv(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, cuComplex * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, cuComplex * dB, cusolver_int_t lddb, cuComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCCgesv(hipsolverHandle_t handle, int n, int nrhs, hipFloatComplex* A, int lda, int* devIpiv, hipFloatComplex* B, int ldb, hipFloatComplex* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnCCgesv(handle, ln, lnrhs, &complexA, ldda, &dipiv, &complexB, lddb, &complexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnCCgesv(handle, ln, lnrhs, &complexA, ldda, &dipiv, &complexB, lddb, &complexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDDgesv(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, double * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, double * dB, cusolver_int_t lddb, double * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDDgesv(hipsolverHandle_t handle, int n, int nrhs, double* A, int lda, int* devIpiv, double* B, int ldb, double* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnDDgesv(handle, ln, lnrhs, &dA, ldda, &dipiv, &dB, lddb, &dX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnDDgesv(handle, ln, lnrhs, &dA, ldda, &dipiv, &dB, lddb, &dX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSSgesv(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, float * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, float * dB, cusolver_int_t lddb, float * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSSgesv(hipsolverHandle_t handle, int n, int nrhs, float* A, int lda, int* devIpiv, float* B, int ldb, float* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnSSgesv(handle, ln, lnrhs, &fA, ldda, &dipiv, &fB, lddb, &fX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnSSgesv(handle, ln, lnrhs, &fA, ldda, &dipiv, &fB, lddb, &fX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZZgesv_bufferSize(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, cuDoubleComplex * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, cuDoubleComplex * dB, cusolver_int_t lddb, cuDoubleComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZZgesv_bufferSize(hipsolverHandle_t handle, int n, int nrhs, hipDoubleComplex* A, int lda, int* devIpiv, hipDoubleComplex* B, int ldb, hipDoubleComplex* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnZZgesv_bufferSize(handle, ln, lnrhs, &dComplexA, ldda, &dipiv, &dComplexB, lddb, &dComplexX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnZZgesv_bufferSize(handle, ln, lnrhs, &dComplexA, ldda, &dipiv, &dComplexB, lddb, &dComplexX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCCgesv_bufferSize(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, cuComplex * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, cuComplex * dB, cusolver_int_t lddb, cuComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCCgesv_bufferSize(hipsolverHandle_t handle, int n, int nrhs, hipFloatComplex* A, int lda, int* devIpiv, hipFloatComplex* B, int ldb, hipFloatComplex* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnCCgesv_bufferSize(handle, ln, lnrhs, &complexA, ldda, &dipiv, &complexB, lddb, &complexX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnCCgesv_bufferSize(handle, ln, lnrhs, &complexA, ldda, &dipiv, &complexB, lddb, &complexX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDDgesv_bufferSize(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, double * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, double * dB, cusolver_int_t lddb, double * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDDgesv_bufferSize(hipsolverHandle_t handle, int n, int nrhs, double* A, int lda, int* devIpiv, double* B, int ldb, double* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnDDgesv_bufferSize(handle, ln, lnrhs, &dA, ldda, &dipiv, &dB, lddb, &dX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnDDgesv_bufferSize(handle, ln, lnrhs, &dA, ldda, &dipiv, &dB, lddb, &dX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSSgesv_bufferSize(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, float * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, float * dB, cusolver_int_t lddb, float * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSSgesv_bufferSize(hipsolverHandle_t handle, int n, int nrhs, float* A, int lda, int* devIpiv, float* B, int ldb, float* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnSSgesv_bufferSize(handle, ln, lnrhs, &fA, ldda, &dipiv, &fB, lddb, &fX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnSSgesv_bufferSize(handle, ln, lnrhs, &fA, ldda, &dipiv, &fB, lddb, &fX, lddx, &Workspace, &lwork_bytes);
#endif

#if CUDA_VERSION >= 11000
  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZZgels(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, cuDoubleComplex * dA, cusolver_int_t ldda, cuDoubleComplex * dB, cusolver_int_t lddb, cuDoubleComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZZgels(hipsolverHandle_t handle, int m, int n, int nrhs, hipDoubleComplex* A, int lda, hipDoubleComplex* B, int ldb, hipDoubleComplex* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnZZgels(handle, lm, ln, lnrhs, &dComplexA, ldda, &dComplexB, lddb, &dComplexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnZZgels(handle, lm, ln, lnrhs, &dComplexA, ldda, &dComplexB, lddb, &dComplexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCCgels(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, cuComplex * dA, cusolver_int_t ldda, cuComplex * dB, cusolver_int_t lddb, cuComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCCgels(hipsolverHandle_t handle, int m, int n, int nrhs, hipFloatComplex* A, int lda, hipFloatComplex* B, int ldb, hipFloatComplex* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnCCgels(handle, lm, ln, lnrhs, &complexA, ldda, &complexB, lddb, &complexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnCCgels(handle, lm, ln, lnrhs, &complexA, ldda, &complexB, lddb, &complexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDDgels(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, double * dA, cusolver_int_t ldda, double * dB, cusolver_int_t lddb, double * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDDgels(hipsolverHandle_t handle, int m, int n, int nrhs, double* A, int lda, double* B, int ldb, double* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnDDgels(handle, lm, ln, lnrhs, &dA, ldda, &dB, lddb, &dX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnDDgels(handle, lm, ln, lnrhs, &dA, ldda, &dB, lddb, &dX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSSgels(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, float * dA, cusolver_int_t ldda, float * dB, cusolver_int_t lddb, float * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSSgels(hipsolverHandle_t handle, int m, int n, int nrhs, float* A, int lda, float* B, int ldb, float* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnSSgels(handle, lm, ln, lnrhs, &fA, ldda, &fB, lddb, &fX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnSSgels(handle, lm, ln, lnrhs, &fA, ldda, &fB, lddb, &fX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZZgels_bufferSize(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, cuDoubleComplex * dA, cusolver_int_t ldda, cuDoubleComplex * dB, cusolver_int_t lddb, cuDoubleComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZZgels_bufferSize(hipsolverHandle_t handle, int m, int n, int nrhs, hipDoubleComplex* A, int lda, hipDoubleComplex* B, int ldb, hipDoubleComplex* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnZZgels_bufferSize(handle, lm, ln, lnrhs, &dComplexA, ldda, &dComplexB, lddb, &dComplexX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnZZgels_bufferSize(handle, lm, ln, lnrhs, &dComplexA, ldda, &dComplexB, lddb, &dComplexX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCCgels_bufferSize(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, cuComplex * dA, cusolver_int_t ldda, cuComplex * dB, cusolver_int_t lddb, cuComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCCgels_bufferSize(hipsolverHandle_t handle, int m, int n, int nrhs, hipFloatComplex* A, int lda, hipFloatComplex* B, int ldb, hipFloatComplex* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnCCgels_bufferSize(handle, lm, ln, lnrhs, &complexA, ldda, &complexB, lddb, &complexX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnCCgels_bufferSize(handle, lm, ln, lnrhs, &complexA, ldda, &complexB, lddb, &complexX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDDgels_bufferSize(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, double * dA, cusolver_int_t ldda, double * dB, cusolver_int_t lddb, double * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDDgels_bufferSize(hipsolverHandle_t handle, int m, int n, int nrhs, double* A, int lda, double* B, int ldb, double* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnDDgels_bufferSize(handle, lm, ln, lnrhs, &dA, ldda, &dB, lddb, &dX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnDDgels_bufferSize(handle, lm, ln, lnrhs, &dA, ldda, &dB, lddb, &dX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSSgels_bufferSize(cusolverDnHandle_t handle, cusolver_int_t m, cusolver_int_t n, cusolver_int_t nrhs, float * dA, cusolver_int_t ldda, float * dB, cusolver_int_t lddb, float * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSSgels_bufferSize(hipsolverHandle_t handle, int m, int n, int nrhs, float* A, int lda, float* B, int ldb, float* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnSSgels_bufferSize(handle, lm, ln, lnrhs, &fA, ldda, &fB, lddb, &fX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnSSgels_bufferSize(handle, lm, ln, lnrhs, &fA, ldda, &fB, lddb, &fX, lddx, &Workspace, &lwork_bytes);
#endif

  return 0;
}
