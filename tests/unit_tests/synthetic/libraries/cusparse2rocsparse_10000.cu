// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18.1. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: rocsparse_status status_t;
  hipsparseStatus_t status_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;

  int m = 0;
  int n = 0;
  int k = 0;
  int nnza = 0;
  int nnzb = 0;
  int nnzc = 0;
  int lda = 0;
  int ldb = 0;
  int ldc = 0;
  int csrRowPtrA = 0;
  int csrRowPtrB = 0;
  int csrRowPtrC = 0;
  int csrColIndA = 0;
  int csrColIndB = 0;
  int csrColIndC = 0;
  int nnzTotalDevHostPtr = 0;
  double dAlpha = 0.f;
  double dBeta = 0.f;
  double dA = 0.f;
  double dB = 0.f;
  double dC = 0.f;
  double dcsrSortedValA = 0.f;
  double dcsrSortedValB = 0.f;
  double dcsrSortedValC = 0.f;
  float fA = 0.f;
  float fB = 0.f;
  float fC = 0.f;
  float csrSortedValA = 0.f;
  float csrSortedValB = 0.f;
  float csrSortedValC = 0.f;
  void *pBuffer = nullptr;
  void *workspace = nullptr;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuDoubleComplex -> rocsparse_double_complex under a new option --sparse
  // CHECK: rocblas_double_complex dcomplex, dcomplexA, dcomplexB, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;
  hipDoubleComplex dcomplex, dcomplexA, dcomplexB, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuComplex -> rocsparse_float_complex under a new option --sparse
  // CHECK: rocblas_float_complex complex, complexA, complexB, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;
  hipComplex complex, complexA, complexB, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;

#if CUDA_VERSION >= 10000
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZcsrgeam2(cusparseHandle_t handle, int m, int n, const cuDoubleComplex* alpha, const cusparseMatDescr_t descrA, int nnzA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cuDoubleComplex* beta, const cusparseMatDescr_t descrB, int nnzB, const cuDoubleComplex* csrSortedValB, const int* csrSortedRowPtrB, const int* csrSortedColIndB, const cusparseMatDescr_t descrC, cuDoubleComplex* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsrgeam(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_double_complex* alpha, const rocsparse_mat_descr descr_A, rocsparse_int nnz_A, const rocsparse_double_complex* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const rocsparse_double_complex* beta, const rocsparse_mat_descr descr_B, rocsparse_int nnz_B, const rocsparse_double_complex* csr_val_B, const rocsparse_int* csr_row_ptr_B, const rocsparse_int* csr_col_ind_B, const rocsparse_mat_descr descr_C, rocsparse_double_complex* csr_val_C, const rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C);
  // CHECK: status_t = rocsparse_zcsrgeam(handle_t, m, n, &dcomplexA, matDescr_A, nnza, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, &dcomplexB, matDescr_B, nnzb, &dComplexcsrSortedValB, &csrRowPtrB, &csrColIndB, matDescr_C, &dComplexcsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseZcsrgeam2(handle_t, m, n, &dcomplexA, matDescr_A, nnza, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, &dcomplexB, matDescr_B, nnzb, &dComplexcsrSortedValB, &csrRowPtrB, &csrColIndB, matDescr_C, &dComplexcsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCcsrgeam2(cusparseHandle_t handle, int m, int n, const cuComplex* alpha, const cusparseMatDescr_t descrA, int nnzA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cuComplex* beta, const cusparseMatDescr_t descrB, int nnzB, const cuComplex* csrSortedValB, const int* csrSortedRowPtrB, const int* csrSortedColIndB, const cusparseMatDescr_t descrC, cuComplex* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsrgeam(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_float_complex* alpha, const rocsparse_mat_descr descr_A, rocsparse_int nnz_A, const rocsparse_float_complex* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const rocsparse_float_complex* beta, const rocsparse_mat_descr descr_B, rocsparse_int nnz_B, const rocsparse_float_complex* csr_val_B, const rocsparse_int* csr_row_ptr_B, const rocsparse_int* csr_col_ind_B, const rocsparse_mat_descr descr_C, rocsparse_float_complex* csr_val_C, const rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C);
  // CHECK: status_t = rocsparse_ccsrgeam(handle_t, m, n, &complexA, matDescr_A, nnza, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, &complexB, matDescr_B, nnzb, &complexcsrSortedValB, &csrRowPtrB, &csrColIndB, matDescr_C, &complexcsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseCcsrgeam2(handle_t, m, n, &complexA, matDescr_A, nnza, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, &complexB, matDescr_B, nnzb, &complexcsrSortedValB, &csrRowPtrB, &csrColIndB, matDescr_C, &complexcsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDcsrgeam2(cusparseHandle_t handle, int m, int n, const double* alpha, const cusparseMatDescr_t descrA, int nnzA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* beta, const cusparseMatDescr_t descrB, int nnzB, const double* csrSortedValB, const int* csrSortedRowPtrB, const int* csrSortedColIndB, const cusparseMatDescr_t descrC, double* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsrgeam(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const double* alpha, const rocsparse_mat_descr descr_A, rocsparse_int nnz_A, const double* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const double* beta, const rocsparse_mat_descr descr_B, rocsparse_int nnz_B, const double* csr_val_B, const rocsparse_int* csr_row_ptr_B, const rocsparse_int* csr_col_ind_B, const rocsparse_mat_descr descr_C, double* csr_val_C, const rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C);
  // CHECK: status_t = rocsparse_dcsrgeam(handle_t, m, n, &dA, matDescr_A, nnza, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, &dB, matDescr_B, nnzb, &dcsrSortedValB, &csrRowPtrB, &csrColIndB, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseDcsrgeam2(handle_t, m, n, &dA, matDescr_A, nnza, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, &dB, matDescr_B, nnzb, &dcsrSortedValB, &csrRowPtrB, &csrColIndB, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScsrgeam2(cusparseHandle_t handle, int m, int n, const float* alpha, const cusparseMatDescr_t descrA, int nnzA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* beta, const cusparseMatDescr_t descrB, int nnzB, const float* csrSortedValB, const int* csrSortedRowPtrB, const int* csrSortedColIndB, const cusparseMatDescr_t descrC, float* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsrgeam(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const float* alpha, const rocsparse_mat_descr descr_A, rocsparse_int nnz_A, const float* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const float* beta, const rocsparse_mat_descr descr_B, rocsparse_int nnz_B, const float* csr_val_B, const rocsparse_int* csr_row_ptr_B, const rocsparse_int* csr_col_ind_B, const rocsparse_mat_descr descr_C, float* csr_val_C, const rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C);
  // CHECK: status_t = rocsparse_scsrgeam(handle_t, m, n, &fA, matDescr_A, nnza, &csrSortedValA, &csrRowPtrA, &csrColIndA, &fB, matDescr_B, nnzb, &csrSortedValB, &csrRowPtrB, &csrColIndB, matDescr_C, &csrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseScsrgeam2(handle_t, m, n, &fA, matDescr_A, nnza, &csrSortedValA, &csrRowPtrA, &csrColIndA, &fB, matDescr_B, nnzb, &csrSortedValB, &csrRowPtrB, &csrColIndB, matDescr_C, &csrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcsrgeam2Nnz(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, int nnzA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrB, int nnzB, const int* csrSortedRowPtrB, const int* csrSortedColIndB, const cusparseMatDescr_t descrC, int* csrSortedRowPtrC, int* nnzTotalDevHostPtr, void* workspace);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_csrgeam_nnz(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr_A, rocsparse_int nnz_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const rocsparse_mat_descr descr_B, rocsparse_int nnz_B, const rocsparse_int* csr_row_ptr_B, const rocsparse_int* csr_col_ind_B, const rocsparse_mat_descr descr_C, rocsparse_int* csr_row_ptr_C, rocsparse_int* nnz_C);
  // CHECK: status_t = rocsparse_csrgeam_nnz(handle_t, m, n, matDescr_A, nnza, &csrRowPtrA, &csrColIndA, matDescr_B, nnzb, &csrRowPtrB, &csrColIndB, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr);
  status_t = hipsparseXcsrgeam2Nnz(handle_t, m, n, matDescr_A, nnza, &csrRowPtrA, &csrColIndA, matDescr_B, nnzb, &csrRowPtrB, &csrColIndB, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, workspace);
#endif

  return 0;
}
