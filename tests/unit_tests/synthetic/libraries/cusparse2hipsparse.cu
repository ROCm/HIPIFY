// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "hipsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "hipsparse.h"

int main() {
  printf("17. cuSPARSE API to hipSPARSE API synthetic test\n");

  // CHECK: hipsparseHandle_t handle_t;
  hipsparseHandle_t handle_t;

  // CHECK: hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_C;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_C;

  // CHECK: hipsparseColorInfo_t colorInfo_t;
  hipsparseColorInfo_t colorInfo_t;

  // CHECK: hipsparseOperation_t sparseOperation_t;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_NON_TRANSPOSE = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_TRANSPOSE = HIPSPARSE_OPERATION_TRANSPOSE;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_CONJUGATE_TRANSPOSE = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
  hipsparseOperation_t sparseOperation_t;
  hipsparseOperation_t OPERATION_NON_TRANSPOSE = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipsparseOperation_t OPERATION_TRANSPOSE = HIPSPARSE_OPERATION_TRANSPOSE;
  hipsparseOperation_t OPERATION_CONJUGATE_TRANSPOSE = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;

  // CHECK: hipsparseIndexBase_t indexBase_t;
  // CHECK-NEXT: hipsparseIndexBase_t INDEX_BASE_ZERO = HIPSPARSE_INDEX_BASE_ZERO;
  // CHECK-NEXT: hipsparseIndexBase_t INDEX_BASE_ONE = HIPSPARSE_INDEX_BASE_ONE;
  hipsparseIndexBase_t indexBase_t;
  hipsparseIndexBase_t INDEX_BASE_ZERO = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseIndexBase_t INDEX_BASE_ONE = HIPSPARSE_INDEX_BASE_ONE;

  // CHECK: hipsparseMatrixType_t matrixType_t;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_GENERAL = HIPSPARSE_MATRIX_TYPE_GENERAL;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_SYMMETRIC = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_HERMITIAN = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_TRIANGULAR = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;
  hipsparseMatrixType_t matrixType_t;
  hipsparseMatrixType_t MATRIX_TYPE_GENERAL = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseMatrixType_t MATRIX_TYPE_SYMMETRIC = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  hipsparseMatrixType_t MATRIX_TYPE_HERMITIAN = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  hipsparseMatrixType_t MATRIX_TYPE_TRIANGULAR = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;

  // CHECK: hipsparseDiagType_t diagType_t;
  // CHECK-NEXT: hipsparseDiagType_t DIAG_TYPE_NON_UNIT = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  // CHECK-NEXT: hipsparseDiagType_t DIAG_TYPE_UNIT = HIPSPARSE_DIAG_TYPE_UNIT;
  hipsparseDiagType_t diagType_t;
  hipsparseDiagType_t DIAG_TYPE_NON_UNIT = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseDiagType_t DIAG_TYPE_UNIT = HIPSPARSE_DIAG_TYPE_UNIT;

  // CHECK: hipsparseFillMode_t fillMode_t;
  // CHECK-NEXT: hipsparseFillMode_t FILL_MODE_LOWER = HIPSPARSE_FILL_MODE_LOWER;
  // CHECK-NEXT: hipsparseFillMode_t FILL_MODE_UPPER = HIPSPARSE_FILL_MODE_UPPER;
  hipsparseFillMode_t fillMode_t;
  hipsparseFillMode_t FILL_MODE_LOWER = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseFillMode_t FILL_MODE_UPPER = HIPSPARSE_FILL_MODE_UPPER;

  // CHECK: hipsparseAction_t action_t;
  // CHECK-NEXT: hipsparseAction_t ACTION_SYMBOLIC = HIPSPARSE_ACTION_SYMBOLIC;
  // CHECK-NEXT: hipsparseAction_t ACTION_NUMERIC = HIPSPARSE_ACTION_NUMERIC;
  hipsparseAction_t action_t;
  hipsparseAction_t ACTION_SYMBOLIC = HIPSPARSE_ACTION_SYMBOLIC;
  hipsparseAction_t ACTION_NUMERIC = HIPSPARSE_ACTION_NUMERIC;

  // CHECK: hipsparseDirection_t direction_t;
  // CHECK-NEXT: hipsparseDirection_t DIRECTION_ROW = HIPSPARSE_DIRECTION_ROW;
  // CHECK-NEXT: hipsparseDirection_t DIRECTION_COLUMN = HIPSPARSE_DIRECTION_COLUMN;
  hipsparseDirection_t direction_t;
  hipsparseDirection_t DIRECTION_ROW = HIPSPARSE_DIRECTION_ROW;
  hipsparseDirection_t DIRECTION_COLUMN = HIPSPARSE_DIRECTION_COLUMN;

  // CHECK: hipsparseSolvePolicy_t solvePolicy_t;
  // CHECK-NEXT: hipsparseSolvePolicy_t SOLVE_POLICY_NO_LEVEL = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
  // CHECK-NEXT: hipsparseSolvePolicy_t SOLVE_POLICY_USE_LEVEL = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
  hipsparseSolvePolicy_t solvePolicy_t;
  hipsparseSolvePolicy_t SOLVE_POLICY_NO_LEVEL = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
  hipsparseSolvePolicy_t SOLVE_POLICY_USE_LEVEL = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

  // CHECK: hipsparsePointerMode_t pointerMode_t;
  // CHECK-NEXT: hipsparsePointerMode_t POINTER_MODE_HOST = HIPSPARSE_POINTER_MODE_HOST;
  // CHECK-NEXT: hipsparsePointerMode_t POINTER_MODE_DEVICE = HIPSPARSE_POINTER_MODE_DEVICE;
  hipsparsePointerMode_t pointerMode_t;
  hipsparsePointerMode_t POINTER_MODE_HOST = HIPSPARSE_POINTER_MODE_HOST;
  hipsparsePointerMode_t POINTER_MODE_DEVICE = HIPSPARSE_POINTER_MODE_DEVICE;

  // CHECK: hipsparseStatus_t status_t;
  // CHECK-NEXT: hipsparseStatus_t STATUS_SUCCESS = HIPSPARSE_STATUS_SUCCESS;
  // CHECK-NEXT: hipsparseStatus_t STATUS_NOT_INITIALIZED = HIPSPARSE_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_ALLOC_FAILED = HIPSPARSE_STATUS_ALLOC_FAILED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_INVALID_VALUE = HIPSPARSE_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipsparseStatus_t STATUS_ARCH_MISMATCH = HIPSPARSE_STATUS_ARCH_MISMATCH;
  // CHECK-NEXT: hipsparseStatus_t STATUS_MAPPING_ERROR = HIPSPARSE_STATUS_MAPPING_ERROR;
  // CHECK-NEXT: hipsparseStatus_t STATUS_EXECUTION_FAILED = HIPSPARSE_STATUS_EXECUTION_FAILED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_INTERNAL_ERROR = HIPSPARSE_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipsparseStatus_t STATUS_MATRIX_TYPE_NOT_SUPPORTED = HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_ZERO_PIVOT = HIPSPARSE_STATUS_ZERO_PIVOT;
  hipsparseStatus_t status_t;
  hipsparseStatus_t STATUS_SUCCESS = HIPSPARSE_STATUS_SUCCESS;
  hipsparseStatus_t STATUS_NOT_INITIALIZED = HIPSPARSE_STATUS_NOT_INITIALIZED;
  hipsparseStatus_t STATUS_ALLOC_FAILED = HIPSPARSE_STATUS_ALLOC_FAILED;
  hipsparseStatus_t STATUS_INVALID_VALUE = HIPSPARSE_STATUS_INVALID_VALUE;
  hipsparseStatus_t STATUS_ARCH_MISMATCH = HIPSPARSE_STATUS_ARCH_MISMATCH;
  hipsparseStatus_t STATUS_MAPPING_ERROR = HIPSPARSE_STATUS_MAPPING_ERROR;
  hipsparseStatus_t STATUS_EXECUTION_FAILED = HIPSPARSE_STATUS_EXECUTION_FAILED;
  hipsparseStatus_t STATUS_INTERNAL_ERROR = HIPSPARSE_STATUS_INTERNAL_ERROR;
  hipsparseStatus_t STATUS_MATRIX_TYPE_NOT_SUPPORTED = HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED;
  hipsparseStatus_t STATUS_ZERO_PIVOT = HIPSPARSE_STATUS_ZERO_PIVOT;

  // CHECK: hipStream_t stream_t;
  hipStream_t stream_t;

  int iVal = 0;
  int batchCount = 0;
  int m = 0;
  int n = 0;
  int mb = 0;
  int nb = 0;
  int nnzb = 0;
  int innz = 0;
  int blockDim = 0;
  int cscRowIndA = 0;
  int cscColPtrA = 0;
  int csrRowPtrA = 0;
  int csrColIndA = 0;
  int ncolors = 0;
  int coloring = 0;
  int reordering = 0;
  int bsrRowPtrA = 0;
  int bsrRowPtrC = 0;
  int csrRowPtrC = 0;
  int bsrColIndA = 0;
  int bsrColIndC = 0;
  int csrColIndC = 0;
  int rowBlockDimA = 0;
  int colBlockDimA = 0;
  int rowBlockDimC = 0;
  int colBlockDimC = 0;
  int bsrSortedRowPtrC = 0;
  int bsrSortedColIndC = 0;
  int bufferSizeInBytes = 0;
  int nnzTotalDevHostPtr = 0;
  int64_t size = 0;
  int64_t nnz = 0;
  int64_t rows = 0;
  int64_t cols = 0;
  int64_t ellCols = 0;
  int64_t ellBlockSize = 0;
  int64_t batchStride = 0;
  int64_t offsetsBatchStride = 0;
  int64_t columnsValuesBatchStride = 0;
  int64_t ld = 0;
  void *indices = nullptr;
  void *values = nullptr;
  void *cooRowInd = nullptr;
  int icooRowInd = 0;
  void *cscRowInd = nullptr;
  void *csrColInd = nullptr;
  void *cooColInd = nullptr;
  void *ellColInd = nullptr;
  void *cooValues = nullptr;
  void *csrValues = nullptr;
  void *cscValues = nullptr;
  void *ellValue = nullptr;
  void *csrRowOffsets = nullptr;
  void *cscColOffsets = nullptr;
  void *cooRows = nullptr;
  int icooRows = 0;
  void *cooColumns = nullptr;
  int icooColumns = 0;
  void *data = nullptr;
  void *alpha = nullptr;
  void *beta = nullptr;
  void *pBuffer = nullptr;
  int *P = nullptr;
  void *tempBuffer = nullptr;
  void *c_coeff = nullptr;
  void *s_coeff = nullptr;
  size_t dataSize = 0;
  size_t bufferSize = 0;
  double dfractionToColor = 0.f;
  float ffractionToColor = 0.f;
  double bsrValA = 0.f;
  double csrValA = 0.f;
  float fcsrValA = 0.f;
  double csrValC = 0.f;
  float csrSortedValA = 0.f;
  double dbsrSortedValA = 0.f;
  double dbsrSortedValC = 0.f;
  float fbsrSortedValA = 0.f;
  float fbsrSortedValC = 0.f;
  float fcsrSortedValC = 0.f;
  double percentage = 0.f;

  pruneInfo_t prune_info;

  // CHECK: hipDoubleComplex dcomplex, dComplexbsrSortedValA, dComplexbsrSortedValC;
  hipDoubleComplex dcomplex, dComplexbsrSortedValA, dComplexbsrSortedValC;

  // CHECK: hipComplex complex, complexbsrValA, complexbsrSortedValC;
  hipComplex complex, complexbsrValA, complexbsrSortedValC;

  // CHECK: hipsparseOperation_t opA, opB;
  hipsparseOperation_t opA, opB;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreate(cusparseHandle_t* handle);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreate(hipsparseHandle_t* handle);
  // CHECK: status_t = hipsparseCreate(&handle_t);
  status_t = hipsparseCreate(&handle_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroy(cusparseHandle_t handle);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroy(hipsparseHandle_t handle);
  // CHECK: status_t = hipsparseDestroy(handle_t);
  status_t = hipsparseDestroy(handle_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetStream(cusparseHandle_t handle, cudaStream_t streamId);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetStream(hipsparseHandle_t handle, hipStream_t streamId);
  // CHECK: status_t = hipsparseSetStream(handle_t, stream_t);
  status_t = hipsparseSetStream(handle_t, stream_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetStream(cusparseHandle_t handle, cudaStream_t* streamId);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseGetStream(hipsparseHandle_t handle, hipStream_t* streamId);
  // CHECK: status_t = hipsparseGetStream(handle_t, &stream_t);
  status_t = hipsparseGetStream(handle_t, &stream_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetPointerMode(cusparseHandle_t handle, cusparsePointerMode_t mode);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetPointerMode(hipsparseHandle_t handle, hipsparsePointerMode_t mode);
  // CHECK: status_t = hipsparseSetPointerMode(handle_t, pointerMode_t);
  status_t = hipsparseSetPointerMode(handle_t, pointerMode_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetPointerMode(cusparseHandle_t handle, cusparsePointerMode_t* mode);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseGetPointerMode(hipsparseHandle_t handle, hipsparsePointerMode_t* mode);
  // CHECK: status_t = hipsparseGetPointerMode(handle_t, &pointerMode_t);
  status_t = hipsparseGetPointerMode(handle_t, &pointerMode_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetVersion(cusparseHandle_t handle, int* version);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseGetVersion(hipsparseHandle_t handle, int* version);
  // CHECK: status_t = hipsparseGetVersion(handle_t, &iVal);
  status_t = hipsparseGetVersion(handle_t, &iVal);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateMatDescr(cusparseMatDescr_t* descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateMatDescr(hipsparseMatDescr_t* descrA);
  // CHECK: status_t = hipsparseCreateMatDescr(&matDescr_t);
  status_t = hipsparseCreateMatDescr(&matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyMatDescr(cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroyMatDescr(hipsparseMatDescr_t descrA);
  // CHECK: status_t = hipsparseDestroyMatDescr(matDescr_t);
  status_t = hipsparseDestroyMatDescr(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatIndexBase(cusparseMatDescr_t descrA, cusparseIndexBase_t base);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetMatIndexBase(hipsparseMatDescr_t descrA, hipsparseIndexBase_t base);
  // CHECK: status_t = hipsparseSetMatIndexBase(matDescr_t, indexBase_t);
  status_t = hipsparseSetMatIndexBase(matDescr_t, indexBase_t);

  // CUDA: cusparseIndexBase_t CUSPARSEAPI cusparseGetMatIndexBase(const cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseIndexBase_t hipsparseGetMatIndexBase(const hipsparseMatDescr_t descrA);
  // CHECK: indexBase_t = hipsparseGetMatIndexBase(matDescr_t);
  indexBase_t = hipsparseGetMatIndexBase(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatType(cusparseMatDescr_t descrA, cusparseMatrixType_t type);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetMatType(hipsparseMatDescr_t descrA, hipsparseMatrixType_t type);
  // CHECK: status_t = hipsparseSetMatType(matDescr_t, matrixType_t);
  status_t = hipsparseSetMatType(matDescr_t, matrixType_t);

  // CUDA: cusparseMatrixType_t CUSPARSEAPI cusparseGetMatType(const cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseMatrixType_t hipsparseGetMatType(const hipsparseMatDescr_t descrA);
  // CHECK: matrixType_t = hipsparseGetMatType(matDescr_t);
  matrixType_t = hipsparseGetMatType(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatFillMode(cusparseMatDescr_t descrA, cusparseFillMode_t fillMode);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetMatFillMode(hipsparseMatDescr_t descrA, hipsparseFillMode_t fillMode);
  // CHECK: status_t = hipsparseSetMatFillMode(matDescr_t, fillMode_t);
  status_t = hipsparseSetMatFillMode(matDescr_t, fillMode_t);

  // CUDA: cusparseFillMode_t CUSPARSEAPI cusparseGetMatFillMode(const cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseFillMode_t hipsparseGetMatFillMode(const hipsparseMatDescr_t descrA);
  // CHECK: fillMode_t = hipsparseGetMatFillMode(matDescr_t);
  fillMode_t = hipsparseGetMatFillMode(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatDiagType(cusparseMatDescr_t descrA, cusparseDiagType_t diagType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetMatDiagType(hipsparseMatDescr_t descrA, hipsparseDiagType_t diagType);
  // CHECK: status_t = hipsparseSetMatDiagType(matDescr_t, diagType_t);
  status_t = hipsparseSetMatDiagType(matDescr_t, diagType_t);

  // CUDA: cusparseDiagType_t CUSPARSEAPI cusparseGetMatDiagType(const cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseDiagType_t hipsparseGetMatDiagType(const hipsparseMatDescr_t descrA);
  // CHECK: diagType_t = hipsparseGetMatDiagType(matDescr_t);
  diagType_t = hipsparseGetMatDiagType(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateColorInfo(cusparseColorInfo_t* info);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateColorInfo(hipsparseColorInfo_t* info);
  // CHECK: status_t = hipsparseCreateColorInfo(&colorInfo_t);
  status_t = hipsparseCreateColorInfo(&colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyColorInfo(cusparseColorInfo_t info);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroyColorInfo(hipsparseColorInfo_t info);
  // CHECK: status_t = hipsparseDestroyColorInfo(colorInfo_t);
  status_t = hipsparseDestroyColorInfo(colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZcsrcolor(cusparseHandle_t handle, int m, int nnz, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* fractionToColor, int* ncolors, int* coloring, int* reordering, const cusparseColorInfo_t info);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZcsrcolor(hipsparseHandle_t handle, int m, int nnz, const hipsparseMatDescr_t descrA, const hipDoubleComplex* csrValA, const int* csrRowPtrA, const int* csrColIndA, const double* fractionToColor, int* ncolors, int* coloring, int* reordering, hipsparseColorInfo_t info);
  // CHECK: status_t = hipsparseZcsrcolor(handle_t, m, innz, matDescr_t, &dcomplex, &csrRowPtrA, &csrColIndA, &dfractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);
  status_t = hipsparseZcsrcolor(handle_t, m, innz, matDescr_t, &dcomplex, &csrRowPtrA, &csrColIndA, &dfractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCcsrcolor(cusparseHandle_t handle, int m, int nnz, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* fractionToColor, int* ncolors, int* coloring, int* reordering, const cusparseColorInfo_t info);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCcsrcolor(hipsparseHandle_t handle, int m, int nnz, const hipsparseMatDescr_t descrA, const hipComplex* csrValA, const int* csrRowPtrA, const int* csrColIndA, const float* fractionToColor, int* ncolors, int* coloring, int* reordering, hipsparseColorInfo_t info);
  // CHECK: status_t = hipsparseCcsrcolor(handle_t, m, innz, matDescr_t, &complex, &csrRowPtrA, &csrColIndA, &ffractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);
  status_t = hipsparseCcsrcolor(handle_t, m, innz, matDescr_t, &complex, &csrRowPtrA, &csrColIndA, &ffractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDcsrcolor(cusparseHandle_t handle, int m, int nnz, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* fractionToColor, int* ncolors, int* coloring, int* reordering, const cusparseColorInfo_t info);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDcsrcolor(hipsparseHandle_t handle, int m, int nnz, const hipsparseMatDescr_t descrA, const double* csrValA, const int* csrRowPtrA, const int* csrColIndA, const double* fractionToColor, int* ncolors, int* coloring, int* reordering, hipsparseColorInfo_t info);
  // CHECK: status_t = hipsparseDcsrcolor(handle_t, m, innz, matDescr_t, &csrValA, &csrRowPtrA, &csrColIndA, &dfractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);
  status_t = hipsparseDcsrcolor(handle_t, m, innz, matDescr_t, &csrValA, &csrRowPtrA, &csrColIndA, &dfractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScsrcolor(cusparseHandle_t handle, int m, int nnz, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* fractionToColor, int* ncolors, int* coloring, int* reordering, const cusparseColorInfo_t info);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseScsrcolor(hipsparseHandle_t handle, int m, int nnz, const hipsparseMatDescr_t descrA, const float* csrValA, const int* csrRowPtrA, const int* csrColIndA, const float* fractionToColor, int* ncolors, int* coloring, int* reordering, hipsparseColorInfo_t info);
  // CHECK: status_t = hipsparseScsrcolor(handle_t, m, innz, matDescr_t, &csrSortedValA, &csrRowPtrA, &csrColIndA, &ffractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);
  status_t = hipsparseScsrcolor(handle_t, m, innz, matDescr_t, &csrSortedValA, &csrRowPtrA, &csrColIndA, &ffractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);

  // CUDA:cusparseStatus_t CUSPARSEAPI cusparseZgebsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const cuDoubleComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, cuDoubleComplex* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDimC, int colBlockDimC, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZgebsr2gebsr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const hipDoubleComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, const hipsparseMatDescr_t descrC, hipDoubleComplex* bsrValC, int* bsrRowPtrC, int* bsrColIndC, int rowBlockDimC, int colBlockDimC, void* buffer);
  // CHECK: status_t = hipsparseZgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dcomplex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dComplexbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, bsrRowPtrC, bsrColIndC, tempBuffer);
  status_t = hipsparseZgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dcomplex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dComplexbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, bsrRowPtrC, bsrColIndC, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgebsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const cuDoubleComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC,int colBlockDimC, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZgebsr2gebsr_bufferSize(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const hipDoubleComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* bufferSize);
  // CHECK: status_t = hipsparseZgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dcomplex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);
  status_t = hipsparseZgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dcomplex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgebsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const cuComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, cuComplex* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDimC, int colBlockDimC, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCgebsr2gebsr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const hipComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, const hipsparseMatDescr_t descrC, hipComplex* bsrValC, int* bsrRowPtrC, int* bsrColIndC, int rowBlockDimC, int colBlockDimC, void* buffer);
  // CHECK: status_t = hipsparseCgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &complex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &complexbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);
  status_t = hipsparseCgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &complex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &complexbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgebsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const cuComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCgebsr2gebsr_bufferSize(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const hipComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* bufferSize);
  // CHECK: status_t = hipsparseCgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &complex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);
  status_t = hipsparseCgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &complex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgebsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const double* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, double* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDimC, int colBlockDimC, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDgebsr2gebsr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const double* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, const hipsparseMatDescr_t descrC, double* bsrValC, int* bsrRowPtrC, int* bsrColIndC, int rowBlockDimC, int colBlockDimC, void* buffer);
  // CHECK: status_t = hipsparseDgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);
  status_t = hipsparseDgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgebsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const double* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDgebsr2gebsr_bufferSize(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const double* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* bufferSize);
  // CHECK: status_t = hipsparseDgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);
  status_t = hipsparseDgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgebsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const float* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, float* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDimC, int colBlockDimC, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSgebsr2gebsr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const float* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, const hipsparseMatDescr_t descrC, float* bsrValC, int* bsrRowPtrC, int* bsrColIndC, int rowBlockDimC, int colBlockDimC, void* buffer);
  // CHECK: status_t = hipsparseSgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &fbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);
  status_t = hipsparseSgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &fbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgebsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const float* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSgebsr2gebsr_bufferSize(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const float* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* bufferSize);
  // CHECK: status_t = hipsparseSgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);
  status_t = hipsparseSgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXgebsr2gebsrNnz(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, int* bsrSortedRowPtrC, int rowBlockDimC, int colBlockDimC, int* nnzTotalDevHostPtr, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXgebsr2gebsrNnz(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, const hipsparseMatDescr_t descrC, int* bsrRowPtrC, int rowBlockDimC, int colBlockDimC, int* nnzTotalDevHostPtr, void* buffer);
  // CHECK: status_t = hipsparseXgebsr2gebsrNnz(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &bsrSortedRowPtrC,  rowBlockDimC, colBlockDimC, &nnzTotalDevHostPtr, &tempBuffer);
  status_t = hipsparseXgebsr2gebsrNnz(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &bsrSortedRowPtrC,  rowBlockDimC, colBlockDimC, &nnzTotalDevHostPtr, &tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgebsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const cuDoubleComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDim, int colBlockDim, const cusparseMatDescr_t descrC, cuDoubleComplex* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZgebsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const hipDoubleComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDim, int colBlockDim, const hipsparseMatDescr_t descrC, hipDoubleComplex* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseZgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &dComplexbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dComplexbsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseZgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &dComplexbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dComplexbsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgebsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const cuComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDim, int colBlockDim, const cusparseMatDescr_t descrC, cuComplex* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCgebsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const hipComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDim, int colBlockDim, const hipsparseMatDescr_t descrC, hipComplex* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseCgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &complexbsrValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &complexbsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseCgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &complexbsrValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &complexbsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgebsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const double* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDim, int colBlockDim, const cusparseMatDescr_t descrC, double* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDgebsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const double* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDim, int colBlockDim, const hipsparseMatDescr_t descrC, double* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseDgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &bsrValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseDgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &bsrValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgebsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const float* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDim, int colBlockDim, const cusparseMatDescr_t descrC, float* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSgebsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const float* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDim, int colBlockDim, const hipsparseMatDescr_t descrC, float* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseSgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseSgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZbsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const cuDoubleComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, cuDoubleComplex* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZbsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const hipDoubleComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int blockDim, const hipsparseMatDescr_t descrC, hipDoubleComplex* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseZbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &dComplexbsrSortedValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &dComplexbsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseZbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &dComplexbsrSortedValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &dComplexbsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCbsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const cuComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, cuComplex* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCbsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const hipComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int blockDim, const hipsparseMatDescr_t descrC, hipComplex* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseCbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &complexbsrValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &complexbsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseCbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &complexbsrValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &complexbsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDbsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const double* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, double* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDbsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const double* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int blockDim, const hipsparseMatDescr_t descrC, double* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseDbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &bsrValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseDbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &bsrValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSbsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const float* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, float* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSbsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const float* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int blockDim, const hipsparseMatDescr_t descrC, float* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseSbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseSbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcoosortByColumn(cusparseHandle_t handle, int m, int n, int nnz, int* cooRowsA, int* cooColsA, int* P, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcoosortByColumn(hipsparseHandle_t handle, int m, int n, int nnz, int* cooRows, int* cooCols, int* P, void* pBuffer);
  // CHECK: status_t = hipsparseXcoosortByColumn(handle_t, m, n, innz, &icooRows, &icooColumns, P, pBuffer);
  status_t = hipsparseXcoosortByColumn(handle_t, m, n, innz, &icooRows, &icooColumns, P, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcoosortByRow(cusparseHandle_t handle, int m, int n, int nnz, int* cooRowsA, int* cooColsA, int* P, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcoosortByRow(hipsparseHandle_t handle, int m, int n, int nnz, int* cooRows, int* cooCols, int* P, void* pBuffer);
  // CHECK: status_t = hipsparseXcoosortByRow(handle_t, m, n, innz, &icooRows, &icooColumns, P, pBuffer);
  status_t = hipsparseXcoosortByRow(handle_t, m, n, innz, &icooRows, &icooColumns, P, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcoosort_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnz, const int* cooRowsA, const int* cooColsA, size_t* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcoosort_bufferSizeExt(hipsparseHandle_t handle, int m, int n, int nnz, const int* cooRows, const int* cooCols, size_t* pBufferSizeInBytes);
  // CHECK: status_t = hipsparseXcoosort_bufferSizeExt(handle_t, m, n, innz, &icooRows, &icooColumns, &bufferSize);
  status_t = hipsparseXcoosort_bufferSizeExt(handle_t, m, n, innz, &icooRows, &icooColumns, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcscsort(cusparseHandle_t handle, int m, int n, int nnz, const cusparseMatDescr_t descrA, const int* cscColPtrA, int* cscRowIndA, int* P, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcscsort(hipsparseHandle_t handle, int m, int n, int nnz, const hipsparseMatDescr_t descrA, const int* cscColPtr, int* cscRowInd, int* P, void* pBuffer);
  // CHECK: status_t = hipsparseXcscsort(handle_t, m, n, innz, matDescr_A, &cscColPtrA, &cscRowIndA, P, pBuffer);
  status_t = hipsparseXcscsort(handle_t, m, n, innz, matDescr_A, &cscColPtrA, &cscRowIndA, P, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcscsort_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnz, const int* cscColPtrA, const int* cscRowIndA, size_t* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcscsort_bufferSizeExt(hipsparseHandle_t handle, int m, int n, int nnz, const int* cscColPtr, const int* cscRowInd, size_t* pBufferSizeInBytes);
  // CHECK: status_t = hipsparseXcscsort_bufferSizeExt(handle_t, m, n, innz, &cscColPtrA, &cscRowIndA, &bufferSize);
  status_t = hipsparseXcscsort_bufferSizeExt(handle_t, m, n, innz, &cscColPtrA, &cscRowIndA, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcsrsort(cusparseHandle_t handle, int m, int n, int nnz, const cusparseMatDescr_t descrA, const int* csrRowPtrA, int* csrColIndA, int* P, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcsrsort(hipsparseHandle_t handle, int m, int n, int nnz, const hipsparseMatDescr_t descrA, const int* csrRowPtr, int* csrColInd, int* P, void* pBuffer);
  // CHECK: status_t = hipsparseXcsrsort(handle_t, m, n, innz, matDescr_A, &cscRowIndA, &cscColPtrA, P, pBuffer);
  status_t = hipsparseXcsrsort(handle_t, m, n, innz, matDescr_A, &cscRowIndA, &cscColPtrA, P, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcsrsort_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnz, const int* csrRowPtrA, const int* csrColIndA, size_t* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcsrsort_bufferSizeExt(hipsparseHandle_t handle, int m, int n, int nnz, const int* csrRowPtr, const int* csrColInd, size_t* pBufferSizeInBytes);
  // CHECK: status_t = hipsparseXcsrsort_bufferSizeExt(handle_t, m, n, innz, &cscRowIndA, &cscColPtrA, &bufferSize);
  status_t = hipsparseXcsrsort_bufferSizeExt(handle_t, m, n, innz, &cscRowIndA, &cscColPtrA, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateIdentityPermutation(cusparseHandle_t handle, int n, int* p);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateIdentityPermutation(hipsparseHandle_t handle, int n, int* p);
  // CHECK: status_t = hipsparseCreateIdentityPermutation(handle_t, n, P);
  status_t = hipsparseCreateIdentityPermutation(handle_t, n, P);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcoo2csr(cusparseHandle_t handle, const int* cooRowInd, int nnz, int m, int* csrSortedRowPtr, cusparseIndexBase_t idxBase);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcoo2csr(hipsparseHandle_t handle, const int* cooRowInd, int nnz, int m, int* csrRowPtr, hipsparseIndexBase_t idxBase);
  // CHECK: status_t = hipsparseXcoo2csr(handle_t, &icooRowInd, nnz, m, &csrRowPtrA, indexBase_t);
  status_t = hipsparseXcoo2csr(handle_t, &icooRowInd, nnz, m, &csrRowPtrA, indexBase_t);

#if CUDA_VERSION >= 8000
  // CHECK: hipDataType dataType_t;
  // CHECK-NEXT: hipDataType dataType;
  hipDataType dataType_t;
  hipDataType dataType;
#endif

#if CUDA_VERSION >= 9000
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDpruneCsr2csrByPercentage(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float percentage, const cusparseMatDescr_t descrC, double* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, pruneInfo_t info, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDpruneCsr2csrByPercentage(hipsparseHandle_t handle, int m, int n, int nnzA, const hipsparseMatDescr_t descrA, const double* csrValA, const int* csrRowPtrA, const int* csrColIndA, double percentage, const hipsparseMatDescr_t descrC, double* csrValC, const int* csrRowPtrC, int* csrColIndC, pruneInfo_t info, void* buffer);
  // CHECK: status_t = hipsparseDpruneCsr2csrByPercentage(handle_t, m, n, nnz, matDescr_A, &csrValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);
  status_t = hipsparseDpruneCsr2csrByPercentage(handle_t, m, n, nnz, matDescr_A, &csrValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);
#endif

#if CUDA_VERSION >= 8000 && CUDA_VERSION < 12000
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCopyMatDescr(cusparseMatDescr_t dest, const cusparseMatDescr_t src);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCopyMatDescr(hipsparseMatDescr_t dest, const hipsparseMatDescr_t src);
  // CHECK: status_t = hipsparseCopyMatDescr(matDescr_t, matDescr_t_2);
  status_t = cusparseCopyMatDescr(matDescr_t, matDescr_t_2);
#endif

#if (CUDA_VERSION >= 10010 && CUDA_VERSION < 11000 && !defined(_WIN32)) || CUDA_VERSION >= 11000
  // CHECK: hipsparseSpMatDescr_t spMatDescr_t, matC;
  hipsparseSpMatDescr_t spMatDescr_t, matC;

  // CHECK: hipsparseDnMatDescr_t dnMatDescr_t, matA, matB;
  hipsparseDnMatDescr_t dnMatDescr_t, matA, matB;

  // CHECK: hipsparseIndexType_t indexType_t;
  // CHECK-NEXT: hipsparseIndexType_t csrRowOffsetsType;
  // CHECK-NEXT: hipsparseIndexType_t cscColOffsetsType;
  // CHECK-NEXT: hipsparseIndexType_t cscRowIndType;
  // CHECK-NEXT: hipsparseIndexType_t csrColIndType;
  // CHECK-NEXT: hipsparseIndexType_t ellIdxType;
  // CHECK-NEXT: hipsparseIndexType_t INDEX_16U = HIPSPARSE_INDEX_16U;
  // CHECK-NEXT: hipsparseIndexType_t INDEX_32I = HIPSPARSE_INDEX_32I;
  // CHECK-NEXT: hipsparseIndexType_t INDEX_64I = HIPSPARSE_INDEX_64I;
  hipsparseIndexType_t indexType_t;
  hipsparseIndexType_t csrRowOffsetsType;
  hipsparseIndexType_t cscColOffsetsType;
  hipsparseIndexType_t cscRowIndType;
  hipsparseIndexType_t csrColIndType;
  hipsparseIndexType_t ellIdxType;
  hipsparseIndexType_t INDEX_16U = HIPSPARSE_INDEX_16U;
  hipsparseIndexType_t INDEX_32I = HIPSPARSE_INDEX_32I;
  hipsparseIndexType_t INDEX_64I = HIPSPARSE_INDEX_64I;

  // CHECK: hipsparseFormat_t format_t;
  // CHECK-NEXT: hipsparseFormat_t FORMAT_CSR = HIPSPARSE_FORMAT_CSR;
  // CHECK-NEXT: hipsparseFormat_t FORMAT_CSC = HIPSPARSE_FORMAT_CSC;
  // CHECK-NEXT: hipsparseFormat_t FORMAT_CSO = HIPSPARSE_FORMAT_COO;
  hipsparseFormat_t format_t;
  hipsparseFormat_t FORMAT_CSR = HIPSPARSE_FORMAT_CSR;
  hipsparseFormat_t FORMAT_CSC = HIPSPARSE_FORMAT_CSC;
  hipsparseFormat_t FORMAT_CSO = HIPSPARSE_FORMAT_COO;

  // CHECK: hipsparseOrder_t order_t;
  // CHECK-NEXT: hipsparseOrder_t ORDER_COL = HIPSPARSE_ORDER_COL;
  // CHECK-NEXT: hipsparseOrder_t ORDER_ROW = HIPSPARSE_ORDER_ROW;
  hipsparseOrder_t order_t;
  hipsparseOrder_t ORDER_COL = HIPSPARSE_ORDER_COL;
  hipsparseOrder_t ORDER_ROW = HIPSPARSE_ORDER_ROW;

  // CHECK: hipsparseSpMMAlg_t spMMAlg_t;
  hipsparseSpMMAlg_t spMMAlg_t;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCoo(cusparseSpMatDescr_t* spMatDescr, int64_t ows, int64_t cols, int64_t nnz, void* cooRowInd, void* cooColInd, void* cooValues, cusparseIndexType_t cooIdxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateCoo(hipsparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cooRowInd, void* cooColInd, void* cooValues, hipsparseIndexType_t cooIdxType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateCoo(&spMatDescr_t, rows, cols, nnz, cooRowInd, cooColInd, cooValues, indexType_t, indexBase_t, dataType);
  status_t = hipsparseCreateCoo(&spMatDescr_t, rows, cols, nnz, cooRowInd, cooColInd, cooValues, indexType_t, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroySpMat(cusparseConstSpMatDescr_t spMatDescr);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroySpMat(hipsparseSpMatDescr_t spMatDescr);
  // CHECK: status_t = hipsparseDestroySpMat(spMatDescr_t);
  status_t = hipsparseDestroySpMat(spMatDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCooGet(cusparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** cooRowInd, void** cooColInd, void** cooValues, cusparseIndexType_t* idxType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // HIP: hipsparseStatus_t hipsparseCooGet(const hipsparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** cooRowInd, void** cooColInd, void** cooValues, hipsparseIndexType_t* idxType, hipsparseIndexBase_t* idxBase, hipDataType* valueType);
  // CHECK: status_t = hipsparseCooGet(spMatDescr_t, &rows, &cols, &nnz, &cooRowInd, &cooColInd, &cooValues, &indexType_t, &indexBase_t, &dataType);
  status_t = hipsparseCooGet(spMatDescr_t, &rows, &cols, &nnz, &cooRowInd, &cooColInd, &cooValues, &indexType_t, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetFormat(cusparseConstSpMatDescr_t spMatDescr, cusparseFormat_t* format);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatGetFormat(const hipsparseSpMatDescr_t spMatDescr, hipsparseFormat_t* format);
  // CHECK: status_t = hipsparseSpMatGetFormat(spMatDescr_t, &format_t);
  status_t = hipsparseSpMatGetFormat(spMatDescr_t, &format_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetIndexBase(cusparseConstSpMatDescr_t spMatDescr, cusparseIndexBase_t* idxBase);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatGetIndexBase(const hipsparseSpMatDescr_t spMatDescr, hipsparseIndexBase_t* idxBase);
  // CHECK: status_t = hipsparseSpMatGetIndexBase(spMatDescr_t, &indexBase_t);
  status_t = hipsparseSpMatGetIndexBase(spMatDescr_t, &indexBase_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateDnMat(cusparseDnMatDescr_t* dnMatDescr, int64_t rows, int64_t cols, int64_t ld, void* values, cudaDataType valueType, cusparseOrder_t order);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateDnMat(hipsparseDnMatDescr_t* dnMatDescr, int64_t rows, int64_t cols, int64_t ld, void* values, hipDataType valueType, hipsparseOrder_t order);
  // CHECK: status_t = hipsparseCreateDnMat(&dnMatDescr_t, rows, cols, ld, values, dataType, order_t);
  status_t = hipsparseCreateDnMat(&dnMatDescr_t, rows, cols, ld, values, dataType, order_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyDnMat(cusparseConstDnMatDescr_t dnMatDescr);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroyDnMat(hipsparseDnMatDescr_t dnMatDescr);
  // CHECK: status_t = hipsparseDestroyDnMat(dnMatDescr_t);
  status_t = hipsparseDestroyDnMat(dnMatDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatGet(cusparseDnMatDescr_t dnMatDescr, int64_t* rows, int64_t* cols, int64_t* ld, void** values, cudaDataType* type, cusparseOrder_t* order);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnMatGet(const hipsparseDnMatDescr_t dnMatDescr, int64_t* rows, int64_t* cols, int64_t* ld, void** values, hipDataType* valueType, hipsparseOrder_t* order);
  // CHECK: status_t = hipsparseDnMatGet(dnMatDescr_t, &rows, &cols, &ld, &values, &dataType, &order_t);
  status_t = hipsparseDnMatGet(dnMatDescr_t, &rows, &cols, &ld, &values, &dataType, &order_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatGetStridedBatch(cusparseConstDnMatDescr_t dnMatDescr, int* batchCount, int64_t* batchStride);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnMatGetStridedBatch(hipsparseDnMatDescr_t dnMatDescr, int* batchCount, int64_t* batchStride);
  // CHECK: status_t = hipsparseDnMatGetStridedBatch(dnMatDescr_t, &batchCount, &batchStride);
  status_t = hipsparseDnMatGetStridedBatch(dnMatDescr_t, &batchCount, &batchStride);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatSetStridedBatch(cusparseDnMatDescr_t dnMatDescr, int batchCount, int64_t batchStride);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnMatSetStridedBatch(hipsparseDnMatDescr_t dnMatDescr, int batchCount, int64_t batchStride);
  // CHECK: status_t = hipsparseDnMatSetStridedBatch(dnMatDescr_t, batchCount, batchStride);
  status_t = hipsparseDnMatSetStridedBatch(dnMatDescr_t, batchCount, batchStride);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipsparseCsr2CscAlg_t Csr2CscAlg_t;
  // CHECK-NEXT: hipsparseCsr2CscAlg_t CSR2CSC_ALG1 = HIPSPARSE_CSR2CSC_ALG1;
  hipsparseCsr2CscAlg_t Csr2CscAlg_t;
  hipsparseCsr2CscAlg_t CSR2CSC_ALG1 = HIPSPARSE_CSR2CSC_ALG1;
#endif

#if (CUDA_VERSION >= 10010 && CUDA_VERSION < 11000 && !defined(_WIN32)) || (CUDA_VERSION >= 11000 && CUDA_VERSION < 12000)
  // CHECK: hipsparseSpMMAlg_t COOMM_ALG1 = HIPSPARSE_COOMM_ALG1;
  // CHECK-NEXT: hipsparseSpMMAlg_t COOMM_ALG2 = HIPSPARSE_COOMM_ALG2;
  // CHECK-NEXT: hipsparseSpMMAlg_t COOMM_ALG3 = HIPSPARSE_COOMM_ALG3;
  cusparseSpMMAlg_t COOMM_ALG1 = CUSPARSE_COOMM_ALG1;
  cusparseSpMMAlg_t COOMM_ALG2 = CUSPARSE_COOMM_ALG2;
  cusparseSpMMAlg_t COOMM_ALG3 = CUSPARSE_COOMM_ALG3;
#endif

#if CUDA_VERSION >= 10010 && CUDA_VERSION < 12000
  // CHECK: hipsparseCsr2CscAlg_t CSR2CSC_ALG2 = HIPSPARSE_CSR2CSC_ALG2;
  cusparseCsr2CscAlg_t CSR2CSC_ALG2 = CUSPARSE_CSR2CSC_ALG2;
#endif

#if (CUDA_VERSION >= 10020 && CUDA_VERSION < 11000 && !defined(_WIN32)) || CUDA_VERSION >= 11000
  // CHECK: hipsparseSpVecDescr_t spVecDescr_t;
  hipsparseSpVecDescr_t spVecDescr_t;

  // CHECK: hipsparseDnVecDescr_t dnVecDescr_t, vecX, vecY;
  hipsparseDnVecDescr_t dnVecDescr_t, vecX, vecY;

  // CHECK: hipsparseSpMVAlg_t spMVAlg_t;
  hipsparseSpMVAlg_t spMVAlg_t;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateSpVec(cusparseSpVecDescr_t* spVecDescr, int64_t size, int64_t nnz, void* indices, void* values, cusparseIndexType_t idxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateSpVec(hipsparseSpVecDescr_t* spVecDescr, int64_t size, int64_t nnz, void* indices, void* values, hipsparseIndexType_t idxType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateSpVec(&spVecDescr_t, size, nnz, indices, values, indexType_t, indexBase_t, dataType);
  status_t = hipsparseCreateSpVec(&spVecDescr_t, size, nnz, indices, values, indexType_t, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroySpVec(cusparseConstSpVecDescr_t spVecDescr);
  // HIP: hipsparseStatus_t hipsparseDestroySpVec(hipsparseSpVecDescr_t spVecDescr);
  // CHECK: status_t = hipsparseDestroySpVec(spVecDescr_t);
  status_t = hipsparseDestroySpVec(spVecDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecGet(cusparseSpVecDescr_t spVecDescr, int64_t* size, int64_t* nnz, void** indices, void** values, cusparseIndexType_t* idxType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpVecGet(const hipsparseSpVecDescr_t spVecDescr, int64_t* size, int64_t* nnz, void** indices, void** values, hipsparseIndexType_t* idxType, hipsparseIndexBase_t* idxBase, hipDataType* valueType);
  // CHECK: status_t = hipsparseSpVecGet(spVecDescr_t, &size, &nnz, &indices, &values, &indexType_t, &indexBase_t, &dataType);
  status_t = hipsparseSpVecGet(spVecDescr_t, &size, &nnz, &indices, &values, &indexType_t, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecGetIndexBase(cusparseConstSpVecDescr_t spVecDescr, cusparseIndexBase_t* idxBase);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpVecGetIndexBase(const hipsparseSpVecDescr_t spVecDescr, hipsparseIndexBase_t* idxBase);
  // CHECK: status_t = hipsparseSpVecGetIndexBase(spVecDescr_t, &indexBase_t);
  status_t = hipsparseSpVecGetIndexBase(spVecDescr_t, &indexBase_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecGetValues(cusparseSpVecDescr_t spVecDescr, void** values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpVecGetValues(const hipsparseSpVecDescr_t spVecDescr, void** values);
  // CHECK: status_t = hipsparseSpVecGetValues(spVecDescr_t, &values);
  status_t = hipsparseSpVecGetValues(spVecDescr_t, &values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecSetValues(cusparseSpVecDescr_t spVecDescr, void* values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpVecSetValues(hipsparseSpVecDescr_t spVecDescr, void* values);
  // CHECK: status_t = hipsparseSpVecSetValues(spVecDescr_t, values);
  status_t = hipsparseSpVecSetValues(spVecDescr_t, values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCsr(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* csrRowOffsets, void* csrColInd, void* csrValues, cusparseIndexType_t csrRowOffsetsType, cusparseIndexType_t csrColIndType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateCsr(hipsparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* csrRowOffsets, void* csrColInd, void* csrValues, hipsparseIndexType_t csrRowOffsetsType, hipsparseIndexType_t csrColIndType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateCsr(&spMatDescr_t, rows, cols, nnz, csrRowOffsets, csrColInd, csrValues, csrRowOffsetsType, csrColIndType, indexBase_t, dataType);
  status_t = hipsparseCreateCsr(&spMatDescr_t, rows, cols, nnz, csrRowOffsets, csrColInd, csrValues, csrRowOffsetsType, csrColIndType, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCsrGet(cusparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** csrRowOffsets, void** csrColInd, void** csrValues, cusparseIndexType_t* csrRowOffsetsType, cusparseIndexType_t* csrColIndType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCsrGet(const hipsparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** csrRowOffsets, void** csrColInd, void** csrValues, hipsparseIndexType_t* csrRowOffsetsType, hipsparseIndexType_t* csrColIndType, hipsparseIndexBase_t* idxBase, hipDataType* valueType);
  // CHECK: status_t = hipsparseCsrGet(spMatDescr_t, &rows, &cols, &nnz, &csrRowOffsets, &csrColInd, &csrValues, &csrRowOffsetsType, &csrColIndType, &indexBase_t, &dataType);
  status_t = hipsparseCsrGet(spMatDescr_t, &rows, &cols, &nnz, &csrRowOffsets, &csrColInd, &csrValues, &csrRowOffsetsType, &csrColIndType, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetValues(cusparseSpMatDescr_t spMatDescr, void** values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatGetValues(hipsparseSpMatDescr_t spMatDescr, void** values);
  // CHECK: status_t = hipsparseSpMatGetValues(spMatDescr_t, &values);
  status_t = hipsparseSpMatGetValues(spMatDescr_t, &values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatSetValues(cusparseSpMatDescr_t spMatDescr, void* values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatSetValues(hipsparseSpMatDescr_t spMatDescr, void* values);
  // CHECK: status_t = hipsparseSpMatSetValues(spMatDescr_t, values);
  status_t = hipsparseSpMatSetValues(spMatDescr_t, values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetStridedBatch(cusparseConstSpMatDescr_t spMatDescr, int* batchCount);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatGetStridedBatch(hipsparseSpMatDescr_t spMatDescr, int* batchCount);
  // CHECK: status_t = hipsparseSpMatGetStridedBatch(spMatDescr_t, &batchCount);
  status_t = hipsparseSpMatGetStridedBatch(spMatDescr_t, &batchCount);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateDnVec(cusparseDnVecDescr_t* dnVecDescr, int64_t size, void* values, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateDnVec(hipsparseDnVecDescr_t* dnVecDescr, int64_t size, void* values, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateDnVec(&dnVecDescr_t, size, values, dataType);
  status_t = hipsparseCreateDnVec(&dnVecDescr_t, size, values, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyDnVec(cusparseConstDnVecDescr_t dnVecDescr);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroyDnVec(hipsparseDnVecDescr_t dnVecDescr);
  // CHECK: status_t = hipsparseDestroyDnVec(dnVecDescr_t);
  status_t = hipsparseDestroyDnVec(dnVecDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnVecGet(cusparseDnVecDescr_t dnVecDescr, int64_t* size, void** values, cudaDataType* valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnVecGet(const hipsparseDnVecDescr_t dnVecDescr, int64_t* size, void** values, hipDataType* valueType);
  // CHECK: status_t = hipsparseDnVecGet(dnVecDescr_t, &size, &values, &dataType);
  status_t = hipsparseDnVecGet(dnVecDescr_t, &size, &values, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnVecGetValues(cusparseDnVecDescr_t dnVecDescr, void** values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnVecGetValues(const hipsparseDnVecDescr_t dnVecDescr, void** values);
  // CHECK: status_t = hipsparseDnVecGetValues(dnVecDescr_t, &values);
  status_t = hipsparseDnVecGetValues(dnVecDescr_t, &values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnVecSetValues(cusparseDnVecDescr_t dnVecDescr, void* values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnVecSetValues(hipsparseDnVecDescr_t dnVecDescr, void* values);
  // CHECK: status_t = hipsparseDnVecSetValues(dnVecDescr_t, values);
  status_t = hipsparseDnVecSetValues(dnVecDescr_t, values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatGetValues(cusparseDnMatDescr_t dnMatDescr, void** values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnMatGetValues(const hipsparseDnMatDescr_t dnMatDescr, void** values);
  // CHECK: status_t = hipsparseDnMatGetValues(dnMatDescr_t, &values);
  status_t = hipsparseDnMatGetValues(dnMatDescr_t, &values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatSetValues(cusparseDnMatDescr_t dnMatDescr, void* values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnMatSetValues(hipsparseDnMatDescr_t dnMatDescr, void* values);
  // CHECK: status_t = hipsparseDnMatSetValues(dnMatDescr_t, values);
  status_t = hipsparseDnMatSetValues(dnMatDescr_t, values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMV(cusparseHandle_t handle, cusparseOperation_t opA, const void* alpha, cusparseConstSpMatDescr_t matA, cusparseConstDnVecDescr_t vecX, const void* beta, cusparseDnVecDescr_t vecY, cudaDataType computeType, cusparseSpMVAlg_t alg, void* externalBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMV(hipsparseHandle_t handle, hipsparseOperation_t opA, const void* alpha, const hipsparseSpMatDescr_t matA, const hipsparseDnVecDescr_t vecX, const void* beta, const hipsparseDnVecDescr_t vecY, hipDataType computeType, hipsparseSpMVAlg_t alg, void* externalBuffer);
  // CHECK: status_t = hipsparseSpMV(handle_t, opA, alpha, spMatDescr_t, vecX, beta, vecY, dataType, spMVAlg_t, tempBuffer);
  status_t = hipsparseSpMV(handle_t, opA, alpha, spMatDescr_t, vecX, beta, vecY, dataType, spMVAlg_t, tempBuffer);
#endif

#if CUDA_VERSION >= 10020
  // CHECK: hipsparseStatus_t STATUS_NOT_SUPPORTED = HIPSPARSE_STATUS_NOT_SUPPORTED;
  hipsparseStatus_t STATUS_NOT_SUPPORTED = HIPSPARSE_STATUS_NOT_SUPPORTED;
#endif

#if (CUDA_VERSION >= 10020 && CUDA_VERSION < 11000 && !defined(_WIN32)) || (CUDA_VERSION >= 11000 && CUDA_VERSION < 12000)
  // CHECK: hipsparseFormat_t FORMAT_COO_AOS = HIPSPARSE_FORMAT_COO_AOS;
  cusparseFormat_t FORMAT_COO_AOS = CUSPARSE_FORMAT_COO_AOS;

  // CHECK: hipsparseSpMVAlg_t MV_ALG_DEFAULT = HIPSPARSE_MV_ALG_DEFAULT;
  cusparseSpMVAlg_t MV_ALG_DEFAULT = CUSPARSE_MV_ALG_DEFAULT;

  // CHECK: hipsparseSpMVAlg_t COOMV_ALG = HIPSPARSE_COOMV_ALG;
  // CHECK-NEXT: hipsparseSpMVAlg_t CSRMV_ALG1 = HIPSPARSE_CSRMV_ALG1;
  // CHECK-NEXT: hipsparseSpMVAlg_t CSRMV_ALG2 = HIPSPARSE_CSRMV_ALG2;
  cusparseSpMVAlg_t COOMV_ALG = CUSPARSE_COOMV_ALG;
  cusparseSpMVAlg_t CSRMV_ALG1 = CUSPARSE_CSRMV_ALG1;
  cusparseSpMVAlg_t CSRMV_ALG2 = CUSPARSE_CSRMV_ALG2;

  // CHECK: hipsparseSpMMAlg_t MM_ALG_DEFAULT = HIPSPARSE_MM_ALG_DEFAULT;
  // CHECK: hipsparseSpMMAlg_t CSRMM_ALG1 = HIPSPARSE_CSRMM_ALG1;
  cusparseSpMMAlg_t MM_ALG_DEFAULT = CUSPARSE_MM_ALG_DEFAULT;
  cusparseSpMMAlg_t CSRMM_ALG1 = CUSPARSE_CSRMM_ALG1;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCooAoS(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cooInd, void* cooValues, cusparseIndexType_t cooIdxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateCooAoS(hipsparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cooInd, void* cooValues, hipsparseIndexType_t cooIdxType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateCooAoS(&spMatDescr_t, rows, cols, nnz, cooColInd, cooValues, indexType_t, indexBase_t, dataType);
  status_t = cusparseCreateCooAoS(&spMatDescr_t, rows, cols, nnz, cooColInd, cooValues, indexType_t, indexBase_t, dataType);

  // CUDA: CUSPARSE_DEPRECATED(cusparseCooGet) cusparseStatus_t CUSPARSEAPI cusparseCooAoSGet(cusparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** cooInd, void** cooValues, cusparseIndexType_t* idxType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCooAoSGet(const hipsparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** cooInd, void** cooValues, hipsparseIndexType_t* idxType, hipsparseIndexBase_t* idxBase, hipDataType* valueType);
  // CHECK: status_t = hipsparseCooAoSGet(spMatDescr_t, &rows, &cols, &nnz, &cooColInd, &cooValues, &indexType_t, &indexBase_t, &dataType);
  status_t = cusparseCooAoSGet(spMatDescr_t, &rows, &cols, &nnz, &cooColInd, &cooValues, &indexType_t, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatSetStridedBatch(cusparseSpMatDescr_t spMatDescr, int batchCount);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatSetStridedBatch(hipsparseSpMatDescr_t spMatDescr, int batchCount);
  // CHECK: status_t = hipsparseSpMatSetStridedBatch(spMatDescr_t, batchCount);
  status_t = cusparseSpMatSetStridedBatch(spMatDescr_t, batchCount);
#endif

#if CUDA_VERSION >= 11000 && CUSPARSE_VERSION >= 11100
  // CHECK: hipsparseSpMMAlg_t SPMM_ALG_DEFAULT = HIPSPARSE_SPMM_ALG_DEFAULT;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG1 = HIPSPARSE_SPMM_COO_ALG1;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG2 = HIPSPARSE_SPMM_COO_ALG2;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG3 = HIPSPARSE_SPMM_COO_ALG3;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_CSR_ALG1 = HIPSPARSE_SPMM_CSR_ALG1;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG4 = HIPSPARSE_SPMM_COO_ALG4;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_CSR_ALG2 = HIPSPARSE_SPMM_CSR_ALG2;
  hipsparseSpMMAlg_t SPMM_ALG_DEFAULT = HIPSPARSE_SPMM_ALG_DEFAULT;
  hipsparseSpMMAlg_t SPMM_COO_ALG1 = HIPSPARSE_SPMM_COO_ALG1;
  hipsparseSpMMAlg_t SPMM_COO_ALG2 = HIPSPARSE_SPMM_COO_ALG2;
  hipsparseSpMMAlg_t SPMM_COO_ALG3 = HIPSPARSE_SPMM_COO_ALG3;
  hipsparseSpMMAlg_t SPMM_CSR_ALG1 = HIPSPARSE_SPMM_CSR_ALG1;
  hipsparseSpMMAlg_t SPMM_COO_ALG4 = HIPSPARSE_SPMM_COO_ALG4;
  hipsparseSpMMAlg_t SPMM_CSR_ALG2 = HIPSPARSE_SPMM_CSR_ALG2;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCooSetStridedBatch(cusparseSpMatDescr_t spMatDescr, int batchCount, int64_t batchStride);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCooSetStridedBatch(hipsparseSpMatDescr_t spMatDescr, int batchCount, int64_t batchStride);
  // CHECK: status_t = hipsparseCooSetStridedBatch(spMatDescr_t, batchCount, batchStride);
  status_t = hipsparseCooSetStridedBatch(spMatDescr_t, batchCount, batchStride);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCsrSetStridedBatch(cusparseSpMatDescr_t spMatDescr, int batchCount, int64_t offsetsBatchStride, int64_t columnsValuesBatchStride);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCsrSetStridedBatch(hipsparseSpMatDescr_t spMatDescr, int batchCount, int64_t offsetsBatchStride, int64_t columnsValuesBatchStride);
  // CHECK: status_t = hipsparseCsrSetStridedBatch(spMatDescr_t, batchCount, offsetsBatchStride, columnsValuesBatchStride);
  status_t = hipsparseCsrSetStridedBatch(spMatDescr_t, batchCount, offsetsBatchStride, columnsValuesBatchStride);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseRot(cusparseHandle_t handle, const void* c_coeff, const void* s_coeff, cusparseSpVecDescr_t vecX, cusparseDnVecDescr_t vecY);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseRot(hipsparseHandle_t handle, const void* c_coeff, const void* s_coeff, hipsparseSpVecDescr_t vecX, hipsparseDnVecDescr_t vecY);
  // CHECK: status_t = hipsparseRot(handle_t, c_coeff, s_coeff, spVecDescr_t, vecY);
  status_t = hipsparseRot(handle_t, c_coeff, s_coeff, spVecDescr_t, vecY);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScatter(cusparseHandle_t handle, cusparseConstSpVecDescr_t vecX, cusparseDnVecDescr_t vecY);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseScatter(hipsparseHandle_t handle, hipsparseSpVecDescr_t vecX, hipsparseDnVecDescr_t vecY);
  // CHECK: status_t = hipsparseScatter(handle_t, spVecDescr_t, vecY);
  status_t = hipsparseScatter(handle_t, spVecDescr_t, vecY);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGather(cusparseHandle_t handle, cusparseConstDnVecDescr_t vecY, cusparseSpVecDescr_t vecX);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseGather(hipsparseHandle_t handle, hipsparseDnVecDescr_t vecY, hipsparseSpVecDescr_t vecX);
  // CHECK: status_t = hipsparseGather(handle_t, vecY, spVecDescr_t);
  status_t = hipsparseGather(handle_t, vecY, spVecDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseAxpby(cusparseHandle_t handle, const void* alpha, cusparseConstSpVecDescr_t vecX, const void* beta, cusparseDnVecDescr_t vecY);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseAxpby(hipsparseHandle_t handle, const void* alpha, hipsparseSpVecDescr_t vecX, const void* beta, hipsparseDnVecDescr_t vecY);
  // CHECK: status_t = hipsparseAxpby(handle_t, alpha, spVecDescr_t, beta, vecY);
  status_t = hipsparseAxpby(handle_t, alpha, spVecDescr_t, beta, vecY);
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipsparseStatus_t STATUS_INSUFFICIENT_RESOURCES = HIPSPARSE_STATUS_INSUFFICIENT_RESOURCES;
  hipsparseStatus_t STATUS_INSUFFICIENT_RESOURCES = HIPSPARSE_STATUS_INSUFFICIENT_RESOURCES;

  // CHECK: hipsparseSpGEMMAlg_t spGEMMAlg_t;
  // CHECK-NEXT: hipsparseSpGEMMAlg_t SPGEMM_DEFAULT = HIPSPARSE_SPGEMM_DEFAULT;
  hipsparseSpGEMMAlg_t spGEMMAlg_t;
  hipsparseSpGEMMAlg_t SPGEMM_DEFAULT = HIPSPARSE_SPGEMM_DEFAULT;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCsrSetPointers(cusparseSpMatDescr_t spMatDescr, void* csrRowOffsets, void* csrColInd, void* csrValues);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCsrSetPointers(hipsparseSpMatDescr_t spMatDescr, void* csrRowOffsets, void* csrColInd, void* csrValues);
  // CHECK: status_t = hipsparseCsrSetPointers(spMatDescr_t, csrRowOffsets, csrColInd, csrValues);
  status_t = hipsparseCsrSetPointers(spMatDescr_t, csrRowOffsets, csrColInd, csrValues);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetSize(cusparseConstSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatGetSize(hipsparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz);
  // CHECK: status_t = hipsparseSpMatGetSize(spMatDescr_t, &rows, &cols, &nnz);
  status_t = hipsparseSpMatGetSize(spMatDescr_t, &rows, &cols, &nnz);
#endif

#if CUDA_VERSION < 11000
  // CHECK: hipsparseHybMat_t hybMat_t;
  cusparseHybMat_t hybMat_t;

  // CHECK: hipsparseHybPartition_t hybPartition_t;
  // CHECK-NEXT: hipsparseHybPartition_t HYB_PARTITION_AUTO = HIPSPARSE_HYB_PARTITION_AUTO;
  // CHECK-NEXT: hipsparseHybPartition_t HYB_PARTITION_USER = HIPSPARSE_HYB_PARTITION_USER;
  // CHECK-NEXT: hipsparseHybPartition_t HYB_PARTITION_MAX = HIPSPARSE_HYB_PARTITION_MAX;
  cusparseHybPartition_t hybPartition_t;
  cusparseHybPartition_t HYB_PARTITION_AUTO = CUSPARSE_HYB_PARTITION_AUTO;
  cusparseHybPartition_t HYB_PARTITION_USER = CUSPARSE_HYB_PARTITION_USER;
  cusparseHybPartition_t HYB_PARTITION_MAX = CUSPARSE_HYB_PARTITION_MAX;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateHybMat(cusparseHybMat_t* hybA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateHybMat(hipsparseHybMat_t* hybA);
  // CHECK: status_t = hipsparseCreateHybMat(&hybMat_t);
  status_t = cusparseCreateHybMat(&hybMat_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyHybMat(cusparseHybMat_t hybA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroyHybMat(hipsparseHybMat_t hybA);
  // CHECK: status_t = hipsparseDestroyHybMat(hybMat_t);
  status_t = cusparseDestroyHybMat(hybMat_t);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseZhyb2csr(cusparseHandle_t handle, const cusparseMatDescr_t descrA, const cusparseHybMat_t hybA, cuDoubleComplex* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZhyb2csr(hipsparseHandle_t handle, const hipsparseMatDescr_t descrA, const hipsparseHybMat_t hybA, hipDoubleComplex* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // CHECK: status_t = hipsparseZhyb2csr(handle_t, matDescr_t, hybMat_t, &dComplexbsrSortedValA, &csrRowPtrA, &csrColIndA);
  status_t = cusparseZhyb2csr(handle_t, matDescr_t, hybMat_t, &dComplexbsrSortedValA, &csrRowPtrA, &csrColIndA);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseChyb2csr(cusparseHandle_t handle, const cusparseMatDescr_t descrA, const cusparseHybMat_t hybA, cuComplex* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseChyb2csr(hipsparseHandle_t handle, const hipsparseMatDescr_t descrA, const hipsparseHybMat_t hybA, hipComplex* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // CHECK: status_t = hipsparseChyb2csr(handle_t, matDescr_t, hybMat_t, &complex, &csrRowPtrA, &csrColIndA);
  status_t = cusparseChyb2csr(handle_t, matDescr_t, hybMat_t, &complex, &csrRowPtrA, &csrColIndA);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDhyb2csr(cusparseHandle_t handle, const cusparseMatDescr_t descrA, const cusparseHybMat_t hybA, double* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDhyb2csr(hipsparseHandle_t handle, const hipsparseMatDescr_t descrA, const hipsparseHybMat_t hybA, double* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // CHECK: status_t = hipsparseDhyb2csr(handle_t, matDescr_t, hybMat_t, &csrValA, &csrRowPtrA, &csrColIndA);
  status_t = cusparseDhyb2csr(handle_t, matDescr_t, hybMat_t, &csrValA, &csrRowPtrA, &csrColIndA);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseShyb2csr(cusparseHandle_t handle, const cusparseMatDescr_t descrA, const cusparseHybMat_t hybA, float* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseShyb2csr(hipsparseHandle_t handle, const hipsparseMatDescr_t descrA, const hipsparseHybMat_t hybA, float* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // CHECK: status_t = hipsparseShyb2csr(handle_t, matDescr_t, hybMat_t, &fcsrValA, &csrRowPtrA, &csrColIndA);
  status_t = cusparseShyb2csr(handle_t, matDescr_t, hybMat_t, &fcsrValA, &csrRowPtrA, &csrColIndA);
#endif

#if CUDA_VERSION >= 11010 && CUSPARSE_VERSION >= 11300
  // CHECK: hipsparseSparseToDenseAlg_t sparseToDenseAlg_t;
  // CHECK-NEXT: hipsparseSparseToDenseAlg_t SPARSETODENSE_ALG_DEFAULT = HIPSPARSE_SPARSETODENSE_ALG_DEFAULT;
  hipsparseSparseToDenseAlg_t sparseToDenseAlg_t;
  hipsparseSparseToDenseAlg_t SPARSETODENSE_ALG_DEFAULT = HIPSPARSE_SPARSETODENSE_ALG_DEFAULT;

  // CHECK: hipsparseDenseToSparseAlg_t denseToSparseAlg_t;
  // CHECK-NEXT: hipsparseDenseToSparseAlg_t DENSETOSPARSE_ALG_DEFAULT = HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT;
  hipsparseDenseToSparseAlg_t denseToSparseAlg_t;
  hipsparseDenseToSparseAlg_t DENSETOSPARSE_ALG_DEFAULT = HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCsc(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cscColOffsets, void* cscRowInd, void* cscValues, cusparseIndexType_t cscColOffsetsType, cusparseIndexType_t cscRowIndType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateCsc(hipsparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cscColOffsets, void* cscRowInd, void* cscValues, hipsparseIndexType_t cscColOffsetsType, hipsparseIndexType_t cscRowIndType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateCsc(&spMatDescr_t, rows, cols, nnz, cscColOffsets, cscRowInd, cscValues, cscColOffsetsType, csrColIndType, indexBase_t, dataType);
  status_t = hipsparseCreateCsc(&spMatDescr_t, rows, cols, nnz, cscColOffsets, cscRowInd, cscValues, cscColOffsetsType, csrColIndType, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCooSetPointers(cusparseSpMatDescr_t spMatDescr, void* cooRows, void* cooColumns, void* cooValues);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCooSetPointers(hipsparseSpMatDescr_t spMatDescr, void* cooRowInd, void* cooColInd, void* cooValues);
  // CHECK: status_t = hipsparseCooSetPointers(spMatDescr_t, cooRows, cooColumns, cooValues);
  status_t = hipsparseCooSetPointers(spMatDescr_t, cooRows, cooColumns, cooValues);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCscSetPointers(cusparseSpMatDescr_t spMatDescr, void* cscColOffsets, void* cscRowInd, void* cscValues);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCscSetPointers(hipsparseSpMatDescr_t spMatDescr, void* cscColOffsets, void* cscRowInd, void* cscValues);
  // CHECK: status_t = hipsparseCscSetPointers(spMatDescr_t, cscColOffsets, cscRowInd, cscValues);
  status_t = hipsparseCscSetPointers(spMatDescr_t, cscColOffsets, cscRowInd, cscValues);
#endif

#if CUDA_VERSION >= 11020 && CUSPARSE_VERSION >= 11400
  // CHECK: hipsparseFormat_t FORMAT_BLOCKED_ELL = HIPSPARSE_FORMAT_BLOCKED_ELL;
  hipsparseFormat_t FORMAT_BLOCKED_ELL = HIPSPARSE_FORMAT_BLOCKED_ELL;

  // CHECK: hipsparseSpMVAlg_t SPMV_ALG_DEFAULT = HIPSPARSE_SPMV_ALG_DEFAULT;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_COO_ALG1 = HIPSPARSE_SPMV_COO_ALG1;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_COO_ALG2 = HIPSPARSE_SPMV_COO_ALG2;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_CSR_ALG1 = HIPSPARSE_SPMV_CSR_ALG1;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_CSR_ALG2 = HIPSPARSE_SPMV_CSR_ALG2;
  hipsparseSpMVAlg_t SPMV_ALG_DEFAULT = HIPSPARSE_SPMV_ALG_DEFAULT;
  hipsparseSpMVAlg_t SPMV_COO_ALG1 = HIPSPARSE_SPMV_COO_ALG1;
  hipsparseSpMVAlg_t SPMV_COO_ALG2 = HIPSPARSE_SPMV_COO_ALG2;
  hipsparseSpMVAlg_t SPMV_CSR_ALG1 = HIPSPARSE_SPMV_CSR_ALG1;
  hipsparseSpMVAlg_t SPMV_CSR_ALG2 = HIPSPARSE_SPMV_CSR_ALG2;

  // CHECK: hipsparseSpMMAlg_t SPMM_CSR_ALG3 = HIPSPARSE_SPMM_CSR_ALG3;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_BLOCKED_ELL_ALG1 = HIPSPARSE_SPMM_BLOCKED_ELL_ALG1;
  hipsparseSpMMAlg_t SPMM_CSR_ALG3 = HIPSPARSE_SPMM_CSR_ALG3;
  hipsparseSpMMAlg_t SPMM_BLOCKED_ELL_ALG1 = HIPSPARSE_SPMM_BLOCKED_ELL_ALG1;

  // CHECK: hipsparseSDDMMAlg_t sDDMMAlg_t;
  // CHECK-NEXT: hipsparseSDDMMAlg_t SDDMM_ALG_DEFAULT = HIPSPARSE_SDDMM_ALG_DEFAULT;
  hipsparseSDDMMAlg_t sDDMMAlg_t;
  hipsparseSDDMMAlg_t SDDMM_ALG_DEFAULT = HIPSPARSE_SDDMM_ALG_DEFAULT;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateBlockedEll(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t ellBlockSize, int64_t ellCols, void* ellColInd, void* ellValue, cusparseIndexType_t ellIdxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateBlockedEll(hipsparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t ellBlockSize, int64_t ellCols, void* ellColInd, void* ellValue, hipsparseIndexType_t ellIdxType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateBlockedEll(&spMatDescr_t, rows, cols, ellBlockSize, ellCols, ellColInd, ellValue, ellIdxType, indexBase_t, dataType);
  status_t = hipsparseCreateBlockedEll(&spMatDescr_t, rows, cols, ellBlockSize, ellCols, ellColInd, ellValue, ellIdxType, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseBlockedEllGet(cusparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* ellBlockSize, int64_t* ellCols, void** ellColInd, void** ellValue, cusparseIndexType_t* ellIdxType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseBlockedEllGet(const hipsparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* ellBlockSize, int64_t* ellCols, void** ellColInd, void** ellValue, hipsparseIndexType_t* ellIdxType, hipsparseIndexBase_t* idxBase, hipDataType* valueType);
  // CHECK: status_t = hipsparseBlockedEllGet(spMatDescr_t, &rows, &cols, &ellBlockSize, &ellCols, &ellColInd, &ellValue, &ellIdxType, &indexBase_t, &dataType);
  status_t = hipsparseBlockedEllGet(spMatDescr_t, &rows, &cols, &ellBlockSize, &ellCols, &ellColInd, &ellValue, &ellIdxType, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSDDMM_preprocess(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstDnMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseSpMatDescr_t matC, cudaDataType computeType, cusparseSDDMMAlg_t alg, void* externalBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSDDMM_preprocess(hipsparseHandle_t handle, hipsparseOperation_t opA, hipsparseOperation_t opB, const void* alpha, const hipsparseDnMatDescr_t A, const hipsparseDnMatDescr_t B, const void* beta, hipsparseSpMatDescr_t C, hipDataType computeType, hipsparseSDDMMAlg_t alg, void* tempBuffer);
  // CHECK: status_t = hipsparseSDDMM_preprocess(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, tempBuffer);
 status_t = hipsparseSDDMM_preprocess(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSDDMM_bufferSize(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstDnMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseSpMatDescr_t matC, cudaDataType computeType, cusparseSDDMMAlg_t alg, size_t* bufferSize);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSDDMM_bufferSize(hipsparseHandle_t handle, hipsparseOperation_t opA, hipsparseOperation_t opB, const void* alpha, const hipsparseDnMatDescr_t A, const hipsparseDnMatDescr_t B, const void* beta, hipsparseSpMatDescr_t C, hipDataType computeType, hipsparseSDDMMAlg_t alg, size_t* bufferSize);
  // CHECK: status_t = hipsparseSDDMM_bufferSize(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, &bufferSize);
 status_t = hipsparseSDDMM_bufferSize(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSDDMM(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstDnMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseSpMatDescr_t matC, cudaDataType computeType, cusparseSDDMMAlg_t alg, void* externalBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSDDMM(hipsparseHandle_t handle, hipsparseOperation_t opA, hipsparseOperation_t opB, const void* alpha, const hipsparseDnMatDescr_t A, const hipsparseDnMatDescr_t B, const void* beta, hipsparseSpMatDescr_t C, hipDataType computeType, hipsparseSDDMMAlg_t alg, void* tempBuffer);
  // CHECK: status_t = hipsparseSDDMM(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, tempBuffer);
 status_t = hipsparseSDDMM(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, tempBuffer);
#endif

#if CUDA_VERSION >= 11030
  // CHECK: hipsparseSpMatAttribute_t spMatAttribute_t;
  // CHECK-NEXT: hipsparseSpMatAttribute_t SPMAT_FILL_MODE = HIPSPARSE_SPMAT_FILL_MODE;
  // CHECK-NEXT: hipsparseSpMatAttribute_t SPMAT_DIAG_TYPE = HIPSPARSE_SPMAT_DIAG_TYPE;
  hipsparseSpMatAttribute_t spMatAttribute_t;
  hipsparseSpMatAttribute_t SPMAT_FILL_MODE = HIPSPARSE_SPMAT_FILL_MODE;
  hipsparseSpMatAttribute_t SPMAT_DIAG_TYPE = HIPSPARSE_SPMAT_DIAG_TYPE;

  // CHECK: hipsparseSpSVAlg_t spSVAlg_t;
  // CHECK-NEXT: hipsparseSpSVAlg_t SPSV_ALG_DEFAULT = HIPSPARSE_SPSV_ALG_DEFAULT;
  hipsparseSpSVAlg_t spSVAlg_t;
  hipsparseSpSVAlg_t SPSV_ALG_DEFAULT = HIPSPARSE_SPSV_ALG_DEFAULT;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetAttribute(cusparseConstSpMatDescr_t spMatDescr, cusparseSpMatAttribute_t attribute, void* data, size_t dataSize);
  // HIP: hipsparseStatus_t hipsparseSpMatGetAttribute(hipsparseSpMatDescr_t spMatDescr, hipsparseSpMatAttribute_t attribute, void* data, size_t dataSize);
  // CHECK: status_t = hipsparseSpMatGetAttribute(spMatDescr_t, spMatAttribute_t, &data, dataSize);
  status_t = hipsparseSpMatGetAttribute(spMatDescr_t, spMatAttribute_t, &data, dataSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatSetAttribute(cusparseSpMatDescr_t spMatDescr, cusparseSpMatAttribute_t attribute, void* data, size_t dataSize);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatSetAttribute(hipsparseSpMatDescr_t spMatDescr, hipsparseSpMatAttribute_t attribute, const void* data, size_t dataSize);
  // CHECK: status_t = hipsparseSpMatSetAttribute(spMatDescr_t, spMatAttribute_t, &data, dataSize);
  status_t = hipsparseSpMatSetAttribute(spMatDescr_t, spMatAttribute_t, &data, dataSize);
#endif

#if CUDA_VERSION >= 11030 && CUSPARSE_VERSION >= 11600
  // CHECK: hipsparseSpSMAlg_t spSMAlg_t;
  // CHECK-NEXT: hipsparseSpSMAlg_t SPSM_ALG_DEFAULT = HIPSPARSE_SPSM_ALG_DEFAULT;
  hipsparseSpSMAlg_t spSMAlg_t;
  hipsparseSpSMAlg_t SPSM_ALG_DEFAULT = HIPSPARSE_SPSM_ALG_DEFAULT;

  // CHECK: hipsparseSpGEMMAlg_t SPGEMM_CSR_ALG_DETERMINITIC = HIPSPARSE_SPGEMM_CSR_ALG_DETERMINISTIC;
  // CHECK-NEXT: hipsparseSpGEMMAlg_t SPGEMM_CSR_ALG_NONDETERMINITIC = HIPSPARSE_SPGEMM_CSR_ALG_NONDETERMINISTIC;
  hipsparseSpGEMMAlg_t SPGEMM_CSR_ALG_DETERMINITIC = HIPSPARSE_SPGEMM_CSR_ALG_DETERMINISTIC;
  hipsparseSpGEMMAlg_t SPGEMM_CSR_ALG_NONDETERMINITIC = HIPSPARSE_SPGEMM_CSR_ALG_NONDETERMINISTIC;
#endif

#if CUDA_VERSION >= 12000
  // CHECK: hipsparseCsr2CscAlg_t CSR2CSC_ALG_DEFAULT = HIPSPARSE_CSR2CSC_ALG_DEFAULT;
  hipsparseCsr2CscAlg_t CSR2CSC_ALG_DEFAULT = HIPSPARSE_CSR2CSC_ALG_DEFAULT;

  // CHECK: hipsparseSpGEMMAlg_t SPGEMM_ALG1 = HIPSPARSE_SPGEMM_ALG1;
  // CHECK: hipsparseSpGEMMAlg_t SPGEMM_ALG2 = HIPSPARSE_SPGEMM_ALG2;
  // CHECK: hipsparseSpGEMMAlg_t SPGEMM_ALG3 = HIPSPARSE_SPGEMM_ALG3;
  hipsparseSpGEMMAlg_t SPGEMM_ALG1 = HIPSPARSE_SPGEMM_ALG1;
  hipsparseSpGEMMAlg_t SPGEMM_ALG2 = HIPSPARSE_SPGEMM_ALG2;
  hipsparseSpGEMMAlg_t SPGEMM_ALG3 = HIPSPARSE_SPGEMM_ALG3;
#endif

  return 0;
}
