// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "hipsparse.h"

int main() {
  printf("17. cuSPARSE API to hipSPARSE API synthetic test\n");

  // CHECK: hipsparseHandle_t handle_t;
  hipsparseHandle_t handle_t;

  // CHECK: hipsparseMatDescr_t matDescr_t, matDescr_t_2;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2;

  // CHECK: hipsparseColorInfo_t colorInfo_t;
  hipsparseColorInfo_t colorInfo_t;

  // CHECK: hipsparseOperation_t sparseOperation_t;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_NON_TRANSPOSE = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_TRANSPOSE = HIPSPARSE_OPERATION_TRANSPOSE;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_CONJUGATE_TRANSPOSE = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
  hipsparseOperation_t sparseOperation_t;
  hipsparseOperation_t OPERATION_NON_TRANSPOSE = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipsparseOperation_t OPERATION_TRANSPOSE = HIPSPARSE_OPERATION_TRANSPOSE;
  hipsparseOperation_t OPERATION_CONJUGATE_TRANSPOSE = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;

  // CHECK: hipsparseIndexBase_t indexBase_t;
  // CHECK-NEXT: hipsparseIndexBase_t INDEX_BASE_ZERO = HIPSPARSE_INDEX_BASE_ZERO;
  // CHECK-NEXT: hipsparseIndexBase_t INDEX_BASE_ONE = HIPSPARSE_INDEX_BASE_ONE;
  hipsparseIndexBase_t indexBase_t;
  hipsparseIndexBase_t INDEX_BASE_ZERO = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseIndexBase_t INDEX_BASE_ONE = HIPSPARSE_INDEX_BASE_ONE;

  // CHECK: hipsparseMatrixType_t matrixType_t;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_GENERAL = HIPSPARSE_MATRIX_TYPE_GENERAL;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_SYMMETRIC = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_HERMITIAN = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_TRIANGULAR = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;
  hipsparseMatrixType_t matrixType_t;
  hipsparseMatrixType_t MATRIX_TYPE_GENERAL = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseMatrixType_t MATRIX_TYPE_SYMMETRIC = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  hipsparseMatrixType_t MATRIX_TYPE_HERMITIAN = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  hipsparseMatrixType_t MATRIX_TYPE_TRIANGULAR = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;

  // CHECK: hipsparseDiagType_t diagType_t;
  // CHECK-NEXT: hipsparseDiagType_t DIAG_TYPE_NON_UNIT = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  // CHECK-NEXT: hipsparseDiagType_t DIAG_TYPE_UNIT = HIPSPARSE_DIAG_TYPE_UNIT;
  hipsparseDiagType_t diagType_t;
  hipsparseDiagType_t DIAG_TYPE_NON_UNIT = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseDiagType_t DIAG_TYPE_UNIT = HIPSPARSE_DIAG_TYPE_UNIT;

  // CHECK: hipsparseFillMode_t fillMode_t;
  // CHECK-NEXT: hipsparseFillMode_t FILL_MODE_LOWER = HIPSPARSE_FILL_MODE_LOWER;
  // CHECK-NEXT: hipsparseFillMode_t FILL_MODE_UPPER = HIPSPARSE_FILL_MODE_UPPER;
  hipsparseFillMode_t fillMode_t;
  hipsparseFillMode_t FILL_MODE_LOWER = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseFillMode_t FILL_MODE_UPPER = HIPSPARSE_FILL_MODE_UPPER;

  // CHECK: hipsparseAction_t action_t;
  // CHECK-NEXT: hipsparseAction_t ACTION_SYMBOLIC = HIPSPARSE_ACTION_SYMBOLIC;
  // CHECK-NEXT: hipsparseAction_t ACTION_NUMERIC = HIPSPARSE_ACTION_NUMERIC;
  hipsparseAction_t action_t;
  hipsparseAction_t ACTION_SYMBOLIC = HIPSPARSE_ACTION_SYMBOLIC;
  hipsparseAction_t ACTION_NUMERIC = HIPSPARSE_ACTION_NUMERIC;

  // CHECK: hipsparseDirection_t direction_t;
  // CHECK-NEXT: hipsparseDirection_t DIRECTION_ROW = HIPSPARSE_DIRECTION_ROW;
  // CHECK-NEXT: hipsparseDirection_t DIRECTION_COLUMN = HIPSPARSE_DIRECTION_COLUMN;
  hipsparseDirection_t direction_t;
  hipsparseDirection_t DIRECTION_ROW = HIPSPARSE_DIRECTION_ROW;
  hipsparseDirection_t DIRECTION_COLUMN = HIPSPARSE_DIRECTION_COLUMN;

  // CHECK: hipsparseSolvePolicy_t solvePolicy_t;
  // CHECK-NEXT: hipsparseSolvePolicy_t SOLVE_POLICY_NO_LEVEL = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
  // CHECK-NEXT: hipsparseSolvePolicy_t SOLVE_POLICY_USE_LEVEL = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
  hipsparseSolvePolicy_t solvePolicy_t;
  hipsparseSolvePolicy_t SOLVE_POLICY_NO_LEVEL = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
  hipsparseSolvePolicy_t SOLVE_POLICY_USE_LEVEL = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

  // CHECK: hipsparsePointerMode_t pointerMode_t;
  // CHECK-NEXT: hipsparsePointerMode_t POINTER_MODE_HOST = HIPSPARSE_POINTER_MODE_HOST;
  // CHECK-NEXT: hipsparsePointerMode_t POINTER_MODE_DEVICE = HIPSPARSE_POINTER_MODE_DEVICE;
  hipsparsePointerMode_t pointerMode_t;
  hipsparsePointerMode_t POINTER_MODE_HOST = HIPSPARSE_POINTER_MODE_HOST;
  hipsparsePointerMode_t POINTER_MODE_DEVICE = HIPSPARSE_POINTER_MODE_DEVICE;

  // CHECK: hipsparseStatus_t status_t;
  // CHECK-NEXT: hipsparseStatus_t STATUS_SUCCESS = HIPSPARSE_STATUS_SUCCESS;
  // CHECK-NEXT: hipsparseStatus_t STATUS_NOT_INITIALIZED = HIPSPARSE_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_ALLOC_FAILED = HIPSPARSE_STATUS_ALLOC_FAILED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_INVALID_VALUE = HIPSPARSE_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipsparseStatus_t STATUS_ARCH_MISMATCH = HIPSPARSE_STATUS_ARCH_MISMATCH;
  // CHECK-NEXT: hipsparseStatus_t STATUS_MAPPING_ERROR = HIPSPARSE_STATUS_MAPPING_ERROR;
  // CHECK-NEXT: hipsparseStatus_t STATUS_EXECUTION_FAILED = HIPSPARSE_STATUS_EXECUTION_FAILED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_INTERNAL_ERROR = HIPSPARSE_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipsparseStatus_t STATUS_MATRIX_TYPE_NOT_SUPPORTED = HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_ZERO_PIVOT = HIPSPARSE_STATUS_ZERO_PIVOT;
  hipsparseStatus_t status_t;
  hipsparseStatus_t STATUS_SUCCESS = HIPSPARSE_STATUS_SUCCESS;
  hipsparseStatus_t STATUS_NOT_INITIALIZED = HIPSPARSE_STATUS_NOT_INITIALIZED;
  hipsparseStatus_t STATUS_ALLOC_FAILED = HIPSPARSE_STATUS_ALLOC_FAILED;
  hipsparseStatus_t STATUS_INVALID_VALUE = HIPSPARSE_STATUS_INVALID_VALUE;
  hipsparseStatus_t STATUS_ARCH_MISMATCH = HIPSPARSE_STATUS_ARCH_MISMATCH;
  hipsparseStatus_t STATUS_MAPPING_ERROR = HIPSPARSE_STATUS_MAPPING_ERROR;
  hipsparseStatus_t STATUS_EXECUTION_FAILED = HIPSPARSE_STATUS_EXECUTION_FAILED;
  hipsparseStatus_t STATUS_INTERNAL_ERROR = HIPSPARSE_STATUS_INTERNAL_ERROR;
  hipsparseStatus_t STATUS_MATRIX_TYPE_NOT_SUPPORTED = HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED;
  hipsparseStatus_t STATUS_ZERO_PIVOT = HIPSPARSE_STATUS_ZERO_PIVOT;

  // CHECK: hipStream_t stream_t;
  hipStream_t stream_t;

  int iVal = 0;
  int64_t size = 0;
  int64_t nnz = 0;
  int64_t rows = 0;
  int64_t cols = 0;
  void *indices = nullptr;
  void *values = nullptr;
  void *cooRowInd = nullptr;
  void *cscRowInd = nullptr;
  void *csrColInd = nullptr;
  void *cooColInd = nullptr;
  void *cooValues = nullptr;
  void *csrValues = nullptr;
  void *cscValues = nullptr;
  void *csrRowOffsets = nullptr;
  void *cscColOffsets = nullptr;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreate(cusparseHandle_t* handle);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreate(hipsparseHandle_t* handle);
  // CHECK: status_t = hipsparseCreate(&handle_t);
  status_t = hipsparseCreate(&handle_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroy(cusparseHandle_t handle);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroy(hipsparseHandle_t handle);
  // CHECK: status_t = hipsparseDestroy(handle_t);
  status_t = hipsparseDestroy(handle_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetStream(cusparseHandle_t handle, cudaStream_t streamId);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetStream(hipsparseHandle_t handle, hipStream_t streamId);
  // CHECK: status_t = hipsparseSetStream(handle_t, stream_t);
  status_t = hipsparseSetStream(handle_t, stream_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetStream(cusparseHandle_t handle, cudaStream_t* streamId);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseGetStream(hipsparseHandle_t handle, hipStream_t* streamId);
  // CHECK: status_t = hipsparseGetStream(handle_t, &stream_t);
  status_t = hipsparseGetStream(handle_t, &stream_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetPointerMode(cusparseHandle_t handle, cusparsePointerMode_t mode);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetPointerMode(hipsparseHandle_t handle, hipsparsePointerMode_t mode);
  // CHECK: status_t = hipsparseSetPointerMode(handle_t, pointerMode_t);
  status_t = hipsparseSetPointerMode(handle_t, pointerMode_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetPointerMode(cusparseHandle_t handle, cusparsePointerMode_t* mode);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseGetPointerMode(hipsparseHandle_t handle, hipsparsePointerMode_t* mode);
  // CHECK: status_t = hipsparseGetPointerMode(handle_t, &pointerMode_t);
  status_t = hipsparseGetPointerMode(handle_t, &pointerMode_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetVersion(cusparseHandle_t handle, int* version);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseGetVersion(hipsparseHandle_t handle, int* version);
  // CHECK: status_t = hipsparseGetVersion(handle_t, &iVal);
  status_t = hipsparseGetVersion(handle_t, &iVal);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateMatDescr(cusparseMatDescr_t* descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateMatDescr(hipsparseMatDescr_t* descrA);
  // CHECK: status_t = hipsparseCreateMatDescr(&matDescr_t);
  status_t = hipsparseCreateMatDescr(&matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyMatDescr(cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroyMatDescr(hipsparseMatDescr_t descrA);
  // CHECK: status_t = hipsparseDestroyMatDescr(matDescr_t);
  status_t = hipsparseDestroyMatDescr(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatIndexBase(cusparseMatDescr_t descrA, cusparseIndexBase_t base);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetMatIndexBase(hipsparseMatDescr_t descrA, hipsparseIndexBase_t base);
  // CHECK: status_t = hipsparseSetMatIndexBase(matDescr_t, indexBase_t);
  status_t = hipsparseSetMatIndexBase(matDescr_t, indexBase_t);

  // CUDA: cusparseIndexBase_t CUSPARSEAPI cusparseGetMatIndexBase(const cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseIndexBase_t hipsparseGetMatIndexBase(const hipsparseMatDescr_t descrA);
  // CHECK: indexBase_t = hipsparseGetMatIndexBase(matDescr_t);
  indexBase_t = hipsparseGetMatIndexBase(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatType(cusparseMatDescr_t descrA, cusparseMatrixType_t type);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetMatType(hipsparseMatDescr_t descrA, hipsparseMatrixType_t type);
  // CHECK: status_t = hipsparseSetMatType(matDescr_t, matrixType_t);
  status_t = hipsparseSetMatType(matDescr_t, matrixType_t);

  // CUDA: cusparseMatrixType_t CUSPARSEAPI cusparseGetMatType(const cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseMatrixType_t hipsparseGetMatType(const hipsparseMatDescr_t descrA);
  // CHECK: matrixType_t = hipsparseGetMatType(matDescr_t);
  matrixType_t = hipsparseGetMatType(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatFillMode(cusparseMatDescr_t descrA, cusparseFillMode_t fillMode);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetMatFillMode(hipsparseMatDescr_t descrA, hipsparseFillMode_t fillMode);
  // CHECK: status_t = hipsparseSetMatFillMode(matDescr_t, fillMode_t);
  status_t = hipsparseSetMatFillMode(matDescr_t, fillMode_t);

  // CUDA: cusparseFillMode_t CUSPARSEAPI cusparseGetMatFillMode(const cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseFillMode_t hipsparseGetMatFillMode(const hipsparseMatDescr_t descrA);
  // CHECK: fillMode_t = hipsparseGetMatFillMode(matDescr_t);
  fillMode_t = hipsparseGetMatFillMode(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatDiagType(cusparseMatDescr_t descrA, cusparseDiagType_t diagType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetMatDiagType(hipsparseMatDescr_t descrA, hipsparseDiagType_t diagType);
  // CHECK: status_t = hipsparseSetMatDiagType(matDescr_t, diagType_t);
  status_t = hipsparseSetMatDiagType(matDescr_t, diagType_t);

  // CUDA: cusparseDiagType_t CUSPARSEAPI cusparseGetMatDiagType(const cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseDiagType_t hipsparseGetMatDiagType(const hipsparseMatDescr_t descrA);
  // CHECK: diagType_t = hipsparseGetMatDiagType(matDescr_t);
  diagType_t = hipsparseGetMatDiagType(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateColorInfo(cusparseColorInfo_t* info);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateColorInfo(hipsparseColorInfo_t* info);
  // CHECK: status_t = hipsparseCreateColorInfo(&colorInfo_t);
  status_t = hipsparseCreateColorInfo(&colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyColorInfo(cusparseColorInfo_t info);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroyColorInfo(hipsparseColorInfo_t info);
  // CHECK: status_t = hipsparseDestroyColorInfo(colorInfo_t);
  status_t = hipsparseDestroyColorInfo(colorInfo_t);

#if CUDA_VERSION >= 8000
  // CHECK: hipDataType dataType_t;
  // CHECK-NEXT: hipDataType dataType;
  hipDataType dataType_t;
  hipDataType dataType;
#endif

#if CUDA_VERSION >= 8000 && CUDA_VERSION < 12000
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCopyMatDescr(cusparseMatDescr_t dest, const cusparseMatDescr_t src);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCopyMatDescr(hipsparseMatDescr_t dest, const hipsparseMatDescr_t src);
  // CHECK: status_t = hipsparseCopyMatDescr(matDescr_t, matDescr_t_2);
  status_t = cusparseCopyMatDescr(matDescr_t, matDescr_t_2);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipsparseSpMatDescr_t spMatDescr_t;
  hipsparseSpMatDescr_t spMatDescr_t;

  // CHECK: hipsparseDnMatDescr_t dnMatDescr_t;
  hipsparseDnMatDescr_t dnMatDescr_t;

  // CHECK: hipsparseIndexType_t indexType_t;
  // CHECK-NEXT: hipsparseIndexType_t csrRowOffsetsType;
  // CHECK-NEXT: hipsparseIndexType_t cscColOffsetsType;
  // CHECK-NEXT: hipsparseIndexType_t cscRowIndType;
  // CHECK-NEXT: hipsparseIndexType_t csrColIndType;
  // CHECK-NEXT: hipsparseIndexType_t INDEX_16U = HIPSPARSE_INDEX_16U;
  // CHECK-NEXT: hipsparseIndexType_t INDEX_32I = HIPSPARSE_INDEX_32I;
  // CHECK-NEXT: hipsparseIndexType_t INDEX_64I = HIPSPARSE_INDEX_64I;
  hipsparseIndexType_t indexType_t;
  hipsparseIndexType_t csrRowOffsetsType;
  hipsparseIndexType_t cscColOffsetsType;
  hipsparseIndexType_t cscRowIndType;
  hipsparseIndexType_t csrColIndType;
  hipsparseIndexType_t INDEX_16U = HIPSPARSE_INDEX_16U;
  hipsparseIndexType_t INDEX_32I = HIPSPARSE_INDEX_32I;
  hipsparseIndexType_t INDEX_64I = HIPSPARSE_INDEX_64I;

  // CHECK: hipsparseFormat_t format_t;
  // CHECK-NEXT: hipsparseFormat_t FORMAT_CSR = HIPSPARSE_FORMAT_CSR;
  // CHECK-NEXT: hipsparseFormat_t FORMAT_CSC = HIPSPARSE_FORMAT_CSC;
  // CHECK-NEXT: hipsparseFormat_t FORMAT_CSO = HIPSPARSE_FORMAT_COO;
  hipsparseFormat_t format_t;
  hipsparseFormat_t FORMAT_CSR = HIPSPARSE_FORMAT_CSR;
  hipsparseFormat_t FORMAT_CSC = HIPSPARSE_FORMAT_CSC;
  hipsparseFormat_t FORMAT_CSO = HIPSPARSE_FORMAT_COO;

  // CHECK: hipsparseOrder_t order_t;
  // CHECK-NEXT: hipsparseOrder_t ORDER_COL = HIPSPARSE_ORDER_COL;
  // CHECK-NEXT: hipsparseOrder_t ORDER_ROW = HIPSPARSE_ORDER_ROW;
  hipsparseOrder_t order_t;
  hipsparseOrder_t ORDER_COL = HIPSPARSE_ORDER_COL;
  hipsparseOrder_t ORDER_ROW = HIPSPARSE_ORDER_ROW;

  // CHECK: hipsparseSpMMAlg_t spMMAlg_t;
  hipsparseSpMMAlg_t spMMAlg_t;

  // CHECK: hipsparseCsr2CscAlg_t Csr2CscAlg_t;
  // CHECK-NEXT: hipsparseCsr2CscAlg_t CSR2CSC_ALG1 = HIPSPARSE_CSR2CSC_ALG1;
  hipsparseCsr2CscAlg_t Csr2CscAlg_t;
  hipsparseCsr2CscAlg_t CSR2CSC_ALG1 = HIPSPARSE_CSR2CSC_ALG1;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCoo(cusparseSpMatDescr_t* spMatDescr, int64_t ows, int64_t cols, int64_t nnz, void* cooRowInd, void* cooColInd, void* cooValues, cusparseIndexType_t cooIdxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateCoo(hipsparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cooRowInd, void* cooColInd, void* cooValues, hipsparseIndexType_t cooIdxType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateCoo(&spMatDescr_t, rows, cols, nnz, cooRowInd, cooColInd, cooValues, indexType_t, indexBase_t, dataType);
  status_t = hipsparseCreateCoo(&spMatDescr_t, rows, cols, nnz, cooRowInd, cooColInd, cooValues, indexType_t, indexBase_t, dataType);
#endif

#if CUDA_VERSION >= 10010 && CUDA_VERSION < 12000
  // CHECK: hipsparseSpMMAlg_t COOMM_ALG1 = HIPSPARSE_COOMM_ALG1;
  // CHECK-NEXT: hipsparseSpMMAlg_t COOMM_ALG2 = HIPSPARSE_COOMM_ALG2;
  // CHECK-NEXT: hipsparseSpMMAlg_t COOMM_ALG3 = HIPSPARSE_COOMM_ALG3;
  cusparseSpMMAlg_t COOMM_ALG1 = CUSPARSE_COOMM_ALG1;
  cusparseSpMMAlg_t COOMM_ALG2 = CUSPARSE_COOMM_ALG2;
  cusparseSpMMAlg_t COOMM_ALG3 = CUSPARSE_COOMM_ALG3;

  // CHECK: hipsparseCsr2CscAlg_t CSR2CSC_ALG2 = HIPSPARSE_CSR2CSC_ALG2;
  cusparseCsr2CscAlg_t CSR2CSC_ALG2 = CUSPARSE_CSR2CSC_ALG2;
#endif

#if CUDA_VERSION >= 10020
  // CHECK: hipsparseSpVecDescr_t spVecDescr_t;
  hipsparseSpVecDescr_t spVecDescr_t;

  // CHECK: hipsparseDnVecDescr_t dnVecDescr_t;
  hipsparseDnVecDescr_t dnVecDescr_t;

  // CHECK: hipsparseStatus_t STATUS_NOT_SUPPORTED = HIPSPARSE_STATUS_NOT_SUPPORTED;
  hipsparseStatus_t STATUS_NOT_SUPPORTED = HIPSPARSE_STATUS_NOT_SUPPORTED;

  // CHECK: hipsparseSpMVAlg_t spMVAlg_t;
  hipsparseSpMVAlg_t spMVAlg_t;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateSpVec(cusparseSpVecDescr_t* spVecDescr, int64_t size, int64_t nnz, void* indices, void* values, cusparseIndexType_t idxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateSpVec(hipsparseSpVecDescr_t* spVecDescr, int64_t size, int64_t nnz, void* indices, void* values, hipsparseIndexType_t idxType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateSpVec(&spVecDescr_t, size, nnz, indices, values, indexType_t, indexBase_t, dataType);
  status_t = hipsparseCreateSpVec(&spVecDescr_t, size, nnz, indices, values, indexType_t, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroySpVec(cusparseConstSpVecDescr_t spVecDescr);
  // HIP: hipsparseStatus_t hipsparseDestroySpVec(hipsparseSpVecDescr_t spVecDescr);
  // CHECK: status_t = hipsparseDestroySpVec(spVecDescr_t);
  status_t = hipsparseDestroySpVec(spVecDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecGet(cusparseSpVecDescr_t spVecDescr, int64_t* size, int64_t* nnz, void** indices, void** values, cusparseIndexType_t* idxType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpVecGet(const hipsparseSpVecDescr_t spVecDescr, int64_t* size, int64_t* nnz, void** indices, void** values, hipsparseIndexType_t* idxType, hipsparseIndexBase_t* idxBase, hipDataType* valueType);
  // CHECK: status_t = hipsparseSpVecGet(spVecDescr_t, &size, &nnz, &indices, &values, &indexType_t, &indexBase_t, &dataType);
  status_t = hipsparseSpVecGet(spVecDescr_t, &size, &nnz, &indices, &values, &indexType_t, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecGetIndexBase(cusparseConstSpVecDescr_t spVecDescr, cusparseIndexBase_t* idxBase);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpVecGetIndexBase(const hipsparseSpVecDescr_t spVecDescr, hipsparseIndexBase_t* idxBase);
  // CHECK: status_t = hipsparseSpVecGetIndexBase(spVecDescr_t, &indexBase_t);
  status_t = hipsparseSpVecGetIndexBase(spVecDescr_t, &indexBase_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecGetValues(cusparseSpVecDescr_t spVecDescr, void** values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpVecGetValues(const hipsparseSpVecDescr_t spVecDescr, void** values);
  // CHECK: status_t = hipsparseSpVecGetValues(spVecDescr_t, &values);
  status_t = hipsparseSpVecGetValues(spVecDescr_t, &values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecSetValues(cusparseSpVecDescr_t spVecDescr, void* values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpVecSetValues(hipsparseSpVecDescr_t spVecDescr, void* values);
  // CHECK: status_t = hipsparseSpVecSetValues(spVecDescr_t, values);
  status_t = hipsparseSpVecSetValues(spVecDescr_t, values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCsr(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* csrRowOffsets, void* csrColInd, void* csrValues, cusparseIndexType_t csrRowOffsetsType, cusparseIndexType_t csrColIndType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateCsr(hipsparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* csrRowOffsets, void* csrColInd, void* csrValues, hipsparseIndexType_t csrRowOffsetsType, hipsparseIndexType_t csrColIndType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateCsr(&spMatDescr_t, rows, cols, nnz, csrRowOffsets, csrColInd, csrValues, csrRowOffsetsType, csrColIndType, indexBase_t, dataType);
  status_t = hipsparseCreateCsr(&spMatDescr_t, rows, cols, nnz, csrRowOffsets, csrColInd, csrValues, csrRowOffsetsType, csrColIndType, indexBase_t, dataType);
#endif

#if CUDA_VERSION >= 10020 && CUDA_VERSION < 12000
  // CHECK: hipsparseFormat_t FORMAT_COO_AOS = HIPSPARSE_FORMAT_COO_AOS;
  cusparseFormat_t FORMAT_COO_AOS = CUSPARSE_FORMAT_COO_AOS;

  // CHECK: hipsparseSpMVAlg_t MV_ALG_DEFAULT = HIPSPARSE_MV_ALG_DEFAULT;
  cusparseSpMVAlg_t MV_ALG_DEFAULT = CUSPARSE_MV_ALG_DEFAULT;

  // CHECK: hipsparseSpMVAlg_t COOMV_ALG = HIPSPARSE_COOMV_ALG;
  // CHECK-NEXT: hipsparseSpMVAlg_t CSRMV_ALG1 = HIPSPARSE_CSRMV_ALG1;
  // CHECK-NEXT: hipsparseSpMVAlg_t CSRMV_ALG2 = HIPSPARSE_CSRMV_ALG2;
  cusparseSpMVAlg_t COOMV_ALG = CUSPARSE_COOMV_ALG;
  cusparseSpMVAlg_t CSRMV_ALG1 = CUSPARSE_CSRMV_ALG1;
  cusparseSpMVAlg_t CSRMV_ALG2 = CUSPARSE_CSRMV_ALG2;

  // CHECK: hipsparseSpMMAlg_t MM_ALG_DEFAULT = HIPSPARSE_MM_ALG_DEFAULT;
  // CHECK: hipsparseSpMMAlg_t CSRMM_ALG1 = HIPSPARSE_CSRMM_ALG1;
  cusparseSpMMAlg_t MM_ALG_DEFAULT = CUSPARSE_MM_ALG_DEFAULT;
  cusparseSpMMAlg_t CSRMM_ALG1 = CUSPARSE_CSRMM_ALG1;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCooAoS(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cooInd, void* cooValues, cusparseIndexType_t cooIdxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateCooAoS(hipsparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cooInd, void* cooValues, hipsparseIndexType_t cooIdxType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateCooAoS(&spMatDescr_t, rows, cols, nnz, cooRowInd, cooColInd, cooValues, indexType_t, indexBase_t, dataType);
  status_t = cusparseCreateCooAoS(&spMatDescr_t, rows, cols, nnz, cooRowInd, cooColInd, cooValues, indexType_t, indexBase_t, dataType);
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipsparseStatus_t STATUS_INSUFFICIENT_RESOURCES = HIPSPARSE_STATUS_INSUFFICIENT_RESOURCES;
  hipsparseStatus_t STATUS_INSUFFICIENT_RESOURCES = HIPSPARSE_STATUS_INSUFFICIENT_RESOURCES;

  // CHECK: hipsparseSpMMAlg_t SPMM_ALG_DEFAULT = HIPSPARSE_SPMM_ALG_DEFAULT;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG1 = HIPSPARSE_SPMM_COO_ALG1;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG2 = HIPSPARSE_SPMM_COO_ALG2;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG3 = HIPSPARSE_SPMM_COO_ALG3;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_CSR_ALG1 = HIPSPARSE_SPMM_CSR_ALG1;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG4 = HIPSPARSE_SPMM_COO_ALG4;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_CSR_ALG2 = HIPSPARSE_SPMM_CSR_ALG2;
  hipsparseSpMMAlg_t SPMM_ALG_DEFAULT = HIPSPARSE_SPMM_ALG_DEFAULT;
  hipsparseSpMMAlg_t SPMM_COO_ALG1 = HIPSPARSE_SPMM_COO_ALG1;
  hipsparseSpMMAlg_t SPMM_COO_ALG2 = HIPSPARSE_SPMM_COO_ALG2;
  hipsparseSpMMAlg_t SPMM_COO_ALG3 = HIPSPARSE_SPMM_COO_ALG3;
  hipsparseSpMMAlg_t SPMM_CSR_ALG1 = HIPSPARSE_SPMM_CSR_ALG1;
  hipsparseSpMMAlg_t SPMM_COO_ALG4 = HIPSPARSE_SPMM_COO_ALG4;
  hipsparseSpMMAlg_t SPMM_CSR_ALG2 = HIPSPARSE_SPMM_CSR_ALG2;

  // CHECK: hipsparseSpGEMMAlg_t spGEMMAlg_t;
  // CHECK-NEXT: hipsparseSpGEMMAlg_t SPGEMM_DEFAULT = HIPSPARSE_SPGEMM_DEFAULT;
  hipsparseSpGEMMAlg_t spGEMMAlg_t;
  hipsparseSpGEMMAlg_t SPGEMM_DEFAULT = HIPSPARSE_SPGEMM_DEFAULT;
#endif

#if CUDA_VERSION < 11000
  // CHECK: hipsparseHybMat_t hybMat_t;
  cusparseHybMat_t hybMat_t;

  // CHECK: hipsparseHybPartition_t hybPartition_t;
  // CHECK-NEXT: hipsparseHybPartition_t HYB_PARTITION_AUTO = HIPSPARSE_HYB_PARTITION_AUTO;
  // CHECK-NEXT: hipsparseHybPartition_t HYB_PARTITION_USER = HIPSPARSE_HYB_PARTITION_USER;
  // CHECK-NEXT: hipsparseHybPartition_t HYB_PARTITION_MAX = HIPSPARSE_HYB_PARTITION_MAX;
  cusparseHybPartition_t hybPartition_t;
  cusparseHybPartition_t HYB_PARTITION_AUTO = CUSPARSE_HYB_PARTITION_AUTO;
  cusparseHybPartition_t HYB_PARTITION_USER = CUSPARSE_HYB_PARTITION_USER;
  cusparseHybPartition_t HYB_PARTITION_MAX = CUSPARSE_HYB_PARTITION_MAX;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateHybMat(cusparseHybMat_t* hybA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateHybMat(hipsparseHybMat_t* hybA);
  // CHECK: status_t = hipsparseCreateHybMat(&hybMat_t);
  status_t = cusparseCreateHybMat(&hybMat_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyHybMat(cusparseHybMat_t hybA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroyHybMat(hipsparseHybMat_t hybA);
  // CHECK: status_t = hipsparseDestroyHybMat(hybMat_t);
  status_t = cusparseDestroyHybMat(hybMat_t);
#endif

#if CUDA_VERSION >= 11010
  // CHECK: hipsparseSparseToDenseAlg_t sparseToDenseAlg_t;
  // CHECK-NEXT: hipsparseSparseToDenseAlg_t SPARSETODENSE_ALG_DEFAULT = HIPSPARSE_SPARSETODENSE_ALG_DEFAULT;
  hipsparseSparseToDenseAlg_t sparseToDenseAlg_t;
  hipsparseSparseToDenseAlg_t SPARSETODENSE_ALG_DEFAULT = HIPSPARSE_SPARSETODENSE_ALG_DEFAULT;

  // CHECK: hipsparseDenseToSparseAlg_t denseToSparseAlg_t;
  // CHECK-NEXT: hipsparseDenseToSparseAlg_t DENSETOSPARSE_ALG_DEFAULT = HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT;
  hipsparseDenseToSparseAlg_t denseToSparseAlg_t;
  hipsparseDenseToSparseAlg_t DENSETOSPARSE_ALG_DEFAULT = HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCsc(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cscColOffsets, void* cscRowInd, void* cscValues, cusparseIndexType_t cscColOffsetsType, cusparseIndexType_t cscRowIndType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateCsc(hipsparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cscColOffsets, void* cscRowInd, void* cscValues, hipsparseIndexType_t cscColOffsetsType, hipsparseIndexType_t cscRowIndType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateCsc(&spMatDescr_t, rows, cols, nnz, cscColOffsets, cscRowInd, cscValues, cscColOffsetsType, csrColIndType, indexBase_t, dataType);
  status_t = hipsparseCreateCsc(&spMatDescr_t, rows, cols, nnz, cscColOffsets, cscRowInd, cscValues, cscColOffsetsType, csrColIndType, indexBase_t, dataType);
#endif

#if CUDA_VERSION >= 11020
  // CHECK: hipsparseFormat_t FORMAT_BLOCKED_ELL = HIPSPARSE_FORMAT_BLOCKED_ELL;
  hipsparseFormat_t FORMAT_BLOCKED_ELL = HIPSPARSE_FORMAT_BLOCKED_ELL;

  // CHECK: hipsparseSpMVAlg_t SPMV_ALG_DEFAULT = HIPSPARSE_SPMV_ALG_DEFAULT;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_COO_ALG1 = HIPSPARSE_SPMV_COO_ALG1;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_COO_ALG2 = HIPSPARSE_SPMV_COO_ALG2;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_CSR_ALG1 = HIPSPARSE_SPMV_CSR_ALG1;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_CSR_ALG2 = HIPSPARSE_SPMV_CSR_ALG2;
  hipsparseSpMVAlg_t SPMV_ALG_DEFAULT = HIPSPARSE_SPMV_ALG_DEFAULT;
  hipsparseSpMVAlg_t SPMV_COO_ALG1 = HIPSPARSE_SPMV_COO_ALG1;
  hipsparseSpMVAlg_t SPMV_COO_ALG2 = HIPSPARSE_SPMV_COO_ALG2;
  hipsparseSpMVAlg_t SPMV_CSR_ALG1 = HIPSPARSE_SPMV_CSR_ALG1;
  hipsparseSpMVAlg_t SPMV_CSR_ALG2 = HIPSPARSE_SPMV_CSR_ALG2;

  // CHECK: hipsparseSpMMAlg_t SPMM_CSR_ALG3 = HIPSPARSE_SPMM_CSR_ALG3;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_BLOCKED_ELL_ALG1 = HIPSPARSE_SPMM_BLOCKED_ELL_ALG1;
  hipsparseSpMMAlg_t SPMM_CSR_ALG3 = HIPSPARSE_SPMM_CSR_ALG3;
  hipsparseSpMMAlg_t SPMM_BLOCKED_ELL_ALG1 = HIPSPARSE_SPMM_BLOCKED_ELL_ALG1;

  // CHECK: hipsparseSDDMMAlg_t sDDMMAlg_t;
  // CHECK-NEXT: hipsparseSDDMMAlg_t SDDMM_ALG_DEFAULT = HIPSPARSE_SDDMM_ALG_DEFAULT;
  hipsparseSDDMMAlg_t sDDMMAlg_t;
  hipsparseSDDMMAlg_t SDDMM_ALG_DEFAULT = HIPSPARSE_SDDMM_ALG_DEFAULT;
#endif

#if CUDA_VERSION >= 11030
  // CHECK: hipsparseSpMatAttribute_t spMatAttribute_t;
  // CHECK-NEXT: hipsparseSpMatAttribute_t SPMAT_FILL_MODE = HIPSPARSE_SPMAT_FILL_MODE;
  // CHECK-NEXT: hipsparseSpMatAttribute_t SPMAT_DIAG_TYPE = HIPSPARSE_SPMAT_DIAG_TYPE;
  hipsparseSpMatAttribute_t spMatAttribute_t;
  hipsparseSpMatAttribute_t SPMAT_FILL_MODE = HIPSPARSE_SPMAT_FILL_MODE;
  hipsparseSpMatAttribute_t SPMAT_DIAG_TYPE = HIPSPARSE_SPMAT_DIAG_TYPE;

  // CHECK: hipsparseSpSVAlg_t spSVAlg_t;
  // CHECK-NEXT: hipsparseSpSVAlg_t SPSV_ALG_DEFAULT = HIPSPARSE_SPSV_ALG_DEFAULT;
  hipsparseSpSVAlg_t spSVAlg_t;
  hipsparseSpSVAlg_t SPSV_ALG_DEFAULT = HIPSPARSE_SPSV_ALG_DEFAULT;

  // CHECK: hipsparseSpSMAlg_t spSMAlg_t;
  // CHECK-NEXT: hipsparseSpSMAlg_t SPSM_ALG_DEFAULT = HIPSPARSE_SPSM_ALG_DEFAULT;
  hipsparseSpSMAlg_t spSMAlg_t;
  hipsparseSpSMAlg_t SPSM_ALG_DEFAULT = HIPSPARSE_SPSM_ALG_DEFAULT;

  // CHECK: hipsparseSpGEMMAlg_t SPGEMM_CSR_ALG_DETERMINITIC = HIPSPARSE_SPGEMM_CSR_ALG_DETERMINISTIC;
  // CHECK: hipsparseSpGEMMAlg_t SPGEMM_CSR_ALG_NONDETERMINITIC = HIPSPARSE_SPGEMM_CSR_ALG_NONDETERMINISTIC;
  hipsparseSpGEMMAlg_t SPGEMM_CSR_ALG_DETERMINITIC = HIPSPARSE_SPGEMM_CSR_ALG_DETERMINISTIC;
  hipsparseSpGEMMAlg_t SPGEMM_CSR_ALG_NONDETERMINITIC = HIPSPARSE_SPGEMM_CSR_ALG_NONDETERMINISTIC;
#endif

#if CUDA_VERSION >= 12000
  // CHECK: hipsparseCsr2CscAlg_t CSR2CSC_ALG_DEFAULT = HIPSPARSE_CSR2CSC_ALG_DEFAULT;
  hipsparseCsr2CscAlg_t CSR2CSC_ALG_DEFAULT = HIPSPARSE_CSR2CSC_ALG_DEFAULT;

  // CHECK: hipsparseSpGEMMAlg_t SPGEMM_ALG1 = HIPSPARSE_SPGEMM_ALG1;
  // CHECK: hipsparseSpGEMMAlg_t SPGEMM_ALG2 = HIPSPARSE_SPGEMM_ALG2;
  // CHECK: hipsparseSpGEMMAlg_t SPGEMM_ALG3 = HIPSPARSE_SPGEMM_ALG3;
  hipsparseSpGEMMAlg_t SPGEMM_ALG1 = HIPSPARSE_SPGEMM_ALG1;
  hipsparseSpGEMMAlg_t SPGEMM_ALG2 = HIPSPARSE_SPGEMM_ALG2;
  hipsparseSpGEMMAlg_t SPGEMM_ALG3 = HIPSPARSE_SPGEMM_ALG3;
#endif

  return 0;
}
