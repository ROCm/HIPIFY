// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --miopen %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "miopen/miopen.h"
#include "cudnn.h"

int main() {
  printf("15. cuDNN API to MIOpen API synthetic test\n");

  // CHECK: miopenStatus_t dnnStatus_t;
  // CHECK-NEXT: miopenStatus_t STATUS_SUCCESS = miopenStatusSuccess;
  // CHECK-NEXT: miopenStatus_t STATUS_NOT_INITIALIZED = miopenStatusNotInitialized;
  // CHECK-NEXT: miopenStatus_t STATUS_ALLOC_FAILED = miopenStatusAllocFailed;
  // CHECK-NEXT: miopenStatus_t STATUS_BAD_PARAM = miopenStatusBadParm;
  // CHECK-NEXT: miopenStatus_t STATUS_INTERNAL_ERROR = miopenStatusInternalError;
  // CHECK-NEXT: miopenStatus_t STATUS_INVALID_VALUE = miopenStatusInvalidValue;
  // CHECK-NEXT: miopenStatus_t STATUS_NOT_SUPPORTED = miopenStatusUnsupportedOp;
  cudnnStatus_t dnnStatus_t;
  cudnnStatus_t STATUS_SUCCESS = CUDNN_STATUS_SUCCESS;
  cudnnStatus_t STATUS_NOT_INITIALIZED = CUDNN_STATUS_NOT_INITIALIZED;
  cudnnStatus_t STATUS_ALLOC_FAILED = CUDNN_STATUS_ALLOC_FAILED;
  cudnnStatus_t STATUS_BAD_PARAM = CUDNN_STATUS_BAD_PARAM;
  cudnnStatus_t STATUS_INTERNAL_ERROR = CUDNN_STATUS_INTERNAL_ERROR;
  cudnnStatus_t STATUS_INVALID_VALUE = CUDNN_STATUS_INVALID_VALUE;
  cudnnStatus_t STATUS_NOT_SUPPORTED = CUDNN_STATUS_NOT_SUPPORTED;

  // CHECK: miopenStatus_t status;
  cudnnStatus_t status;

  // CHECK: miopenHandle_t handle;
  cudnnHandle_t handle;

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnCreate(cudnnHandle_t *handle);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreate(miopenHandle_t* handle);
  // CHECK: status = miopenCreate(&handle);
  status = cudnnCreate(&handle);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnDestroy(cudnnHandle_t handle);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroy(miopenHandle_t handle);
  // CHECK: status = miopenDestroy(handle);
  status = cudnnDestroy(handle);

  const char* const_ch = nullptr;

  // CUDA: const char *CUDNNWINAPI cudnnGetErrorString(cudnnStatus_t status);
  // MIOPEN: MIOPEN_EXPORT const char* miopenGetErrorString(miopenStatus_t error);
  // CHECK: const_ch = miopenGetErrorString(status);
  const_ch = cudnnGetErrorString(status);

  // CHECK: miopenAcceleratorQueue_t streamId;
  hipStream_t streamId;

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnSetStream(cudnnHandle_t handle, cudaStream_t streamId);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSetStream(miopenHandle_t handle, miopenAcceleratorQueue_t streamID);
  // CHECK: status = miopenSetStream(handle, streamId);
  status = cudnnSetStream(handle, streamId);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnGetStream(cudnnHandle_t handle, cudaStream_t *streamId);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetStream(miopenHandle_t handle, miopenAcceleratorQueue_t* streamID);
  // CHECK: status = miopenGetStream(handle, &streamId);
  status = cudnnGetStream(handle, &streamId);

  // CHECK: miopenTensorDescriptor_t tensorDescriptor;
  // CHECK-NEXT: miopenTensorDescriptor_t filterDescriptor;
  cudnnTensorDescriptor_t tensorDescriptor;
  cudnnFilterDescriptor_t filterDescriptor;

  // CHECK: miopenConvolutionDescriptor_t convolutionDescriptor;
  cudnnConvolutionDescriptor_t convolutionDescriptor;

  // CHECK: miopenPoolingDescriptor_t poolingDescriptor;
  cudnnPoolingDescriptor_t poolingDescriptor;

  // CHECK: miopenLRNDescriptor_t LRNDescriptor;
  cudnnLRNDescriptor_t LRNDescriptor;

  // CHECK: miopenActivationDescriptor_t activationDescriptor;
  cudnnActivationDescriptor_t activationDescriptor;

  // CHECK: miopenRNNDescriptor_t RNNDescriptor;
  cudnnRNNDescriptor_t RNNDescriptor;

  // CHECK: miopenCTCLossDescriptor_t CTCLossDescriptor;
  cudnnCTCLossDescriptor_t CTCLossDescriptor;

  // CHECK: miopenDropoutDescriptor_t DropoutDescriptor;
  cudnnDropoutDescriptor_t DropoutDescriptor;

  // CHECK: miopenReduceTensorDescriptor_t ReduceTensorDescriptor;
  cudnnReduceTensorDescriptor_t ReduceTensorDescriptor;

  // CHECK: miopenDataType_t dataType;
  // CHECK-NEXT: miopenDataType_t DATA_FLOAT = miopenFloat;
  // CHECK-NEXT: miopenDataType_t DATA_DOUBLE = miopenDouble;
  // CHECK-NEXT: miopenDataType_t DATA_HALF = miopenHalf;
  // CHECK-NEXT: miopenDataType_t DATA_INT8 = miopenInt8;
  // CHECK-NEXT: miopenDataType_t DATA_INT32 = miopenInt32;
  // CHECK-NEXT: miopenDataType_t DATA_INT8x4 = miopenInt8x4;
  // CHECK-NEXT: miopenDataType_t DATA_BFLOAT16 = miopenBFloat16;
  cudnnDataType_t dataType;
  cudnnDataType_t DATA_FLOAT = CUDNN_DATA_FLOAT;
  cudnnDataType_t DATA_DOUBLE = CUDNN_DATA_DOUBLE;
  cudnnDataType_t DATA_HALF = CUDNN_DATA_HALF;
  cudnnDataType_t DATA_INT8 = CUDNN_DATA_INT8;
  cudnnDataType_t DATA_INT32 = CUDNN_DATA_INT32;
  cudnnDataType_t DATA_INT8x4 = CUDNN_DATA_INT8x4;
  cudnnDataType_t DATA_BFLOAT16 = CUDNN_DATA_BFLOAT16;

  // CHECK: miopenTensorOp_t tensorOp;
  // CHECK-NEXT: miopenTensorOp_t OP_TENSOR_ADD = miopenTensorOpAdd;
  // CHECK-NEXT: miopenTensorOp_t OP_TENSOR_MUL = miopenTensorOpMul;
  // CHECK-NEXT: miopenTensorOp_t OP_TENSOR_MIN = miopenTensorOpMin;
  // CHECK-NEXT: miopenTensorOp_t OP_TENSOR_MAX = miopenTensorOpMax;
  cudnnOpTensorOp_t tensorOp;
  cudnnOpTensorOp_t OP_TENSOR_ADD = CUDNN_OP_TENSOR_ADD;
  cudnnOpTensorOp_t OP_TENSOR_MUL = CUDNN_OP_TENSOR_MUL;
  cudnnOpTensorOp_t OP_TENSOR_MIN = CUDNN_OP_TENSOR_MIN;
  cudnnOpTensorOp_t OP_TENSOR_MAX = CUDNN_OP_TENSOR_MAX;

  // CHECK: miopenConvolutionMode_t convolutionMode;
  cudnnConvolutionMode_t convolutionMode;

  // CHECK: miopenPoolingMode_t poolingMode;
  // CHECK-NEXT: miopenPoolingMode_t POOLING_MAX = miopenPoolingMax;
  cudnnPoolingMode_t poolingMode;
  cudnnPoolingMode_t POOLING_MAX = CUDNN_POOLING_MAX;

  // CHECK: miopenLRNMode_t LRNMode;
  // CHECK-NEXT: miopenLRNMode_t LRN_CROSS_CHANNEL_DIM1 = miopenLRNCrossChannel;
  cudnnLRNMode_t LRNMode;
  cudnnLRNMode_t LRN_CROSS_CHANNEL_DIM1 = CUDNN_LRN_CROSS_CHANNEL_DIM1;

  // CHECK: miopenBatchNormMode_t batchNormMode;
  // CHECK-NEXT: miopenBatchNormMode_t BATCHNORM_PER_ACTIVATION = miopenBNPerActivation;
  // CHECK-NEXT: miopenBatchNormMode_t BATCHNORM_SPATIAL = miopenBNSpatial;
  cudnnBatchNormMode_t batchNormMode;
  cudnnBatchNormMode_t BATCHNORM_PER_ACTIVATION = CUDNN_BATCHNORM_PER_ACTIVATION;
  cudnnBatchNormMode_t BATCHNORM_SPATIAL = CUDNN_BATCHNORM_SPATIAL;

  // CHECK: miopenActivationMode_t activationMode;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_RELU = miopenActivationRELU;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_TANH = miopenActivationTANH;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_CLIPPED_RELU = miopenActivationCLIPPEDRELU;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_ELU = miopenActivationELU;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_IDENTITY = miopenActivationPASTHRU;
  cudnnActivationMode_t activationMode;
  cudnnActivationMode_t ACTIVATION_RELU = CUDNN_ACTIVATION_RELU;
  cudnnActivationMode_t ACTIVATION_TANH = CUDNN_ACTIVATION_TANH;
  cudnnActivationMode_t ACTIVATION_CLIPPED_RELU = CUDNN_ACTIVATION_CLIPPED_RELU;
  cudnnActivationMode_t ACTIVATION_ELU = CUDNN_ACTIVATION_ELU;
  cudnnActivationMode_t ACTIVATION_IDENTITY = CUDNN_ACTIVATION_IDENTITY;

  // CHECK: miopenSoftmaxAlgorithm_t softmaxAlgorithm;
  // CHECK-NEXT: miopenSoftmaxAlgorithm_t SOFTMAX_FAST = MIOPEN_SOFTMAX_FAST;
  // CHECK-NEXT: miopenSoftmaxAlgorithm_t SOFTMAX_ACCURATE = MIOPEN_SOFTMAX_ACCURATE;
  // CHECK-NEXT: miopenSoftmaxAlgorithm_t SOFTMAX_LOG = MIOPEN_SOFTMAX_LOG;
  cudnnSoftmaxAlgorithm_t softmaxAlgorithm;
  cudnnSoftmaxAlgorithm_t SOFTMAX_FAST = CUDNN_SOFTMAX_FAST;
  cudnnSoftmaxAlgorithm_t SOFTMAX_ACCURATE = CUDNN_SOFTMAX_ACCURATE;
  cudnnSoftmaxAlgorithm_t SOFTMAX_LOG = CUDNN_SOFTMAX_LOG;

  // CHECK: miopenReduceTensorOp_t reduceTensorOp;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_ADD = MIOPEN_REDUCE_TENSOR_ADD;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_MUL = MIOPEN_REDUCE_TENSOR_MUL;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_MIN = MIOPEN_REDUCE_TENSOR_MIN;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_MAX = MIOPEN_REDUCE_TENSOR_MAX;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_AMAX = MIOPEN_REDUCE_TENSOR_AMAX;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_AVG = MIOPEN_REDUCE_TENSOR_AVG;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_NORM1 = MIOPEN_REDUCE_TENSOR_NORM1;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_NORM2 = MIOPEN_REDUCE_TENSOR_NORM2;
  cudnnReduceTensorOp_t reduceTensorOp;
  cudnnReduceTensorOp_t REDUCE_TENSOR_ADD = CUDNN_REDUCE_TENSOR_ADD;
  cudnnReduceTensorOp_t REDUCE_TENSOR_MUL = CUDNN_REDUCE_TENSOR_MUL;
  cudnnReduceTensorOp_t REDUCE_TENSOR_MIN = CUDNN_REDUCE_TENSOR_MIN;
  cudnnReduceTensorOp_t REDUCE_TENSOR_MAX = CUDNN_REDUCE_TENSOR_MAX;
  cudnnReduceTensorOp_t REDUCE_TENSOR_AMAX = CUDNN_REDUCE_TENSOR_AMAX;
  cudnnReduceTensorOp_t REDUCE_TENSOR_AVG = CUDNN_REDUCE_TENSOR_AVG;
  cudnnReduceTensorOp_t REDUCE_TENSOR_NORM1 = CUDNN_REDUCE_TENSOR_NORM1;
  cudnnReduceTensorOp_t REDUCE_TENSOR_NORM2 = CUDNN_REDUCE_TENSOR_NORM2;

  // CHECK: miopenConvFwdAlgorithm_t convolutionFwdAlgo;
  // CHECK-NEXT: miopenConvFwdAlgorithm_t CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM = miopenConvolutionFwdAlgoImplicitGEMM;
  // CHECK-NEXT: miopenConvFwdAlgorithm_t CONVOLUTION_FWD_ALGO_GEMM = miopenConvolutionFwdAlgoGEMM;
  // CHECK-NEXT: miopenConvFwdAlgorithm_t CONVOLUTION_FWD_ALGO_DIRECT = miopenConvolutionFwdAlgoDirect;
  // CHECK-NEXT: miopenConvFwdAlgorithm_t CONVOLUTION_FWD_ALGO_FFT = miopenConvolutionFwdAlgoFFT;
  // CHECK-NEXT: miopenConvFwdAlgorithm_t CONVOLUTION_FWD_ALGO_WINOGRAD = miopenConvolutionFwdAlgoWinograd;
  cudnnConvolutionFwdAlgo_t convolutionFwdAlgo;
  cudnnConvolutionFwdAlgo_t CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM = CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
  cudnnConvolutionFwdAlgo_t CONVOLUTION_FWD_ALGO_GEMM = CUDNN_CONVOLUTION_FWD_ALGO_GEMM;
  cudnnConvolutionFwdAlgo_t CONVOLUTION_FWD_ALGO_DIRECT = CUDNN_CONVOLUTION_FWD_ALGO_DIRECT;
  cudnnConvolutionFwdAlgo_t CONVOLUTION_FWD_ALGO_FFT = CUDNN_CONVOLUTION_FWD_ALGO_FFT;
  cudnnConvolutionFwdAlgo_t CONVOLUTION_FWD_ALGO_WINOGRAD = CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD;

  // CHECK: miopenNanPropagation_t nanPropagation_t;
  // CHECK-NEXT: miopenNanPropagation_t NOT_PROPAGATE_NAN = MIOPEN_NOT_PROPAGATE_NAN;
  // CHECK-NEXT: miopenNanPropagation_t PROPAGATE_NAN = MIOPEN_PROPAGATE_NAN;
  cudnnNanPropagation_t nanPropagation_t;
  cudnnNanPropagation_t NOT_PROPAGATE_NAN = CUDNN_NOT_PROPAGATE_NAN;
  cudnnNanPropagation_t PROPAGATE_NAN = CUDNN_PROPAGATE_NAN;

  // CHECK: miopenReduceTensorIndices_t reduceTensorIndices;
  // CHECK-NEXT: miopenReduceTensorIndices_t REDUCE_TENSOR_NO_INDICES = MIOPEN_REDUCE_TENSOR_NO_INDICES;
  // CHECK-NEXT: miopenReduceTensorIndices_t REDUCE_TENSOR_FLATTENED_INDICES = MIOPEN_REDUCE_TENSOR_FLATTENED_INDICES;
  cudnnReduceTensorIndices_t reduceTensorIndices;
  cudnnReduceTensorIndices_t REDUCE_TENSOR_NO_INDICES = CUDNN_REDUCE_TENSOR_NO_INDICES;
  cudnnReduceTensorIndices_t REDUCE_TENSOR_FLATTENED_INDICES = CUDNN_REDUCE_TENSOR_FLATTENED_INDICES;

  // CHECK: miopenIndicesType_t indicesType;
  // CHECK-NEXT: miopenIndicesType_t _32BIT_INDICES = MIOPEN_32BIT_INDICES;
  // CHECK-NEXT: miopenIndicesType_t _64BIT_INDICES = MIOPEN_64BIT_INDICES;
  // CHECK-NEXT: miopenIndicesType_t _16BIT_INDICES = MIOPEN_16BIT_INDICES;
  // CHECK-NEXT: miopenIndicesType_t _8BIT_INDICES = MIOPEN_8BIT_INDICES;
  cudnnIndicesType_t indicesType;
  cudnnIndicesType_t _32BIT_INDICES = CUDNN_32BIT_INDICES;
  cudnnIndicesType_t _64BIT_INDICES = CUDNN_64BIT_INDICES;
  cudnnIndicesType_t _16BIT_INDICES = CUDNN_16BIT_INDICES;
  cudnnIndicesType_t _8BIT_INDICES = CUDNN_8BIT_INDICES;

  // CHECK: miopenConvBwdDataAlgorithm_t ConvolutionBwdDataAlgo_t;
  // CHECK-NEXT: miopenConvBwdDataAlgorithm_t CONVOLUTION_BWD_DATA_ALGO_0 = miopenConvolutionBwdDataAlgoGEMM;
  // CHECK-NEXT: miopenConvBwdDataAlgorithm_t CONVOLUTION_BWD_DATA_ALGO_1 = miopenConvolutionBwdDataAlgoDirect;
  // CHECK-NEXT: miopenConvBwdDataAlgorithm_t CONVOLUTION_BWD_DATA_ALGO_FFT = miopenConvolutionBwdDataAlgoFFT;
  // CHECK-NEXT: miopenConvBwdDataAlgorithm_t CONVOLUTION_BWD_DATA_ALGO_WINOGRAD = miopenConvolutionBwdDataAlgoWinograd;
  cudnnConvolutionBwdDataAlgo_t ConvolutionBwdDataAlgo_t;
  cudnnConvolutionBwdDataAlgo_t CONVOLUTION_BWD_DATA_ALGO_0 = CUDNN_CONVOLUTION_BWD_DATA_ALGO_0;
  cudnnConvolutionBwdDataAlgo_t CONVOLUTION_BWD_DATA_ALGO_1 = CUDNN_CONVOLUTION_BWD_DATA_ALGO_1;
  cudnnConvolutionBwdDataAlgo_t CONVOLUTION_BWD_DATA_ALGO_FFT = CUDNN_CONVOLUTION_BWD_DATA_ALGO_FFT;
  cudnnConvolutionBwdDataAlgo_t CONVOLUTION_BWD_DATA_ALGO_WINOGRAD = CUDNN_CONVOLUTION_BWD_DATA_ALGO_WINOGRAD;

  // CHECK: miopenConvAlgoPerf_t ConvolutionFwdAlgoPerf_t;
  // CHECK-NEXT: miopenConvAlgoPerf_t ConvolutionFwdAlgoPerfStruct;
  cudnnConvolutionFwdAlgoPerf_t ConvolutionFwdAlgoPerf_t;
  cudnnConvolutionFwdAlgoPerfStruct ConvolutionFwdAlgoPerfStruct;

  // CHECK: miopenConvAlgoPerf_t ConvolutionBwdDataAlgoPerf_t;
  // CHECK-NEXT: miopenConvAlgoPerf_t ConvolutionBwdDataAlgoPerfStruct;
  cudnnConvolutionBwdDataAlgoPerf_t ConvolutionBwdDataAlgoPerf_t;
  cudnnConvolutionBwdDataAlgoPerfStruct ConvolutionBwdDataAlgoPerfStruct;

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnCreateTensorDescriptor(cudnnTensorDescriptor_t* tensorDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreateTensorDescriptor(miopenTensorDescriptor_t* tensorDesc);
  // CHECK: status = miopenCreateTensorDescriptor(&tensorDescriptor);
  status = cudnnCreateTensorDescriptor(&tensorDescriptor);

  // TODO: cudnnSetTensor4dDescriptor -> miopenSet4dTensorDescriptor: different signatures
  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnSetTensor4dDescriptor(cudnnTensorDescriptor_t tensorDesc, cudnnTensorFormat_t format, cudnnDataType_t dataType, int n, int c, int h, int w);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSet4dTensorDescriptor(miopenTensorDescriptor_t tensorDesc, miopenDataType_t dataType, int n, int c, int h, int w);

  int n = 0;
  int c = 0;
  int h = 0;
  int w = 0;
  int nStride = 0;
  int cStride = 0;
  int hStride = 0;
  int wStride = 0;

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnSetTensor4dDescriptorEx(cudnnTensorDescriptor_t tensorDesc, cudnnDataType_t dataType, int n, int c, int h, int w, int nStride, int cStride, int hStride, int wStride);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSet4dTensorDescriptorEx(miopenTensorDescriptor_t tensorDesc, miopenDataType_t dataType, int n, int c, int h, int w, int nStride, int cStride, int hStride, int wStride);
  // CHECK: status = miopenSet4dTensorDescriptorEx(tensorDescriptor, dataType, n, c, h, w, nStride, cStride, hStride, wStride);
  status = cudnnSetTensor4dDescriptorEx(tensorDescriptor, dataType, n, c, h, w, nStride, cStride, hStride, wStride);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnGetTensor4dDescriptor(const cudnnTensorDescriptor_t tensorDesc, cudnnDataType_t* dataType, int* n, int* c, int* h, int* w, int* nStride, int* cStride, int* hStride, int* wStride);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGet4dTensorDescriptor(miopenTensorDescriptor_t tensorDesc, miopenDataType_t* dataType, int* n, int* c, int* h, int* w, int* nStride, int* cStride, int* hStride, int* wStride);
  // CHECK: status = miopenGet4dTensorDescriptor(tensorDescriptor, &dataType, &n, &c, &h, &w, &nStride, &cStride, &hStride, &wStride);
  status = cudnnGetTensor4dDescriptor(tensorDescriptor, &dataType, &n, &c, &h, &w, &nStride, &cStride, &hStride, &wStride);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnDestroyTensorDescriptor(cudnnTensorDescriptor_t tensorDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroyTensorDescriptor(miopenTensorDescriptor_t tensorDesc);
  // CHECK: status = miopenDestroyTensorDescriptor(tensorDescriptor);
  status = cudnnDestroyTensorDescriptor(tensorDescriptor);


  // CHECK: miopenTensorDescriptor_t aD;
  // CHECK-NEXT: miopenTensorDescriptor_t bD;
  // CHECK-NEXT: miopenTensorDescriptor_t cD;
  // CHECK-NEXT: miopenTensorDescriptor_t xD;
  // CHECK-NEXT: miopenTensorDescriptor_t yD;
  // CHECK-NEXT: miopenTensorDescriptor_t wD;
  // CHECK-NEXT: miopenTensorDescriptor_t inputD;
  // CHECK-NEXT: miopenTensorDescriptor_t dbD;
  cudnnTensorDescriptor_t aD;
  cudnnTensorDescriptor_t bD;
  cudnnTensorDescriptor_t cD;
  cudnnTensorDescriptor_t xD;
  cudnnTensorDescriptor_t yD;
  cudnnTensorDescriptor_t wD;
  cudnnTensorDescriptor_t inputD;
  cudnnTensorDescriptor_t dbD;
  void* A = nullptr;
  void* B = nullptr;
  void* C = nullptr;
  void* alpha = nullptr;
  void* alpha1 = nullptr;
  void* alpha2 = nullptr;
  void* beta = nullptr;
  void* x = nullptr;
  void* dx = nullptr;
  void* y = nullptr;
  void* dy = nullptr;
  void* W = nullptr;
  void* db = nullptr;
  int groupCount = 0;
  int requestedAlgoCount = 0;
  int returnedAlgoCount = 0;
  void* workSpace = nullptr;
  size_t workSpaceSizeInBytes = 0;

  // TODO: cudnnOpTensor -> miopenOpTensor: different signatures: cudnnOpTensorDescriptor_t != miopenTensorOp_t
  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnOpTensor(cudnnHandle_t handle, const cudnnOpTensorDescriptor_t opTensorDesc, const void* alpha1, const cudnnTensorDescriptor_t aDesc, const void* A, const void* alpha2, const cudnnTensorDescriptor_t bDesc, const void* B, const void* beta, const cudnnTensorDescriptor_t cDesc, void* C);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenOpTensor(miopenHandle_t handle, miopenTensorOp_t tensorOp, const void* alpha1, const miopenTensorDescriptor_t aDesc, const void* A, const void* alpha2, const miopenTensorDescriptor_t bDesc, const void* B, const void* beta, const miopenTensorDescriptor_t cDesc, void* C);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnSetTensor(cudnnHandle_t handle, const cudnnTensorDescriptor_t yDesc, void* y, const void* valuePtr);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSetTensor(miopenHandle_t handle, const miopenTensorDescriptor_t yDesc, void* y, const void* alpha);
  // CHECK: status = miopenSetTensor(handle, tensorDescriptor, y, alpha);
  status = cudnnSetTensor(handle, tensorDescriptor, y, alpha);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnScaleTensor(cudnnHandle_t handle, const cudnnTensorDescriptor_t yDesc, void* y, const void* alpha);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenScaleTensor(miopenHandle_t handle, const miopenTensorDescriptor_t yDesc, void* y, const void* alpha);
  // CHECK: status = miopenScaleTensor(handle, tensorDescriptor, y, alpha);
  status = cudnnScaleTensor(handle, tensorDescriptor, y, alpha);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnTransformTensor(cudnnHandle_t handle, const void* alpha, const cudnnTensorDescriptor_t xDesc, const void* x, const void* beta, const cudnnTensorDescriptor_t yDesc, void* y);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenTransformTensor(miopenHandle_t handle, const void* alpha, const miopenTensorDescriptor_t xDesc, const void* x, const void* beta, const miopenTensorDescriptor_t yDesc, void* y);
  // CHECK: status = miopenTransformTensor(handle, alpha, xD, x, beta, yD, y);
  status = cudnnTransformTensor(handle, alpha, xD, x, beta, yD, y);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnCreateConvolutionDescriptor(cudnnConvolutionDescriptor_t* convDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreateConvolutionDescriptor(miopenConvolutionDescriptor_t* convDesc);
  // CHECK: status = miopenCreateConvolutionDescriptor(&convolutionDescriptor);
  status = cudnnCreateConvolutionDescriptor(&convolutionDescriptor);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnSetConvolutionGroupCount(cudnnConvolutionDescriptor_t convDesc, int groupCount);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSetConvolutionGroupCount(miopenConvolutionDescriptor_t convDesc, int groupCount);
  // CHECK: status = miopenSetConvolutionGroupCount(convolutionDescriptor, groupCount);
  status = cudnnSetConvolutionGroupCount(convolutionDescriptor, groupCount);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnGetConvolution2dForwardOutputDim(const cudnnConvolutionDescriptor_t convDesc, const cudnnTensorDescriptor_t inputTensorDesc, const cudnnFilterDescriptor_t filterDesc, int* n, int* c, int* h, int* w);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetConvolutionForwardOutputDim(miopenConvolutionDescriptor_t convDesc, const miopenTensorDescriptor_t inputTensorDesc, const miopenTensorDescriptor_t filterDesc, int* n, int* c, int* h, int* w);
  // CHECK: status = miopenGetConvolutionForwardOutputDim(convolutionDescriptor, inputD, filterDescriptor, &n, &c, &h, &w);
  status = cudnnGetConvolution2dForwardOutputDim(convolutionDescriptor, inputD, filterDescriptor, &n, &c, &h, &w);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnDestroyConvolutionDescriptor(cudnnConvolutionDescriptor_t convDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroyConvolutionDescriptor(miopenConvolutionDescriptor_t convDesc);
  // CHECK: status = miopenDestroyConvolutionDescriptor(convolutionDescriptor);
  status = cudnnDestroyConvolutionDescriptor(convolutionDescriptor);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnFindConvolutionForwardAlgorithmEx(cudnnHandle_t handle, const cudnnTensorDescriptor_t xDesc, const void* x, const cudnnFilterDescriptor_t wDesc, const void* w, const cudnnConvolutionDescriptor_t convDesc, const cudnnTensorDescriptor_t yDesc, void* y, const int requestedAlgoCount, int* returnedAlgoCount, cudnnConvolutionFwdAlgoPerf_t* perfResults, void* workSpace, size_t workSpaceSizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenFindConvolutionForwardAlgorithm(miopenHandle_t handle, const miopenTensorDescriptor_t xDesc, const void* x, const miopenTensorDescriptor_t wDesc, const void* w, const miopenConvolutionDescriptor_t convDesc, const miopenTensorDescriptor_t yDesc, void* y, const int requestAlgoCount, int* returnedAlgoCount, miopenConvAlgoPerf_t* perfResults, void* workSpace, size_t workSpaceSize, bool exhaustiveSearch);
  // CHECK: status = miopenFindConvolutionForwardAlgorithm(handle, xD, x, filterDescriptor, W, convolutionDescriptor, yD, y, requestedAlgoCount, &returnedAlgoCount, &ConvolutionFwdAlgoPerf_t, workSpace, workSpaceSizeInBytes, true);
  status = cudnnFindConvolutionForwardAlgorithmEx(handle, xD, x, filterDescriptor, W, convolutionDescriptor, yD, y, requestedAlgoCount, &returnedAlgoCount, &ConvolutionFwdAlgoPerf_t, workSpace, workSpaceSizeInBytes);

  // TODO: swap 2 and 3 arguments (const miopenTensorDescriptor_t wDesc and const miopenTensorDescriptor_t xDesc)
  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnGetConvolutionForwardWorkspaceSize(cudnnHandle_t handle, const cudnnTensorDescriptor_t xDesc, const cudnnFilterDescriptor_t wDesc, const cudnnConvolutionDescriptor_t convDesc, const cudnnTensorDescriptor_t yDesc, cudnnConvolutionFwdAlgo_t algo, size_t* sizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenConvolutionForwardGetWorkSpaceSize(miopenHandle_t handle, const miopenTensorDescriptor_t wDesc, const miopenTensorDescriptor_t xDesc, const miopenConvolutionDescriptor_t convDesc, const miopenTensorDescriptor_t yDesc, size_t* workSpaceSize);
  // CHECK: status = miopenConvolutionForwardGetWorkSpaceSize(handle, xD, filterDescriptor, convolutionDescriptor, yD,  &workSpaceSizeInBytes);
  status = cudnnGetConvolutionForwardWorkspaceSize(handle, xD, filterDescriptor, convolutionDescriptor, yD, convolutionFwdAlgo, &workSpaceSizeInBytes);

  // TODO: swap correctly last 5 arguments
  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnConvolutionForward(cudnnHandle_t handle, const void* alpha, const cudnnTensorDescriptor_t xDesc, const void* x, const cudnnFilterDescriptor_t wDesc, const void* w, const cudnnConvolutionDescriptor_t convDesc, cudnnConvolutionFwdAlgo_t algo, void* workSpace, size_t workSpaceSizeInBytes, const void* beta, const cudnnTensorDescriptor_t yDesc, void* y);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenConvolutionForward(miopenHandle_t handle, const void* alpha, const miopenTensorDescriptor_t xDesc, const void* x, const miopenTensorDescriptor_t wDesc, const void* w, const miopenConvolutionDescriptor_t convDesc, miopenConvFwdAlgorithm_t algo, const void* beta, const miopenTensorDescriptor_t yDesc, void* y, void* workSpace, size_t workSpaceSize);
  // CHECK: status = miopenConvolutionForward(handle, alpha, xD, x, filterDescriptor, W, convolutionDescriptor, convolutionFwdAlgo, workSpace, workSpaceSizeInBytes, beta, yD, y);
  status = cudnnConvolutionForward(handle, alpha, xD, x, filterDescriptor, W, convolutionDescriptor, convolutionFwdAlgo, workSpace, workSpaceSizeInBytes, beta, yD, y);

  // TODO: swap 2 and 3 arguments
  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnGetConvolutionBackwardDataWorkspaceSize(cudnnHandle_t handle, const cudnnFilterDescriptor_t wDesc, const cudnnTensorDescriptor_t dyDesc, const cudnnConvolutionDescriptor_t convDesc, const cudnnTensorDescriptor_t dxDesc, cudnnConvolutionBwdDataAlgo_t algo, size_t* sizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenConvolutionBackwardDataGetWorkSpaceSize(miopenHandle_t handle, const miopenTensorDescriptor_t dyDesc, const miopenTensorDescriptor_t wDesc, const miopenConvolutionDescriptor_t convDesc, const miopenTensorDescriptor_t dxDesc, size_t* workSpaceSize);
  // CHECK: status = miopenConvolutionBackwardDataGetWorkSpaceSize(handle, filterDescriptor, yD, convolutionDescriptor, xD,  &workSpaceSizeInBytes);
  status = cudnnGetConvolutionBackwardDataWorkspaceSize(handle, filterDescriptor, yD, convolutionDescriptor, xD, ConvolutionBwdDataAlgo_t, &workSpaceSizeInBytes);

  // TODO: swap correctly all args, starting from 3rd
  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnConvolutionBackwardData(cudnnHandle_t handle, const void* alpha, const cudnnFilterDescriptor_t wDesc, const void* w, const cudnnTensorDescriptor_t dyDesc, const void* dy, const cudnnConvolutionDescriptor_t convDesc, cudnnConvolutionBwdDataAlgo_t algo, void* workSpace, size_t workSpaceSizeInBytes, const void* beta, const cudnnTensorDescriptor_t dxDesc, void* dx);
  // MIOPEN MIOPEN_EXPORT miopenStatus_t miopenConvolutionBackwardData(miopenHandle_t handle, const void* alpha, const miopenTensorDescriptor_t dyDesc, const void* dy, const miopenTensorDescriptor_t wDesc, const void* w, const miopenConvolutionDescriptor_t convDesc, miopenConvBwdDataAlgorithm_t algo, const void* beta, const miopenTensorDescriptor_t dxDesc, void* dx, void* workSpace, size_t workSpaceSize);
  // CHECK: status = miopenConvolutionBackwardData(handle, alpha, filterDescriptor, W, yD, dy, convolutionDescriptor, ConvolutionBwdDataAlgo_t, workSpace, workSpaceSizeInBytes, beta, xD, dx);
  status = cudnnConvolutionBackwardData(handle, alpha, filterDescriptor, W, yD, dy, convolutionDescriptor, ConvolutionBwdDataAlgo_t, workSpace, workSpaceSizeInBytes, beta, xD, dx);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnConvolutionBackwardBias(cudnnHandle_t handle, const void* alpha, const cudnnTensorDescriptor_t dyDesc, const void* dy, const void* beta, const cudnnTensorDescriptor_t dbDesc, void* db);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenConvolutionBackwardBias(miopenHandle_t handle, const void* alpha, const miopenTensorDescriptor_t dyDesc, const void* dy, const void* beta, const miopenTensorDescriptor_t dbDesc, void* db);
  // CHECK: status = miopenConvolutionBackwardBias(handle, alpha, yD, dy, beta, dbD, db);
  status = cudnnConvolutionBackwardBias(handle, alpha, yD, dy, beta, dbD, db);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnCreatePoolingDescriptor(cudnnPoolingDescriptor_t* poolingDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreatePoolingDescriptor(miopenPoolingDescriptor_t* poolDesc);
  // CHECK: status = miopenCreatePoolingDescriptor(&poolingDescriptor);
  status = cudnnCreatePoolingDescriptor(&poolingDescriptor);

  return 0;
}
