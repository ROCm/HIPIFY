// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --miopen %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "miopen/miopen.h"
#include "hipDNN.h"

int main() {
  printf("15. cuDNN API to MIOpen API synthetic test\n");

  // CHECK: miopenStatus_t dnnStatus_t;
  // CHECK-NEXT: miopenStatus_t STATUS_SUCCESS = miopenStatusSuccess;
  // CHECK-NEXT: miopenStatus_t STATUS_NOT_INITIALIZED = miopenStatusNotInitialized;
  // CHECK-NEXT: miopenStatus_t STATUS_ALLOC_FAILED = miopenStatusAllocFailed;
  // CHECK-NEXT: miopenStatus_t STATUS_BAD_PARAM = miopenStatusBadParm;
  // CHECK-NEXT: miopenStatus_t STATUS_INTERNAL_ERROR = miopenStatusInternalError;
  // CHECK-NEXT: miopenStatus_t STATUS_INVALID_VALUE = miopenStatusInvalidValue;
  // CHECK-NEXT: miopenStatus_t STATUS_NOT_SUPPORTED = miopenStatusUnsupportedOp;
  hipdnnStatus_t dnnStatus_t;
  hipdnnStatus_t STATUS_SUCCESS = HIPDNN_STATUS_SUCCESS;
  hipdnnStatus_t STATUS_NOT_INITIALIZED = HIPDNN_STATUS_NOT_INITIALIZED;
  hipdnnStatus_t STATUS_ALLOC_FAILED = HIPDNN_STATUS_ALLOC_FAILED;
  hipdnnStatus_t STATUS_BAD_PARAM = HIPDNN_STATUS_BAD_PARAM;
  hipdnnStatus_t STATUS_INTERNAL_ERROR = HIPDNN_STATUS_INTERNAL_ERROR;
  hipdnnStatus_t STATUS_INVALID_VALUE = HIPDNN_STATUS_INVALID_VALUE;
  hipdnnStatus_t STATUS_NOT_SUPPORTED = HIPDNN_STATUS_NOT_SUPPORTED;

  // CHECK: miopenStatus_t status;
  hipdnnStatus_t status;

  // CHECK: miopenHandle_t handle;
  hipdnnHandle_t handle;

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnCreate(hipdnnHandle_t *handle);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreate(miopenHandle_t* handle);
  // CHECK: status = miopenCreate(&handle);
  status = hipdnnCreate(&handle);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnDestroy(hipdnnHandle_t handle);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroy(miopenHandle_t handle);
  // CHECK: status = miopenDestroy(handle);
  status = hipdnnDestroy(handle);

  const char* const_ch = nullptr;

  // CUDA: const char *CUDNNWINAPI hipdnnGetErrorString(hipdnnStatus_t status);
  // MIOPEN: MIOPEN_EXPORT const char* miopenGetErrorString(miopenStatus_t error);
  // CHECK: const_ch = miopenGetErrorString(status);
  const_ch = hipdnnGetErrorString(status);

  // CHECK: miopenAcceleratorQueue_t streamId;
  hipStream_t streamId;

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnSetStream(hipdnnHandle_t handle, hipStream_t streamId);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSetStream(miopenHandle_t handle, miopenAcceleratorQueue_t streamID);
  // CHECK: status = miopenSetStream(handle, streamId);
  status = hipdnnSetStream(handle, streamId);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnGetStream(hipdnnHandle_t handle, hipStream_t *streamId);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetStream(miopenHandle_t handle, miopenAcceleratorQueue_t* streamID);
  // CHECK: status = miopenGetStream(handle, &streamId);
  status = hipdnnGetStream(handle, &streamId);

  // CHECK: miopenTensorDescriptor_t tensorDescriptor;
  // CHECK-NEXT: miopenTensorDescriptor_t filterDescriptor;
  hipdnnTensorDescriptor_t tensorDescriptor;
  hipdnnFilterDescriptor_t filterDescriptor;

  // CHECK: miopenConvolutionDescriptor_t convolutionDescriptor;
  hipdnnConvolutionDescriptor_t convolutionDescriptor;

  // CHECK: miopenPoolingDescriptor_t poolingDescriptor;
  hipdnnPoolingDescriptor_t poolingDescriptor;

  // CHECK: miopenLRNDescriptor_t LRNDescriptor;
  hipdnnLRNDescriptor_t LRNDescriptor;

  // CHECK: miopenActivationDescriptor_t activationDescriptor;
  hipdnnActivationDescriptor_t activationDescriptor;

  // CHECK: miopenRNNDescriptor_t RNNDescriptor;
  hipdnnRNNDescriptor_t RNNDescriptor;

  // CHECK: miopenCTCLossDescriptor_t CTCLossDescriptor;
  cudnnCTCLossDescriptor_t CTCLossDescriptor;

  // CHECK: miopenDropoutDescriptor_t DropoutDescriptor;
  hipdnnDropoutDescriptor_t DropoutDescriptor;

  // CHECK: miopenReduceTensorDescriptor_t ReduceTensorDescriptor;
  hipdnnReduceTensorDescriptor_t ReduceTensorDescriptor;

  // CHECK: miopenDataType_t dataType;
  // CHECK-NEXT: miopenDataType_t DATA_FLOAT = miopenFloat;
  // CHECK-NEXT: miopenDataType_t DATA_DOUBLE = miopenDouble;
  // CHECK-NEXT: miopenDataType_t DATA_HALF = miopenHalf;
  // CHECK-NEXT: miopenDataType_t DATA_INT8 = miopenInt8;
  // CHECK-NEXT: miopenDataType_t DATA_INT32 = miopenInt32;
  // CHECK-NEXT: miopenDataType_t DATA_INT8x4 = miopenInt8x4;
  // CHECK-NEXT: miopenDataType_t DATA_BFLOAT16 = miopenBFloat16;
  hipdnnDataType_t dataType;
  hipdnnDataType_t DATA_FLOAT = HIPDNN_DATA_FLOAT;
  hipdnnDataType_t DATA_DOUBLE = HIPDNN_DATA_DOUBLE;
  hipdnnDataType_t DATA_HALF = HIPDNN_DATA_HALF;
  hipdnnDataType_t DATA_INT8 = HIPDNN_DATA_INT8;
  hipdnnDataType_t DATA_INT32 = HIPDNN_DATA_INT32;
  hipdnnDataType_t DATA_INT8x4 = HIPDNN_DATA_INT8x4;
  hipdnnDataType_t DATA_BFLOAT16 = CUDNN_DATA_BFLOAT16;

  // CHECK: miopenRNNMode_t RNNMode;
  // CHECK-NEXT: miopenRNNMode_t RNN_RELU = miopenRNNRELU;
  // CHECK-NEXT: miopenRNNMode_t RNN_TANH = miopenRNNTANH;
  // CHECK-NEXT: miopenRNNMode_t LSTM = miopenLSTM;
  // CHECK-NEXT: miopenRNNMode_t GRU = miopenGRU;
  hipdnnRNNMode_t RNNMode;
  hipdnnRNNMode_t RNN_RELU = HIPDNN_RNN_RELU;
  hipdnnRNNMode_t RNN_TANH = HIPDNN_RNN_TANH;
  hipdnnRNNMode_t LSTM = HIPDNN_LSTM;
  hipdnnRNNMode_t GRU = HIPDNN_GRU;

  // CHECK: miopenTensorOp_t tensorOp;
  // CHECK-NEXT: miopenTensorOp_t OP_TENSOR_ADD = miopenTensorOpAdd;
  // CHECK-NEXT: miopenTensorOp_t OP_TENSOR_MUL = miopenTensorOpMul;
  // CHECK-NEXT: miopenTensorOp_t OP_TENSOR_MIN = miopenTensorOpMin;
  // CHECK-NEXT: miopenTensorOp_t OP_TENSOR_MAX = miopenTensorOpMax;
  hipdnnOpTensorOp_t tensorOp;
  hipdnnOpTensorOp_t OP_TENSOR_ADD = HIPDNN_OP_TENSOR_ADD;
  hipdnnOpTensorOp_t OP_TENSOR_MUL = HIPDNN_OP_TENSOR_MUL;
  hipdnnOpTensorOp_t OP_TENSOR_MIN = HIPDNN_OP_TENSOR_MIN;
  hipdnnOpTensorOp_t OP_TENSOR_MAX = HIPDNN_OP_TENSOR_MAX;

  // CHECK: miopenConvolutionMode_t convolutionMode;
  hipdnnConvolutionMode_t convolutionMode;

  // CHECK: miopenPoolingMode_t poolingMode;
  // CHECK-NEXT: miopenPoolingMode_t POOLING_MAX = miopenPoolingMax;
  hipdnnPoolingMode_t poolingMode;
  hipdnnPoolingMode_t POOLING_MAX = HIPDNN_POOLING_MAX;

  // CHECK: miopenRNNInputMode_t RNNInputMode;
  // CHECK-NEXT: miopenRNNInputMode_t LINEAR_INPUT = miopenRNNlinear;
  // CHECK-NEXT: miopenRNNInputMode_t SKIP_INPUT = miopenRNNskip;
  hipdnnRNNInputMode_t RNNInputMode;
  hipdnnRNNInputMode_t LINEAR_INPUT = HIPDNN_LINEAR_INPUT;
  hipdnnRNNInputMode_t SKIP_INPUT = HIPDNN_SKIP_INPUT;

  // CHECK: miopenRNNAlgo_t RNNAlgo;
  // CHECK-NEXT: miopenRNNAlgo_t RNN_ALGO_STANDARD = miopenRNNdefault;
  hipdnnRNNAlgo_t RNNAlgo;
  hipdnnRNNAlgo_t RNN_ALGO_STANDARD = HIPDNN_RNN_ALGO_STANDARD;

  // CHECK: miopenRNNBiasMode_t RNNBiasMode;
  // CHECK-NEXT: miopenRNNBiasMode_t RNN_NO_BIAS = miopenRNNNoBias;
  // CHECK-NEXT: miopenRNNBiasMode_t RNN_SINGLE_INP_BIAS = miopenRNNwithBias;
  // CHECK-NEXT: miopenRNNBiasMode_t RNN_DOUBLE_BIAS = miopenRNNwithBias;
  // CHECK-NEXT: miopenRNNBiasMode_t RNN_SINGLE_REC_BIAS = miopenRNNwithBias;
  hipdnnRNNBiasMode_t RNNBiasMode;
  hipdnnRNNBiasMode_t RNN_NO_BIAS = HIPDNN_RNN_NO_BIAS;
  hipdnnRNNBiasMode_t RNN_SINGLE_INP_BIAS = HIPDNN_RNN_WITH_BIAS;
  hipdnnRNNBiasMode_t RNN_DOUBLE_BIAS = HIPDNN_RNN_WITH_BIAS;
  hipdnnRNNBiasMode_t RNN_SINGLE_REC_BIAS = HIPDNN_RNN_WITH_BIAS;

  // CHECK: miopenLRNMode_t LRNMode;
  // CHECK-NEXT: miopenLRNMode_t LRN_CROSS_CHANNEL_DIM1 = miopenLRNCrossChannel;
  hipdnnLRNMode_t LRNMode;
  hipdnnLRNMode_t LRN_CROSS_CHANNEL_DIM1 = HIPDNN_LRN_CROSS_CHANNEL;

  // CHECK: miopenBatchNormMode_t batchNormMode;
  // CHECK-NEXT: miopenBatchNormMode_t BATCHNORM_PER_ACTIVATION = miopenBNPerActivation;
  // CHECK-NEXT: miopenBatchNormMode_t BATCHNORM_SPATIAL = miopenBNSpatial;
  hipdnnBatchNormMode_t batchNormMode;
  hipdnnBatchNormMode_t BATCHNORM_PER_ACTIVATION = HIPDNN_BATCHNORM_PER_ACTIVATION;
  hipdnnBatchNormMode_t BATCHNORM_SPATIAL = HIPDNN_BATCHNORM_SPATIAL;

  // CHECK: miopenActivationMode_t activationMode;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_RELU = miopenActivationRELU;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_TANH = miopenActivationTANH;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_CLIPPED_RELU = miopenActivationCLIPPEDRELU;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_ELU = miopenActivationELU;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_IDENTITY = miopenActivationPASTHRU;
  hipdnnActivationMode_t activationMode;
  hipdnnActivationMode_t ACTIVATION_RELU = HIPDNN_ACTIVATION_RELU;
  hipdnnActivationMode_t ACTIVATION_TANH = HIPDNN_ACTIVATION_TANH;
  hipdnnActivationMode_t ACTIVATION_CLIPPED_RELU = HIPDNN_ACTIVATION_CLIPPED_RELU;
  hipdnnActivationMode_t ACTIVATION_ELU = HIPDNN_ACTIVATION_ELU;
  hipdnnActivationMode_t ACTIVATION_IDENTITY = HIPDNN_ACTIVATION_PATHTRU;

  // CHECK: miopenSoftmaxAlgorithm_t softmaxAlgorithm;
  // CHECK-NEXT: miopenSoftmaxAlgorithm_t SOFTMAX_FAST = MIOPEN_SOFTMAX_FAST;
  // CHECK-NEXT: miopenSoftmaxAlgorithm_t SOFTMAX_ACCURATE = MIOPEN_SOFTMAX_ACCURATE;
  // CHECK-NEXT: miopenSoftmaxAlgorithm_t SOFTMAX_LOG = MIOPEN_SOFTMAX_LOG;
  hipdnnSoftmaxAlgorithm_t softmaxAlgorithm;
  hipdnnSoftmaxAlgorithm_t SOFTMAX_FAST = HIPDNN_SOFTMAX_FAST;
  hipdnnSoftmaxAlgorithm_t SOFTMAX_ACCURATE = HIPDNN_SOFTMAX_ACCURATE;
  hipdnnSoftmaxAlgorithm_t SOFTMAX_LOG = HIPDNN_SOFTMAX_LOG;

  // CHECK: miopenSoftmaxMode_t softmaxMode;
  // CHECK-NEXT: miopenSoftmaxMode_t SOFTMAX_MODE_INSTANCE = MIOPEN_SOFTMAX_MODE_INSTANCE;
  // CHECK-NEXT: miopenSoftmaxMode_t SOFTMAX_MODE_CHANNEL = MIOPEN_SOFTMAX_MODE_CHANNEL;
  hipdnnSoftmaxMode_t softmaxMode;
  hipdnnSoftmaxMode_t SOFTMAX_MODE_INSTANCE = HIPDNN_SOFTMAX_MODE_INSTANCE;
  hipdnnSoftmaxMode_t SOFTMAX_MODE_CHANNEL = HIPDNN_SOFTMAX_MODE_CHANNEL;

  // CHECK: miopenReduceTensorOp_t reduceTensorOp;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_ADD = MIOPEN_REDUCE_TENSOR_ADD;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_MUL = MIOPEN_REDUCE_TENSOR_MUL;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_MIN = MIOPEN_REDUCE_TENSOR_MIN;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_MAX = MIOPEN_REDUCE_TENSOR_MAX;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_AMAX = MIOPEN_REDUCE_TENSOR_AMAX;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_AVG = MIOPEN_REDUCE_TENSOR_AVG;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_NORM1 = MIOPEN_REDUCE_TENSOR_NORM1;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_NORM2 = MIOPEN_REDUCE_TENSOR_NORM2;
  hipdnnReduceTensorOp_t reduceTensorOp;
  hipdnnReduceTensorOp_t REDUCE_TENSOR_ADD = HIPDNN_REDUCE_TENSOR_ADD;
  hipdnnReduceTensorOp_t REDUCE_TENSOR_MUL = HIPDNN_REDUCE_TENSOR_MUL;
  hipdnnReduceTensorOp_t REDUCE_TENSOR_MIN = HIPDNN_REDUCE_TENSOR_MIN;
  hipdnnReduceTensorOp_t REDUCE_TENSOR_MAX = HIPDNN_REDUCE_TENSOR_MAX;
  hipdnnReduceTensorOp_t REDUCE_TENSOR_AMAX = HIPDNN_REDUCE_TENSOR_AMAX;
  hipdnnReduceTensorOp_t REDUCE_TENSOR_AVG = HIPDNN_REDUCE_TENSOR_AVG;
  hipdnnReduceTensorOp_t REDUCE_TENSOR_NORM1 = HIPDNN_REDUCE_TENSOR_NORM1;
  hipdnnReduceTensorOp_t REDUCE_TENSOR_NORM2 = HIPDNN_REDUCE_TENSOR_NORM2;

  // CHECK: miopenConvFwdAlgorithm_t convolutionFwdAlgo;
  // CHECK-NEXT: miopenConvFwdAlgorithm_t CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM = miopenConvolutionFwdAlgoImplicitGEMM;
  // CHECK-NEXT: miopenConvFwdAlgorithm_t CONVOLUTION_FWD_ALGO_GEMM = miopenConvolutionFwdAlgoGEMM;
  // CHECK-NEXT: miopenConvFwdAlgorithm_t CONVOLUTION_FWD_ALGO_DIRECT = miopenConvolutionFwdAlgoDirect;
  // CHECK-NEXT: miopenConvFwdAlgorithm_t CONVOLUTION_FWD_ALGO_FFT = miopenConvolutionFwdAlgoFFT;
  // CHECK-NEXT: miopenConvFwdAlgorithm_t CONVOLUTION_FWD_ALGO_WINOGRAD = miopenConvolutionFwdAlgoWinograd;
  hipdnnConvolutionFwdAlgo_t convolutionFwdAlgo;
  hipdnnConvolutionFwdAlgo_t CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
  hipdnnConvolutionFwdAlgo_t CONVOLUTION_FWD_ALGO_GEMM = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;
  hipdnnConvolutionFwdAlgo_t CONVOLUTION_FWD_ALGO_DIRECT = HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT;
  hipdnnConvolutionFwdAlgo_t CONVOLUTION_FWD_ALGO_FFT = HIPDNN_CONVOLUTION_FWD_ALGO_FFT;
  hipdnnConvolutionFwdAlgo_t CONVOLUTION_FWD_ALGO_WINOGRAD = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD;

  // CHECK: miopenNanPropagation_t nanPropagation_t;
  // CHECK-NEXT: miopenNanPropagation_t NOT_PROPAGATE_NAN = MIOPEN_NOT_PROPAGATE_NAN;
  // CHECK-NEXT: miopenNanPropagation_t PROPAGATE_NAN = MIOPEN_PROPAGATE_NAN;
  hipdnnNanPropagation_t nanPropagation_t;
  hipdnnNanPropagation_t NOT_PROPAGATE_NAN = HIPDNN_NOT_PROPAGATE_NAN;
  hipdnnNanPropagation_t PROPAGATE_NAN = HIPDNN_PROPAGATE_NAN;

  // CHECK: miopenReduceTensorIndices_t reduceTensorIndices;
  // CHECK-NEXT: miopenReduceTensorIndices_t REDUCE_TENSOR_NO_INDICES = MIOPEN_REDUCE_TENSOR_NO_INDICES;
  // CHECK-NEXT: miopenReduceTensorIndices_t REDUCE_TENSOR_FLATTENED_INDICES = MIOPEN_REDUCE_TENSOR_FLATTENED_INDICES;
  hipdnnReduceTensorIndices_t reduceTensorIndices;
  hipdnnReduceTensorIndices_t REDUCE_TENSOR_NO_INDICES = HIPDNN_REDUCE_TENSOR_NO_INDICES;
  hipdnnReduceTensorIndices_t REDUCE_TENSOR_FLATTENED_INDICES = HIPDNN_REDUCE_TENSOR_FLATTENED_INDICES;

  // CHECK: miopenIndicesType_t indicesType;
  // CHECK-NEXT: miopenIndicesType_t _32BIT_INDICES = MIOPEN_32BIT_INDICES;
  // CHECK-NEXT: miopenIndicesType_t _64BIT_INDICES = MIOPEN_64BIT_INDICES;
  // CHECK-NEXT: miopenIndicesType_t _16BIT_INDICES = MIOPEN_16BIT_INDICES;
  // CHECK-NEXT: miopenIndicesType_t _8BIT_INDICES = MIOPEN_8BIT_INDICES;
  hipdnnIndicesType_t indicesType;
  hipdnnIndicesType_t _32BIT_INDICES = HIPDNN_32BIT_INDICES;
  hipdnnIndicesType_t _64BIT_INDICES = HIPDNN_64BIT_INDICES;
  hipdnnIndicesType_t _16BIT_INDICES = HIPDNN_16BIT_INDICES;
  hipdnnIndicesType_t _8BIT_INDICES = HIPDNN_8BIT_INDICES;

  // CHECK: miopenConvBwdDataAlgorithm_t ConvolutionBwdDataAlgo_t;
  // CHECK-NEXT: miopenConvBwdDataAlgorithm_t CONVOLUTION_BWD_DATA_ALGO_0 = miopenConvolutionBwdDataAlgoGEMM;
  // CHECK-NEXT: miopenConvBwdDataAlgorithm_t CONVOLUTION_BWD_DATA_ALGO_1 = miopenConvolutionBwdDataAlgoDirect;
  // CHECK-NEXT: miopenConvBwdDataAlgorithm_t CONVOLUTION_BWD_DATA_ALGO_FFT = miopenConvolutionBwdDataAlgoFFT;
  // CHECK-NEXT: miopenConvBwdDataAlgorithm_t CONVOLUTION_BWD_DATA_ALGO_WINOGRAD = miopenConvolutionBwdDataAlgoWinograd;
  hipdnnConvolutionBwdDataAlgo_t ConvolutionBwdDataAlgo_t;
  hipdnnConvolutionBwdDataAlgo_t CONVOLUTION_BWD_DATA_ALGO_0 = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0;
  hipdnnConvolutionBwdDataAlgo_t CONVOLUTION_BWD_DATA_ALGO_1 = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1;
  hipdnnConvolutionBwdDataAlgo_t CONVOLUTION_BWD_DATA_ALGO_FFT = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_FFT;
  hipdnnConvolutionBwdDataAlgo_t CONVOLUTION_BWD_DATA_ALGO_WINOGRAD = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_WINOGRAD;

  // CHECK: miopenRNNDirectionMode_t DirectionMode;
  // CHECK-NEXT: miopenRNNDirectionMode_t UNIDIRECTIONAL = miopenRNNunidirection;
  // CHECK-NEXT: miopenRNNDirectionMode_t BIDIRECTIONAL = miopenRNNbidirection;
  hipdnnDirectionMode_t DirectionMode;
  hipdnnDirectionMode_t UNIDIRECTIONAL = HIPDNN_UNIDIRECTIONAL;
  hipdnnDirectionMode_t BIDIRECTIONAL = HIPDNN_BIDIRECTIONAL;

  // CHECK: miopenConvAlgoPerf_t ConvolutionFwdAlgoPerf_t;
  // CHECK-NEXT: miopenConvAlgoPerf_t ConvolutionFwdAlgoPerfStruct;
  hipdnnConvolutionFwdAlgoPerf_t ConvolutionFwdAlgoPerf_t;
  hipdnnConvolutionFwdAlgoPerf_t ConvolutionFwdAlgoPerfStruct;

  // CHECK: miopenConvAlgoPerf_t ConvolutionBwdDataAlgoPerf_t;
  // CHECK-NEXT: miopenConvAlgoPerf_t ConvolutionBwdDataAlgoPerfStruct;
  hipdnnConvolutionBwdDataAlgoPerf_t ConvolutionBwdDataAlgoPerf_t;
  hipdnnConvolutionBwdDataAlgoPerf_t ConvolutionBwdDataAlgoPerfStruct;

  // CHECK: miopenCTCLossAlgo_t CTCLossAlgo;
  // CHECK-NEXT: miopenCTCLossAlgo_t CTC_LOSS_ALGO_DETERMINISTIC = MIOPEN_CTC_LOSS_ALGO_DETERMINISTIC;
  cudnnCTCLossAlgo_t CTCLossAlgo;
  cudnnCTCLossAlgo_t CTC_LOSS_ALGO_DETERMINISTIC = CUDNN_CTC_LOSS_ALGO_DETERMINISTIC;

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnCreateTensorDescriptor(hipdnnTensorDescriptor_t* tensorDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreateTensorDescriptor(miopenTensorDescriptor_t* tensorDesc);
  // CHECK: status = miopenCreateTensorDescriptor(&tensorDescriptor);
  status = hipdnnCreateTensorDescriptor(&tensorDescriptor);

  // TODO: hipdnnSetTensor4dDescriptor -> miopenSet4dTensorDescriptor: different signatures
  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnSetTensor4dDescriptor(hipdnnTensorDescriptor_t tensorDesc, hipdnnTensorFormat_t format, hipdnnDataType_t dataType, int n, int c, int h, int w);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSet4dTensorDescriptor(miopenTensorDescriptor_t tensorDesc, miopenDataType_t dataType, int n, int c, int h, int w);

  int n = 0;
  int c = 0;
  int h = 0;
  int w = 0;
  int nStride = 0;
  int cStride = 0;
  int hStride = 0;
  int wStride = 0;

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnSetTensor4dDescriptorEx(hipdnnTensorDescriptor_t tensorDesc, hipdnnDataType_t dataType, int n, int c, int h, int w, int nStride, int cStride, int hStride, int wStride);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSet4dTensorDescriptorEx(miopenTensorDescriptor_t tensorDesc, miopenDataType_t dataType, int n, int c, int h, int w, int nStride, int cStride, int hStride, int wStride);
  // CHECK: status = miopenSet4dTensorDescriptorEx(tensorDescriptor, dataType, n, c, h, w, nStride, cStride, hStride, wStride);
  status = hipdnnSetTensor4dDescriptorEx(tensorDescriptor, dataType, n, c, h, w, nStride, cStride, hStride, wStride);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnGetTensor4dDescriptor(const hipdnnTensorDescriptor_t tensorDesc, hipdnnDataType_t* dataType, int* n, int* c, int* h, int* w, int* nStride, int* cStride, int* hStride, int* wStride);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGet4dTensorDescriptor(miopenTensorDescriptor_t tensorDesc, miopenDataType_t* dataType, int* n, int* c, int* h, int* w, int* nStride, int* cStride, int* hStride, int* wStride);
  // CHECK: status = miopenGet4dTensorDescriptor(tensorDescriptor, &dataType, &n, &c, &h, &w, &nStride, &cStride, &hStride, &wStride);
  status = hipdnnGetTensor4dDescriptor(tensorDescriptor, &dataType, &n, &c, &h, &w, &nStride, &cStride, &hStride, &wStride);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnDestroyTensorDescriptor(hipdnnTensorDescriptor_t tensorDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroyTensorDescriptor(miopenTensorDescriptor_t tensorDesc);
  // CHECK: status = miopenDestroyTensorDescriptor(tensorDescriptor);
  status = hipdnnDestroyTensorDescriptor(tensorDescriptor);

  // CHECK: miopenTensorDescriptor_t aD;
  // CHECK-NEXT: miopenTensorDescriptor_t bD;
  // CHECK-NEXT: miopenTensorDescriptor_t cD;
  // CHECK-NEXT: miopenTensorDescriptor_t xD;
  // CHECK-NEXT: miopenTensorDescriptor_t hxD;
  // CHECK-NEXT: miopenTensorDescriptor_t dhxD;
  // CHECK-NEXT: miopenTensorDescriptor_t cxD;
  // CHECK-NEXT: miopenTensorDescriptor_t dcxD;
  // CHECK-NEXT: miopenTensorDescriptor_t yD;
  // CHECK-NEXT: miopenTensorDescriptor_t dyD;
  // CHECK-NEXT: miopenTensorDescriptor_t hyD;
  // CHECK-NEXT: miopenTensorDescriptor_t dhyD;
  // CHECK-NEXT: miopenTensorDescriptor_t cyD;
  // CHECK-NEXT: miopenTensorDescriptor_t dcyD;
  // CHECK-NEXT: miopenTensorDescriptor_t wD;
  // CHECK-NEXT: miopenTensorDescriptor_t zD;
  // CHECK-NEXT: miopenTensorDescriptor_t inputD;
  // CHECK-NEXT: miopenTensorDescriptor_t dbD;
  // CHECK-NEXT: miopenTensorDescriptor_t dxD;
  // CHECK-NEXT: miopenTensorDescriptor_t biasD;
  // CHECK-NEXT: miopenTensorDescriptor_t probsD;
  // CHECK-NEXT: miopenTensorDescriptor_t gradientsD;
  hipdnnTensorDescriptor_t aD;
  hipdnnTensorDescriptor_t bD;
  hipdnnTensorDescriptor_t cD;
  hipdnnTensorDescriptor_t xD;
  hipdnnTensorDescriptor_t hxD;
  hipdnnTensorDescriptor_t dhxD;
  hipdnnTensorDescriptor_t cxD;
  hipdnnTensorDescriptor_t dcxD;
  hipdnnTensorDescriptor_t yD;
  hipdnnTensorDescriptor_t dyD;
  hipdnnTensorDescriptor_t hyD;
  hipdnnTensorDescriptor_t dhyD;
  hipdnnTensorDescriptor_t cyD;
  hipdnnTensorDescriptor_t dcyD;
  hipdnnTensorDescriptor_t wD;
  hipdnnTensorDescriptor_t zD;
  hipdnnTensorDescriptor_t inputD;
  hipdnnTensorDescriptor_t dbD;
  hipdnnTensorDescriptor_t dxD;
  hipdnnTensorDescriptor_t biasD;
  hipdnnTensorDescriptor_t probsD;
  hipdnnTensorDescriptor_t gradientsD;
  void* A = nullptr;
  void* B = nullptr;
  void* C = nullptr;
  void* alpha = nullptr;
  void* alpha1 = nullptr;
  void* alpha2 = nullptr;
  void* beta = nullptr;
  void* x = nullptr;
  void* dx = nullptr;
  void* hx = nullptr;
  void* dhx = nullptr;
  void* cx = nullptr;
  void* dcx = nullptr;
  void* y = nullptr;
  void* dy = nullptr;
  void* hy = nullptr;
  void* cy = nullptr;
  void* dcy = nullptr;
  void* z = nullptr;
  void* dhy = nullptr;
  void* W = nullptr;
  void* dw = nullptr;
  void* db = nullptr;
  void* bias = nullptr;
  void* workSpace = nullptr;
  void* indices = nullptr;
  void* reserveSpace = nullptr;
  void* probs = nullptr;
  void* gradients = nullptr;
  void* losses = nullptr;
  int groupCount = 0;
  int requestedAlgoCount = 0;
  int returnedAlgoCount = 0;
  size_t workSpaceSizeInBytes = 0;
  size_t reserveSpaceNumBytes = 0;
  size_t indicesSizeInBytes = 0;

  // TODO: hipdnnOpTensor -> miopenOpTensor: different signatures: hipdnnOpTensorDescriptor_t != miopenTensorOp_t
  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnOpTensor(hipdnnHandle_t handle, const hipdnnOpTensorDescriptor_t opTensorDesc, const void* alpha1, const hipdnnTensorDescriptor_t aDesc, const void* A, const void* alpha2, const hipdnnTensorDescriptor_t bDesc, const void* B, const void* beta, const hipdnnTensorDescriptor_t cDesc, void* C);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenOpTensor(miopenHandle_t handle, miopenTensorOp_t tensorOp, const void* alpha1, const miopenTensorDescriptor_t aDesc, const void* A, const void* alpha2, const miopenTensorDescriptor_t bDesc, const void* B, const void* beta, const miopenTensorDescriptor_t cDesc, void* C);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnSetTensor(hipdnnHandle_t handle, const hipdnnTensorDescriptor_t yDesc, void* y, const void* valuePtr);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSetTensor(miopenHandle_t handle, const miopenTensorDescriptor_t yDesc, void* y, const void* alpha);
  // CHECK: status = miopenSetTensor(handle, tensorDescriptor, y, alpha);
  status = hipdnnSetTensor(handle, tensorDescriptor, y, alpha);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnScaleTensor(hipdnnHandle_t handle, const hipdnnTensorDescriptor_t yDesc, void* y, const void* alpha);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenScaleTensor(miopenHandle_t handle, const miopenTensorDescriptor_t yDesc, void* y, const void* alpha);
  // CHECK: status = miopenScaleTensor(handle, tensorDescriptor, y, alpha);
  status = hipdnnScaleTensor(handle, tensorDescriptor, y, alpha);

  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnTransformTensor(hipdnnHandle_t handle, const void* alpha, const hipdnnTensorDescriptor_t xDesc, const void* x, const void* beta, const hipdnnTensorDescriptor_t yDesc, void* y);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenTransformTensor(miopenHandle_t handle, const void* alpha, const miopenTensorDescriptor_t xDesc, const void* x, const void* beta, const miopenTensorDescriptor_t yDesc, void* y);
  // CHECK: status = miopenTransformTensor(handle, alpha, xD, x, beta, yD, y);
  status = cudnnTransformTensor(handle, alpha, xD, x, beta, yD, y);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnCreateConvolutionDescriptor(hipdnnConvolutionDescriptor_t* convDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreateConvolutionDescriptor(miopenConvolutionDescriptor_t* convDesc);
  // CHECK: status = miopenCreateConvolutionDescriptor(&convolutionDescriptor);
  status = hipdnnCreateConvolutionDescriptor(&convolutionDescriptor);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnSetConvolutionGroupCount(hipdnnConvolutionDescriptor_t convDesc, int groupCount);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSetConvolutionGroupCount(miopenConvolutionDescriptor_t convDesc, int groupCount);
  // CHECK: status = miopenSetConvolutionGroupCount(convolutionDescriptor, groupCount);
  status = hipdnnSetConvolutionGroupCount(convolutionDescriptor, groupCount);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnGetConvolution2dForwardOutputDim(const hipdnnConvolutionDescriptor_t convDesc, const hipdnnTensorDescriptor_t inputTensorDesc, const hipdnnFilterDescriptor_t filterDesc, int* n, int* c, int* h, int* w);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetConvolutionForwardOutputDim(miopenConvolutionDescriptor_t convDesc, const miopenTensorDescriptor_t inputTensorDesc, const miopenTensorDescriptor_t filterDesc, int* n, int* c, int* h, int* w);
  // CHECK: status = miopenGetConvolutionForwardOutputDim(convolutionDescriptor, inputD, filterDescriptor, &n, &c, &h, &w);
  status = hipdnnGetConvolution2dForwardOutputDim(convolutionDescriptor, inputD, filterDescriptor, &n, &c, &h, &w);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnDestroyConvolutionDescriptor(hipdnnConvolutionDescriptor_t convDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroyConvolutionDescriptor(miopenConvolutionDescriptor_t convDesc);
  // CHECK: status = miopenDestroyConvolutionDescriptor(convolutionDescriptor);
  status = hipdnnDestroyConvolutionDescriptor(convolutionDescriptor);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnFindConvolutionForwardAlgorithmEx(hipdnnHandle_t handle, const hipdnnTensorDescriptor_t xDesc, const void* x, const hipdnnFilterDescriptor_t wDesc, const void* w, const hipdnnConvolutionDescriptor_t convDesc, const hipdnnTensorDescriptor_t yDesc, void* y, const int requestedAlgoCount, int* returnedAlgoCount, hipdnnConvolutionFwdAlgoPerf_t* perfResults, void* workSpace, size_t workSpaceSizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenFindConvolutionForwardAlgorithm(miopenHandle_t handle, const miopenTensorDescriptor_t xDesc, const void* x, const miopenTensorDescriptor_t wDesc, const void* w, const miopenConvolutionDescriptor_t convDesc, const miopenTensorDescriptor_t yDesc, void* y, const int requestAlgoCount, int* returnedAlgoCount, miopenConvAlgoPerf_t* perfResults, void* workSpace, size_t workSpaceSize, bool exhaustiveSearch);
  // CHECK: status = miopenFindConvolutionForwardAlgorithm(handle, xD, x, filterDescriptor, W, convolutionDescriptor, yD, y, requestedAlgoCount, &returnedAlgoCount, &ConvolutionFwdAlgoPerf_t, workSpace, workSpaceSizeInBytes, true);
  status = hipdnnFindConvolutionForwardAlgorithmEx(handle, xD, x, filterDescriptor, W, convolutionDescriptor, yD, y, requestedAlgoCount, &returnedAlgoCount, &ConvolutionFwdAlgoPerf_t, workSpace, workSpaceSizeInBytes);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnGetConvolutionForwardWorkspaceSize(hipdnnHandle_t handle, const hipdnnTensorDescriptor_t xDesc, const hipdnnFilterDescriptor_t wDesc, const hipdnnConvolutionDescriptor_t convDesc, const hipdnnTensorDescriptor_t yDesc, hipdnnConvolutionFwdAlgo_t algo, size_t* sizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenConvolutionForwardGetWorkSpaceSize(miopenHandle_t handle, const miopenTensorDescriptor_t wDesc, const miopenTensorDescriptor_t xDesc, const miopenConvolutionDescriptor_t convDesc, const miopenTensorDescriptor_t yDesc, size_t* workSpaceSize);
  // CHECK: status = miopenConvolutionForwardGetWorkSpaceSize(handle, filterDescriptor, xD, convolutionDescriptor, yD, &workSpaceSizeInBytes);
  status = hipdnnGetConvolutionForwardWorkspaceSize(handle, xD, filterDescriptor, convolutionDescriptor, yD, convolutionFwdAlgo, &workSpaceSizeInBytes);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnConvolutionForward(hipdnnHandle_t handle, const void* alpha, const hipdnnTensorDescriptor_t xDesc, const void* x, const hipdnnFilterDescriptor_t wDesc, const void* w, const hipdnnConvolutionDescriptor_t convDesc, hipdnnConvolutionFwdAlgo_t algo, void* workSpace, size_t workSpaceSizeInBytes, const void* beta, const hipdnnTensorDescriptor_t yDesc, void* y);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenConvolutionForward(miopenHandle_t handle, const void* alpha, const miopenTensorDescriptor_t xDesc, const void* x, const miopenTensorDescriptor_t wDesc, const void* w, const miopenConvolutionDescriptor_t convDesc, miopenConvFwdAlgorithm_t algo, const void* beta, const miopenTensorDescriptor_t yDesc, void* y, void* workSpace, size_t workSpaceSize);
  // CHECK: status = miopenConvolutionForward(handle, alpha, xD, x, filterDescriptor, W, convolutionDescriptor, convolutionFwdAlgo, beta, yD, y, workSpace, workSpaceSizeInBytes);
  status = hipdnnConvolutionForward(handle, alpha, xD, x, filterDescriptor, W, convolutionDescriptor, convolutionFwdAlgo, workSpace, workSpaceSizeInBytes, beta, yD, y);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnGetConvolutionBackwardDataWorkspaceSize(hipdnnHandle_t handle, const hipdnnFilterDescriptor_t wDesc, const hipdnnTensorDescriptor_t dyDesc, const hipdnnConvolutionDescriptor_t convDesc, const hipdnnTensorDescriptor_t dxDesc, hipdnnConvolutionBwdDataAlgo_t algo, size_t* sizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenConvolutionBackwardDataGetWorkSpaceSize(miopenHandle_t handle, const miopenTensorDescriptor_t dyDesc, const miopenTensorDescriptor_t wDesc, const miopenConvolutionDescriptor_t convDesc, const miopenTensorDescriptor_t dxDesc, size_t* workSpaceSize);
  // CHECK: status = miopenConvolutionBackwardDataGetWorkSpaceSize(handle, yD, filterDescriptor, convolutionDescriptor, xD, &workSpaceSizeInBytes);
  status = hipdnnGetConvolutionBackwardDataWorkspaceSize(handle, filterDescriptor, yD, convolutionDescriptor, xD, ConvolutionBwdDataAlgo_t, &workSpaceSizeInBytes);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnConvolutionBackwardData(hipdnnHandle_t handle, const void* alpha, const hipdnnFilterDescriptor_t wDesc, const void* w, const hipdnnTensorDescriptor_t dyDesc, const void* dy, const hipdnnConvolutionDescriptor_t convDesc, hipdnnConvolutionBwdDataAlgo_t algo, void* workSpace, size_t workSpaceSizeInBytes, const void* beta, const hipdnnTensorDescriptor_t dxDesc, void* dx);
  // MIOPEN MIOPEN_EXPORT miopenStatus_t miopenConvolutionBackwardData(miopenHandle_t handle, const void* alpha, const miopenTensorDescriptor_t dyDesc, const void* dy, const miopenTensorDescriptor_t wDesc, const void* w, const miopenConvolutionDescriptor_t convDesc, miopenConvBwdDataAlgorithm_t algo, const void* beta, const miopenTensorDescriptor_t dxDesc, void* dx, void* workSpace, size_t workSpaceSize);
  // CHECK: status = miopenConvolutionBackwardData(handle, alpha, yD, dy, filterDescriptor, W, convolutionDescriptor, ConvolutionBwdDataAlgo_t, beta, xD, dx, workSpace, workSpaceSizeInBytes);
  status = hipdnnConvolutionBackwardData(handle, alpha, filterDescriptor, W, yD, dy, convolutionDescriptor, ConvolutionBwdDataAlgo_t, workSpace, workSpaceSizeInBytes, beta, xD, dx);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnConvolutionBackwardBias(hipdnnHandle_t handle, const void* alpha, const hipdnnTensorDescriptor_t dyDesc, const void* dy, const void* beta, const hipdnnTensorDescriptor_t dbDesc, void* db);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenConvolutionBackwardBias(miopenHandle_t handle, const void* alpha, const miopenTensorDescriptor_t dyDesc, const void* dy, const void* beta, const miopenTensorDescriptor_t dbDesc, void* db);
  // CHECK: status = miopenConvolutionBackwardBias(handle, alpha, yD, dy, beta, dbD, db);
  status = hipdnnConvolutionBackwardBias(handle, alpha, yD, dy, beta, dbD, db);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnCreatePoolingDescriptor(hipdnnPoolingDescriptor_t* poolingDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreatePoolingDescriptor(miopenPoolingDescriptor_t* poolDesc);
  // CHECK: status = miopenCreatePoolingDescriptor(&poolingDescriptor);
  status = hipdnnCreatePoolingDescriptor(&poolingDescriptor);

  // CHECK: miopenNanPropagation_t maxpoolingNanOpt;
  hipdnnNanPropagation_t maxpoolingNanOpt;
  int wH = 0;
  int wW = 0;
  int pad_h = 0;
  int pad_w = 0;
  int stride_h = 0;
  int stride_w = 0;

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnSetPooling2dDescriptor(hipdnnPoolingDescriptor_t poolingDesc, hipdnnPoolingMode_t mode, hipdnnNanPropagation_t maxpoolingNanOpt, int windowHeight, int windowWidth, int verticalPadding, int horizontalPadding, int verticalStride, int horizontalStride);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSet2dPoolingDescriptor(miopenPoolingDescriptor_t poolDesc, miopenPoolingMode_t mode, int windowHeight, int windowWidth, int pad_h, int pad_w, int stride_h, int stride_w);
  // CHECK: status = miopenSet2dPoolingDescriptor(poolingDescriptor, poolingMode, wH, wW, pad_h, pad_w, stride_h, stride_w);
  status = hipdnnSetPooling2dDescriptor(poolingDescriptor, poolingMode, maxpoolingNanOpt, wH, wW, pad_h, pad_w, stride_h, stride_w);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnGetPooling2dDescriptor(const hipdnnPoolingDescriptor_t poolingDesc, hipdnnPoolingMode_t* mode, hipdnnNanPropagation_t* maxpoolingNanOpt, int* windowHeight, int* windowWidth, int* verticalPadding, int* horizontalPadding, int* verticalStride, int* horizontalStride);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGet2dPoolingDescriptor(const miopenPoolingDescriptor_t poolDesc, miopenPoolingMode_t* mode, int* windowHeight, int* windowWidth, int* pad_h, int* pad_w, int* stride_h, int* stride_w);
  // CHECK: status = miopenGet2dPoolingDescriptor(poolingDescriptor, &poolingMode, &wH, &wW, &pad_h, &pad_w, &stride_h, &stride_w);
  status = hipdnnGetPooling2dDescriptor(poolingDescriptor, &poolingMode, &maxpoolingNanOpt, &wH, &wW, &pad_h, &pad_w, &stride_h, &stride_w);

  int nbDims = 0;
  int nbDimsRequested = 0;
  int* windowDimA = nullptr;
  int* padA = nullptr;
  int* stridesA = nullptr;
  int* tensorDimArr = nullptr;

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnGetPooling2dForwardOutputDim(const hipdnnPoolingDescriptor_t poolingDesc, const hipdnnTensorDescriptor_t inputTensorDesc, int* n, int* c, int* h, int* w);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetPoolingForwardOutputDim(const miopenPoolingDescriptor_t poolDesc, const miopenTensorDescriptor_t tensorDesc, int* n, int* c, int* h, int* w);
  // CHECK: status = miopenGetPoolingForwardOutputDim(poolingDescriptor, tensorDescriptor, &n, &c, &h, &w);
  status = hipdnnGetPooling2dForwardOutputDim(poolingDescriptor, tensorDescriptor, &n, &c, &h, &w);

  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnGetPoolingNdForwardOutputDim(const hipdnnPoolingDescriptor_t poolingDesc, const hipdnnTensorDescriptor_t inputTensorDesc, int nbDims, int outputTensorDimA[]);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetPoolingNdForwardOutputDim(const miopenPoolingDescriptor_t poolDesc, const miopenTensorDescriptor_t tensorDesc, int dims, int* tensorDimArr);
  // CHECK: status = miopenGetPoolingNdForwardOutputDim(poolingDescriptor, tensorDescriptor, nbDims, tensorDimArr);
  status = cudnnGetPoolingNdForwardOutputDim(poolingDescriptor, tensorDescriptor, nbDims, tensorDimArr);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnSetPoolingNdDescriptor(hipdnnPoolingDescriptor_t poolingDesc, const hipdnnPoolingMode_t mode, const hipdnnNanPropagation_t maxpoolingNanOpt, int nbDims, const int windowDimA[], const int paddingA[], const int strideA[]);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSetNdPoolingDescriptor(miopenPoolingDescriptor_t poolDesc, const miopenPoolingMode_t mode, int nbDims, int* windowDimA, int* padA, int* stridesA);
  // CHECK: status = miopenSetNdPoolingDescriptor(poolingDescriptor, poolingMode, nbDims, windowDimA, padA, stridesA);
  status = hipdnnSetPoolingNdDescriptor(poolingDescriptor, poolingMode, maxpoolingNanOpt, nbDims, windowDimA, padA, stridesA);

  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnGetPoolingNdDescriptor(const hipdnnPoolingDescriptor_t poolingDesc, int nbDimsRequested, hipdnnPoolingMode_t* mode, hipdnnNanPropagation_t* maxpoolingNanOpt, int* nbDims, int windowDimA[], int paddingA[], int strideA[]);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetNdPoolingDescriptor(const miopenPoolingDescriptor_t poolDesc, int nbDimsRequested, miopenPoolingMode_t* mode, int* nbDims, int* windowDimA, int* padA, int* stridesA);
  // CHECK: status = miopenGetNdPoolingDescriptor(poolingDescriptor, nbDimsRequested, &poolingMode, &nbDims, windowDimA, padA, stridesA);
  status = cudnnGetPoolingNdDescriptor(poolingDescriptor, nbDimsRequested, &poolingMode, &maxpoolingNanOpt, &nbDims, windowDimA, padA, stridesA);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnDestroyPoolingDescriptor(hipdnnPoolingDescriptor_t poolingDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroyPoolingDescriptor(miopenPoolingDescriptor_t poolDesc);
  // CHECK: status = miopenDestroyPoolingDescriptor(poolingDescriptor);
  status = hipdnnDestroyPoolingDescriptor(poolingDescriptor);

  unsigned lrnN = 0;
  double lrnAlpha = 0.0f;
  double lrnBeta = 0.0f;
  double lrnK = 0.0f;

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnCreateLRNDescriptor(hipdnnLRNDescriptor_t* normDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreateLRNDescriptor(miopenLRNDescriptor_t* lrnDesc);
  // CHECK: status = miopenCreateLRNDescriptor(&LRNDescriptor);
  status = hipdnnCreateLRNDescriptor(&LRNDescriptor);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnSetLRNDescriptor(hipdnnLRNDescriptor_t normDesc, unsigned lrnN, double lrnAlpha, double lrnBeta, double lrnK);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSetLRNDescriptor(const miopenLRNDescriptor_t lrnDesc, miopenLRNMode_t mode, unsigned int lrnN, double lrnAlpha, double lrnBeta, double lrnK);
  // CHECK: status = miopenSetLRNDescriptor(LRNDescriptor, miopenLRNCrossChannel, lrnN, lrnAlpha, lrnBeta, lrnK);
  status = hipdnnSetLRNDescriptor(LRNDescriptor, lrnN, lrnAlpha, lrnBeta, lrnK);

  // TODO: add a referrence to miopenLRNMode_t as a 2nd arg
  // TODO: [feature] Add a new type of transformation by declaring a var before the function call to add that var reference as an arg to the below function call
  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnGetLRNDescriptor(hipdnnLRNDescriptor_t normDesc, unsigned* lrnN, double* lrnAlpha, double* lrnBeta, double* lrnK);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetLRNDescriptor(const miopenLRNDescriptor_t lrnDesc, miopenLRNMode_t* mode, unsigned int* lrnN, double* lrnAlpha, double* lrnBeta, double* lrnK);
  // CHECK: status = miopenGetLRNDescriptor(LRNDescriptor, &lrnN, &lrnAlpha, &lrnBeta, &lrnK);
  status = hipdnnGetLRNDescriptor(LRNDescriptor, &lrnN, &lrnAlpha, &lrnBeta, &lrnK);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnDestroyLRNDescriptor(hipdnnLRNDescriptor_t lrnDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroyLRNDescriptor(miopenLRNDescriptor_t lrnDesc);
  // CHECK: status = miopenDestroyLRNDescriptor(LRNDescriptor);
  status = hipdnnDestroyLRNDescriptor(LRNDescriptor);

  // CHECK: miopenTensorDescriptor_t bnScaleBiasMeanVarDesc;
  // CHECK: miopenTensorDescriptor_t bnScaleBiasDiffDesc;
  hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc;
  hipdnnTensorDescriptor_t bnScaleBiasDiffDesc;
  void *bnScale = nullptr;
  void *bnBias = nullptr;
  double expAvgFactor = 0.0f;
  void *resultRunningMean = nullptr;
  void *resultRunningVariance = nullptr;
  double epsilon = 0.0f;
  void *resultSaveMean = nullptr;
  void *resultSaveInvVariance = nullptr;
  void *estimatedMean = nullptr;
  void *estimatedVariance = nullptr;
  void *alphaDataDiff = nullptr;
  void *betaDataDiff = nullptr;
  void *alphaParamDiff = nullptr;
  void *betaParamDiff = nullptr;
  void *resultBnScaleDiff = nullptr;
  void *resultBnBiasDiff = nullptr;
  void *savedMean = nullptr;
  void *savedInvVariance = nullptr;

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnDeriveBNTensorDescriptor(hipdnnTensorDescriptor_t derivedBnDesc, const hipdnnTensorDescriptor_t xDesc, hipdnnBatchNormMode_t mode);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDeriveBNTensorDescriptor(miopenTensorDescriptor_t derivedBnDesc, const miopenTensorDescriptor_t xDesc, miopenBatchNormMode_t bn_mode);
  // CHECK: status = miopenDeriveBNTensorDescriptor(tensorDescriptor, xD, batchNormMode);
  status = hipdnnDeriveBNTensorDescriptor(tensorDescriptor, xD, batchNormMode);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnBatchNormalizationForwardTraining(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, const void* alpha, const void* beta, const hipdnnTensorDescriptor_t xDesc, const void* x, const hipdnnTensorDescriptor_t yDesc, void* y, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void* bnScale, const void* bnBias, double exponentialAverageFactor, void* resultRunningMean, double epsilon, void* resultSaveMean, void* resultSaveInvVariance);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenBatchNormalizationForwardTraining(miopenHandle_t handle, miopenBatchNormMode_t bn_mode, void* alpha, void* beta, const miopenTensorDescriptor_t xDesc, const void* x, const miopenTensorDescriptor_t yDesc, void* y, const miopenTensorDescriptor_t bnScaleBiasMeanVarDesc, void* bnScale, void* bnBias, double expAvgFactor, void* resultRunningMean, void* resultRunningVariance, double epsilon, void* resultSaveMean, void* resultSaveInvVariance);
  // CHECK: status = miopenBatchNormalizationForwardTraining(handle, batchNormMode, alpha, beta, xD, x, yD, y, bnScaleBiasMeanVarDesc, bnScale, bnBias, expAvgFactor, resultRunningMean, resultRunningVariance, epsilon, resultSaveMean, resultSaveInvVariance);
  status = hipdnnBatchNormalizationForwardTraining(handle, batchNormMode, alpha, beta, xD, x, yD, y, bnScaleBiasMeanVarDesc, bnScale, bnBias, expAvgFactor, resultRunningMean, resultRunningVariance, epsilon, resultSaveMean, resultSaveInvVariance);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnBatchNormalizationForwardInference(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, const void* alpha, const void* beta, const hipdnnTensorDescriptor_t xDesc, const void* x, const hipdnnTensorDescriptor_t yDesc, void* y, const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc, const void* bnScale, const void* bnBias, const void* estimatedMean, const void* estimatedVariance, double epsilon);
  // MIOPEN: miopenBatchNormalizationForwardInference(miopenHandle_t handle, miopenBatchNormMode_t bn_mode, void* alpha, void* beta, const miopenTensorDescriptor_t xDesc, const void* x, const miopenTensorDescriptor_t yDesc, void* y, const miopenTensorDescriptor_t bnScaleBiasMeanVarDesc, void* bnScale, void* bnBias, void* estimatedMean, void* estimatedVariance, double epsilon);
  // CHECK: status = miopenBatchNormalizationForwardInference(handle, batchNormMode, alpha, beta, xD, x, yD, y, bnScaleBiasMeanVarDesc, bnScale, bnBias, estimatedMean, estimatedVariance, epsilon);
  status = hipdnnBatchNormalizationForwardInference(handle, batchNormMode, alpha, beta, xD, x, yD, y, bnScaleBiasMeanVarDesc, bnScale, bnBias, estimatedMean, estimatedVariance, epsilon);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnBatchNormalizationBackward(hipdnnHandle_t handle, hipdnnBatchNormMode_t mode, const void* alphaDataDiff, const void* betaDataDiff, const void* alphaParamDiff, const void* betaParamDiff, const hipdnnTensorDescriptor_t xDesc, const void* x, const hipdnnTensorDescriptor_t dyDesc, const void* dy, const hipdnnTensorDescriptor_t dxDesc, void* dx, const hipdnnTensorDescriptor_t dBnScaleBiasDesc, const void* bnScale, void* dBnScaleResult, void* dBnBiasResult, double epsilon, const void* savedMean, const void* savedInvVariance);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenBatchNormalizationBackward(miopenHandle_t handle, miopenBatchNormMode_t bn_mode, const void* alphaDataDiff, const void* betaDataDiff, const void* alphaParamDiff, const void* betaParamDiff, const miopenTensorDescriptor_t xDesc, const void* x, const miopenTensorDescriptor_t dyDesc, const void* dy, const miopenTensorDescriptor_t dxDesc, void* dx, const miopenTensorDescriptor_t bnScaleBiasDiffDesc, const void* bnScale, void* resultBnScaleDiff, void* resultBnBiasDiff, double epsilon, const void* savedMean, const void* savedInvVariance);
  // CHECK: status = miopenBatchNormalizationBackward(handle, batchNormMode, alphaDataDiff, betaDataDiff, alphaParamDiff, betaParamDiff, xD, x, yD, y, dxD, dx, bnScaleBiasDiffDesc, bnScale, resultBnScaleDiff, resultBnBiasDiff, epsilon, savedMean, savedInvVariance);
  status = hipdnnBatchNormalizationBackward(handle, batchNormMode, alphaDataDiff, betaDataDiff, alphaParamDiff, betaParamDiff, xD, x, yD, y, dxD, dx, bnScaleBiasDiffDesc, bnScale, resultBnScaleDiff, resultBnBiasDiff, epsilon, savedMean, savedInvVariance);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnCreateActivationDescriptor(hipdnnActivationDescriptor_t* activationDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreateActivationDescriptor(miopenActivationDescriptor_t* activDesc);
  // CHECK: status = miopenCreateActivationDescriptor(&activationDescriptor);
  status = hipdnnCreateActivationDescriptor(&activationDescriptor);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnDestroyActivationDescriptor(hipdnnActivationDescriptor_t activationDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroyActivationDescriptor(miopenActivationDescriptor_t activDesc);
  // CHECK: status = miopenDestroyActivationDescriptor(activationDescriptor);
  status = hipdnnDestroyActivationDescriptor(activationDescriptor);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnActivationForward(hipdnnHandle_t handle, hipdnnActivationDescriptor_t activationDesc, const void* alpha, const hipdnnTensorDescriptor_t xDesc, const void* x, const void* beta, const hipdnnTensorDescriptor_t yDesc, void* y);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenActivationForward(miopenHandle_t handle, const miopenActivationDescriptor_t activDesc, const void* alpha, const miopenTensorDescriptor_t xDesc, const void* x, const void* beta, const miopenTensorDescriptor_t yDesc, void* y);
  // CHECK: status = miopenActivationForward(handle, activationDescriptor, alpha, xD, x, beta, yD, y);
  status = hipdnnActivationForward(handle, activationDescriptor, alpha, xD, x, beta, yD, y);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnActivationBackward(hipdnnHandle_t handle, hipdnnActivationDescriptor_t activationDesc, const void* alpha, const hipdnnTensorDescriptor_t yDesc, const void* y, const hipdnnTensorDescriptor_t dyDesc, const void* dy, const hipdnnTensorDescriptor_t xDesc, const void* x, const void* beta, const hipdnnTensorDescriptor_t dxDesc, void* dx);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenActivationBackward(miopenHandle_t handle, const miopenActivationDescriptor_t activDesc, const void* alpha, const miopenTensorDescriptor_t yDesc, const void* y, const miopenTensorDescriptor_t dyDesc, const void* dy, const miopenTensorDescriptor_t xDesc, const void* x, const void* beta, const miopenTensorDescriptor_t dxDesc, void* dx);
  // CHECK: status = miopenActivationBackward(handle, activationDescriptor, alpha, yD, y, dyD, dy, xD, x, beta, dxD, dx);
  status = hipdnnActivationBackward(handle, activationDescriptor, alpha, yD, y, dyD, dy, xD, x, beta, dxD, dx);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnSoftmaxForward(hipdnnHandle_t handle, hipdnnSoftmaxAlgorithm_t algo, hipdnnSoftmaxMode_t mode, const void* alpha, const hipdnnTensorDescriptor_t xDesc, const void* x, const void* beta, const hipdnnTensorDescriptor_t yDesc, void* y);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSoftmaxForward_V2(miopenHandle_t handle, const void* alpha, const miopenTensorDescriptor_t xDesc, const void* x, const void* beta, const miopenTensorDescriptor_t yDesc, void* y, miopenSoftmaxAlgorithm_t algorithm, miopenSoftmaxMode_t mode);
  // CHECK: status = miopenSoftmaxForward_V2(handle, alpha, xD, x, beta, yD, y, softmaxAlgorithm, softmaxMode);
  status = hipdnnSoftmaxForward(handle, softmaxAlgorithm, softmaxMode, alpha, xD, x, beta, yD, y);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnSoftmaxBackward(hipdnnHandle_t handle, hipdnnSoftmaxAlgorithm_t algo, hipdnnSoftmaxMode_t mode, const void* alpha, const hipdnnTensorDescriptor_t yDesc, const void* y, const hipdnnTensorDescriptor_t dyDesc, const void* dy, const void* beta, const hipdnnTensorDescriptor_t dxDesc, void* dx);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSoftmaxBackward_V2(miopenHandle_t handle, const void* alpha, const miopenTensorDescriptor_t yDesc, const void* y, const miopenTensorDescriptor_t dyDesc, const void* dy, const void* beta, const miopenTensorDescriptor_t dxDesc, void* dx, miopenSoftmaxAlgorithm_t algorithm, miopenSoftmaxMode_t mode);
  // CHECK: status = miopenSoftmaxBackward_V2(handle, alpha, yD, y, dyD, dy, beta, dxD, dx, softmaxAlgorithm, softmaxMode);
  status = hipdnnSoftmaxBackward(handle, softmaxAlgorithm, softmaxMode, alpha, yD, y, dyD, dy, beta, dxD, dx);

  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnConvolutionBiasActivationForward(hipdnnHandle_t handle, const void* alpha1, const hipdnnTensorDescriptor_t xDesc, const void* x, const hipdnnFilterDescriptor_t wDesc, const void* w, const hipdnnConvolutionDescriptor_t convDesc, hipdnnConvolutionFwdAlgo_t algo, void* workSpace, size_t workSpaceSizeInBytes, const void* alpha2, const hipdnnTensorDescriptor_t zDesc, const void* z, const hipdnnTensorDescriptor_t biasDesc, const void* bias, const hipdnnActivationDescriptor_t activationDesc, const hipdnnTensorDescriptor_t yDesc, void* y);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenConvolutionBiasActivationForward(miopenHandle_t handle, const void* alpha1, const miopenTensorDescriptor_t xDesc, const void* x, const miopenTensorDescriptor_t wDesc, const void* w, const miopenConvolutionDescriptor_t convDesc, miopenConvFwdAlgorithm_t algo, void* workspace, size_t workspaceSizeInBytes, const void* alpha2, const miopenTensorDescriptor_t zDesc, const void* z, const miopenTensorDescriptor_t biasDesc, const void* bias, const miopenActivationDescriptor_t activationDesc, const miopenTensorDescriptor_t yDesc, void* y);
  // CHECK: status = miopenConvolutionBiasActivationForward(handle, alpha1, xD, x, filterDescriptor, W, convolutionDescriptor, convolutionFwdAlgo, workSpace, workSpaceSizeInBytes, alpha2, zD, z, biasD, bias, activationDescriptor, yD, y);
  status = cudnnConvolutionBiasActivationForward(handle, alpha1, xD, x, filterDescriptor, W, convolutionDescriptor, convolutionFwdAlgo, workSpace, workSpaceSizeInBytes, alpha2, zD, z, biasD, bias, activationDescriptor, yD, y);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnCreateRNNDescriptor(hipdnnRNNDescriptor_t* rnnDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreateRNNDescriptor(miopenRNNDescriptor_t* rnnDesc);
  // CHECK: status = miopenCreateRNNDescriptor(&RNNDescriptor);
  status = hipdnnCreateRNNDescriptor(&RNNDescriptor);

  // NOTE: hipdnnGetRNNDescriptor - removed after cuDNN 7.6.5
  // TODO: add hipdnnGetRNNDescriptor -> miopenGetRNNDescriptor_V2 mapping after implementing cuDNN versioning in tests

  int hiddenSize = 0;
  int layer = 0;

  // TODO [#837]: Insert miopenRNNBiasMode_t* biasMode in the hipified miopenGetRNNDescriptor_V2 after miopenRNNMode_t* rnnMode: will need variable declaration
  // CUDA: CUDNN_DEPRECATED hipdnnStatus_t CUDNNWINAPI cudnnGetRNNDescriptor_v6(hipdnnHandle_t handle, hipdnnRNNDescriptor_t rnnDesc, int* hiddenSize, int* numLayers, hipdnnDropoutDescriptor_t* dropoutDesc, hipdnnRNNInputMode_t* inputMode, hipdnnDirectionMode_t* direction, hipdnnRNNMode_t* cellMode, hipdnnRNNAlgo_t* algo, hipdnnDataType_t* mathPrec);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetRNNDescriptor_V2(miopenRNNDescriptor_t rnnDesc, int* hiddenSize, int* layer, miopenDropoutDescriptor_t* dropoutDesc, miopenRNNInputMode_t* inputMode, miopenRNNDirectionMode_t* dirMode, miopenRNNMode_t* rnnMode, miopenRNNBiasMode_t* biasMode, miopenRNNAlgo_t* algoMode, miopenDataType_t* dataType);
  // CHECK: status = miopenGetRNNDescriptor_V2(RNNDescriptor, &hiddenSize, &layer, &DropoutDescriptor, &RNNInputMode, &DirectionMode, &RNNMode, &RNNAlgo, &dataType);
  status = cudnnGetRNNDescriptor_v6(handle, RNNDescriptor, &hiddenSize, &layer, &DropoutDescriptor, &RNNInputMode, &DirectionMode, &RNNMode, &RNNAlgo, &dataType);

  // NOTE: hipdnnSetRNNDescriptor - removed after cuDNN 7.6.5
  // NOTE: hipdnnSetRNNDescriptor_v5 - removed after cuDNN 7.6.5
  // TODO: add hipdnnSetRNNDescriptor -> miopenSetRNNDescriptor_V2 mapping after implementing cuDNN versioning in tests

  // TODO [#837]: Insert miopenRNNBiasMode_t biasMode in the hipified miopenSetRNNDescriptor_V2 after miopenRNNMode_t rnnMode: will need variable declaration
  // CUDA: CUDNN_DEPRECATED hipdnnStatus_t CUDNNWINAPI hipdnnSetRNNDescriptor_v6(hipdnnHandle_t handle, hipdnnRNNDescriptor_t rnnDesc, const int hiddenSize, const int numLayers, hipdnnDropoutDescriptor_t dropoutDesc, hipdnnRNNInputMode_t inputMode, hipdnnDirectionMode_t direction, hipdnnRNNMode_t cellMode, hipdnnRNNAlgo_t algo, hipdnnDataType_t mathPrec);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSetRNNDescriptor_V2(miopenRNNDescriptor_t rnnDesc, const int hsize, const int nlayers, miopenDropoutDescriptor_t dropoutDesc, miopenRNNInputMode_t inMode, miopenRNNDirectionMode_t direction, miopenRNNMode_t rnnMode, miopenRNNBiasMode_t biasMode, miopenRNNAlgo_t algo, miopenDataType_t dataType);
  // CHECK: status = miopenSetRNNDescriptor_V2(RNNDescriptor, hiddenSize, layer, DropoutDescriptor, RNNInputMode, DirectionMode, RNNMode, RNNAlgo, dataType);
  status = hipdnnSetRNNDescriptor_v6(handle, RNNDescriptor, hiddenSize, layer, DropoutDescriptor, RNNInputMode, DirectionMode, RNNMode, RNNAlgo, dataType);

  int seqLength = 0;

  // CUDA: CUDNN_DEPRECATED hipdnnStatus_t CUDNNWINAPI hipdnnGetRNNWorkspaceSize(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t* xDesc, size_t* sizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetRNNWorkspaceSize(miopenHandle_t handle, const miopenRNNDescriptor_t rnnDesc, const int sequenceLen, const miopenTensorDescriptor_t* xDesc, size_t* numBytes);
  // CHECK: status = miopenGetRNNWorkspaceSize(handle, RNNDescriptor, seqLength, &xD, &workSpaceSizeInBytes);
  status = hipdnnGetRNNWorkspaceSize(handle, RNNDescriptor, seqLength, &xD, &workSpaceSizeInBytes);

  // CUDA: CUDNN_DEPRECATED hipdnnStatus_t CUDNNWINAPI hipdnnGetRNNTrainingReserveSize(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t* xDesc, size_t* sizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetRNNTrainingReserveSize(miopenHandle_t handle, miopenRNNDescriptor_t rnnDesc, const int sequenceLen, const miopenTensorDescriptor_t* xDesc, size_t* numBytes);
  // CHECK: status = miopenGetRNNTrainingReserveSize(handle, RNNDescriptor, seqLength, &xD, &workSpaceSizeInBytes);
  status = hipdnnGetRNNTrainingReserveSize(handle, RNNDescriptor, seqLength, &xD, &workSpaceSizeInBytes);

  // CUDA: CUDNN_DEPRECATED hipdnnStatus_t CUDNNWINAPI hipdnnGetRNNParamsSize(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const hipdnnTensorDescriptor_t xDesc, size_t* sizeInBytes, hipdnnDataType_t dataType);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetRNNParamsSize(miopenHandle_t handle, miopenRNNDescriptor_t rnnDesc, miopenTensorDescriptor_t xDesc, size_t* numBytes, miopenDataType_t dtype);
  // CHECK: status = miopenGetRNNParamsSize(handle, RNNDescriptor, xD, &workSpaceSizeInBytes, dataType);
  status = hipdnnGetRNNParamsSize(handle, RNNDescriptor, xD, &workSpaceSizeInBytes, dataType);

  // CUDA: CUDNN_DEPRECATED hipdnnStatus_t CUDNNWINAPI hipdnnRNNForwardTraining(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t* xDesc, const void* x, const hipdnnTensorDescriptor_t hxDesc, const void* hx, const hipdnnTensorDescriptor_t cxDesc, const void* cx, const hipdnnFilterDescriptor_t wDesc, const void* w, const hipdnnTensorDescriptor_t* yDesc, void* y, const hipdnnTensorDescriptor_t hyDesc, void* hy, const hipdnnTensorDescriptor_t cyDesc, void* cy, void* workSpace, size_t workSpaceSizeInBytes, void* reserveSpace, size_t reserveSpaceSizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenRNNForwardTraining(miopenHandle_t handle, const miopenRNNDescriptor_t rnnDesc, const int sequenceLen, const miopenTensorDescriptor_t* xDesc, const void* x, const miopenTensorDescriptor_t hxDesc, const void* hx, const miopenTensorDescriptor_t cxDesc, const void* cx, const miopenTensorDescriptor_t wDesc, const void* w, const miopenTensorDescriptor_t* yDesc, void* y, const miopenTensorDescriptor_t hyDesc, void* hy, const miopenTensorDescriptor_t cyDesc, void* cy, void* workSpace, size_t workSpaceNumBytes, void* reserveSpace, size_t reserveSpaceNumBytes);
  // CHECK: status = miopenRNNForwardTraining(handle, RNNDescriptor, seqLength, &xD, x, hxD, hx, cxD, cx, filterDescriptor, W, &yD, y, hyD, hy, cyD, cy, workSpace, workSpaceSizeInBytes, reserveSpace, reserveSpaceNumBytes);
  status = hipdnnRNNForwardTraining(handle, RNNDescriptor, seqLength, &xD, x, hxD, hx, cxD, cx, filterDescriptor, W, &yD, y, hyD, hy, cyD, cy, workSpace, workSpaceSizeInBytes, reserveSpace, reserveSpaceNumBytes);

  // CUDA: CUDNN_DEPRECATED hipdnnStatus_t CUDNNWINAPI hipdnnRNNBackwardData(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t* yDesc, const void* y, const hipdnnTensorDescriptor_t* dyDesc, const void* dy, const hipdnnTensorDescriptor_t dhyDesc, const void* dhy, const hipdnnTensorDescriptor_t dcyDesc, const void* dcy, const hipdnnFilterDescriptor_t wDesc, const void* w, const hipdnnTensorDescriptor_t hxDesc, const void* hx, const hipdnnTensorDescriptor_t cxDesc, const void* cx, const hipdnnTensorDescriptor_t* dxDesc, void* dx, const hipdnnTensorDescriptor_t dhxDesc, void* dhx, const hipdnnTensorDescriptor_t dcxDesc, void* dcx, void* workSpace, size_t workSpaceSizeInBytes, void* reserveSpace, size_t reserveSpaceSizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenRNNBackwardData(miopenHandle_t handle, const miopenRNNDescriptor_t rnnDesc, const int sequenceLen, const miopenTensorDescriptor_t* yDesc, const void* y, const miopenTensorDescriptor_t* dyDesc, const void* dy, const miopenTensorDescriptor_t dhyDesc, const void* dhy, const miopenTensorDescriptor_t dcyDesc, const void* dcy, const miopenTensorDescriptor_t wDesc, const void* w, const miopenTensorDescriptor_t hxDesc, const void* hx, const miopenTensorDescriptor_t cxDesc, const void* cx, const miopenTensorDescriptor_t* dxDesc, void* dx, const miopenTensorDescriptor_t dhxDesc, void* dhx, const miopenTensorDescriptor_t dcxDesc, void* dcx, void* workSpace, size_t workSpaceNumBytes, void* reserveSpace, size_t reserveSpaceNumBytes);
  // CHECK: status = miopenRNNBackwardData(handle, RNNDescriptor, seqLength, &yD, y, &dyD, dy, dhyD, dhy, dcyD, dcy, filterDescriptor, W, hxD, hx, cxD, cx, &dxD, dx, dhxD, dhx, dcxD, dcx, workSpace, workSpaceSizeInBytes, &reserveSpace, reserveSpaceNumBytes);
  status = hipdnnRNNBackwardData(handle, RNNDescriptor, seqLength, &yD, y, &dyD, dy, dhyD, dhy, dcyD, dcy, filterDescriptor, W, hxD, hx, cxD, cx, &dxD, dx, dhxD, dhx, dcxD, dcx, workSpace, workSpaceSizeInBytes, &reserveSpace, reserveSpaceNumBytes);

  // CUDA: CUDNN_DEPRECATED hipdnnStatus_t CUDNNWINAPI hipdnnRNNBackwardWeights(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t* xDesc, const void* x, const hipdnnTensorDescriptor_t hxDesc, const void* hx, const hipdnnTensorDescriptor_t* yDesc, const void* y, const void* workSpace, size_t workSpaceSizeInBytes, const hipdnnFilterDescriptor_t dwDesc, void* dw, const void* reserveSpace, size_t reserveSpaceSizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenRNNBackwardWeights(miopenHandle_t handle, const miopenRNNDescriptor_t rnnDesc, const int sequenceLen, const miopenTensorDescriptor_t* xDesc, const void* x, const miopenTensorDescriptor_t hxDesc, const void* hx, const miopenTensorDescriptor_t* yDesc, const void* y, const miopenTensorDescriptor_t dwDesc, void* dw, void* workSpace, size_t workSpaceNumBytes, const void* reserveSpace, size_t reserveSpaceNumBytes);
  // CHECK: status = miopenRNNBackwardWeights(handle, RNNDescriptor, seqLength, &xD, x, hxD, hx, &yD, y, filterDescriptor, dw, workSpace, workSpaceSizeInBytes, &reserveSpace, reserveSpaceNumBytes);
  status = hipdnnRNNBackwardWeights(handle, RNNDescriptor, seqLength, &xD, x, hxD, hx, &yD, y, workSpace, workSpaceSizeInBytes, filterDescriptor, dw, &reserveSpace, reserveSpaceNumBytes);

  // CUDA: CUDNN_DEPRECATED hipdnnStatus_t CUDNNWINAPI hipdnnRNNForwardInference(hipdnnHandle_t handle, const hipdnnRNNDescriptor_t rnnDesc, const int seqLength, const hipdnnTensorDescriptor_t* xDesc, const void* x, const hipdnnTensorDescriptor_t hxDesc, const void* hx, const hipdnnTensorDescriptor_t cxDesc, const void* cx, const hipdnnFilterDescriptor_t wDesc, const void* w, const hipdnnTensorDescriptor_t* yDesc, void* y, const hipdnnTensorDescriptor_t hyDesc, void* hy, const hipdnnTensorDescriptor_t cyDesc, void* cy, void* workSpace, size_t workSpaceSizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenRNNForwardInference(miopenHandle_t handle, miopenRNNDescriptor_t rnnDesc, const int sequenceLen, const miopenTensorDescriptor_t* xDesc, const void* x, const miopenTensorDescriptor_t hxDesc, const void* hx, const miopenTensorDescriptor_t cxDesc, const void* cx, const miopenTensorDescriptor_t wDesc, const void* w, const miopenTensorDescriptor_t* yDesc, void* y, const miopenTensorDescriptor_t hyDesc, void* hy, const miopenTensorDescriptor_t cyDesc, void* cy, void* workSpace, size_t workSpaceNumBytes);
  // CHECK: status = miopenRNNForwardInference(handle, RNNDescriptor, seqLength, &xD, x, hxD, hx, cxD, cx, filterDescriptor, W, &yD, y, hyD, hy, cyD, cy, workSpace, workSpaceSizeInBytes);
  status = hipdnnRNNForwardInference(handle, RNNDescriptor, seqLength, &xD, x, hxD, hx, cxD, cx, filterDescriptor, W, &yD, y, hyD, hy, cyD, cy, workSpace, workSpaceSizeInBytes);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnDestroyRNNDescriptor(hipdnnRNNDescriptor_t rnnDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroyRNNDescriptor(miopenRNNDescriptor_t rnnDesc);
  // CHECK: status = miopenDestroyRNNDescriptor(RNNDescriptor);
  status = hipdnnDestroyRNNDescriptor(RNNDescriptor);

  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnCreateCTCLossDescriptor(cudnnCTCLossDescriptor_t* ctcLossDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreateCTCLossDescriptor(miopenCTCLossDescriptor_t* ctcLossDesc);
  // CHECK: status = miopenCreateCTCLossDescriptor(&CTCLossDescriptor);
  status = cudnnCreateCTCLossDescriptor(&CTCLossDescriptor);

  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnDestroyCTCLossDescriptor(cudnnCTCLossDescriptor_t ctcLossDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroyCTCLossDescriptor(miopenCTCLossDescriptor_t ctcLossDesc);
  // CHECK: status = miopenDestroyCTCLossDescriptor(CTCLossDescriptor);
  status = cudnnDestroyCTCLossDescriptor(CTCLossDescriptor);

  // TODO [#837]: Insert int* blank_label_id, bool* apply_softmax_layer in the hipified miopenGetCTCLossDescriptor: will need variable declaration
  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnGetCTCLossDescriptor(cudnnCTCLossDescriptor_t ctcLossDesc, hipdnnDataType_t* compType);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetCTCLossDescriptor(miopenCTCLossDescriptor_t ctcLossDesc, miopenDataType_t* dataType, int* blank_label_id, bool* apply_softmax_layer);
  // CHECK: status = miopenGetCTCLossDescriptor(CTCLossDescriptor, &dataType);
  status = cudnnGetCTCLossDescriptor(CTCLossDescriptor, &dataType);

  // TODO [#837]: Insert int blank_label_id, bool apply_softmax_layer in the hipified miopenSetCTCLossDescriptor: will need variable declaration
  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnSetCTCLossDescriptor(cudnnCTCLossDescriptor_t ctcLossDesc, hipdnnDataType_t compType);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSetCTCLossDescriptor(miopenCTCLossDescriptor_t ctcLossDesc, miopenDataType_t dataType, const int blank_label_id, bool apply_softmax_layer);
  // CHECK: status = miopenSetCTCLossDescriptor(CTCLossDescriptor, dataType);
  status = cudnnSetCTCLossDescriptor(CTCLossDescriptor, dataType);

  int labels = 0;
  int labelLengths = 0;
  int inputLengths = 0;

  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnGetCTCLossWorkspaceSize(hipdnnHandle_t handle, const hipdnnTensorDescriptor_t probsDesc, const hipdnnTensorDescriptor_t gradientsDesc, const int* labels, const int* labelLengths, const int* inputLengths, cudnnCTCLossAlgo_t algo, cudnnCTCLossDescriptor_t ctcLossDesc, size_t* sizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetCTCLossWorkspaceSize(miopenHandle_t handle, const miopenTensorDescriptor_t probsDesc, const miopenTensorDescriptor_t gradientsDesc, const int* labels, const int* labelLengths, const int* inputLengths, miopenCTCLossAlgo_t algo, const miopenCTCLossDescriptor_t ctcLossDesc, size_t* workSpaceSize);
  // CHECK: status = miopenGetCTCLossWorkspaceSize(handle, probsD, gradientsD, &labels, &labelLengths, &inputLengths, CTCLossAlgo, CTCLossDescriptor, &workSpaceSizeInBytes);
  status = cudnnGetCTCLossWorkspaceSize(handle, probsD, gradientsD, &labels, &labelLengths, &inputLengths, CTCLossAlgo, CTCLossDescriptor, &workSpaceSizeInBytes);

  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnCTCLoss(hipdnnHandle_t handle, const hipdnnTensorDescriptor_t probsDesc, const void* probs, const int hostLabels[], const int hostLabelLengths[], const int hostInputLengths[], void* costs, const hipdnnTensorDescriptor_t gradientsDesc, void* gradients, cudnnCTCLossAlgo_t algo, cudnnCTCLossDescriptor_t ctcLossDesc, void* workspace, size_t workSpaceSizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCTCLoss(miopenHandle_t handle, const miopenTensorDescriptor_t probsDesc, const void* probs, const int* labels, const int* labelLengths, const int* inputLengths, void* losses, const miopenTensorDescriptor_t gradientsDesc, void* gradients, miopenCTCLossAlgo_t algo, const miopenCTCLossDescriptor_t ctcLossDesc, void* workSpace, size_t workSpaceSize);
  // CHECK: status = miopenCTCLoss(handle, probsD, probs, &labels, &labelLengths, &inputLengths, losses, gradientsD, gradients, CTCLossAlgo, CTCLossDescriptor, workSpace , workSpaceSizeInBytes);
  status = cudnnCTCLoss(handle, probsD, probs, &labels, &labelLengths, &inputLengths, losses, gradientsD, gradients, CTCLossAlgo, CTCLossDescriptor, workSpace , workSpaceSizeInBytes);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnCreateDropoutDescriptor(hipdnnDropoutDescriptor_t* dropoutDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreateDropoutDescriptor(miopenDropoutDescriptor_t* dropoutDesc);
  // CHECK: status = miopenCreateDropoutDescriptor(&DropoutDescriptor);
  status = hipdnnCreateDropoutDescriptor(&DropoutDescriptor);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnDestroyDropoutDescriptor(hipdnnDropoutDescriptor_t dropoutDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroyDropoutDescriptor(miopenDropoutDescriptor_t dropoutDesc);
  // CHECK: status = miopenDestroyDropoutDescriptor(DropoutDescriptor);
  status = hipdnnDestroyDropoutDescriptor(DropoutDescriptor);

  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnDropoutGetReserveSpaceSize(hipdnnTensorDescriptor_t xdesc, size_t* sizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDropoutGetReserveSpaceSize(const miopenTensorDescriptor_t xDesc, size_t* reserveSpaceSizeInBytes);
  // CHECK: status = miopenDropoutGetReserveSpaceSize(xD, &reserveSpaceNumBytes);
  status = cudnnDropoutGetReserveSpaceSize(xD, &reserveSpaceNumBytes);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnDropoutGetStatesSize(hipdnnHandle_t handle, size_t* sizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDropoutGetStatesSize(miopenHandle_t handle, size_t* stateSizeInBytes);
  // CHECK: status = miopenDropoutGetStatesSize(handle, &reserveSpaceNumBytes);
  status = hipdnnDropoutGetStatesSize(handle, &reserveSpaceNumBytes);

  float dropout = 0.0f;
  void* states = nullptr;
  unsigned long long seed = 0;

  // TODO [#837]: Insert float* dropout, void** states, unsigned long long* seed in the hipified miopenGetDropoutDescriptor: will need variable declaration
  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnGetDropoutDescriptor(hipdnnDropoutDescriptor_t dropoutDesc, hipdnnHandle_t handle, float* dropout, void** states, unsigned long long* seed);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetDropoutDescriptor(miopenDropoutDescriptor_t dropoutDesc, miopenHandle_t handle, float* dropout, void** states, unsigned long long* seed, bool* use_mask, bool* state_evo, miopenRNGType_t* rng_mode);
  // CHECK: status = miopenGetDropoutDescriptor(DropoutDescriptor, handle, &dropout, &states, &seed);
  status = cudnnGetDropoutDescriptor(DropoutDescriptor, handle, &dropout, &states, &seed);

  // TODO [#837]: Insert bool use_mask, bool state_evo, miopenRNGType_t rng_mode in the hipified miopenGetDropoutDescriptor: will need variable declaration
  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnSetDropoutDescriptor(hipdnnDropoutDescriptor_t dropoutDesc, hipdnnHandle_t handle, float dropout, void* states, size_t stateSizeInBytes, unsigned long long seed);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSetDropoutDescriptor(miopenDropoutDescriptor_t dropoutDesc, miopenHandle_t handle, float dropout, void* states, size_t stateSizeInBytes, unsigned long long seed, bool use_mask, bool state_evo, miopenRNGType_t rng_mode);
  // CHECK: status = miopenSetDropoutDescriptor(DropoutDescriptor, handle, dropout, states, reserveSpaceNumBytes, seed);
  status = hipdnnSetDropoutDescriptor(DropoutDescriptor, handle, dropout, states, reserveSpaceNumBytes, seed);

  // TODO [#837]: Insert bool use_mask, bool state_evo, miopenRNGType_t rng_mode in the hipified miopenRestoreDropoutDescriptor: will need variable declaration
  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnRestoreDropoutDescriptor(hipdnnDropoutDescriptor_t dropoutDesc, hipdnnHandle_t handle, float dropout, void* states, size_t stateSizeInBytes, unsigned long long seed);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenRestoreDropoutDescriptor(miopenDropoutDescriptor_t dropoutDesc, miopenHandle_t handle, float dropout, void* states, size_t stateSizeInBytes, unsigned long long seed, bool use_mask, bool state_evo, miopenRNGType_t rng_mode);
  // CHECK: status = miopenRestoreDropoutDescriptor(DropoutDescriptor, handle, dropout, states, reserveSpaceNumBytes, seed);
  status = cudnnRestoreDropoutDescriptor(DropoutDescriptor, handle, dropout, states, reserveSpaceNumBytes, seed);

  // TODO [#837]: Insert const miopenTensorDescriptor_t noise_shape in the hipified miopenDropoutForward: will need variable declaration
  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnDropoutForward(hipdnnHandle_t handle, const hipdnnDropoutDescriptor_t dropoutDesc, const hipdnnTensorDescriptor_t xdesc, const void* x, const hipdnnTensorDescriptor_t ydesc, void* y, void* reserveSpace, size_t reserveSpaceSizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDropoutForward(miopenHandle_t handle, const miopenDropoutDescriptor_t dropoutDesc, const miopenTensorDescriptor_t noise_shape, const miopenTensorDescriptor_t xDesc, const void* x, const miopenTensorDescriptor_t yDesc, void* y, void* reserveSpace, size_t reserveSpaceSizeInBytes);
  // CHECK: status = miopenDropoutForward(handle, DropoutDescriptor, xD, x, yD, y, reserveSpace, reserveSpaceNumBytes);
  status = cudnnDropoutForward(handle, DropoutDescriptor, xD, x, yD, y, reserveSpace, reserveSpaceNumBytes);

  // TODO [#837]: Insert const miopenTensorDescriptor_t noise_shape in the hipified miopenDropoutBackward: will need variable declaration
  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnDropoutBackward(hipdnnHandle_t handle, const hipdnnDropoutDescriptor_t dropoutDesc, const hipdnnTensorDescriptor_t dydesc, const void* dy, const hipdnnTensorDescriptor_t dxdesc, void* dx, void* reserveSpace, size_t reserveSpaceSizeInBytes);
  // MIOPNE: MIOPEN_EXPORT miopenStatus_t miopenDropoutBackward(miopenHandle_t handle, const miopenDropoutDescriptor_t dropoutDesc, const miopenTensorDescriptor_t noise_shape, const miopenTensorDescriptor_t dyDesc, const void* dy, const miopenTensorDescriptor_t dxDesc, void* dx, void* reserveSpace, size_t reserveSpaceSizeInBytes);
  // CHECK: status = miopenDropoutBackward(handle, DropoutDescriptor, yD, y, xD, x, reserveSpace, reserveSpaceNumBytes);
  status = cudnnDropoutBackward(handle, DropoutDescriptor, yD, y, xD, x, reserveSpace, reserveSpaceNumBytes);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnCreateReduceTensorDescriptor(hipdnnReduceTensorDescriptor_t* reduceTensorDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreateReduceTensorDescriptor(miopenReduceTensorDescriptor_t* reduceTensorDesc);
  // CHECK: status = miopenCreateReduceTensorDescriptor(&ReduceTensorDescriptor);
  status = hipdnnCreateReduceTensorDescriptor(&ReduceTensorDescriptor);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnDestroyReduceTensorDescriptor(hipdnnReduceTensorDescriptor_t reduceTensorDesc);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroyReduceTensorDescriptor(miopenReduceTensorDescriptor_t reduceTensorDesc);
  // CHECK: status = miopenDestroyReduceTensorDescriptor(ReduceTensorDescriptor);
  status = hipdnnDestroyReduceTensorDescriptor(ReduceTensorDescriptor);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnSetReduceTensorDescriptor(hipdnnReduceTensorDescriptor_t reduceTensorDesc, hipdnnReduceTensorOp_t reduceTensorOp, hipdnnDataType_t reduceTensorCompType, hipdnnNanPropagation_t reduceTensorNanOpt, hipdnnReduceTensorIndices_t reduceTensorIndices, hipdnnIndicesType_t reduceTensorIndicesType);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSetReduceTensorDescriptor(miopenReduceTensorDescriptor_t reduceTensorDesc, miopenReduceTensorOp_t reduceTensorOp, miopenDataType_t reduceTensorCompType, miopenNanPropagation_t reduceTensorNanOpt, miopenReduceTensorIndices_t reduceTensorIndices, miopenIndicesType_t reduceTensorIndicesType);
  // CHECK: status = miopenSetReduceTensorDescriptor(ReduceTensorDescriptor, reduceTensorOp, dataType, nanPropagation_t, reduceTensorIndices, indicesType);
  status = hipdnnSetReduceTensorDescriptor(ReduceTensorDescriptor, reduceTensorOp, dataType, nanPropagation_t, reduceTensorIndices, indicesType);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnGetReduceTensorDescriptor(const hipdnnReduceTensorDescriptor_t reduceTensorDesc, hipdnnReduceTensorOp_t* reduceTensorOp, hipdnnDataType_t* reduceTensorCompType, hipdnnNanPropagation_t* reduceTensorNanOpt, hipdnnReduceTensorIndices_t* reduceTensorIndices, hipdnnIndicesType_t* reduceTensorIndicesType);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetReduceTensorDescriptor(const miopenReduceTensorDescriptor_t reduceTensorDesc, miopenReduceTensorOp_t* reduceTensorOp, miopenDataType_t* reduceTensorCompType, miopenNanPropagation_t* reduceTensorNanOpt, miopenReduceTensorIndices_t* reduceTensorIndices, miopenIndicesType_t* reduceTensorIndicesType);
  // CHECK: status = miopenGetReduceTensorDescriptor(ReduceTensorDescriptor, &reduceTensorOp, &dataType, &nanPropagation_t, &reduceTensorIndices, &indicesType);
  status = hipdnnGetReduceTensorDescriptor(ReduceTensorDescriptor, &reduceTensorOp, &dataType, &nanPropagation_t, &reduceTensorIndices, &indicesType);

  // CUDA: hipdnnStatus_t CUDNNWINAPI cudnnGetReductionIndicesSize(hipdnnHandle_t handle, const hipdnnReduceTensorDescriptor_t reduceTensorDesc, const hipdnnTensorDescriptor_t aDesc, const hipdnnTensorDescriptor_t cDesc, size_t* sizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetReductionIndicesSize(miopenHandle_t handle, const miopenReduceTensorDescriptor_t reduceTensorDesc, const miopenTensorDescriptor_t aDesc, const miopenTensorDescriptor_t cDesc, size_t* sizeInBytes);
  // CHECK: status = miopenGetReductionIndicesSize(handle, ReduceTensorDescriptor, aD, cD, &workSpaceSizeInBytes);
  status = cudnnGetReductionIndicesSize(handle, ReduceTensorDescriptor, aD, cD, &workSpaceSizeInBytes);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnGetReductionWorkspaceSize(hipdnnHandle_t handle, const hipdnnReduceTensorDescriptor_t reduceTensorDesc, const hipdnnTensorDescriptor_t aDesc, const hipdnnTensorDescriptor_t cDesc, size_t* sizeInBytes);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetReductionWorkspaceSize(miopenHandle_t handle, const miopenReduceTensorDescriptor_t reduceTensorDesc, const miopenTensorDescriptor_t aDesc, const miopenTensorDescriptor_t cDesc, size_t* sizeInBytes);
  // CHECK: status = miopenGetReductionWorkspaceSize(handle, ReduceTensorDescriptor, aD, cD, &workSpaceSizeInBytes);
  status = hipdnnGetReductionWorkspaceSize(handle, ReduceTensorDescriptor, aD, cD, &workSpaceSizeInBytes);

  // CUDA: hipdnnStatus_t CUDNNWINAPI hipdnnReduceTensor(hipdnnHandle_t handle, const hipdnnReduceTensorDescriptor_t reduceTensorDesc, void* indices, size_t indicesSizeInBytes, void* workspace, size_t workspaceSizeInBytes, const void* alpha, const hipdnnTensorDescriptor_t aDesc, const void* A, const void* beta, const hipdnnTensorDescriptor_t cDesc, void* C);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenReduceTensor(miopenHandle_t handle, const miopenReduceTensorDescriptor_t reduceTensorDesc, void* indices, size_t indicesSizeInBytes, void* workspace, size_t workspaceSizeInBytes, const void* alpha, const miopenTensorDescriptor_t aDesc, const void* A, const void* beta, const miopenTensorDescriptor_t cDesc, void* C);
  // CHECK: status = miopenReduceTensor(handle, ReduceTensorDescriptor, indices, indicesSizeInBytes, workSpace, workSpaceSizeInBytes, alpha, aD, A, beta, cD, C);
  status = hipdnnReduceTensor(handle, ReduceTensorDescriptor, indices, indicesSizeInBytes, workSpace, workSpaceSizeInBytes, alpha, aD, A, beta, cD, C);

  return 0;
}
