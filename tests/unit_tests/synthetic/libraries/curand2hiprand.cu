// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --default-preprocessor --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hiprand/hiprand.h"
// CHECK-NEXT: #include "hiprand/hiprand_kernel.h"
#include "hiprand/hiprand.h"
#include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "hiprand/hiprand.h"
// CHECK-NOT: #include "hiprand/hiprand_kernel.h"

#if defined(_WIN32) && CUDA_VERSION < 9000
  typedef signed   __int64 int64_t;
  typedef unsigned __int64 uint64_t;
#endif

int main() {
  printf("21. cuRAND API to hipRAND API synthetic test\n");

  unsigned int *outputPtr = nullptr;
  size_t num = 0;

  // CHECK: hiprandStatus randStatus;
  // CHECK-NEXT: hiprandStatus_t status;
  // CHECK-NEXT: hiprandStatus_t STATUS_SUCCESS = HIPRAND_STATUS_SUCCESS;
  // CHECK-NEXT: hiprandStatus_t STATUS_VERSION_MISMATCH = HIPRAND_STATUS_VERSION_MISMATCH;
  // CHECK-NEXT: hiprandStatus_t STATUS_NOT_INITIALIZED = HIPRAND_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT: hiprandStatus_t STATUS_ALLOCATION_FAILED = HIPRAND_STATUS_ALLOCATION_FAILED;
  // CHECK-NEXT: hiprandStatus_t STATUS_TYPE_ERROR = HIPRAND_STATUS_TYPE_ERROR;
  // CHECK-NEXT: hiprandStatus_t STATUS_OUT_OF_RANGE = HIPRAND_STATUS_OUT_OF_RANGE;
  // CHECK-NEXT: hiprandStatus_t STATUS_LENGTH_NOT_MULTIPLE = HIPRAND_STATUS_LENGTH_NOT_MULTIPLE;
  // CHECK-NEXT: hiprandStatus_t STATUS_DOUBLE_PRECISION_REQUIRED = HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED;
  // CHECK-NEXT: hiprandStatus_t STATUS_LAUNCH_FAILURE = HIPRAND_STATUS_LAUNCH_FAILURE;
  // CHECK-NEXT: hiprandStatus_t STATUS_PREEXISTING_FAILURE = HIPRAND_STATUS_PREEXISTING_FAILURE;
  // CHECK-NEXT: hiprandStatus_t STATUS_INITIALIZATION_FAILED = HIPRAND_STATUS_INITIALIZATION_FAILED;
  // CHECK-NEXT: hiprandStatus_t STATUS_ARCH_MISMATCH = HIPRAND_STATUS_ARCH_MISMATCH;
  // CHECK-NEXT: hiprandStatus_t STATUS_INTERNAL_ERROR = HIPRAND_STATUS_INTERNAL_ERROR;
  hiprandStatus randStatus;
  hiprandStatus_t status;
  hiprandStatus_t STATUS_SUCCESS = HIPRAND_STATUS_SUCCESS;
  hiprandStatus_t STATUS_VERSION_MISMATCH = HIPRAND_STATUS_VERSION_MISMATCH;
  hiprandStatus_t STATUS_NOT_INITIALIZED = HIPRAND_STATUS_NOT_INITIALIZED;
  hiprandStatus_t STATUS_ALLOCATION_FAILED = HIPRAND_STATUS_ALLOCATION_FAILED;
  hiprandStatus_t STATUS_TYPE_ERROR = HIPRAND_STATUS_TYPE_ERROR;
  hiprandStatus_t STATUS_OUT_OF_RANGE = HIPRAND_STATUS_OUT_OF_RANGE;
  hiprandStatus_t STATUS_LENGTH_NOT_MULTIPLE = HIPRAND_STATUS_LENGTH_NOT_MULTIPLE;
  hiprandStatus_t STATUS_DOUBLE_PRECISION_REQUIRED = HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED;
  hiprandStatus_t STATUS_LAUNCH_FAILURE = HIPRAND_STATUS_LAUNCH_FAILURE;
  hiprandStatus_t STATUS_PREEXISTING_FAILURE = HIPRAND_STATUS_PREEXISTING_FAILURE;
  hiprandStatus_t STATUS_INITIALIZATION_FAILED = HIPRAND_STATUS_INITIALIZATION_FAILED;
  hiprandStatus_t STATUS_ARCH_MISMATCH = HIPRAND_STATUS_ARCH_MISMATCH;
  hiprandStatus_t STATUS_INTERNAL_ERROR = HIPRAND_STATUS_INTERNAL_ERROR;

  // CHECK: hiprandRngType_t randRngType;
  // CHECK-NEXT: hiprandRngType_t randRngType_t;
  // CHECK-NEXT: hiprandRngType_t RNG_TEST = HIPRAND_RNG_TEST;
  // CHECK-NEXT: hiprandRngType_t RNG_PSEUDO_DEFAULT = HIPRAND_RNG_PSEUDO_DEFAULT;
  // CHECK-NEXT: hiprandRngType_t RNG_PSEUDO_XORWOW = HIPRAND_RNG_PSEUDO_XORWOW;
  // CHECK-NEXT: hiprandRngType_t RNG_PSEUDO_MRG32K3A = HIPRAND_RNG_PSEUDO_MRG32K3A;
  // CHECK-NEXT: hiprandRngType_t RNG_PSEUDO_MTGP32 = HIPRAND_RNG_PSEUDO_MTGP32;
  // CHECK-NEXT: hiprandRngType_t RNG_PSEUDO_MT19937 = HIPRAND_RNG_PSEUDO_MT19937;
  // CHECK-NEXT: hiprandRngType_t RNG_PSEUDO_PHILOX4_32_10 = HIPRAND_RNG_PSEUDO_PHILOX4_32_10;
  // CHECK-NEXT: hiprandRngType_t RNG_QUASI_DEFAULT = HIPRAND_RNG_QUASI_DEFAULT;
  // CHECK-NEXT: hiprandRngType_t RNG_QUASI_SOBOL32 = HIPRAND_RNG_QUASI_SOBOL32;
  // CHECK-NEXT: hiprandRngType_t RNG_QUASI_SCRAMBLED_SOBOL32 = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32;
  // CHECK-NEXT: hiprandRngType_t RNG_QUASI_SOBOL64 = HIPRAND_RNG_QUASI_SOBOL64;
  // CHECK-NEXT: hiprandRngType_t RNG_QUASI_SCRAMBLED_SOBOL64 = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64;
  hiprandRngType_t randRngType;
  hiprandRngType_t randRngType_t;
  hiprandRngType_t RNG_TEST = HIPRAND_RNG_TEST;
  hiprandRngType_t RNG_PSEUDO_DEFAULT = HIPRAND_RNG_PSEUDO_DEFAULT;
  hiprandRngType_t RNG_PSEUDO_XORWOW = HIPRAND_RNG_PSEUDO_XORWOW;
  hiprandRngType_t RNG_PSEUDO_MRG32K3A = HIPRAND_RNG_PSEUDO_MRG32K3A;
  hiprandRngType_t RNG_PSEUDO_MTGP32 = HIPRAND_RNG_PSEUDO_MTGP32;
  hiprandRngType_t RNG_PSEUDO_MT19937 = HIPRAND_RNG_PSEUDO_MT19937;
  hiprandRngType_t RNG_PSEUDO_PHILOX4_32_10 = HIPRAND_RNG_PSEUDO_PHILOX4_32_10;
  hiprandRngType_t RNG_QUASI_DEFAULT = HIPRAND_RNG_QUASI_DEFAULT;
  hiprandRngType_t RNG_QUASI_SOBOL32 = HIPRAND_RNG_QUASI_SOBOL32;
  hiprandRngType_t RNG_QUASI_SCRAMBLED_SOBOL32 = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32;
  hiprandRngType_t RNG_QUASI_SOBOL64 = HIPRAND_RNG_QUASI_SOBOL64;
  hiprandRngType_t RNG_QUASI_SCRAMBLED_SOBOL64 = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64;

  // CHECK: hiprandOrdering randOrdering;
  // CHECK-NEXT: hiprandOrdering_t randOrdering_t;
  // CHECK-NEXT: hiprandOrdering_t RAND_ORDERING_PSEUDO_BEST = HIPRAND_ORDERING_PSEUDO_BEST;
  // CHECK-NEXT: hiprandOrdering_t RAND_ORDERING_PSEUDO_DEFAULT = HIPRAND_ORDERING_PSEUDO_DEFAULT;
  // CHECK-NEXT: hiprandOrdering_t RAND_ORDERING_PSEUDO_SEEDED = HIPRAND_ORDERING_PSEUDO_SEEDED;
  // CHECK-NEXT: hiprandOrdering_t RAND_ORDERING_QUASI_DEFAULT = HIPRAND_ORDERING_QUASI_DEFAULT;
  hiprandOrdering randOrdering;
  hiprandOrdering_t randOrdering_t;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_BEST = HIPRAND_ORDERING_PSEUDO_BEST;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_DEFAULT = HIPRAND_ORDERING_PSEUDO_DEFAULT;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_SEEDED = HIPRAND_ORDERING_PSEUDO_SEEDED;
  hiprandOrdering_t RAND_ORDERING_QUASI_DEFAULT = HIPRAND_ORDERING_QUASI_DEFAULT;

  // CHECK: hiprandGenerator_st *randGenerator_st = nullptr;
  // CHECK-NEXT: hiprandGenerator_t randGenerator;
  hiprandGenerator_st *randGenerator_st = nullptr;
  hiprandGenerator_t randGenerator;

  // CHECK: hiprandStateSobol64 randStateSobol64;
  // CHECK-NEXT: hiprandStateSobol64_t randStateSobol64_t;
  hiprandStateSobol64 randStateSobol64;
  hiprandStateSobol64_t randStateSobol64_t;

  // CHECK: hiprandStateScrambledSobol64 randStateScrambledSobol64;
  // CHECK-NEXT: hiprandStateScrambledSobol64_t randStateScrambledSobol64_t;
  hiprandStateScrambledSobol64 randStateScrambledSobol64;
  hiprandStateScrambledSobol64_t randStateScrambledSobol64_t;

  // CHECK: hiprandStateScrambledSobol32 randStateScrambledSobol32;
  // CHECK-NEXT: hiprandStateScrambledSobol32_t randStateScrambledSobol32_t;
  hiprandStateScrambledSobol32 randStateScrambledSobol32;
  hiprandStateScrambledSobol32_t randStateScrambledSobol32_t;

  // CHECK: hiprandDirectionVectors32_t directions32;
  // CHECK-NEXT: hiprandDirectionVectors64_t directions64;
  // CHECK-NEXT: hiprandDirectionVectors64_t *pDirections64 = nullptr;
  hiprandDirectionVectors32_t directions32;
  hiprandDirectionVectors64_t directions64;
  hiprandDirectionVectors64_t *pDirections64 = nullptr;

  // CHECK: hiprandDirectionVectorSet_t directionVectorSet;
  // CHECK-NEXT: hiprandDirectionVectorSet_t directionVectorSet_t;
  // CHECK-NEXT: hiprandDirectionVectorSet_t DIRECTION_VECTORS_32_JOEKUO6 = HIPRAND_DIRECTION_VECTORS_32_JOEKUO6;
  // CHECK-NEXT: hiprandDirectionVectorSet_t SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6 = HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6;
  // CHECK-NEXT: hiprandDirectionVectorSet_t DIRECTION_VECTORS_64_JOEKUO6 = HIPRAND_DIRECTION_VECTORS_64_JOEKUO6;
  // CHECK-NEXT: hiprandDirectionVectorSet_t SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6 = HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6;
  hiprandDirectionVectorSet_t directionVectorSet;
  hiprandDirectionVectorSet_t directionVectorSet_t;
  hiprandDirectionVectorSet_t DIRECTION_VECTORS_32_JOEKUO6 = HIPRAND_DIRECTION_VECTORS_32_JOEKUO6;
  hiprandDirectionVectorSet_t SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6 = HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6;
  hiprandDirectionVectorSet_t DIRECTION_VECTORS_64_JOEKUO6 = HIPRAND_DIRECTION_VECTORS_64_JOEKUO6;
  hiprandDirectionVectorSet_t SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6 = HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6;

  // CUDA: curandStatus_t CURANDAPI curandCreateGenerator(curandGenerator_t *generator, curandRngType_t rng_type);
  // HIP: hiprandStatus_t HIPRANDAPI hiprandCreateGenerator(hiprandGenerator_t* generator, hiprandRngType_t rng_type)
  // CHECK: status = hiprandCreateGenerator(&randGenerator, randRngType_t);
  status = hiprandCreateGenerator(&randGenerator, randRngType_t);

  // CUDA: curandStatus_t CURANDAPI curandDestroyGenerator(curandGenerator_t generator);
  // HIP: hiprandStatus_t HIPRANDAPI hiprandDestroyGenerator(hiprandGenerator_t generator);
  // CHECK: status = hiprandDestroyGenerator(randGenerator);
  status = hiprandDestroyGenerator(randGenerator);

  // CUDA: curandStatus_t CURANDAPI curandCreateGeneratorHost(curandGenerator_t *generator, curandRngType_t rng_type);
  // HIP: hiprandStatus_t HIPRANDAPI hiprandCreateGeneratorHost(hiprandGenerator_t * generator, hiprandRngType_t rng_type);
  // CHECK: status = hiprandCreateGeneratorHost(&randGenerator, randRngType_t);
  status = hiprandCreateGeneratorHost(&randGenerator, randRngType_t);

  // CUDA: curandStatus_t CURANDAPI curandSetGeneratorOrdering(curandGenerator_t generator, curandOrdering_t order);
  // HIP: hiprandStatus_t HIPRANDAPI hiprandSetGeneratorOrdering(hiprandGenerator_t generator, hiprandOrdering_t order);
  // CHECK: status = hiprandSetGeneratorOrdering(randGenerator, randOrdering_t);
  status = hiprandSetGeneratorOrdering(randGenerator, randOrdering_t);

  // CUDA: curandStatus_t CURANDAPI curandGetDirectionVectors64(curandDirectionVectors64_t *vectors[], curandDirectionVectorSet_t set);
  // HIP: hiprandStatus_t HIPRANDAPI hiprandGetDirectionVectors64(hiprandDirectionVectors64_t** vectors, hiprandDirectionVectorSet_t set);
  // CHECK: status = hiprandGetDirectionVectors64(&pDirections64, directionVectorSet_t);
  status = hiprandGetDirectionVectors64(&pDirections64, directionVectorSet_t);

  // CUDA: curandStatus_t CURANDAPI curandGenerate(curandGenerator_t generator, unsigned int *outputPtr, size_t num);
  // HIP: hiprandStatus_t HIPRANDAPI hiprandGenerate(hiprandGenerator_t generator, unsigned int * output_data, size_t n);
  // CHECK: status = hiprandGenerate(randGenerator, outputPtr, num);
  status = hiprandGenerate(randGenerator, outputPtr, num);

#if CUDA_VERSION >= 11000 && CURAND_VERSION >= 10200
  // CHECK: hiprandOrdering_t RAND_ORDERING_PSEUDO_LEGACY = HIPRAND_ORDERING_PSEUDO_LEGACY;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_LEGACY = HIPRAND_ORDERING_PSEUDO_LEGACY;
#endif

#if CUDA_VERSION >= 11050 && CURAND_VERSION >= 10207
  // CHECK: hiprandOrdering_t RAND_ORDERING_PSEUDO_DYNAMIC = HIPRAND_ORDERING_PSEUDO_DYNAMIC;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_DYNAMIC = HIPRAND_ORDERING_PSEUDO_DYNAMIC;
#endif

  return 0;
}
