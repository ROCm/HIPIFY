// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipblas.h"
#include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"

int main() {
  printf("14. cuBLAS API to hipBLAS API synthetic test\n");

  // CHECK: hipblasOperation_t blasOperation;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;
  hipblasOperation_t blasOperation;
  hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;

  // CHECK: hipblasStatus_t blasStatus;
  // CHECK-NEXT: hipblasStatus_t blasStatus_t;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;
  hipblasStatus_t blasStatus;
  hipblasStatus_t blasStatus_t;
  hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;

  // CHECK: hipblasFillMode_t blasFillMode;
  // CHECK-NEXT: hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  // CHECK-NEXT: hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;
  hipblasFillMode_t blasFillMode;
  hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;

  // CHECK: hipblasDiagType_t blasDiagType;
  // CHECK-NEXT: hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  // CHECK-NEXT: hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;
  hipblasDiagType_t blasDiagType;
  hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;

  // CHECK: hipblasSideMode_t blasSideMode;
  // CHECK-NEXT: hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  // CHECK-NEXT: hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;
  hipblasSideMode_t blasSideMode;
  hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;

  // CHECK: hipblasPointerMode_t blasPointerMode;
  // CHECK-NEXT: hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  // CHECK-NEXT: hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;
  hipblasPointerMode_t blasPointerMode;
  hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;

  // CHECK: hipblasAtomicsMode_t blasAtomicsMode;
  // CHECK-NEXT: hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  // CHECK-NEXT: hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;
  hipblasAtomicsMode_t blasAtomicsMode;
  hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;

  // CHECK: hipblasHandle_t blasHandle;
  hipblasHandle_t blasHandle;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetAtomicsMode(cublasHandle_t handle, cublasAtomicsMode_t* mode);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t* atomics_mode);
  // CHECK: blasStatus = hipblasGetAtomicsMode(blasHandle, &blasAtomicsMode);
  blasStatus = hipblasGetAtomicsMode(blasHandle, &blasAtomicsMode);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetAtomicsMode(cublasHandle_t handle, cublasAtomicsMode_t mode);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t atomics_mode);
  // CHECK: blasStatus = hipblasSetAtomicsMode(blasHandle, blasAtomicsMode);
  blasStatus = hipblasSetAtomicsMode(blasHandle, blasAtomicsMode);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCreate_v2(cublasHandle_t* handle);
  // CUDA: #define cublasCreate cublasCreate_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCreate(hipblasHandle_t* handle);
  // CHECK: blasStatus = hipblasCreate(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDestroy_v2(cublasHandle_t handle);
  // CUDA: #define cublasDestroy cublasDestroy_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDestroy(hipblasHandle_t handle);
  // CHECK: blasStatus = hipblasDestroy(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);

  // CHECK: hipStream_t stream;
  hipStream_t stream;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetStream_v2(cublasHandle_t handle, cudaStream_t streamId);
  // CUDA: #define cublasSetStream cublasSetStream_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetStream(hipblasHandle_t handle, hipStream_t streamId);
  // CHECK: blasStatus = hipblasSetStream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetStream_v2(cublasHandle_t handle, cudaStream_t* streamId);
  // CUDA: #define cublasGetStream cublasGetStream_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetStream(hipblasHandle_t handle, hipStream_t* streamId);
  // CHECK: blasStatus = hipblasGetStream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetPointerMode_v2(cublasHandle_t handle, cublasPointerMode_t mode);
  // CUDA: #define cublasSetPointerMode cublasSetPointerMode_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetPointerMode(hipblasHandle_t handle, hipblasPointerMode_t mode);
  // CHECK: blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);
  blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetPointerMode_v2(cublasHandle_t handle, cublasPointerMode_t* mode);
  // CUDA: #define cublasGetPointerMode cublasGetPointerMode_v2
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetPointerMode(hipblasHandle_t handle, hipblasPointerMode_t* mode);
  // CHECK: blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);
  blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);

  int n = 0;
  int nrhs = 0;
  int m = 0;
  int num = 0;
  int lda = 0;
  int ldb = 0;
  int ldc = 0;
  int res = 0;
  int incx = 0;
  int incy = 0;
  int k = 0;
  int kl = 0;
  int ku = 0;
  int batchCount = 0;
  int P = 0;
  int info = 0;
  void* image = nullptr;
  void* image_2 = nullptr;
  void* deviceptr = nullptr;

  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetVector(int n, int elemSize, const void* x, int incx, void* devicePtr, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // CHECK: blasStatus = hipblasSetVector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasSetVector(n, num, image, incx, image_2, incy);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // CHECK: blasStatus = hipblasGetVector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasGetVector(n, num, image, incx, image_2, incy);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetVectorAsync(int n, int elemSize, const void* hostPtr, int incx, void* devicePtr, int incy, cudaStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetVectorAsync(int n, int elemSize, const void* x, int incx, void* y, int incy, hipStream_t stream);
  // CHECK: blasStatus = hipblasSetVectorAsync(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetVectorAsync(n, num, image, incx, image_2, incy, stream);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetVectorAsync(int n, int elemSize, const void* devicePtr, int incx, void* hostPtr, int incy, cudaStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetVectorAsync(int n, int elemSize, const void* x, int incx, void* y, int incy, hipStream_t stream);
  // CHECK: blasStatus = hipblasGetVectorAsync(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetVectorAsync(n, num, image, incx, image_2, incy, stream);

  int rows = 0;
  int cols = 0;

  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetMatrix(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb);
  // CHECK: blasStatus = hipblasSetMatrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasSetMatrix(rows, cols, num, image, incx, image_2, incy);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetMatrix(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb);
  // CHECK: blasStatus = hipblasGetMatrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasGetMatrix(rows, cols, num, image, incx, image_2, incy);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, cudaStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetMatrixAsync(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb, hipStream_t stream);
  // CHECK: blasStatus = hipblasSetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, cudaStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetMatrixAsync(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb, hipStream_t stream);
  // CHECK: blasStatus = hipblasGetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  float fa = 0;
  float fA = 0;
  float fb = 0;
  float fB = 0;
  float fx = 0;
  float fx1 = 0;
  float fy = 0;
  float fy1 = 0;
  float fc = 0;
  float fC = 0;
  float fs = 0;
  float fd1 = 0;
  float fd2 = 0;
  float fresult = 0;

  float** fAarray = 0;
  const float** const fAarray_const = const_cast<const float**>(fAarray);
  float** fBarray = 0;
  const float** const fBarray_const = const_cast<const float**>(fBarray);
  float** fCarray = 0;
  float** fTauarray = 0;

  double da = 0;
  double dA = 0;
  double db = 0;
  double dB = 0;
  double dx = 0;
  double dx1 = 0;
  double dy = 0;
  double dy1 = 0;
  double dc = 0;
  double dC = 0;
  double ds = 0;
  double dd1 = 0;
  double dd2 = 0;
  double dresult = 0;

  double** dAarray = 0;
  const double** const dAarray_const = const_cast<const double**>(dAarray);
  double** dBarray = 0;
  const double** const dBarray_const = const_cast<const double**>(dBarray);
  double** dCarray = 0;
  double** dTauarray = 0;

  void** voidAarray = nullptr;
  const void** const voidAarray_const = const_cast<const void**>(voidAarray);
  void** voidBarray = nullptr;
  const void** const voidBarray_const = const_cast<const void**>(voidBarray);
  void** voidCarray = nullptr;

  // NOTE: float CUBLASWINAPI cublasSnrm2(int n, const float* x, int incx) is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSnrm2_v2(cublasHandle_t handle, int n, const float* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSnrm2(hipblasHandle_t handle, int n, const float* x, int incx, float* result);
  // CHECK: blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);

  // NOTE: double CUBLASWINAPI cublasDnrm2(int n, const double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDnrm2_v2(cublasHandle_t handle, int n, const double* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDnrm2(hipblasHandle_t handle, int n, const double* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);

  // CHECK: hipComplex complex, complexa, complexA, complexB, complexC, complexx, complexy, complexs, complexb;
  hipComplex complex, complexa, complexA, complexB, complexC, complexx, complexy, complexs, complexb;
  // CHECK: hipDoubleComplex dcomplex, dcomplexa, dcomplexA, dcomplexB, dcomplexC, dcomplexx, dcomplexy, dcomplexs, dcomplexb;
  hipDoubleComplex dcomplex, dcomplexa, dcomplexA, dcomplexB, dcomplexC, dcomplexx, dcomplexy, dcomplexs, dcomplexb;

  // CHECK: hipComplex** complexAarray = 0;
  // CHECK: const hipComplex** const complexAarray_const = const_cast<const hipComplex**>(complexAarray);
  // CHECK-NEXT: hipComplex** complexBarray = 0;
  // CHECK: const hipComplex** const complexBarray_const = const_cast<const hipComplex**>(complexBarray);
  // CHECK-NEXT: hipComplex** complexCarray = 0;
  // CHECK-NEXT: hipComplex** complexTauarray = 0;
  hipComplex** complexAarray = 0;
  const hipComplex** const complexAarray_const = const_cast<const hipComplex**>(complexAarray);
  hipComplex** complexBarray = 0;
  const hipComplex** const complexBarray_const = const_cast<const hipComplex**>(complexBarray);
  hipComplex** complexCarray = 0;
  hipComplex** complexTauarray = 0;

  // CHECK: hipDoubleComplex** dcomplexAarray = 0;
  // CHECK: const hipDoubleComplex** const dcomplexAarray_const = const_cast<const hipDoubleComplex**>(dcomplexAarray);
  // CHECK-NEXT: hipDoubleComplex** dcomplexBarray = 0;
  // CHECK: const hipDoubleComplex** const dcomplexBarray_const = const_cast<const hipDoubleComplex**>(dcomplexBarray);
  // CHECK-NEXT: hipDoubleComplex** dcomplexCarray = 0;
  // CHECK-NEXT: hipDoubleComplex** dcomplexTauarray = 0;
  hipDoubleComplex** dcomplexAarray = 0;
  const hipDoubleComplex** const dcomplexAarray_const = const_cast<const hipDoubleComplex**>(dcomplexAarray);
  hipDoubleComplex** dcomplexBarray = 0;
  const hipDoubleComplex** const dcomplexBarray_const = const_cast<const hipDoubleComplex**>(dcomplexBarray);
  hipDoubleComplex** dcomplexCarray = 0;
  hipDoubleComplex** dcomplexTauarray = 0;

  // NOTE: float CUBLASWINAPI cublasScnrm2(int n, const cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScnrm2_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScnrm2(hipblasHandle_t handle, int n, const hipblasComplex* x, int incx, float* result);
  // CHECK: blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);
  blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);

  // NOTE: double CUBLASWINAPI cublasDznrm2(int n, const cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDznrm2_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDznrm2(hipblasHandle_t handle, int n, const hipblasDoubleComplex* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);

  // NOTE: float CUBLASWINAPI cublasSdot(int n, const float* x, int incx, const float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSdot_v2(cublasHandle_t handle, int n, const float* x, int incx, const float* y, int incy, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSdot(hipblasHandle_t handle, int n, const float* x, int incx, const float* y, int incy, float* result);
  // CHECK: blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);

  // NOTE: double CUBLASWINAPI cublasDdot(int n, const double* x, int incx, const double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDdot_v2(cublasHandle_t handle, int n, const double* x, int incx, const double* y, int incy, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDdot(hipblasHandle_t handle, int n, const double* x, int incx, const double* y, int incy, double* result);
  // CHECK: blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);

  // NOTE: cuComplex CUBLASWINAPI cublasCdotu(int n, const cuComplex* x, int incx, const cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdotu_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCdotu(hipblasHandle_t handle, int n, const hipblasComplex* x, int incx, const hipblasComplex* y, int incy, hipblasComplex* result);
  // CHECK: blasStatus = hipblasCdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);

  // NOTE: cuComplex CUBLASWINAPI cublasCdotc(int n, const cuComplex* x, int incx, const cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdotc_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCdotc(hipblasHandle_t handle, int n, const hipblasComplex* x, int incx, const hipblasComplex* y, int incy, hipblasComplex* result);
  // CHECK: blasStatus = hipblasCdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);

  // NOTE: cuDoubleComplex CUBLASWINAPI cublasZdotu(int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdotu_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdotu(hipblasHandle_t handle, int n, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* y, int incy, hipblasDoubleComplex* result);
  // CHECK: blasStatus = hipblasZdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);

  // NOTE: cuDoubleComplex CUBLASWINAPI cublasZdotc(int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdotc_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdotc(hipblasHandle_t handle, int n, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* y, int incy, hipblasDoubleComplex* result);
  // CHECK: blasStatus = hipblasZdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);

  // NOTE: void CUBLASWINAPI cublasSscal(int n, float alpha, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSscal_v2(cublasHandle_t handle, int n, const float* alpha, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSscal(hipblasHandle_t handle, int n, const float* alpha, float* x, int incx);
  // CHECK: blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);
  blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);

  // NOTE: void CUBLASWINAPI cublasDscal(int n, double alpha, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDscal_v2(cublasHandle_t handle, int n, const double* alpha, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDscal(hipblasHandle_t handle, int n, const double* alpha, double* x, int incx);
  // CHECK: blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);
  blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);

  // NOTE: void CUBLASWINAPI cublasCscal(int n, cuComplex alpha, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCscal_v2(cublasHandle_t handle, int n, const cuComplex* alpha, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCscal(hipblasHandle_t handle, int n, const hipblasComplex* alpha, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCscal(blasHandle, n, &complexa, &complexx, incx);
  blasStatus = hipblasCscal(blasHandle, n, &complexa, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasCsscal(int n, float alpha, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsscal_v2(cublasHandle_t handle, int n, const float* alpha, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsscal(hipblasHandle_t handle, int n, const float* alpha, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCsscal(blasHandle, n, &fx, &complexx, incx);
  blasStatus = hipblasCsscal(blasHandle, n, &fx, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasZscal(int n, cuDoubleComplex alpha, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZscal_v2(cublasHandle_t handle, int n, const cuDoubleComplex* alpha, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZscal(hipblasHandle_t handle, int n, const hipblasDoubleComplex* alpha, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  blasStatus = hipblasZscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasZdscal(int n, double alpha, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdscal_v2(cublasHandle_t handle, int n, const double* alpha, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdscal(hipblasHandle_t handle, int n, const double* alpha, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZdscal(blasHandle, n, &dx, &dcomplexx, incx);
  blasStatus = hipblasZdscal(blasHandle, n, &dx, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasSaxpy(int n, float alpha, const float* x, int incx, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSaxpy_v2(cublasHandle_t handle, int n, const float* alpha, const float* x, int incx, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSaxpy(hipblasHandle_t handle, int n, const float* alpha, const float* x, int incx, float* y, int incy);
  // CHECK: blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDaxpy(int n, double alpha, const double* x, int incx, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDaxpy_v2(cublasHandle_t handle, int n, const double* alpha, const double* x, int incx, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDaxpy(hipblasHandle_t handle, int n, const double* alpha, const double* x, int incx, double* y, int incy);
  // CHECK: blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);

  // NOTE: void CUBLASWINAPI cublasCaxpy(int n, cuComplex alpha, const cuComplex* x, int incx, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCaxpy_v2(cublasHandle_t handle, int n, const cuComplex* alpha, const cuComplex* x, int incx, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCaxpy(hipblasHandle_t handle, int n, const hipblasComplex* alpha, const hipblasComplex* x, int incx, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasCaxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCaxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZaxpy(int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZaxpy_v2(cublasHandle_t handle, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZaxpy(hipblasHandle_t handle, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* x, int incx, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasScopy(int n, const float* x, int incx, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScopy_v2(cublasHandle_t handle, int n, const float* x, int incx, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScopy(hipblasHandle_t handle, int n, const float* x, int incx, float* y, int incy);
  // CHECK: blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDcopy(int n, const double* x, int incx, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDcopy_v2(cublasHandle_t handle, int n, const double* x, int incx, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDcopy(hipblasHandle_t handle, int n, const double* x, int incx, double* y, int incy);
  // CHECK: blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);

  // NOTE: void CUBLASWINAPI cublasCcopy(int n, const cuComplex* x, int incx, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCcopy_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCcopy(hipblasHandle_t handle, int n, const hipblasComplex* x, int incx, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasCcopy(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCcopy(blasHandle, n, &complexx, incx, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZcopy(int n, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZcopy_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZcopy(hipblasHandle_t handle, int n, const hipblasDoubleComplex* x, int incx, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasSswap(int n, float* x, int incx, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSswap_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSswap(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy);
  // CHECK: blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDswap(int n, double* x, int incx, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDswap_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDswap(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy);
  // CHECK: blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);

  // NOTE: void CUBLASWINAPI cublasCswap(int n, cuComplex* x, int incx, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCswap_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCswap(hipblasHandle_t handle, int n, hipblasComplex* x, int incx, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasCswap(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCswap(blasHandle, n, &complexx, incx, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZswap(int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZswap_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZswap(hipblasHandle_t handle, int n, hipblasDoubleComplex* x, int incx, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);

  // NOTE: int CUBLASWINAPI cublasIsamax(int n, const float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIsamax_v2(cublasHandle_t handle, int n, const float* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIsamax(hipblasHandle_t handle, int n, const float* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);

  // NOTE: int CUBLASWINAPI cublasIdamax(int n, const double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIdamax_v2(cublasHandle_t handle, int n, const double* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIdamax(hipblasHandle_t handle, int n, const double* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);

  // NOTE: int CUBLASWINAPI cublasIcamax(int n, const cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIcamax_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIcamax(hipblasHandle_t handle, int n, const hipblasComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIcamax(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamax(blasHandle, n, &complexx, incx, &res);

  // NOTE: int CUBLASWINAPI cublasIzamax(int n, const cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIzamax_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIzamax(hipblasHandle_t handle, int n, const hipblasDoubleComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIzamax(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamax(blasHandle, n, &dcomplexx, incx, &res);

  // NOTE: int CUBLASWINAPI cublasIsamin(int n, const float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIsamin_v2(cublasHandle_t handle, int n, const float* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIsamin(hipblasHandle_t handle, int n, const float* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);

  // NOTE: int CUBLASWINAPI cublasIdamin(int n, const double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIdamin_v2(cublasHandle_t handle, int n, const double* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIdamin(hipblasHandle_t handle, int n, const double* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);

  // NOTE: int CUBLASWINAPI cublasIcamin(int n, const cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIcamin_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIcamin(hipblasHandle_t handle, int n, const hipblasComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIcamin(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamin(blasHandle, n, &complexx, incx, &res);

  // NOTE: int CUBLASWINAPI cublasIzamin(int n, const cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIzamin_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIzamin(hipblasHandle_t handle, int n, const hipblasDoubleComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIzamin(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamin(blasHandle, n, &dcomplexx, incx, &res);

  // NOTE: float CUBLASWINAPI cublasSasum(int n, const float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSasum_v2(cublasHandle_t handle, int n, const float* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSasum(hipblasHandle_t handle, int n, const float* x, int incx, float* result);
  // CHECK: blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);

  // NOTE: double CUBLASWINAPI cublasDasum(int n, const double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDasum_v2(cublasHandle_t handle, int n, const double* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDasum(hipblasHandle_t handle, int n, const double* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);

  // NOTE: float CUBLASWINAPI cublasScasum(int n, const cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScasum_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScasum(hipblasHandle_t handle, int n, const hipblasComplex* x, int incx, float* result);
  // CHECK: blasStatus = hipblasScasum(blasHandle, n, &complexx, incx, &fresult);
  blasStatus = hipblasScasum(blasHandle, n, &complexx, incx, &fresult);

  // NOTE: double CUBLASWINAPI cublasDzasum(int n, const cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDzasum_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDzasum(hipblasHandle_t handle, int n, const hipblasDoubleComplex* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDzasum(blasHandle, n, &dcomplexx, incx, &dresult);
  blasStatus = hipblasDzasum(blasHandle, n, &dcomplexx, incx, &dresult);

  // NOTE: void CUBLASWINAPI cublasSrot(int n, float* x, int incx, float* y, int incy, float sc, float ss); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrot_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* c, const float* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrot(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* c, const float* s);
  // CHECK: blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);
  blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);

  // NOTE: void CUBLASWINAPI cublasDrot(int n, double* x, int incx, double* y, int incy, double sc, double ss); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrot_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* c, const double* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrot(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* c, const double* s);
  // CHECK: blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);
  blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);

  // NOTE: void CUBLASWINAPI cublasCrot(int n, cuComplex* x, int incx, cuComplex* y, int incy, float c, cuComplex s); is not supported by HIP
  // CUDA: CUBLASAPI CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCrot_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy, const float* c, const cuComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCrot(hipblasHandle_t handle, int n, hipblasComplex* x, int incx, hipblasComplex* y, int incy, const float* c, const hipblasComplex* s);
  // CHECK: blasStatus = hipblasCrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);
  blasStatus = hipblasCrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);

  // NOTE: void CUBLASWINAPI cublasCsrot(int n, cuComplex* x, int incx, cuComplex* y, int incy, float c, float s); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsrot_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy, const float* c, const float* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsrot(hipblasHandle_t handle, int n, hipblasComplex* x, int incx, hipblasComplex* y, int incy, const float* c, const float* s);
  // CHECK: blasStatus = hipblasCsrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);
  blasStatus = hipblasCsrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);

  // NOTE: void CUBLASWINAPI cublasZrot(int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, double sc, cuDoubleComplex cs); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZrot_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, const double* c, const cuDoubleComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZrot(hipblasHandle_t handle, int n, hipblasDoubleComplex* x, int incx, hipblasDoubleComplex* y, int incy, const double* c, const hipblasDoubleComplex* s);
  // CHECK: blasStatus = hipblasZrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);
  blasStatus = hipblasZrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);

  // NOTE: void CUBLASWINAPI cublasZdrot(int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, double c, double s); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdrot_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, const double* c, const double* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdrot(hipblasHandle_t handle, int n, hipblasDoubleComplex* x, int incx, hipblasDoubleComplex* y, int incy, const double* c, const double* s);
  // CHECK: blasStatus = hipblasZdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);
  blasStatus = hipblasZdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);

  // NOTE: void CUBLASWINAPI cublasSrotg(float* sa, float* sb, float* sc, float* ss); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotg_v2(cublasHandle_t handle, float* a, float* b, float* c, float* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrotg(hipblasHandle_t handle, float* a, float* b, float* c, float* s);
  // CHECK: blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);
  blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);

  // NOTE: void CUBLASWINAPI cublasDrotg(double* sa, double* sb, double* sc, double* ss); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotg_v2(cublasHandle_t handle, double* a, double* b, double* c, double* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrotg(hipblasHandle_t handle, double* a, double* b, double* c, double* s);
  // CHECK: blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);
  blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);

  // NOTE: void CUBLASWINAPI cublasCrotg(cuComplex* ca, cuComplex cb, float* sc, cuComplex* cs); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCrotg_v2(cublasHandle_t handle, cuComplex* a, cuComplex* b, float* c, cuComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCrotg(hipblasHandle_t handle, hipblasComplex* a, hipblasComplex* b, float* c, hipblasComplex* s);
  // CHECK: blasStatus = hipblasCrotg(blasHandle, &complexa, &complexb, &fc, &complexs);
  blasStatus = hipblasCrotg(blasHandle, &complexa, &complexb, &fc, &complexs);

  // NOTE: void CUBLASWINAPI cublasZrotg(cuDoubleComplex* ca, cuDoubleComplex cb, double* sc, cuDoubleComplex* cs); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZrotg_v2(cublasHandle_t handle, cuDoubleComplex* a, cuDoubleComplex* b, double* c, cuDoubleComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZrotg(hipblasHandle_t handle, hipblasDoubleComplex* a, hipblasDoubleComplex* b, double* c, hipblasDoubleComplex* s);
  // CHECK: blasStatus = hipblasZrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);
  blasStatus = hipblasZrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);

  // NOTE: void CUBLASWINAPI cublasSrotm(int n, float* x, int incx, float* y, int incy, const float* sparam); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotm_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrotm(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* param);
  // CHECK: blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);

  // NOTE: void CUBLASWINAPI cublasDrotm(int n, double* x, int incx, double* y, int incy, const double* sparam); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotm_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrotm(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* param);
  // CHECK: blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);

  // NOTE: void CUBLASWINAPI cublasSrotmg(float* sd1, float* sd2, float* sx1, const float* sy1, float* sparam); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotmg_v2(cublasHandle_t handle, float* d1, float* d2, float* x1, const float* y1, float* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrotmg(hipblasHandle_t handle, float* d1, float* d2, float* x1, const float* y1, float* param);
  // CHECK: blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);
  blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);

  // NOTE: void CUBLASWINAPI cublasDrotmg(double* sd1, double* sd2, double* sx1, const double* sy1, double* sparam); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotmg_v2(cublasHandle_t handle, double* d1, double* d2, double* x1, const double* y1, double* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrotmg(hipblasHandle_t handle, double* d1, double* d2, double* x1, const double* y1, double* param);
  // CHECK: blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);
  blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);

  // NOTE: void CUBLASWINAPI cublasSgemv(char trans, int m, int n, float alpha, const float* A, int lda, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDgemv(char trans, int m, int n, double alpha, const double* A, int lda, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // NOTE: void CUBLASWINAPI cublasCgemv(char trans, int m, int n, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex beta, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* x, int incx, const hipblasComplex* beta, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasCgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZgemv(char trans, int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* beta, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasSgbmv(char trans, int m, int n, int kl, int ku, float alpha, const float* A, int lda, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDgbmv(char trans, int m, int n, int kl, int ku, double alpha, const double* A, int lda, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // NOTE: void CUBLASWINAPI cublasCgbmv(char trans, int m, int n, int kl, int ku, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex beta, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* x, int incx, const hipblasComplex* beta, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasCgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZgbmv(char trans, int m, int n, int kl, int ku, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* beta, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasStrmv(char uplo, char trans, char diag, int n, const float* A, int lda, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);

  // NOTE: void CUBLASWINAPI cublasDtrmv(char uplo, char trans, char diag, int n, const double* A, int lda, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);

  // NOTE: void CUBLASWINAPI cublasCtrmv(char uplo, char trans, char diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasComplex* AP, int lda, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasZtrmv(char uplo, char trans, char diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasDoubleComplex* AP, int lda, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasStbmv(char uplo, char trans, char diag, int n, int k, const float* A, int lda, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);

  // NOTE: void CUBLASWINAPI cublasDtbmv(char uplo, char trans, char diag, int n, int k, const double* A, int lda, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);

  // NOTE: void CUBLASWINAPI cublasCtbmv(char uplo, char trans, char diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const hipblasComplex* AP, int lda, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasZtbmv(char uplo, char trans, char diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const hipblasDoubleComplex* AP, int lda, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasStpmv(char uplo, char trans, char diag, int n, const float* AP, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* AP, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, float* x, int incx);
  // CHECK: blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);

  // NOTE: void CUBLASWINAPI cublasDtpmv(char uplo, char trans, char diag, int n, const double* AP, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* AP, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, double* x, int incx);
  // CHECK: blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);

  // NOTE: void CUBLASWINAPI cublasCtpmv(char uplo, char trans, char diag, int n, const cuComplex* AP, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* AP, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasComplex* AP, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasZtpmv(char uplo, char trans, char diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasDoubleComplex* AP, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasStrsv(char uplo, char trans, char diag, int n, const float* A, int lda, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);

  // NOTE: void CUBLASWINAPI cublasDtrsv(char uplo, char trans, char diag, int n, const double* A, int lda, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);

  // NOTE: void CUBLASWINAPI cublasCtrsv(char uplo, char trans, char diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasComplex* AP, int lda, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasZtrsv(char uplo, char trans, char diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasDoubleComplex* AP, int lda, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasStpsv(char uplo, char trans, char diag, int n, const float* AP, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* AP, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, float* x, int incx);
  // CHECK: blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);

  // NOTE: void CUBLASWINAPI cublasDtpsv(char uplo, char trans, char diag, int n, const double* AP, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* AP, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, double* x, int incx);
  // CHECK: blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);

  // NOTE: void CUBLASWINAPI cublasCtpsv(char uplo, char trans, char diag, int n, const cuComplex* AP, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* AP, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasComplex* AP, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasZtpsv(char uplo, char trans, char diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasDoubleComplex* AP, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasStbsv(char uplo, char trans, char diag, int n, int k, const float* A, int lda, float* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);

  // NOTE: void CUBLASWINAPI cublasDtbsv(char uplo, char trans, char diag, int n, int k, const double* A, int lda, double* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);

  // NOTE: void CUBLASWINAPI cublasCtbsv(char uplo, char trans, char diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const hipblasComplex* AP, int lda, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);

  // NOTE: void CUBLASWINAPI cublasZtbsv(char uplo, char trans, char diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const hipblasDoubleComplex* AP, int lda, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);

  // NOTE: void CUBLASWINAPI cublasSsymv(char uplo, int n, float alpha, const float* A, int lda, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDsymv(char uplo, int n, double alpha, const double* A, int lda, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* x, int incx, const hipblasComplex* beta, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasCsymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCsymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* beta, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasChemv(char uplo, int n, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex beta, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChemv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChemv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* x, int incx, const hipblasComplex* beta, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasChemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZhemv(char uplo, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhemv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhemv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* beta, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasSsbmv(char uplo, int n, int k, float alpha, const float* A, int lda, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDsbmv(char uplo, int n, int k, double alpha, const double* A, int lda, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // NOTE: void CUBLASWINAPI cublasChbmv(char uplo, int n, int k, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex beta, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* x, int incx, const hipblasComplex* beta, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasChbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZhbmv(char uplo, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* beta, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasSspmv(char uplo, int n, float alpha, const float* AP, const float* x, int incx, float beta, float* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* AP, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSspmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* AP, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);

  // NOTE: void CUBLASWINAPI cublasDspmv(char uplo, int n, double alpha, const double* AP, const double* x, int incx, double beta, double* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* AP, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDspmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* AP, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);

  // NOTE: void CUBLASWINAPI cublasChpmv(char uplo, int n, cuComplex alpha, const cuComplex* AP, const cuComplex* x, int incx, cuComplex beta, cuComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* AP, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasComplex* alpha, const hipblasComplex* AP, const hipblasComplex* x, int incx, const hipblasComplex* beta, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasChpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);

  // NOTE: void CUBLASWINAPI cublasZhpmv(char uplo, int n, cuDoubleComplex alpha, const cuDoubleComplex* AP, const cuDoubleComplex* x, int incx, cuDoubleComplex beta, cuDoubleComplex* y, int incy); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* AP, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* beta, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // NOTE: void CUBLASWINAPI cublasSger(int m, int n, float alpha, const float* x, int incx, const float* y, int incy, float* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSger_v2(cublasHandle_t handle, int m, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSger(hipblasHandle_t handle, int m, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* AP, int lda);
  // CHECK: blasStatus = hipblasSger(blasHandle, m, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  blasStatus = hipblasSger(blasHandle, m, n, &fa, &fx, incx, &fy, incy, &fA, lda);

  // NOTE: void CUBLASWINAPI cublasDger(int m, int n, double alpha, const double* x, int incx, const double* y, int incy, double* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDger_v2(cublasHandle_t handle, int m, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDger(hipblasHandle_t handle, int m, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* AP, int lda);
  // CHECK: blasStatus = hipblasDger(blasHandle, m, n, &da, &dx, incx, &dy, incy, &dA, lda);
  blasStatus = hipblasDger(blasHandle, m, n, &da, &dx, incx, &dy, incy, &dA, lda);

  // NOTE: void CUBLASWINAPI cublasCgeru(int m, int n, cuComplex alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgeru_v2(cublasHandle_t handle, int m, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgeru(hipblasHandle_t handle, int m, int n, const hipblasComplex* alpha, const hipblasComplex* x, int incx, const hipblasComplex* y, int incy, hipblasComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCgeru(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCgeru(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // NOTE: void CUBLASWINAPI cublasCgerc(int m, int n, cuComplex alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgerc_v2(cublasHandle_t handle, int m, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgerc(hipblasHandle_t handle, int m, int n, const hipblasComplex* alpha, const hipblasComplex* x, int incx, const hipblasComplex* y, int incy, hipblasComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCgerc(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCgerc(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // NOTE: void CUBLASWINAPI cublasZgeru(int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgeru_v2(cublasHandle_t handle, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgeru(hipblasHandle_t handle, int m, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* y, int incy, hipblasDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZgeru(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZgeru(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // NOTE: void CUBLASWINAPI cublasZgerc(int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgerc_v2(cublasHandle_t handle, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgerc(hipblasHandle_t handle, int m, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* y, int incy, hipblasDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZgerc(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZgerc(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // NOTE: void CUBLASWINAPI cublasSsyr(char uplo, int n, float alpha, const float* x, int incx, float* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* AP, int lda);
  // CHECK: blasStatus = hipblasSsyr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA, lda);
  blasStatus = hipblasSsyr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA, lda);

  // NOTE: void CUBLASWINAPI cublasDsyr(char uplo, int n, double alpha, const double* x, int incx, double* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* AP, int lda);
  // CHECK: blasStatus = hipblasDsyr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA, lda);
  blasStatus = hipblasDsyr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasComplex* alpha, const hipblasComplex* x, int incx, hipblasComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCsyr(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexA, lda);
  blasStatus = hipblasCsyr(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* x, int incx, hipblasDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZsyr(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexA, lda);
  blasStatus = hipblasZsyr(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexA, lda);

  // NOTE: void CUBLASWINAPI cublasCher(char uplo, int n, float alpha, const cuComplex* x, int incx, cuComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const cuComplex* x, int incx, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCher(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const hipblasComplex* x, int incx, hipblasComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCher(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA, lda);
  blasStatus = hipblasCher(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA, lda);

  // NOTE: void CUBLASWINAPI cublasZher(char uplo, int n, double alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZher(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const hipblasDoubleComplex* x, int incx, hipblasDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZher(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA, lda);
  blasStatus = hipblasZher(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA, lda);

  // NOTE: void CUBLASWINAPI cublasSspr(char uplo, int n, float alpha, const float* x, int incx, float* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSspr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* AP);
  // CHECK: blasStatus = hipblasSspr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA);
  blasStatus = hipblasSspr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA);

  // NOTE: void CUBLASWINAPI cublasDspr(char uplo, int n, double alpha, const double* x, int incx, double* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, double* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDspr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, double* AP);
  // CHECK: blasStatus = hipblasDspr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA);
  blasStatus = hipblasDspr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA);

  // NOTE: void CUBLASWINAPI cublasChpr(char uplo, int n, float alpha, const cuComplex* x, int incx, cuComplex* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const cuComplex* x, int incx, cuComplex* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChpr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const hipblasComplex* x, int incx, hipblasComplex* AP);
  // CHECK: blasStatus = hipblasChpr(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA);
  blasStatus = hipblasChpr(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA);

  // NOTE: void CUBLASWINAPI cublasZhpr(char uplo, int n, double alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhpr(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const hipblasDoubleComplex* x, int incx, hipblasDoubleComplex* AP);
  // CHECK: blasStatus = hipblasZhpr(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA);
  blasStatus = hipblasZhpr(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA);

  // NOTE: void CUBLASWINAPI cublasSsyr2(char uplo, int n, float alpha, const float* x, int incx, const float* y, int incy, float* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* AP, int lda);
  // CHECK: blasStatus = hipblasSsyr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  blasStatus = hipblasSsyr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA, lda);

  // NOTE: void CUBLASWINAPI cublasDsyr2(char uplo, int n, double alpha, const double* x, int incx, const double* y, int incy, double* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* AP, int lda);
  // CHECK: blasStatus = hipblasDsyr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA, lda);
  blasStatus = hipblasDsyr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasComplex* alpha, const hipblasComplex* x, int incx, const hipblasComplex* y, int incy, hipblasComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCsyr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCsyr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* y, int incy, hipblasDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZsyr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZsyr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // NOTE: void CUBLASWINAPI cublasCher2(char uplo, int n, cuComplex alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCher2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasComplex* alpha, const hipblasComplex* x, int incx, const hipblasComplex* y, int incy, hipblasComplex* AP, int lda);
  // CHECK: blasStatus = hipblasCher2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCher2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // NOTE: void CUBLASWINAPI cublasZher2(char uplo, int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZher2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* y, int incy, hipblasDoubleComplex* AP, int lda);
  // CHECK: blasStatus = hipblasZher2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZher2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // NOTE: void CUBLASWINAPI cublasSspr2(char uplo, int n, float alpha, const float* x, int incx, const float* y, int incy, float* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSspr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* AP);
  // CHECK: blasStatus = hipblasSspr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA);
  blasStatus = hipblasSspr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA);

  // NOTE: void CUBLASWINAPI cublasDspr2(char uplo, int n, double alpha, const double* x, int incx, const double* y, int incy, double* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDspr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* AP);
  // CHECK: blasStatus = hipblasDspr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA);
  blasStatus = hipblasDspr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA);

  // NOTE: void CUBLASWINAPI cublasChpr2(char uplo, int n, cuComplex alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChpr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasComplex* alpha, const hipblasComplex* x, int incx, const hipblasComplex* y, int incy, hipblasComplex* AP);
  // CHECK: blasStatus = hipblasChpr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA);
  blasStatus = hipblasChpr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA);

  // NOTE: void CUBLASWINAPI cublasZhpr2(char uplo, int n, cuDoubleComplex alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* AP); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* AP);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhpr2(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* y, int incy, hipblasDoubleComplex* AP);
  // CHECK: blasStatus = hipblasZhpr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA);
  blasStatus = hipblasZhpr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA);

  hipblasOperation_t transa, transb;

  // NOTE: void CUBLASWINAPI cublasSgemm(char transa, char transb, int m, int n, int k, float alpha, const float* A, int lda, const float* B, int ldb, float beta, float* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgemm(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const float* alpha, const float* AP, int lda, const float* BP, int ldb, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSgemm(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSgemm(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // NOTE: void CUBLASWINAPI cublasDgemm(char transa, char transb, int m, int n, int k, double alpha, const double* A, int lda, const double* B, int ldb, double beta, double* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgemm(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const double* alpha, const double* AP, int lda, const double* BP, int ldb, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDgemm(blasHandle, transa, transb, m, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDgemm(blasHandle, transa, transb, m, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // NOTE: void CUBLASWINAPI cublasCgemm(char transa, char transb, int m, int n, int k, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemm(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* BP, int ldb, const hipblasComplex* beta, hipblasComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCgemm(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCgemm(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZgemm(char transa, char transb, int m, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemm(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* BP, int ldb, const hipblasDoubleComplex* beta, hipblasDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZgemm(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZgemm(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // TODO: __half -> hipblasHalf
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemm(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* A, int lda, const __half* B, int ldb, const __half* beta, __half* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasHgemm(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasHalf* alpha, const hipblasHalf* AP, int lda, const hipblasHalf* BP, int ldb, const hipblasHalf* beta, hipblasHalf* CP, int ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* const Aarray[], int lda, const float* const Barray[], int ldb, const float* beta, float* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgemmBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const float* alpha, const float* const AP[], int lda, const float* const BP[], int ldb, const float* beta, float* const CP[], int ldc, int batchCount);
  // CHECK: blasStatus = hipblasSgemmBatched(blasHandle, transa, transb, m, n, k, &fa, fAarray_const, lda, fBarray_const, ldb, &fb, fCarray, ldc, batchCount);
  blasStatus = hipblasSgemmBatched(blasHandle, transa, transb, m, n, k, &fa, fAarray_const, lda, fBarray_const, ldb, &fb, fCarray, ldc, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* const Aarray[], int lda, const double* const Barray[], int ldb, const double* beta, double* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgemmBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const double* alpha, const double* const AP[], int lda, const double* const BP[], int ldb, const double* beta, double* const CP[], int ldc, int batchCount);
  // CHECK: blasStatus = hipblasDgemmBatched(blasHandle, transa, transb, m, n, k, &da, dAarray_const, lda, dBarray_const, ldb, &db, dCarray, ldc, batchCount);
  blasStatus = hipblasDgemmBatched(blasHandle, transa, transb, m, n, k, &da, dAarray_const, lda, dBarray_const, ldb, &db, dCarray, ldc, batchCount);

  // TODO: __half -> hipblasHalf
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* const Aarray[], int lda, const __half* const Barray[], int ldb, const __half* beta, __half* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasHgemmBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasHalf* alpha, const hipblasHalf* const AP[], int lda, const hipblasHalf* const BP[], int ldb, const hipblasHalf* beta, hipblasHalf* const CP[], int ldc, int batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* const Aarray[], int lda, const cuComplex* const Barray[], int ldb, const cuComplex* beta, cuComplex* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemmBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasComplex* alpha, const hipblasComplex* const AP[], int lda, const hipblasComplex* const BP[], int ldb, const hipblasComplex* beta, hipblasComplex* const CP[], int ldc, int batchCount);
  // CHECK: blasStatus = hipblasCgemmBatched(blasHandle, transa, transb, m, n, k, &complexa, complexAarray_const, lda, complexBarray_const, ldb, &complexb, complexCarray, ldc, batchCount);
  blasStatus = hipblasCgemmBatched(blasHandle, transa, transb, m, n, k, &complexa, complexAarray_const, lda, complexBarray_const, ldb, &complexb, complexCarray, ldc, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* const Aarray[], int lda, const cuDoubleComplex* const Barray[], int ldb, const cuDoubleComplex* beta, cuDoubleComplex* const Carray[], int ldc, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemmBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* const AP[], int lda, const hipblasDoubleComplex* const BP[], int ldb, const hipblasDoubleComplex* beta, hipblasDoubleComplex* const CP[], int ldc, int batchCount);
  // CHECK: blasStatus = hipblasZgemmBatched(blasHandle, transa, transb, m, n, k, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray_const, ldb, &dcomplexb, dcomplexCarray, ldc, batchCount);
  blasStatus = hipblasZgemmBatched(blasHandle, transa, transb, m, n, k, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray_const, ldb, &dcomplexb, dcomplexCarray, ldc, batchCount);

  // NOTE: void CUBLASWINAPI cublasSsyrk(char uplo, char trans, int n, int k, float alpha, const float* A, int lda, float beta, float* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyrk(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const float* alpha, const float* AP, int lda, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSsyrk(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, &fC, ldc);
  blasStatus = hipblasSsyrk(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, &fC, ldc);

  // NOTE: void CUBLASWINAPI cublasDsyrk(char uplo, char trans, int n, int k, double alpha, const double* A, int lda, double beta, double* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyrk(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const double* alpha, const double* AP, int lda, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDsyrk(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, &dC, ldc);
  blasStatus = hipblasDsyrk(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, &dC, ldc);

  // NOTE: void CUBLASWINAPI cublasCsyrk(char uplo, char trans, int n, int k, cuComplex alpha, const cuComplex* A, int lda, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyrk(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* beta, hipblasComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCsyrk(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyrk(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZsyrk(char uplo, char trans, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyrk(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* beta, hipblasDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZsyrk(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyrk(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexC, ldc);

  // NOTE: void CUBLASWINAPI cublasCherk(char uplo, char trans, int n, int k, float alpha, const cuComplex* A, int lda, float beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCherk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const cuComplex* A, int lda, const float* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCherk(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const float* alpha, const hipblasComplex* AP, int lda, const float* beta, hipblasComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCherk(blasHandle, blasFillMode, transa, n, k, &fa, &complexA, lda, &fb, &complexC, ldc);
  blasStatus = hipblasCherk(blasHandle, blasFillMode, transa, n, k, &fa, &complexA, lda, &fb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZherk(char uplo, char trans, int n, int k, double alpha, const cuDoubleComplex* A, int lda, double beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZherk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const cuDoubleComplex* A, int lda, const double* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZherk(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const double* alpha, const hipblasDoubleComplex* AP, int lda, const double* beta, hipblasDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZherk(blasHandle, blasFillMode, transa, n, k, &da, &dcomplexA, lda, &db, &dcomplexC, ldc);
  blasStatus = hipblasZherk(blasHandle, blasFillMode, transa, n, k, &da, &dcomplexA, lda, &db, &dcomplexC, ldc);

  // NOTE: void CUBLASWINAPI cublasSsyr2k(char uplo, char trans, int n, int k, float alpha, const float* A, int lda, const float* B, int ldb, float beta, float* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyr2k(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const float* alpha, const float* AP, int lda, const float* BP, int ldb, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSsyr2k(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsyr2k(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, ldb, &fb, &fC, ldc);

  // NOTE: void CUBLASWINAPI cublasDsyr2k(char uplo, char trans, int n, int k, double alpha, const double* A, int lda, const double* B, int ldb, double beta, double* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyr2k(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const double* alpha, const double* AP, int lda, const double* BP, int ldb, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDsyr2k(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsyr2k(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, ldb, &db, &dC, ldc);

  // NOTE: void CUBLASWINAPI cublasCsyr2k(char uplo, char trans, int n, int k, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyr2k(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* BP, int ldb, const hipblasComplex* beta, hipblasComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCsyr2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyr2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &complexb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZsyr2k(char uplo, char trans, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyr2k(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* BP, int ldb, const hipblasDoubleComplex* beta, hipblasDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZsyr2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyr2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &dcomplexb, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsyrkx(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const float* alpha, const float* AP, int lda, const float* BP, int ldb, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSsyrkx(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsyrkx(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsyrkx(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const double* alpha, const double* AP, int lda, const double* BP, int ldb, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDsyrkx(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsyrkx(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsyrkx(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* BP, int ldb, const hipblasComplex* beta, hipblasComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCsyrkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyrkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsyrkx(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* BP, int ldb, const hipblasDoubleComplex* beta, hipblasDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZsyrkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyrkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // NOTE: void CUBLASWINAPI cublasCher2k(char uplo, char trans, int n, int k, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, float beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const float* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCher2k(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* BP, int ldb, const float* beta, hipblasComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCher2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &fb, &complexC, ldc);
  blasStatus = hipblasCher2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &fb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZher2k(char uplo, char trans, int n, int k, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, double beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const double* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZher2k(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* BP, int ldb, const double* beta, hipblasDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZher2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &db, &dcomplexC, ldc);
  blasStatus = hipblasZher2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &db, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCherkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const float* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCherkx(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* BP, int ldb, const float* beta, hipblasComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCherkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &fb, &complexC, ldc);
  blasStatus = hipblasCherkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &fb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZherkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const double* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZherkx(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, int n, int k, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* BP, int ldb, const double* beta, hipblasDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZherkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &db, &dcomplexC, ldc);
  blasStatus = hipblasZherkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &db, &dcomplexC, ldc);

  // NOTE: void CUBLASWINAPI cublasSsymm(char side, char uplo, int m, int n, float alpha, const float* A, int lda, const float* B, int ldb, float beta, float* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsymm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int m, int n, const float* alpha, const float* AP, int lda, const float* BP, int ldb, const float* beta, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSsymm(blasHandle, blasSideMode, blasFillMode, m, n, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsymm(blasHandle, blasSideMode, blasFillMode, m, n, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // NOTE: void CUBLASWINAPI cublasDsymm(char side, char uplo, int m, int n, double alpha, const double* A, int lda, const double* B, int ldb, double beta, double* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsymm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int m, int n, const double* alpha, const double* AP, int lda, const double* BP, int ldb, const double* beta, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDsymm(blasHandle, blasSideMode, blasFillMode, m, n, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsymm(blasHandle, blasSideMode, blasFillMode, m, n, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // NOTE: void CUBLASWINAPI cublasCsymm(char side, char uplo, int m, int n, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsymm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int m, int n, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* BP, int ldb, const hipblasComplex* beta, hipblasComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCsymm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsymm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZsymm(char side, char uplo, int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsymm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int m, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* BP, int ldb, const hipblasDoubleComplex* beta, hipblasDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZsymm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsymm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // NOTE: void CUBLASWINAPI cublasChemm(char side, char uplo, int m, int n, cuComplex alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex beta, cuComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChemm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChemm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int m, int n, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* BP, int ldb, const hipblasComplex* beta, hipblasComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasChemm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasChemm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // NOTE: void CUBLASWINAPI cublasZhemm(char side, char uplo, int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex beta, cuDoubleComplex* C, int ldc); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhemm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhemm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, int m, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* BP, int ldb, const hipblasDoubleComplex* beta, hipblasDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZhemm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZhemm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // NOTE: void CUBLASWINAPI cublasStrsm(char side, char uplo, char transa, char diag, int m, int n, float alpha, const float* A, int lda, float* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const float* alpha, const float* A, int lda, float* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrsm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const float* alpha, float* AP, int lda, float* BP, int ldb);
  // CHECK: blasStatus = hipblasStrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb);
  blasStatus = hipblasStrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb);

  // NOTE: void CUBLASWINAPI cublasDtrsm(char side, char uplo, char transa, char diag, int m, int n, double alpha, const double* A, int lda, double* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const double* alpha, const double* A, int lda, double* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrsm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const double* alpha, double* AP, int lda, double* BP, int ldb);
  // CHECK: blasStatus = hipblasDtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb);
  blasStatus = hipblasDtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb);

  // NOTE: void CUBLASWINAPI cublasCtrsm(char side, char uplo, char transa, char diag, int m, int n, cuComplex alpha, const cuComplex* A, int lda, cuComplex* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, cuComplex* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrsm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const hipblasComplex* alpha, hipblasComplex* AP, int lda, hipblasComplex* BP, int ldb);
  // CHECK: blasStatus = hipblasCtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb);
  blasStatus = hipblasCtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb);

  // NOTE: void CUBLASWINAPI cublasZtrsm(char side, char uplo, char transa, char diag, int m, int n, cuDoubleComplex alpha, const cuDoubleComplex* A, int lda, cuDoubleComplex* B, int ldb); is not supported by HIP
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, cuDoubleComplex* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrsm(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const hipblasDoubleComplex* alpha, hipblasDoubleComplex* AP, int lda, hipblasDoubleComplex* BP, int ldb);
  // CHECK: blasStatus = hipblasZtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb);
  blasStatus = hipblasZtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const float* alpha, const float* A, int lda, const float* beta, const float* B, int ldb, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgeam(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, const float* alpha, const float* AP, int lda, const float* beta, const float* BP, int ldb, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSgeam(blasHandle, transa, transb, m, n, &fa, &fA, lda, &fb, &fB, ldb, &fC, ldc);
  blasStatus = hipblasSgeam(blasHandle, transa, transb, m, n, &fa, &fA, lda, &fb, &fB, ldb, &fC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const double* alpha, const double* A, int lda, const double* beta, const double* B, int ldb, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgeam(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, const double* alpha, const double* AP, int lda, const double* beta, const double* BP, int ldb, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDgeam(blasHandle, transa, transb, m, n, &da, &dA, lda, &db, &dB, ldb, &dC, ldc);
  blasStatus = hipblasDgeam(blasHandle, transa, transb, m, n, &da, &dA, lda, &db, &dB, ldb, &dC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* beta, const cuComplex* B, int ldb, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgeam(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* beta, const hipblasComplex* BP, int ldb, hipblasComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCgeam(blasHandle, transa, transb, m, n, &complexa, &complexA, lda, &complexb, &complexB, ldb, &complexC, ldc);
  blasStatus = hipblasCgeam(blasHandle, transa, transb, m, n, &complexa, &complexA, lda, &complexb, &complexB, ldb, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* beta, const cuDoubleComplex* B, int ldb, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgeam(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* beta, const hipblasDoubleComplex* BP, int ldb, hipblasDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZgeam(blasHandle, transa, transb, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexB, ldb, &dcomplexC, ldc);
  blasStatus = hipblasZgeam(blasHandle, transa, transb, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexB, ldb, &dcomplexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgetrfBatched(cublasHandle_t handle, int n, float* const A[], int lda, int* P, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgetrfBatched(hipblasHandle_t handle, const int n, float* const A[], const int lda, int* ipiv, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasSgetrfBatched(blasHandle, n, fAarray, lda, &P, &info, batchCount);
  blasStatus = hipblasSgetrfBatched(blasHandle, n, fAarray, lda, &P, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgetrfBatched(cublasHandle_t handle, int n, double* const A[], int lda, int* P, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgetrfBatched(hipblasHandle_t handle, const int n, double* const A[], const int lda, int* ipiv, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasDgetrfBatched(blasHandle, n, dAarray, lda, &P, &info, batchCount);
  blasStatus = hipblasDgetrfBatched(blasHandle, n, dAarray, lda, &P, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgetrfBatched(cublasHandle_t handle, int n, cuComplex* const A[], int lda, int* P, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgetrfBatched(hipblasHandle_t handle, const int n, hipblasComplex* const A[], const int lda, int* ipiv, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasCgetrfBatched(blasHandle, n, complexAarray, lda, &P, &info, batchCount);
  blasStatus = hipblasCgetrfBatched(blasHandle, n, complexAarray, lda, &P, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgetrfBatched(cublasHandle_t handle, int n, cuDoubleComplex* const A[], int lda, int* P, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgetrfBatched(hipblasHandle_t handle, const int n, hipblasDoubleComplex* const A[], const int lda, int* ipiv, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasZgetrfBatched(blasHandle, n, dcomplexAarray, lda, &P, &info, batchCount);
  blasStatus = hipblasZgetrfBatched(blasHandle, n, dcomplexAarray, lda, &P, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgetriBatched(cublasHandle_t handle, int n, const float* const A[], int lda, const int* P, float* const C[], int ldc, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgetriBatched(hipblasHandle_t handle, const int n, float* const A[], const int lda, int* ipiv, float* const C[], const int ldc, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasSgetriBatched(blasHandle, n, fAarray_const, lda, &P, fCarray, ldc, &info, batchCount);
  blasStatus = hipblasSgetriBatched(blasHandle, n, fAarray_const, lda, &P, fCarray, ldc, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgetriBatched(cublasHandle_t handle, int n, const double* const A[], int lda, const int* P, double* const C[], int ldc, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgetriBatched(hipblasHandle_t handle, const int n, double* const A[], const int lda, int* ipiv, double* const C[], const int ldc, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasDgetriBatched(blasHandle, n, dAarray_const, lda, &P, dCarray, ldc, &info, batchCount);
  blasStatus = hipblasDgetriBatched(blasHandle, n, dAarray_const, lda, &P, dCarray, ldc, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgetriBatched(cublasHandle_t handle, int n, const cuComplex* const A[], int lda, const int* P, cuComplex* const C[], int ldc, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgetriBatched(hipblasHandle_t handle, const int n, hipblasComplex* const A[], const int lda, int* ipiv, hipblasComplex* const C[], const int ldc, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasCgetriBatched(blasHandle, n, complexAarray_const, lda, &P, complexCarray, ldc, &info, batchCount);
  blasStatus = hipblasCgetriBatched(blasHandle, n, complexAarray_const, lda, &P, complexCarray, ldc, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgetriBatched(cublasHandle_t handle, int n, const cuDoubleComplex* const A[], int lda, const int* P, cuDoubleComplex* const C[], int ldc, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgetriBatched(hipblasHandle_t handle, const int n, hipblasDoubleComplex* const A[], const int lda, int* ipiv, hipblasDoubleComplex* const C[], const int ldc, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasZgetriBatched(blasHandle, n, dcomplexAarray_const, lda, &P, dcomplexCarray, ldc, &info, batchCount);
  blasStatus = hipblasZgetriBatched(blasHandle, n, dcomplexAarray_const, lda, &P, dcomplexCarray, ldc, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgetrsBatched(cublasHandle_t handle, cublasOperation_t trans, int n, int nrhs, const float* const Aarray[], int lda, const int* devIpiv, float* const Barray[], int ldb, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgetrsBatched(hipblasHandle_t handle, const hipblasOperation_t trans, const int n, const int nrhs, float* const A[], const int lda, const int* ipiv, float* const B[], const int ldb, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasSgetrsBatched(blasHandle, transa, n, nrhs, fAarray_const, lda, &P, fBarray, ldb, &info, batchCount);
  blasStatus = hipblasSgetrsBatched(blasHandle, transa, n, nrhs, fAarray_const, lda, &P, fBarray, ldb, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgetrsBatched(cublasHandle_t handle, cublasOperation_t trans, int n, int nrhs, const double* const Aarray[], int lda, const int* devIpiv, double* const Barray[], int ldb, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgetrsBatched(hipblasHandle_t handle, const hipblasOperation_t trans, const int n, const int nrhs, double* const A[], const int lda, const int* ipiv, double* const B[], const int ldb, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasDgetrsBatched(blasHandle, transa, n, nrhs, dAarray_const, lda, &P, dBarray, ldb, &info, batchCount);
  blasStatus = hipblasDgetrsBatched(blasHandle, transa, n, nrhs, dAarray_const, lda, &P, dBarray, ldb, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgetrsBatched(cublasHandle_t handle, cublasOperation_t trans, int n, int nrhs, const cuComplex* const Aarray[], int lda, const int* devIpiv, cuComplex* const Barray[], int ldb, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgetrsBatched(hipblasHandle_t handle, const hipblasOperation_t trans, const int n, const int nrhs, hipblasComplex* const A[], const int lda, const int* ipiv, hipblasComplex* const B[], const int ldb, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasCgetrsBatched(blasHandle, transa, n, nrhs, complexAarray_const, lda, &P, complexBarray, ldb, &info, batchCount);
  blasStatus = hipblasCgetrsBatched(blasHandle, transa, n, nrhs, complexAarray_const, lda, &P, complexBarray, ldb, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgetrsBatched(cublasHandle_t handle, cublasOperation_t trans, int n, int nrhs, const cuDoubleComplex* const Aarray[], int lda, const int* devIpiv, cuDoubleComplex* const Barray[], int ldb, int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgetrsBatched(hipblasHandle_t handle, const hipblasOperation_t trans, const int n, const int nrhs, hipblasDoubleComplex* const A[], const int lda, const int* ipiv, hipblasDoubleComplex* const B[], const int ldb, int* info, const int batchCount);
  // CHECK: blasStatus = hipblasZgetrsBatched(blasHandle, transa, n, nrhs, dcomplexAarray_const, lda, &P, dcomplexBarray, ldb, &info, batchCount);
  blasStatus = hipblasZgetrsBatched(blasHandle, transa, n, nrhs, dcomplexAarray_const, lda, &P, dcomplexBarray, ldb, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const float* alpha, const float* const A[], int lda, float* const B[], int ldb, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrsmBatched(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const float* alpha, float* const AP[], int lda, float* BP[], int ldb, int batchCount);
  // CHECK: blasStatus = hipblasStrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, fAarray_const, lda, fBarray, ldb, batchCount);
  blasStatus = hipblasStrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, fAarray_const, lda, fBarray, ldb, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const double* alpha, const double* const A[], int lda, double* const B[], int ldb, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrsmBatched(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const double* alpha, double* const AP[], int lda, double* BP[], int ldb, int batchCount);
  // CHECK: blasStatus = hipblasDtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, dAarray_const, lda, dBarray, ldb, batchCount);
  blasStatus = hipblasDtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, dAarray_const, lda, dBarray, ldb, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuComplex* alpha, const cuComplex* const A[], int lda, cuComplex* const B[], int ldb, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrsmBatched(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const hipblasComplex* alpha, hipblasComplex* const AP[], int lda, hipblasComplex* BP[], int ldb, int batchCount);
  // CHECK: blasStatus = hipblasCtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, complexAarray_const, lda, complexBarray, ldb, batchCount);
  blasStatus = hipblasCtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, complexAarray_const, lda, complexBarray, ldb, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* const A[], int lda, cuDoubleComplex* const B[], int ldb, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrsmBatched(hipblasHandle_t handle, hipblasSideMode_t side, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int n, const hipblasDoubleComplex* alpha, hipblasDoubleComplex* const AP[], int lda, hipblasDoubleComplex* BP[], int ldb, int batchCount);
  // CHECK: blasStatus = hipblasZtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray, ldb, batchCount);
  blasStatus = hipblasZtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray, ldb, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgeqrfBatched(cublasHandle_t handle, int m, int n, float* const Aarray[], int lda, float* const TauArray[], int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgeqrfBatched(hipblasHandle_t handle, const int m, const int n, float* const A[], const int lda, float* const ipiv[], int* info, const int batchCount);
  // CHECK: blasStatus = hipblasSgeqrfBatched(blasHandle, m, n, fAarray, lda, fTauarray, &info, batchCount);
  blasStatus = hipblasSgeqrfBatched(blasHandle, m, n, fAarray, lda, fTauarray, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgeqrfBatched(cublasHandle_t handle, int m, int n, double* const Aarray[], int lda, double* const TauArray[], int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgeqrfBatched(hipblasHandle_t handle, const int m, const int n, double* const A[], const int lda, double* const ipiv[], int* info, const int batchCount);
  // CHECK: blasStatus = hipblasDgeqrfBatched(blasHandle, m, n, dAarray, lda, dTauarray, &info, batchCount);
  blasStatus = hipblasDgeqrfBatched(blasHandle, m, n, dAarray, lda, dTauarray, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgeqrfBatched(cublasHandle_t handle, int m, int n, cuComplex* const Aarray[], int lda, cuComplex* const TauArray[], int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgeqrfBatched(hipblasHandle_t handle, const int m, const int n, hipblasComplex* const A[], const int lda, hipblasComplex* const ipiv[], int* info, const int batchCount);
  // CHECK: blasStatus = hipblasCgeqrfBatched(blasHandle, m, n, complexAarray, lda, complexTauarray, &info, batchCount);
  blasStatus = hipblasCgeqrfBatched(blasHandle, m, n, complexAarray, lda, complexTauarray, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgeqrfBatched(cublasHandle_t handle, int m, int n, cuDoubleComplex* const Aarray[], int lda, cuDoubleComplex* const TauArray[], int* info, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgeqrfBatched(hipblasHandle_t handle, const int m, const int n, hipblasDoubleComplex* const A[], const int lda, hipblasDoubleComplex* const ipiv[], int* info, const int batchCount);
  // CHECK: blasStatus = hipblasZgeqrfBatched(blasHandle, m, n, dcomplexAarray, lda, dcomplexTauarray, &info, batchCount);
  blasStatus = hipblasZgeqrfBatched(blasHandle, m, n, dcomplexAarray, lda, dcomplexTauarray, &info, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const float* A, int lda, const float* x, int incx, float* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSdgmm(hipblasHandle_t handle, hipblasSideMode_t side, int m, int n, const float* AP, int lda, const float* x, int incx, float* CP, int ldc);
  // CHECK: blasStatus = hipblasSdgmm(blasHandle, blasSideMode, m, n, &fa, lda, &fx, incx, &fC, ldc);
  blasStatus = hipblasSdgmm(blasHandle, blasSideMode, m, n, &fa, lda, &fx, incx, &fC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const double* A, int lda, const double* x, int incx, double* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDdgmm(hipblasHandle_t handle, hipblasSideMode_t side, int m, int n, const double* AP, int lda, const double* x, int incx, double* CP, int ldc);
  // CHECK: blasStatus = hipblasDdgmm(blasHandle, blasSideMode, m, n, &da, lda, &dx, incx, &dC, ldc);
  blasStatus = hipblasDdgmm(blasHandle, blasSideMode, m, n, &da, lda, &dx, incx, &dC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCdgmm(hipblasHandle_t handle, hipblasSideMode_t side, int m, int n, const hipblasComplex* AP, int lda, const hipblasComplex* x, int incx, hipblasComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasCdgmm(blasHandle, blasSideMode, m, n, &complexa, lda, &complexx, incx, &complexC, ldc);
  blasStatus = hipblasCdgmm(blasHandle, blasSideMode, m, n, &complexa, lda, &complexx, incx, &complexC, ldc);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex* C, int ldc);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdgmm(hipblasHandle_t handle, hipblasSideMode_t side, int m, int n, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* x, int incx, hipblasDoubleComplex* CP, int ldc);
  // CHECK: blasStatus = hipblasZdgmm(blasHandle, blasSideMode, m, n, &dcomplexa, lda, &dcomplexx, incx, &dcomplexC, ldc);
  blasStatus = hipblasZdgmm(blasHandle, blasSideMode, m, n, &dcomplexa, lda, &dcomplexx, incx, &dcomplexC, ldc);

  int deviceInfo = 0;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgelsBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int nrhs, float* const Aarray[], int lda, float* const Carray[], int ldc, int* info, int* devInfoArray, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgelsBatched(hipblasHandle_t handle, hipblasOperation_t trans, const int m, const int n, const int nrhs, float* const A[], const int lda, float* const B[], const int ldb, int* info, int* deviceInfo, const int batchCount);
  // CHECK: blasStatus = hipblasSgelsBatched(blasHandle, blasOperation, m, n, nrhs, fAarray, lda, fCarray, ldc, &info, &deviceInfo, batchCount);
  blasStatus = hipblasSgelsBatched(blasHandle, blasOperation, m, n, nrhs, fAarray, lda, fCarray, ldc, &info, &deviceInfo, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgelsBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int nrhs, double* const Aarray[], int lda, double* const Carray[], int ldc, int* info, int* devInfoArray, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgelsBatched(hipblasHandle_t handle, hipblasOperation_t trans, const int m, const int n, const int nrhs, double* const A[], const int lda, double* const B[], const int ldb, int* info, int* deviceInfo, const int batchCount);
  // CHECK: blasStatus = hipblasDgelsBatched(blasHandle, blasOperation, m, n, nrhs, dAarray, lda, dCarray, ldc, &info, &deviceInfo, batchCount);
  blasStatus = hipblasDgelsBatched(blasHandle, blasOperation, m, n, nrhs, dAarray, lda, dCarray, ldc, &info, &deviceInfo, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgelsBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int nrhs, cuComplex* const Aarray[], int lda, cuComplex* const Carray[], int ldc, int* info, int* devInfoArray, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgelsBatched(hipblasHandle_t handle, hipblasOperation_t trans, const int m, const int n, const int nrhs, hipblasComplex* const A[], const int lda, hipblasComplex* const B[], const int ldb, int* info, int* deviceInfo, const int batchCount);
  // CHECK: blasStatus = hipblasCgelsBatched(blasHandle, blasOperation, m, n, nrhs, complexAarray, lda, complexCarray, ldc, &info, &deviceInfo, batchCount);
  blasStatus = hipblasCgelsBatched(blasHandle, blasOperation, m, n, nrhs, complexAarray, lda, complexCarray, ldc, &info, &deviceInfo, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgelsBatched(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int nrhs, cuDoubleComplex* const Aarray[], int lda, cuDoubleComplex* const Carray[], int ldc, int* info, int* devInfoArray, int batchSize);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgelsBatched(hipblasHandle_t handle, hipblasOperation_t trans, const int m, const int n, const int nrhs, hipblasDoubleComplex* const A[], const int lda, hipblasDoubleComplex* const B[], const int ldb, int* info, int* deviceInfo, const int batchCount);
  // CHECK: blasStatus = hipblasZgelsBatched(blasHandle, blasOperation, m, n, nrhs, dcomplexAarray, lda, dcomplexCarray, ldc, &info, &deviceInfo, batchCount);
  blasStatus = hipblasZgelsBatched(blasHandle, blasOperation, m, n, nrhs, dcomplexAarray, lda, dcomplexCarray, ldc, &info, &deviceInfo, batchCount);

  long long int strideA = 0;
  long long int strideB = 0;
  long long int strideC = 0;

#if CUDA_VERSION >= 8000
  // CHECK: hipblasDatatype_t DataType;
  // CHECK-NEXT: hipblasDatatype_t DataType_t;
  // CHECK-NEXT: hipblasDatatype_t blasDataType;
  // CHECK-NEXT: hipblasDatatype_t R_16F = HIPBLAS_R_16F;
  // CHECK-NEXT: hipblasDatatype_t C_16F = HIPBLAS_C_16F;
  // CHECK-NEXT: hipblasDatatype_t R_32F = HIPBLAS_R_32F;
  // CHECK-NEXT: hipblasDatatype_t C_32F = HIPBLAS_C_32F;
  // CHECK-NEXT: hipblasDatatype_t R_64F = HIPBLAS_R_64F;
  // CHECK-NEXT: hipblasDatatype_t C_64F = HIPBLAS_C_64F;
  // CHECK-NEXT: hipblasDatatype_t R_8I = HIPBLAS_R_8I;
  // CHECK-NEXT: hipblasDatatype_t C_8I = HIPBLAS_C_8I;
  // CHECK-NEXT: hipblasDatatype_t R_8U = HIPBLAS_R_8U;
  // CHECK-NEXT: hipblasDatatype_t C_8U = HIPBLAS_C_8U;
  // CHECK-NEXT: hipblasDatatype_t R_32I = HIPBLAS_R_32I;
  // CHECK-NEXT: hipblasDatatype_t C_32I = HIPBLAS_C_32I;
  // CHECK-NEXT: hipblasDatatype_t R_32U = HIPBLAS_R_32U;
  // CHECK-NEXT: hipblasDatatype_t C_32U = HIPBLAS_C_32U;
  hipDataType DataType;
  hipDataType DataType_t;
  hipDataType blasDataType;
  hipDataType R_16F = HIP_R_16F;
  hipDataType C_16F = HIP_C_16F;
  hipDataType R_32F = HIP_R_32F;
  hipDataType C_32F = HIP_C_32F;
  hipDataType R_64F = HIP_R_64F;
  hipDataType C_64F = HIP_C_64F;
  hipDataType R_8I = HIP_R_8I;
  hipDataType C_8I = HIP_C_8I;
  hipDataType R_8U = HIP_R_8U;
  hipDataType C_8U = HIP_C_8U;
  hipDataType R_32I = HIP_R_32I;
  hipDataType C_32I = HIP_C_32I;
  hipDataType R_32U = HIP_R_32U;
  hipDataType C_32U = HIP_C_32U;

  // CHECK: hipblasDatatype_t DataType_2, DataType_3;
  hipDataType DataType_2, DataType_3;

  // CHECK: hipblasGemmAlgo_t blasGemmAlgo;
  // CHECK-NEXT: hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;
  hipblasGemmAlgo_t blasGemmAlgo;
  hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasNrm2Ex(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, void* result, cudaDataType resultType, cudaDataType executionType);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasNrm2Ex(hipblasHandle_t handle, int n, const void* x, hipblasDatatype_t xType, int incx, void* result, hipblasDatatype_t resultType, hipblasDatatype_t executionType);
  // CHECK: blasStatus = hipblasNrm2Ex(blasHandle, n, image, DataType, incx, image_2, DataType_2, DataType_3);
  blasStatus = hipblasNrm2Ex(blasHandle, n, image, DataType, incx, image_2, DataType_2, DataType_3);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* A, int lda, long long int strideA, const float* B, int ldb, long long int strideB, const float* beta, float* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgemmStridedBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const float* alpha, const float* AP, int lda, long long strideA, const float* BP, int ldb, long long strideB, const float* beta, float* CP, int ldc, long long strideC, int batchCount);
  // CHECK: blasStatus = hipblasSgemmStridedBatched(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, strideA, &fB, ldb, strideB, &fb, &fC, ldc, strideC, batchCount);
  blasStatus = hipblasSgemmStridedBatched(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, strideA, &fB, ldb, strideB, &fb, &fC, ldc, strideC, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* A, int lda, long long int strideA, const double* B, int ldb, long long int strideB, const double* beta, double* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgemmStridedBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const double* alpha, const double* AP, int lda, long long strideA, const double* BP, int ldb, long long strideB, const double* beta, double* CP, int ldc, long long strideC, int batchCount);
  // CHECK: blasStatus = hipblasDgemmStridedBatched(blasHandle, transa, transb, m, n, k, &da, &dA, lda, strideA, &dB, ldb, strideB, &db, &dC, ldc, strideC, batchCount);
  blasStatus = hipblasDgemmStridedBatched(blasHandle, transa, transb, m, n, k, &da, &dA, lda, strideA, &dB, ldb, strideB, &db, &dC, ldc, strideC, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, long long int strideA, const cuComplex* B, int ldb, long long int strideB, const cuComplex* beta, cuComplex* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemmStridedBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, long long strideA, const hipblasComplex* BP, int ldb, long long strideB, const hipblasComplex* beta, hipblasComplex* CP, int ldc, long long strideC, int batchCount);
  // CHECK: blasStatus = hipblasCgemmStridedBatched(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, strideA, &complexB, ldb, strideB, &complexb, &complexC, ldc, strideC, batchCount);
  blasStatus = hipblasCgemmStridedBatched(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, strideA, &complexB, ldb, strideB, &complexb, &complexC, ldc, strideC, batchCount);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, long long int strideA, const cuDoubleComplex* B, int ldb, long long int strideB, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemmStridedBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, long long strideA, const hipblasDoubleComplex* BP, int ldb, long long strideB, const hipblasDoubleComplex* beta, hipblasDoubleComplex* CP, int ldc, long long strideC, int batchCount);
  // CHECK: blasStatus = hipblasZgemmStridedBatched(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, strideA, &dcomplexB, ldb, strideB, &dcomplexb, &dcomplexC, ldc, strideC, batchCount);
  blasStatus = hipblasZgemmStridedBatched(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, strideA, &dcomplexB, ldb, strideB, &dcomplexb, &dcomplexC, ldc, strideC, batchCount);

  // TODO: __half -> hipblasHalf
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* A, int lda, long long int strideA, const __half* B, int ldb, long long int strideB, const __half* beta, __half* C, int ldc, long long int strideC, int batchCount);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasHgemmStridedBatched(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const hipblasHalf* alpha, const hipblasHalf* AP, int lda, long long strideA, const hipblasHalf* BP, int ldb, long long strideB, const hipblasHalf* beta, hipblasHalf* CP, int ldc, long long strideC, int batchCount);

  void* aptr = nullptr;
  void* Aptr = nullptr;
  void* bptr = nullptr;
  void* Bptr = nullptr;
  void* cptr = nullptr;
  void* Cptr = nullptr;
  void* xptr = nullptr;
  void* yptr = nullptr;
  void* sptr = nullptr;

  // CHECK: hipblasDatatype_t Atype;
  // CHECK-NEXT: hipblasDatatype_t Btype;
  // CHECK-NEXT: hipblasDatatype_t Ctype;
  // CHECK-NEXT: hipblasDatatype_t Xtype;
  // CHECK-NEXT: hipblasDatatype_t Ytype;
  // CHECK-NEXT: hipblasDatatype_t CStype;
  // CHECK-NEXT: hipblasDatatype_t Executiontype;
  hipDataType Atype;
  hipDataType Btype;
  hipDataType Ctype;
  hipDataType Xtype;
  hipDataType Ytype;
  hipDataType CStype;
  hipDataType Executiontype;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScalEx(cublasHandle_t handle, int n, const void* alpha, cudaDataType alphaType, void* x, cudaDataType xType, int incx, cudaDataType executionType);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScalEx(hipblasHandle_t handle, int n, const void* alpha, hipblasDatatype_t alphaType, void* x, hipblasDatatype_t xType, int incx, hipblasDatatype_t executionType);
  // CHECK: blasStatus = hipblasScalEx(blasHandle, n, aptr, Atype, xptr, Xtype, incx, Executiontype);
  blasStatus = hipblasScalEx(blasHandle, n, aptr, Atype, xptr, Xtype, incx, Executiontype);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasAxpyEx(cublasHandle_t handle, int n, const void* alpha, cudaDataType alphaType, const void* x, cudaDataType xType, int incx, void* y, cudaDataType yType, int incy, cudaDataType executiontype);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasAxpyEx(hipblasHandle_t handle, int n, const void* alpha, hipblasDatatype_t alphaType, const void* x, hipblasDatatype_t xType, int incx, void* y, hipblasDatatype_t yType, int incy, hipblasDatatype_t executionType);
  // CHECK: blasStatus = hipblasAxpyEx(blasHandle, n, aptr, Atype, xptr, Xtype, incx, yptr, Ytype, incy, Executiontype);
  blasStatus = hipblasAxpyEx(blasHandle, n, aptr, Atype, xptr, Xtype, incx, yptr, Ytype, incy, Executiontype);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDotEx(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, const void* y, cudaDataType yType, int incy, void* result, cudaDataType resultType, cudaDataType executionType);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDotEx(hipblasHandle_t handle, int n, const void* x, hipblasDatatype_t xType, int incx, const void* y, hipblasDatatype_t yType, int incy, void* result, hipblasDatatype_t resultType, hipblasDatatype_t executionType);
  // CHECK: blasStatus = hipblasDotEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
  blasStatus = hipblasDotEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDotcEx(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, const void* y, cudaDataType yType, int incy, void* result, cudaDataType resultType, cudaDataType executionType);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDotcEx(hipblasHandle_t handle, int n, const void* x, hipblasDatatype_t xType, int incx, const void* y, hipblasDatatype_t yType, int incy, void* result, hipblasDatatype_t resultType, hipblasDatatype_t executionType);
  // CHECK: blasStatus = hipblasDotcEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
  blasStatus = hipblasDotcEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
#endif

#if CUDA_VERSION >= 8000 && CUDA_VERSION < 11000
  // CHECK: hipblasDatatype_t computeType;
  cudaDataType computeType;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, const void* B, cudaDataType Btype, int ldb, const void* beta, void* C, cudaDataType Ctype, int ldc, cudaDataType computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmEx(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A, hipblasDatatype_t aType, int lda, const void* B, hipblasDatatype_t bType, int ldb, const void* beta, void* C, hipblasDatatype_t cType, int ldc, hipblasDatatype_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, computeType, blasGemmAlgo);
  blasStatus = cublasGemmEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, computeType, blasGemmAlgo);
#endif

#if CUDA_VERSION >= 9000
  // CHECK: hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;
  hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;
#endif

#if CUDA_VERSION >= 9010 && CUDA_VERSION < 11000
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* const Aarray[], cudaDataType Atype, int lda, const void* const Barray[], cudaDataType Btype, int ldb, const void* beta, void* const Carray[], cudaDataType Ctype, int ldc, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmBatchedEx(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A[], hipblasDatatype_t aType, int lda, const void* B[], hipblasDatatype_t bType, int ldb, const void* beta, void* C[], hipblasDatatype_t cType, int ldc, int batchCount, hipblasDatatype_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmBatchedEx(blasHandle, transa, transb, m, n, k, aptr, voidAarray_const, Atype, lda, voidBarray_const, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, computeType, blasGemmAlgo);
  blasStatus = cublasGemmBatchedEx(blasHandle, transa, transb, m, n, k, aptr, voidAarray_const, Atype, lda, voidBarray_const, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, computeType, blasGemmAlgo);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmStridedBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, long long int strideA, const void* B, cudaDataType Btype, int ldb, long long int strideB, const void* beta, void* C, cudaDataType Ctype, int ldc, long long int strideC, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmStridedBatchedEx(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A, hipblasDatatype_t aType, int lda, hipblasStride strideA, const void* B, hipblasDatatype_t bType, int ldb, hipblasStride strideB, const void* beta, void* C, hipblasDatatype_t cType, int ldc, hipblasStride strideC, int batchCount, hipblasDatatype_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmStridedBatchedEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, computeType, blasGemmAlgo);
  blasStatus = cublasGemmStridedBatchedEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, computeType, blasGemmAlgo);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;
  hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;

  // CHECK: hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;
  hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasRotEx(cublasHandle_t handle, int n, void* x, cudaDataType xType, int incx, void* y, cudaDataType yType, int incy, const void* c, const void* s, cudaDataType csType, cudaDataType executiontype);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasRotEx(hipblasHandle_t handle, int n, void* x, hipblasDatatype_t xType, int incx, void* y, hipblasDatatype_t yType, int incy, const void* c, const void* s, hipblasDatatype_t csType, hipblasDatatype_t executionType);
  // CHECK: blasStatus = hipblasRotEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, cptr, sptr, CStype, Executiontype);
  blasStatus = hipblasRotEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, cptr, sptr, CStype, Executiontype);
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipblasDatatype_t R_16BF = HIPBLAS_R_16B;
  // CHECK-NEXT: hipblasDatatype_t C_16BF = HIPBLAS_C_16B;
  hipDataType R_16BF = HIP_R_16BF;
  hipDataType C_16BF = HIP_C_16BF;

  // NOTE: WORKAROUND: cublasComputeType_t is not actually supported by hipBLAS
  // TODO: Fix it after fixing https://github.com/ROCmSoftwarePlatform/hipBLAS/issues/529
  // CHECK: hipblasDatatype_t blasComputeType;
  hipblasComputeType_t blasComputeType;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, const void* B, cudaDataType Btype, int ldb, const void* beta, void* C, cudaDataType Ctype, int ldc, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmEx(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A, hipblasDatatype_t aType, int lda, const void* B, hipblasDatatype_t bType, int ldb, const void* beta, void* C, hipblasDatatype_t cType, int ldc, hipblasDatatype_t computeType, ipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, blasComputeType, blasGemmAlgo);
  blasStatus = hipblasGemmEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, blasComputeType, blasGemmAlgo);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* const Aarray[], cudaDataType Atype, int lda, const void* const Barray[], cudaDataType Btype, int ldb, const void* beta, void* const Carray[], cudaDataType Ctype, int ldc, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmBatchedEx(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A[], hipblasDatatype_t aType, int lda, const void* B[], hipblasDatatype_t bType, int ldb, const void* beta, void* C[], hipblasDatatype_t cType, int ldc, int batchCount, hipblasDatatype_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmBatchedEx(blasHandle, transa, transb, m, n, k, aptr, voidAarray, Atype, lda, voidBarray, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, blasComputeType, blasGemmAlgo);
  blasStatus = hipblasGemmBatchedEx(blasHandle, transa, transb, m, n, k, aptr, voidAarray, Atype, lda, voidBarray, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, blasComputeType, blasGemmAlgo);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmStridedBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, long long int strideA, const void* B, cudaDataType Btype, int ldb, long long int strideB, const void* beta, void* C, cudaDataType Ctype, int ldc, long long int strideC, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGemmStridedBatchedEx(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB, int m, int n, int k, const void* alpha, const void* A, hipblasDatatype_t aType, int lda, hipblasStride strideA, const void* B, hipblasDatatype_t bType, int ldb, hipblasStride strideB, const void* beta, void* C, hipblasDatatype_t cType, int ldc, hipblasStride strideC, int batchCount, hipblasDatatype_t computeType, hipblasGemmAlgo_t algo);
  // CHECK: blasStatus = hipblasGemmStridedBatchedEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, blasComputeType, blasGemmAlgo);
  blasStatus = hipblasGemmStridedBatchedEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, blasComputeType, blasGemmAlgo);
#endif

  return 0;
}
