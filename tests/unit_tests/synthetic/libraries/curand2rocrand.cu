// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --amap --default-preprocessor --experimental --roc %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocrand/rocrand.h"
#include "hiprand/hiprand.h"
// CHECK-NOT: #include "rocrand/rocrand.h"

int main() {
  printf("21.1. cuRAND API to rocRAND API synthetic test\n");

  unsigned int *outputPtr = nullptr;
  float *outputPtrFloat = nullptr;
  double *outputPtrDouble = nullptr;
  unsigned long long *outputPtrUll = nullptr;
  unsigned long long offset = 0;
  size_t num = 0;
  float mean = 0.f;
  double dmean = 0.f;
  float stddev = 0.f;
  double dstddev = 0.f;
  double dlambda = 0.f;

  // CHECK: hipStream_t stream;
  hipStream_t stream;

  // CHECK: rocrand_status randStatus;
  // CHECK-NEXT: rocrand_status status;
  // CHECK-NEXT: rocrand_status STATUS_SUCCESS = ROCRAND_STATUS_SUCCESS;
  // CHECK-NEXT: rocrand_status STATUS_VERSION_MISMATCH = ROCRAND_STATUS_VERSION_MISMATCH;
  // CHECK-NEXT: rocrand_status STATUS_NOT_INITIALIZED = ROCRAND_STATUS_NOT_CREATED;
  // CHECK-NEXT: rocrand_status STATUS_ALLOCATION_FAILED = ROCRAND_STATUS_ALLOCATION_FAILED;
  // CHECK-NEXT: rocrand_status STATUS_TYPE_ERROR = ROCRAND_STATUS_TYPE_ERROR;
  // CHECK-NEXT: rocrand_status STATUS_OUT_OF_RANGE = ROCRAND_STATUS_OUT_OF_RANGE;
  // CHECK-NEXT: rocrand_status STATUS_LENGTH_NOT_MULTIPLE = ROCRAND_STATUS_LENGTH_NOT_MULTIPLE;
  // CHECK-NEXT: rocrand_status STATUS_DOUBLE_PRECISION_REQUIRED = ROCRAND_STATUS_DOUBLE_PRECISION_REQUIRED;
  // CHECK-NEXT: rocrand_status STATUS_LAUNCH_FAILURE = ROCRAND_STATUS_LAUNCH_FAILURE;
  // CHECK-NEXT: rocrand_status STATUS_INTERNAL_ERROR = ROCRAND_STATUS_INTERNAL_ERROR;
  hiprandStatus randStatus;
  hiprandStatus_t status;
  hiprandStatus_t STATUS_SUCCESS = HIPRAND_STATUS_SUCCESS;
  hiprandStatus_t STATUS_VERSION_MISMATCH = HIPRAND_STATUS_VERSION_MISMATCH;
  hiprandStatus_t STATUS_NOT_INITIALIZED = HIPRAND_STATUS_NOT_INITIALIZED;
  hiprandStatus_t STATUS_ALLOCATION_FAILED = HIPRAND_STATUS_ALLOCATION_FAILED;
  hiprandStatus_t STATUS_TYPE_ERROR = HIPRAND_STATUS_TYPE_ERROR;
  hiprandStatus_t STATUS_OUT_OF_RANGE = HIPRAND_STATUS_OUT_OF_RANGE;
  hiprandStatus_t STATUS_LENGTH_NOT_MULTIPLE = HIPRAND_STATUS_LENGTH_NOT_MULTIPLE;
  hiprandStatus_t STATUS_DOUBLE_PRECISION_REQUIRED = HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED;
  hiprandStatus_t STATUS_LAUNCH_FAILURE = HIPRAND_STATUS_LAUNCH_FAILURE;
  hiprandStatus_t STATUS_INTERNAL_ERROR = HIPRAND_STATUS_INTERNAL_ERROR;

  // CHECK: rocrand_rng_type randRngType;
  // CHECK-NEXT: rocrand_rng_type randRngType_t;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_DEFAULT = ROCRAND_RNG_PSEUDO_DEFAULT;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_XORWOW = ROCRAND_RNG_PSEUDO_XORWOW;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_MRG32K3A = ROCRAND_RNG_PSEUDO_MRG32K3A;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_MTGP32 = ROCRAND_RNG_PSEUDO_MTGP32;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_MT19937 = ROCRAND_RNG_PSEUDO_MT19937;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_PHILOX4_32_10 = ROCRAND_RNG_PSEUDO_PHILOX4_32_10;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_DEFAULT = ROCRAND_RNG_QUASI_DEFAULT;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_SOBOL32 = ROCRAND_RNG_QUASI_SOBOL32;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_SCRAMBLED_SOBOL32 = ROCRAND_RNG_QUASI_SCRAMBLED_SOBOL32;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_SOBOL64 = ROCRAND_RNG_QUASI_SOBOL64;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_SCRAMBLED_SOBOL64 = ROCRAND_RNG_QUASI_SCRAMBLED_SOBOL64;
  hiprandRngType_t randRngType;
  hiprandRngType_t randRngType_t;
  hiprandRngType_t RNG_PSEUDO_DEFAULT = HIPRAND_RNG_PSEUDO_DEFAULT;
  hiprandRngType_t RNG_PSEUDO_XORWOW = HIPRAND_RNG_PSEUDO_XORWOW;
  hiprandRngType_t RNG_PSEUDO_MRG32K3A = HIPRAND_RNG_PSEUDO_MRG32K3A;
  hiprandRngType_t RNG_PSEUDO_MTGP32 = HIPRAND_RNG_PSEUDO_MTGP32;
  hiprandRngType_t RNG_PSEUDO_MT19937 = HIPRAND_RNG_PSEUDO_MT19937;
  hiprandRngType_t RNG_PSEUDO_PHILOX4_32_10 = HIPRAND_RNG_PSEUDO_PHILOX4_32_10;
  hiprandRngType_t RNG_QUASI_DEFAULT = HIPRAND_RNG_QUASI_DEFAULT;
  hiprandRngType_t RNG_QUASI_SOBOL32 = HIPRAND_RNG_QUASI_SOBOL32;
  hiprandRngType_t RNG_QUASI_SCRAMBLED_SOBOL32 = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32;
  hiprandRngType_t RNG_QUASI_SOBOL64 = HIPRAND_RNG_QUASI_SOBOL64;
  hiprandRngType_t RNG_QUASI_SCRAMBLED_SOBOL64 = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64;

  // CHECK: rocrand_ordering randOrdering;
  // CHECK-NEXT: rocrand_ordering RAND_ORDERING_PSEUDO_BEST = ROCRAND_ORDERING_PSEUDO_BEST;
  // CHECK-NEXT: rocrand_ordering RAND_ORDERING_PSEUDO_DEFAULT = ROCRAND_ORDERING_PSEUDO_DEFAULT;
  // CHECK-NEXT: rocrand_ordering RAND_ORDERING_PSEUDO_SEEDED = ROCRAND_ORDERING_PSEUDO_SEEDED;
  // CHECK-NEXT: rocrand_ordering RAND_ORDERING_QUASI_DEFAULT = ROCRAND_ORDERING_QUASI_DEFAULT;
  hiprandOrdering randOrdering;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_BEST = HIPRAND_ORDERING_PSEUDO_BEST;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_DEFAULT = HIPRAND_ORDERING_PSEUDO_DEFAULT;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_SEEDED = HIPRAND_ORDERING_PSEUDO_SEEDED;
  hiprandOrdering_t RAND_ORDERING_QUASI_DEFAULT = HIPRAND_ORDERING_QUASI_DEFAULT;

  // CHECK: rocrand_direction_vector_set directionVectorSet;
  // CHECK-NEXT: rocrand_direction_vector_set directionVectorSet_t;
  // CHECK-NEXT: rocrand_direction_vector_set DIRECTION_VECTORS_32_JOEKUO6 = ROCRAND_DIRECTION_VECTORS_32_JOEKUO6;
  // CHECK-NEXT: rocrand_direction_vector_set SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6 = ROCRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6;
  // CHECK-NEXT: rocrand_direction_vector_set DIRECTION_VECTORS_64_JOEKUO6 = ROCRAND_DIRECTION_VECTORS_64_JOEKUO6;
  // CHECK-NEXT: rocrand_direction_vector_set SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6 = ROCRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6;
  hiprandDirectionVectorSet_t directionVectorSet;
  hiprandDirectionVectorSet_t directionVectorSet_t;
  hiprandDirectionVectorSet_t DIRECTION_VECTORS_32_JOEKUO6 = HIPRAND_DIRECTION_VECTORS_32_JOEKUO6;
  hiprandDirectionVectorSet_t SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6 = HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6;
  hiprandDirectionVectorSet_t DIRECTION_VECTORS_64_JOEKUO6 = HIPRAND_DIRECTION_VECTORS_64_JOEKUO6;
  hiprandDirectionVectorSet_t SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6 = HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6;

  // CHECK: rocrand_generator_base_type *randGenerator_st = nullptr;
  // CHECK-NEXT: rocrand_generator randGenerator;
  hiprandGenerator_st *randGenerator_st = nullptr;
  hiprandGenerator_t randGenerator;

  // CUDA: curandStatus_t CURANDAPI curandCreateGenerator(curandGenerator_t *generator, curandRngType_t rng_type);
  // ROC: rocrand_status ROCRANDAPI rocrand_create_generator(rocrand_generator * generator, rocrand_rng_type rng_type);
  // CHECK: status = rocrand_create_generator(&randGenerator, randRngType_t);
  status = hiprandCreateGenerator(&randGenerator, randRngType_t);

  // CUDA: curandStatus_t CURANDAPI curandDestroyGenerator(curandGenerator_t generator);
  // ROC: rocrand_status ROCRANDAPI rocrand_destroy_generator(rocrand_generator generator);
  // CHECK: status = rocrand_destroy_generator(randGenerator);
  status = hiprandDestroyGenerator(randGenerator);

  // CUDA: curandStatus_t CURANDAPI curandCreateGeneratorHost(curandGenerator_t *generator, curandRngType_t rng_type);
  // ROC: rocrand_status ROCRANDAPI rocrand_create_generator_host_blocking(rocrand_generator* generator, rocrand_rng_type rng_type);
  // CHECK: status = rocrand_create_generator_host_blocking(&randGenerator, randRngType_t);
  status = hiprandCreateGeneratorHost(&randGenerator, randRngType_t);

  // CUDA: curandStatus_t CURANDAPI curandGenerate(curandGenerator_t generator, unsigned int *outputPtr, size_t num);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate(rocrand_generator generator, unsigned int * output_data, size_t n);
  // CHECK: status = rocrand_generate(randGenerator, outputPtr, num);
  status = hiprandGenerate(randGenerator, outputPtr, num);

  // CUDA: curandStatus_t CURANDAPI curandGenerateLogNormal(curandGenerator_t generator, float *outputPtr, size_t n, float mean, float stddev);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_log_normal(rocrand_generator generator, float * output_data, size_t n, float mean, float stddev);
  // CHECK: status = rocrand_generate_log_normal(randGenerator, outputPtrFloat, num, mean, stddev);
  status = hiprandGenerateLogNormal(randGenerator, outputPtrFloat, num, mean, stddev);

  // CUDA: curandStatus_t CURANDAPI curandGenerateLongLong(curandGenerator_t generator, unsigned long long *outputPtr, size_t num);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_long_long(rocrand_generator generator, unsigned long long int* output_data, size_t n);
  // CHECK: status = rocrand_generate_long_long(randGenerator, outputPtrUll, num);
  status = hiprandGenerateLongLong(randGenerator, outputPtrUll, num);

  // CUDA: curandStatus_t CURANDAPI curandGenerateNormal(curandGenerator_t generator, float *outputPtr, size_t n, float mean, float stddev);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_normal(rocrand_generator generator, float * output_data, size_t n, float mean, float stddev);
  // CHECK: status = rocrand_generate_normal(randGenerator, outputPtrFloat, num, mean, stddev);
  status = hiprandGenerateNormal(randGenerator, outputPtrFloat, num, mean, stddev);

  // CUDA: curandStatus_t CURANDAPI curandGenerateNormalDouble(curandGenerator_t generator, double *outputPtr, size_t n, double mean, double stddev);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_normal_double(rocrand_generator generator, double * output_data, size_t n, double mean, double stddev);
  // CHECK: status = rocrand_generate_normal_double(randGenerator, outputPtrDouble, num, dmean, dstddev);
  status = hiprandGenerateNormalDouble(randGenerator, outputPtrDouble, num, dmean, dstddev);

  // CUDA: curandStatus_t CURANDAPI curandGenerateUniform(curandGenerator_t generator, float *outputPtr, size_t num);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_uniform(rocrand_generator generator, float * output_data, size_t n);
  // CHECK: status = rocrand_generate_uniform(randGenerator, outputPtrFloat, num);
  status = hiprandGenerateUniform(randGenerator, outputPtrFloat, num);

  // CUDA: curandStatus_t CURANDAPI curandGenerateUniformDouble(curandGenerator_t generator, double *outputPtr, size_t num);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_uniform_double(rocrand_generator generator, double * output_data, size_t n);
  // CHECK: status = rocrand_generate_uniform_double(randGenerator, outputPtrDouble, num);
  status = hiprandGenerateUniformDouble(randGenerator, outputPtrDouble, num);

  // CUDA: curandStatus_t CURANDAPI curandGenerateLogNormalDouble(curandGenerator_t generator, double *outputPtr, size_t n, double mean, double stddev);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_log_normal_double(rocrand_generator generator, double * output_data, size_t n, double mean, double stddev);
  // CHECK: status = rocrand_generate_log_normal_double(randGenerator, outputPtrDouble, num, dmean, dstddev);
  status = hiprandGenerateLogNormalDouble(randGenerator, outputPtrDouble, num, dmean, dstddev);

  // CUDA: curandStatus_t CURANDAPI curandGeneratePoisson(curandGenerator_t generator, unsigned int *outputPtr, size_t n, double lambda);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_poisson(rocrand_generator generator, unsigned int * output_data, size_t n, double lambda);
  // CHECK: status = rocrand_generate_poisson(randGenerator, outputPtr, num, dlambda);
  status = hiprandGeneratePoisson(randGenerator, outputPtr, num, dlambda);

  // CUDA: curandStatus_t CURANDAPI curandGenerateSeeds(curandGenerator_t generator);
  // ROC: rocrand_status ROCRANDAPI rocrand_initialize_generator(rocrand_generator generator);
  // CHECK: status = rocrand_initialize_generator(randGenerator);
  status = hiprandGenerateSeeds(randGenerator);

  // CUDA: curandStatus_t CURANDAPI curandSetGeneratorOffset(curandGenerator_t generator, unsigned long long offset);
  // ROC: rocrand_status ROCRANDAPI rocrand_set_offset(rocrand_generator generator, unsigned long long offset);
  // CHECK: status = rocrand_set_offset(randGenerator, offset);
  status = hiprandSetGeneratorOffset(randGenerator, offset);

  // CUDA: curandStatus_t CURANDAPI curandSetPseudoRandomGeneratorSeed(curandGenerator_t generator, unsigned long long seed);
  // ROC: rocrand_status ROCRANDAPI rocrand_set_seed(rocrand_generator generator, unsigned long long seed);
  // CHECK: status = rocrand_set_seed(randGenerator, offset);
  status = hiprandSetPseudoRandomGeneratorSeed(randGenerator, offset);

  // CUDA: curandStatus_t CURANDAPI curandSetStream(curandGenerator_t generator, cudaStream_t stream);
  // ROC: rocrand_status ROCRANDAPI rocrand_set_stream(rocrand_generator generator, hipStream_t stream);
  // CHECK: status = rocrand_set_stream(randGenerator, stream);
  status = hiprandSetStream(randGenerator, stream);

#if CUDA_VERSION >= 11000 && CURAND_VERSION >= 10200
  // CHECK: rocrand_ordering RAND_ORDERING_PSEUDO_LEGACY = ROCRAND_ORDERING_PSEUDO_LEGACY;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_LEGACY = HIPRAND_ORDERING_PSEUDO_LEGACY;
#endif

#if CUDA_VERSION >= 11050 && CURAND_VERSION >= 10207
  // CHECK: rocrand_ordering RAND_ORDERING_PSEUDO_DYNAMIC = ROCRAND_ORDERING_PSEUDO_DYNAMIC;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_DYNAMIC = HIPRAND_ORDERING_PSEUDO_DYNAMIC;
#endif

  return 0;
}
