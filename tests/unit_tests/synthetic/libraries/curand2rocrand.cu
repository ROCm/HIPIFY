// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --amap --default-preprocessor --experimental --roc %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocrand/rocrand.h"
#include "hiprand/hiprand.h"
// CHECK-NOT: #include "rocrand/rocrand.h"

int main() {
  printf("21.1. cuRAND API to rocRAND API synthetic test\n");

  unsigned int *outputPtr = nullptr;
  size_t num = 0;

  // CHECK: rocrand_status randStatus;
  // CHECK-NEXT: rocrand_status status;
  // CHECK-NEXT: rocrand_status STATUS_SUCCESS = ROCRAND_STATUS_SUCCESS;
  // CHECK-NEXT: rocrand_status STATUS_VERSION_MISMATCH = ROCRAND_STATUS_VERSION_MISMATCH;
  // CHECK-NEXT: rocrand_status STATUS_NOT_INITIALIZED = ROCRAND_STATUS_NOT_CREATED;
  // CHECK-NEXT: rocrand_status STATUS_ALLOCATION_FAILED = ROCRAND_STATUS_ALLOCATION_FAILED;
  // CHECK-NEXT: rocrand_status STATUS_TYPE_ERROR = ROCRAND_STATUS_TYPE_ERROR;
  // CHECK-NEXT: rocrand_status STATUS_OUT_OF_RANGE = ROCRAND_STATUS_OUT_OF_RANGE;
  // CHECK-NEXT: rocrand_status STATUS_LENGTH_NOT_MULTIPLE = ROCRAND_STATUS_LENGTH_NOT_MULTIPLE;
  // CHECK-NEXT: rocrand_status STATUS_DOUBLE_PRECISION_REQUIRED = ROCRAND_STATUS_DOUBLE_PRECISION_REQUIRED;
  // CHECK-NEXT: rocrand_status STATUS_LAUNCH_FAILURE = ROCRAND_STATUS_LAUNCH_FAILURE;
  // CHECK-NEXT: rocrand_status STATUS_INTERNAL_ERROR = ROCRAND_STATUS_INTERNAL_ERROR;
  hiprandStatus randStatus;
  hiprandStatus_t status;
  hiprandStatus_t STATUS_SUCCESS = HIPRAND_STATUS_SUCCESS;
  hiprandStatus_t STATUS_VERSION_MISMATCH = HIPRAND_STATUS_VERSION_MISMATCH;
  hiprandStatus_t STATUS_NOT_INITIALIZED = HIPRAND_STATUS_NOT_INITIALIZED;
  hiprandStatus_t STATUS_ALLOCATION_FAILED = HIPRAND_STATUS_ALLOCATION_FAILED;
  hiprandStatus_t STATUS_TYPE_ERROR = HIPRAND_STATUS_TYPE_ERROR;
  hiprandStatus_t STATUS_OUT_OF_RANGE = HIPRAND_STATUS_OUT_OF_RANGE;
  hiprandStatus_t STATUS_LENGTH_NOT_MULTIPLE = HIPRAND_STATUS_LENGTH_NOT_MULTIPLE;
  hiprandStatus_t STATUS_DOUBLE_PRECISION_REQUIRED = HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED;
  hiprandStatus_t STATUS_LAUNCH_FAILURE = HIPRAND_STATUS_LAUNCH_FAILURE;
  hiprandStatus_t STATUS_INTERNAL_ERROR = HIPRAND_STATUS_INTERNAL_ERROR;


  // CHECK: rocrand_rng_type randRngType;
  // CHECK-NEXT: rocrand_rng_type randRngType_t;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_DEFAULT = ROCRAND_RNG_PSEUDO_DEFAULT;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_XORWOW = ROCRAND_RNG_PSEUDO_XORWOW;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_MRG32K3A = ROCRAND_RNG_PSEUDO_MRG32K3A;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_MTGP32 = ROCRAND_RNG_PSEUDO_MTGP32;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_MT19937 = ROCRAND_RNG_PSEUDO_MT19937;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_PHILOX4_32_10 = ROCRAND_RNG_PSEUDO_PHILOX4_32_10;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_DEFAULT = ROCRAND_RNG_QUASI_DEFAULT;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_SOBOL32 = ROCRAND_RNG_QUASI_SOBOL32;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_SCRAMBLED_SOBOL32 = ROCRAND_RNG_QUASI_SCRAMBLED_SOBOL32;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_SOBOL64 = ROCRAND_RNG_QUASI_SOBOL64;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_SCRAMBLED_SOBOL64 = ROCRAND_RNG_QUASI_SCRAMBLED_SOBOL64;
  hiprandRngType_t randRngType;
  hiprandRngType_t randRngType_t;
  hiprandRngType_t RNG_PSEUDO_DEFAULT = HIPRAND_RNG_PSEUDO_DEFAULT;
  hiprandRngType_t RNG_PSEUDO_XORWOW = HIPRAND_RNG_PSEUDO_XORWOW;
  hiprandRngType_t RNG_PSEUDO_MRG32K3A = HIPRAND_RNG_PSEUDO_MRG32K3A;
  hiprandRngType_t RNG_PSEUDO_MTGP32 = HIPRAND_RNG_PSEUDO_MTGP32;
  hiprandRngType_t RNG_PSEUDO_MT19937 = HIPRAND_RNG_PSEUDO_MT19937;
  hiprandRngType_t RNG_PSEUDO_PHILOX4_32_10 = HIPRAND_RNG_PSEUDO_PHILOX4_32_10;
  hiprandRngType_t RNG_QUASI_DEFAULT = HIPRAND_RNG_QUASI_DEFAULT;
  hiprandRngType_t RNG_QUASI_SOBOL32 = HIPRAND_RNG_QUASI_SOBOL32;
  hiprandRngType_t RNG_QUASI_SCRAMBLED_SOBOL32 = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32;
  hiprandRngType_t RNG_QUASI_SOBOL64 = HIPRAND_RNG_QUASI_SOBOL64;
  hiprandRngType_t RNG_QUASI_SCRAMBLED_SOBOL64 = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64;

  // CHECK: rocrand_ordering randOrdering;
  // CHECK-NEXT: rocrand_ordering RAND_ORDERING_PSEUDO_BEST = ROCRAND_ORDERING_PSEUDO_BEST;
  // CHECK-NEXT: rocrand_ordering RAND_ORDERING_PSEUDO_DEFAULT = ROCRAND_ORDERING_PSEUDO_DEFAULT;
  // CHECK-NEXT: rocrand_ordering RAND_ORDERING_PSEUDO_SEEDED = ROCRAND_ORDERING_PSEUDO_SEEDED;
  // CHECK-NEXT: rocrand_ordering RAND_ORDERING_QUASI_DEFAULT = ROCRAND_ORDERING_QUASI_DEFAULT;
  hiprandOrdering randOrdering;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_BEST = HIPRAND_ORDERING_PSEUDO_BEST;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_DEFAULT = HIPRAND_ORDERING_PSEUDO_DEFAULT;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_SEEDED = HIPRAND_ORDERING_PSEUDO_SEEDED;
  hiprandOrdering_t RAND_ORDERING_QUASI_DEFAULT = HIPRAND_ORDERING_QUASI_DEFAULT;

  // CHECK: rocrand_direction_vector_set directionVectorSet;
  // CHECK-NEXT: rocrand_direction_vector_set directionVectorSet_t;
  // CHECK-NEXT: rocrand_direction_vector_set DIRECTION_VECTORS_32_JOEKUO6 = ROCRAND_DIRECTION_VECTORS_32_JOEKUO6;
  // CHECK-NEXT: rocrand_direction_vector_set SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6 = ROCRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6;
  // CHECK-NEXT: rocrand_direction_vector_set DIRECTION_VECTORS_64_JOEKUO6 = ROCRAND_DIRECTION_VECTORS_64_JOEKUO6;
  // CHECK-NEXT: rocrand_direction_vector_set SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6 = ROCRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6;
  hiprandDirectionVectorSet_t directionVectorSet;
  hiprandDirectionVectorSet_t directionVectorSet_t;
  hiprandDirectionVectorSet_t DIRECTION_VECTORS_32_JOEKUO6 = HIPRAND_DIRECTION_VECTORS_32_JOEKUO6;
  hiprandDirectionVectorSet_t SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6 = HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6;
  hiprandDirectionVectorSet_t DIRECTION_VECTORS_64_JOEKUO6 = HIPRAND_DIRECTION_VECTORS_64_JOEKUO6;
  hiprandDirectionVectorSet_t SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6 = HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6;

  // CHECK: rocrand_generator_base_type *randGenerator_st = nullptr;
  // CHECK-NEXT: rocrand_generator randGenerator;
  hiprandGenerator_st *randGenerator_st = nullptr;
  hiprandGenerator_t randGenerator;

  // CUDA: curandStatus_t CURANDAPI curandCreateGenerator(curandGenerator_t *generator, curandRngType_t rng_type);
  // ROC: rocrand_status ROCRANDAPI rocrand_create_generator(rocrand_generator * generator, rocrand_rng_type rng_type);
  // CHECK: status = rocrand_create_generator(&randGenerator, randRngType_t);
  status = hiprandCreateGenerator(&randGenerator, randRngType_t);

  // CUDA: curandStatus_t CURANDAPI curandDestroyGenerator(curandGenerator_t generator);
  // ROC: rocrand_status ROCRANDAPI rocrand_destroy_generator(rocrand_generator generator);
  // CHECK: status = rocrand_destroy_generator(randGenerator);
  status = hiprandDestroyGenerator(randGenerator);

  // CUDA: curandStatus_t CURANDAPI curandCreateGeneratorHost(curandGenerator_t *generator, curandRngType_t rng_type);
  // ROC: rocrand_status ROCRANDAPI rocrand_create_generator_host_blocking(rocrand_generator* generator, rocrand_rng_type rng_type);
  // CHECK: status = rocrand_create_generator_host_blocking(&randGenerator, randRngType_t);
  status = hiprandCreateGeneratorHost(&randGenerator, randRngType_t);

  // CUDA: curandStatus_t CURANDAPI curandGenerate(curandGenerator_t generator, unsigned int *outputPtr, size_t num);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate(rocrand_generator generator, unsigned int * output_data, size_t n);
  // CHECK: status = rocrand_generate(randGenerator, outputPtr, num);
  status = hiprandGenerate(randGenerator, outputPtr, num);

#if CUDA_VERSION >= 11000 && CURAND_VERSION >= 10200
  // CHECK: rocrand_ordering RAND_ORDERING_PSEUDO_LEGACY = ROCRAND_ORDERING_PSEUDO_LEGACY;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_LEGACY = HIPRAND_ORDERING_PSEUDO_LEGACY;
#endif

#if CUDA_VERSION >= 11050 && CURAND_VERSION >= 10207
  // CHECK: rocrand_ordering RAND_ORDERING_PSEUDO_DYNAMIC = ROCRAND_ORDERING_PSEUDO_DYNAMIC;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_DYNAMIC = HIPRAND_ORDERING_PSEUDO_DYNAMIC;
#endif

  return 0;
}
