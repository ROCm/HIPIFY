// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18.1. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: rocsparse_status status_t;
  hipsparseStatus_t status_t;

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;

  // CHECK: rocsparse_operation opA, opB, opX;
  hipsparseOperation_t opA, opB, opX;

  // CHECK: rocsparse_solve_policy solvePolicy_t;
  hipsparseSolvePolicy_t solvePolicy_t;

  int m = 0;
  int n = 0;
  int k = 0;
  int innz = 0;
  int nnza = 0;
  int nnzb = 0;
  int nnzc = 0;
  int nnzd = 0;
  int csrRowPtrA = 0;
  int csrRowPtrB = 0;
  int csrRowPtrC = 0;
  int csrRowPtrD = 0;
  int csrColIndA = 0;
  int csrColIndB = 0;
  int csrColIndC = 0;
  int csrColIndD = 0;
  int bufferSizeInBytes = 0;
  size_t bufferSize = 0;
  double dA = 0.f;
  double dB = 0.f;
  double dAlpha = 0.f;
  double dF = 0.f;
  double dX = 0.f;
  double dcsrSortedValA = 0.f;
  double dcsrSortedValB = 0.f;
  double dcsrSortedValC = 0.f;
  double dcsrSortedValD = 0.f;
  float fAlpha = 0.f;
  float fA = 0.f;
  float fB = 0.f;
  float fF = 0.f;
  float fX = 0.f;
  float csrSortedValA = 0.f;
  float csrSortedValB = 0.f;
  float csrSortedValC = 0.f;
  float csrSortedValD = 0.f;
  void *pBuffer = nullptr;
  void *tempBuffer = nullptr;
  void *alpha = nullptr;
  void *beta = nullptr;
  void* result = nullptr;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuDoubleComplex -> rocsparse_double_complex under a new option --sparse
  // CHECK: rocblas_double_complex dcomplex, dcomplexA, dcomplexAlpha, dcomplexB, dcomplexBeta, dcomplexC, dcomplexF, dcomplexX, dcomplexY, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dComplexcsrSortedValD, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;
  hipDoubleComplex dcomplex, dcomplexA, dcomplexAlpha, dcomplexB, dcomplexBeta, dcomplexC, dcomplexF, dcomplexX, dcomplexY, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dComplexcsrSortedValD, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuComplex -> rocsparse_float_complex under a new option --sparse
  // CHECK: rocblas_float_complex complex, complexA, complexAlpha, complexB, complexBeta, complexC, complexF, complexX, complexY, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complexcsrSortedValD, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;
  hipComplex complex, complexA, complexAlpha, complexB, complexBeta, complexC, complexF, complexX, complexY, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complexcsrSortedValD, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;

#if CUDA_VERSION >= 8000
  // TODO: [#899] There should be rocsparse_datatype instead of hipDataType
  hipDataType dataType_t;
  hipDataType dataType;
#endif

#if (CUDA_VERSION >= 10010 && CUSPARSE_VERSION >= 10200 && CUDA_VERSION < 11000 && !defined(_WIN32)) || CUDA_VERSION >= 11000
  // CHECK: rocsparse_spmat_descr spMatDescr_t, spmatA, spmatB, spmatC;
  hipsparseSpMatDescr_t spMatDescr_t, spmatA, spmatB, spmatC;

  // CHECK: rocsparse_dnmat_descr dnMatDescr_t, dnmatA, dnmatB, dnmatC;
  hipsparseDnMatDescr_t dnMatDescr_t, dnmatA, dnmatB, dnmatC;

  // CHECK: rocsparse_spmm_alg spMMAlg_t;
  hipsparseSpMMAlg_t spMMAlg_t;

  // CHECK: _rocsparse_dnvec_descr *dnVecDescr = nullptr;
  // CHECK-NEXT: rocsparse_dnvec_descr dnVecDescr_t, vecX, vecY;
  cusparseDnVecDescr *dnVecDescr = nullptr;
  hipsparseDnVecDescr_t dnVecDescr_t, vecX, vecY;

  // CHECK: rocsparse_spmv_alg spMVAlg_t;
  hipsparseSpMVAlg_t spMVAlg_t;
#endif

#if CUDA_VERSION >= 11010 && CUSPARSE_VERSION >= 11300
  // CHECK: rocsparse_sparse_to_dense_alg sparseToDenseAlg_t;
  hipsparseSparseToDenseAlg_t sparseToDenseAlg_t;

  // CHECK: rocsparse_dense_to_sparse_alg denseToSparseAlg_t;
  hipsparseDenseToSparseAlg_t denseToSparseAlg_t;
#endif

#if CUDA_VERSION >= 11020 && CUSPARSE_VERSION >= 11400
  // CHECK: rocsparse_sddmm_alg sDDMMAlg_t;
  // CHECK-NEXT: rocsparse_sddmm_alg SDDMM_ALG_DEFAULT = rocsparse_sddmm_alg_default;
  hipsparseSDDMMAlg_t sDDMMAlg_t;
  hipsparseSDDMMAlg_t SDDMM_ALG_DEFAULT = HIPSPARSE_SDDMM_ALG_DEFAULT;
#endif

#if CUDA_VERSION >= 11030 && CUSPARSE_VERSION >= 11600
  // CHECK: rocsparse_spsm_alg spSMAlg_t;
  // CHECK-NEXT: rocsparse_spsm_alg SPSM_ALG_DEFAULT = rocsparse_spsm_alg_default;
  hipsparseSpSMAlg_t spSMAlg_t;
  hipsparseSpSMAlg_t SPSM_ALG_DEFAULT = HIPSPARSE_SPSM_ALG_DEFAULT;

  // NOTE:cusparseSpSMDescr_t doesn't have a correspondence in rocSPARSE, the corresponding function argument is removed in the hipified call of the rocsparse_spsm function
  hipsparseSpSMDescr_t spSMDescr;
#endif

#if CUDA_VERSION >= 12000
  // CHECK: rocsparse_spsv_alg spSVAlg_t;
  // CHECK-NEXT: rocsparse_spsv_alg SPSV_ALG_DEFAULT = rocsparse_spsv_alg_default;
  hipsparseSpSVAlg_t spSVAlg_t;
  hipsparseSpSVAlg_t SPSV_ALG_DEFAULT = HIPSPARSE_SPSV_ALG_DEFAULT;

  // TODO: remove decalration of cusparseSpSVDescr_t, as it is not mirroved and not used in rocSPARSE
  hipsparseSpSVDescr_t spSVDescr;

  // CHECK: rocsparse_const_spmat_descr constSpMatDescr = nullptr;
  // CHECK-NEXT: rocsparse_const_spmat_descr constSpMatDescrB = nullptr;
  hipsparseConstSpMatDescr_t constSpMatDescr = nullptr;
  hipsparseConstSpMatDescr_t constSpMatDescrB = nullptr;

  // CHECK: rocsparse_const_dnmat_descr constDnMatDescr = nullptr;
  // CHECK-NEXT: rocsparse_const_dnmat_descr constDnMatDescrB = nullptr;
  hipsparseConstDnMatDescr_t constDnMatDescr = nullptr;
  hipsparseConstDnMatDescr_t constDnMatDescrB = nullptr;

  // CHECK: rocsparse_const_spvec_descr constSpVecDescr = nullptr;
  hipsparseConstSpVecDescr_t constSpVecDescr = nullptr;

  // CHECK: rocsparse_const_dnvec_descr constDnVecDescr = nullptr;
  hipsparseConstDnVecDescr_t constDnVecDescr = nullptr;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSparseToDense(cusparseHandle_t handle, cusparseConstSpMatDescr_t matA, cusparseDnMatDescr_t matB, cusparseSparseToDenseAlg_t alg, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sparse_to_dense(rocsparse_handle handle, rocsparse_const_spmat_descr mat_A, rocsparse_dnmat_descr mat_B, rocsparse_sparse_to_dense_alg alg, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_sparse_to_dense(handle_t, constSpMatDescr, dnmatB, sparseToDenseAlg_t, nullptr, tempBuffer);
  status_t = hipsparseSparseToDense(handle_t, constSpMatDescr, dnmatB, sparseToDenseAlg_t, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSparseToDense_bufferSize(cusparseHandle_t handle, cusparseConstSpMatDescr_t matA, cusparseDnMatDescr_t matB, cusparseSparseToDenseAlg_t alg, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sparse_to_dense(rocsparse_handle handle, rocsparse_const_spmat_descr mat_A, rocsparse_dnmat_descr mat_B, rocsparse_sparse_to_dense_alg alg, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_sparse_to_dense(handle_t, constSpMatDescr, dnmatB, sparseToDenseAlg_t, &bufferSize, nullptr);
  status_t = hipsparseSparseToDense_bufferSize(handle_t, constSpMatDescr, dnmatB, sparseToDenseAlg_t, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDenseToSparse_bufferSize(cusparseHandle_t handle, cusparseConstDnMatDescr_t matA, cusparseSpMatDescr_t matB, cusparseDenseToSparseAlg_t alg, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dense_to_sparse(rocsparse_handle handle, rocsparse_const_dnmat_descr mat_A, rocsparse_spmat_descr mat_B, rocsparse_dense_to_sparse_alg alg, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_dense_to_sparse(handle_t, dnmatB, spMatDescr_t, denseToSparseAlg_t, &bufferSize, nullptr);
  status_t = hipsparseDenseToSparse_bufferSize(handle_t, dnmatB, spMatDescr_t, denseToSparseAlg_t, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDenseToSparse_analysis(cusparseHandle_t handle, cusparseConstDnMatDescr_t matA, cusparseSpMatDescr_t matB, cusparseDenseToSparseAlg_t alg, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dense_to_sparse(rocsparse_handle handle, rocsparse_const_dnmat_descr mat_A, rocsparse_spmat_descr mat_B, rocsparse_dense_to_sparse_alg alg, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_dense_to_sparse(handle_t, constDnMatDescr, spmatB, denseToSparseAlg_t, nullptr, tempBuffer);
  status_t = hipsparseDenseToSparse_analysis(handle_t, constDnMatDescr, spmatB, denseToSparseAlg_t, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMM_bufferSize(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstSpMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseDnMatDescr_t matC, cudaDataType computeType, cusparseSpMMAlg_t alg, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, const void* alpha, rocsparse_const_spmat_descr mat_A, rocsparse_const_dnmat_descr mat_B, const void* beta, const rocsparse_dnmat_descr mat_C, rocsparse_datatype compute_type, rocsparse_spmm_alg alg, rocsparse_spmm_stage stage, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spmm(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescr, beta, dnmatC, dataType, spMMAlg_t, rocsparse_spmm_stage_compute, &bufferSize, nullptr);
  status_t = hipsparseSpMM_bufferSize(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescr, beta, dnmatC, dataType, spMMAlg_t, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpSM_analysis(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstSpMatDescr_t matA, cusparseConstDnMatDescr_t matB, cusparseDnMatDescr_t matC, cudaDataType computeType, cusparseSpSMAlg_t alg, cusparseSpSMDescr_t spsmDescr, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spsm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, const void* alpha, rocsparse_const_spmat_descr matA, rocsparse_const_dnmat_descr matB, const rocsparse_dnmat_descr matC, rocsparse_datatype compute_type, rocsparse_spsm_alg alg, rocsparse_spsm_stage stage, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spsm(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescrB, dnmatC, dataType, spSMAlg_t, rocsparse_spsm_stage_compute, nullptr, tempBuffer);
  status_t = hipsparseSpSM_analysis(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescrB, dnmatC, dataType, spSMAlg_t, spSMDescr, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpSM_solve(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstSpMatDescr_t matA, cusparseConstDnMatDescr_t matB, cusparseDnMatDescr_t matC, cudaDataType computeType, cusparseSpSMAlg_t alg, cusparseSpSMDescr_t spsmDescr);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spsm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, const void* alpha, rocsparse_const_spmat_descr matA, rocsparse_const_dnmat_descr matB, const rocsparse_dnmat_descr matC, rocsparse_datatype compute_type, rocsparse_spsm_alg alg, rocsparse_spsm_stage stage, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spsm(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescrB, dnmatC, dataType, spSMAlg_t, rocsparse_spsm_stage_compute, nullptr, nullptr);
  status_t = hipsparseSpSM_solve(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescrB, dnmatC, dataType, spSMAlg_t, spSMDescr, nullptr);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMM(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstSpMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseDnMatDescr_t matC, cudaDataType computeType, cusparseSpMMAlg_t alg, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, const void* alpha, rocsparse_const_spmat_descr mat_A, rocsparse_const_dnmat_descr mat_B, const void* beta, const rocsparse_dnmat_descr mat_C, rocsparse_datatype compute_type, rocsparse_spmm_alg alg, rocsparse_spmm_stage stage, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spmm(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescr, beta, dnmatC, dataType, spMMAlg_t, rocsparse_spmm_stage_compute, nullptr, tempBuffer);
  status_t = hipsparseSpMM(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescr, beta, dnmatC, dataType, spMMAlg_t, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVV(cusparseHandle_t handle, cusparseOperation_t opX, cusparseConstSpVecDescr_t vecX, cusparseConstDnVecDescr_t vecY, void* result, cudaDataType computeType, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spvv(rocsparse_handle handle, rocsparse_operation trans, rocsparse_const_spvec_descr x, rocsparse_const_dnvec_descr y, void* result, rocsparse_datatype compute_type, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spvv(handle_t, opX, constSpVecDescr, constDnVecDescr, result, dataType, nullptr, tempBuffer);
  status_t = hipsparseSpVV(handle_t, opX, constSpVecDescr, constDnVecDescr, result, dataType, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVV_bufferSize(cusparseHandle_t handle, cusparseOperation_t opX, cusparseConstSpVecDescr_t vecX, cusparseConstDnVecDescr_t vecY, const void* result, cudaDataType computeType, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spvv(rocsparse_handle handle, rocsparse_operation trans, rocsparse_const_spvec_descr x, rocsparse_const_dnvec_descr y, void* result, rocsparse_datatype compute_type, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spvv(handle_t, opX, constSpVecDescr, constDnVecDescr, result, dataType, &bufferSize, nullptr);
  status_t = hipsparseSpVV_bufferSize(handle_t, opX, constSpVecDescr, constDnVecDescr, result, dataType, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMV(cusparseHandle_t handle, cusparseOperation_t opA, const void* alpha, cusparseConstSpMatDescr_t matA, cusparseConstDnVecDescr_t vecX, const void* beta, cusparseDnVecDescr_t vecY, cudaDataType computeType, cusparseSpMVAlg_t alg, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmv(rocsparse_handle handle, rocsparse_operation trans, const void* alpha, rocsparse_const_spmat_descr mat, rocsparse_const_dnvec_descr x, const void* beta, const rocsparse_dnvec_descr y, rocsparse_datatype compute_type, rocsparse_spmv_alg alg, rocsparse_spmv_stage stage, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spmv(handle_t, opA, alpha, constSpMatDescr, constDnVecDescr, beta, vecY, dataType, spMVAlg_t, rocsparse_spmv_stage_compute, tempBuffer);
  status_t = hipsparseSpMV(handle_t, opA, alpha, constSpMatDescr, constDnVecDescr, beta, vecY, dataType, spMVAlg_t, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMV_bufferSize(cusparseHandle_t handle, cusparseOperation_t opA, const void* alpha, cusparseConstSpMatDescr_t matA, cusparseConstDnVecDescr_t vecX, const void* beta, cusparseDnVecDescr_t vecY, cudaDataType computeType, cusparseSpMVAlg_t alg, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmv(rocsparse_handle handle, rocsparse_operation trans, const void* alpha, rocsparse_const_spmat_descr mat, rocsparse_const_dnvec_descr x, const void* beta, const rocsparse_dnvec_descr y, rocsparse_datatype compute_type, rocsparse_spmv_alg alg, rocsparse_spmv_stage stage, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spmv(handle_t, opA, alpha, constSpMatDescr, constDnVecDescr, beta, vecY, dataType, spMVAlg_t, rocsparse_spmv_stage_buffer_size, &bufferSize, nullptr);
  status_t = hipsparseSpMV_bufferSize(handle_t, opA, alpha, constSpMatDescr, constDnVecDescr, beta, vecY, dataType, spMVAlg_t, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMM_preprocess(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstSpMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseDnMatDescr_t matC, cudaDataType computeType, cusparseSpMMAlg_t alg, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, const void* alpha, const rocsparse_spmat_descr mat_A, const rocsparse_dnmat_descr mat_B, const void* beta, const rocsparse_dnmat_descr mat_C, rocsparse_datatype compute_type, rocsparse_spmm_alg alg, rocsparse_spmm_stage stage, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spmm(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescr, beta, dnmatC, dataType, spMMAlg_t, rocsparse_spmm_stage_preprocess, nullptr, tempBuffer);
  status_t = hipsparseSpMM_preprocess(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescr, beta, dnmatC, dataType, spMMAlg_t, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSDDMM_bufferSize(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstDnMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseSpMatDescr_t matC, cudaDataType computeType, cusparseSDDMMAlg_t alg, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sddmm_buffer_size(rocsparse_handle handle, rocsparse_operation opA, rocsparse_operation opB, const void* alpha, const rocsparse_dnmat_descr A, const rocsparse_dnmat_descr B, const void* beta, rocsparse_spmat_descr C, rocsparse_datatype compute_type, rocsparse_sddmm_alg alg, size_t* buffer_size);
  // CHECK: status_t = rocsparse_sddmm_buffer_size(handle_t, opA, opB, alpha, constDnMatDescr, constDnMatDescrB, beta, spmatC, dataType, sDDMMAlg_t, &bufferSize);
  status_t = hipsparseSDDMM_bufferSize(handle_t, opA, opB, alpha, constDnMatDescr, constDnMatDescrB, beta, spmatC, dataType, sDDMMAlg_t, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSDDMM_preprocess(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstDnMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseSpMatDescr_t matC, cudaDataType computeType, cusparseSDDMMAlg_t alg, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sddmm_preprocess(rocsparse_handle handle, rocsparse_operation opA, rocsparse_operation opB, const void* alpha, rocsparse_const_dnmat_descr A, rocsparse_const_dnmat_descr B, const void* beta, rocsparse_spmat_descr C, rocsparse_datatype compute_type, rocsparse_sddmm_alg alg, void* temp_buffer);
  // CHECK: status_t = rocsparse_sddmm_preprocess(handle_t, opA, opB, alpha, constDnMatDescr, constDnMatDescrB, beta, spmatC, dataType, sDDMMAlg_t, tempBuffer);
  status_t = hipsparseSDDMM_preprocess(handle_t, opA, opB, alpha, constDnMatDescr, constDnMatDescrB, beta, spmatC, dataType, sDDMMAlg_t, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSDDMM(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstDnMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseSpMatDescr_t matC, cudaDataType computeType, cusparseSDDMMAlg_t alg, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sddmm(rocsparse_handle handle, rocsparse_operation opA, rocsparse_operation opB, const void* alpha, rocsparse_const_dnmat_descr A, rocsparse_const_dnmat_descr B, const void* beta, rocsparse_spmat_descr C, rocsparse_datatype compute_type, rocsparse_sddmm_alg alg, void* temp_buffer);
  // CHECK: status_t = rocsparse_sddmm(handle_t, opA, opB, alpha, constDnMatDescr, constDnMatDescrB, beta, spmatC, dataType, sDDMMAlg_t, tempBuffer);
  status_t = hipsparseSDDMM(handle_t, opA, opB, alpha, constDnMatDescr, constDnMatDescrB, beta, spmatC, dataType, sDDMMAlg_t, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpSV_bufferSize(cusparseHandle_t handle, cusparseOperation_t opA, const void* alpha, cusparseConstSpMatDescr_t matA, cusparseConstDnVecDescr_t vecX, cusparseDnVecDescr_t vecY, cudaDataType computeType, cusparseSpSVAlg_t alg, cusparseSpSVDescr_t spsvDescr, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spsv(rocsparse_handle handle, rocsparse_operation trans, const void* alpha, rocsparse_const_spmat_descr mat, rocsparse_const_dnvec_descr x, const rocsparse_dnvec_descr y, rocsparse_datatype compute_type, rocsparse_spsv_alg alg, rocsparse_spsv_stage stage, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spsv(handle_t, opA, alpha, constSpMatDescr, constDnVecDescr, vecY, dataType, spSVAlg_t, rocsparse_spsv_stage_buffer_size, &bufferSize, nullptr);
  status_t = hipsparseSpSV_bufferSize(handle_t, opA, alpha, constSpMatDescr, constDnVecDescr, vecY, dataType, spSVAlg_t, spSVDescr, &bufferSize);
#endif

  return 0;
}
