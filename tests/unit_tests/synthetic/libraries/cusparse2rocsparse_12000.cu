// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18.1. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: rocsparse_status status_t;
  hipsparseStatus_t status_t;

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;

  // CHECK: rocsparse_operation opA, opB, opX;
  hipsparseOperation_t opA, opB, opX;

  // CHECK: rocsparse_solve_policy solvePolicy_t;
  hipsparseSolvePolicy_t solvePolicy_t;

  int m = 0;
  int n = 0;
  int k = 0;
  int innz = 0;
  int nnza = 0;
  int nnzb = 0;
  int nnzc = 0;
  int nnzd = 0;
  int csrRowPtrA = 0;
  int csrRowPtrB = 0;
  int csrRowPtrC = 0;
  int csrRowPtrD = 0;
  int csrColIndA = 0;
  int csrColIndB = 0;
  int csrColIndC = 0;
  int csrColIndD = 0;
  int bufferSizeInBytes = 0;
  size_t bufferSize = 0;
  double dA = 0.f;
  double dB = 0.f;
  double dAlpha = 0.f;
  double dF = 0.f;
  double dX = 0.f;
  double dcsrSortedValA = 0.f;
  double dcsrSortedValB = 0.f;
  double dcsrSortedValC = 0.f;
  double dcsrSortedValD = 0.f;
  float fAlpha = 0.f;
  float fA = 0.f;
  float fB = 0.f;
  float fF = 0.f;
  float fX = 0.f;
  float csrSortedValA = 0.f;
  float csrSortedValB = 0.f;
  float csrSortedValC = 0.f;
  float csrSortedValD = 0.f;
  void *pBuffer = nullptr;
  void *tempBuffer = nullptr;
  void *alpha = nullptr;
  void *beta = nullptr;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuDoubleComplex -> rocsparse_double_complex under a new option --sparse
  // CHECK: rocblas_double_complex dcomplex, dcomplexA, dcomplexAlpha, dcomplexB, dcomplexBeta, dcomplexC, dcomplexF, dcomplexX, dcomplexY, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dComplexcsrSortedValD, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;
  hipDoubleComplex dcomplex, dcomplexA, dcomplexAlpha, dcomplexB, dcomplexBeta, dcomplexC, dcomplexF, dcomplexX, dcomplexY, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dComplexcsrSortedValD, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuComplex -> rocsparse_float_complex under a new option --sparse
  // CHECK: rocblas_float_complex complex, complexA, complexAlpha, complexB, complexBeta, complexC, complexF, complexX, complexY, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complexcsrSortedValD, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;
  hipComplex complex, complexA, complexAlpha, complexB, complexBeta, complexC, complexF, complexX, complexY, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complexcsrSortedValD, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;

#if CUDA_VERSION >= 8000
  // TODO: [#899] There should be rocsparse_datatype instead of hipDataType
  hipDataType dataType_t;
  hipDataType dataType;
#endif

#if (CUDA_VERSION >= 10010 && CUDA_VERSION < 11000 && !defined(_WIN32)) || CUDA_VERSION >= 11000
  // CHECK: rocsparse_spmat_descr spMatDescr_t, spmatA, spmatB, spmatC;
  hipsparseSpMatDescr_t spMatDescr_t, spmatA, spmatB, spmatC;

  // CHECK: rocsparse_dnmat_descr dnMatDescr_t, dnmatA, dnmatB, dnmatC;
  hipsparseDnMatDescr_t dnMatDescr_t, dnmatA, dnmatB, dnmatC;

  // CHECK: rocsparse_spmm_alg spMMAlg_t;
  hipsparseSpMMAlg_t spMMAlg_t;
#endif

#if CUDA_VERSION >= 11010 && CUSPARSE_VERSION >= 11300
  // CHECK: rocsparse_sparse_to_dense_alg sparseToDenseAlg_t;
  hipsparseSparseToDenseAlg_t sparseToDenseAlg_t;

  // CHECK: rocsparse_dense_to_sparse_alg denseToSparseAlg_t;
  hipsparseDenseToSparseAlg_t denseToSparseAlg_t;
#endif

#if CUDA_VERSION >= 11030 && CUSPARSE_VERSION >= 11600
  // CHECK: rocsparse_spsm_alg spSMAlg_t;
  // CHECK-NEXT: rocsparse_spsm_alg SPSM_ALG_DEFAULT = rocsparse_spsm_alg_default;
  hipsparseSpSMAlg_t spSMAlg_t;
  hipsparseSpSMAlg_t SPSM_ALG_DEFAULT = HIPSPARSE_SPSM_ALG_DEFAULT;

  // NOTE:cusparseSpSMDescr_t doesn't have a correspondence in rocSPARSE, the corresponding function argument is removed in the hipified call of the rocsparse_spsm function
  hipsparseSpSMDescr_t spSMDescr;
#endif

#if CUDA_VERSION >= 12000
  // CHECK: rocsparse_const_spmat_descr constSpMatDescr = nullptr;
  // CHECK-NEXT: rocsparse_const_spmat_descr constSpMatDescrB = nullptr;
  hipsparseConstSpMatDescr_t constSpMatDescr = nullptr;
  hipsparseConstSpMatDescr_t constSpMatDescrB = nullptr;

  // CHECK: rocsparse_const_dnmat_descr constDnMatDescr = nullptr;
  // CHECK-NEXT: rocsparse_const_dnmat_descr constDnMatDescrB = nullptr;
  hipsparseConstDnMatDescr_t constDnMatDescr = nullptr;
  hipsparseConstDnMatDescr_t constDnMatDescrB = nullptr;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSparseToDense(cusparseHandle_t handle, cusparseConstSpMatDescr_t matA, cusparseDnMatDescr_t matB, cusparseSparseToDenseAlg_t alg, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sparse_to_dense(rocsparse_handle handle, rocsparse_const_spmat_descr mat_A, rocsparse_dnmat_descr mat_B, rocsparse_sparse_to_dense_alg alg, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_sparse_to_dense(handle_t, constSpMatDescr, dnmatB, sparseToDenseAlg_t, nullptr, tempBuffer);
  status_t = hipsparseSparseToDense(handle_t, constSpMatDescr, dnmatB, sparseToDenseAlg_t, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSparseToDense_bufferSize(cusparseHandle_t handle, cusparseConstSpMatDescr_t matA, cusparseDnMatDescr_t matB, cusparseSparseToDenseAlg_t alg, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sparse_to_dense(rocsparse_handle handle, rocsparse_const_spmat_descr mat_A, rocsparse_dnmat_descr mat_B, rocsparse_sparse_to_dense_alg alg, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_sparse_to_dense(handle_t, constSpMatDescr, dnmatB, sparseToDenseAlg_t, &bufferSize, nullptr);
  status_t = hipsparseSparseToDense_bufferSize(handle_t, constSpMatDescr, dnmatB, sparseToDenseAlg_t, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDenseToSparse_bufferSize(cusparseHandle_t handle, cusparseConstDnMatDescr_t matA, cusparseSpMatDescr_t matB, cusparseDenseToSparseAlg_t alg, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dense_to_sparse(rocsparse_handle handle, rocsparse_const_dnmat_descr mat_A, rocsparse_spmat_descr mat_B, rocsparse_dense_to_sparse_alg alg, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_dense_to_sparse(handle_t, dnmatB, spMatDescr_t, denseToSparseAlg_t, &bufferSize, nullptr);
  status_t = hipsparseDenseToSparse_bufferSize(handle_t, dnmatB, spMatDescr_t, denseToSparseAlg_t, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDenseToSparse_analysis(cusparseHandle_t handle, cusparseConstDnMatDescr_t matA, cusparseSpMatDescr_t matB, cusparseDenseToSparseAlg_t alg, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dense_to_sparse(rocsparse_handle handle, rocsparse_const_dnmat_descr mat_A, rocsparse_spmat_descr mat_B, rocsparse_dense_to_sparse_alg alg, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_dense_to_sparse(handle_t, constDnMatDescr, spmatB, denseToSparseAlg_t, nullptr, tempBuffer);
  status_t = hipsparseDenseToSparse_analysis(handle_t, constDnMatDescr, spmatB, denseToSparseAlg_t, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMM_bufferSize(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstSpMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseDnMatDescr_t matC, cudaDataType computeType, cusparseSpMMAlg_t alg, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, const void* alpha, rocsparse_const_spmat_descr mat_A, rocsparse_const_dnmat_descr mat_B, const void* beta, const rocsparse_dnmat_descr mat_C, rocsparse_datatype compute_type, rocsparse_spmm_alg alg, rocsparse_spmm_stage stage, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spmm(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescr, beta, dnmatC, dataType, spMMAlg_t, rocsparse_spmm_stage_compute, &bufferSize, nullptr);
  status_t = hipsparseSpMM_bufferSize(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescr, beta, dnmatC, dataType, spMMAlg_t, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpSM_analysis(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstSpMatDescr_t matA, cusparseConstDnMatDescr_t matB, cusparseDnMatDescr_t matC, cudaDataType computeType, cusparseSpSMAlg_t alg, cusparseSpSMDescr_t spsmDescr, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spsm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, const void* alpha, rocsparse_const_spmat_descr matA, rocsparse_const_dnmat_descr matB, const rocsparse_dnmat_descr matC, rocsparse_datatype compute_type, rocsparse_spsm_alg alg, rocsparse_spsm_stage stage, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spsm(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescrB, dnmatC, dataType, spSMAlg_t, rocsparse_spsm_stage_compute, nullptr, tempBuffer);
  status_t = hipsparseSpSM_analysis(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescrB, dnmatC, dataType, spSMAlg_t, spSMDescr, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpSM_solve(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstSpMatDescr_t matA, cusparseConstDnMatDescr_t matB, cusparseDnMatDescr_t matC, cudaDataType computeType, cusparseSpSMAlg_t alg, cusparseSpSMDescr_t spsmDescr);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spsm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, const void* alpha, rocsparse_const_spmat_descr matA, rocsparse_const_dnmat_descr matB, const rocsparse_dnmat_descr matC, rocsparse_datatype compute_type, rocsparse_spsm_alg alg, rocsparse_spsm_stage stage, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spsm(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescrB, dnmatC, dataType, spSMAlg_t, rocsparse_spsm_stage_compute, nullptr, nullptr);
  status_t = hipsparseSpSM_solve(handle_t, opA, opB, alpha, constSpMatDescr, constDnMatDescrB, dnmatC, dataType, spSMAlg_t, spSMDescr, nullptr);
#endif

  return 0;
}
