// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18.1. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: rocsparse_status status_t;
  hipsparseStatus_t status_t;

  // CHECK: rocsparse_operation opA, opB, opX;
  hipsparseOperation_t opA, opB, opX;

  int batchCount = 0;
  int m = 0;
  int n = 0;
  int innz = 0;
  int algo = 0;
  int bufferSizeInBytes = 0;
  double dds = 0.f;
  double ddl = 0.f;
  double dd = 0.f;
  double ddu = 0.f;
  double ddw = 0.f;
  double dx = 0.f;
  float fds = 0.f;
  float fdl = 0.f;
  float fd = 0.f;
  float fdu = 0.f;
  float fdw = 0.f;
  float fx = 0.f;
  size_t bufferSize = 0;
  void *pBuffer = nullptr;

#if CUDA_VERSION >= 7050
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgemvi_bufferSize(cusparseHandle_t handle, cusparseOperation_t transA, int m, int n, int nnz, int* pBufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zgemvi_buffer_size(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int n, rocsparse_int nnz, size_t* buffer_size);
  // CHECK: status_t = rocsparse_zgemvi_buffer_size(handle_t, opA, m, n, innz, reinterpret_cast<size_t*>(&bufferSizeInBytes));
  status_t = hipsparseZgemvi_bufferSize(handle_t, opA, m, n, innz, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgemvi_bufferSize(cusparseHandle_t handle, cusparseOperation_t transA, int m, int n, int nnz, int* pBufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cgemvi_buffer_size(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int n, rocsparse_int nnz, size_t* buffer_size);
  // CHECK: status_t = rocsparse_cgemvi_buffer_size(handle_t, opA, m, n, innz, reinterpret_cast<size_t*>(&bufferSizeInBytes));
  status_t = hipsparseCgemvi_bufferSize(handle_t, opA, m, n, innz, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgemvi_bufferSize(cusparseHandle_t handle, cusparseOperation_t transA, int m, int n, int nnz, int* pBufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dgemvi_buffer_size(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int n, rocsparse_int nnz, size_t* buffer_size);
  // CHECK: status_t = rocsparse_dgemvi_buffer_size(handle_t, opA, m, n, innz, reinterpret_cast<size_t*>(&bufferSizeInBytes));
  status_t = hipsparseDgemvi_bufferSize(handle_t, opA, m, n, innz, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgemvi_bufferSize(cusparseHandle_t handle, cusparseOperation_t transA, int m, int n, int nnz, int* pBufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sgemvi_buffer_size(rocsparse_handle handle, rocsparse_operation trans, rocsparse_int m, rocsparse_int n, rocsparse_int nnz, size_t* buffer_size);
  // CHECK: status_t = rocsparse_sgemvi_buffer_size(handle_t, opA, m, n, innz, reinterpret_cast<size_t*>(&bufferSizeInBytes));
  status_t = hipsparseSgemvi_bufferSize(handle_t, opA, m, n, innz, &bufferSizeInBytes);
#endif

  return 0;
}
