// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --amap --skip-excluded-preprocessor-conditional-blocks --experimental --roc %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocblas.h"
// CHECK-NOT: #include "cublas_v2.h"
#include "hipblas.h"
// CHECK-NOT: #include "rocblas.h"

#if defined(_WIN32) && CUDA_VERSION < 9000
  typedef signed   __int64 int64_t;
  typedef unsigned __int64 uint64_t;
#endif

int main() {
  printf("16.v2. cuBLAS API to hipBLAS API synthetic test\n");

  // CHECK: rocblas_operation blasOperation;
  // CHECK-NEXT: rocblas_operation BLAS_OP_N = rocblas_operation_none;
  // CHECK-NEXT: rocblas_operation BLAS_OP_T = rocblas_operation_transpose;
  // CHECK-NEXT: rocblas_operation BLAS_OP_C = rocblas_operation_conjugate_transpose;
  hipblasOperation_t blasOperation;
  hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;

  // CHECK: rocblas_status blasStatus;
  // CHECK-NEXT: rocblas_status blasStatus_t;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_SUCCESS = rocblas_status_success;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_NOT_INITIALIZED = rocblas_status_invalid_handle;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_ALLOC_FAILED = rocblas_status_not_implemented;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_INVALID_VALUE = rocblas_status_invalid_value;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_MAPPING_ERROR = rocblas_status_invalid_size;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_EXECUTION_FAILED = rocblas_status_memory_error;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_INTERNAL_ERROR = rocblas_status_internal_error;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_NOT_SUPPORTED = rocblas_status_perf_degraded;
  // CHECK-NEXT: rocblas_status BLAS_STATUS_ARCH_MISMATCH = rocblas_status_arch_mismatch;
  hipblasStatus_t blasStatus;
  hipblasStatus_t blasStatus_t;
  hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;

  // CHECK: rocblas_fill blasFillMode;
  // CHECK-NEXT: rocblas_fill BLAS_FILL_MODE_LOWER = rocblas_fill_lower;
  // CHECK-NEXT: rocblas_fill BLAS_FILL_MODE_UPPER = rocblas_fill_upper;
  hipblasFillMode_t blasFillMode;
  hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;

  // CHECK: rocblas_diagonal blasDiagType;
  // CHECK-NEXT: rocblas_diagonal BLAS_DIAG_NON_UNIT = rocblas_diagonal_non_unit;
  // CHECK-NEXT: rocblas_diagonal BLAS_DIAG_UNIT = rocblas_diagonal_unit;
  hipblasDiagType_t blasDiagType;
  hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;

  // CHECK: rocblas_side blasSideMode;
  // CHECK-NEXT: rocblas_side BLAS_SIDE_LEFT = rocblas_side_left;
  // CHECK-NEXT: rocblas_side BLAS_SIDE_RIGHT = rocblas_side_right;
  hipblasSideMode_t blasSideMode;
  hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;

  // CHECK: rocblas_pointer_mode blasPointerMode;
  // CHECK-NEXT: rocblas_pointer_mode BLAS_POINTER_MODE_HOST = rocblas_pointer_mode_host;
  // CHECK-NEXT: rocblas_pointer_mode BLAS_POINTER_MODE_DEVICE = rocblas_pointer_mode_device;
  hipblasPointerMode_t blasPointerMode;
  hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;

  // CHECK: rocblas_atomics_mode blasAtomicsMode;
  // CHECK-NEXT: rocblas_atomics_mode BLAS_ATOMICS_NOT_ALLOWED = rocblas_atomics_not_allowed;
  // CHECK-NEXT: rocblas_atomics_mode BLAS_ATOMICS_ALLOWED = rocblas_atomics_allowed;
  hipblasAtomicsMode_t blasAtomicsMode;
  hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;

  // CHECK: rocblas_handle blasHandle;
  hipblasHandle_t blasHandle;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetAtomicsMode(cublasHandle_t handle, cublasAtomicsMode_t* mode);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_atomics_mode(rocblas_handle handle, rocblas_atomics_mode* atomics_mode);
  // CHECK: blasStatus = rocblas_get_atomics_mode(blasHandle, &blasAtomicsMode);
  blasStatus = hipblasGetAtomicsMode(blasHandle, &blasAtomicsMode);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetAtomicsMode(cublasHandle_t handle, cublasAtomicsMode_t mode);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_atomics_mode(rocblas_handle handle, rocblas_atomics_mode atomics_mode);
  // CHECK: blasStatus = rocblas_set_atomics_mode(blasHandle, blasAtomicsMode);
  blasStatus = hipblasSetAtomicsMode(blasHandle, blasAtomicsMode);

  const char* const_ch = nullptr;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCreate_v2(cublasHandle_t* handle);
  // CUDA: #define cublasCreate cublasCreate_v2
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_create_handle(rocblas_handle* handle);
  // CHECK: blasStatus = rocblas_create_handle(&blasHandle);
  // CHECK-NEXT: blasStatus = rocblas_create_handle(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDestroy_v2(cublasHandle_t handle);
  // CUDA: #define cublasDestroy cublasDestroy_v2
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_destroy_handle(rocblas_handle handle);
  // CHECK: blasStatus = rocblas_destroy_handle(blasHandle);
  // CHECK-NEXT: blasStatus = rocblas_destroy_handle(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);

  // CHECK: hipStream_t stream;
  hipStream_t stream;

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetStream_v2(cublasHandle_t handle, cudaStream_t streamId);
  // CUDA: #define cublasSetStream cublasSetStream_v2
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_stream(rocblas_handle handle, hipStream_t stream);
  // CHECK: blasStatus = rocblas_set_stream(blasHandle, stream);
  // CHECK-NEXT: blasStatus = rocblas_set_stream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetStream_v2(cublasHandle_t handle, cudaStream_t* streamId);
  // CUDA: #define cublasGetStream cublasGetStream_v2
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_stream(rocblas_handle handle, hipStream_t* stream);
  // CHECK: blasStatus = rocblas_get_stream(blasHandle, &stream);
  // CHECK-NEXT: blasStatus = rocblas_get_stream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSetPointerMode_v2(cublasHandle_t handle, cublasPointerMode_t mode);
  // CUDA: #define cublasSetPointerMode cublasSetPointerMode_v2
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_pointer_mode(rocblas_handle handle, rocblas_pointer_mode pointer_mode);
  // CHECK: blasStatus = rocblas_set_pointer_mode(blasHandle, blasPointerMode);
  // CHECK-NEXT: blasStatus = rocblas_set_pointer_mode(blasHandle, blasPointerMode);
  blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);
  blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGetPointerMode_v2(cublasHandle_t handle, cublasPointerMode_t* mode);
  // CUDA: #define cublasGetPointerMode cublasGetPointerMode_v2
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_pointer_mode(rocblas_handle handle, rocblas_pointer_mode* pointer_mode);
  // CHECK: blasStatus = rocblas_get_pointer_mode(blasHandle, &blasPointerMode);
  // CHECK-NEXT: blasStatus = rocblas_get_pointer_mode(blasHandle, &blasPointerMode);
  blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);
  blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);

  int n = 0;
  int64_t n_64 = 0;
  int nrhs = 0;
  int m = 0;
  int num = 0;
  int lda = 0;
  int ldb = 0;
  int ldc = 0;
  int res = 0;
  int64_t res_64 = 0;
  int incx = 0;
  int64_t incx_64 = 0;
  int incy = 0;
  int64_t incy_64 = 0;
  int k = 0;
  int kl = 0;
  int ku = 0;
  int batchCount = 0;
  void *image = nullptr;
  void *image_2 = nullptr;
  void *valpha = nullptr;
  void *vc = nullptr;
  void *vs = nullptr;
  void *vx = nullptr;
  void *vy = nullptr;
  void *vresult = nullptr;

  // https://github.com/ROCmSoftwarePlatform/rocBLAS/issues/1281
  // TODO: Apply the chosen typecasting of int to rocblas_int arguments

  /*
  #if defined(rocblas_ILP64)
    typedef int64_t rocblas_int;
  #else
    typedef int32_t rocblas_int;
  #endif
  */

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetVector(int n, int elemSize, const void* x, int incx, void* devicePtr, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_vector(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_set_vector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasSetVector(n, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_vector(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_get_vector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasGetVector(n, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetVectorAsync(int n, int elemSize, const void* hostPtr, int incx, void* devicePtr, int incy, cudaStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_vector_async(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy, hipStream_t stream);
  // CHECK: blasStatus = rocblas_set_vector_async(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetVectorAsync(n, num, image, incx, image_2, incy, stream);

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetVectorAsync(int n, int elemSize, const void* devicePtr, int incx, void* hostPtr, int incy, cudaStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_vector_async(rocblas_int n, rocblas_int elem_size, const void* x, rocblas_int incx, void* y, rocblas_int incy, hipStream_t stream);
  // CHECK: blasStatus = rocblas_get_vector_async(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetVectorAsync(n, num, image, incx, image_2, incy, stream);

  int rows = 0;
  int cols = 0;

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_matrix(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_set_matrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasSetMatrix(rows, cols, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_matrix(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_get_matrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasGetMatrix(rows, cols, num, image, incx, image_2, incy);

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasSetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, cudaStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_set_matrix_async(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb, hipStream_t stream);
  // CHECK: blasStatus = rocblas_set_matrix_async(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  // TODO: #1281
  // CUDA: cublasStatus_t CUBLASWINAPI cublasGetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, cudaStream_t stream);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_get_matrix_async(rocblas_int rows, rocblas_int cols, rocblas_int elem_size, const void* a, rocblas_int lda, void* b, rocblas_int ldb, hipStream_t stream);
  // CHECK: blasStatus = rocblas_get_matrix_async(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  float fa = 0;
  float fA = 0;
  float fb = 0;
  float fB = 0;
  float fx = 0;
  float fx1 = 0;
  float fy = 0;
  float fy1 = 0;
  float fc = 0;
  float fC = 0;
  float fs = 0;
  float fd1 = 0;
  float fd2 = 0;
  float fresult = 0;
  float fparam = 0;

  float** fAarray = 0;
  const float** const fAarray_const = const_cast<const float**>(fAarray);
  float** fBarray = 0;
  const float** const fBarray_const = const_cast<const float**>(fBarray);
  float** fCarray = 0;
  float** fTauarray = 0;

  double da = 0;
  double dA = 0;
  double db = 0;
  double dB = 0;
  double dx = 0;
  double dx1 = 0;
  double dy = 0;
  double dy1 = 0;
  double dc = 0;
  double dC = 0;
  double ds = 0;
  double dd1 = 0;
  double dd2 = 0;
  double dresult = 0;
  double dparam = 0;

  double** dAarray = 0;
  const double** const dAarray_const = const_cast<const double**>(dAarray);
  double** dBarray = 0;
  const double** const dBarray_const = const_cast<const double**>(dBarray);
  double** dCarray = 0;
  double** dTauarray = 0;

  void** voidAarray = nullptr;
  const void** const voidAarray_const = const_cast<const void**>(voidAarray);
  void** voidBarray = nullptr;
  const void** const voidBarray_const = const_cast<const void**>(voidBarray);
  void** voidCarray = nullptr;

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSnrm2_v2(cublasHandle_t handle, int n, const float* x, int incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_snrm2(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, float* result);
  // CHECK: blasStatus = rocblas_snrm2(blasHandle, n, &fx, incx, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_snrm2(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDnrm2_v2(cublasHandle_t handle, int n, const double* x, int incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dnrm2(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, double* result);
  // CHECK: blasStatus = rocblas_dnrm2(blasHandle, n, &dx, incx, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_dnrm2(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);

  // CHECK: rocblas_float_complex complex, complexa, complexA, complexB, complexC, complexx, complexy, complexs, complexb, complexresult;
  hipComplex complex, complexa, complexA, complexB, complexC, complexx, complexy, complexs, complexb, complexresult;
  // CHECK: rocblas_double_complex dcomplex, dcomplexa, dcomplexA, dcomplexB, dcomplexC, dcomplexx, dcomplexy, dcomplexs, dcomplexb, dcomplexresult;
  hipDoubleComplex dcomplex, dcomplexa, dcomplexA, dcomplexB, dcomplexC, dcomplexx, dcomplexy, dcomplexs, dcomplexb, dcomplexresult;

  // CHECK: rocblas_float_complex** complexAarray = 0;
  // CHECK: const rocblas_float_complex** const complexAarray_const = const_cast<const rocblas_float_complex**>(complexAarray);
  // CHECK-NEXT: rocblas_float_complex** complexBarray = 0;
  // CHECK: const rocblas_float_complex** const complexBarray_const = const_cast<const rocblas_float_complex**>(complexBarray);
  // CHECK-NEXT: rocblas_float_complex** complexCarray = 0;
  // CHECK-NEXT: rocblas_float_complex** complexTauarray = 0;
  hipComplex** complexAarray = 0;
  const hipComplex** const complexAarray_const = const_cast<const hipComplex**>(complexAarray);
  hipComplex** complexBarray = 0;
  const hipComplex** const complexBarray_const = const_cast<const hipComplex**>(complexBarray);
  hipComplex** complexCarray = 0;
  hipComplex** complexTauarray = 0;

  // CHECK: rocblas_double_complex** dcomplexAarray = 0;
  // CHECK: const rocblas_double_complex** const dcomplexAarray_const = const_cast<const rocblas_double_complex**>(dcomplexAarray);
  // CHECK-NEXT: rocblas_double_complex** dcomplexBarray = 0;
  // CHECK: const rocblas_double_complex** const dcomplexBarray_const = const_cast<const rocblas_double_complex**>(dcomplexBarray);
  // CHECK-NEXT: rocblas_double_complex** dcomplexCarray = 0;
  // CHECK-NEXT: rocblas_double_complex** dcomplexTauarray = 0;
  hipDoubleComplex** dcomplexAarray = 0;
  const hipDoubleComplex** const dcomplexAarray_const = const_cast<const hipDoubleComplex**>(dcomplexAarray);
  hipDoubleComplex** dcomplexBarray = 0;
  const hipDoubleComplex** const dcomplexBarray_const = const_cast<const hipDoubleComplex**>(dcomplexBarray);
  hipDoubleComplex** dcomplexCarray = 0;
  hipDoubleComplex** dcomplexTauarray = 0;

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScnrm2_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scnrm2(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, float* result);
  // CHECK: blasStatus = rocblas_scnrm2(blasHandle, n, &complex, incx, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_scnrm2(blasHandle, n, &complex, incx, &fresult);
  blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);
  blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDznrm2_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dznrm2(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, double* result);
  // CHECK: blasStatus = rocblas_dznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_dznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSdot_v2(cublasHandle_t handle, int n, const float* x, int incx, const float* y, int incy, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sdot(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, const float* y, rocblas_int incy, float* result);
  // CHECK: blasStatus = rocblas_sdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_sdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDdot_v2(cublasHandle_t handle, int n, const double* x, int incx, const double* y, int incy, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ddot(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, const double* y, rocblas_int incy, double* result);
  // CHECK: blasStatus = rocblas_ddot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_ddot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdotu_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cdotu(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* result);
  // CHECK: blasStatus = rocblas_cdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  // CHECK-NEXT: blasStatus = rocblas_cdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdotc_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cdotc(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* result);
  // CHECK: blasStatus = rocblas_cdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  // CHECK-NEXT: blasStatus = rocblas_cdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdotu_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdotu(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* result);
  // CHECK: blasStatus = rocblas_zdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  // CHECK-NEXT: blasStatus = rocblas_zdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdotc_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdotc(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* result);
  // CHECK: blasStatus = rocblas_zdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  // CHECK-NEXT: blasStatus = rocblas_zdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSscal_v2(cublasHandle_t handle, int n, const float* alpha, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sscal(rocblas_handle handle, rocblas_int n, const float* alpha, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_sscal(blasHandle, n, &fy, &fx, incx);
  // CHECK-NEXT: blasStatus = rocblas_sscal(blasHandle, n, &fy, &fx, incx);
  blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);
  blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDscal_v2(cublasHandle_t handle, int n, const double* alpha, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dscal(rocblas_handle handle, rocblas_int n, const double* alpha, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dscal(blasHandle, n, &dx, &dy, incx);
  // CHECK-NEXT: blasStatus = rocblas_dscal(blasHandle, n, &dx, &dy, incx);
  blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);
  blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCscal_v2(cublasHandle_t handle, int n, const cuComplex* alpha, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cscal(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* alpha, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_cscal(blasHandle, n, &complexa, &complexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_cscal(blasHandle, n, &complexa, &complexx, incx);
  blasStatus = hipblasCscal(blasHandle, n, &complexa, &complexx, incx);
  blasStatus = hipblasCscal(blasHandle, n, &complexa, &complexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsscal_v2(cublasHandle_t handle, int n, const float* alpha, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csscal(rocblas_handle handle, rocblas_int n, const float* alpha, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_csscal(blasHandle, n, &fx, &complexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_csscal(blasHandle, n, &fx, &complexx, incx);
  blasStatus = hipblasCsscal(blasHandle, n, &fx, &complexx, incx);
  blasStatus = hipblasCsscal(blasHandle, n, &fx, &complexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZscal_v2(cublasHandle_t handle, int n, const cuDoubleComplex* alpha, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zscal(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* alpha, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_zscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_zscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  blasStatus = hipblasZscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  blasStatus = hipblasZscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdscal_v2(cublasHandle_t handle, int n, const double* alpha, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdscal(rocblas_handle handle, rocblas_int n, const double* alpha, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_zdscal(blasHandle, n, &dx, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_zdscal(blasHandle, n, &dx, &dcomplexx, incx);
  blasStatus = hipblasZdscal(blasHandle, n, &dx, &dcomplexx, incx);
  blasStatus = hipblasZdscal(blasHandle, n, &dx, &dcomplexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSaxpy_v2(cublasHandle_t handle, int n, const float* alpha, const float* x, int incx, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_saxpy(rocblas_handle handle, rocblas_int n, const float* alpha, const float* x, rocblas_int incx, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_saxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  // CHECK-NEXT: blasStatus = rocblas_saxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDaxpy_v2(cublasHandle_t handle, int n, const double* alpha, const double* x, int incx, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_daxpy(rocblas_handle handle, rocblas_int n, const double* alpha, const double* x, rocblas_int incx, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_daxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  // CHECK-NEXT: blasStatus = rocblas_daxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCaxpy_v2(cublasHandle_t handle, int n, const cuComplex* alpha, const cuComplex* x, int incx, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_caxpy(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_caxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_caxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCaxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCaxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZaxpy_v2(cublasHandle_t handle, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zaxpy(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_zaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScopy_v2(cublasHandle_t handle, int n, const float* x, int incx, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scopy(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_scopy(blasHandle, n, &fx, incx, &fy, incy);
  // CHECK-NEXT: blasStatus = rocblas_scopy(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDcopy_v2(cublasHandle_t handle, int n, const double* x, int incx, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dcopy(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dcopy(blasHandle, n, &dx, incx, &dy, incy);
  // CHECK-NEXT: blasStatus = rocblas_dcopy(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCcopy_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ccopy(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_ccopy(blasHandle, n, &complexx, incx, &complexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_ccopy(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCcopy(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCcopy(blasHandle, n, &complexx, incx, &complexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZcopy_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zcopy(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_zcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSswap_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sswap(rocblas_handle handle, rocblas_int n, float* x, rocblas_int incx, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_sswap(blasHandle, n, &fx, incx, &fy, incy);
  // CHECK-NEXT: blasStatus = rocblas_sswap(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDswap_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dswap(rocblas_handle handle, rocblas_int n, double* x, rocblas_int incx, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dswap(blasHandle, n, &dx, incx, &dy, incy);
  // CHECK-NEXT: blasStatus = rocblas_dswap(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCswap_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cswap(rocblas_handle handle, rocblas_int n, rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_cswap(blasHandle, n, &complexx, incx, &complexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_cswap(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCswap(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCswap(blasHandle, n, &complexx, incx, &complexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZswap_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zswap(rocblas_handle handle, rocblas_int n, rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_zswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIsamax_v2(cublasHandle_t handle, int n, const float* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_isamax(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_isamax(blasHandle, n, &fx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_isamax(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIdamax_v2(cublasHandle_t handle, int n, const double* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_idamax(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_idamax(blasHandle, n, &dx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_idamax(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIcamax_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_icamax(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_icamax(blasHandle, n, &complexx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_icamax(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamax(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamax(blasHandle, n, &complexx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIzamax_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_izamax(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_izamax(blasHandle, n, &dcomplexx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_izamax(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamax(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamax(blasHandle, n, &dcomplexx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIsamin_v2(cublasHandle_t handle, int n, const float* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_isamin(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_isamin(blasHandle, n, &fx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_isamin(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIdamin_v2(cublasHandle_t handle, int n, const double* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_idamin(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_idamin(blasHandle, n, &dx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_idamin(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIcamin_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_icamin(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_icamin(blasHandle, n, &complexx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_icamin(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamin(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamin(blasHandle, n, &complexx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIzamin_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, int* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_izamin(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, rocblas_int* result);
  // CHECK: blasStatus = rocblas_izamin(blasHandle, n, &dcomplexx, incx, &res);
  // CHECK-NEXT: blasStatus = rocblas_izamin(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamin(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamin(blasHandle, n, &dcomplexx, incx, &res);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSasum_v2(cublasHandle_t handle, int n, const float* x, int incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sasum(rocblas_handle handle, rocblas_int n, const float* x, rocblas_int incx, float* result);
  // CHECK: blasStatus = rocblas_sasum(blasHandle, n, &fx, incx, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_sasum(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDasum_v2(cublasHandle_t handle, int n, const double* x, int incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dasum(rocblas_handle handle, rocblas_int n, const double* x, rocblas_int incx, double* result);
  // CHECK: blasStatus = rocblas_dasum(blasHandle, n, &dx, incx, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_dasum(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScasum_v2(cublasHandle_t handle, int n, const cuComplex* x, int incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scasum(rocblas_handle handle, rocblas_int n, const rocblas_float_complex* x, rocblas_int incx, float* result);
  // CHECK: blasStatus = rocblas_scasum(blasHandle, n, &complexx, incx, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_scasum(blasHandle, n, &complexx, incx, &fresult);
  blasStatus = hipblasScasum(blasHandle, n, &complexx, incx, &fresult);
  blasStatus = hipblasScasum(blasHandle, n, &complexx, incx, &fresult);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDzasum_v2(cublasHandle_t handle, int n, const cuDoubleComplex* x, int incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dzasum(rocblas_handle handle, rocblas_int n, const rocblas_double_complex* x, rocblas_int incx, double* result);
  // CHECK: blasStatus = rocblas_dzasum(blasHandle, n, &dcomplexx, incx, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_dzasum(blasHandle, n, &dcomplexx, incx, &dresult);
  blasStatus = hipblasDzasum(blasHandle, n, &dcomplexx, incx, &dresult);
  blasStatus = hipblasDzasum(blasHandle, n, &dcomplexx, incx, &dresult);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrot_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* c, const float* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_srot(rocblas_handle handle, rocblas_int n, float* x, rocblas_int incx, float* y, rocblas_int incy, const float* c, const float* s);
  // CHECK: blasStatus = rocblas_srot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);
  // CHECK-NEXT: blasStatus = rocblas_srot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);
  blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);
  blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrot_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* c, const double* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_drot(rocblas_handle handle, rocblas_int n, double* x, rocblas_int incx, double* y, rocblas_int incy, const double* c, const double* s);
  // CHECK: blasStatus = rocblas_drot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);
  // CHECK-NEXT: blasStatus = rocblas_drot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);
  blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);
  blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);

  // TODO: #1281
  // CUDA: CUBLASAPI CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCrot_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy, const float* c, const cuComplex* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_crot(rocblas_handle handle, rocblas_int n, rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* y, rocblas_int incy, const float* c, const rocblas_float_complex* s);
  // CHECK: blasStatus = rocblas_crot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);
  // CHECK-NEXT: blasStatus = rocblas_crot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);
  blasStatus = hipblasCrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);
  blasStatus = hipblasCrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsrot_v2(cublasHandle_t handle, int n, cuComplex* x, int incx, cuComplex* y, int incy, const float* c, const float* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csrot(rocblas_handle handle, rocblas_int n, rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* y, rocblas_int incy, const float* c, const float* s);
  // CHECK: blasStatus = rocblas_csrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);
  // CHECK-NEXT: blasStatus = rocblas_csrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);
  blasStatus = hipblasCsrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);
  blasStatus = hipblasCsrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZrot_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, const double* c, const cuDoubleComplex* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zrot(rocblas_handle handle, rocblas_int n, rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* y, rocblas_int incy, const double* c, const rocblas_double_complex* s);
  // CHECK: blasStatus = rocblas_zrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);
  // CHECK-NEXT: blasStatus = rocblas_zrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);
  blasStatus = hipblasZrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);
  blasStatus = hipblasZrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdrot_v2(cublasHandle_t handle, int n, cuDoubleComplex* x, int incx, cuDoubleComplex* y, int incy, const double* c, const double* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdrot(rocblas_handle handle, rocblas_int n, rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* y, rocblas_int incy, const double* c, const double* s);
  // CHECK: blasStatus = rocblas_zdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);
  // CHECK-NEXT: blasStatus = rocblas_zdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);
  blasStatus = hipblasZdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);
  blasStatus = hipblasZdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotg_v2(cublasHandle_t handle, float* a, float* b, float* c, float* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_srotg(rocblas_handle handle, float* a, float* b, float* c, float* s);
  // CHECK: blasStatus = rocblas_srotg(blasHandle, &fa, &fb, &fc, &fs);
  // CHECK-NEXT: blasStatus = rocblas_srotg(blasHandle, &fa, &fb, &fc, &fs);
  blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);
  blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotg_v2(cublasHandle_t handle, double* a, double* b, double* c, double* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_drotg(rocblas_handle handle, double* a, double* b, double* c, double* s);
  // CHECK: blasStatus = rocblas_drotg(blasHandle, &da, &db, &dc, &ds);
  // CHECK-NEXT: blasStatus = rocblas_drotg(blasHandle, &da, &db, &dc, &ds);
  blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);
  blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCrotg_v2(cublasHandle_t handle, cuComplex* a, cuComplex* b, float* c, cuComplex* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_crotg(rocblas_handle handle, rocblas_float_complex* a, rocblas_float_complex* b, float* c, rocblas_float_complex* s);
  // CHECK: blasStatus = rocblas_crotg(blasHandle, &complexa, &complexb, &fc, &complexs);
  // CHECK-NEXT: blasStatus = rocblas_crotg(blasHandle, &complexa, &complexb, &fc, &complexs);
  blasStatus = hipblasCrotg(blasHandle, &complexa, &complexb, &fc, &complexs);
  blasStatus = hipblasCrotg(blasHandle, &complexa, &complexb, &fc, &complexs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZrotg_v2(cublasHandle_t handle, cuDoubleComplex* a, cuDoubleComplex* b, double* c, cuDoubleComplex* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zrotg(rocblas_handle handle, rocblas_double_complex* a, rocblas_double_complex* b, double* c, rocblas_double_complex* s);
  // CHECK: blasStatus = rocblas_zrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);
  // CHECK-NEXT: blasStatus = rocblas_zrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);
  blasStatus = hipblasZrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);
  blasStatus = hipblasZrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotm_v2(cublasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* param);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_srotm(rocblas_handle handle, rocblas_int n, float* x, rocblas_int incx, float* y, rocblas_int incy, const float* param);
  // CHECK: blasStatus = rocblas_srotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_srotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotm_v2(cublasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* param);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_drotm(rocblas_handle handle, rocblas_int n, double* x, rocblas_int incx, double* y, rocblas_int incy, const double* param);
  // CHECK: blasStatus = rocblas_drotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_drotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotmg_v2(cublasHandle_t handle, float* d1, float* d2, float* x1, const float* y1, float* param);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_srotmg(rocblas_handle handle, float* d1, float* d2, float* x1, const float* y1, float* param);
  // CHECK: blasStatus = rocblas_srotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_srotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);
  blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);
  blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotmg_v2(cublasHandle_t handle, double* d1, double* d2, double* x1, const double* y1, double* param);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_drotmg(rocblas_handle handle, double* d1, double* d2, double* x1, const double* y1, double* param);
  // CHECK: blasStatus = rocblas_drotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_drotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);
  blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);
  blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sgemv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, const float* alpha, const float* A, rocblas_int lda, const float* x, rocblas_int incx, const float* beta, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_sgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = rocblas_sgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dgemv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, const double* alpha, const double* A, rocblas_int lda, const double* x, rocblas_int incx, const double* beta, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = rocblas_dgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgemv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* beta, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_cgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_cgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgemv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* beta, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_zgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sgbmv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, rocblas_int kl, rocblas_int ku, const float* alpha, const float* A, rocblas_int lda, const float* x, rocblas_int incx, const float* beta, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_sgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = rocblas_sgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dgbmv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, rocblas_int kl, rocblas_int ku, const double* alpha, const double* A, rocblas_int lda, const double* x, rocblas_int incx, const double* beta, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = rocblas_dgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgbmv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, rocblas_int kl, rocblas_int ku, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* beta, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_cgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_cgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgbmv_v2(cublasHandle_t handle, cublasOperation_t trans, int m, int n, int kl, int ku, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgbmv(rocblas_handle handle, rocblas_operation trans, rocblas_int m, rocblas_int n, rocblas_int kl, rocblas_int ku, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* beta, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_zgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* A, int lda, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_strmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const float* A, rocblas_int lda, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_strmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  // CHECK-NEXT: blasStatus = rocblas_strmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* A, int lda, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtrmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const double* A, rocblas_int lda, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  // CHECK-NEXT: blasStatus = rocblas_dtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctrmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const rocblas_float_complex* A, rocblas_int lda, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ctrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_ctrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztrmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const rocblas_double_complex* A, rocblas_int lda, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ztrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_ztrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const float* A, int lda, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_stbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_diagonal diag, rocblas_int m, rocblas_int k, const float* A, rocblas_int lda, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_stbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  // CHECK-NEXT: blasStatus = rocblas_stbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const double* A, int lda, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_diagonal diag, rocblas_int m, rocblas_int k, const double* A, rocblas_int lda, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  // CHECK-NEXT: blasStatus = rocblas_dtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_diagonal diag, rocblas_int m, rocblas_int k, const rocblas_float_complex* A, rocblas_int lda, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ctbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_ctbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_diagonal diag, rocblas_int m, rocblas_int k, const rocblas_double_complex* A, rocblas_int lda, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ztbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_ztbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* AP, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_stpmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const float* A, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_stpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  // CHECK-NEXT: blasStatus = rocblas_stpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* AP, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtpmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const double* A, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  // CHECK-NEXT: blasStatus = rocblas_dtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* AP, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctpmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const rocblas_float_complex* A, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ctpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_ctpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztpmv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const rocblas_double_complex* A, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ztpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_ztpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* A, int lda, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_strsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const float* A, rocblas_int lda, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_strsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  // CHECK-NEXT: blasStatus = rocblas_strsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* A, int lda, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtrsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const double* A, rocblas_int lda, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  // CHECK-NEXT: blasStatus = rocblas_dtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* A, int lda, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctrsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const rocblas_float_complex* A, rocblas_int lda, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ctrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_ctrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztrsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, const rocblas_double_complex* A, rocblas_int lda, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ztrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_ztrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const float* AP, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_stpsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, const float* AP, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_stpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  // CHECK-NEXT: blasStatus = rocblas_stpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const double* AP, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtpsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, const double* AP, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  // CHECK-NEXT: blasStatus = rocblas_dtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuComplex* AP, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctpsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, const rocblas_float_complex* AP, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ctpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_ctpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtpsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, const cuDoubleComplex* AP, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztpsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, const rocblas_double_complex* AP, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ztpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_ztpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const float* A, int lda, float* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_stbsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, rocblas_int k, const float* A, rocblas_int lda, float* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_stbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  // CHECK-NEXT: blasStatus = rocblas_stbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const double* A, int lda, double* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtbsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, rocblas_int k, const double* A, rocblas_int lda, double* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_dtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  // CHECK-NEXT: blasStatus = rocblas_dtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuComplex* A, int lda, cuComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctbsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, rocblas_int k, const rocblas_float_complex* A, rocblas_int lda, rocblas_float_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ctbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_ctbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtbsv_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int n, int k, const cuDoubleComplex* A, int lda, cuDoubleComplex* x, int incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztbsv(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int n, rocblas_int k, const rocblas_double_complex* A, rocblas_int lda, rocblas_double_complex* x, rocblas_int incx);
  // CHECK: blasStatus = rocblas_ztbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = rocblas_ztbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssymv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const float* A, rocblas_int lda, const float* x, rocblas_int incx, const float* beta, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_ssymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = rocblas_ssymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsymv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const double* A, rocblas_int lda, const double* x, rocblas_int incx, const double* beta, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = rocblas_dsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csymv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* beta, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_csymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_csymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCsymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCsymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsymv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zsymv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* beta, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_zsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChemv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_chemv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* beta, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_chemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_chemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhemv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zhemv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* beta, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_zhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, rocblas_int k, const float* alpha, const float* A, rocblas_int lda, const float* x, rocblas_int incx, const float* beta, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_ssbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = rocblas_ssbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, rocblas_int k, const double* alpha, const double* A, rocblas_int lda, const double* x, rocblas_int incx, const double* beta, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = rocblas_dsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_chbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* beta, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_chbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_chbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhbmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zhbmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* beta, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_zhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* AP, const float* x, int incx, const float* beta, float* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sspmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const float* A, const float* x, rocblas_int incx, const float* beta, float* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_sspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = rocblas_sspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* AP, const double* x, int incx, const double* beta, double* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dspmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const double* A, const double* x, rocblas_int incx, const double* beta, double* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_dspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = rocblas_dspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* AP, const cuComplex* x, int incx, const cuComplex* beta, cuComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_chpmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* AP, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* beta, rocblas_float_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_chpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_chpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpmv_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* AP, const cuDoubleComplex* x, int incx, const cuDoubleComplex* beta, cuDoubleComplex* y, int incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zhpmv(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* AP, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* beta, rocblas_double_complex* y, rocblas_int incy);
  // CHECK: blasStatus = rocblas_zhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = rocblas_zhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSger_v2(cublasHandle_t handle, int m, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sger(rocblas_handle handle, rocblas_int m, rocblas_int n, const float* alpha, const float* x, rocblas_int incx, const float* y, rocblas_int incy, float* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_sger(blasHandle, m, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  // CHECK-NEXT: blasStatus = rocblas_sger(blasHandle, m, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  blasStatus = hipblasSger(blasHandle, m, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  blasStatus = hipblasSger(blasHandle, m, n, &fa, &fx, incx, &fy, incy, &fA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDger_v2(cublasHandle_t handle, int m, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dger(rocblas_handle handle, rocblas_int m, rocblas_int n, const double* alpha, const double* x, rocblas_int incx, const double* y, rocblas_int incy, double* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_dger(blasHandle, m, n, &da, &dx, incx, &dy, incy, &dA, lda);
  // CHECK-NEXT: blasStatus = rocblas_dger(blasHandle, m, n, &da, &dx, incx, &dy, incy, &dA, lda);
  blasStatus = hipblasDger(blasHandle, m, n, &da, &dx, incx, &dy, incy, &dA, lda);
  blasStatus = hipblasDger(blasHandle, m, n, &da, &dx, incx, &dy, incy, &dA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgeru_v2(cublasHandle_t handle, int m, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgeru(rocblas_handle handle, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_cgeru(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  // CHECK-NEXT: blasStatus = rocblas_cgeru(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCgeru(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCgeru(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgerc_v2(cublasHandle_t handle, int m, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgerc(rocblas_handle handle, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_cgerc(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  // CHECK-NEXT: blasStatus = rocblas_cgerc(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCgerc(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCgerc(blasHandle, m, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgeru_v2(cublasHandle_t handle, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgeru(rocblas_handle handle, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_zgeru(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  // CHECK-NEXT: blasStatus = rocblas_zgeru(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZgeru(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZgeru(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgerc_v2(cublasHandle_t handle, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgerc(rocblas_handle handle, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_zgerc(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  // CHECK-NEXT: blasStatus = rocblas_zgerc(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZgerc(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZgerc(blasHandle, m, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssyr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const float* x, rocblas_int incx, float* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_ssyr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA, lda);
  // CHECK-NEXT: blasStatus = rocblas_ssyr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA, lda);
  blasStatus = hipblasSsyr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA, lda);
  blasStatus = hipblasSsyr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsyr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const double* x, rocblas_int incx, double* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_dsyr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA, lda);
  // CHECK-NEXT: blasStatus = rocblas_dsyr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA, lda);
  blasStatus = hipblasDsyr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA, lda);
  blasStatus = hipblasDsyr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, cuComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csyr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_csyr(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexA, lda);
  // CHECK-NEXT: blasStatus = rocblas_csyr(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexA, lda);
  blasStatus = hipblasCsyr(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexA, lda);
  blasStatus = hipblasCsyr(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zsyr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_zsyr(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexA, lda);
  // CHECK-NEXT: blasStatus = rocblas_zsyr(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexA, lda);
  blasStatus = hipblasZsyr(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexA, lda);
  blasStatus = hipblasZsyr(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const cuComplex* x, int incx, cuComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cher(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_cher(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA, lda);
  // CHECK-NEXT: blasStatus = rocblas_cher(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA, lda);
  blasStatus = hipblasCher(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA, lda);
  blasStatus = hipblasCher(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zher(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_zher(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA, lda);
  // CHECK-NEXT: blasStatus = rocblas_zher(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA, lda);
  blasStatus = hipblasZher(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA, lda);
  blasStatus = hipblasZher(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, float* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sspr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const float* x, rocblas_int incx, float* AP);
  // CHECK: blasStatus = rocblas_sspr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA);
  // CHECK-NEXT: blasStatus = rocblas_sspr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA);
  blasStatus = hipblasSspr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA);
  blasStatus = hipblasSspr(blasHandle, blasFillMode, n, &fa, &fx, incx, &fA);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, double* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dspr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const double* x, rocblas_int incx, double* AP);
  // CHECK: blasStatus = rocblas_dspr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA);
  // CHECK-NEXT: blasStatus = rocblas_dspr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA);
  blasStatus = hipblasDspr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA);
  blasStatus = hipblasDspr(blasHandle, blasFillMode, n, &da, &dx, incx, &dA);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const cuComplex* x, int incx, cuComplex* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_chpr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* AP);
  // CHECK: blasStatus = rocblas_chpr(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA);
  // CHECK-NEXT: blasStatus = rocblas_chpr(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA);
  blasStatus = hipblasChpr(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA);
  blasStatus = hipblasChpr(blasHandle, blasFillMode, n, &fa, &complexx, incx, &complexA);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpr_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const cuDoubleComplex* x, int incx, cuDoubleComplex* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zhpr(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* AP);
  // CHECK: blasStatus = rocblas_zhpr(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA);
  // CHECK-NEXT: blasStatus = rocblas_zhpr(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA);
  blasStatus = hipblasZhpr(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA);
  blasStatus = hipblasZhpr(blasHandle, blasFillMode, n, &da, &dcomplexx, incx, &dcomplexA);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssyr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const float* x, rocblas_int incx, const float* y, rocblas_int incy, float* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_ssyr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  // CHECK-NEXT: blasStatus = rocblas_ssyr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  blasStatus = hipblasSsyr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA, lda);
  blasStatus = hipblasSsyr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsyr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const double* x, rocblas_int incx, const double* y, rocblas_int incy, double* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_dsyr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA, lda);
  // CHECK-NEXT: blasStatus = rocblas_dsyr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA, lda);
  blasStatus = hipblasDsyr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA, lda);
  blasStatus = hipblasDsyr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csyr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_csyr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  // CHECK-NEXT: blasStatus = rocblas_csyr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCsyr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCsyr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zsyr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_zsyr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  // CHECK-NEXT: blasStatus = rocblas_zsyr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZsyr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZsyr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cher2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_cher2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  // CHECK-NEXT: blasStatus = rocblas_cher2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCher2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);
  blasStatus = hipblasCher2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* A, int lda);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zher2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* A, rocblas_int lda);
  // CHECK: blasStatus = rocblas_zher2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  // CHECK-NEXT: blasStatus = rocblas_zher2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZher2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);
  blasStatus = hipblasZher2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA, lda);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSspr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const float* alpha, const float* x, int incx, const float* y, int incy, float* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sspr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const float* alpha, const float* x, rocblas_int incx, const float* y, rocblas_int incy, float* AP);
  // CHECK: blasStatus = rocblas_sspr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA);
  // CHECK-NEXT: blasStatus = rocblas_sspr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA);
  blasStatus = hipblasSspr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA);
  blasStatus = hipblasSspr2(blasHandle, blasFillMode, n, &fa, &fx, incx, &fy, incy, &fA);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDspr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const double* alpha, const double* x, int incx, const double* y, int incy, double* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dspr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const double* alpha, const double* x, rocblas_int incx, const double* y, rocblas_int incy, double* AP);
  // CHECK: blasStatus = rocblas_dspr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA);
  // CHECK-NEXT: blasStatus = rocblas_dspr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA);
  blasStatus = hipblasDspr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA);
  blasStatus = hipblasDspr2(blasHandle, blasFillMode, n, &da, &dx, incx, &dy, incy, &dA);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChpr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuComplex* alpha, const cuComplex* x, int incx, const cuComplex* y, int incy, cuComplex* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_chpr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, rocblas_int incx, const rocblas_float_complex* y, rocblas_int incy, rocblas_float_complex* AP);
  // CHECK: blasStatus = rocblas_chpr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA);
  // CHECK-NEXT: blasStatus = rocblas_chpr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA);
  blasStatus = hipblasChpr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA);
  blasStatus = hipblasChpr2(blasHandle, blasFillMode, n, &complexa, &complexx, incx, &complexy, incy, &complexA);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhpr2_v2(cublasHandle_t handle, cublasFillMode_t uplo, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int incx, const cuDoubleComplex* y, int incy, cuDoubleComplex* AP);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zhpr2(rocblas_handle handle, rocblas_fill uplo, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, rocblas_int incx, const rocblas_double_complex* y, rocblas_int incy, rocblas_double_complex* AP);
  // CHECK: blasStatus = rocblas_zhpr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA);
  // CHECK-NEXT: blasStatus = rocblas_zhpr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA);
  blasStatus = hipblasZhpr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA);
  blasStatus = hipblasZhpr2(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy, &dcomplexA);

  // CHECK rocblas_operation transa, transb;
  hipblasOperation_t transa, transb;

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sgemm(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const float* alpha, const float* A, rocblas_int lda, const float* B, rocblas_int ldb, const float* beta, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_sgemm(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_sgemm(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSgemm(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSgemm(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dgemm(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const double* alpha, const double* A, rocblas_int lda, const double* B, rocblas_int ldb, const double* beta, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_dgemm(blasHandle, transa, transb, m, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_dgemm(blasHandle, transa, transb, m, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDgemm(blasHandle, transa, transb, m, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDgemm(blasHandle, transa, transb, m, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgemm(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, const rocblas_float_complex* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_cgemm(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_cgemm(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCgemm(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCgemm(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemm_v2(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgemm(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, const rocblas_double_complex* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zgemm(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_zgemm(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZgemm(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZgemm(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* const Aarray[], int lda, const float* const Barray[], int ldb, const float* beta, float* const Carray[], int ldc, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sgemm_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const float* alpha, const float* const A[], rocblas_int lda, const float* const B[], rocblas_int ldb, const float* beta, float* const C[], rocblas_int ldc, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_sgemm_batched(blasHandle, transa, transb, m, n, k, &fa, fAarray_const, lda, fBarray_const, ldb, &fb, fCarray, ldc, batchCount);
  blasStatus = hipblasSgemmBatched(blasHandle, transa, transb, m, n, k, &fa, fAarray_const, lda, fBarray_const, ldb, &fb, fCarray, ldc, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* const Aarray[], int lda, const double* const Barray[], int ldb, const double* beta, double* const Carray[], int ldc, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dgemm_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const double* alpha, const double* const A[], rocblas_int lda, const double* const B[], rocblas_int ldb, const double* beta, double* const C[], rocblas_int ldc, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_dgemm_batched(blasHandle, transa, transb, m, n, k, &da, dAarray_const, lda, dBarray_const, ldb, &db, dCarray, ldc, batchCount);
  blasStatus = hipblasDgemmBatched(blasHandle, transa, transb, m, n, k, &da, dAarray_const, lda, dBarray_const, ldb, &db, dCarray, ldc, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* const Aarray[], int lda, const cuComplex* const Barray[], int ldb, const cuComplex* beta, cuComplex* const Carray[], int ldc, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgemm_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* const A[], rocblas_int lda, const rocblas_float_complex* const B[], rocblas_int ldb, const rocblas_float_complex* beta, rocblas_float_complex* const C[], rocblas_int ldc, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_cgemm_batched(blasHandle, transa, transb, m, n, k, &complexa, complexAarray_const, lda, complexBarray_const, ldb, &complexb, complexCarray, ldc, batchCount);
  blasStatus = hipblasCgemmBatched(blasHandle, transa, transb, m, n, k, &complexa, complexAarray_const, lda, complexBarray_const, ldb, &complexb, complexCarray, ldc, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* const Aarray[], int lda, const cuDoubleComplex* const Barray[], int ldb, const cuDoubleComplex* beta, cuDoubleComplex* const Carray[], int ldc, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgemm_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* const A[], rocblas_int lda, const rocblas_double_complex* const B[], rocblas_int ldb, const rocblas_double_complex* beta, rocblas_double_complex* const C[], rocblas_int ldc, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_zgemm_batched(blasHandle, transa, transb, m, n, k, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray_const, ldb, &dcomplexb, dcomplexCarray, ldc, batchCount);
  blasStatus = hipblasZgemmBatched(blasHandle, transa, transb, m, n, k, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray_const, ldb, &dcomplexb, dcomplexCarray, ldc, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* beta, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssyrk(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_int n, rocblas_int k, const float* alpha, const float* A, rocblas_int lda, const float* beta, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_ssyrk(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, &fC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_ssyrk(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, &fC, ldc);
  blasStatus = hipblasSsyrk(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, &fC, ldc);
  blasStatus = hipblasSsyrk(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, &fC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* beta, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsyrk(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_int n, rocblas_int k, const double* alpha, const double* A, rocblas_int lda, const double* beta, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_dsyrk(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, &dC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_dsyrk(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, &dC, ldc);
  blasStatus = hipblasDsyrk(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, &dC, ldc);
  blasStatus = hipblasDsyrk(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, &dC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csyrk(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_csyrk(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_csyrk(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyrk(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyrk(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyrk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zsyrk(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zsyrk(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_zsyrk(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyrk(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyrk(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCherk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const cuComplex* A, int lda, const float* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cherk(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_int n, rocblas_int k, const float* alpha, const rocblas_float_complex* A, rocblas_int lda, const float* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_cherk(blasHandle, blasFillMode, transa, n, k, &fa, &complexA, lda, &fb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_cherk(blasHandle, blasFillMode, transa, n, k, &fa, &complexA, lda, &fb, &complexC, ldc);
  blasStatus = hipblasCherk(blasHandle, blasFillMode, transa, n, k, &fa, &complexA, lda, &fb, &complexC, ldc);
  blasStatus = hipblasCherk(blasHandle, blasFillMode, transa, n, k, &fa, &complexA, lda, &fb, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZherk_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const cuDoubleComplex* A, int lda, const double* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zherk(rocblas_handle handle, rocblas_fill uplo, rocblas_operation transA, rocblas_int n, rocblas_int k, const double* alpha, const rocblas_double_complex* A, rocblas_int lda, const double* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zherk(blasHandle, blasFillMode, transa, n, k, &da, &dcomplexA, lda, &db, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_zherk(blasHandle, blasFillMode, transa, n, k, &da, &dcomplexA, lda, &db, &dcomplexC, ldc);
  blasStatus = hipblasZherk(blasHandle, blasFillMode, transa, n, k, &da, &dcomplexA, lda, &db, &dcomplexC, ldc);
  blasStatus = hipblasZherk(blasHandle, blasFillMode, transa, n, k, &da, &dcomplexA, lda, &db, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssyr2k(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const float* alpha, const float* A, rocblas_int lda, const float* B, rocblas_int ldb, const float* beta, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_ssyr2k(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, ldb, &fb, &fC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_ssyr2k(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsyr2k(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsyr2k(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fb, ldb, &fb, &fC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsyr2k(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const double* alpha, const double* A, rocblas_int lda, const double* B, rocblas_int ldb, const double* beta, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_dsyr2k(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, ldb, &db, &dC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_dsyr2k(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsyr2k(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsyr2k(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &db, ldb, &db, &dC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csyr2k(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, const rocblas_float_complex* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_csyr2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &complexb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_csyr2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyr2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyr2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &complexb, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyr2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zsyr2k(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, const rocblas_double_complex* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zsyr2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &dcomplexb, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_zsyr2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyr2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyr2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &dcomplexb, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssyrkx(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const float* alpha, const float* A, rocblas_int lda, const float* B, rocblas_int ldb, const float* beta, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_ssyrkx(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsyrkx(blasHandle, blasFillMode, transa, n, k, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsyrkx(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const double* alpha, const double* A, rocblas_int lda, const double* B, rocblas_int ldb, const double* beta, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_dsyrkx(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsyrkx(blasHandle, blasFillMode, transa, n, k, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csyrkx(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, const rocblas_float_complex* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_csyrkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsyrkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsyrkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zsyrkx(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, const rocblas_double_complex* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zsyrkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsyrkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCher2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const float* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cher2k(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, const float* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_cher2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &fb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_cher2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &fb, &complexC, ldc);
  blasStatus = hipblasCher2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &fb, &complexC, ldc);
  blasStatus = hipblasCher2k(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexb, ldb, &fb, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZher2k_v2(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const double* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zher2k(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, const double* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zher2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &db, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_zher2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &db, &dcomplexC, ldc);
  blasStatus = hipblasZher2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &db, &dcomplexC, ldc);
  blasStatus = hipblasZher2k(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexb, ldb, &db, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCherkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const float* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cherkx(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, const float* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_cherkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &fb, &complexC, ldc);
  blasStatus = hipblasCherkx(blasHandle, blasFillMode, transa, n, k, &complexa, &complexA, lda, &complexB, ldb, &fb, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZherkx(cublasHandle_t handle, cublasFillMode_t uplo, cublasOperation_t trans, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const double* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zherkx(rocblas_handle handle, rocblas_fill uplo, rocblas_operation trans, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, const double* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zherkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &db, &dcomplexC, ldc);
  blasStatus = hipblasZherkx(blasHandle, blasFillMode, transa, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &db, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ssymm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_int m, rocblas_int n, const float* alpha, const float* A, rocblas_int lda, const float* B, rocblas_int ldb, const float* beta, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_ssymm(blasHandle, blasSideMode, blasFillMode, m, n, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_ssymm(blasHandle, blasSideMode, blasFillMode, m, n, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsymm(blasHandle, blasSideMode, blasFillMode, m, n, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);
  blasStatus = hipblasSsymm(blasHandle, blasSideMode, blasFillMode, m, n, &fa, &fA, lda, &fB, ldb, &fb, &fC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dsymm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_int m, rocblas_int n, const double* alpha, const double* A, rocblas_int lda, const double* B, rocblas_int ldb, const double* beta, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_dsymm(blasHandle, blasSideMode, blasFillMode, m, n, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_dsymm(blasHandle, blasSideMode, blasFillMode, m, n, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsymm(blasHandle, blasSideMode, blasFillMode, m, n, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);
  blasStatus = hipblasDsymm(blasHandle, blasSideMode, blasFillMode, m, n, &da, &dA, lda, &dB, ldb, &db, &dC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csymm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, const rocblas_float_complex* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_csymm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_csymm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsymm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasCsymm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZsymm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zsymm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, const rocblas_double_complex* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zsymm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_zsymm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsymm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZsymm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasChemm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, const cuComplex* beta, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_chemm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, const rocblas_float_complex* beta, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_chemm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_chemm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasChemm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);
  blasStatus = hipblasChemm(blasHandle, blasSideMode, blasFillMode, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexb, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZhemm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zhemm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, const rocblas_double_complex* beta, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zhemm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_zhemm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZhemm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);
  blasStatus = hipblasZhemm(blasHandle, blasSideMode, blasFillMode, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexb, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const float* alpha, const float* A, int lda, float* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_strsm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const float* alpha, const float* A, rocblas_int lda, float* B, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_strsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb);
  // CHECK-NEXT: blasStatus = rocblas_strsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb);
  blasStatus = hipblasStrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb);
  blasStatus = hipblasStrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const double* alpha, const double* A, int lda, double* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtrsm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const double* alpha, const double* A, rocblas_int lda, double* B, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_dtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb);
  // CHECK-NEXT: blasStatus = rocblas_dtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb);
  blasStatus = hipblasDtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb);
  blasStatus = hipblasDtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, cuComplex* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctrsm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, rocblas_float_complex* B, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_ctrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb);
  // CHECK-NEXT: blasStatus = rocblas_ctrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb);
  blasStatus = hipblasCtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb);
  blasStatus = hipblasCtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, cuDoubleComplex* B, int ldb);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztrsm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, rocblas_double_complex* B, rocblas_int ldb);
  // CHECK: blasStatus = rocblas_ztrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb);
  // CHECK-NEXT: blasStatus = rocblas_ztrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb);
  blasStatus = hipblasZtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb);
  blasStatus = hipblasZtrsm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const float* alpha, const float* A, int lda, const float* B, int ldb, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_strmm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const float* alpha, const float* A, rocblas_int lda, const float* B, rocblas_int ldb, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_strmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb, &fC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_strmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb, &fC, ldc);
  blasStatus = hipblasStrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb, &fC, ldc);
  blasStatus = hipblasStrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, &fA, lda, &fB, ldb, &fC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const double* alpha, const double* A, int lda, const double* B, int ldb, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtrmm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const double* alpha, const double* A, rocblas_int lda, const double* B, rocblas_int ldb, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_dtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb, &dC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_dtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb, &dC, ldc);
  blasStatus = hipblasDtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb, &dC, ldc);
  blasStatus = hipblasDtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, &dA, lda, &dB, ldb, &dC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* B, int ldb, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctrmm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* B, rocblas_int ldb, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_ctrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_ctrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexC, ldc);
  blasStatus = hipblasCtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexC, ldc);
  blasStatus = hipblasCtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, &complexA, lda, &complexB, ldb, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrmm_v2(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* B, int ldb, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztrmm(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* B, rocblas_int ldb, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_ztrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexC, ldc);
  // CHECK-NEXT: blasStatus = rocblas_ztrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexC, ldc);
  blasStatus = hipblasZtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexC, ldc);
  blasStatus = hipblasZtrmm(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexB, ldb, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const float* alpha, const float* A, int lda, const float* beta, const float* B, int ldb, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sgeam(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, const float* alpha, const float* A, rocblas_int lda, const float* beta, const float* B, rocblas_int ldb, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_sgeam(blasHandle, transa, transb, m, n, &fa, &fA, lda, &fb, &fB, ldb, &fC, ldc);
  blasStatus = hipblasSgeam(blasHandle, transa, transb, m, n, &fa, &fA, lda, &fb, &fB, ldb, &fC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const double* alpha, const double* A, int lda, const double* beta, const double* B, int ldb, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dgeam(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, const double* alpha, const double* A, rocblas_int lda, const double* beta, const double* B, rocblas_int ldb, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_dgeam(blasHandle, transa, transb, m, n, &da, &dA, lda, &db, &dB, ldb, &dC, ldc);
  blasStatus = hipblasDgeam(blasHandle, transa, transb, m, n, &da, &dA, lda, &db, &dB, ldb, &dC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const cuComplex* alpha, const cuComplex* A, int lda, const cuComplex* beta, const cuComplex* B, int ldb, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgeam(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* beta, const rocblas_float_complex* B, rocblas_int ldb, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_cgeam(blasHandle, transa, transb, m, n, &complexa, &complexA, lda, &complexb, &complexB, ldb, &complexC, ldc);
  blasStatus = hipblasCgeam(blasHandle, transa, transb, m, n, &complexa, &complexA, lda, &complexb, &complexB, ldb, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgeam(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, const cuDoubleComplex* beta, const cuDoubleComplex* B, int ldb, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgeam(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* beta, const rocblas_double_complex* B, rocblas_int ldb, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zgeam(blasHandle, transa, transb, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexB, ldb, &dcomplexC, ldc);
  blasStatus = hipblasZgeam(blasHandle, transa, transb, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexb, &dcomplexB, ldb, &dcomplexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasStrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const float* alpha, const float* const A[], int lda, float* const B[], int ldb, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_strsm_batched(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const float* alpha, const float* const A[], rocblas_int lda, float* const B[], rocblas_int ldb, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_strsm_batched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, fAarray_const, lda, fBarray, ldb, batchCount);
  blasStatus = hipblasStrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &fa, fAarray_const, lda, fBarray, ldb, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const double* alpha, const double* const A[], int lda, double* const B[], int ldb, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dtrsm_batched(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const double* alpha, const double* const A[], rocblas_int lda, double* const B[], rocblas_int ldb, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_dtrsm_batched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, dAarray_const, lda, dBarray, ldb, batchCount);
  blasStatus = hipblasDtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &da, dAarray_const, lda, dBarray, ldb, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuComplex* alpha, const cuComplex* const A[], int lda, cuComplex* const B[], int ldb, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ctrsm_batched(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const rocblas_float_complex* alpha, const rocblas_float_complex* const A[], rocblas_int lda, rocblas_float_complex* const B[], rocblas_int ldb, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_ctrsm_batched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, complexAarray_const, lda, complexBarray, ldb, batchCount);
  blasStatus = hipblasCtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &complexa, complexAarray_const, lda, complexBarray, ldb, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZtrsmBatched(cublasHandle_t handle, cublasSideMode_t side, cublasFillMode_t uplo, cublasOperation_t trans, cublasDiagType_t diag, int m, int n, const cuDoubleComplex* alpha, const cuDoubleComplex* const A[], int lda, cuDoubleComplex* const B[], int ldb, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ztrsm_batched(rocblas_handle handle, rocblas_side side, rocblas_fill uplo, rocblas_operation transA, rocblas_diagonal diag, rocblas_int m, rocblas_int n, const rocblas_double_complex* alpha, const rocblas_double_complex* const A[], rocblas_int lda, rocblas_double_complex* const B[], rocblas_int ldb, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_ztrsm_batched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray, ldb, batchCount);
  blasStatus = hipblasZtrsmBatched(blasHandle, blasSideMode, blasFillMode, transa, blasDiagType, m, n, &dcomplexa, dcomplexAarray_const, lda, dcomplexBarray, ldb, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const float* A, int lda, const float* x, int incx, float* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sdgmm(rocblas_handle handle, rocblas_side side, rocblas_int m, rocblas_int n, const float* A, rocblas_int lda, const float* x, rocblas_int incx, float* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_sdgmm(blasHandle, blasSideMode, m, n, &fa, lda, &fx, incx, &fC, ldc);
  blasStatus = hipblasSdgmm(blasHandle, blasSideMode, m, n, &fa, lda, &fx, incx, &fC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const double* A, int lda, const double* x, int incx, double* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ddgmm(rocblas_handle handle, rocblas_side side, rocblas_int m, rocblas_int n, const double* A, rocblas_int lda, const double* x, rocblas_int incx, double* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_ddgmm(blasHandle, blasSideMode, m, n, &da, lda, &dx, incx, &dC, ldc);
  blasStatus = hipblasDdgmm(blasHandle, blasSideMode, m, n, &da, lda, &dx, incx, &dC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const cuComplex* A, int lda, const cuComplex* x, int incx, cuComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cdgmm(rocblas_handle handle, rocblas_side side, rocblas_int m, rocblas_int n, const rocblas_float_complex* A, rocblas_int lda, const rocblas_float_complex* x, rocblas_int incx, rocblas_float_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_cdgmm(blasHandle, blasSideMode, m, n, &complexa, lda, &complexx, incx, &complexC, ldc);
  blasStatus = hipblasCdgmm(blasHandle, blasSideMode, m, n, &complexa, lda, &complexx, incx, &complexC, ldc);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdgmm(cublasHandle_t handle, cublasSideMode_t mode, int m, int n, const cuDoubleComplex* A, int lda, const cuDoubleComplex* x, int incx, cuDoubleComplex* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdgmm(rocblas_handle handle, rocblas_side side, rocblas_int m, rocblas_int n, const rocblas_double_complex* A, rocblas_int lda, const rocblas_double_complex* x, rocblas_int incx, rocblas_double_complex* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_zdgmm(blasHandle, blasSideMode, m, n, &dcomplexa, lda, &dcomplexx, incx, &dcomplexC, ldc);
  blasStatus = hipblasZdgmm(blasHandle, blasSideMode, m, n, &dcomplexa, lda, &dcomplexx, incx, &dcomplexC, ldc);

  long long int strideA = 0;
  long long int strideB = 0;
  long long int strideC = 0;

#if CUDA_VERSION >= 7050
  // CHECK: rocblas_half* ha = 0;
  __half* ha = 0;
  // CHECK: rocblas_half* hA = 0;
  __half* hA = 0;
  // CHECK: rocblas_half* hb = 0;
  __half* hb = 0;
  // CHECK: rocblas_half* hB = 0;
  __half* hB = 0;
  // CHECK: rocblas_half* hc = 0;
  __half* hc = 0;
  // CHECK: rocblas_half* hC = 0;
  __half* hC = 0;

  // CHECK: rocblas_half** hAarray = 0;
  __half** hAarray = 0;
  // CHECK: const rocblas_half** const hAarray_const = const_cast<const rocblas_half**>(hAarray);
  const __half** const hAarray_const = const_cast<const __half**>(hAarray);
  // CHECK: rocblas_half** hBarray = 0;
  __half** hBarray = 0;
  // CHECK: const rocblas_half** const hBarray_const = const_cast<const rocblas_half**>(hBarray);
  const __half** const hBarray_const = const_cast<const __half**>(hBarray);
  // CHECK: rocblas_half** hCarray = 0;
  __half** hCarray = 0;
  // CHECK: const rocblas_half** const hCarray_const = const_cast<const rocblas_half**>(hCarray);
  const __half** const hCarray_const = const_cast<const __half**>(hCarray);
  // CHECK: rocblas_half** hxarray = 0;
  __half** hxarray = 0;
  // CHECK: const rocblas_half** const hxarray_const = const_cast<const rocblas_half**>(hxarray_const);
  const __half** const hxarray_const = const_cast<const __half**>(hxarray_const);
  // CHECK: rocblas_half** hyarray = 0;
  __half** hyarray = 0;

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemm(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* A, int lda, const __half* B, int ldb, const __half* beta, __half* C, int ldc);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_hgemm(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_half* alpha, const rocblas_half* A, rocblas_int lda, const rocblas_half* B, rocblas_int ldb, const rocblas_half* beta, rocblas_half* C, rocblas_int ldc);
  // CHECK: blasStatus = rocblas_hgemm(blasHandle, transa, transb, m, n, k, ha, hA, lda, hB, ldb, hb, hC, ldc);
  blasStatus = hipblasHgemm(blasHandle, transa, transb, m, n, k, ha, hA, lda, hB, ldb, hb, hC, ldc);
#endif

#if CUDA_VERSION >= 8000
  // CHECK: rocblas_datatype DataType;
  // CHECK-NEXT: rocblas_datatype_ DataType_t;
  // CHECK-NEXT: rocblas_datatype blasDataType;
  // CHECK-NEXT: rocblas_datatype R_16F = rocblas_datatype_f16_r;
  // CHECK-NEXT: rocblas_datatype C_16F = rocblas_datatype_f16_c;
  // CHECK-NEXT: rocblas_datatype R_32F = rocblas_datatype_f32_r;
  // CHECK-NEXT: rocblas_datatype C_32F = rocblas_datatype_f32_c;
  // CHECK-NEXT: rocblas_datatype R_64F = rocblas_datatype_f64_r;
  // CHECK-NEXT: rocblas_datatype C_64F = rocblas_datatype_f64_c;
  // CHECK-NEXT: rocblas_datatype R_8I = rocblas_datatype_i8_r;
  // CHECK-NEXT: rocblas_datatype C_8I = rocblas_datatype_i8_c;
  // CHECK-NEXT: rocblas_datatype R_8U = rocblas_datatype_u8_r;
  // CHECK-NEXT: rocblas_datatype C_8U = rocblas_datatype_u8_c;
  // CHECK-NEXT: rocblas_datatype R_32I = rocblas_datatype_i32_r;
  // CHECK-NEXT: rocblas_datatype C_32I = rocblas_datatype_i32_c;
  // CHECK-NEXT: rocblas_datatype R_32U = rocblas_datatype_u32_r;
  // CHECK-NEXT: rocblas_datatype C_32U = rocblas_datatype_u32_c;
  hipDataType DataType;
  hipDataType DataType_t;
  hipDataType blasDataType;
  hipDataType R_16F = HIP_R_16F;
  hipDataType C_16F = HIP_C_16F;
  hipDataType R_32F = HIP_R_32F;
  hipDataType C_32F = HIP_C_32F;
  hipDataType R_64F = HIP_R_64F;
  hipDataType C_64F = HIP_C_64F;
  hipDataType R_8I = HIP_R_8I;
  hipDataType C_8I = HIP_C_8I;
  hipDataType R_8U = HIP_R_8U;
  hipDataType C_8U = HIP_C_8U;
  hipDataType R_32I = HIP_R_32I;
  hipDataType C_32I = HIP_C_32I;
  hipDataType R_32U = HIP_R_32U;
  hipDataType C_32U = HIP_C_32U;

  // CHECK: rocblas_datatype DataType_2, DataType_3, alpha_type, cs_type, x_type, y_type, execution_type, result_type;
  hipDataType DataType_2, DataType_3, alpha_type, cs_type, x_type, y_type, execution_type, result_type;

  // CHECK: rocblas_gemm_algo blasGemmAlgo;
  // CHECK-NEXT: rocblas_gemm_algo BLAS_GEMM_DFALT = rocblas_gemm_algo_standard;
  hipblasGemmAlgo_t blasGemmAlgo;
  hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasNrm2Ex(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, void* result, cudaDataType resultType, cudaDataType executionType);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_nrm2_ex(rocblas_handle handle, rocblas_int n, const void* x, rocblas_datatype x_type, rocblas_int incx, void* results, rocblas_datatype result_type, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_nrm2_ex(blasHandle, n, image, DataType, incx, image_2, DataType_2, DataType_3);
  blasStatus = hipblasNrm2Ex(blasHandle, n, image, DataType, incx, image_2, DataType_2, DataType_3);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const float* alpha, const float* A, int lda, long long int strideA, const float* B, int ldb, long long int strideB, const float* beta, float* C, int ldc, long long int strideC, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sgemm_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const float* alpha, const float* A, rocblas_int lda, rocblas_stride stride_a, const float* B, rocblas_int ldb, rocblas_stride stride_b, const float* beta, float* C, rocblas_int ldc, rocblas_stride stride_c, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_sgemm_strided_batched(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, strideA, &fB, ldb, strideB, &fb, &fC, ldc, strideC, batchCount);
  blasStatus = hipblasSgemmStridedBatched(blasHandle, transa, transb, m, n, k, &fa, &fA, lda, strideA, &fB, ldb, strideB, &fb, &fC, ldc, strideC, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const double* alpha, const double* A, int lda, long long int strideA, const double* B, int ldb, long long int strideB, const double* beta, double* C, int ldc, long long int strideC, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dgemm_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const double* alpha, const double* A, rocblas_int lda, rocblas_stride stride_a, const double* B, rocblas_int ldb, rocblas_stride stride_b, const double* beta, double* C, rocblas_int ldc, rocblas_stride stride_c, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_dgemm_strided_batched(blasHandle, transa, transb, m, n, k, &da, &dA, lda, strideA, &dB, ldb, strideB, &db, &dC, ldc, strideC, batchCount);
  blasStatus = hipblasDgemmStridedBatched(blasHandle, transa, transb, m, n, k, &da, &dA, lda, strideA, &dB, ldb, strideB, &db, &dC, ldc, strideC, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuComplex* alpha, const cuComplex* A, int lda, long long int strideA, const cuComplex* B, int ldb, long long int strideB, const cuComplex* beta, cuComplex* C, int ldc, long long int strideC, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cgemm_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_float_complex* alpha, const rocblas_float_complex* A, rocblas_int lda, rocblas_stride stride_a, const rocblas_float_complex* B, rocblas_int ldb, rocblas_stride stride_b, const rocblas_float_complex* beta, rocblas_float_complex* C, rocblas_int ldc, rocblas_stride stride_c, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_cgemm_strided_batched(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, strideA, &complexB, ldb, strideB, &complexb, &complexC, ldc, strideC, batchCount);
  blasStatus = hipblasCgemmStridedBatched(blasHandle, transa, transb, m, n, k, &complexa, &complexA, lda, strideA, &complexB, ldb, strideB, &complexb, &complexC, ldc, strideC, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const cuDoubleComplex* alpha, const cuDoubleComplex* A, int lda, long long int strideA, const cuDoubleComplex* B, int ldb, long long int strideB, const cuDoubleComplex* beta, cuDoubleComplex* C, int ldc, long long int strideC, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zgemm_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_double_complex* alpha, const rocblas_double_complex* A, rocblas_int lda, rocblas_stride stride_a, const rocblas_double_complex* B, rocblas_int ldb, rocblas_stride stride_b, const rocblas_double_complex* beta, rocblas_double_complex* C, rocblas_int ldc, rocblas_stride stride_c, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_zgemm_strided_batched(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, strideA, &dcomplexB, ldb, strideB, &dcomplexb, &dcomplexC, ldc, strideC, batchCount);
  blasStatus = hipblasZgemmStridedBatched(blasHandle, transa, transb, m, n, k, &dcomplexa, &dcomplexA, lda, strideA, &dcomplexB, ldb, strideB, &dcomplexb, &dcomplexC, ldc, strideC, batchCount);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemmStridedBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* A, int lda, long long int strideA, const __half* B, int ldb, long long int strideB, const __half* beta, __half* C, int ldc, long long int strideC, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_hgemm_strided_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_half* alpha, const rocblas_half* A, rocblas_int lda, rocblas_stride stride_a, const rocblas_half* B, rocblas_int ldb, rocblas_stride stride_b, const rocblas_half* beta, rocblas_half* C, rocblas_int ldc, rocblas_stride stride_c, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_hgemm_strided_batched(blasHandle, transa, transb, m, n, k, ha, hA, lda, strideA, hB, ldb, strideB, hb, hC, ldc, strideC, batchCount);
  blasStatus = hipblasHgemmStridedBatched(blasHandle, transa, transb, m, n, k, ha, hA, lda, strideA, hB, ldb, strideB, hb, hC, ldc, strideC, batchCount);

  void* aptr = nullptr;
  void* Aptr = nullptr;
  void* bptr = nullptr;
  void* Bptr = nullptr;
  void* cptr = nullptr;
  void* Cptr = nullptr;
  void* xptr = nullptr;
  void* yptr = nullptr;
  void* sptr = nullptr;

  // CHECK: rocblas_datatype Atype;
  // CHECK-NEXT: rocblas_datatype Btype;
  // CHECK-NEXT: rocblas_datatype Ctype;
  // CHECK-NEXT: rocblas_datatype Xtype;
  // CHECK-NEXT: rocblas_datatype Ytype;
  // CHECK-NEXT: rocblas_datatype CStype;
  // CHECK-NEXT: rocblas_datatype Executiontype;
  hipDataType Atype;
  hipDataType Btype;
  hipDataType Ctype;
  hipDataType Xtype;
  hipDataType Ytype;
  hipDataType CStype;
  hipDataType Executiontype;

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScalEx(cublasHandle_t handle, int n, const void* alpha, cudaDataType alphaType, void* x, cudaDataType xType, int incx, cudaDataType executionType);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scal_ex(rocblas_handle handle, rocblas_int n, const void* alpha, rocblas_datatype alpha_type, void* x, rocblas_datatype x_type, rocblas_int incx, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_scal_ex(blasHandle, n, aptr, Atype, xptr, Xtype, incx, Executiontype);
  blasStatus = hipblasScalEx(blasHandle, n, aptr, Atype, xptr, Xtype, incx, Executiontype);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasAxpyEx(cublasHandle_t handle, int n, const void* alpha, cudaDataType alphaType, const void* x, cudaDataType xType, int incx, void* y, cudaDataType yType, int incy, cudaDataType executiontype);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_axpy_ex(rocblas_handle handle, rocblas_int n, const void* alpha, rocblas_datatype alpha_type, const void* x, rocblas_datatype x_type, rocblas_int incx, void* y, rocblas_datatype y_type, rocblas_int incy, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_axpy_ex(blasHandle, n, aptr, Atype, xptr, Xtype, incx, yptr, Ytype, incy, Executiontype);
  blasStatus = hipblasAxpyEx(blasHandle, n, aptr, Atype, xptr, Xtype, incx, yptr, Ytype, incy, Executiontype);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDotEx(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, const void* y, cudaDataType yType, int incy, void* result, cudaDataType resultType, cudaDataType executionType);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dot_ex(rocblas_handle handle, rocblas_int n, const void* x, rocblas_datatype x_type, rocblas_int incx, const void* y, rocblas_datatype y_type, rocblas_int incy, void* result, rocblas_datatype result_type, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_dot_ex(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
  blasStatus = hipblasDotEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDotcEx(cublasHandle_t handle, int n, const void* x, cudaDataType xType, int incx, const void* y, cudaDataType yType, int incy, void* result, cudaDataType resultType, cudaDataType executionType);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dotc_ex(rocblas_handle handle, rocblas_int n, const void* x, rocblas_datatype x_type, rocblas_int incx, const void* y, rocblas_datatype y_type, rocblas_int incy, void* result, rocblas_datatype result_type, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_dotc_ex(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
  blasStatus = hipblasDotcEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, image, DataType, Executiontype);
#endif

#if CUDA_VERSION >= 8000 && CUDA_VERSION < 11000
  // CHECK: rocblas_datatype computeType;
  cudaDataType computeType;

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, const void* B, cudaDataType Btype, int ldb, const void* beta, void* C, cudaDataType Ctype, int ldc, cudaDataType computeType, cublasGemmAlgo_t algo);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_gemm_ex(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const void* alpha, const void* a, rocblas_datatype a_type, rocblas_int lda, const void* b, rocblas_datatype b_type, rocblas_int ldb, const void* beta, const void* c, rocblas_datatype c_type, rocblas_int ldc, void* d, rocblas_datatype d_type, rocblas_int ldd, rocblas_datatype compute_type, rocblas_gemm_algo algo, int32_t solution_index, uint32_t flags);
  // CHECK: blasStatus = rocblas_gemm_ex(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, computeType, blasGemmAlgo);
  blasStatus = cublasGemmEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, Bptr, Btype, ldb, bptr, Cptr, Ctype, ldc, computeType, blasGemmAlgo);
#endif

#if CUDA_VERSION >= 9000
  // CHECK: rocblas_gemm_algo BLAS_GEMM_DEFAULT = rocblas_gemm_algo_standard;
  hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasHgemmBatched(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const __half* alpha, const __half* const Aarray[], int lda, const __half* const Barray[], int ldb, const __half* beta, __half* const Carray[], int ldc, int batchCount);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_hgemm_batched(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const rocblas_half* alpha, const rocblas_half* const A[], rocblas_int lda, const rocblas_half* const B[], rocblas_int ldb, const rocblas_half* beta, rocblas_half* const C[], rocblas_int ldc, rocblas_int batch_count);
  // CHECK: blasStatus = rocblas_hgemm_batched(blasHandle, transa, transb, m, n, k, ha, hAarray_const, lda, hBarray_const, ldb, hb, hCarray, ldc, batchCount);
  blasStatus = hipblasHgemmBatched(blasHandle, transa, transb, m, n, k, ha, hAarray_const, lda, hBarray_const, ldb, hb, hCarray, ldc, batchCount);
#endif

#if CUDA_VERSION >= 9010 && CUDA_VERSION < 11000
  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* const Aarray[], cudaDataType Atype, int lda, const void* const Barray[], cudaDataType Btype, int ldb, const void* beta, void* const Carray[], cudaDataType Ctype, int ldc, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_gemm_batched_ex(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const void* alpha, const void* a, rocblas_datatype a_type, rocblas_int lda, const void* b, rocblas_datatype b_type, rocblas_int ldb, const void* beta, const void* c, rocblas_datatype c_type, rocblas_int ldc, void* d, rocblas_datatype d_type, rocblas_int ldd, rocblas_int batch_count, rocblas_datatype compute_type, rocblas_gemm_algo algo, int32_t solution_index, uint32_t flags);
  // CHECK: blasStatus = rocblas_gemm_batched_ex(blasHandle, transa, transb, m, n, k, aptr, voidAarray_const, Atype, lda, voidBarray_const, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, computeType, blasGemmAlgo);
  blasStatus = cublasGemmBatchedEx(blasHandle, transa, transb, m, n, k, aptr, voidAarray_const, Atype, lda, voidBarray_const, Btype, ldb, bptr, voidCarray, Ctype, ldc, batchCount, computeType, blasGemmAlgo);

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasGemmStridedBatchedEx(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, int m, int n, int k, const void* alpha, const void* A, cudaDataType Atype, int lda, long long int strideA, const void* B, cudaDataType Btype, int ldb, long long int strideB, const void* beta, void* C, cudaDataType Ctype, int ldc, long long int strideC, int batchCount, cublasComputeType_t computeType, cublasGemmAlgo_t algo);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_gemm_strided_batched_ex(rocblas_handle handle, rocblas_operation transA, rocblas_operation transB, rocblas_int m, rocblas_int n, rocblas_int k, const void* alpha, const void* a, rocblas_datatype a_type, rocblas_int lda, rocblas_stride stride_a, const void* b, rocblas_datatype b_type, rocblas_int ldb, rocblas_stride stride_b, const void* beta, const void* c, rocblas_datatype c_type, rocblas_int ldc, rocblas_stride stride_c, void* d, rocblas_datatype d_type, rocblas_int ldd, rocblas_stride stride_d, rocblas_int batch_count, rocblas_datatype compute_type, rocblas_gemm_algo algo, int32_t solution_index, uint32_t flags);
  // CHECK: blasStatus = rocblas_gemm_strided_batched_ex(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, computeType, blasGemmAlgo);
  blasStatus = cublasGemmStridedBatchedEx(blasHandle, transa, transb, m, n, k, aptr, Aptr, Atype, lda, strideA, Bptr, Btype, ldb, strideB, bptr, Cptr, Ctype, ldc, strideC, batchCount, computeType, blasGemmAlgo);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: rocblas_operation BLAS_OP_HERMITAN = rocblas_operation_conjugate_transpose;
  hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;

  // CHECK: rocblas_fill BLAS_FILL_MODE_FULL = rocblas_fill_full;
  hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;

  // TODO: #1281
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasRotEx(cublasHandle_t handle, int n, void* x, cudaDataType xType, int incx, void* y, cudaDataType yType, int incy, const void* c, const void* s, cudaDataType csType, cudaDataType executiontype);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_rot_ex(rocblas_handle handle, rocblas_int n, void* x, rocblas_datatype x_type, rocblas_int incx, void* y, rocblas_datatype y_type, rocblas_int incy, const void* c, const void* s, rocblas_datatype cs_type, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_rot_ex(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, cptr, sptr, CStype, Executiontype);
  blasStatus = hipblasRotEx(blasHandle, n, xptr, Xtype, incx, yptr, Ytype, incy, cptr, sptr, CStype, Executiontype);
#endif

#if CUDA_VERSION >= 11000
  // CHECK: rocblas_datatype R_16BF = rocblas_datatype_bf16_r;
  // CHECK-NEXT: rocblas_datatype C_16BF = rocblas_datatype_bf16_c;
  hipDataType R_16BF = HIP_R_16BF;
  hipDataType C_16BF = HIP_C_16BF;
#endif

#if CUDA_VERSION >= 11040 && CUBLAS_VERSION >= 11600
  // CUDA: CUBLASAPI const char* CUBLASWINAPI cublasGetStatusString(cublasStatus_t status);
  // ROC: ROCBLAS_EXPORT const char* rocblas_status_to_string(rocblas_status status);
  // CHECK: const_ch = rocblas_status_to_string(blasStatus);
  const_ch = cublasGetStatusString(blasStatus);
#endif

#if CUDA_VERSION >= 12000
  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIsamax_v2_64(cublasHandle_t handle, int64_t n, const float* x, int64_t incx, int64_t* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_isamax_64(rocblas_handle handle, int64_t n, const float* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = rocblas_isamax_64(blasHandle, n_64, &fx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = rocblas_isamax_64(blasHandle, n_64, &fx, incx_64, &res_64);
  blasStatus = hipblasIsamax_64(blasHandle, n_64, &fx, incx_64, &res_64);
  blasStatus = hipblasIsamax_64(blasHandle, n_64, &fx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIdamax_v2_64(cublasHandle_t handle, int64_t n, const double* x, int64_t incx, int64_t* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_idamax_64(rocblas_handle handle, int64_t n, const double* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = rocblas_idamax_64(blasHandle, n_64, &dx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = rocblas_idamax_64(blasHandle, n_64, &dx, incx_64, &res_64);
  blasStatus = hipblasIdamax_64(blasHandle, n_64, &dx, incx_64, &res_64);
  blasStatus = hipblasIdamax_64(blasHandle, n_64, &dx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIcamax_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* x, int64_t incx, int64_t* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_icamax_64(rocblas_handle handle, int64_t n, const rocblas_float_complex* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = rocblas_icamax_64(blasHandle, n_64, &complexx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = rocblas_icamax_64(blasHandle, n_64, &complexx, incx_64, &res_64);
  blasStatus = hipblasIcamax_64(blasHandle, n_64, &complexx, incx_64, &res_64);
  blasStatus = hipblasIcamax_64(blasHandle, n_64, &complexx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIzamax_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* x, int64_t incx, int64_t* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_izamax_64(rocblas_handle handle, int64_t n, const rocblas_double_complex* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = rocblas_izamax_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = rocblas_izamax_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);
  blasStatus = hipblasIzamax_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);
  blasStatus = hipblasIzamax_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIsamin_v2_64(cublasHandle_t handle, int64_t n, const float* x, int64_t incx, int64_t* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_isamin_64(rocblas_handle handle, int64_t n, const float* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = rocblas_isamin_64(blasHandle, n_64, &fx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = rocblas_isamin_64(blasHandle, n_64, &fx, incx_64, &res_64);
  blasStatus = hipblasIsamin_64(blasHandle, n_64, &fx, incx_64, &res_64);
  blasStatus = hipblasIsamin_64(blasHandle, n_64, &fx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIdamin_v2_64(cublasHandle_t handle, int64_t n, const double* x, int64_t incx, int64_t* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_idamin_64(rocblas_handle handle, int64_t n, const double* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = rocblas_idamin_64(blasHandle, n_64, &dx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = rocblas_idamin_64(blasHandle, n_64, &dx, incx_64, &res_64);
  blasStatus = hipblasIdamin_64(blasHandle, n_64, &dx, incx_64, &res_64);
  blasStatus = hipblasIdamin_64(blasHandle, n_64, &dx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIcamin_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* x, int64_t incx, int64_t* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_icamin_64(rocblas_handle handle, int64_t n, const rocblas_float_complex* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = rocblas_icamin_64(blasHandle, n_64, &complexx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = rocblas_icamin_64(blasHandle, n_64, &complexx, incx_64, &res_64);
  blasStatus = hipblasIcamin_64(blasHandle, n_64, &complexx, incx_64, &res_64);
  blasStatus = hipblasIcamin_64(blasHandle, n_64, &complexx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasIzamin_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* x, int64_t incx, int64_t* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_izamin_64(rocblas_handle handle, int64_t n, const rocblas_double_complex* x, int64_t incx, int64_t* result);
  // CHECK: blasStatus = rocblas_izamin_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);
  // CHECK-NEXT: blasStatus = rocblas_izamin_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);
  blasStatus = hipblasIzamin_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);
  blasStatus = hipblasIzamin_64(blasHandle, n_64, &dcomplexx, incx_64, &res_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSasum_v2_64(cublasHandle_t handle, int64_t n, const float* x, int64_t incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sasum_64(rocblas_handle handle, int64_t n, const float* x, int64_t incx, float* result);
  // CHECK: blasStatus = rocblas_sasum_64(blasHandle, n_64, &fx, incx_64, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_sasum_64(blasHandle, n_64, &fx, incx_64, &fresult);
  blasStatus = hipblasSasum_64(blasHandle, n_64, &fx, incx_64, &fresult);
  blasStatus = hipblasSasum_64(blasHandle, n_64, &fx, incx_64, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDasum_v2_64(cublasHandle_t handle, int64_t n, const double* x, int64_t incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dasum_64(rocblas_handle handle, int64_t n, const double* x, int64_t incx, double* result);
  // CHECK: blasStatus = rocblas_dasum_64(blasHandle, n_64, &dx, incx_64, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_dasum_64(blasHandle, n_64, &dx, incx_64, &dresult);
  blasStatus = hipblasDasum_64(blasHandle, n_64, &dx, incx_64, &dresult);
  blasStatus = hipblasDasum_64(blasHandle, n_64, &dx, incx_64, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScasum_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* x, int64_t incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scasum_64(rocblas_handle handle, int64_t n, const rocblas_float_complex* x, int64_t incx, float* result);
  // CHECK: blasStatus = rocblas_scasum_64(blasHandle, n_64, &complexx, incx_64, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_scasum_64(blasHandle, n_64, &complexx, incx_64, &fresult);
  blasStatus = hipblasScasum_64(blasHandle, n_64, &complexx, incx_64, &fresult);
  blasStatus = hipblasScasum_64(blasHandle, n_64, &complexx, incx_64, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDzasum_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* x, int64_t incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dzasum_64(rocblas_handle handle, int64_t n, const rocblas_double_complex* x, int64_t incx, double* result);
  // CHECK: blasStatus = rocblas_dzasum_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_dzasum_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);
  blasStatus = hipblasDzasum_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);
  blasStatus = hipblasDzasum_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSaxpy_v2_64(cublasHandle_t handle, int64_t n, const float* alpha, const float* x, int64_t incx, float* y, int64_t incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_saxpy_64(rocblas_handle handle, int64_t n, const float* alpha, const float* x, int64_t incx, float* y, int64_t incy);
  // CHECK: blasStatus = rocblas_saxpy_64(blasHandle, n_64, &fa, &fx, incx_64, &fy, incy_64);
  // CHECK-NEXT: blasStatus = rocblas_saxpy_64(blasHandle, n_64, &fa, &fx, incx_64, &fy, incy_64);
  blasStatus = hipblasSaxpy_64(blasHandle, n_64, &fa, &fx, incx_64, &fy, incy_64);
  blasStatus = hipblasSaxpy_64(blasHandle, n_64, &fa, &fx, incx_64, &fy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDaxpy_v2_64(cublasHandle_t handle, int64_t n, const double* alpha, const double* x, int64_t incx, double* y, int64_t incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_daxpy_64(rocblas_handle handle, int64_t n, const double* alpha, const double* x, int64_t incx, double* y, int64_t incy);
  // CHECK: blasStatus = rocblas_daxpy_64(blasHandle, n_64, &da, &dx, incx_64, &dy, incy_64);
  // CHECK-NEXT: blasStatus = rocblas_daxpy_64(blasHandle, n_64, &da, &dx, incx_64, &dy, incy_64);
  blasStatus = hipblasDaxpy_64(blasHandle, n_64, &da, &dx, incx_64, &dy, incy_64);
  blasStatus = hipblasDaxpy_64(blasHandle, n_64, &da, &dx, incx_64, &dy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCaxpy_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* alpha, const cuComplex* x, int64_t incx, cuComplex* y, int64_t incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_caxpy_64(rocblas_handle handle, int64_t n, const rocblas_float_complex* alpha, const rocblas_float_complex* x, int64_t incx, rocblas_float_complex* y, int64_t incy);
  // CHECK: blasStatus = rocblas_caxpy_64(blasHandle, n_64, &complexa, &complexx, incx_64, &complexy, incy_64);
  // CHECK-NEXT: blasStatus = rocblas_caxpy_64(blasHandle, n_64, &complexa, &complexx, incx_64, &complexy, incy_64);
  blasStatus = hipblasCaxpy_64(blasHandle, n_64, &complexa, &complexx, incx_64, &complexy, incy_64);
  blasStatus = hipblasCaxpy_64(blasHandle, n_64, &complexa, &complexx, incx_64, &complexy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZaxpy_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* alpha, const cuDoubleComplex* x, int64_t incx, cuDoubleComplex* y, int64_t incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zaxpy_64(rocblas_handle handle, int64_t n, const rocblas_double_complex* alpha, const rocblas_double_complex* x, int64_t incx, rocblas_double_complex* y, int64_t incy);
  // CHECK: blasStatus = rocblas_zaxpy_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64, &dcomplexy, incy_64);
  // CHECK-NEXT: blasStatus = rocblas_zaxpy_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64, &dcomplexy, incy_64);
  blasStatus = hipblasZaxpy_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64, &dcomplexy, incy_64);
  blasStatus = hipblasZaxpy_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64, &dcomplexy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScopy_v2_64(cublasHandle_t handle, int64_t n, const float* x, int64_t incx, float* y, int64_t incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scopy_64(rocblas_handle handle, int64_t n, const float* x, int64_t incx, float* y, int64_t incy);
  // CHECK: blasStatus = rocblas_scopy_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);
  // CHECK-NEXT: blasStatus = rocblas_scopy_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);
  blasStatus = hipblasScopy_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);
  blasStatus = hipblasScopy_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDcopy_v2_64(cublasHandle_t handle, int64_t n, const double* x, int64_t incx, double* y, int64_t incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dcopy_64(rocblas_handle handle, int64_t n, const double* x, int64_t incx, double* y, int64_t incy);
  // CHECK: blasStatus = rocblas_dcopy_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);
  // CHECK-NEXT: blasStatus = rocblas_dcopy_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);
  blasStatus = hipblasDcopy_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);
  blasStatus = hipblasDcopy_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCcopy_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* x, int64_t incx, cuComplex* y, int64_t incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ccopy_64(rocblas_handle handle, int64_t n, const rocblas_float_complex* x, int64_t incx, rocblas_float_complex* y, int64_t incy);
  // CHECK: blasStatus = rocblas_ccopy_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);
  // CHECK-NEXT: blasStatus = rocblas_ccopy_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);
  blasStatus = hipblasCcopy_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);
  blasStatus = hipblasCcopy_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZcopy_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* x, int64_t incx, cuDoubleComplex* y, int64_t incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zcopy_64(rocblas_handle handle, int64_t n, const rocblas_double_complex* x, int64_t incx, rocblas_double_complex* y, int64_t incy);
  // CHECK: blasStatus = rocblas_zcopy_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);
  // CHECK-NEXT: blasStatus = rocblas_zcopy_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);
  blasStatus = hipblasZcopy_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);
  blasStatus = hipblasZcopy_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSdot_v2_64(cublasHandle_t handle, int64_t n, const float* x, int64_t incx, const float* y, int64_t incy, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sdot_64(rocblas_handle handle, int64_t n, const float* x, int64_t incx, const float* y, int64_t incy, float* result);
  // CHECK: blasStatus = rocblas_sdot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_sdot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fresult);
  blasStatus = hipblasSdot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fresult);
  blasStatus = hipblasSdot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDdot_v2_64(cublasHandle_t handle, int64_t n, const double* x, int64_t incx, const double* y, int64_t incy, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_ddot_64(rocblas_handle handle, int64_t n, const double* x, int64_t incx, const double* y, int64_t incy, double* result);
  // CHECK: blasStatus = rocblas_ddot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_ddot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dresult);
  blasStatus = hipblasDdot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dresult);
  blasStatus = hipblasDdot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdotc_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* x, int64_t incx, const cuComplex* y, int64_t incy, cuComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cdotc_64(rocblas_handle handle, int64_t n, const rocblas_float_complex* x, int64_t incx, const rocblas_float_complex* y, int64_t incy, rocblas_float_complex* result);
  // CHECK: blasStatus = rocblas_cdotc_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);
  // CHECK-NEXT: blasStatus = rocblas_cdotc_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);
  blasStatus = hipblasCdotc_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);
  blasStatus = hipblasCdotc_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCdotu_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* x, int64_t incx, const cuComplex* y, int64_t incy, cuComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cdotu_64(rocblas_handle handle, int64_t n, const rocblas_float_complex* x, int64_t incx, const rocblas_float_complex* y, int64_t incy, rocblas_float_complex* result);
  // CHECK: blasStatus = rocblas_cdotu_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);
  // CHECK-NEXT: blasStatus = rocblas_cdotu_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);
  blasStatus = hipblasCdotu_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);
  blasStatus = hipblasCdotu_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &complexresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdotc_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* x, int64_t incx, const cuDoubleComplex* y, int64_t incy, cuDoubleComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdotc_64(rocblas_handle handle, int64_t n, const rocblas_double_complex* x, int64_t incx, const rocblas_double_complex* y, int64_t incy, rocblas_double_complex* result);
  // CHECK: blasStatus = rocblas_zdotc_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);
  // CHECK-NEXT: blasStatus = rocblas_zdotc_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);
  blasStatus = hipblasZdotc_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);
  blasStatus = hipblasZdotc_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdotu_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* x, int64_t incx, const cuDoubleComplex* y, int64_t incy, cuDoubleComplex* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdotu_64(rocblas_handle handle, int64_t n, const rocblas_double_complex* x, int64_t incx, const rocblas_double_complex* y, int64_t incy, rocblas_double_complex* result);
  // CHECK: blasStatus = rocblas_zdotu_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);
  // CHECK-NEXT: blasStatus = rocblas_zdotu_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);
  blasStatus = hipblasZdotu_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);
  blasStatus = hipblasZdotu_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dcomplexresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSnrm2_v2_64(cublasHandle_t handle, int64_t n, const float* x, int64_t incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_snrm2_64(rocblas_handle handle, int64_t n, const float* x, int64_t incx, float* result);
  // CHECK: blasStatus = rocblas_snrm2_64(blasHandle, n_64, &fx, incx_64, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_snrm2_64(blasHandle, n_64, &fx, incx_64, &fresult);
  blasStatus = hipblasSnrm2_64(blasHandle, n_64, &fx, incx_64, &fresult);
  blasStatus = hipblasSnrm2_64(blasHandle, n_64, &fx, incx_64, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDnrm2_v2_64(cublasHandle_t handle, int64_t n, const double* x, int64_t incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dnrm2_64(rocblas_handle handle, int64_t n, const double* x, int64_t incx, double* result);
  // CHECK: blasStatus = rocblas_dnrm2_64(blasHandle, n_64, &dx, incx_64, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_dnrm2_64(blasHandle, n_64, &dx, incx_64, &dresult);
  blasStatus = hipblasDnrm2_64(blasHandle, n_64, &dx, incx_64, &dresult);
  blasStatus = hipblasDnrm2_64(blasHandle, n_64, &dx, incx_64, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScnrm2_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* x, int64_t incx, float* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scnrm2_64(rocblas_handle handle, int64_t n, const rocblas_float_complex* x, int64_t incx, float* result);
  // CHECK: blasStatus = rocblas_scnrm2_64(blasHandle, n_64, &complexx, incx_64, &fresult);
  // CHECK-NEXT: blasStatus = rocblas_scnrm2_64(blasHandle, n_64, &complexx, incx_64, &fresult);
  blasStatus = hipblasScnrm2_64(blasHandle, n_64, &complexx, incx_64, &fresult);
  blasStatus = hipblasScnrm2_64(blasHandle, n_64, &complexx, incx_64, &fresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDznrm2_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* x, int64_t incx, double* result);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dznrm2_64(rocblas_handle handle, int64_t n, const rocblas_double_complex* x, int64_t incx, double* result);
  // CHECK: blasStatus = rocblas_dznrm2_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);
  // CHECK-NEXT: blasStatus = rocblas_dznrm2_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);
  blasStatus = hipblasDznrm2_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);
  blasStatus = hipblasDznrm2_64(blasHandle, n_64, &dcomplexx, incx_64, &dresult);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrot_v2_64(cublasHandle_t handle, int64_t n, float* x, int64_t incx, float* y, int64_t incy, const float* c, const float* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_srot_64(rocblas_handle handle, int64_t n, float* x, int64_t incx, float* y, int64_t incy, const float* c, const float* s);
  // CHECK: blasStatus = rocblas_srot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fc, &fs);
  // CHECK-NEXT: blasStatus = rocblas_srot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fc, &fs);
  blasStatus = hipblasSrot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fc, &fs);
  blasStatus = hipblasSrot_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fc, &fs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrot_v2_64(cublasHandle_t handle, int64_t n, double* x, int64_t incx, double* y, int64_t incy, const double* c, const double* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_drot_64(rocblas_handle handle, int64_t n, double* x, int64_t incx, double* y, int64_t incy, const double* c, const double* s);
  // CHECK: blasStatus = rocblas_drot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dc, &ds);
  // CHECK-NEXT: blasStatus = rocblas_drot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dc, &ds);
  blasStatus = hipblasDrot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dc, &ds);
  blasStatus = hipblasDrot_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dc, &ds);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCrot_v2_64(cublasHandle_t handle, int64_t n, cuComplex* x, int64_t incx, cuComplex* y, int64_t incy, const float* c, const cuComplex* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_crot_64(rocblas_handle handle, int64_t n, rocblas_float_complex* x, int64_t incx, rocblas_float_complex* y, int64_t incy, const float* c, const rocblas_float_complex* s);
  // CHECK: blasStatus = rocblas_crot_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &complexs);
  // CHECK-NEXT: blasStatus = rocblas_crot_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &complexs);
  blasStatus = hipblasCrot_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &complexs);
  blasStatus = hipblasCrot_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &complexs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsrot_v2_64(cublasHandle_t handle, int64_t n, cuComplex* x, int64_t incx, cuComplex* y, int64_t incy, const float* c, const float* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csrot_64(rocblas_handle handle, int64_t n, rocblas_float_complex* x, int64_t incx, rocblas_float_complex* y, int64_t incy, const float* c, const float* s);
  // CHECK: blasStatus = rocblas_csrot_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &fs);
  // CHECK-NEXT: blasStatus = rocblas_csrot_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &fs);
  blasStatus = hipblasCsrot_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &fs);
  blasStatus = hipblasCsrot_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64, &fc, &fs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZrot_v2_64(cublasHandle_t handle, int64_t n, cuDoubleComplex* x, int64_t incx, cuDoubleComplex* y, int64_t incy, const double* c, const cuDoubleComplex* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zrot_64(rocblas_handle handle, int64_t n, rocblas_double_complex* x, int64_t incx, rocblas_double_complex* y, int64_t incy, const double* c, const rocblas_double_complex* s);
  // CHECK: blasStatus = rocblas_zrot_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &dcomplexs);
  // CHECK-NEXT: blasStatus = rocblas_zrot_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &dcomplexs);
  blasStatus = hipblasZrot_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &dcomplexs);
  blasStatus = hipblasZrot_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &dcomplexs);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdrot_v2_64(cublasHandle_t handle, int64_t n, cuDoubleComplex* x, int64_t incx, cuDoubleComplex* y, int64_t incy, const double* c, const double* s);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdrot_64(rocblas_handle handle, int64_t n, rocblas_double_complex* x, int64_t incx, rocblas_double_complex* y, int64_t incy, const double* c, const double* s);
  // CHECK: blasStatus = rocblas_zdrot_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &ds);
  // CHECK-NEXT: blasStatus = rocblas_zdrot_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &ds);
  blasStatus = hipblasZdrot_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &ds);
  blasStatus = hipblasZdrot_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64, &dc, &ds);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSrotm_v2_64(cublasHandle_t handle, int64_t n, float* x, int64_t incx, float* y, int64_t incy, const float* param);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_srotm_64(rocblas_handle handle, int64_t n, float* x, int64_t incx, float* y, int64_t incy, const float* param);
  // CHECK: blasStatus = rocblas_srotm_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fparam);
  // CHECK-NEXT: blasStatus = rocblas_srotm_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fparam);
  blasStatus = hipblasSrotm_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fparam);
  blasStatus = hipblasSrotm_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64, &fparam);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDrotm_v2_64(cublasHandle_t handle, int64_t n, double* x, int64_t incx, double* y, int64_t incy, const double* param);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_drotm_64(rocblas_handle handle, int64_t n, double* x, int64_t incx, double* y, int64_t incy, const double* param);
  // CHECK: blasStatus = rocblas_drotm_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dparam);
  // CHECK-NEXT: blasStatus = rocblas_drotm_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dparam);
  blasStatus = hipblasDrotm_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dparam);
  blasStatus = hipblasDrotm_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64, &dparam);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSscal_v2_64(cublasHandle_t handle, int64_t n, const float* alpha, float* x, int64_t incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sscal_64(rocblas_handle handle, int64_t n, const float* alpha, float* x, int64_t incx);
  // CHECK: blasStatus = rocblas_sscal_64(blasHandle, n_64, &fa, &fx, incx_64);
  // CHECK-NEXT: blasStatus = rocblas_sscal_64(blasHandle, n_64, &fa, &fx, incx_64);
  blasStatus = hipblasSscal_64(blasHandle, n_64, &fa, &fx, incx_64);
  blasStatus = hipblasSscal_64(blasHandle, n_64, &fa, &fx, incx_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDscal_v2_64(cublasHandle_t handle, int64_t n, const double* alpha, double* x, int64_t incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dscal_64(rocblas_handle handle, int64_t n, const double* alpha, double* x, int64_t incx);
  // CHECK: blasStatus = rocblas_dscal_64(blasHandle, n_64, &da, &dx, incx_64);
  // CHECK-NEXT: blasStatus = rocblas_dscal_64(blasHandle, n_64, &da, &dx, incx_64);
  blasStatus = hipblasDscal_64(blasHandle, n_64, &da, &dx, incx_64);
  blasStatus = hipblasDscal_64(blasHandle, n_64, &da, &dx, incx_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCscal_v2_64(cublasHandle_t handle, int64_t n, const cuComplex* alpha, cuComplex* x, int64_t incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cscal_64(rocblas_handle handle, int64_t n, const rocblas_float_complex* alpha, rocblas_float_complex* x, int64_t incx);
  // CHECK: blasStatus = rocblas_cscal_64(blasHandle, n_64, &complexa, &complexx, incx_64);
  // CHECK-NEXT: blasStatus = rocblas_cscal_64(blasHandle, n_64, &complexa, &complexx, incx_64);
  blasStatus = hipblasCscal_64(blasHandle, n_64, &complexa, &complexx, incx_64);
  blasStatus = hipblasCscal_64(blasHandle, n_64, &complexa, &complexx, incx_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCsscal_v2_64(cublasHandle_t handle, int64_t n, const float* alpha, cuComplex* x, int64_t incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_csscal_64(rocblas_handle handle, int64_t n, const float* alpha, rocblas_float_complex* x, int64_t incx);
  // CHECK: blasStatus = rocblas_csscal_64(blasHandle, n_64, &fa, &complexx, incx_64);
  // CHECK-NEXT: blasStatus = rocblas_csscal_64(blasHandle, n_64, &fa, &complexx, incx_64);
  blasStatus = hipblasCsscal_64(blasHandle, n_64, &fa, &complexx, incx_64);
  blasStatus = hipblasCsscal_64(blasHandle, n_64, &fa, &complexx, incx_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZscal_v2_64(cublasHandle_t handle, int64_t n, const cuDoubleComplex* alpha, cuDoubleComplex* x, int64_t incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zscal_64(rocblas_handle handle, int64_t n, const rocblas_double_complex* alpha, rocblas_double_complex* x, int64_t incx);
  // CHECK: blasStatus = rocblas_zscal_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64);
  // CHECK-NEXT: blasStatus = rocblas_zscal_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64);
  blasStatus = hipblasZscal_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64);
  blasStatus = hipblasZscal_64(blasHandle, n_64, &dcomplexa, &dcomplexx, incx_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZdscal_v2_64(cublasHandle_t handle, int64_t n, const double* alpha, cuDoubleComplex* x, int64_t incx);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zdscal_64(rocblas_handle handle, int64_t n, const double* alpha, rocblas_double_complex* x, int64_t incx);
  // CHECK: blasStatus = rocblas_zdscal_64(blasHandle, n_64, &da, &dcomplexx, incx_64);
  // CHECK-NEXT: blasStatus = rocblas_zdscal_64(blasHandle, n_64, &da, &dcomplexx, incx_64);
  blasStatus = hipblasZdscal_64(blasHandle, n_64, &da, &dcomplexx, incx_64);
  blasStatus = hipblasZdscal_64(blasHandle, n_64, &da, &dcomplexx, incx_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSswap_v2_64(cublasHandle_t handle, int64_t n, float* x, int64_t incx, float* y, int64_t incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_sswap_64(rocblas_handle handle, int64_t n, float* x, int64_t incx, float* y, int64_t incy);
  // CHECK: blasStatus = rocblas_sswap_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);
  // CHECK-NEXT: blasStatus = rocblas_sswap_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);
  blasStatus = hipblasSswap_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);
  blasStatus = hipblasSswap_64(blasHandle, n_64, &fx, incx_64, &fy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDswap_v2_64(cublasHandle_t handle, int64_t n, double* x, int64_t incx, double* y, int64_t incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dswap_64(rocblas_handle handle, int64_t n, double* x, int64_t incx, double* y, int64_t incy);
  // CHECK: blasStatus = rocblas_dswap_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);
  // CHECK-NEXT: blasStatus = rocblas_dswap_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);
  blasStatus = hipblasDswap_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);
  blasStatus = hipblasDswap_64(blasHandle, n_64, &dx, incx_64, &dy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasCswap_v2_64(cublasHandle_t handle, int64_t n, cuComplex* x, int64_t incx, cuComplex* y, int64_t incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_cswap_64(rocblas_handle handle, int64_t n, rocblas_float_complex* x, int64_t incx, rocblas_float_complex* y, int64_t incy);
  // CHECK: blasStatus = rocblas_cswap_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);
  // CHECK-NEXT: blasStatus = rocblas_cswap_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);
  blasStatus = hipblasCswap_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);
  blasStatus = hipblasCswap_64(blasHandle, n_64, &complexx, incx_64, &complexy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasZswap_v2_64(cublasHandle_t handle, int64_t n, cuDoubleComplex* x, int64_t incx, cuDoubleComplex* y, int64_t incy);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_zswap_64(rocblas_handle handle, int64_t n, rocblas_double_complex* x, int64_t incx, rocblas_double_complex* y, int64_t incy);
  // CHECK: blasStatus = rocblas_zswap_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);
  // CHECK-NEXT: blasStatus = rocblas_zswap_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);
  blasStatus = hipblasZswap_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);
  blasStatus = hipblasZswap_64(blasHandle, n_64, &dcomplexx, incx_64, &dcomplexy, incy_64);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasAxpyEx_64(cublasHandle_t handle, int64_t n, const void* alpha, cudaDataType alphaType, const void* x, cudaDataType xType, int64_t incx, void* y, cudaDataType yType, int64_t incy, cudaDataType executiontype);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_axpy_ex_64(rocblas_handle handle, int64_t n, const void* alpha, rocblas_datatype alpha_type, const void* x, rocblas_datatype x_type, int64_t incx, void* y, rocblas_datatype y_type, int64_t incy, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_axpy_ex_64(blasHandle, n_64, valpha, alpha_type, vx, x_type, incx_64, vy, y_type, incy_64, execution_type);
  blasStatus = hipblasAxpyEx_64(blasHandle, n_64, valpha, alpha_type, vx, x_type, incx_64, vy, y_type, incy_64, execution_type);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDotEx_64(cublasHandle_t handle, int64_t n, const void* x, cudaDataType xType, int64_t incx, const void* y, cudaDataType yType, int64_t incy, void* result, cudaDataType resultType, cudaDataType executionType);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dot_ex_64(rocblas_handle handle, int64_t n, const void* x, rocblas_datatype x_type, int64_t incx, const void* y, rocblas_datatype y_type, int64_t incy, void* result, rocblas_datatype result_type, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_dot_ex_64(blasHandle, n_64, vx, x_type, incx_64, vy, y_type, incy_64, vresult, result_type, execution_type);
  blasStatus = hipblasDotEx_64(blasHandle, n_64, vx, x_type, incx_64, vy, y_type, incy_64, vresult, result_type, execution_type);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasDotcEx_64(cublasHandle_t handle, int64_t n, const void* x, cudaDataType xType, int64_t incx, const void* y, cudaDataType yType, int64_t incy, void* result, cudaDataType resultType, cudaDataType executionType);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_dotc_ex_64(rocblas_handle handle, int64_t n, const void* x, rocblas_datatype x_type, int64_t incx, const void* y, rocblas_datatype y_type, int64_t incy, void* result, rocblas_datatype result_type, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_dotc_ex_64(blasHandle, n_64, vx, x_type, incx_64, vy, y_type, incy_64, vresult, result_type, execution_type);
  blasStatus = hipblasDotcEx_64(blasHandle, n_64, vx, x_type, incx_64, vy, y_type, incy_64, vresult, result_type, execution_type);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasNrm2Ex_64(cublasHandle_t handle, int64_t n, const void* x, cudaDataType xType, int64_t incx, void* result, cudaDataType resultType, cudaDataType executionType);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_nrm2_ex_64(rocblas_handle handle, int64_t n, const void* x, rocblas_datatype x_type, int64_t incx, void* results, rocblas_datatype result_type, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_nrm2_ex_64(blasHandle, n_64, vx, x_type, incx_64, vresult, result_type, execution_type);
  blasStatus = hipblasNrm2Ex_64(blasHandle, n_64, vx, x_type, incx_64, vresult, result_type, execution_type);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasRotEx_64(cublasHandle_t handle, int64_t n, void* x, cudaDataType xType, int64_t incx, void* y, cudaDataType yType, int64_t incy, const void* c, const void* s, cudaDataType csType, cudaDataType executiontype);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_rot_ex_64(rocblas_handle handle, int64_t n, void* x, rocblas_datatype x_type, int64_t incx, void* y, rocblas_datatype y_type, int64_t incy, const void* c, const void* s, rocblas_datatype cs_type, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_rot_ex_64(blasHandle, n_64, vx, x_type, incx_64, vy, y_type, incy_64, vc, vs, cs_type, execution_type);
  blasStatus = hipblasRotEx_64(blasHandle, n_64, vx, x_type, incx_64, vy, y_type, incy_64, vc, vs, cs_type, execution_type);

  // CUDA: CUBLASAPI cublasStatus_t CUBLASWINAPI cublasScalEx_64(cublasHandle_t handle, int64_t n, const void* alpha, cudaDataType alphaType, void* x, cudaDataType xType, int64_t incx, cudaDataType executionType);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_scal_ex_64(rocblas_handle handle, int64_t n, const void* alpha, rocblas_datatype alpha_type, void* x, rocblas_datatype x_type, int64_t incx, rocblas_datatype execution_type);
  // CHECK: blasStatus = rocblas_scal_ex_64(blasHandle, n_64, valpha, alpha_type, vx, x_type, incx_64, execution_type);
  blasStatus = hipblasScalEx_64(blasHandle, n_64, valpha, alpha_type, vx, x_type, incx_64, execution_type);
#endif

  return 0;
}
