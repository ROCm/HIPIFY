// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipblaslt.h"
#include "hipblaslt.h"
// CHECK-NOT: #include "hipblaslt.h"

int main() {
  printf("20. cuBLASLt API to hipBLASLt API synthetic test\n");

  // CHECK: hipblasLtHandle_t blasLtHandle;
  hipblasLtHandle_t blasLtHandle;

  // CHECK: hipblasStatus_t status;
  hipblasStatus_t status;

  const char *const_ch = nullptr;

#if CUDA_VERSION >= 10010
  // CUDA: cublasStatus_t CUBLASWINAPI cublasLtCreate(cublasLtHandle_t* lightHandle);
  // HIP: HIPBLASLT_EXPORT hipblasStatus_t hipblasLtCreate(hipblasLtHandle_t* handle);
  // CHECK: status = hipblasLtCreate(&blasLtHandle);
  status = hipblasLtCreate(&blasLtHandle);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasLtDestroy(cublasLtHandle_t lightHandle);
  // HIP: HIPBLASLT_EXPORT hipblasStatus_t hipblasLtDestroy(const hipblasLtHandle_t handle);
  // CHECK: status = hipblasLtDestroy(blasLtHandle);
  status = hipblasLtDestroy(blasLtHandle);
#endif

#if CUDA_VERSION >= 11000 && CUBLAS_VERSION >= 11000
  // CHECK: hipblasLtMatrixLayoutOpaque_t blasLtMatrixLayoutOpaque;
  hipblasLtMatrixLayoutOpaque_t blasLtMatrixLayoutOpaque;
#endif

  return 0;
}
