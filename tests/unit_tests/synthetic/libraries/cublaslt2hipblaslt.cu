// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipblaslt.h"
#include "hipblaslt.h"
// CHECK-NOT: #include "hipblaslt.h"

int main() {
  printf("20. cuBLASLt API to hipBLASLt API synthetic test\n");

  // CHECK: hipblasLtHandle_t blasLtHandle;
  hipblasLtHandle_t blasLtHandle;

  // CHECK: hipblasStatus_t status;
  hipblasStatus_t status;

  const char *const_ch = nullptr;

#if CUDA_VERSION >= 10010
  // CHECK: hipblasLtMatmulAlgo_t blasLtMatmulAlgo;
  hipblasLtMatmulAlgo_t blasLtMatmulAlgo;

  // CHECK: hipblasLtMatmulDesc_t blasLtMatmulDesc;
  hipblasLtMatmulDesc_t blasLtMatmulDesc;

  // CHECK: hipblasLtMatrixTransformDesc_t blasLtMatrixTransformDesc;
  hipblasLtMatrixTransformDesc_t blasLtMatrixTransformDesc;

  // CHECK: hipblasLtMatmulPreference_t blasLtMatmulPreference;
  hipblasLtMatmulPreference_t blasLtMatmulPreference;

  // CUDA: cublasStatus_t CUBLASWINAPI cublasLtCreate(cublasLtHandle_t* lightHandle);
  // HIP: HIPBLASLT_EXPORT hipblasStatus_t hipblasLtCreate(hipblasLtHandle_t* handle);
  // CHECK: status = hipblasLtCreate(&blasLtHandle);
  status = hipblasLtCreate(&blasLtHandle);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasLtDestroy(cublasLtHandle_t lightHandle);
  // HIP: HIPBLASLT_EXPORT hipblasStatus_t hipblasLtDestroy(const hipblasLtHandle_t handle);
  // CHECK: status = hipblasLtDestroy(blasLtHandle);
  status = hipblasLtDestroy(blasLtHandle);

#if CUBLAS_VERSION >= 10200
  // CHECK: hipblasLtPointerMode_t blasLtPointerMode;
  // CHECK-NEXT: hipblasLtPointerMode_t BLASLT_POINTER_MODE_HOST = HIPBLASLT_POINTER_MODE_HOST;
  // CHECK-NEXT: hipblasLtPointerMode_t BLASLT_POINTER_MODE_DEVICE = HIPBLASLT_POINTER_MODE_DEVICE;
  hipblasLtPointerMode_t blasLtPointerMode;
  hipblasLtPointerMode_t BLASLT_POINTER_MODE_HOST = HIPBLASLT_POINTER_MODE_HOST;
  hipblasLtPointerMode_t BLASLT_POINTER_MODE_DEVICE = HIPBLASLT_POINTER_MODE_DEVICE;
#endif
#endif

#if CUDA_VERSION >= 11000 && CUBLAS_VERSION >= 11000
  // CHECK: hipblasLtMatrixLayoutOpaque_t blasLtMatrixLayoutOpaque;
  hipblasLtMatrixLayoutOpaque_t blasLtMatrixLayoutOpaque;

  // CHECK: hipblasLtMatmulDescOpaque_t blasLtMatmulDescOpaque;
  hipblasLtMatmulDescOpaque_t blasLtMatmulDescOpaque;

  // CHECK: hipblasLtMatrixTransformDescOpaque_t blasLtMatrixTransformDescOpaque;
  hipblasLtMatrixTransformDescOpaque_t blasLtMatrixTransformDescOpaque;

  // CHECK: hipblasLtMatmulPreferenceOpaque_t blasLtMatmulPreferenceOpaque;
  hipblasLtMatmulPreferenceOpaque_t blasLtMatmulPreferenceOpaque;
#endif

#if CUDA_VERSION >= 11040 && CUBLAS_VERSION >= 11601
  // CHECK: hipblasLtPointerMode_t BLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_HOST = HIPBLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_HOST;
  hipblasLtPointerMode_t BLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_HOST = HIPBLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_HOST;
#endif
  return 0;
}
