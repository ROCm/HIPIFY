// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

int main() {
  printf("05. CUDA Runtime API Structs synthetic test\n");

  // CHECK: hipChannelFormatDesc ChannelFormatDesc;
  hipChannelFormatDesc ChannelFormatDesc;

  // CHECK: hipDeviceProp_t DeviceProp;
  hipDeviceProp_t DeviceProp;

  // CHECK: hipExtent Extent;
  hipExtent Extent;

  // CHECK: hipFuncAttributes FuncAttributes;
  hipFuncAttributes FuncAttributes;

  // CHECK: hipIpcEventHandle_st IpcEventHandle_st;
  // CHECK-NEXT: hipIpcEventHandle_t IpcEventHandle_t;
  hipIpcEventHandle_st IpcEventHandle_st;
  hipIpcEventHandle_t IpcEventHandle_t;

  // CHECK: hipIpcMemHandle_st IpcMemHandle_st;
  // CHECK-NEXT: hipIpcMemHandle_t IpcMemHandle_t;
  hipIpcMemHandle_st IpcMemHandle_st;
  hipIpcMemHandle_t IpcMemHandle_t;

  // CHECK: hipMemcpy3DParms Memcpy3DParms;
  hipMemcpy3DParms Memcpy3DParms;

  // CHECK: hipPitchedPtr PitchedPtr;
  hipPitchedPtr PitchedPtr;

  // CHECK: hipPointerAttribute_t PointerAttributes;
  hipPointerAttribute_t PointerAttributes;

  // CHECK: hipPos Pos;
  hipPos Pos;

  // CHECK: hipResourceDesc ResourceDesc;
  hipResourceDesc ResourceDesc;

  // CHECK: hipResourceViewDesc ResourceViewDesc;
  hipResourceViewDesc ResourceViewDesc;

  // CHECK: hipTextureDesc TextureDesc;
  hipTextureDesc TextureDesc;

  // CHECK: ihipEvent_t* event_st;
  // CHECK-NEXT: hipEvent_t Event_t;
  ihipEvent_t* event_st;
  hipEvent_t Event_t;

  // CHECK: hipGraphicsResource* GraphicsResource;
  // CHECK-NEXT: hipGraphicsResource_t GraphicsResource_t;
  hipGraphicsResource* GraphicsResource;
  hipGraphicsResource_t GraphicsResource_t;

  // CHECK: hipArray* Array;
  // CHECK-NEXT: hipArray_t Array_t;
  // CHECK-NEXT: hipArray_const_t Array_const_t;
  hipArray* Array;
  hipArray_t Array_t;
  hipArray_const_t Array_const_t;

  // CHECK: hipMipmappedArray* MipmappedArray;
  // CHECK-NEXT: hipMipmappedArray_t MipmappedArray_t;
  // CHECK-NEXT: hipMipmappedArray_const_t MipmappedArray_const_t;
  hipMipmappedArray* MipmappedArray;
  hipMipmappedArray_t MipmappedArray_t;
  hipMipmappedArray_const_t MipmappedArray_const_t;

  // CHECK: ihipStream_t* stream_st;
  // CHECK-NEXT: hipStream_t Stream_t;
  ihipStream_t* stream_st;
  hipStream_t Stream_t;

  // CHECK: ihipModuleSymbol_t* func_st_ptr;
  ihipModuleSymbol_t* func_st_ptr;

  // CHECK: hipUUID_t uuid_st;
  hipUUID_t uuid_st;

#if CUDA_VERSION >= 9000
  // CHECK: hipLaunchParams LaunchParams;
  hipLaunchParams LaunchParams;
#endif

#if CUDA_VERSION >= 10000
  // CHECK: hipExternalMemoryBufferDesc ExternalMemoryBufferDesc;
  hipExternalMemoryBufferDesc ExternalMemoryBufferDesc;

  // CHECK: hipExternalMemoryHandleDesc ExternalMemoryHandleDesc;
  hipExternalMemoryHandleDesc ExternalMemoryHandleDesc;

  // CHECK: hipExternalSemaphoreHandleDesc ExternalSemaphoreHandleDesc;
  hipExternalSemaphoreHandleDesc ExternalSemaphoreHandleDesc;

  // CHECK: hipExternalSemaphoreSignalParams ExternalSemaphoreSignalParams;
  hipExternalSemaphoreSignalParams ExternalSemaphoreSignalParams;

  // CHECK: hipExternalSemaphoreWaitParams ExternalSemaphoreWaitParams;
  hipExternalSemaphoreWaitParams ExternalSemaphoreWaitParams;

  // CHECK: hipHostNodeParams HostNodeParams;
  hipHostNodeParams HostNodeParams;

  // CHECK: hipKernelNodeParams KernelNodeParams;
  hipKernelNodeParams KernelNodeParams;

  // CHECK: hipMemsetParams MemsetParams;
  hipMemsetParams MemsetParams;

  // CHECK: hipExternalMemory_t ExternalMemory_t;
  hipExternalMemory_t ExternalMemory_t;

  // CHECK: hipExternalSemaphore_t ExternalSemaphore_t;
  hipExternalSemaphore_t ExternalSemaphore_t;

  // CHECK: ihipGraph* graph_st;
  // CHECK-NEXT: hipGraph_t Graph_t;
  ihipGraph* graph_st;
  hipGraph_t Graph_t;

  // CHECK: hipGraphExec* graphExec_st;
  // CHECK-NEXT: hipGraphExec_t GraphExec_t;
  hipGraphExec* graphExec_st;
  hipGraphExec_t GraphExec_t;

  // CHECK: hipGraphNode* graphNode_st;
  // CHECK-NEXT: hipGraphNode_t GraphNode_t;
  hipGraphNode* graphNode_st;
  hipGraphNode_t GraphNode_t;
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipFunction_t func;
  hipFunction_t func;

  // CHECK: hipAccessPolicyWindow AccessPolicyWindow;
  hipAccessPolicyWindow AccessPolicyWindow;
#endif

#if CUDA_VERSION >= 11020
  // CHECK: hipExternalSemaphoreSignalParams ExternalSemaphoreSignalParams_v1;
  hipExternalSemaphoreSignalParams ExternalSemaphoreSignalParams_v1;

  // CHECK: hipExternalSemaphoreWaitParams ExternalSemaphoreWaitParams_v1;
  hipExternalSemaphoreWaitParams ExternalSemaphoreWaitParams_v1;

  // CHECK: hipMemPool_t memPool_t;
  hipMemPool_t memPool_t;

  // CHECK: hipMemLocation memLocation;
  hipMemLocation memLocation;

  // CHECK: hipMemAccessDesc MemAccessDesc;
  hipMemAccessDesc MemAccessDesc;

  // CHECK: hipMemPoolProps MemPoolProps;
  hipMemPoolProps MemPoolProps;

  // CHECK: hipExternalSemaphoreSignalNodeParams ExternalSemaphoreSignalNodeParams;
  hipExternalSemaphoreSignalNodeParams ExternalSemaphoreSignalNodeParams;

  // CHECK: hipExternalSemaphoreWaitNodeParams ExternalSemaphoreWaitNodeParams;
  hipExternalSemaphoreWaitNodeParams ExternalSemaphoreWaitNodeParams;
#endif

#if CUDA_VERSION >= 11030
  // CHECK: hipMemPoolPtrExportData memPoolPtrExportData;
  hipMemPoolPtrExportData memPoolPtrExportData;

  // CHECK: hipUserObject_t userObject;
  hipUserObject_t userObject;
#endif

#if CUDA_VERSION >= 11040
  // CHECK: hipMemAllocNodeParams MemAllocNodeParams;
  hipMemAllocNodeParams MemAllocNodeParams;
#endif

#if CUDA_VERSION < 12000
  // CHECK: surfaceReference surfaceRef;
  surfaceReference surfaceRef;
#endif

#if CUDA_VERSION >= 12000
  // CHECK: hipGraphInstantiateParams GRAPH_INSTANTIATE_PARAMS_st;
  // CHECK-NEXT: hipGraphInstantiateParams GRAPH_INSTANTIATE_PARAMS;
  hipGraphInstantiateParams GRAPH_INSTANTIATE_PARAMS_st;
  hipGraphInstantiateParams GRAPH_INSTANTIATE_PARAMS;
#endif

#if CUDA_VERSION >= 12020
  // CHECK: hipExternalSemaphoreSignalNodeParams ExternalSemaphoreSignalNodeParams_v2;
  hipExternalSemaphoreSignalNodeParams ExternalSemaphoreSignalNodeParams_v2;

  // CHECK: hipExternalSemaphoreWaitNodeParams ExternalSemaphoreWaitNodeParams_v2;
  hipExternalSemaphoreWaitNodeParams ExternalSemaphoreWaitNodeParams_v2;

  // CHECK: hipMemFreeNodeParams MemFreeNodeParams;
  hipMemFreeNodeParams MemFreeNodeParams;

  // CHECK: hipChildGraphNodeParams ChildGraphNodeParams;
  hipChildGraphNodeParams ChildGraphNodeParams;

  // CHECK: hipEventRecordNodeParams EventRecordNodeParams;
  hipEventRecordNodeParams EventRecordNodeParams;

  // CHECK: hipEventWaitNodeParams EventWaitNodeParams;
  hipEventWaitNodeParams EventWaitNodeParams;

  // CHECK: hipGraphNodeParams *GraphNodeParams = nullptr
  hipGraphNodeParams *GraphNodeParams = nullptr;
#endif

  return 0;
}
