// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#if defined(_WIN32)
#include "windows.h"
#endif
#include "cuda_gl_interop.h"

int main() {
  printf("06. CUDA Runtime API Enums synthetic test\n");

  // CHECK: hipChannelFormatKind ChannelFormatKind;
  // CHECK-NEXT: hipChannelFormatKind ChannelFormatKindSigned = hipChannelFormatKindSigned;
  // CHECK-NEXT: hipChannelFormatKind ChannelFormatKindUnsigned = hipChannelFormatKindUnsigned;
  // CHECK-NEXT: hipChannelFormatKind ChannelFormatKindFloat = hipChannelFormatKindFloat;
  // CHECK-NEXT: hipChannelFormatKind ChannelFormatKindNone = hipChannelFormatKindNone;
  hipChannelFormatKind ChannelFormatKind;
  hipChannelFormatKind ChannelFormatKindSigned = hipChannelFormatKindSigned;
  hipChannelFormatKind ChannelFormatKindUnsigned = hipChannelFormatKindUnsigned;
  hipChannelFormatKind ChannelFormatKindFloat = hipChannelFormatKindFloat;
  hipChannelFormatKind ChannelFormatKindNone = hipChannelFormatKindNone;

  // CHECK: hipComputeMode ComputeMode;
  // CHECK-NEXT: hipComputeMode ComputeModeDefault = hipComputeModeDefault;
  // CHECK-NEXT: hipComputeMode ComputeModeExclusive = hipComputeModeExclusive;
  // CHECK-NEXT: hipComputeMode ComputeModeProhibited = hipComputeModeProhibited;
  // CHECK-NEXT: hipComputeMode ComputeModeExclusiveProcess = hipComputeModeExclusiveProcess;
  hipComputeMode ComputeMode;
  hipComputeMode ComputeModeDefault = hipComputeModeDefault;
  hipComputeMode ComputeModeExclusive = hipComputeModeExclusive;
  hipComputeMode ComputeModeProhibited = hipComputeModeProhibited;
  hipComputeMode ComputeModeExclusiveProcess = hipComputeModeExclusiveProcess;

  // CHECK: hipDeviceAttribute_t DeviceAttr;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxThreadsPerBlock = hipDeviceAttributeMaxThreadsPerBlock;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxBlockDimX = hipDeviceAttributeMaxBlockDimX;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxBlockDimY = hipDeviceAttributeMaxBlockDimY;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxBlockDimZ = hipDeviceAttributeMaxBlockDimZ;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxGridDimX = hipDeviceAttributeMaxGridDimX;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxGridDimY = hipDeviceAttributeMaxGridDimY;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxGridDimZ = hipDeviceAttributeMaxGridDimZ;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxSharedMemoryPerBlock = hipDeviceAttributeMaxSharedMemoryPerBlock;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrTotalConstantMemory = hipDeviceAttributeTotalConstantMemory;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrWarpSize = hipDeviceAttributeWarpSize;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxPitch = hipDeviceAttributeMaxPitch;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxRegistersPerBlock = hipDeviceAttributeMaxRegistersPerBlock;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrClockRate = hipDeviceAttributeClockRate;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrTextureAlignment = hipDeviceAttributeTextureAlignment;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrGpuOverlap = hipDeviceAttributeAsyncEngineCount;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMultiProcessorCount = hipDeviceAttributeMultiprocessorCount;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrKernelExecTimeout = hipDeviceAttributeKernelExecTimeout;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrIntegrated = hipDeviceAttributeIntegrated;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrCanMapHostMemory = hipDeviceAttributeCanMapHostMemory;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrComputeMode = hipDeviceAttributeComputeMode;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture1DWidth = hipDeviceAttributeMaxTexture1DWidth;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture2DWidth = hipDeviceAttributeMaxTexture2DWidth;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture2DHeight = hipDeviceAttributeMaxTexture2DHeight;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture3DWidth = hipDeviceAttributeMaxTexture3DWidth;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture3DHeight = hipDeviceAttributeMaxTexture3DHeight;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture3DDepth = hipDeviceAttributeMaxTexture3DDepth;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture2DLayeredWidth = hipDeviceAttributeMaxTexture2DLayered;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture2DLayeredHeight = hipDeviceAttributeMaxTexture2DLayered;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrSurfaceAlignment = hipDeviceAttributeSurfaceAlignment;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrConcurrentKernels = hipDeviceAttributeConcurrentKernels;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrEccEnabled = hipDeviceAttributeEccEnabled;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrPciBusId = hipDeviceAttributePciBusId;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrPciDeviceId = hipDeviceAttributePciDeviceId;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrTccDriver = hipDeviceAttributeTccDriver;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMemoryClockRate = hipDeviceAttributeMemoryClockRate;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrGlobalMemoryBusWidth = hipDeviceAttributeMemoryBusWidth;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrL2CacheSize = hipDeviceAttributeL2CacheSize;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxThreadsPerMultiProcessor = hipDeviceAttributeMaxThreadsPerMultiProcessor;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrAsyncEngineCount = hipDeviceAttributeAsyncEngineCount;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrUnifiedAddressing = hipDeviceAttributeUnifiedAddressing;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture1DLayeredWidth = hipDeviceAttributeMaxTexture1DLayered;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture2DGatherWidth = hipDeviceAttributeMaxTexture2DGather;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture2DGatherHeight = hipDeviceAttributeMaxTexture2DGather;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture3DWidthAlt = hipDeviceAttributeMaxTexture3DAlt;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture3DHeightAlt = hipDeviceAttributeMaxTexture3DAlt;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture3DDepthAlt = hipDeviceAttributeMaxTexture3DAlt;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrPciDomainId = hipDeviceAttributePciDomainID;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrTexturePitchAlignment = hipDeviceAttributeTexturePitchAlignment;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTextureCubemapWidth = hipDeviceAttributeMaxTextureCubemap;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTextureCubemapLayeredWidth = hipDeviceAttributeMaxTextureCubemapLayered;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxSurface1DWidth = hipDeviceAttributeMaxSurface1D;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxSurface2DWidth = hipDeviceAttributeMaxSurface2D;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxSurface2DHeight = hipDeviceAttributeMaxSurface2D;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxSurface3DWidth = hipDeviceAttributeMaxSurface3D;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxSurface3DHeight = hipDeviceAttributeMaxSurface3D;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxSurface3DDepth = hipDeviceAttributeMaxSurface3D;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxSurface1DLayeredWidth = hipDeviceAttributeMaxSurface1DLayered;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxSurface2DLayeredWidth = hipDeviceAttributeMaxSurface2DLayered;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxSurface2DLayeredHeight = hipDeviceAttributeMaxSurface2DLayered;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxSurfaceCubemapWidth = hipDeviceAttributeMaxSurfaceCubemap;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxSurfaceCubemapLayeredWidth = hipDeviceAttributeMaxSurfaceCubemapLayered;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture1DLinearWidth = hipDeviceAttributeMaxTexture1DLinear;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture2DLinearWidth = hipDeviceAttributeMaxTexture2DLinear;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture2DLinearHeight = hipDeviceAttributeMaxTexture2DLinear;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture2DLinearPitch = hipDeviceAttributeMaxTexture2DLinear;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture2DMipmappedWidth = hipDeviceAttributeMaxTexture2DMipmap;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture2DMipmappedHeight = hipDeviceAttributeMaxTexture2DMipmap;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrComputeCapabilityMajor = hipDeviceAttributeComputeCapabilityMajor;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrComputeCapabilityMinor = hipDeviceAttributeComputeCapabilityMinor;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxTexture1DMipmappedWidth = hipDeviceAttributeMaxTexture1DMipmap;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrStreamPrioritiesSupported = hipDeviceAttributeStreamPrioritiesSupported;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrGlobalL1CacheSupported = hipDeviceAttributeGlobalL1CacheSupported;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrLocalL1CacheSupported = hipDeviceAttributeLocalL1CacheSupported;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxSharedMemoryPerMultiprocessor = hipDeviceAttributeMaxSharedMemoryPerMultiprocessor;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxRegistersPerMultiprocessor = hipDeviceAttributeMaxRegistersPerMultiprocessor;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrManagedMemory = hipDeviceAttributeManagedMemory;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrIsMultiGpuBoard = hipDeviceAttributeIsMultiGpuBoard;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMultiGpuBoardGroupID = hipDeviceAttributeMultiGpuBoardGroupID;
  hipDeviceAttribute_t DeviceAttr;
  hipDeviceAttribute_t DevAttrMaxThreadsPerBlock = hipDeviceAttributeMaxThreadsPerBlock;
  hipDeviceAttribute_t DevAttrMaxBlockDimX = hipDeviceAttributeMaxBlockDimX;
  hipDeviceAttribute_t DevAttrMaxBlockDimY = hipDeviceAttributeMaxBlockDimY;
  hipDeviceAttribute_t DevAttrMaxBlockDimZ = hipDeviceAttributeMaxBlockDimZ;
  hipDeviceAttribute_t DevAttrMaxGridDimX = hipDeviceAttributeMaxGridDimX;
  hipDeviceAttribute_t DevAttrMaxGridDimY = hipDeviceAttributeMaxGridDimY;
  hipDeviceAttribute_t DevAttrMaxGridDimZ = hipDeviceAttributeMaxGridDimZ;
  hipDeviceAttribute_t DevAttrMaxSharedMemoryPerBlock = hipDeviceAttributeMaxSharedMemoryPerBlock;
  hipDeviceAttribute_t DevAttrTotalConstantMemory = hipDeviceAttributeTotalConstantMemory;
  hipDeviceAttribute_t DevAttrWarpSize = hipDeviceAttributeWarpSize;
  hipDeviceAttribute_t DevAttrMaxPitch = hipDeviceAttributeMaxPitch;
  hipDeviceAttribute_t DevAttrMaxRegistersPerBlock = hipDeviceAttributeMaxRegistersPerBlock;
  hipDeviceAttribute_t DevAttrClockRate = hipDeviceAttributeClockRate;
  hipDeviceAttribute_t DevAttrTextureAlignment = hipDeviceAttributeTextureAlignment;
  hipDeviceAttribute_t DevAttrGpuOverlap = hipDeviceAttributeAsyncEngineCount;
  hipDeviceAttribute_t DevAttrMultiProcessorCount = hipDeviceAttributeMultiprocessorCount;
  hipDeviceAttribute_t DevAttrKernelExecTimeout = hipDeviceAttributeKernelExecTimeout;
  hipDeviceAttribute_t DevAttrIntegrated = hipDeviceAttributeIntegrated;
  hipDeviceAttribute_t DevAttrCanMapHostMemory = hipDeviceAttributeCanMapHostMemory;
  hipDeviceAttribute_t DevAttrComputeMode = hipDeviceAttributeComputeMode;
  hipDeviceAttribute_t DevAttrMaxTexture1DWidth = hipDeviceAttributeMaxTexture1DWidth;
  hipDeviceAttribute_t DevAttrMaxTexture2DWidth = hipDeviceAttributeMaxTexture2DWidth;
  hipDeviceAttribute_t DevAttrMaxTexture2DHeight = hipDeviceAttributeMaxTexture2DHeight;
  hipDeviceAttribute_t DevAttrMaxTexture3DWidth = hipDeviceAttributeMaxTexture3DWidth;
  hipDeviceAttribute_t DevAttrMaxTexture3DHeight = hipDeviceAttributeMaxTexture3DHeight;
  hipDeviceAttribute_t DevAttrMaxTexture3DDepth = hipDeviceAttributeMaxTexture3DDepth;
  hipDeviceAttribute_t DevAttrMaxTexture2DLayeredWidth = hipDeviceAttributeMaxTexture2DLayered;
  hipDeviceAttribute_t DevAttrMaxTexture2DLayeredHeight = hipDeviceAttributeMaxTexture2DLayered;
  hipDeviceAttribute_t DevAttrSurfaceAlignment = hipDeviceAttributeSurfaceAlignment;
  hipDeviceAttribute_t DevAttrConcurrentKernels = hipDeviceAttributeConcurrentKernels;
  hipDeviceAttribute_t DevAttrEccEnabled = hipDeviceAttributeEccEnabled;
  hipDeviceAttribute_t DevAttrPciBusId = hipDeviceAttributePciBusId;
  hipDeviceAttribute_t DevAttrPciDeviceId = hipDeviceAttributePciDeviceId;
  hipDeviceAttribute_t DevAttrTccDriver = hipDeviceAttributeTccDriver;
  hipDeviceAttribute_t DevAttrMemoryClockRate = hipDeviceAttributeMemoryClockRate;
  hipDeviceAttribute_t DevAttrGlobalMemoryBusWidth = hipDeviceAttributeMemoryBusWidth;
  hipDeviceAttribute_t DevAttrL2CacheSize = hipDeviceAttributeL2CacheSize;
  hipDeviceAttribute_t DevAttrMaxThreadsPerMultiProcessor = hipDeviceAttributeMaxThreadsPerMultiProcessor;
  hipDeviceAttribute_t DevAttrAsyncEngineCount = hipDeviceAttributeAsyncEngineCount;
  hipDeviceAttribute_t DevAttrUnifiedAddressing = hipDeviceAttributeUnifiedAddressing;
  hipDeviceAttribute_t DevAttrMaxTexture1DLayeredWidth = hipDeviceAttributeMaxTexture1DLayered;
  hipDeviceAttribute_t DevAttrMaxTexture2DGatherWidth = hipDeviceAttributeMaxTexture2DGather;
  hipDeviceAttribute_t DevAttrMaxTexture2DGatherHeight = hipDeviceAttributeMaxTexture2DGather;
  hipDeviceAttribute_t DevAttrMaxTexture3DWidthAlt = hipDeviceAttributeMaxTexture3DAlt;
  hipDeviceAttribute_t DevAttrMaxTexture3DHeightAlt = hipDeviceAttributeMaxTexture3DAlt;
  hipDeviceAttribute_t DevAttrMaxTexture3DDepthAlt = hipDeviceAttributeMaxTexture3DAlt;
  hipDeviceAttribute_t DevAttrPciDomainId = hipDeviceAttributePciDomainId;
  hipDeviceAttribute_t DevAttrTexturePitchAlignment = hipDeviceAttributeTexturePitchAlignment;
  hipDeviceAttribute_t DevAttrMaxTextureCubemapWidth = hipDeviceAttributeMaxTextureCubemap;
  hipDeviceAttribute_t DevAttrMaxTextureCubemapLayeredWidth = hipDeviceAttributeMaxTextureCubemapLayered;
  hipDeviceAttribute_t DevAttrMaxSurface1DWidth = hipDeviceAttributeMaxSurface1D;
  hipDeviceAttribute_t DevAttrMaxSurface2DWidth = hipDeviceAttributeMaxSurface2D;
  hipDeviceAttribute_t DevAttrMaxSurface2DHeight = hipDeviceAttributeMaxSurface2D;
  hipDeviceAttribute_t DevAttrMaxSurface3DWidth = hipDeviceAttributeMaxSurface3D;
  hipDeviceAttribute_t DevAttrMaxSurface3DHeight = hipDeviceAttributeMaxSurface3D;
  hipDeviceAttribute_t DevAttrMaxSurface3DDepth = hipDeviceAttributeMaxSurface3D;
  hipDeviceAttribute_t DevAttrMaxSurface1DLayeredWidth = hipDeviceAttributeMaxSurface1DLayered;
  hipDeviceAttribute_t DevAttrMaxSurface2DLayeredWidth = hipDeviceAttributeMaxSurface2DLayered;
  hipDeviceAttribute_t DevAttrMaxSurface2DLayeredHeight = hipDeviceAttributeMaxSurface2DLayered;
  hipDeviceAttribute_t DevAttrMaxSurfaceCubemapWidth = hipDeviceAttributeMaxSurfaceCubemap;
  hipDeviceAttribute_t DevAttrMaxSurfaceCubemapLayeredWidth = hipDeviceAttributeMaxSurfaceCubemapLayered;
  hipDeviceAttribute_t DevAttrMaxTexture1DLinearWidth = hipDeviceAttributeMaxTexture1DLinear;
  hipDeviceAttribute_t DevAttrMaxTexture2DLinearWidth = hipDeviceAttributeMaxTexture2DLinear;
  hipDeviceAttribute_t DevAttrMaxTexture2DLinearHeight = hipDeviceAttributeMaxTexture2DLinear;
  hipDeviceAttribute_t DevAttrMaxTexture2DLinearPitch = hipDeviceAttributeMaxTexture2DLinear;
  hipDeviceAttribute_t DevAttrMaxTexture2DMipmappedWidth = hipDeviceAttributeMaxTexture2DMipmap;
  hipDeviceAttribute_t DevAttrMaxTexture2DMipmappedHeight = hipDeviceAttributeMaxTexture2DMipmap;
  hipDeviceAttribute_t DevAttrComputeCapabilityMajor = hipDeviceAttributeComputeCapabilityMajor;
  hipDeviceAttribute_t DevAttrComputeCapabilityMinor = hipDeviceAttributeComputeCapabilityMinor;
  hipDeviceAttribute_t DevAttrMaxTexture1DMipmappedWidth = hipDeviceAttributeMaxTexture1DMipmap;
  hipDeviceAttribute_t DevAttrStreamPrioritiesSupported = hipDeviceAttributeStreamPrioritiesSupported;
  hipDeviceAttribute_t DevAttrGlobalL1CacheSupported = hipDeviceAttributeGlobalL1CacheSupported;
  hipDeviceAttribute_t DevAttrLocalL1CacheSupported = hipDeviceAttributeLocalL1CacheSupported;
  hipDeviceAttribute_t DevAttrMaxSharedMemoryPerMultiprocessor = hipDeviceAttributeMaxSharedMemoryPerMultiprocessor;
  hipDeviceAttribute_t DevAttrMaxRegistersPerMultiprocessor = hipDeviceAttributeMaxRegistersPerMultiprocessor;
  hipDeviceAttribute_t DevAttrManagedMemory = hipDeviceAttributeManagedMemory;
  hipDeviceAttribute_t DevAttrIsMultiGpuBoard = hipDeviceAttributeIsMultiGpuBoard;
  hipDeviceAttribute_t DevAttrMultiGpuBoardGroupID = hipDeviceAttributeMultiGpuBoardGroupID;

  // CHECK: hipError_t Error;
  // CHECK-NEXT: hipError_t Error_t;
  // CHECK-NEXT: hipError_t Success = hipSuccess;
  // CHECK-NEXT: hipError_t ErrorInvalidValue = hipErrorInvalidValue;
  // CHECK-NEXT: hipError_t ErrorMemoryAllocation = hipErrorOutOfMemory;
  // CHECK-NEXT: hipError_t ErrorInitializationError = hipErrorNotInitialized;
  // CHECK-NEXT: hipError_t ErrorCudartUnloading = hipErrorDeinitialized;
  // CHECK-NEXT: hipError_t ErrorProfilerDisabled = hipErrorProfilerDisabled;
  // CHECK-NEXT: hipError_t ErrorProfilerNotInitialized = hipErrorProfilerNotInitialized;
  // CHECK-NEXT: hipError_t ErrorProfilerAlreadyStarted = hipErrorProfilerAlreadyStarted;
  // CHECK-NEXT: hipError_t ErrorProfilerAlreadyStopped = hipErrorProfilerAlreadyStopped;
  // CHECK-NEXT: hipError_t ErrorInvalidConfiguration = hipErrorInvalidConfiguration;
  // CHECK-NEXT: hipError_t ErrorInvalidPitchValue = hipErrorInvalidPitchValue;
  // CHECK-NEXT: hipError_t ErrorInvalidSymbol = hipErrorInvalidSymbol;
  // CHECK-NEXT: hipError_t ErrorInvalidDevicePointer = hipErrorInvalidDevicePointer;
  // CHECK-NEXT: hipError_t ErrorInvalidMemcpyDirection = hipErrorInvalidMemcpyDirection;
  // CHECK-NEXT: hipError_t ErrorInsufficientDriver = hipErrorInsufficientDriver;
  // CHECK-NEXT: hipError_t ErrorMissingConfiguration = hipErrorMissingConfiguration;
  // CHECK-NEXT: hipError_t ErrorPriorLaunchFailure = hipErrorPriorLaunchFailure;
  // CHECK-NEXT: hipError_t ErrorInvalidDeviceFunction = hipErrorInvalidDeviceFunction;
  // CHECK-NEXT: hipError_t ErrorNoDevice = hipErrorNoDevice;
  // CHECK-NEXT: hipError_t ErrorInvalidDevice = hipErrorInvalidDevice;
  // CHECK-NEXT: hipError_t ErrorInvalidKernelImage = hipErrorInvalidImage;
  // CHECK-NEXT: hipError_t ErrorMapBufferObjectFailed = hipErrorMapFailed;
  // CHECK-NEXT: hipError_t ErrorUnmapBufferObjectFailed = hipErrorUnmapFailed;
  // CHECK-NEXT: hipError_t ErrorNoKernelImageForDevice = hipErrorNoBinaryForGpu;
  // CHECK-NEXT: hipError_t ErrorECCUncorrectable = hipErrorECCNotCorrectable;
  // CHECK-NEXT: hipError_t ErrorUnsupportedLimit = hipErrorUnsupportedLimit;
  // CHECK-NEXT: hipError_t ErrorDeviceAlreadyInUse = hipErrorContextAlreadyInUse;
  // CHECK-NEXT: hipError_t ErrorPeerAccessUnsupported = hipErrorPeerAccessUnsupported;
  // CHECK-NEXT: hipError_t ErrorInvalidPtx = hipErrorInvalidKernelFile;
  // CHECK-NEXT: hipError_t ErrorInvalidGraphicsContext = hipErrorInvalidGraphicsContext;
  // CHECK-NEXT: hipError_t ErrorSharedObjectSymbolNotFound = hipErrorSharedObjectSymbolNotFound;
  // CHECK-NEXT: hipError_t ErrorSharedObjectInitFailed = hipErrorSharedObjectInitFailed;
  // CHECK-NEXT: hipError_t ErrorOperatingSystem = hipErrorOperatingSystem;
  // CHECK-NEXT: hipError_t ErrorInvalidResourceHandle = hipErrorInvalidHandle;
  // CHECK-NEXT: hipError_t ErrorNotReady = hipErrorNotReady;
  // CHECK-NEXT: hipError_t ErrorIllegalAddress = hipErrorIllegalAddress;
  // CHECK-NEXT: hipError_t ErrorLaunchOutOfResources = hipErrorLaunchOutOfResources;
  // CHECK-NEXT: hipError_t ErrorLaunchTimeout = hipErrorLaunchTimeOut;
  // CHECK-NEXT: hipError_t ErrorPeerAccessAlreadyEnabled = hipErrorPeerAccessAlreadyEnabled;
  // CHECK-NEXT: hipError_t ErrorPeerAccessNotEnabled = hipErrorPeerAccessNotEnabled;
  // CHECK-NEXT: hipError_t ErrorSetOnActiveProcess = hipErrorSetOnActiveProcess;
  // CHECK-NEXT: hipError_t ErrorAssert = hipErrorAssert;
  // CHECK-NEXT: hipError_t ErrorHostMemoryAlreadyRegistered = hipErrorHostMemoryAlreadyRegistered;
  // CHECK-NEXT: hipError_t ErrorHostMemoryNotRegistered = hipErrorHostMemoryNotRegistered;
  // CHECK-NEXT: hipError_t ErrorLaunchFailure = hipErrorLaunchFailure;
  // CHECK-NEXT: hipError_t ErrorNotSupported = hipErrorNotSupported;
  hipError_t Error;
  hipError_t Error_t;
  hipError_t Success = hipSuccess;
  hipError_t ErrorInvalidValue = hipErrorInvalidValue;
  hipError_t ErrorMemoryAllocation = hipErrorOutOfMemory;
  hipError_t ErrorInitializationError = hipErrorNotInitialized;
  hipError_t ErrorCudartUnloading = hipErrorDeinitialized;
  hipError_t ErrorProfilerDisabled = hipErrorProfilerDisabled;
  hipError_t ErrorProfilerNotInitialized = hipErrorProfilerNotInitialized;
  hipError_t ErrorProfilerAlreadyStarted = hipErrorProfilerAlreadyStarted;
  hipError_t ErrorProfilerAlreadyStopped = hipErrorProfilerAlreadyStopped;
  hipError_t ErrorInvalidConfiguration = hipErrorInvalidConfiguration;
  hipError_t ErrorInvalidPitchValue = hipErrorInvalidPitchValue;
  hipError_t ErrorInvalidSymbol = hipErrorInvalidSymbol;
  hipError_t ErrorInvalidDevicePointer = hipErrorInvalidDevicePointer;
  hipError_t ErrorInvalidMemcpyDirection = hipErrorInvalidMemcpyDirection;
  hipError_t ErrorInsufficientDriver = hipErrorInsufficientDriver;
  hipError_t ErrorMissingConfiguration = hipErrorMissingConfiguration;
  hipError_t ErrorPriorLaunchFailure = hipErrorPriorLaunchFailure;
  hipError_t ErrorInvalidDeviceFunction = hipErrorInvalidDeviceFunction;
  hipError_t ErrorNoDevice = hipErrorNoDevice;
  hipError_t ErrorInvalidDevice = hipErrorInvalidDevice;
  hipError_t ErrorInvalidKernelImage = hipErrorInvalidImage;
  hipError_t ErrorMapBufferObjectFailed = hipErrorMapFailed;
  hipError_t ErrorUnmapBufferObjectFailed = hipErrorUnmapFailed;
  hipError_t ErrorNoKernelImageForDevice = hipErrorNoBinaryForGpu;
  hipError_t ErrorECCUncorrectable = hipErrorECCNotCorrectable;
  hipError_t ErrorUnsupportedLimit = hipErrorUnsupportedLimit;
  hipError_t ErrorDeviceAlreadyInUse = hipErrorContextAlreadyInUse;
  hipError_t ErrorPeerAccessUnsupported = hipErrorPeerAccessUnsupported;
  hipError_t ErrorInvalidPtx = hipErrorInvalidKernelFile;
  hipError_t ErrorInvalidGraphicsContext = hipErrorInvalidGraphicsContext;
  hipError_t ErrorSharedObjectSymbolNotFound = hipErrorSharedObjectSymbolNotFound;
  hipError_t ErrorSharedObjectInitFailed = hipErrorSharedObjectInitFailed;
  hipError_t ErrorOperatingSystem = hipErrorOperatingSystem;
  hipError_t ErrorInvalidResourceHandle = hipErrorInvalidHandle;
  hipError_t ErrorNotReady = hipErrorNotReady;
  hipError_t ErrorIllegalAddress = hipErrorIllegalAddress;
  hipError_t ErrorLaunchOutOfResources = hipErrorLaunchOutOfResources;
  hipError_t ErrorLaunchTimeout = hipErrorLaunchTimeOut;
  hipError_t ErrorPeerAccessAlreadyEnabled = hipErrorPeerAccessAlreadyEnabled;
  hipError_t ErrorPeerAccessNotEnabled = hipErrorPeerAccessNotEnabled;
  hipError_t ErrorSetOnActiveProcess = hipErrorSetOnActiveProcess;
  hipError_t ErrorAssert = hipErrorAssert;
  hipError_t ErrorHostMemoryAlreadyRegistered = hipErrorHostMemoryAlreadyRegistered;
  hipError_t ErrorHostMemoryNotRegistered = hipErrorHostMemoryNotRegistered;
  hipError_t ErrorLaunchFailure = hipErrorLaunchFailure;
  hipError_t ErrorNotSupported = hipErrorNotSupported;

  // CHECK: hipError_t ErrorUnknown = hipErrorUnknown;
  hipError_t ErrorUnknown = hipErrorUnknown;

  // CHECK: hipFuncCache_t FuncCache;
  // CHECK-NEXT: hipFuncCache_t FuncCachePreferNone = hipFuncCachePreferNone;
  // CHECK-NEXT: hipFuncCache_t FuncCachePreferShared = hipFuncCachePreferShared;
  // CHECK-NEXT: hipFuncCache_t FuncCachePreferL1 = hipFuncCachePreferL1;
  // CHECK-NEXT: hipFuncCache_t FuncCachePreferEqual = hipFuncCachePreferEqual;
  hipFuncCache_t FuncCache;
  hipFuncCache_t FuncCachePreferNone = hipFuncCachePreferNone;
  hipFuncCache_t FuncCachePreferShared = hipFuncCachePreferShared;
  hipFuncCache_t FuncCachePreferL1 = hipFuncCachePreferL1;
  hipFuncCache_t FuncCachePreferEqual = hipFuncCachePreferEqual;

  // CHECK: hipGraphicsRegisterFlags GraphicsRegisterFlags;
  // CHECK-NEXT: hipGraphicsRegisterFlags GraphicsRegisterFlagsNone = hipGraphicsRegisterFlagsNone;
  // CHECK-NEXT: hipGraphicsRegisterFlags GraphicsRegisterFlagsReadOnly = hipGraphicsRegisterFlagsReadOnly;
  // CHECK-NEXT: hipGraphicsRegisterFlags GraphicsRegisterFlagsWriteDiscard = hipGraphicsRegisterFlagsWriteDiscard;
  // CHECK-NEXT: hipGraphicsRegisterFlags GraphicsRegisterFlagsSurfaceLoadStore = hipGraphicsRegisterFlagsSurfaceLoadStore;
  // CHECK-NEXT: hipGraphicsRegisterFlags GraphicsRegisterFlagsTextureGather = hipGraphicsRegisterFlagsTextureGather;
  hipGraphicsRegisterFlags GraphicsRegisterFlags;
  hipGraphicsRegisterFlags GraphicsRegisterFlagsNone = hipGraphicsRegisterFlagsNone;
  hipGraphicsRegisterFlags GraphicsRegisterFlagsReadOnly = hipGraphicsRegisterFlagsReadOnly;
  hipGraphicsRegisterFlags GraphicsRegisterFlagsWriteDiscard = hipGraphicsRegisterFlagsWriteDiscard;
  hipGraphicsRegisterFlags GraphicsRegisterFlagsSurfaceLoadStore = hipGraphicsRegisterFlagsSurfaceLoadStore;
  hipGraphicsRegisterFlags GraphicsRegisterFlagsTextureGather = hipGraphicsRegisterFlagsTextureGather;

  // CHECK: hipLimit_t Limit;
  // CHECK-NEXT: hipLimit_t LimitStackSize = hipLimitStackSize;
  // CHECK-NEXT: hipLimit_t LimitPrintfFifoSize = hipLimitPrintfFifoSize;
  // CHECK-NEXT: hipLimit_t LimitMallocHeapSize = hipLimitMallocHeapSize;
  hipLimit_t Limit;
  hipLimit_t LimitStackSize = hipLimitStackSize;
  hipLimit_t LimitPrintfFifoSize = hipLimitPrintfFifoSize;
  hipLimit_t LimitMallocHeapSize = hipLimitMallocHeapSize;

  // CHECK: hipMemcpyKind MemcpyKind;
  // CHECK-NEXT: hipMemcpyKind MemcpyHostToHost = hipMemcpyHostToHost;
  // CHECK-NEXT: hipMemcpyKind MemcpyHostToDevice = hipMemcpyHostToDevice;
  // CHECK-NEXT: hipMemcpyKind MemcpyDeviceToHost = hipMemcpyDeviceToHost;
  // CHECK-NEXT: hipMemcpyKind MemcpyDeviceToDevice = hipMemcpyDeviceToDevice;
  // CHECK-NEXT: hipMemcpyKind MemcpyDefault = hipMemcpyDefault;
  hipMemcpyKind MemcpyKind;
  hipMemcpyKind MemcpyHostToHost = hipMemcpyHostToHost;
  hipMemcpyKind MemcpyHostToDevice = hipMemcpyHostToDevice;
  hipMemcpyKind MemcpyDeviceToHost = hipMemcpyDeviceToHost;
  hipMemcpyKind MemcpyDeviceToDevice = hipMemcpyDeviceToDevice;
  hipMemcpyKind MemcpyDefault = hipMemcpyDefault;

  // CHECK: hipMemoryType MemoryType;
  // CHECK-NEXT: hipMemoryType MemoryTypeHost = hipMemoryTypeHost;
  // CHECK-NEXT: hipMemoryType MemoryTypeDevice = hipMemoryTypeDevice;
  hipMemoryType MemoryType;
  hipMemoryType MemoryTypeHost = hipMemoryTypeHost;
  hipMemoryType MemoryTypeDevice = hipMemoryTypeDevice;

  // CHECK: hipResourceType ResourceType;
  // CHECK-NEXT: hipResourceType ResourceTypeArray = hipResourceTypeArray;
  // CHECK-NEXT: hipResourceType ResourceTypeMipmappedArray = hipResourceTypeMipmappedArray;
  // CHECK-NEXT: hipResourceType ResourceTypeLinear = hipResourceTypeLinear;
  // CHECK-NEXT: hipResourceType ResourceTypePitch2D = hipResourceTypePitch2D;
  hipResourceType ResourceType;
  hipResourceType ResourceTypeArray = hipResourceTypeArray;
  hipResourceType ResourceTypeMipmappedArray = hipResourceTypeMipmappedArray;
  hipResourceType ResourceTypeLinear = hipResourceTypeLinear;
  hipResourceType ResourceTypePitch2D = hipResourceTypePitch2D;

  // CHECK: hipResourceViewFormat ResourceViewFormat;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatNone = hipResViewFormatNone;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedChar1 = hipResViewFormatUnsignedChar1;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedChar2 = hipResViewFormatUnsignedChar2;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedChar4 = hipResViewFormatUnsignedChar4;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatSignedChar1 = hipResViewFormatSignedChar1;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatSignedChar2 = hipResViewFormatSignedChar2;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatSignedChar4 = hipResViewFormatSignedChar4;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedShort1 = hipResViewFormatUnsignedShort1;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedShort2 = hipResViewFormatUnsignedShort2;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedShort4 = hipResViewFormatUnsignedShort4;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatSignedShort1 = hipResViewFormatSignedShort1;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatSignedShort2 = hipResViewFormatSignedShort2;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatSignedShort4 = hipResViewFormatSignedShort4;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedInt1 = hipResViewFormatUnsignedInt1;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedInt2 = hipResViewFormatUnsignedInt2;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedInt4 = hipResViewFormatUnsignedInt4;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatSignedInt1 = hipResViewFormatSignedInt1;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatSignedInt2 = hipResViewFormatSignedInt2;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatSignedInt4 = hipResViewFormatSignedInt4;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatHalf1 = hipResViewFormatHalf1;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatHalf2 = hipResViewFormatHalf2;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatHalf4 = hipResViewFormatHalf4;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatFloat1 = hipResViewFormatFloat1;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatFloat2 = hipResViewFormatFloat2;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatFloat4 = hipResViewFormatFloat4;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedBlockCompressed1 = hipResViewFormatUnsignedBlockCompressed1;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedBlockCompressed2 = hipResViewFormatUnsignedBlockCompressed2;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedBlockCompressed3 = hipResViewFormatUnsignedBlockCompressed3;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedBlockCompressed4 = hipResViewFormatUnsignedBlockCompressed4;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatSignedBlockCompressed4 = hipResViewFormatSignedBlockCompressed4;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedBlockCompressed5 = hipResViewFormatUnsignedBlockCompressed5;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatSignedBlockCompressed5 = hipResViewFormatSignedBlockCompressed5;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedBlockCompressed6H = hipResViewFormatUnsignedBlockCompressed6H;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatSignedBlockCompressed6H = hipResViewFormatSignedBlockCompressed6H;
  // CHECK-NEXT: hipResourceViewFormat ResViewFormatUnsignedBlockCompressed7 = hipResViewFormatUnsignedBlockCompressed7;
  hipResourceViewFormat ResourceViewFormat;
  hipResourceViewFormat ResViewFormatNone = hipResViewFormatNone;
  hipResourceViewFormat ResViewFormatUnsignedChar1 = hipResViewFormatUnsignedChar1;
  hipResourceViewFormat ResViewFormatUnsignedChar2 = hipResViewFormatUnsignedChar2;
  hipResourceViewFormat ResViewFormatUnsignedChar4 = hipResViewFormatUnsignedChar4;
  hipResourceViewFormat ResViewFormatSignedChar1 = hipResViewFormatSignedChar1;
  hipResourceViewFormat ResViewFormatSignedChar2 = hipResViewFormatSignedChar2;
  hipResourceViewFormat ResViewFormatSignedChar4 = hipResViewFormatSignedChar4;
  hipResourceViewFormat ResViewFormatUnsignedShort1 = hipResViewFormatUnsignedShort1;
  hipResourceViewFormat ResViewFormatUnsignedShort2 = hipResViewFormatUnsignedShort2;
  hipResourceViewFormat ResViewFormatUnsignedShort4 = hipResViewFormatUnsignedShort4;
  hipResourceViewFormat ResViewFormatSignedShort1 = hipResViewFormatSignedShort1;
  hipResourceViewFormat ResViewFormatSignedShort2 = hipResViewFormatSignedShort2;
  hipResourceViewFormat ResViewFormatSignedShort4 = hipResViewFormatSignedShort4;
  hipResourceViewFormat ResViewFormatUnsignedInt1 = hipResViewFormatUnsignedInt1;
  hipResourceViewFormat ResViewFormatUnsignedInt2 = hipResViewFormatUnsignedInt2;
  hipResourceViewFormat ResViewFormatUnsignedInt4 = hipResViewFormatUnsignedInt4;
  hipResourceViewFormat ResViewFormatSignedInt1 = hipResViewFormatSignedInt1;
  hipResourceViewFormat ResViewFormatSignedInt2 = hipResViewFormatSignedInt2;
  hipResourceViewFormat ResViewFormatSignedInt4 = hipResViewFormatSignedInt4;
  hipResourceViewFormat ResViewFormatHalf1 = hipResViewFormatHalf1;
  hipResourceViewFormat ResViewFormatHalf2 = hipResViewFormatHalf2;
  hipResourceViewFormat ResViewFormatHalf4 = hipResViewFormatHalf4;
  hipResourceViewFormat ResViewFormatFloat1 = hipResViewFormatFloat1;
  hipResourceViewFormat ResViewFormatFloat2 = hipResViewFormatFloat2;
  hipResourceViewFormat ResViewFormatFloat4 = hipResViewFormatFloat4;
  hipResourceViewFormat ResViewFormatUnsignedBlockCompressed1 = hipResViewFormatUnsignedBlockCompressed1;
  hipResourceViewFormat ResViewFormatUnsignedBlockCompressed2 = hipResViewFormatUnsignedBlockCompressed2;
  hipResourceViewFormat ResViewFormatUnsignedBlockCompressed3 = hipResViewFormatUnsignedBlockCompressed3;
  hipResourceViewFormat ResViewFormatUnsignedBlockCompressed4 = hipResViewFormatUnsignedBlockCompressed4;
  hipResourceViewFormat ResViewFormatSignedBlockCompressed4 = hipResViewFormatSignedBlockCompressed4;
  hipResourceViewFormat ResViewFormatUnsignedBlockCompressed5 = hipResViewFormatUnsignedBlockCompressed5;
  hipResourceViewFormat ResViewFormatSignedBlockCompressed5 = hipResViewFormatSignedBlockCompressed5;
  hipResourceViewFormat ResViewFormatUnsignedBlockCompressed6H = hipResViewFormatUnsignedBlockCompressed6H;
  hipResourceViewFormat ResViewFormatSignedBlockCompressed6H = hipResViewFormatSignedBlockCompressed6H;
  hipResourceViewFormat ResViewFormatUnsignedBlockCompressed7 = hipResViewFormatUnsignedBlockCompressed7;

  // CHECK: hipSharedMemConfig SharedMemConfig;
  // CHECK-NEXT: hipSharedMemConfig SharedMemBankSizeDefault = hipSharedMemBankSizeDefault;
  // CHECK-NEXT: hipSharedMemConfig SharedMemBankSizeFourByte = hipSharedMemBankSizeFourByte;
  // CHECK-NEXT: hipSharedMemConfig SharedMemBankSizeEightByte = hipSharedMemBankSizeEightByte;
  hipSharedMemConfig SharedMemConfig;
  hipSharedMemConfig SharedMemBankSizeDefault = hipSharedMemBankSizeDefault;
  hipSharedMemConfig SharedMemBankSizeFourByte = hipSharedMemBankSizeFourByte;
  hipSharedMemConfig SharedMemBankSizeEightByte = hipSharedMemBankSizeEightByte;

  // CHECK: hipSurfaceBoundaryMode SurfaceBoundaryMode;
  // CHECK-NEXT: hipSurfaceBoundaryMode BoundaryModeZero = hipBoundaryModeZero;
  // CHECK-NEXT: hipSurfaceBoundaryMode BoundaryModeClamp = hipBoundaryModeClamp;
  // CHECK-NEXT: hipSurfaceBoundaryMode BoundaryModeTrap = hipBoundaryModeTrap;
  hipSurfaceBoundaryMode SurfaceBoundaryMode;
  hipSurfaceBoundaryMode BoundaryModeZero = hipBoundaryModeZero;
  hipSurfaceBoundaryMode BoundaryModeClamp = hipBoundaryModeClamp;
  hipSurfaceBoundaryMode BoundaryModeTrap = hipBoundaryModeTrap;

  // CHECK: hipTextureAddressMode TextureAddressMode;
  // CHECK-NEXT: hipTextureAddressMode AddressModeWrap = hipAddressModeWrap;
  // CHECK-NEXT: hipTextureAddressMode AddressModeClamp = hipAddressModeClamp;
  // CHECK-NEXT: hipTextureAddressMode AddressModeMirror = hipAddressModeMirror;
  // CHECK-NEXT: hipTextureAddressMode AddressModeBorder = hipAddressModeBorder;
  hipTextureAddressMode TextureAddressMode;
  hipTextureAddressMode AddressModeWrap = hipAddressModeWrap;
  hipTextureAddressMode AddressModeClamp = hipAddressModeClamp;
  hipTextureAddressMode AddressModeMirror = hipAddressModeMirror;
  hipTextureAddressMode AddressModeBorder = hipAddressModeBorder;

  // CHECK: hipTextureFilterMode TextureFilterMode;
  // CHECK-NEXT: hipTextureFilterMode FilterModePoint = hipFilterModePoint;
  // CHECK-NEXT: hipTextureFilterMode FilterModeLinear = hipFilterModeLinear;
  hipTextureFilterMode TextureFilterMode;
  hipTextureFilterMode FilterModePoint = hipFilterModePoint;
  hipTextureFilterMode FilterModeLinear = hipFilterModeLinear;

  // CHECK: hipTextureReadMode TextureReadMode;
  // CHECK-NEXT: hipTextureReadMode ReadModeElementType = hipReadModeElementType;
  // CHECK-NEXT: hipTextureReadMode ReadModeNormalizedFloat = hipReadModeNormalizedFloat;
  hipTextureReadMode TextureReadMode;
  hipTextureReadMode ReadModeElementType = hipReadModeElementType;
  hipTextureReadMode ReadModeNormalizedFloat = hipReadModeNormalizedFloat;

  // CHECK: hipGLDeviceList GLDeviceList;
  // CHECK-NEXT: hipGLDeviceList GLDeviceListAll = hipGLDeviceListAll;
  // CHECK-NEXT: hipGLDeviceList GLDeviceListCurrentFrame = hipGLDeviceListCurrentFrame;
  // CHECK-NEXT: hipGLDeviceList GLDeviceListNextFrame = hipGLDeviceListNextFrame;
  hipGLDeviceList GLDeviceList;
  hipGLDeviceList GLDeviceListAll = hipGLDeviceListAll;
  hipGLDeviceList GLDeviceListCurrentFrame = hipGLDeviceListCurrentFrame;
  hipGLDeviceList GLDeviceListNextFrame = hipGLDeviceListNextFrame;

#if CUDA_VERSION >= 8000
  // CHECK: hipDeviceAttribute_t DevAttrHostNativeAtomicSupported = hipDeviceAttributeHostNativeAtomicSupported;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrSingleToDoublePrecisionPerfRatio = hipDeviceAttributeSingleToDoublePrecisionPerfRatio;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrPageableMemoryAccess = hipDeviceAttributePageableMemoryAccess;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrConcurrentManagedAccess = hipDeviceAttributeConcurrentManagedAccess;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrComputePreemptionSupported = hipDeviceAttributeComputePreemptionSupported;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrCanUseHostPointerForRegisteredMem = hipDeviceAttributeCanUseHostPointerForRegisteredMem;
  hipDeviceAttribute_t DevAttrHostNativeAtomicSupported = hipDeviceAttributeHostNativeAtomicSupported;
  hipDeviceAttribute_t DevAttrSingleToDoublePrecisionPerfRatio = hipDeviceAttributeSingleToDoublePrecisionPerfRatio;
  hipDeviceAttribute_t DevAttrPageableMemoryAccess = hipDeviceAttributePageableMemoryAccess;
  hipDeviceAttribute_t DevAttrConcurrentManagedAccess = hipDeviceAttributeConcurrentManagedAccess;
  hipDeviceAttribute_t DevAttrComputePreemptionSupported = hipDeviceAttributeComputePreemptionSupported;
  hipDeviceAttribute_t DevAttrCanUseHostPointerForRegisteredMem = hipDeviceAttributeCanUseHostPointerForRegisteredMem;

  // CHECK: hipDeviceP2PAttr DeviceP2PAttr;
  // CHECK-NEXT: hipDeviceP2PAttr DevP2PAttrPerformanceRank = hipDevP2PAttrPerformanceRank;
  // CHECK-NEXT: hipDeviceP2PAttr DevP2PAttrAccessSupported = hipDevP2PAttrAccessSupported;
  // CHECK-NEXT: hipDeviceP2PAttr DevP2PAttrNativeAtomicSupported = hipDevP2PAttrNativeAtomicSupported;
  hipDeviceP2PAttr DeviceP2PAttr;
  hipDeviceP2PAttr DevP2PAttrPerformanceRank = hipDevP2PAttrPerformanceRank;
  hipDeviceP2PAttr DevP2PAttrAccessSupported = hipDevP2PAttrAccessSupported;
  hipDeviceP2PAttr DevP2PAttrNativeAtomicSupported = hipDevP2PAttrNativeAtomicSupported;

  // CHECK: hipMemoryAdvise MemoryAdvise;
  // CHECK-NEXT: hipMemoryAdvise MemAdviseSetReadMostly = hipMemAdviseSetReadMostly;
  // CHECK-NEXT: hipMemoryAdvise MemAdviseUnsetReadMostly = hipMemAdviseUnsetReadMostly;
  // CHECK-NEXT: hipMemoryAdvise MemAdviseSetPreferredLocation = hipMemAdviseSetPreferredLocation;
  // CHECK-NEXT: hipMemoryAdvise MemAdviseUnsetPreferredLocation = hipMemAdviseUnsetPreferredLocation;
  // CHECK-NEXT: hipMemoryAdvise MemAdviseSetAccessedBy = hipMemAdviseSetAccessedBy;
  // CHECK-NEXT: hipMemoryAdvise MemAdviseUnsetAccessedBy = hipMemAdviseUnsetAccessedBy;
  hipMemoryAdvise MemoryAdvise;
  hipMemoryAdvise MemAdviseSetReadMostly = hipMemAdviseSetReadMostly;
  hipMemoryAdvise MemAdviseUnsetReadMostly = hipMemAdviseUnsetReadMostly;
  hipMemoryAdvise MemAdviseSetPreferredLocation = hipMemAdviseSetPreferredLocation;
  hipMemoryAdvise MemAdviseUnsetPreferredLocation = hipMemAdviseUnsetPreferredLocation;
  hipMemoryAdvise MemAdviseSetAccessedBy = hipMemAdviseSetAccessedBy;
  hipMemoryAdvise MemAdviseUnsetAccessedBy = hipMemAdviseUnsetAccessedBy;

  // CHECK: hipMemRangeAttribute MemRangeAttribute;
  // CHECK-NEXT: hipMemRangeAttribute MemRangeAttributeReadMostly = hipMemRangeAttributeReadMostly;
  // CHECK-NEXT: hipMemRangeAttribute MemRangeAttributePreferredLocation = hipMemRangeAttributePreferredLocation;
  // CHECK-NEXT: hipMemRangeAttribute MemRangeAttributeAccessedBy = hipMemRangeAttributeAccessedBy;
  // CHECK-NEXT: hipMemRangeAttribute MemRangeAttributeLastPrefetchLocation = hipMemRangeAttributeLastPrefetchLocation;
  hipMemRangeAttribute MemRangeAttribute;
  hipMemRangeAttribute MemRangeAttributeReadMostly = hipMemRangeAttributeReadMostly;
  hipMemRangeAttribute MemRangeAttributePreferredLocation = hipMemRangeAttributePreferredLocation;
  hipMemRangeAttribute MemRangeAttributeAccessedBy = hipMemRangeAttributeAccessedBy;
  hipMemRangeAttribute MemRangeAttributeLastPrefetchLocation = hipMemRangeAttributeLastPrefetchLocation;
#endif

#if CUDA_VERSION >= 9000
  // CHECK: hipDeviceAttribute_t DevAttrReserved94 = hipDeviceAttributeCanUseStreamWaitValue;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrCooperativeLaunch = hipDeviceAttributeCooperativeLaunch;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrCooperativeMultiDeviceLaunch = hipDeviceAttributeCooperativeMultiDeviceLaunch;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrMaxSharedMemoryPerBlockOptin = hipDeviceAttributeSharedMemPerBlockOptin;
  hipDeviceAttribute_t DevAttrReserved94 = hipDeviceAttributeCanUseStreamWaitValue;
  hipDeviceAttribute_t DevAttrCooperativeLaunch = hipDeviceAttributeCooperativeLaunch;
  hipDeviceAttribute_t DevAttrCooperativeMultiDeviceLaunch = hipDeviceAttributeCooperativeMultiDeviceLaunch;
  hipDeviceAttribute_t DevAttrMaxSharedMemoryPerBlockOptin = hipDeviceAttributeSharedMemPerBlockOptin;

  // CHECK: hipError_t ErrorCooperativeLaunchTooLarge = hipErrorCooperativeLaunchTooLarge;
  hipError_t ErrorCooperativeLaunchTooLarge = hipErrorCooperativeLaunchTooLarge;

  // CHECK: hipFuncAttribute FuncAttribute;
  // CHECK-NEXT: hipFuncAttribute FuncAttributeMaxDynamicSharedMemorySize = hipFuncAttributeMaxDynamicSharedMemorySize;
  // CHECK-NEXT: hipFuncAttribute FuncAttributePreferredSharedMemoryCarveout = hipFuncAttributePreferredSharedMemoryCarveout;
  // CHECK-NEXT: hipFuncAttribute FuncAttributeMax = hipFuncAttributeMax;
  hipFuncAttribute FuncAttribute;
  hipFuncAttribute FuncAttributeMaxDynamicSharedMemorySize = hipFuncAttributeMaxDynamicSharedMemorySize;
  hipFuncAttribute FuncAttributePreferredSharedMemoryCarveout = hipFuncAttributePreferredSharedMemoryCarveout;
  hipFuncAttribute FuncAttributeMax = hipFuncAttributeMax;
#endif

#if CUDA_VERSION >= 9020
  // CHECK: hipDeviceAttribute_t DevAttrPageableMemoryAccessUsesHostPageTables = hipDeviceAttributePageableMemoryAccessUsesHostPageTables;
  // CHECK-NEXT: hipDeviceAttribute_t DevAttrDirectManagedMemAccessFromHost = hipDeviceAttributeDirectManagedMemAccessFromHost;
  hipDeviceAttribute_t DevAttrPageableMemoryAccessUsesHostPageTables = hipDeviceAttributePageableMemoryAccessUsesHostPageTables;
  hipDeviceAttribute_t DevAttrDirectManagedMemAccessFromHost = hipDeviceAttributeDirectManagedMemAccessFromHost;

  // CHECK: hipDeviceP2PAttr DevP2PAttrCudaArrayAccessSupported = hipDevP2PAttrHipArrayAccessSupported;
  hipDeviceP2PAttr DevP2PAttrCudaArrayAccessSupported = hipDevP2PAttrHipArrayAccessSupported;

  // CHECK: hipDeviceAttribute_t DevAttrHostRegisterSupported = hipDeviceAttributeHostRegisterSupported;
  hipDeviceAttribute_t DevAttrHostRegisterSupported = hipDeviceAttributeHostRegisterSupported;
#endif

#if CUDA_VERSION >= 10000
  // CHECK: hipError_t ErrorStreamCaptureUnsupported = hipErrorStreamCaptureUnsupported;
  // CHECK-NEXT: hipError_t ErrorStreamCaptureInvalidated = hipErrorStreamCaptureInvalidated;
  // CHECK-NEXT: hipError_t ErrorStreamCaptureMerge = hipErrorStreamCaptureMerge;
  // CHECK-NEXT: hipError_t ErrorStreamCaptureUnmatched = hipErrorStreamCaptureUnmatched;
  // CHECK-NEXT: hipError_t ErrorStreamCaptureUnjoined = hipErrorStreamCaptureUnjoined;
  // CHECK-NEXT: hipError_t ErrorStreamCaptureIsolation = hipErrorStreamCaptureIsolation;
  // CHECK-NEXT: hipError_t ErrorStreamCaptureImplicit = hipErrorStreamCaptureImplicit;
  // CHECK-NEXT: hipError_t ErrorCapturedEvent = hipErrorCapturedEvent;
  // CHECK-NEXT: hipError_t ErrorIllegalState = hipErrorIllegalState;
  hipError_t ErrorStreamCaptureUnsupported = hipErrorStreamCaptureUnsupported;
  hipError_t ErrorStreamCaptureInvalidated = hipErrorStreamCaptureInvalidated;
  hipError_t ErrorStreamCaptureMerge = hipErrorStreamCaptureMerge;
  hipError_t ErrorStreamCaptureUnmatched = hipErrorStreamCaptureUnmatched;
  hipError_t ErrorStreamCaptureUnjoined = hipErrorStreamCaptureUnjoined;
  hipError_t ErrorStreamCaptureIsolation = hipErrorStreamCaptureIsolation;
  hipError_t ErrorStreamCaptureImplicit = hipErrorStreamCaptureImplicit;
  hipError_t ErrorCapturedEvent = hipErrorCapturedEvent;
  hipError_t ErrorIllegalState = hipErrorIllegalState;

  // CHECK: hipExternalMemoryHandleType ExternalMemoryHandleType;
  // CHECK-NEXT: hipExternalMemoryHandleType ExternalMemoryHandleTypeOpaqueFd = hipExternalMemoryHandleTypeOpaqueFd;
  // CHECK-NEXT: hipExternalMemoryHandleType ExternalMemoryHandleTypeOpaqueWin32 = hipExternalMemoryHandleTypeOpaqueWin32;
  // CHECK-NEXT: hipExternalMemoryHandleType ExternalMemoryHandleTypeOpaqueWin32Kmt = hipExternalMemoryHandleTypeOpaqueWin32Kmt;
  // CHECK-NEXT: hipExternalMemoryHandleType ExternalMemoryHandleTypeD3D12Heap = hipExternalMemoryHandleTypeD3D12Heap;
  // CHECK-NEXT: hipExternalMemoryHandleType ExternalMemoryHandleTypeD3D12Resource = hipExternalMemoryHandleTypeD3D12Resource;
  hipExternalMemoryHandleType ExternalMemoryHandleType;
  hipExternalMemoryHandleType ExternalMemoryHandleTypeOpaqueFd = hipExternalMemoryHandleTypeOpaqueFd;
  hipExternalMemoryHandleType ExternalMemoryHandleTypeOpaqueWin32 = hipExternalMemoryHandleTypeOpaqueWin32;
  hipExternalMemoryHandleType ExternalMemoryHandleTypeOpaqueWin32Kmt = hipExternalMemoryHandleTypeOpaqueWin32Kmt;
  hipExternalMemoryHandleType ExternalMemoryHandleTypeD3D12Heap = hipExternalMemoryHandleTypeD3D12Heap;
  hipExternalMemoryHandleType ExternalMemoryHandleTypeD3D12Resource = hipExternalMemoryHandleTypeD3D12Resource;

  // CHECK: hipExternalSemaphoreHandleType ExternalSemaphoreHandleType;
  // CHECK-NEXT: hipExternalSemaphoreHandleType ExternalSemaphoreHandleTypeOpaqueFd = hipExternalSemaphoreHandleTypeOpaqueFd;
  // CHECK-NEXT: hipExternalSemaphoreHandleType ExternalSemaphoreHandleTypeOpaqueWin32 = hipExternalSemaphoreHandleTypeOpaqueWin32;
  // CHECK-NEXT: hipExternalSemaphoreHandleType ExternalSemaphoreHandleTypeOpaqueWin32Kmt = hipExternalSemaphoreHandleTypeOpaqueWin32Kmt;
  // CHECK-NEXT: hipExternalSemaphoreHandleType ExternalSemaphoreHandleTypeD3D12Fence = hipExternalSemaphoreHandleTypeD3D12Fence;
  hipExternalSemaphoreHandleType ExternalSemaphoreHandleType;
  hipExternalSemaphoreHandleType ExternalSemaphoreHandleTypeOpaqueFd = hipExternalSemaphoreHandleTypeOpaqueFd;
  hipExternalSemaphoreHandleType ExternalSemaphoreHandleTypeOpaqueWin32 = hipExternalSemaphoreHandleTypeOpaqueWin32;
  hipExternalSemaphoreHandleType ExternalSemaphoreHandleTypeOpaqueWin32Kmt = hipExternalSemaphoreHandleTypeOpaqueWin32Kmt;
  hipExternalSemaphoreHandleType ExternalSemaphoreHandleTypeD3D12Fence = hipExternalSemaphoreHandleTypeD3D12Fence;

  // CHECK: hipGraphNodeType GraphNodeType;
  // CHECK-NEXT: hipGraphNodeType GraphNodeTypeKernel = hipGraphNodeTypeKernel;
  // CHECK-NEXT: hipGraphNodeType GraphNodeTypeMemcpy = hipGraphNodeTypeMemcpy;
  // CHECK-NEXT: hipGraphNodeType GraphNodeTypeMemset = hipGraphNodeTypeMemset;
  // CHECK-NEXT: hipGraphNodeType GraphNodeTypeHost = hipGraphNodeTypeHost;
  // CHECK-NEXT: hipGraphNodeType GraphNodeTypeGraph = hipGraphNodeTypeGraph;
  // CHECK-NEXT: hipGraphNodeType GraphNodeTypeEmpty = hipGraphNodeTypeEmpty;
  hipGraphNodeType GraphNodeType;
  hipGraphNodeType GraphNodeTypeKernel = hipGraphNodeTypeKernel;
  hipGraphNodeType GraphNodeTypeMemcpy = hipGraphNodeTypeMemcpy;
  hipGraphNodeType GraphNodeTypeMemset = hipGraphNodeTypeMemset;
  hipGraphNodeType GraphNodeTypeHost = hipGraphNodeTypeHost;
  hipGraphNodeType GraphNodeTypeGraph = hipGraphNodeTypeGraph;
  hipGraphNodeType GraphNodeTypeEmpty = hipGraphNodeTypeEmpty;

  // CHECK: hipGraphNodeType GraphNodeTypeCount = hipGraphNodeTypeCount;
  hipGraphNodeType GraphNodeTypeCount = hipGraphNodeTypeCount;

  // CHECK: hipMemoryType MemoryTypeManaged = hipMemoryTypeManaged;
  hipMemoryType MemoryTypeManaged = hipMemoryTypeManaged;

  // CHECK: hipStreamCaptureStatus StreamCaptureStatus;
  // CHECK-NEXT: hipStreamCaptureStatus StreamCaptureStatusNone = hipStreamCaptureStatusNone;
  // CHECK-NEXT: hipStreamCaptureStatus StreamCaptureStatusActive = hipStreamCaptureStatusActive;
  // CHECK-NEXT: hipStreamCaptureStatus StreamCaptureStatusInvalidated = hipStreamCaptureStatusInvalidated;
  hipStreamCaptureStatus StreamCaptureStatus;
  hipStreamCaptureStatus StreamCaptureStatusNone = hipStreamCaptureStatusNone;
  hipStreamCaptureStatus StreamCaptureStatusActive = hipStreamCaptureStatusActive;
  hipStreamCaptureStatus StreamCaptureStatusInvalidated = hipStreamCaptureStatusInvalidated;
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipError_t ErrorArrayIsMapped = hipErrorArrayIsMapped;
  // CHECK-NEXT: hipError_t ErrorAlreadyMapped = hipErrorAlreadyMapped;
  // CHECK-NEXT: hipError_t ErrorAlreadyAcquired = hipErrorAlreadyAcquired;
  // CHECK-NEXT: hipError_t ErrorNotMapped = hipErrorNotMapped;
  // CHECK-NEXT: hipError_t ErrorNotMappedAsArray = hipErrorNotMappedAsArray;
  // CHECK-NEXT: hipError_t ErrorNotMappedAsPointer = hipErrorNotMappedAsPointer;
  // CHECK-NEXT: hipError_t ErrorInvalidSource = hipErrorInvalidSource;
  // CHECK-NEXT: hipError_t ErrorFileNotFound = hipErrorFileNotFound;
  // CHECK-NEXT: hipError_t ErrorSymbolNotFound = hipErrorNotFound;
  // CHECK-NEXT: hipError_t ErrorContextIsDestroyed = hipErrorContextIsDestroyed;
  // CHECK-NEXT: hipError_t ErrorStreamCaptureWrongThread = hipErrorStreamCaptureWrongThread;
  hipError_t ErrorArrayIsMapped = hipErrorArrayIsMapped;
  hipError_t ErrorAlreadyMapped = hipErrorAlreadyMapped;
  hipError_t ErrorAlreadyAcquired = hipErrorAlreadyAcquired;
  hipError_t ErrorNotMapped = hipErrorNotMapped;
  hipError_t ErrorNotMappedAsArray = hipErrorNotMappedAsArray;
  hipError_t ErrorNotMappedAsPointer = hipErrorNotMappedAsPointer;
  hipError_t ErrorInvalidSource = hipErrorInvalidSource;
  hipError_t ErrorFileNotFound = hipErrorFileNotFound;
  hipError_t ErrorSymbolNotFound = hipErrorNotFound;
  hipError_t ErrorContextIsDestroyed = hipErrorContextIsDestroyed;
  hipError_t ErrorStreamCaptureWrongThread = hipErrorStreamCaptureWrongThread;

  // CHECK: hipStreamCaptureMode StreamCaptureMode;
  // CHECK-NEXT: hipStreamCaptureMode StreamCaptureModeGlobal = hipStreamCaptureModeGlobal;
  // CHECK-NEXT: hipStreamCaptureMode StreamCaptureModeThreadLocal = hipStreamCaptureModeThreadLocal;
  // CHECK-NEXT: hipStreamCaptureMode StreamCaptureModeRelaxed = hipStreamCaptureModeRelaxed;
  hipStreamCaptureMode StreamCaptureMode;
  hipStreamCaptureMode StreamCaptureModeGlobal = hipStreamCaptureModeGlobal;
  hipStreamCaptureMode StreamCaptureModeThreadLocal = hipStreamCaptureModeThreadLocal;
  hipStreamCaptureMode StreamCaptureModeRelaxed = hipStreamCaptureModeRelaxed;
#endif

#if CUDA_VERSION >= 10020
  // CHECK: hipError_t ErrorDeviceUninitialized = hipErrorInvalidContext;
  // CHECK: hipError_t ErrorGraphExecUpdateFailure = hipErrorGraphExecUpdateFailure;
  hipError_t ErrorDeviceUninitialized = hipErrorInvalidContext;
  hipError_t ErrorGraphExecUpdateFailure = hipErrorGraphExecUpdateFailure;

  // CHECK: hipExternalMemoryHandleType ExternalMemoryHandleTypeD3D11Resource = hipExternalMemoryHandleTypeD3D11Resource;
  // CHECK-NEXT: hipExternalMemoryHandleType ExternalMemoryHandleTypeD3D11ResourceKmt = hipExternalMemoryHandleTypeD3D11ResourceKmt;
  hipExternalMemoryHandleType ExternalMemoryHandleTypeD3D11Resource = hipExternalMemoryHandleTypeD3D11Resource;
  hipExternalMemoryHandleType ExternalMemoryHandleTypeD3D11ResourceKmt = hipExternalMemoryHandleTypeD3D11ResourceKmt;

  // CHECK: hipGraphExecUpdateResult GraphExecUpdateResult;
  // CHECK-NEXT: hipGraphExecUpdateResult GraphExecUpdateSuccess = hipGraphExecUpdateSuccess;
  // CHECK-NEXT: hipGraphExecUpdateResult GraphExecUpdateError = hipGraphExecUpdateError;
  // CHECK-NEXT: hipGraphExecUpdateResult GraphExecUpdateErrorTopologyChanged = hipGraphExecUpdateErrorTopologyChanged;
  // CHECK-NEXT: hipGraphExecUpdateResult GraphExecUpdateErrorNodeTypeChanged = hipGraphExecUpdateErrorNodeTypeChanged;
  // CHECK-NEXT: hipGraphExecUpdateResult GraphExecUpdateErrorFunctionChanged = hipGraphExecUpdateErrorFunctionChanged;
  // CHECK-NEXT: hipGraphExecUpdateResult GraphExecUpdateErrorParametersChanged = hipGraphExecUpdateErrorParametersChanged;
  // CHECK-NEXT: hipGraphExecUpdateResult GraphExecUpdateErrorNotSupported = hipGraphExecUpdateErrorNotSupported;
  hipGraphExecUpdateResult GraphExecUpdateResult;
  hipGraphExecUpdateResult GraphExecUpdateSuccess = hipGraphExecUpdateSuccess;
  hipGraphExecUpdateResult GraphExecUpdateError = hipGraphExecUpdateError;
  hipGraphExecUpdateResult GraphExecUpdateErrorTopologyChanged = hipGraphExecUpdateErrorTopologyChanged;
  hipGraphExecUpdateResult GraphExecUpdateErrorNodeTypeChanged = hipGraphExecUpdateErrorNodeTypeChanged;
  hipGraphExecUpdateResult GraphExecUpdateErrorFunctionChanged = hipGraphExecUpdateErrorFunctionChanged;
  hipGraphExecUpdateResult GraphExecUpdateErrorParametersChanged = hipGraphExecUpdateErrorParametersChanged;
  hipGraphExecUpdateResult GraphExecUpdateErrorNotSupported = hipGraphExecUpdateErrorNotSupported;
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipDeviceAttribute_t DevAttrMaxBlocksPerMultiprocessor = hipDeviceAttributeMaxBlocksPerMultiprocessor;
  hipDeviceAttribute_t DevAttrMaxBlocksPerMultiprocessor = hipDeviceAttributeMaxBlocksPerMultiprocessor;

  // CHECK: hipKernelNodeAttrID kernelNodeAttrID;
  // CHECK-NEXT: hipKernelNodeAttrID KernelNodeAttributeAccessPolicyWindow = hipKernelNodeAttributeAccessPolicyWindow;
  // CHECK-NEXT: hipKernelNodeAttrID KernelNodeAttributeCooperative = hipKernelNodeAttributeCooperative;
  hipKernelNodeAttrID kernelNodeAttrID;
  hipKernelNodeAttrID KernelNodeAttributeAccessPolicyWindow = hipKernelNodeAttributeAccessPolicyWindow;
  hipKernelNodeAttrID KernelNodeAttributeCooperative = hipKernelNodeAttributeCooperative;

  // CHECK: hipAccessProperty accessProperty;
  // CHECK-NEXT: hipAccessProperty AccessPropertyNormal = hipAccessPropertyNormal;
  // CHECK-NEXT: hipAccessProperty AccessPropertyStreaming = hipAccessPropertyStreaming;
  // CHECK-NEXT: hipAccessProperty AccessPropertyPersisting = hipAccessPropertyPersisting;
  hipAccessProperty accessProperty;
  hipAccessProperty AccessPropertyNormal = hipAccessPropertyNormal;
  hipAccessProperty AccessPropertyStreaming = hipAccessPropertyStreaming;
  hipAccessProperty AccessPropertyPersisting = hipAccessPropertyPersisting;
#endif

#if CUDA_VERSION >= 11010
  // CHECK: hipGraphNodeType GraphNodeTypeWaitEvent = hipGraphNodeTypeWaitEvent;
  // CHECK-NEXT: hipGraphNodeType GraphNodeTypeEventRecord = hipGraphNodeTypeEventRecord;
  hipGraphNodeType GraphNodeTypeWaitEvent = hipGraphNodeTypeWaitEvent;
  hipGraphNodeType GraphNodeTypeEventRecord = hipGraphNodeTypeEventRecord;
#endif

#if CUDA_VERSION >= 11020
  // CHECK: hipDeviceAttribute_t DevAttrMemoryPoolsSupported = hipDeviceAttributeMemoryPoolsSupported;
  hipDeviceAttribute_t DevAttrMemoryPoolsSupported = hipDeviceAttributeMemoryPoolsSupported;

  // CHECK: hipGraphExecUpdateResult GraphExecUpdateErrorUnsupportedFunctionChange = hipGraphExecUpdateErrorUnsupportedFunctionChange;
  hipGraphExecUpdateResult GraphExecUpdateErrorUnsupportedFunctionChange = hipGraphExecUpdateErrorUnsupportedFunctionChange;

  // CHECK: hipMemPoolAttr MemPoolAttr;
  // CHECK-NEXT: hipMemPoolAttr MemPoolReuseFollowEventDependencies = hipMemPoolReuseFollowEventDependencies;
  // CHECK-NEXT: hipMemPoolAttr MemPoolReuseAllowOpportunistic = hipMemPoolReuseAllowOpportunistic;
  // CHECK-NEXT: hipMemPoolAttr MemPoolReuseAllowInternalDependencies = hipMemPoolReuseAllowInternalDependencies;
  // CHECK-NEXT: hipMemPoolAttr MemPoolAttrReleaseThreshold = hipMemPoolAttrReleaseThreshold;
  hipMemPoolAttr MemPoolAttr;
  hipMemPoolAttr MemPoolReuseFollowEventDependencies = hipMemPoolReuseFollowEventDependencies;
  hipMemPoolAttr MemPoolReuseAllowOpportunistic = hipMemPoolReuseAllowOpportunistic;
  hipMemPoolAttr MemPoolReuseAllowInternalDependencies = hipMemPoolReuseAllowInternalDependencies;
  hipMemPoolAttr MemPoolAttrReleaseThreshold = hipMemPoolAttrReleaseThreshold;

  // CHECK: hipMemLocationType memLocationType;
  // CHECK-NEXT: hipMemLocationType MemLocationTypeInvalid = hipMemLocationTypeInvalid;
  // CHECK-NEXT: hipMemLocationType MemLocationTypeDevice = hipMemLocationTypeDevice;
  hipMemLocationType memLocationType;
  hipMemLocationType MemLocationTypeInvalid = hipMemLocationTypeInvalid;
  hipMemLocationType MemLocationTypeDevice = hipMemLocationTypeDevice;

  // CHECK: hipMemAccessFlags MemAccessFlags;
  // CHECK-NEXT: hipMemAccessFlags MemAccessFlagsProtNone = hipMemAccessFlagsProtNone;
  // CHECK-NEXT: hipMemAccessFlags MemAccessFlagsProtRead = hipMemAccessFlagsProtRead;
  // CHECK-NEXT: hipMemAccessFlags MemAccessFlagsProtReadWrite = hipMemAccessFlagsProtReadWrite;
  hipMemAccessFlags MemAccessFlags;
  hipMemAccessFlags MemAccessFlagsProtNone = hipMemAccessFlagsProtNone;
  hipMemAccessFlags MemAccessFlagsProtRead = hipMemAccessFlagsProtRead;
  hipMemAccessFlags MemAccessFlagsProtReadWrite = hipMemAccessFlagsProtReadWrite;

  // CHECK: hipMemAllocationType memAllocationType;
  // CHECK-NEXT: hipMemAllocationType MemAllocationTypeInvalid = hipMemAllocationTypeInvalid;
  // CHECK-NEXT: hipMemAllocationType MemAllocationTypePinned = hipMemAllocationTypePinned;
  // CHECK-NEXT: hipMemAllocationType MemAllocationTypeMax = hipMemAllocationTypeMax;
  hipMemAllocationType memAllocationType;
  hipMemAllocationType MemAllocationTypeInvalid = hipMemAllocationTypeInvalid;
  hipMemAllocationType MemAllocationTypePinned = hipMemAllocationTypePinned;
  hipMemAllocationType MemAllocationTypeMax = hipMemAllocationTypeMax;

  // CHECK: hipMemAllocationHandleType memAllocationHandleType;
  // CHECK-NEXT: hipMemAllocationHandleType MEM_HANDLE_TYPE_NONE = hipMemHandleTypeNone;
  // CHECK-NEXT: hipMemAllocationHandleType MEM_HANDLE_TYPE_POSIX_FILE_DESCRIPTOR = hipMemHandleTypePosixFileDescriptor;
  // CHECK-NEXT: hipMemAllocationHandleType MEM_HANDLE_TYPE_WIN32 = hipMemHandleTypeWin32;
  // CHECK-NEXT: hipMemAllocationHandleType MEM_HANDLE_TYPE_WIN32_KMT = hipMemHandleTypeWin32Kmt;
  hipMemAllocationHandleType memAllocationHandleType;
  hipMemAllocationHandleType MEM_HANDLE_TYPE_NONE = hipMemHandleTypeNone;
  hipMemAllocationHandleType MEM_HANDLE_TYPE_POSIX_FILE_DESCRIPTOR = hipMemHandleTypePosixFileDescriptor;
  hipMemAllocationHandleType MEM_HANDLE_TYPE_WIN32 = hipMemHandleTypeWin32;
  hipMemAllocationHandleType MEM_HANDLE_TYPE_WIN32_KMT = hipMemHandleTypeWin32Kmt;
#endif

#if CUDA_VERSION >= 11030
  // CHECK: hipStreamUpdateCaptureDependenciesFlags StreamUpdateCaptureDependenciesFlags;
  // CHECK-NEXT: hipStreamUpdateCaptureDependenciesFlags StreamAddCaptureDependencies = hipStreamAddCaptureDependencies;
  // CHECK-NEXT: hipStreamUpdateCaptureDependenciesFlags StreamSetCaptureDependencies = hipStreamSetCaptureDependencies;
  hipStreamUpdateCaptureDependenciesFlags StreamUpdateCaptureDependenciesFlags;
  hipStreamUpdateCaptureDependenciesFlags StreamAddCaptureDependencies = hipStreamAddCaptureDependencies;
  hipStreamUpdateCaptureDependenciesFlags StreamSetCaptureDependencies = hipStreamSetCaptureDependencies;

  // CHECK: hipMemPoolAttr MemPoolAttrReservedMemCurrent = hipMemPoolAttrReservedMemCurrent;
  // CHECK-NEXT: hipMemPoolAttr MemPoolAttrReservedMemHigh = hipMemPoolAttrReservedMemHigh;
  // CHECK-NEXT: hipMemPoolAttr MemPoolAttrUsedMemCurrent = hipMemPoolAttrUsedMemCurrent;
  // CHECK-NEXT: hipMemPoolAttr MemPoolAttrUsedMemHigh = hipMemPoolAttrUsedMemHigh;
  hipMemPoolAttr MemPoolAttrReservedMemCurrent = hipMemPoolAttrReservedMemCurrent;
  hipMemPoolAttr MemPoolAttrReservedMemHigh = hipMemPoolAttrReservedMemHigh;
  hipMemPoolAttr MemPoolAttrUsedMemCurrent = hipMemPoolAttrUsedMemCurrent;
  hipMemPoolAttr MemPoolAttrUsedMemHigh = hipMemPoolAttrUsedMemHigh;

  // CHECK: hipUserObjectFlags UserObjectFlags;
  // CHECK-NEXT: hipUserObjectFlags UserObjectNoDestructorSync = hipUserObjectNoDestructorSync;
  hipUserObjectFlags UserObjectFlags;
  hipUserObjectFlags UserObjectNoDestructorSync = hipUserObjectNoDestructorSync;

  // CHECK: hipUserObjectRetainFlags UserObjectRetainFlags;
  // CHECK-NEXT: hipUserObjectRetainFlags GraphUserObjectMove = hipGraphUserObjectMove;
  hipUserObjectRetainFlags UserObjectRetainFlags;
  hipUserObjectRetainFlags GraphUserObjectMove = hipGraphUserObjectMove;

  // CHECK: hipGraphDebugDotFlags graphDebugDot_flags;
  // CHECK-NEXT: hipGraphDebugDotFlags graphDebugDot_flags_enum;
  // CHECK-NEXT: hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_VERBOSE = hipGraphDebugDotFlagsVerbose;
  // CHECK-NEXT: hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_RUNTIME_TYPES = hipGraphDebugDotFlagsRuntimeTypes;
  // CHECK-NEXT: hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_KERNEL_NODE_PARAMS = hipGraphDebugDotFlagsKernelNodeParams;
  // CHECK-NEXT: hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_MEMCPY_NODE_PARAMS = hipGraphDebugDotFlagsMemcpyNodeParams;
  // CHECK-NEXT: hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_MEMSET_NODE_PARAMS = hipGraphDebugDotFlagsMemsetNodeParams;
  // CHECK-NEXT: hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_HOST_NODE_PARAMS = hipGraphDebugDotFlagsHostNodeParams;
  // CHECK-NEXT: hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_EVENT_NODE_PARAMS = hipGraphDebugDotFlagsEventNodeParams;
  // CHECK-NEXT: hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_EXT_SEMAS_SIGNAL_NODE_PARAMS = hipGraphDebugDotFlagsExtSemasSignalNodeParams;
  // CHECK-NEXT: hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_EXT_SEMAS_WAIT_NODE_PARAMS = hipGraphDebugDotFlagsExtSemasWaitNodeParams;
  // CHECK-NEXT: hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_KERNEL_NODE_ATTRIBUTES = hipGraphDebugDotFlagsKernelNodeAttributes;
  // CHECK-NEXT: hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_HANDLES = hipGraphDebugDotFlagsHandles;
  hipGraphDebugDotFlags graphDebugDot_flags;
  hipGraphDebugDotFlags graphDebugDot_flags_enum;
  hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_VERBOSE = hipGraphDebugDotFlagsVerbose;
  hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_RUNTIME_TYPES = hipGraphDebugDotFlagsRuntimeTypes;
  hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_KERNEL_NODE_PARAMS = hipGraphDebugDotFlagsKernelNodeParams;
  hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_MEMCPY_NODE_PARAMS = hipGraphDebugDotFlagsMemcpyNodeParams;
  hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_MEMSET_NODE_PARAMS = hipGraphDebugDotFlagsMemsetNodeParams;
  hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_HOST_NODE_PARAMS = hipGraphDebugDotFlagsHostNodeParams;
  hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_EVENT_NODE_PARAMS = hipGraphDebugDotFlagsEventNodeParams;
  hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_EXT_SEMAS_SIGNAL_NODE_PARAMS = hipGraphDebugDotFlagsExtSemasSignalNodeParams;
  hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_EXT_SEMAS_WAIT_NODE_PARAMS = hipGraphDebugDotFlagsExtSemasWaitNodeParams;
  hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_KERNEL_NODE_ATTRIBUTES = hipGraphDebugDotFlagsKernelNodeAttributes;
  hipGraphDebugDotFlags GRAPH_DEBUG_DOT_FLAGS_HANDLES = hipGraphDebugDotFlagsHandles;

  // CHECK: hipFlushGPUDirectRDMAWritesOptions flushGPUDirectRDMAWritesOptions;
  // CHECK-NEXT: hipFlushGPUDirectRDMAWritesOptions FLUSH_GPU_DIRECT_RDMA_WRITES_OPTION_HOST = hipFlushGPUDirectRDMAWritesOptionHost;
  // CHECK-NEXT: hipFlushGPUDirectRDMAWritesOptions FLUSH_GPU_DIRECT_RDMA_WRITES_OPTION_MEMOPS = hipFlushGPUDirectRDMAWritesOptionMemOps;
  hipFlushGPUDirectRDMAWritesOptions flushGPUDirectRDMAWritesOptions;
  hipFlushGPUDirectRDMAWritesOptions FLUSH_GPU_DIRECT_RDMA_WRITES_OPTION_HOST = hipFlushGPUDirectRDMAWritesOptionHost;
  hipFlushGPUDirectRDMAWritesOptions FLUSH_GPU_DIRECT_RDMA_WRITES_OPTION_MEMOPS = hipFlushGPUDirectRDMAWritesOptionMemOps;

  // CHECK: hipGPUDirectRDMAWritesOrdering GPUDirectRDMAWritesOrdering;
  // CHECK-NEXT: hipGPUDirectRDMAWritesOrdering GPU_DIRECT_RDMA_WRITES_ORDERING_NONE = hipGPUDirectRDMAWritesOrderingNone;
  // CHECK-NEXT: hipGPUDirectRDMAWritesOrdering GPU_DIRECT_RDMA_WRITES_ORDERING_OWNER = hipGPUDirectRDMAWritesOrderingOwner;
  // CHECK-NEXT: hipGPUDirectRDMAWritesOrdering GPU_DIRECT_RDMA_WRITES_ORDERING_ALL_DEVICES = hipGPUDirectRDMAWritesOrderingAllDevices;
  hipGPUDirectRDMAWritesOrdering GPUDirectRDMAWritesOrdering;
  hipGPUDirectRDMAWritesOrdering GPU_DIRECT_RDMA_WRITES_ORDERING_NONE = hipGPUDirectRDMAWritesOrderingNone;
  hipGPUDirectRDMAWritesOrdering GPU_DIRECT_RDMA_WRITES_ORDERING_OWNER = hipGPUDirectRDMAWritesOrderingOwner;
  hipGPUDirectRDMAWritesOrdering GPU_DIRECT_RDMA_WRITES_ORDERING_ALL_DEVICES = hipGPUDirectRDMAWritesOrderingAllDevices;
#endif

#if CUDA_VERSION >= 11040
  // CHECK: hipGraphInstantiateFlags GraphInstantiateFlags;
  // CHECK-NEXT: hipGraphInstantiateFlags GraphInstantiateFlagAutoFreeOnLaunch = hipGraphInstantiateFlagAutoFreeOnLaunch;
  hipGraphInstantiateFlags GraphInstantiateFlags;
  hipGraphInstantiateFlags GraphInstantiateFlagAutoFreeOnLaunch = hipGraphInstantiateFlagAutoFreeOnLaunch;

  // CHECK: hipGraphMemAttributeType GraphMemAttributeType;
  // CHECK-NEXT: hipGraphMemAttributeType GraphMemAttrUsedMemCurrent = hipGraphMemAttrUsedMemCurrent;
  // CHECK-NEXT: hipGraphMemAttributeType GraphMemAttrUsedMemHigh = hipGraphMemAttrUsedMemHigh;
  // CHECK-NEXT: hipGraphMemAttributeType GraphMemAttrReservedMemCurrent = hipGraphMemAttrReservedMemCurrent;
  // CHECK-NEXT: hipGraphMemAttributeType GraphMemAttrReservedMemHigh = hipGraphMemAttrReservedMemHigh;
  hipGraphMemAttributeType GraphMemAttributeType;
  hipGraphMemAttributeType GraphMemAttrUsedMemCurrent = hipGraphMemAttrUsedMemCurrent;
  hipGraphMemAttributeType GraphMemAttrUsedMemHigh = hipGraphMemAttrUsedMemHigh;
  hipGraphMemAttributeType GraphMemAttrReservedMemCurrent = hipGraphMemAttrReservedMemCurrent;
  hipGraphMemAttributeType GraphMemAttrReservedMemHigh = hipGraphMemAttrReservedMemHigh;

  // CHECK: hipGraphNodeType GraphNodeTypeExtSemaphoreSignal = hipGraphNodeTypeExtSemaphoreSignal;
  // CHECK-NEXT: hipGraphNodeType GraphNodeTypeExtSemaphoreWait = hipGraphNodeTypeExtSemaphoreWait;
  // CHECK-NEXT: hipGraphNodeType GraphNodeTypeMemAlloc = hipGraphNodeTypeMemAlloc;
  // CHECK-NEXT: hipGraphNodeType GraphNodeTypeMemFree = hipGraphNodeTypeMemFree;
  hipGraphNodeType GraphNodeTypeExtSemaphoreSignal = hipGraphNodeTypeExtSemaphoreSignal;
  hipGraphNodeType GraphNodeTypeExtSemaphoreWait = hipGraphNodeTypeExtSemaphoreWait;
  hipGraphNodeType GraphNodeTypeMemAlloc = hipGraphNodeTypeMemAlloc;
  hipGraphNodeType GraphNodeTypeMemFree = hipGraphNodeTypeMemFree;
#endif

#if CUDA_VERSION >= 11070
  // CHECK: hipGraphInstantiateFlags GraphInstantiateFlagUseNodePriority = hipGraphInstantiateFlagUseNodePriority;
  hipGraphInstantiateFlags GraphInstantiateFlagUseNodePriority = hipGraphInstantiateFlagUseNodePriority;
#endif

#if CUDA_VERSION >= 12000
  // CHECK: hipGraphInstantiateFlags GraphInstantiateFlagUpload = hipGraphInstantiateFlagUpload;
  // CHECK-NEXT: hipGraphInstantiateFlags GraphInstantiateFlagDeviceLaunch = hipGraphInstantiateFlagDeviceLaunch;
  hipGraphInstantiateFlags GraphInstantiateFlagUpload = hipGraphInstantiateFlagUpload;
  hipGraphInstantiateFlags GraphInstantiateFlagDeviceLaunch = hipGraphInstantiateFlagDeviceLaunch;

  // CHECK: hipGraphInstantiateResult graphInstantiateResult;
  // CHECK-NEXT: hipGraphInstantiateResult GRAPH_INSTANTIATE_SUCCESS = hipGraphInstantiateSuccess;
  // CHECK-NEXT: hipGraphInstantiateResult GRAPH_INSTANTIATE_ERROR = hipGraphInstantiateError;
  // CHECK-NEXT: hipGraphInstantiateResult GRAPH_INSTANTIATE_INVALID_STRUCTURE = hipGraphInstantiateInvalidStructure;
  // CHECK-NEXT: hipGraphInstantiateResult GRAPH_INSTANTIATE_NODE_OPERATION_NOT_SUPPORTED = hipGraphInstantiateNodeOperationNotSupported;
  // CHECK-NEXT: hipGraphInstantiateResult GRAPH_INSTANTIATE_MULTIPLE_CTXS_NOT_SUPPORTED = hipGraphInstantiateMultipleDevicesNotSupported;
  hipGraphInstantiateResult graphInstantiateResult;
  hipGraphInstantiateResult GRAPH_INSTANTIATE_SUCCESS = hipGraphInstantiateSuccess;
  hipGraphInstantiateResult GRAPH_INSTANTIATE_ERROR = hipGraphInstantiateError;
  hipGraphInstantiateResult GRAPH_INSTANTIATE_INVALID_STRUCTURE = hipGraphInstantiateInvalidStructure;
  hipGraphInstantiateResult GRAPH_INSTANTIATE_NODE_OPERATION_NOT_SUPPORTED = hipGraphInstantiateNodeOperationNotSupported;
  hipGraphInstantiateResult GRAPH_INSTANTIATE_MULTIPLE_CTXS_NOT_SUPPORTED = hipGraphInstantiateMultipleDevicesNotSupported;
#endif

  return 0;
}
