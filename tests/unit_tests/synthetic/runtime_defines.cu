// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

int main() {
  printf("08. CUDA Runtime API Defines synthetic test\n");

  // CHECK: int IPC_HANDLE_SIZE = HIP_IPC_HANDLE_SIZE;
  // CHECK-NEXT: int ArrayDefault = hipArrayDefault;
  // CHECK-NEXT: int ArrayLayered = hipArrayLayered;
  // CHECK-NEXT: int ArraySurfaceLoadStore = hipArraySurfaceLoadStore;
  // CHECK-NEXT: int ArrayCubemap = hipArrayCubemap;
  // CHECK-NEXT: int ArrayTextureGather = hipArrayTextureGather;
  // CHECK-NEXT: int DeviceBlockingSync = hipDeviceScheduleBlockingSync;
  // CHECK-NEXT: int DeviceLmemResizeToMax = hipDeviceLmemResizeToMax;
  // CHECK-NEXT: int DeviceMapHost = hipDeviceMapHost;
  // CHECK-NEXT: int DeviceScheduleAuto = hipDeviceScheduleAuto;
  // CHECK-NEXT: int DeviceScheduleSpin = hipDeviceScheduleSpin;
  // CHECK-NEXT: int DeviceScheduleYield = hipDeviceScheduleYield;
  // CHECK-NEXT: int DeviceScheduleBlockingSync = hipDeviceScheduleBlockingSync;
  // CHECK-NEXT: int DeviceScheduleMask = hipDeviceScheduleMask;
  // CHECK-NEXT: int EventDefault = hipEventDefault;
  // CHECK-NEXT: int EventBlockingSync = hipEventBlockingSync;
  // CHECK-NEXT: int EventDisableTiming = hipEventDisableTiming;
  // CHECK-NEXT: int EventInterprocess = hipEventInterprocess;
  // CHECK-NEXT: int HostAllocDefault = hipHostMallocDefault;
  // CHECK-NEXT: int HostAllocPortable = hipHostMallocPortable;
  // CHECK-NEXT: int HostAllocMapped = hipHostMallocMapped;
  // CHECK-NEXT: int HostAllocWriteCombined = hipHostMallocWriteCombined;
  // CHECK-NEXT: int HostRegisterDefault = hipHostRegisterDefault;
  // CHECK-NEXT: int HostRegisterPortable = hipHostRegisterPortable;
  // CHECK-NEXT: int HostRegisterMapped = hipHostRegisterMapped;
  // CHECK-NEXT: int IpcMemLazyEnablePeerAccess = hipIpcMemLazyEnablePeerAccess;
  // CHECK-NEXT: int MemAttachGlobal = hipMemAttachGlobal;
  // CHECK-NEXT: int MemAttachHost = hipMemAttachHost;
  // CHECK-NEXT: int MemAttachSingle = hipMemAttachSingle;
  // CHECK-NEXT: int TextureType1D = hipTextureType1D;
  // CHECK-NEXT: int TextureType2D = hipTextureType2D;
  // CHECK-NEXT: int TextureType3D = hipTextureType3D;
  // CHECK-NEXT: int TextureTypeCubemap = hipTextureTypeCubemap;
  // CHECK-NEXT: int TextureType1DLayered = hipTextureType1DLayered;
  // CHECK-NEXT: int TextureType2DLayered = hipTextureType2DLayered;
  // CHECK-NEXT: int TextureTypeCubemapLayered = hipTextureTypeCubemapLayered;
  // CHECK-NEXT: int OccupancyDefault = hipOccupancyDefault;
  // CHECK-NEXT: int OccupancyDisableCachingOverride = hipOccupancyDisableCachingOverride;
  // CHECK-NEXT: int StreamDefault = hipStreamDefault;
  // CHECK-NEXT: int StreamNonBlocking = hipStreamNonBlocking;
  // CHECK-NEXT: hipStream_t StreamPerThread = hipStreamPerThread;
  int IPC_HANDLE_SIZE = HIP_IPC_HANDLE_SIZE;
  int ArrayDefault = hipArrayDefault;
  int ArrayLayered = hipArrayLayered;
  int ArraySurfaceLoadStore = hipArraySurfaceLoadStore;
  int ArrayCubemap = hipArrayCubemap;
  int ArrayTextureGather = hipArrayTextureGather;
  int DeviceBlockingSync = hipDeviceScheduleBlockingSync;
  int DeviceLmemResizeToMax = hipDeviceLmemResizeToMax;
  int DeviceMapHost = hipDeviceMapHost;
  int DeviceScheduleAuto = hipDeviceScheduleAuto;
  int DeviceScheduleSpin = hipDeviceScheduleSpin;
  int DeviceScheduleYield = hipDeviceScheduleYield;
  int DeviceScheduleBlockingSync = hipDeviceScheduleBlockingSync;
  int DeviceScheduleMask = hipDeviceScheduleMask;
  int EventDefault = hipEventDefault;
  int EventBlockingSync = hipEventBlockingSync;
  int EventDisableTiming = hipEventDisableTiming;
  int EventInterprocess = hipEventInterprocess;
  int HostAllocDefault = hipHostMallocDefault;
  int HostAllocPortable = hipHostMallocPortable;
  int HostAllocMapped = hipHostMallocMapped;
  int HostAllocWriteCombined = hipHostMallocWriteCombined;
  int HostRegisterDefault = hipHostRegisterDefault;
  int HostRegisterPortable = hipHostRegisterPortable;
  int HostRegisterMapped = hipHostRegisterMapped;
  int IpcMemLazyEnablePeerAccess = hipIpcMemLazyEnablePeerAccess;
  int MemAttachGlobal = hipMemAttachGlobal;
  int MemAttachHost = hipMemAttachHost;
  int MemAttachSingle = hipMemAttachSingle;
  int TextureType1D = hipTextureType1D;
  int TextureType2D = hipTextureType2D;
  int TextureType3D = hipTextureType3D;
  int TextureTypeCubemap = hipTextureTypeCubemap;
  int TextureType1DLayered = hipTextureType1DLayered;
  int TextureType2DLayered = hipTextureType2DLayered;
  int TextureTypeCubemapLayered = hipTextureTypeCubemapLayered;
  int OccupancyDefault = hipOccupancyDefault;
  int OccupancyDisableCachingOverride = hipOccupancyDisableCachingOverride;
  int StreamDefault = hipStreamDefault;
  int StreamNonBlocking = hipStreamNonBlocking;
  hipStream_t StreamPerThread = hipStreamPerThread;

#if CUDA_VERSION >= 7050
  // CHECK: int HostRegisterIoMemory = hipHostRegisterIoMemory;
  int HostRegisterIoMemory = hipHostRegisterIoMemory;
#endif

#if CUDA_VERSION >= 8000
  // CHECK: int CpuDeviceId = hipCpuDeviceId;
  // CHECK-NEXT: int InvalidDeviceId = hipInvalidDeviceId;
  int CpuDeviceId = hipCpuDeviceId;
  int InvalidDeviceId = hipInvalidDeviceId;
#endif

#if CUDA_VERSION >= 9000
  // CHECK: int CooperativeLaunchMultiDeviceNoPreSync = hipCooperativeLaunchMultiDeviceNoPreSync;
  // CHECK-NEXT: int CooperativeLaunchMultiDeviceNoPostSync = hipCooperativeLaunchMultiDeviceNoPostSync;
  int CooperativeLaunchMultiDeviceNoPreSync = hipCooperativeLaunchMultiDeviceNoPreSync;
  int CooperativeLaunchMultiDeviceNoPostSync = hipCooperativeLaunchMultiDeviceNoPostSync;
#endif

#if CUDA_VERSION >= 10000
  // CHECK: int EXTERNAL_MEMORY_DEDICATED = hipExternalMemoryDedicated;
  int EXTERNAL_MEMORY_DEDICATED = hipExternalMemoryDedicated;
#endif

#if CUDA_VERSION >= 11010
  // CHECK: int HostRegisterReadOnly = hipHostRegisterReadOnly;
  int HostRegisterReadOnly = hipHostRegisterReadOnly;
#endif

  return 0;
}
