#include "hip/hip_runtime.h"
// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime_api.h>
// CHECK-NEXT: #include <hip/hip/device_functions.h>
// CHECK-NEXT: #include <hip/hip_math_constants.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_math_constants.h>
#include <stdio.h>

  __global__ __constant__ int INF_F;
  __global__ __constant__ int NAN_F;
  __global__ __constant__ int MIN_DENORM_F;
  __global__ __constant__ int MAX_NORMAL_F;
  __global__ __constant__ int NEG_ZERO_F;
  __global__ __constant__ int ZERO_F;
  __global__ __constant__ int ONE_F;
  __global__ __constant__ int SQRT_HALF_F;
  __global__ __constant__ int SQRT_HALF_HI_F;
  __global__ __constant__ int SQRT_HALF_LO_F;
  __global__ __constant__ int SQRT_TWO_F;
  __global__ __constant__ int THIRD_F;
  __global__ __constant__ int PIO4_F;
  __global__ __constant__ int PIO2_F;
  __global__ __constant__ int _3PIO4_F;
  __global__ __constant__ int _2_OVER_PI_F;
  __global__ __constant__ int SQRT_2_OVER_PI_F;
  __global__ __constant__ int PI_F;
  __global__ __constant__ int L2E_F;
  __global__ __constant__ int L2T_F;
  __global__ __constant__ int LG2_F;
  __global__ __constant__ int LGE_F;
  __global__ __constant__ int LN2_F;
  __global__ __constant__ int LNT_F;
  __global__ __constant__ int LNPI_F;
  __global__ __constant__ int TWO_TO_M126_F;
  __global__ __constant__ int TWO_TO_126_F;
  __global__ __constant__ int NORM_HUGE_F;
  __global__ __constant__ int TWO_TO_23_F;
  __global__ __constant__ int TWO_TO_24_F;
  __global__ __constant__ int TWO_TO_31_F;
  __global__ __constant__ int TWO_TO_32_F;
  __global__ __constant__ int REMQUO_BITS_F;
  __global__ __constant__ int REMQUO_MASK_F;
  __global__ __constant__ int TRIG_PLOSS_F;

__global__ void init() {
  // CHECK: INF_F = HIP_INF_F;
  // CHECK-NEXT: NAN_F = HIP_NAN_F;
  // CHECK-NEXT: MIN_DENORM_F = HIP_MIN_DENORM_F;
  // CHECK-NEXT: MAX_NORMAL_F = HIP_MAX_NORMAL_F;
  // CHECK-NEXT: NEG_ZERO_F = HIP_NEG_ZERO_F;
  // CHECK-NEXT: ZERO_F = HIP_ZERO_F;
  // CHECK-NEXT: ONE_F = HIP_ONE_F;
  // CHECK-NEXT: SQRT_HALF_F = HIP_SQRT_HALF_F;
  // CHECK-NEXT: SQRT_HALF_HI_F = HIP_SQRT_HALF_HI_F;
  // CHECK-NEXT: SQRT_HALF_LO_F = HIP_SQRT_HALF_LO_F;
  // CHECK-NEXT: SQRT_TWO_F = HIP_SQRT_TWO_F;
  // CHECK-NEXT: THIRD_F = HIP_THIRD_F;
  // CHECK-NEXT: PIO4_F = HIP_PIO4_F;
  // CHECK-NEXT: PIO2_F = HIP_PIO2_F;
  // CHECK-NEXT: _3PIO4_F = HIP_3PIO4_F;
  // CHECK-NEXT: _2_OVER_PI_F = HIP_2_OVER_PI_F;
  // CHECK-NEXT: SQRT_2_OVER_PI_F = HIP_SQRT_2_OVER_PI_F;
  // CHECK-NEXT: PI_F = HIP_PI_F;
  // CHECK-NEXT: L2E_F = HIP_L2E_F;
  // CHECK-NEXT: L2T_F = HIP_L2T_F;
  // CHECK-NEXT: LG2_F = HIP_LG2_F;
  // CHECK-NEXT: LGE_F = HIP_LGE_F;
  // CHECK-NEXT: LN2_F = HIP_LN2_F;
  // CHECK-NEXT: LNT_F = HIP_LNT_F;
  // CHECK-NEXT: LNPI_F = HIP_LNPI_F;
  // CHECK-NEXT: TWO_TO_M126_F = HIP_TWO_TO_M126_F;
  // CHECK-NEXT: TWO_TO_126_F = HIP_TWO_TO_126_F;
  // CHECK-NEXT: NORM_HUGE_F = HIP_NORM_HUGE_F;
  // CHECK-NEXT: TWO_TO_23_F = HIP_TWO_TO_23_F;
  // CHECK-NEXT: TWO_TO_24_F = HIP_TWO_TO_24_F;
  // CHECK-NEXT: TWO_TO_31_F = HIP_TWO_TO_31_F;
  // CHECK-NEXT: TWO_TO_32_F = HIP_TWO_TO_32_F;
  // CHECK-NEXT: REMQUO_BITS_F = HIP_REMQUO_BITS_F;
  // CHECK-NEXT: REMQUO_MASK_F = HIP_REMQUO_MASK_F;
  // CHECK-NEXT: TRIG_PLOSS_F = HIP_TRIG_PLOSS_F;
  INF_F = HIP_INF_F;
  NAN_F = HIP_NAN_F;
  MIN_DENORM_F = HIP_MIN_DENORM_F;
  MAX_NORMAL_F = HIP_MAX_NORMAL_F;
  NEG_ZERO_F = HIP_NEG_ZERO_F;
  ZERO_F = HIP_ZERO_F;
  ONE_F = HIP_ONE_F;
  SQRT_HALF_F = HIP_SQRT_HALF_F;
  SQRT_HALF_HI_F = HIP_SQRT_HALF_HI_F;
  SQRT_HALF_LO_F = HIP_SQRT_HALF_LO_F;
  SQRT_TWO_F = HIP_SQRT_TWO_F;
  THIRD_F = HIP_THIRD_F;
  PIO4_F = HIP_PIO4_F;
  PIO2_F = HIP_PIO2_F;
  _3PIO4_F = HIP_3PIO4_F;
  _2_OVER_PI_F = HIP_2_OVER_PI_F;
  SQRT_2_OVER_PI_F = HIP_SQRT_2_OVER_PI_F;
  PI_F = HIP_PI_F;
  L2E_F = HIP_L2E_F;
  L2T_F = HIP_L2T_F;
  LG2_F = HIP_LG2_F;
  LGE_F = HIP_LGE_F;
  LN2_F = HIP_LN2_F;
  LNT_F = HIP_LNT_F;
  LNPI_F = HIP_LNPI_F;
  TWO_TO_M126_F = HIP_TWO_TO_M126_F;
  TWO_TO_126_F = HIP_TWO_TO_126_F;
  NORM_HUGE_F = HIP_NORM_HUGE_F;
  TWO_TO_23_F = HIP_TWO_TO_23_F;
  TWO_TO_24_F = HIP_TWO_TO_24_F;
  TWO_TO_31_F = HIP_TWO_TO_31_F;
  TWO_TO_32_F = HIP_TWO_TO_32_F;
  REMQUO_BITS_F = HIP_REMQUO_BITS_F;
  REMQUO_MASK_F = HIP_REMQUO_MASK_F;
  TRIG_PLOSS_F = HIP_TRIG_PLOSS_F;
}

int main() {
  printf("08. CUDA Runtime API Defines synthetic test\n");

  // CHECK: int IPC_HANDLE_SIZE = HIP_IPC_HANDLE_SIZE;
  // CHECK-NEXT: int ArrayDefault = hipArrayDefault;
  // CHECK-NEXT: int ArrayLayered = hipArrayLayered;
  // CHECK-NEXT: int ArraySurfaceLoadStore = hipArraySurfaceLoadStore;
  // CHECK-NEXT: int ArrayCubemap = hipArrayCubemap;
  // CHECK-NEXT: int ArrayTextureGather = hipArrayTextureGather;
  // CHECK-NEXT: int DeviceBlockingSync = hipDeviceScheduleBlockingSync;
  // CHECK-NEXT: int DeviceLmemResizeToMax = hipDeviceLmemResizeToMax;
  // CHECK-NEXT: int DeviceMapHost = hipDeviceMapHost;
  // CHECK-NEXT: int DeviceScheduleAuto = hipDeviceScheduleAuto;
  // CHECK-NEXT: int DeviceScheduleSpin = hipDeviceScheduleSpin;
  // CHECK-NEXT: int DeviceScheduleYield = hipDeviceScheduleYield;
  // CHECK-NEXT: int DeviceScheduleBlockingSync = hipDeviceScheduleBlockingSync;
  // CHECK-NEXT: int DeviceScheduleMask = hipDeviceScheduleMask;
  // CHECK-NEXT: int EventDefault = hipEventDefault;
  // CHECK-NEXT: int EventBlockingSync = hipEventBlockingSync;
  // CHECK-NEXT: int EventDisableTiming = hipEventDisableTiming;
  // CHECK-NEXT: int EventInterprocess = hipEventInterprocess;
  // CHECK-NEXT: int HostAllocDefault = hipHostMallocDefault;
  // CHECK-NEXT: int HostAllocPortable = hipHostMallocPortable;
  // CHECK-NEXT: int HostAllocMapped = hipHostMallocMapped;
  // CHECK-NEXT: int HostAllocWriteCombined = hipHostMallocWriteCombined;
  // CHECK-NEXT: int HostRegisterDefault = hipHostRegisterDefault;
  // CHECK-NEXT: int HostRegisterPortable = hipHostRegisterPortable;
  // CHECK-NEXT: int HostRegisterMapped = hipHostRegisterMapped;
  // CHECK-NEXT: int IpcMemLazyEnablePeerAccess = hipIpcMemLazyEnablePeerAccess;
  // CHECK-NEXT: int MemAttachGlobal = hipMemAttachGlobal;
  // CHECK-NEXT: int MemAttachHost = hipMemAttachHost;
  // CHECK-NEXT: int MemAttachSingle = hipMemAttachSingle;
  // CHECK-NEXT: int TextureType1D = hipTextureType1D;
  // CHECK-NEXT: int TextureType2D = hipTextureType2D;
  // CHECK-NEXT: int TextureType3D = hipTextureType3D;
  // CHECK-NEXT: int TextureTypeCubemap = hipTextureTypeCubemap;
  // CHECK-NEXT: int TextureType1DLayered = hipTextureType1DLayered;
  // CHECK-NEXT: int TextureType2DLayered = hipTextureType2DLayered;
  // CHECK-NEXT: int TextureTypeCubemapLayered = hipTextureTypeCubemapLayered;
  // CHECK-NEXT: int OccupancyDefault = hipOccupancyDefault;
  // CHECK-NEXT: int OccupancyDisableCachingOverride = hipOccupancyDisableCachingOverride;
  // CHECK-NEXT: int StreamDefault = hipStreamDefault;
  // CHECK-NEXT: int StreamNonBlocking = hipStreamNonBlocking;
  // CHECK-NEXT: hipStream_t StreamPerThread = hipStreamPerThread;
  int IPC_HANDLE_SIZE = HIP_IPC_HANDLE_SIZE;
  int ArrayDefault = hipArrayDefault;
  int ArrayLayered = hipArrayLayered;
  int ArraySurfaceLoadStore = hipArraySurfaceLoadStore;
  int ArrayCubemap = hipArrayCubemap;
  int ArrayTextureGather = hipArrayTextureGather;
  int DeviceBlockingSync = hipDeviceScheduleBlockingSync;
  int DeviceLmemResizeToMax = hipDeviceLmemResizeToMax;
  int DeviceMapHost = hipDeviceMapHost;
  int DeviceScheduleAuto = hipDeviceScheduleAuto;
  int DeviceScheduleSpin = hipDeviceScheduleSpin;
  int DeviceScheduleYield = hipDeviceScheduleYield;
  int DeviceScheduleBlockingSync = hipDeviceScheduleBlockingSync;
  int DeviceScheduleMask = hipDeviceScheduleMask;
  int EventDefault = hipEventDefault;
  int EventBlockingSync = hipEventBlockingSync;
  int EventDisableTiming = hipEventDisableTiming;
  int EventInterprocess = hipEventInterprocess;
  int HostAllocDefault = hipHostMallocDefault;
  int HostAllocPortable = hipHostMallocPortable;
  int HostAllocMapped = hipHostMallocMapped;
  int HostAllocWriteCombined = hipHostMallocWriteCombined;
  int HostRegisterDefault = hipHostRegisterDefault;
  int HostRegisterPortable = hipHostRegisterPortable;
  int HostRegisterMapped = hipHostRegisterMapped;
  int IpcMemLazyEnablePeerAccess = hipIpcMemLazyEnablePeerAccess;
  int MemAttachGlobal = hipMemAttachGlobal;
  int MemAttachHost = hipMemAttachHost;
  int MemAttachSingle = hipMemAttachSingle;
  int TextureType1D = hipTextureType1D;
  int TextureType2D = hipTextureType2D;
  int TextureType3D = hipTextureType3D;
  int TextureTypeCubemap = hipTextureTypeCubemap;
  int TextureType1DLayered = hipTextureType1DLayered;
  int TextureType2DLayered = hipTextureType2DLayered;
  int TextureTypeCubemapLayered = hipTextureTypeCubemapLayered;
  int OccupancyDefault = hipOccupancyDefault;
  int OccupancyDisableCachingOverride = hipOccupancyDisableCachingOverride;
  int StreamDefault = hipStreamDefault;
  int StreamNonBlocking = hipStreamNonBlocking;
  hipStream_t StreamPerThread = hipStreamPerThread;

#if CUDA_VERSION >= 7050
  // CHECK: int HostRegisterIoMemory = hipHostRegisterIoMemory;
  int HostRegisterIoMemory = hipHostRegisterIoMemory;
#endif

#if CUDA_VERSION >= 8000
  // CHECK: int CpuDeviceId = hipCpuDeviceId;
  // CHECK-NEXT: int InvalidDeviceId = hipInvalidDeviceId;
  int CpuDeviceId = hipCpuDeviceId;
  int InvalidDeviceId = hipInvalidDeviceId;
#endif

#if CUDA_VERSION >= 9000
  // CHECK: int CooperativeLaunchMultiDeviceNoPreSync = hipCooperativeLaunchMultiDeviceNoPreSync;
  // CHECK-NEXT: int CooperativeLaunchMultiDeviceNoPostSync = hipCooperativeLaunchMultiDeviceNoPostSync;
  int CooperativeLaunchMultiDeviceNoPreSync = hipCooperativeLaunchMultiDeviceNoPreSync;
  int CooperativeLaunchMultiDeviceNoPostSync = hipCooperativeLaunchMultiDeviceNoPostSync;

  // CHECK: hipStream_t StreamLegacy = hipStreamLegacy;
  hipStream_t StreamLegacy = hipStreamLegacy;
#endif

#if CUDA_VERSION >= 10000
  // CHECK: int EXTERNAL_MEMORY_DEDICATED = hipExternalMemoryDedicated;
  int EXTERNAL_MEMORY_DEDICATED = hipExternalMemoryDedicated;
#endif

#if CUDA_VERSION >= 11010
  // CHECK: int HostRegisterReadOnly = hipHostRegisterReadOnly;
  int HostRegisterReadOnly = hipHostRegisterReadOnly;
#endif

#if CUDA_VERSION >= 12030
  // CHECK: int GRAPH_KERNEL_NODE_PORT_DEFAULT = hipGraphKernelNodePortDefault;
  int GRAPH_KERNEL_NODE_PORT_DEFAULT = hipGraphKernelNodePortDefault;

  // CHECK: int GRAPH_KERNEL_NODE_PORT_LAUNCH_ORDER = hipGraphKernelNodePortLaunchCompletion;
  int GRAPH_KERNEL_NODE_PORT_LAUNCH_ORDER = hipGraphKernelNodePortLaunchCompletion;

  // CHECK: int GRAPH_KERNEL_NODE_PORT_PROGRAMMATIC = hipGraphKernelNodePortProgrammatic;
  int GRAPH_KERNEL_NODE_PORT_PROGRAMMATIC = hipGraphKernelNodePortProgrammatic;
#endif

  return 0;
}
